#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <host_utils.h>
#include <command_line_params.h>

// include because of nasty globals used in the tests
#include <dslash_reference.h>

// google test
#include <gtest/gtest.h>

constexpr int Nkernels = 43;

using namespace quda;

ColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *mH, *lH;
ColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *mD, *lD, *xmD, *ymD, *zmD;
std::vector<cpuColorSpinorField*> xmH;
std::vector<cpuColorSpinorField*> ymH;
std::vector<cpuColorSpinorField*> zmH;
int Nspin;
int Ncolor;

void setPrec(ColorSpinorParam &param, QudaPrecision precision, int order = 0)
{
  param.setPrecision(precision);
  if (order == 2) {
    param.fieldOrder = QUDA_FLOAT8_FIELD_ORDER;
  } else if (order == 1) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else if (Nspin == 1 || Nspin == 2 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void display_test_info()
{
  printfQuda("running the following test:\n");
  printfQuda("S_dimension T_dimension Nspin Ncolor\n");
  printfQuda("%3d /%3d / %3d   %3d      %d     %d\n", xdim, ydim, zdim, tdim, Nspin, Ncolor);
  printfQuda("Grid partition info:     X  Y  Z  T\n");
  printfQuda("                         %d  %d  %d  %d\n",
	     dimPartitioned(0),
	     dimPartitioned(1),
	     dimPartitioned(2),
	     dimPartitioned(3));
  return;
}

int Nprec = 4;

const char *prec_str[] = {"quarter", "half", "single", "double"};
const char *order_str[] = {"default", "float2", "float8"};

// For googletest names must be non-empty, unique, and may only contain ASCII
// alphanumeric characters or underscore
const char *names[] = {"copyHS",
                       "copyMS",
                       "copyLS",
                       "axpby",
                       "xpy",
                       "axpy",
                       "xpay",
                       "mxpy",
                       "ax",
                       "caxpy",
                       "caxpby",
                       "cxpaypbz",
                       "axpyBzpcx",
                       "axpyZpbx",
                       "caxpbypzYmbw",
                       "cabxpyAx",
                       "caxpyXmaz",
                       "norm",
                       "reDotProduct",
                       "axpyNorm",
                       "xmyNorm",
                       "caxpyNorm",
                       "caxpyXmazNormX",
                       "cabxpyzAxNorm",
                       "cDotProduct",
                       "caxpyDotzy",
                       "cDotProductNormA",
                       "cDotProductNormB",
                       "caxpbypzYmbwcDotProductUYNormY",
                       "HeavyQuarkResidualNorm",
                       "xpyHeavyQuarkResidualNorm",
                       "tripleCGReduction",
                       "tripleCGUpdate",
                       "axpyReDot",
                       "caxpy_block",
                       "axpyBzpcx_block",
                       "caxpyBxpz",
                       "caxpyBzpx",
                       "cDotProductNorm_block",
                       "cDotProduct_block",
                       "reDotProductNorm_block",
                       "reDotProduct_block",
                       "axpy_block"};

// kernels that utilize multi-blas
bool is_multi(int kernel) { return std::string(names[kernel]).find("_block") != std::string::npos ? true : false; }

// kernels that require site unrolling
bool is_site_unroll(int kernel) { return std::string(names[kernel]).find("HeavyQuark") != std::string::npos ? true : false; }

bool skip_kernel(int precision, int kernel, int order)
{
  if ((QUDA_PRECISION & getPrecision(precision)) == 0) return true;

  // if we've selected a given kernel then make sure we only run that
  if (test_type != -1 && kernel != test_type) return true;

  // if we've selected a given precision then make sure we only run that
  auto this_prec = getPrecision(precision);
  if (prec != QUDA_INVALID_PRECISION && this_prec != prec) return true;

  if ( Nspin == 2 && ( precision == 0 || precision ==1 ) ) {
    // avoid quarter, half precision tests if doing coarse fields
    return true;
  } else if (Nspin == 2 && (kernel == 1 || kernel == 2)) {
    // avoid low-precision copy if doing coarse fields
    return true;
  } else if (Ncolor != 3 && is_site_unroll(kernel)) {
    // only benchmark heavy-quark norm if doing 3 colors
    return true;
  } else if ((Nprec < 4) && (kernel == 0)) {
    // only benchmark high-precision copy() if double is supported
    return true;
  }

  if (order == 1) {
#ifdef GPU_MULTIGRID
    // order == 1 represents the case of multigrid testing for float-2
    // ordered nspin-4 fields in single precision and less, skip all other cases
    if (Nspin == 1 || Nspin == 2 || this_prec == QUDA_DOUBLE_PRECISION) {
      return true;
    } else if (Nspin == 4 && (this_prec != QUDA_DOUBLE_PRECISION && is_multi(kernel) ||
                              this_prec == QUDA_SINGLE_PRECISION && is_site_unroll(kernel))) {
      // we don't instantiate multi-blas kernels for float-2 nspin-4
      // fields, so skip these
      return true;
    }
#else
    return true;
#endif
  }

  // this is for float-8 testing
  if (order == 2) {
#ifdef FLOAT8
    // order == 2 represents the case of float-8 nspin-4 fields
    // only run fixed-precision fields, skip all other cases
    if (Nspin == 1 || Nspin == 2 || this_prec >= QUDA_HALF_PRECISION) {
      return true;
    } else if (Nspin == 4 && is_multi(kernel)) {
      // we currently don't instantiate multi-blas kernels for float-8
      // fields, so skip these
      return true;
    }
#else
    return true;
#endif
  }

  return false;
}

void initFields(int prec, int order)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec = QUDA_INVALID_PRECISION;
  QudaPrecision mid_aux_prec = QUDA_INVALID_PRECISION;
  QudaPrecision low_aux_prec = QUDA_INVALID_PRECISION;

  ColorSpinorParam param;
  param.nColor = Ncolor;
  param.nSpin = Nspin;
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields

  switch (solve_type) {
  case QUDA_DIRECT_PC_SOLVE:
  case QUDA_NORMOP_PC_SOLVE: param.siteSubset = QUDA_PARITY_SITE_SUBSET; break;
  case QUDA_DIRECT_SOLVE:
  case QUDA_NORMOP_SOLVE: param.siteSubset = QUDA_FULL_SITE_SUBSET; break;
  default: errorQuda("Unexpected solve_type=%d\n", solve_type);
  }

  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = xdim/2;
  else param.x[0] = xdim;
  param.x[1] = ydim;
  param.x[2] = zdim;
  param.x[3] = tdim;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.setPrecision(QUDA_DOUBLE_PRECISION);
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  mH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

  // create composite fields

  // xmH = new cpuColorSpinorField(param);
  // ymH = new cpuColorSpinorField(param);

  xmH.reserve(Nsrc);
  for (int cid = 0; cid < Nsrc; cid++) xmH.push_back(new cpuColorSpinorField(param));
  ymH.reserve(Msrc);
  for (int cid = 0; cid < Msrc; cid++) ymH.push_back(new cpuColorSpinorField(param));
  zmH.reserve(Nsrc);
  for (int cid = 0; cid < Nsrc; cid++) zmH.push_back(new cpuColorSpinorField(param));

  static_cast<cpuColorSpinorField*>(vH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(wH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(xH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(yH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(zH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(hH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(mH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(lH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  for(int i=0; i<Nsrc; i++){
    static_cast<cpuColorSpinorField*>(xmH[i])->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  }
  for(int i=0; i<Msrc; i++){
    static_cast<cpuColorSpinorField*>(ymH[i])->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  }
  // Now set the parameters for the cuda fields
  //param.pad = xdim*ydim*zdim/2;

  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_QUARTER_PRECISION, order);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    mid_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_HALF_PRECISION, order);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    mid_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_QUARTER_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_SINGLE_PRECISION, order);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    mid_aux_prec = QUDA_HALF_PRECISION;
    low_aux_prec = QUDA_QUARTER_PRECISION;
    break;
  case 3:
    setPrec(param, QUDA_DOUBLE_PRECISION, order);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    mid_aux_prec = QUDA_HALF_PRECISION;
    low_aux_prec = QUDA_QUARTER_PRECISION;
    break;
  default:
    errorQuda("Precision option not defined");
  }

  // ensure we don't enable copying between precisions that are not compiled
  if ( (high_aux_prec != QUDA_DOUBLE_PRECISION) && !(high_aux_prec & QUDA_PRECISION) ) high_aux_prec = getPrecision(prec);
  if ( (mid_aux_prec != QUDA_DOUBLE_PRECISION) && !(mid_aux_prec & QUDA_PRECISION) ) mid_aux_prec = getPrecision(prec);
  if ( (low_aux_prec != QUDA_DOUBLE_PRECISION) && !(low_aux_prec & QUDA_PRECISION) ) low_aux_prec = getPrecision(prec);

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  param.is_composite = true;
  param.is_component = false;

  // create composite fields
  param.composite_dim = Nsrc;
  xmD = new cudaColorSpinorField(param);

  param.composite_dim = Msrc;
  ymD = new cudaColorSpinorField(param);

  param.composite_dim = Nsrc;
  zmD = new cudaColorSpinorField(param);

  param.is_composite = false;
  param.is_component = false;
  param.composite_dim = 1;

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, mid_aux_prec);
  mD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  // only do copy if not doing half precision with mg
  bool flag = !(param.nSpin == 2 &&
		(prec == 0 || prec == 1 || low_aux_prec == QUDA_HALF_PRECISION || mid_aux_prec == QUDA_HALF_PRECISION ||
                                low_aux_prec == QUDA_QUARTER_PRECISION || mid_aux_prec == QUDA_QUARTER_PRECISION) );

  if ( flag ) {

    *vD = *vH;
    *wD = *wH;
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *hD = *hH;
    *mD = *mH;
    *lD = *lH;

    // for (int i=0; i < Nsrc; i++){
    //   xmD->Component(i) = *(xmH[i]);
    //   ymD->Component(i) = *(ymH[i]);
    // }
    // *ymD = *ymH;
  }
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete mD;
  delete lD;
  delete xmD;
  delete ymD;
  delete zmD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete mH;
  delete lH;
  for (int i=0; i < Nsrc; i++) delete xmH[i];
  for (int i=0; i < Msrc; i++) delete ymH[i];
  for (int i=0; i < Nsrc; i++) delete zmH[i];
  xmH.clear();
  ymH.clear();
  zmH.clear();
}


double benchmark(int kernel, const int niter) {

  double a = 1.0, b = 2.0, c = 3.0;
  quda::Complex a2, b2;
  quda::Complex * A = new quda::Complex[Nsrc*Msrc];
  quda::Complex * B = new quda::Complex[Nsrc*Msrc];
  quda::Complex * C = new quda::Complex[Nsrc*Msrc];
  quda::Complex * A2 = new quda::Complex[Nsrc*Nsrc]; // for the block cDotProductNorm test
  double *Ar = new double[Nsrc * Msrc];

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  {
    switch (kernel) {

    case 0:
      for (int i=0; i < niter; ++i) blas::copy(*yD, *hD);
      break;

    case 1:
      for (int i=0; i < niter; ++i) blas::copy(*yD, *mD);
      break;

    case 2:
      for (int i=0; i < niter; ++i) blas::copy(*yD, *lD);
      break;

    case 3:
      for (int i=0; i < niter; ++i) blas::axpby(a, *xD, b, *yD);
      break;

    case 4:
      for (int i=0; i < niter; ++i) blas::xpy(*xD, *yD);
      break;

    case 5:
      for (int i=0; i < niter; ++i) blas::axpy(a, *xD, *yD);
      break;

    case 6:
      for (int i=0; i < niter; ++i) blas::xpay(*xD, a, *yD);
      break;

    case 7:
      for (int i=0; i < niter; ++i) blas::mxpy(*xD, *yD);
      break;

    case 8:
      for (int i=0; i < niter; ++i) blas::ax(a, *xD);
      break;

    case 9:
      for (int i=0; i < niter; ++i) blas::caxpy(a2, *xD, *yD);
      break;

    case 10:
      for (int i=0; i < niter; ++i) blas::caxpby(a2, *xD, b2, *yD);
      break;

    case 11:
      for (int i=0; i < niter; ++i) blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
      break;

    case 12:
      for (int i=0; i < niter; ++i) blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
      break;

    case 13:
      for (int i=0; i < niter; ++i) blas::axpyZpbx(a, *xD, *yD, *zD, b);
      break;

    case 14:
      for (int i=0; i < niter; ++i) blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
      break;

    case 15:
      for (int i=0; i < niter; ++i) blas::cabxpyAx(a, b2, *xD, *yD);
      break;

    case 16:
      for (int i=0; i < niter; ++i) blas::caxpyXmaz(a2, *xD, *yD, *zD);
      break;

      // double
    case 17:
      for (int i=0; i < niter; ++i) blas::norm2(*xD);
      break;

    case 18:
      for (int i=0; i < niter; ++i) blas::reDotProduct(*xD, *yD);
      break;

    case 19:
      for (int i=0; i < niter; ++i) blas::axpyNorm(a, *xD, *yD);
      break;

    case 20:
      for (int i=0; i < niter; ++i) blas::xmyNorm(*xD, *yD);
      break;

    case 21:
      for (int i=0; i < niter; ++i) blas::caxpyNorm(a2, *xD, *yD);
      break;

    case 22:
      for (int i=0; i < niter; ++i) blas::caxpyXmazNormX(a2, *xD, *yD, *zD);
      break;

    case 23:
      for (int i=0; i < niter; ++i) blas::cabxpyzAxNorm(a, b2, *xD, *yD, *yD);
      break;

    // double2
    case 24:
      for (int i=0; i < niter; ++i) blas::cDotProduct(*xD, *yD);
      break;

    case 25:
      for (int i=0; i < niter; ++i) blas::caxpyDotzy(a2, *xD, *yD, *zD);
      break;

    // double3
    case 26:
      for (int i=0; i < niter; ++i) blas::cDotProductNormA(*xD, *yD);
      break;

    case 27:
      for (int i=0; i < niter; ++i) blas::cDotProductNormB(*xD, *yD);
      break;

    case 28:
      for (int i=0; i < niter; ++i) blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    case 29:
      for (int i=0; i < niter; ++i) blas::HeavyQuarkResidualNorm(*xD, *yD);
      break;

    case 30:
      for (int i=0; i < niter; ++i) blas::xpyHeavyQuarkResidualNorm(*xD, *yD, *zD);
      break;

    case 31:
      for (int i=0; i < niter; ++i) blas::tripleCGReduction(*xD, *yD, *zD);
      break;

    case 32:
      for (int i=0; i < niter; ++i) blas::tripleCGUpdate(a, b, *xD, *yD, *zD, *wD);
      break;

    case 33:
      for (int i=0; i < niter; ++i) blas::axpyReDot(a, *xD, *yD);
      break;

    case 34:
      for (int i=0; i < niter; ++i) blas::caxpy(A, *xmD,* ymD);
      break;

    case 35:
      for (int i=0; i < niter; ++i) blas::axpyBzpcx((double*)A, xmD->Components(), zmD->Components(), (double*)B, *yD, (double*)C);
      break;

    case 36:
      for (int i=0; i < niter; ++i) blas::caxpyBxpz(a2, *xD, *yD, b2, *zD);
      break;

    case 37:
      for (int i=0; i < niter; ++i) blas::caxpyBzpx(a2, *xD, *yD, b2, *zD);
      break;

    case 38:
      for (int i=0; i < niter; ++i) blas::cDotProduct(A2, xmD->Components(), xmD->Components());
      break;

    case 39:
      for (int i=0; i < niter; ++i) blas::cDotProduct(A, xmD->Components(), ymD->Components());
      break;

    case 40:
      for (int i=0; i < niter; ++i) blas::reDotProduct((double*)A2, xmD->Components(), xmD->Components());
      break;

    case 41:
      for (int i=0; i < niter; ++i) blas::reDotProduct((double*)A, xmD->Components(), ymD->Components());
      break;

    case 42:
      for (int i = 0; i < niter; ++i) blas::axpy(Ar, xmD->Components(), ymD->Components());
      break;

    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }

  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  delete[] A;
  delete[] B;
  delete[] C;
  delete[] A2;
  delete[] Ar;
  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(blas::norm2(*a##D) - blas::norm2(*a##H)) / blas::norm2(*a##H)

double test(int kernel) {

  double a = M_PI, b = M_PI*exp(1.0), c = sqrt(M_PI);
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;
  quda::Complex * A = new quda::Complex[Nsrc*Msrc];
  quda::Complex * B = new quda::Complex[Nsrc*Msrc];
  quda::Complex * C = new quda::Complex[Nsrc*Msrc];
  quda::Complex * A2 = new quda::Complex[Nsrc*Nsrc]; // for the block cDotProductNorm test
  quda::Complex * B2 = new quda::Complex[Nsrc*Nsrc]; // for the block cDotProductNorm test
  double *Ar = new double[Nsrc * Msrc];

  for (int i = 0; i < Nsrc * Msrc; i++) {
    A[i] = a2 * (1.0 * ((i / (double)Nsrc) + i)) + b2 * (1.0 * i) + c2 * (1.0 * (0.5 * Nsrc * Msrc - i));
    B[i] = a2 * (1.0 * ((i / (double)Nsrc) + i)) - b2 * (M_PI * i) + c2 * (1.0 * (0.5 * Nsrc * Msrc - i));
    C[i] = a2 * (1.0 * ((M_PI / (double)Nsrc) + i)) + b2 * (1.0 * i) + c2 * (1.0 * (0.5 * Nsrc * Msrc - i));
    Ar[i] = A[i].real();
  }
  for (int i = 0; i < Nsrc * Nsrc; i++) {
    A2[i] = a2 * (1.0 * ((i / (double)Nsrc) + i)) + b2 * (1.0 * i) + c2 * (1.0 * (0.5 * Nsrc * Nsrc - i));
    B2[i] = a2 * (1.0 * ((i / (double)Nsrc) + i)) - b2 * (M_PI * i) + c2 * (1.0 * (0.5 * Nsrc * Nsrc - i));
  }
  // A[0] = a2;
  // A[1] = 0.;
  // A[2] = 0.;
  // A[3] = 0.;

  switch (kernel) {

  case 0:
    *hD = *hH;
    blas::copy(*yD, *hD);
    blas::copy(*yH, *hH);
    error = ERROR(y);
    break;

  case 1:
    *mD = *mH;
    blas::copy(*yD, *mD);
    blas::copy(*yH, *mH);
    error = ERROR(y);
    break;

  case 2:
    *lD = *lH;
    blas::copy(*yD, *lD);
    blas::copy(*yH, *lH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    blas::axpby(a, *xD, b, *yD);
    blas::axpby(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    blas::xpy(*xD, *yD);
    blas::xpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    blas::axpy(a, *xD, *yD);
    blas::axpy(a, *xH, *yH);
    *zH = *yD;
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    blas::xpay(*xD, a, *yD);
    blas::xpay(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    *yD = *yH;
    blas::mxpy(*xD, *yD);
    blas::mxpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 8:
    *xD = *xH;
    blas::ax(a, *xD);
    blas::ax(a, *xH);
    error = ERROR(x);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    blas::caxpy(a2, *xD, *yD);
    blas::caxpy(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    blas::caxpby(a2, *xD, b2, *yD);
    blas::caxpby(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
    blas::cxpaypbz(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
    blas::axpyBzpcx(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyZpbx(a, *xD, *yD, *zD, b);
    blas::axpyZpbx(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 14:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
    blas::caxpbypzYmbw(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    blas::cabxpyAx(a, b2, *xD, *yD);
    blas::cabxpyAx(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyXmaz(a, *xD, *yD, *zD);
     blas::caxpyXmaz(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

  case 17:
    *xD = *xH;
    *yH = *xD;
    error = fabs(blas::norm2(*xD) - blas::norm2(*xH)) / blas::norm2(*xH);
    break;

  case 18:
    *xD = *xH;
    *yD = *yH;
    error = fabs(blas::reDotProduct(*xD, *yD) - blas::reDotProduct(*xH, *yH)) / fabs(blas::reDotProduct(*xH, *yH));
    break;

  case 19:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::axpyNorm(a, *xD, *yD);
    double h = blas::axpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::xmyNorm(*xD, *yD);
    double h = blas::xmyNorm(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::caxpyNorm(a, *xD, *yD);
    double h = blas::caxpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 22:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = blas::caxpyXmazNormX(a, *xD, *yD, *zD);
      double h = blas::caxpyXmazNormX(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::cabxpyzAxNorm(a, b2, *xD, *yD, *yD);
      double h = blas::cabxpyzAxNorm(a, b2, *xH, *yH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    error = abs(blas::cDotProduct(*xD, *yD) - blas::cDotProduct(*xH, *yH)) / abs(blas::cDotProduct(*xH, *yH));
    break;

  case 25:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = blas::caxpyDotzy(a, *xD, *yD, *zD);
      quda::Complex h = blas::caxpyDotzy(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

  case 26:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormA(*xD, *yD);
      double3 h = blas::cDotProductNormA(*xH, *yH);
      error = abs(Complex(d.x - h.x, d.y - h.y)) / abs(Complex(h.x, h.y)) + fabs(d.z - h.z) / fabs(h.z);
    }
    break;

  case 27:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormB(*xD, *yD);
      double3 h = blas::cDotProductNormB(*xH, *yH);
      error = abs(Complex(d.x - h.x, d.y - h.y)) / abs(Complex(h.x, h.y)) + fabs(d.z - h.z) / fabs(h.z);
    }
    break;

  case 28:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + abs(Complex(d.x - h.x, d.y - h.y)) / abs(Complex(h.x, h.y))
          + fabs(d.z - h.z) / fabs(h.z);
    }
    break;

  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::HeavyQuarkResidualNorm(*xD, *yD);
      double3 h = blas::HeavyQuarkResidualNorm(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { double3 d = blas::xpyHeavyQuarkResidualNorm(*xD, *yD, *zD);
      double3 h = blas::xpyHeavyQuarkResidualNorm(*xH, *yH, *zH);
      error = ERROR(y) + fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 31:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { double3 d = blas::tripleCGReduction(*xD, *yD, *zD);
      double3 h = make_double3(blas::norm2(*xH), blas::norm2(*yH), blas::reDotProduct(*yH, *zH));
      error = fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 32:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    { blas::tripleCGUpdate(a, b, *xD, *yD, *zD, *wD);
      blas::tripleCGUpdate(a, b, *xH, *yH, *zH, *wH);
      error = ERROR(y) + ERROR(z) + ERROR(w); }
    break;

  case 33:
    *xD = *xH;
    *yD = *yH;
    { double d = blas::axpyReDot(a, *xD, *yD);
      double h = blas::axpyReDot(a, *xH, *yH);
      error = ERROR(y) + fabs(d-h)/fabs(h); }
    break;

  case 34:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    for (int i=0; i < Msrc; i++) ymD->Component(i) = *(ymH[i]);

    blas::caxpy(A, *xmD, *ymD);
    for (int i=0; i < Nsrc; i++){
      for(int j=0; j < Msrc; j++){
	blas::caxpy(A[Msrc*i+j], *(xmH[i]), *(ymH[j]));
      }
    }
    error = 0;
    for (int i=0; i < Msrc; i++){
      error+= fabs(blas::norm2((ymD->Component(i))) - blas::norm2(*(ymH[i]))) / blas::norm2(*(ymH[i]));
    }
    error/= Msrc;
    break;

  case 35:
    for (int i=0; i < Nsrc; i++) {
      xmD->Component(i) = *(xmH[i]);
      zmD->Component(i) = *(zmH[i]);
    }
    *yD = *yH;

    blas::axpyBzpcx((double*)A, xmD->Components(), zmD->Components(), (double*)B, *yD, (const double*)C);

    for (int i=0; i<Nsrc; i++) {
      blas::axpyBzpcx(((double*)A)[i], *xmH[i], *zmH[i], ((double*)B)[i], *yH, ((double*)C)[i]);
    }

    error = 0;
    for (int i=0; i < Nsrc; i++){
      error+= fabs(blas::norm2((xmD->Component(i))) - blas::norm2(*(xmH[i]))) / blas::norm2(*(xmH[i]));
      //error+= fabs(blas::norm2((zmD->Component(i))) - blas::norm2(*(zmH[i]))) / blas::norm2(*(zmH[i]));
    }
    error/= Nsrc;
    break;

  case 36:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyBxpz(a, *xD, *yD, b2, *zD);
     blas::caxpyBxpz(a, *xH, *yH, b2, *zH);
     error = ERROR(x) + ERROR(z);}
    break;

  case 37:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyBzpx(a, *xD, *yD, b2, *zD);
     blas::caxpyBzpx(a, *xH, *yH, b2, *zH);
     error = ERROR(x) + ERROR(z);}
    break;

  case 38:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    blas::cDotProduct(A2, xmD->Components(), xmD->Components());
    error = 0.0;
    for (int i = 0; i < Nsrc; i++) {
      for (int j = 0; j < Nsrc; j++) {
	B2[i*Nsrc+j] = blas::cDotProduct(xmD->Component(i), xmD->Component(j));
	error += std::abs(A2[i*Nsrc+j] - B2[i*Nsrc+j])/std::abs(B2[i*Nsrc+j]);
      }
    }
    error /= Nsrc*Nsrc;
    break;

  case 39:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    for (int i=0; i < Msrc; i++) ymD->Component(i) = *(ymH[i]);
    blas::cDotProduct(A, xmD->Components(), ymD->Components());
    error = 0.0;
    for (int i = 0; i < Nsrc; i++) {
      for (int j = 0; j < Msrc; j++) {
	B[i*Msrc+j] = blas::cDotProduct(xmD->Component(i), ymD->Component(j));
	error += std::abs(A[i*Msrc+j] - B[i*Msrc+j])/std::abs(B[i*Msrc+j]);
      }
    }
    error /= Nsrc*Msrc;
    break;

  case 40:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    blas::reDotProduct((double*)A2, xmD->Components(), xmD->Components());
    error = 0.0;
    for (int i = 0; i < Nsrc; i++) {
      for (int j = 0; j < Nsrc; j++) {
        ((double*)B2)[i*Nsrc+j] = blas::reDotProduct(xmD->Component(i), xmD->Component(j));
        error += std::abs(((double*)A2)[i*Nsrc+j] - ((double*)B2)[i*Nsrc+j])/std::abs(((double*)B2)[i*Nsrc+j]);
      }
    }
    error /= Nsrc*Nsrc;
    break;

  case 41:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    for (int i=0; i < Msrc; i++) ymD->Component(i) = *(ymH[i]);
    blas::reDotProduct((double*)A, xmD->Components(), ymD->Components());
    error = 0.0;
    for (int i = 0; i < Nsrc; i++) {
      for (int j = 0; j < Msrc; j++) {
        ((double*)B)[i*Msrc+j] = blas::reDotProduct(xmD->Component(i), ymD->Component(j));
        error += std::abs(((double*)A)[i*Msrc+j] - ((double*)B)[i*Msrc+j])/std::abs(((double*)B)[i*Msrc+j]);
      }
    }
    error /= Nsrc*Msrc;
    break;

  case 42:
    for (int i = 0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    for (int i = 0; i < Msrc; i++) ymD->Component(i) = *(ymH[i]);

    blas::axpy(Ar, *xmD, *ymD);
    for (int i = 0; i < Nsrc; i++) {
      for (int j = 0; j < Msrc; j++) { blas::axpy(Ar[Msrc * i + j], *(xmH[i]), *(ymH[j])); }
    }

    error = 0;
    for (int i = 0; i < Msrc; i++) {
      error += fabs(blas::norm2((ymD->Component(i))) - blas::norm2(*(ymH[i]))) / blas::norm2(*(ymH[i]));
    }
    error /= Msrc;
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }
  delete[] A;
  delete[] B;
  delete[] C;
  delete[] A2;
  delete[] B2;
  delete[] Ar;
  return error;
}

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  int result = 0;

  prec = QUDA_INVALID_PRECISION;
  test_type = -1;

  // command line options
  auto app = make_app();
  // add_eigen_option_group(app);
  // add_deflation_option_group(app);
  // add_multigrid_option_group(app);

  app->add_option("--test", test_type, "Kernel to test (-1: -> all kernels)")->check(CLI::Range(0, Nkernels - 1));
  try {
    app->parse(argc, argv);
  } catch (const CLI::ParseError &e) {
    return app->exit(e);
  }

  // override spin setting if mg solver is set to test coarse grids
  if (inv_type == QUDA_MG_INVERTER) {
    Nspin = 2;
    Ncolor = nvec[0];
    if (Ncolor == 0) Ncolor = 24;
  } else {
    // set spin according to the type of dslash
    Nspin = (dslash_type == QUDA_ASQTAD_DSLASH ||
	     dslash_type == QUDA_STAGGERED_DSLASH) ? 1 : 4;
    Ncolor = 3;
  }

  setSpinorSiteSize(24);
  initComms(argc, argv, gridsize_from_cmdline);
  display_test_info();
  initQuda(device);

  setVerbosity(verbosity);

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  ::testing::TestEventListeners &listeners = ::testing::UnitTest::GetInstance()->listeners();
  if (comm_rank() != 0) { delete listeners.Release(listeners.default_result_printer()); }
  result = RUN_ALL_TESTS();

  endQuda();

  finalizeComms();
  return result;
}

// The following tests each kernel at each precision using the google testing framework

using ::testing::TestWithParam;
using ::testing::Bool;
using ::testing::Values;
using ::testing::Range;
using ::testing::Combine;

class BlasTest : public ::testing::TestWithParam<::testing::tuple<int, int, int>>
{
protected:
  ::testing::tuple<int, int, int> param;
  const int &prec;
  const int &kernel;
  const int &order;

public:
  BlasTest() :
    param(GetParam()),
    prec(::testing::get<0>(param)),
    kernel(::testing::get<1>(param)),
    order(::testing::get<2>(param))
  {
  }
  virtual void SetUp() {
    if (!skip_kernel(prec, kernel, order)) initFields(prec, order);
  }
  virtual void TearDown()
  {
    if (!skip_kernel(prec, kernel, order)) {
      freeFields();
    }
  }
};

TEST_P(BlasTest, verify) {
  int prec = ::testing::get<0>(GetParam());
  int kernel = ::testing::get<1>(GetParam());
  int order = ::testing::get<2>(GetParam());
  if (skip_kernel(prec, kernel, order)) GTEST_SKIP();

  // certain tests will fail to run for coarse grids so mark these as
  // failed without running
  double deviation = test(kernel);
  // printfQuda("%-35s error = %e\n", names[kernel], deviation);
  double tol = (prec == 3 ? 1e-12 : (prec == 2 ? 1e-6 : (prec == 1 ? 1e-4 : 1e-2)));
  tol = (kernel < 4) ? 5e-2 : tol; // use different tolerance for copy
  EXPECT_LE(deviation, tol) << "CPU and CUDA implementations do not agree";
}

TEST_P(BlasTest, benchmark) {
  int prec = ::testing::get<0>(GetParam());
  int kernel = ::testing::get<1>(GetParam());
  int order = ::testing::get<2>(GetParam());
  if (skip_kernel(prec, kernel, order)) GTEST_SKIP();

  // do the initial tune
  benchmark(kernel, 1);

  // now rerun with more iterations to get accurate speed measurements
  quda::blas::flops = 0;
  quda::blas::bytes = 0;

  double secs = benchmark(kernel, niter);

  double gflops = (quda::blas::flops*1e-9)/(secs);
  double gbytes = quda::blas::bytes/(secs*1e9);
  RecordProperty("Gflops", std::to_string(gflops));
  RecordProperty("GBs", std::to_string(gbytes));
  printfQuda("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
}

std::string getblasname(testing::TestParamInfo<::testing::tuple<int, int, int>> param)
{
  int prec = ::testing::get<0>(param.param);
  int kernel = ::testing::get<1>(param.param);
  int order = ::testing::get<2>(param.param);
  std::string str(names[kernel]);
  str += std::string("_") + std::string(prec_str[prec]);
  str += std::string("_") + std::string(order_str[order]);
  return str;
}

// instantiate all test cases
INSTANTIATE_TEST_SUITE_P(QUDA, BlasTest, Combine(Range(0, Nprec), Range(0, Nkernels), Range(0, 3)), getblasname);
