#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <test_util.h>
#include <face_quda.h>

// include because of nasty globals used in the tests
#include <dslash_util.h>

// google test
#include <gtest.h>

extern QudaDslashType dslash_type;
extern QudaInverterType inv_type;
extern int nvec;
extern int device;
extern int xdim;
extern int ydim;
extern int zdim;
extern int tdim;
extern int gridsize_from_cmdline[];
extern int niter;

extern bool verify_results;
extern int Nsrc;
extern int Msrc;

extern void usage(char** );

const int Nkernels = 40;

using namespace quda;

ColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *lH;
ColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *lD, *xmD, *ymD, *zmD;
std::vector<cpuColorSpinorField*> xmH;
std::vector<cpuColorSpinorField*> ymH;
std::vector<cpuColorSpinorField*> zmH;
int Nspin;
int Ncolor;

void setPrec(ColorSpinorParam &param, const QudaPrecision precision)
{
  param.precision = precision;
  if (Nspin == 1 || Nspin == 2 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void
display_test_info()
{
  printfQuda("running the following test:\n");
  printfQuda("S_dimension T_dimension Nspin Ncolor\n");
  printfQuda("%3d /%3d / %3d   %3d      %d     %d\n", xdim, ydim, zdim, tdim, Nspin, Ncolor);
  printfQuda("Grid partition info:     X  Y  Z  T\n");
  printfQuda("                         %d  %d  %d  %d\n",
	     dimPartitioned(0),
	     dimPartitioned(1),
	     dimPartitioned(2),
	     dimPartitioned(3));
  return;
}

int Nprec = 3;

bool skip_kernel(int precision, int kernel) {
  if ( Nspin == 2 && precision == 0) {
    // avoid half precision tests if doing coarse fields
    return true;
  } else if (Nspin == 2 && kernel == 1) {
    // avoid low-precision copy if doing coarse fields
    return true;
  } else if (Ncolor != 3 && (kernel == 31 || kernel == 32)) {
    // only benchmark heavy-quark norm if doing 3 colors
    return true;
  } else if ((Nprec < 3) && (kernel == 0)) {
    // only benchmark high-precision copy() if double is supported
    return true;
  }

  return false;
}

void initFields(int prec)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec = QUDA_INVALID_PRECISION;
  QudaPrecision low_aux_prec = QUDA_INVALID_PRECISION;

  ColorSpinorParam param;
  param.nColor = Ncolor;
  param.nSpin = Nspin;
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields
  param.siteSubset = QUDA_PARITY_SITE_SUBSET;
  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = xdim/2;
  else param.x[0] = xdim;
  param.x[1] = ydim;
  param.x[2] = zdim;
  param.x[3] = tdim;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.precision = QUDA_DOUBLE_PRECISION;
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

// create composite fields

  // xmH = new cpuColorSpinorField(param);
  // ymH = new cpuColorSpinorField(param);



  xmH.reserve(Nsrc);
  for (int cid = 0; cid < Nsrc; cid++) xmH.push_back(new cpuColorSpinorField(param));
  ymH.reserve(Msrc);
  for (int cid = 0; cid < Msrc; cid++) ymH.push_back(new cpuColorSpinorField(param));
  zmH.reserve(Nsrc);
  for (int cid = 0; cid < Nsrc; cid++) zmH.push_back(new cpuColorSpinorField(param));


  static_cast<cpuColorSpinorField*>(vH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(wH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(xH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(yH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(zH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(hH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  static_cast<cpuColorSpinorField*>(lH)->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  for(int i=0; i<Nsrc; i++){
    static_cast<cpuColorSpinorField*>(xmH[i])->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  }
  for(int i=0; i<Msrc; i++){
    static_cast<cpuColorSpinorField*>(ymH[i])->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  }
  // Now set the parameters for the cuda fields
  //param.pad = xdim*ydim*zdim/2;

  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_HALF_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_SINGLE_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_SINGLE_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_DOUBLE_PRECISION);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  default:
    errorQuda("Precision option not defined");
  }

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  param.is_composite = true;
  param.is_component = false;

// create composite fields
  param.composite_dim = Nsrc;
  xmD = new cudaColorSpinorField(param);

  param.composite_dim = Msrc;
  ymD = new cudaColorSpinorField(param);

  param.composite_dim = Nsrc;
  zmD = new cudaColorSpinorField(param);

  param.is_composite = false;
  param.is_component = false;
  param.composite_dim = 1;

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  // only do copy if not doing half precision with mg
  bool flag = !(param.nSpin == 2 &&
		(prec == 0 || low_aux_prec == QUDA_HALF_PRECISION) );

  if ( flag ) {
    *vD = *vH;
    *wD = *wH;
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *hD = *hH;
    *lD = *lH;
    // for (int i=0; i < Nsrc; i++){
    //   xmD->Component(i) = *(xmH[i]);
    //   ymD->Component(i) = *(ymH[i]);
    // }
    // *ymD = *ymH;
  }
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete lD;
  delete xmD;
  delete ymD;
  delete zmD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete lH;
  for (int i=0; i < Nsrc; i++) delete xmH[i];
  for (int i=0; i < Msrc; i++) delete ymH[i];
  for (int i=0; i < Nsrc; i++) delete zmH[i];
  xmH.clear();
  ymH.clear();
  zmH.clear();
}


double benchmark(int kernel, const int niter) {

  double a, b, c;
  quda::Complex a2, b2, c2;
  quda::Complex * A = new quda::Complex[Nsrc*Msrc];
  quda::Complex * B = new quda::Complex[Nsrc*Msrc];
  quda::Complex * C = new quda::Complex[Nsrc*Msrc];

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  {
    switch (kernel) {

    case 0:
      for (int i=0; i < niter; ++i) blas::copy(*yD, *hD);
      break;

    case 1:
      for (int i=0; i < niter; ++i) blas::copy(*yD, *lD);
      break;

    case 2:
      for (int i=0; i < niter; ++i) blas::axpby(a, *xD, b, *yD);
      break;

    case 3:
      for (int i=0; i < niter; ++i) blas::xpy(*xD, *yD);
      break;

    case 4:
      for (int i=0; i < niter; ++i) blas::axpy(a, *xD, *yD);
      break;

    case 5:
      for (int i=0; i < niter; ++i) blas::xpay(*xD, a, *yD);
      break;

    case 6:
      for (int i=0; i < niter; ++i) blas::mxpy(*xD, *yD);
      break;

    case 7:
      for (int i=0; i < niter; ++i) blas::ax(a, *xD);
      break;

    case 8:
      for (int i=0; i < niter; ++i) blas::caxpy(a2, *xD, *yD);
      break;

    case 9:
      for (int i=0; i < niter; ++i) blas::caxpby(a2, *xD, b2, *yD);
      break;

    case 10:
      for (int i=0; i < niter; ++i) blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
      break;

    case 11:
      for (int i=0; i < niter; ++i) blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
      break;

    case 12:
      for (int i=0; i < niter; ++i) blas::axpyZpbx(a, *xD, *yD, *zD, b);
      break;

    case 13:
      for (int i=0; i < niter; ++i) blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
      break;

    case 14:
      for (int i=0; i < niter; ++i) blas::cabxpyAx(a, b2, *xD, *yD);
      break;

    case 15:
      for (int i=0; i < niter; ++i) blas::caxpbypz(a2, *xD, b2, *yD, *zD);
      break;

    case 16:
      for (int i=0; i < niter; ++i) blas::caxpbypczpw(a2, *xD, b2, *yD, c2, *zD, *wD);
      break;

    case 17:
      for (int i=0; i < niter; ++i) blas::caxpyXmaz(a2, *xD, *yD, *zD);
      break;

      // double
    case 18:
      for (int i=0; i < niter; ++i) blas::norm2(*xD);
      break;

    case 19:
      for (int i=0; i < niter; ++i) blas::reDotProduct(*xD, *yD);
      break;

    case 20:
      for (int i=0; i < niter; ++i) blas::axpyNorm(a, *xD, *yD);
      break;

    case 21:
      for (int i=0; i < niter; ++i) blas::xmyNorm(*xD, *yD);
      break;

    case 22:
      for (int i=0; i < niter; ++i) blas::caxpyNorm(a2, *xD, *yD);
      break;

    case 23:
      for (int i=0; i < niter; ++i) blas::caxpyXmazNormX(a2, *xD, *yD, *zD);
      break;

    case 24:
      for (int i=0; i < niter; ++i) blas::cabxpyAxNorm(a, b2, *xD, *yD);
      break;

    // double2
    case 25:
      for (int i=0; i < niter; ++i) blas::cDotProduct(*xD, *yD);
      break;

    case 26:
      for (int i=0; i < niter; ++i) blas::xpaycDotzy(*xD, a, *yD, *zD);
      break;

    case 27:
      for (int i=0; i < niter; ++i) blas::caxpyDotzy(a2, *xD, *yD, *zD);
      break;

    // double3
    case 28:
      for (int i=0; i < niter; ++i) blas::cDotProductNormA(*xD, *yD);
      break;

    case 29:
      for (int i=0; i < niter; ++i) blas::cDotProductNormB(*xD, *yD);
      break;

    case 30:
      for (int i=0; i < niter; ++i) blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    case 31:
      for (int i=0; i < niter; ++i) blas::HeavyQuarkResidualNorm(*xD, *yD);
      break;

    case 32:
      for (int i=0; i < niter; ++i) blas::xpyHeavyQuarkResidualNorm(*xD, *yD, *zD);
      break;

    case 33:
      for (int i=0; i < niter; ++i) blas::tripleCGReduction(*xD, *yD, *zD);
      break;

    case 34:
      for (int i=0; i < niter; ++i) blas::tripleCGUpdate(a, b, *xD, *yD, *zD, *wD);
      break;

    case 35:
      for (int i=0; i < niter; ++i) blas::axpyReDot(a, *xD, *yD);
      break;

    case 36:
      for (int i=0; i < niter; ++i) blas::caxpy(A, *xmD,* ymD);
      break;

    case 37:
      for (int i=0; i < niter; ++i) blas::axpyBzpcx((double*)A, xmD->Components(), zmD->Components(), (double*)B, *yD, (double*)C);
      break;
        
    case 38:
      for (int i=0; i < niter; ++i) blas::caxpyBxpz(a2, *xD, *yD, b2, *zD); 
      break; 

    case 39:
      for (int i=0; i < niter; ++i) blas::caxpyBxpz(a2, *xD, *yD, b2, *zD); 
      break; 
        
    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }

  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  delete[] A;
  delete[] B;
  delete[] C;
  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(blas::norm2(*a##D) - blas::norm2(*a##H)) / blas::norm2(*a##H)

double test(int kernel) {

  double a = M_PI, b = M_PI*exp(1.0), c = sqrt(M_PI);
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;
  quda::Complex * A = new quda::Complex[Nsrc*Msrc];
  quda::Complex * B = new quda::Complex[Nsrc*Msrc];
  quda::Complex * C = new quda::Complex[Nsrc*Msrc];
  for(int i=0; i < Nsrc*Msrc; i++){
    A[i] = a2*  (1.0*((i/Nsrc) + i)) + b2 * (1.0*i) + c2 *(1.0*(Nsrc*Msrc/2-i));
    B[i] = a2*  (1.0*((i/Nsrc) + i)) - b2 * (M_PI*i) + c2 *(1.0*(Nsrc*Msrc/2-i));
    C[i] = a2*  (1.0*((M_PI/Nsrc) + i)) + b2 * (1.0*i) + c2 *(1.0*(Nsrc*Msrc/2-i));
  }
  // A[0] = a2;
  // A[1] = 0.;
  // A[2] = 0.;
  // A[3] = 0.;

  switch (kernel) {

  case 0:
    *hD = *hH;
    blas::copy(*yD, *hD);
    blas::copy(*yH, *hH);
    error = ERROR(y);
    break;

  case 1:
    *lD = *lH;
    blas::copy(*yD, *lD);
    blas::copy(*yH, *lH);
    error = ERROR(y);
    break;

  case 2:
    *xD = *xH;
    *yD = *yH;
    blas::axpby(a, *xD, b, *yD);
    blas::axpby(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    blas::xpy(*xD, *yD);
    blas::xpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    blas::axpy(a, *xD, *yD);
    blas::axpy(a, *xH, *yH);
    *zH = *yD;
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    blas::xpay(*xD, a, *yD);
    blas::xpay(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    blas::mxpy(*xD, *yD);
    blas::mxpy(*xH, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    blas::ax(a, *xD);
    blas::ax(a, *xH);
    error = ERROR(x);
    break;

  case 8:
    *xD = *xH;
    *yD = *yH;
    blas::caxpy(a2, *xD, *yD);
    blas::caxpy(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    blas::caxpby(a2, *xD, b2, *yD);
    blas::caxpby(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::cxpaypbz(*xD, a2, *yD, b2, *zD);
    blas::cxpaypbz(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyBzpcx(a, *xD, *yD, b, *zD, c);
    blas::axpyBzpcx(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    blas::axpyZpbx(a, *xD, *yD, *zD, b);
    blas::axpyZpbx(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    blas::caxpbypzYmbw(a2, *xD, b2, *yD, *zD, *wD);
    blas::caxpbypzYmbw(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;

  case 14:
    *xD = *xH;
    *yD = *yH;
    blas::cabxpyAx(a, b2, *xD, *yD);
    blas::cabxpyAx(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpbypz(a2, *xD, b2, *yD, *zD);
      blas::caxpbypz(a2, *xH, b2, *yH, *zH);
      error = ERROR(z); }
    break;

  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    {blas::caxpbypczpw(a2, *xD, b2, *yD, c2, *zD, *wD);
      blas::caxpbypczpw(a2, *xH, b2, *yH, c2, *zH, *wH);
      error = ERROR(w); }
    break;

  case 17:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyXmaz(a, *xD, *yD, *zD);
     blas::caxpyXmaz(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

    // double
  case 18:
    *xD = *xH;
    *yH = *xD;
    error = fabs(blas::norm2(*xD) - blas::norm2(*xH)) / blas::norm2(*xH);
    break;

  case 19:
    *xD = *xH;
    *yD = *yH;
    error = fabs(blas::reDotProduct(*xD, *yD) - blas::reDotProduct(*xH, *yH)) / fabs(blas::reDotProduct(*xH, *yH));
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::axpyNorm(a, *xD, *yD);
    double h = blas::axpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::xmyNorm(*xD, *yD);
    double h = blas::xmyNorm(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 22:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::caxpyNorm(a, *xD, *yD);
    double h = blas::caxpyNorm(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = blas::caxpyXmazNormX(a, *xD, *yD, *zD);
      double h = blas::caxpyXmazNormX(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    {double d = blas::cabxpyAxNorm(a, b2, *xD, *yD);
      double h = blas::cabxpyAxNorm(a, b2, *xH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

    // double2
  case 25:
    *xD = *xH;
    *yD = *yH;
    error = abs(blas::cDotProduct(*xD, *yD) - blas::cDotProduct(*xH, *yH)) / abs(blas::cDotProduct(*xH, *yH));
    break;

  case 26:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { quda::Complex d = blas::xpaycDotzy(*xD, a, *yD, *zD);
      quda::Complex h = blas::xpaycDotzy(*xH, a, *yH, *zH);
      error =  fabs(blas::norm2(*yD) - blas::norm2(*yH)) / blas::norm2(*yH) + abs(d-h)/abs(h);
    }
    break;

  case 27:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = blas::caxpyDotzy(a, *xD, *yD, *zD);
      quda::Complex h = blas::caxpyDotzy(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

    // double3
  case 28:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormA(*xD, *yD);
      double3 h = blas::cDotProductNormA(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::cDotProductNormB(*xD, *yD);
      double3 h = blas::cDotProductNormB(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = blas::caxpbypzYmbwcDotProductUYNormY(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 31:
    *xD = *xH;
    *yD = *yH;
    { double3 d = blas::HeavyQuarkResidualNorm(*xD, *yD);
      double3 h = blas::HeavyQuarkResidualNorm(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 32:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { double3 d = blas::xpyHeavyQuarkResidualNorm(*xD, *yD, *zD);
      double3 h = blas::xpyHeavyQuarkResidualNorm(*xH, *yH, *zH);
      error = ERROR(y) + fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 33:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { double3 d = blas::tripleCGReduction(*xD, *yD, *zD);
      double3 h = make_double3(blas::norm2(*xH), blas::norm2(*yH), blas::reDotProduct(*yH, *zH));
      error = fabs(d.x - h.x) / fabs(h.x) +
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 34:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    { blas::tripleCGUpdate(a, b, *xD, *yD, *zD, *wD);
      blas::tripleCGUpdate(a, b, *xH, *yH, *zH, *wH);
      error = ERROR(y) + ERROR(z) + ERROR(w); }
    break;

  case 35:
    *xD = *xH;
    *yD = *yH;
    { double d = blas::axpyReDot(a, *xD, *yD);
      double h = blas::axpyReDot(a, *xH, *yH);
      error = ERROR(y) + fabs(d-h)/fabs(h); }
    break;

  case 36:
    for (int i=0; i < Nsrc; i++) xmD->Component(i) = *(xmH[i]);
    for (int i=0; i < Msrc; i++) ymD->Component(i) = *(ymH[i]);

    blas::caxpy(A, *xmD, *ymD);
    for (int i=0; i < Nsrc; i++){
      for(int j=0; j < Msrc; j++){
	blas::caxpy(A[Msrc*i+j], *(xmH[i]), *(ymH[j]));
      }
    }
    error = 0;
    for (int i=0; i < Msrc; i++){
      error+= fabs(blas::norm2((ymD->Component(i))) - blas::norm2(*(ymH[i]))) / blas::norm2(*(ymH[i]));
    }
    error/= Msrc;
    break;

  case 37:
    for (int i=0; i < Nsrc; i++) {
      xmD->Component(i) = *(xmH[i]);
      zmD->Component(i) = *(zmH[i]);
    }
    *yD = *yH;

    blas::axpyBzpcx((double*)A, xmD->Components(), zmD->Components(), (double*)B, *yD, (const double*)C);

    for (int i=0; i<Nsrc; i++) {
      blas::axpyBzpcx(((double*)A)[i], *xmH[i], *zmH[i], ((double*)B)[i], *yH, ((double*)C)[i]);
    }

    error = 0;
    for (int i=0; i < Nsrc; i++){
      error+= fabs(blas::norm2((xmD->Component(i))) - blas::norm2(*(xmH[i]))) / blas::norm2(*(xmH[i]));
      //error+= fabs(blas::norm2((zmD->Component(i))) - blas::norm2(*(zmH[i]))) / blas::norm2(*(zmH[i]));
    }
    error/= Nsrc;
    break;
      
  case 38:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyBxpz(a, *xD, *yD, b2, *zD);
     blas::caxpyBxpz(a, *xH, *yH, b2, *zH);
     error = ERROR(x) + ERROR(z);}
    break;

  case 39:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {blas::caxpyBzpx(a, *xD, *yD, b2, *zD);
     blas::caxpyBzpx(a, *xH, *yH, b2, *zH);
     error = ERROR(x) + ERROR(z);}
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }
  delete[] A;
  delete[] B;
  delete[] C;
  return error;
}

const char *prec_str[] = {"half", "single", "double"};

const char *names[] = {
  "copyHS",
  "copyLS",
  "axpby",
  "xpy",
  "axpy",
  "xpay",
  "mxpy",
  "ax",
  "caxpy",
  "caxpby",
  "cxpaypbz",
  "axpyBzpcx",
  "axpyZpbx",
  "caxpbypzYmbw",
  "cabxpyAx",
  "caxpbypz",
  "caxpbypczpw",
  "caxpyXmaz",
  "norm",
  "reDotProduct",
  "axpyNorm",
  "xmyNorm",
  "caxpyNorm",
  "caxpyXmazNormX",
  "cabxpyAxNorm",
  "cDotProduct",
  "xpaycDotzy",
  "caxpyDotzy",
  "cDotProductNormA",
  "cDotProductNormB",
  "caxpbypzYmbwcDotProductUYNormY",
  "HeavyQuarkResidualNorm",
  "xpyHeavyQuarkResidualNorm",
  "tripleCGReduction",
  "tripleCGUpdate",
  "axpyReDot",
  "caxpy (block)",
  "axpyBzpcx (block)",
  "caxpyBxpz",
  "caxpyBzpx"
};

int main(int argc, char** argv)
{
  for (int i = 1; i < argc; i++){
    if(process_command_line_option(argc, argv, &i) == 0){
      continue;
    }
    printfQuda("ERROR: Invalid option:%s\n", argv[i]);
    usage(argv);
  }

  // override spin setting if mg solver is set to test coarse grids
  if (inv_type == QUDA_MG_INVERTER) {
    Nspin = 2;
    Ncolor = nvec;
  } else {
    // set spin according to the type of dslash
    Nspin = (dslash_type == QUDA_ASQTAD_DSLASH ||
	     dslash_type == QUDA_STAGGERED_DSLASH) ? 1 : 4;
    Ncolor = 3;
  }

  setSpinorSiteSize(24);
  initComms(argc, argv, gridsize_from_cmdline);
  display_test_info();
  initQuda(device);

  setVerbosity(QUDA_SILENT);

  for (int prec = 0; prec < Nprec; prec++) {
    if (Nspin == 2 && prec == 0) continue;

    printfQuda("\nBenchmarking %s precision with %d iterations...\n\n", prec_str[prec], niter);
    initFields(prec);

    for (int kernel = 0; kernel < Nkernels; kernel++) {
      if (skip_kernel(prec, kernel)) continue;

      // do the initial tune
      benchmark(kernel, 1);

      // now rerun with more iterations to get accurate speed measurements
      quda::blas::flops = 0;
      quda::blas::bytes = 0;

      double secs = benchmark(kernel, niter);

      double gflops = (quda::blas::flops*1e-9)/(secs);
      double gbytes = quda::blas::bytes/(secs*1e9);

      printfQuda("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
    }
    freeFields();
  }

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  if (verify_results) {
    ::testing::InitGoogleTest(&argc, argv);
    if (RUN_ALL_TESTS() != 0) warningQuda("Tests failed");
  }

  endQuda();

  finalizeComms();
}

// The following tests each kernel at each precision using the google testing framework

class BlasTest : public ::testing::TestWithParam<int2> {
protected:
  int2 param;

public:
  virtual ~BlasTest() { }
  virtual void SetUp() {
    param = GetParam();
    initFields(param.x);
  }
  virtual void TearDown() { freeFields(); }

  virtual void NormalExit() { printf("monkey\n"); }

};

TEST_P(BlasTest, verify) {
  int prec = param.x;
  int kernel = param.y;

  // certain tests will fail to run for coarse grids so mark these as
  // failed without running
  double deviation =  skip_kernel(prec,kernel) ? 1.0 : test(kernel);
  printfQuda("%-35s error = %e\n", names[kernel], deviation);
  double tol = (prec == 2 ? 1e-11 : (prec == 1 ? 1e-5 : 1e-3));
  tol = (kernel < 2) ? 1e-4 : tol; // use different tolerance for copy
  EXPECT_LE(deviation, tol) << "CPU and CUDA implementations do not agree";
}

// half precision
INSTANTIATE_TEST_CASE_P(copyHS_half, BlasTest, ::testing::Values( make_int2(0,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_half, BlasTest, ::testing::Values( make_int2(0,1) ));
INSTANTIATE_TEST_CASE_P(axpby_half, BlasTest, ::testing::Values( make_int2(0,2) ));
INSTANTIATE_TEST_CASE_P(xpy_half, BlasTest, ::testing::Values( make_int2(0,3) ));
INSTANTIATE_TEST_CASE_P(axpy_half, BlasTest, ::testing::Values( make_int2(0,4) ));
INSTANTIATE_TEST_CASE_P(xpay_half, BlasTest, ::testing::Values( make_int2(0,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_half, BlasTest, ::testing::Values( make_int2(0,6) ));
INSTANTIATE_TEST_CASE_P(ax_half, BlasTest, ::testing::Values( make_int2(0,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_half, BlasTest, ::testing::Values( make_int2(0,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_half, BlasTest, ::testing::Values( make_int2(0,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_half, BlasTest, ::testing::Values( make_int2(0,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_half, BlasTest, ::testing::Values( make_int2(0,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_half, BlasTest, ::testing::Values( make_int2(0,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_half, BlasTest, ::testing::Values( make_int2(0,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_half, BlasTest, ::testing::Values( make_int2(0,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_half, BlasTest, ::testing::Values( make_int2(0,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_half, BlasTest, ::testing::Values( make_int2(0,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_half, BlasTest, ::testing::Values( make_int2(0,17) ));
INSTANTIATE_TEST_CASE_P(norm2_half, BlasTest, ::testing::Values( make_int2(0,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_half, BlasTest, ::testing::Values( make_int2(0,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_half, BlasTest, ::testing::Values( make_int2(0,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_half, BlasTest, ::testing::Values( make_int2(0,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_half, BlasTest, ::testing::Values( make_int2(0,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_half, BlasTest, ::testing::Values( make_int2(0,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_half, BlasTest, ::testing::Values( make_int2(0,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_half, BlasTest, ::testing::Values( make_int2(0,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_half, BlasTest, ::testing::Values( make_int2(0,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_half, BlasTest, ::testing::Values( make_int2(0,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_half, BlasTest, ::testing::Values( make_int2(0,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_half, BlasTest, ::testing::Values( make_int2(0,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_half, BlasTest, ::testing::Values( make_int2(0,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_half, BlasTest, ::testing::Values( make_int2(0,31) ));
INSTANTIATE_TEST_CASE_P(xpyHeavyQuarkResidualNorm_half, BlasTest, ::testing::Values( make_int2(0,32) ));
INSTANTIATE_TEST_CASE_P(TripleCGReduction_half, BlasTest, ::testing::Values( make_int2(0,33) ));
INSTANTIATE_TEST_CASE_P(TripleCGUpdate_half, BlasTest, ::testing::Values( make_int2(0,34) ));
INSTANTIATE_TEST_CASE_P(axpyReDot_half, BlasTest, ::testing::Values( make_int2(0,35) ));
INSTANTIATE_TEST_CASE_P(multicaxpy_half, BlasTest, ::testing::Values( make_int2(0,36) ));
INSTANTIATE_TEST_CASE_P(multiaxpyBzpcx_half, BlasTest, ::testing::Values( make_int2(0,37) ));
INSTANTIATE_TEST_CASE_P(caxpyBxpz_half, BlasTest, ::testing::Values( make_int2(0,38) ));
INSTANTIATE_TEST_CASE_P(caxpyBzpx_half, BlasTest, ::testing::Values( make_int2(0,39) ));

// single precision
INSTANTIATE_TEST_CASE_P(copyHS_single, BlasTest, ::testing::Values( make_int2(1,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_single, BlasTest, ::testing::Values( make_int2(1,1) ));
INSTANTIATE_TEST_CASE_P(axpby_single, BlasTest, ::testing::Values( make_int2(1,2) ));
INSTANTIATE_TEST_CASE_P(xpy_single, BlasTest, ::testing::Values( make_int2(1,3) ));
INSTANTIATE_TEST_CASE_P(axpy_single, BlasTest, ::testing::Values( make_int2(1,4) ));
INSTANTIATE_TEST_CASE_P(xpay_single, BlasTest, ::testing::Values( make_int2(1,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_single, BlasTest, ::testing::Values( make_int2(1,6) ));
INSTANTIATE_TEST_CASE_P(ax_single, BlasTest, ::testing::Values( make_int2(1,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_single, BlasTest, ::testing::Values( make_int2(1,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_single, BlasTest, ::testing::Values( make_int2(1,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_single, BlasTest, ::testing::Values( make_int2(1,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_single, BlasTest, ::testing::Values( make_int2(1,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_single, BlasTest, ::testing::Values( make_int2(1,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_single, BlasTest, ::testing::Values( make_int2(1,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_single, BlasTest, ::testing::Values( make_int2(1,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_single, BlasTest, ::testing::Values( make_int2(1,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_single, BlasTest, ::testing::Values( make_int2(1,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_single, BlasTest, ::testing::Values( make_int2(1,17) ));
INSTANTIATE_TEST_CASE_P(norm2_single, BlasTest, ::testing::Values( make_int2(1,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_single, BlasTest, ::testing::Values( make_int2(1,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_single, BlasTest, ::testing::Values( make_int2(1,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_single, BlasTest, ::testing::Values( make_int2(1,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_single, BlasTest, ::testing::Values( make_int2(1,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_single, BlasTest, ::testing::Values( make_int2(1,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_single, BlasTest, ::testing::Values( make_int2(1,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_single, BlasTest, ::testing::Values( make_int2(1,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_single, BlasTest, ::testing::Values( make_int2(1,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_single, BlasTest, ::testing::Values( make_int2(1,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_single, BlasTest, ::testing::Values( make_int2(1,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_single, BlasTest, ::testing::Values( make_int2(1,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_single, BlasTest, ::testing::Values( make_int2(1,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_single, BlasTest, ::testing::Values( make_int2(1,31) ));
INSTANTIATE_TEST_CASE_P(xpyHeavyQuarkResidualNorm_single, BlasTest, ::testing::Values( make_int2(1,32) ));
INSTANTIATE_TEST_CASE_P(TripleCGReduction_single, BlasTest, ::testing::Values( make_int2(1,33) ));
INSTANTIATE_TEST_CASE_P(TripleCGUpdate_single, BlasTest, ::testing::Values( make_int2(1,34) ));
INSTANTIATE_TEST_CASE_P(axpyReDot_single, BlasTest, ::testing::Values( make_int2(1,35) ));
INSTANTIATE_TEST_CASE_P(multicaxpy_single, BlasTest, ::testing::Values( make_int2(1,36) ));
INSTANTIATE_TEST_CASE_P(multiaxpyBzpcx_single, BlasTest, ::testing::Values( make_int2(1,37) ));
INSTANTIATE_TEST_CASE_P(caxpyBxpz_single, BlasTest, ::testing::Values( make_int2(1,38) ));
INSTANTIATE_TEST_CASE_P(caxpyBzpx_single, BlasTest, ::testing::Values( make_int2(1,39) ));

// double precision
INSTANTIATE_TEST_CASE_P(copyHS_double, BlasTest, ::testing::Values( make_int2(2,0) ));
INSTANTIATE_TEST_CASE_P(copyLS_double, BlasTest, ::testing::Values( make_int2(2,1) ));
INSTANTIATE_TEST_CASE_P(axpby_double, BlasTest, ::testing::Values( make_int2(2,2) ));
INSTANTIATE_TEST_CASE_P(xpy_double, BlasTest, ::testing::Values( make_int2(2,3) ));
INSTANTIATE_TEST_CASE_P(axpy_double, BlasTest, ::testing::Values( make_int2(2,4) ));
INSTANTIATE_TEST_CASE_P(xpay_double, BlasTest, ::testing::Values( make_int2(2,5) ));
INSTANTIATE_TEST_CASE_P(mxpy_double, BlasTest, ::testing::Values( make_int2(2,6) ));
INSTANTIATE_TEST_CASE_P(ax_double, BlasTest, ::testing::Values( make_int2(2,7) ));
INSTANTIATE_TEST_CASE_P(caxpy_double, BlasTest, ::testing::Values( make_int2(2,8) ));
INSTANTIATE_TEST_CASE_P(caxpby_double, BlasTest, ::testing::Values( make_int2(2,9) ));
INSTANTIATE_TEST_CASE_P(cxpaypbz_double, BlasTest, ::testing::Values( make_int2(2,10) ));
INSTANTIATE_TEST_CASE_P(axpyBzpcx_double, BlasTest, ::testing::Values( make_int2(2,11) ));
INSTANTIATE_TEST_CASE_P(axpyZpbx_double, BlasTest, ::testing::Values( make_int2(2,12) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbw_double, BlasTest, ::testing::Values( make_int2(2,13) ));
INSTANTIATE_TEST_CASE_P(cabxpyAx_double, BlasTest, ::testing::Values( make_int2(2,14) ));
INSTANTIATE_TEST_CASE_P(caxpbypz_double, BlasTest, ::testing::Values( make_int2(2,15) ));
INSTANTIATE_TEST_CASE_P(caxpbypczpw_double, BlasTest, ::testing::Values( make_int2(2,16) ));
INSTANTIATE_TEST_CASE_P(caxpyXmaz_double, BlasTest, ::testing::Values( make_int2(2,17) ));
INSTANTIATE_TEST_CASE_P(norm2_double, BlasTest, ::testing::Values( make_int2(2,18) ));
INSTANTIATE_TEST_CASE_P(reDotProduct_double, BlasTest, ::testing::Values( make_int2(2,19) ));
INSTANTIATE_TEST_CASE_P(axpyNorm_double, BlasTest, ::testing::Values( make_int2(2,20) ));
INSTANTIATE_TEST_CASE_P(xmyNorm_double, BlasTest, ::testing::Values( make_int2(2,21) ));
INSTANTIATE_TEST_CASE_P(caxpyNorm_double, BlasTest, ::testing::Values( make_int2(2,22) ));
INSTANTIATE_TEST_CASE_P(caxpyXmazNormX_double, BlasTest, ::testing::Values( make_int2(2,23) ));
INSTANTIATE_TEST_CASE_P(cabxpyAxNorm_double, BlasTest, ::testing::Values( make_int2(2,24) ));
INSTANTIATE_TEST_CASE_P(cDotProduct_double, BlasTest, ::testing::Values( make_int2(2,25) ));
INSTANTIATE_TEST_CASE_P(xpaycDotzy_double, BlasTest, ::testing::Values( make_int2(2,26) ));
INSTANTIATE_TEST_CASE_P(caxpyDotzy_double, BlasTest, ::testing::Values( make_int2(2,27) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormA_double, BlasTest, ::testing::Values( make_int2(2,28) ));
INSTANTIATE_TEST_CASE_P(cDotProductNormB_double, BlasTest, ::testing::Values( make_int2(2,29) ));
INSTANTIATE_TEST_CASE_P(caxpbypzYmbwcDotProductWYNormY_double, BlasTest, ::testing::Values( make_int2(2,30) ));
INSTANTIATE_TEST_CASE_P(HeavyQuarkResidualNorm_double, BlasTest, ::testing::Values( make_int2(2,31) ));
INSTANTIATE_TEST_CASE_P(xpyHeavyQuarkResidualNorm_double, BlasTest, ::testing::Values( make_int2(2,32) ));
INSTANTIATE_TEST_CASE_P(TripleCGReduction_double, BlasTest, ::testing::Values( make_int2(2,33) ));
INSTANTIATE_TEST_CASE_P(TripleCGUpdate_double, BlasTest, ::testing::Values( make_int2(2,34) ));
INSTANTIATE_TEST_CASE_P(axpyReDot_double, BlasTest, ::testing::Values( make_int2(2,35) ));
INSTANTIATE_TEST_CASE_P(multicaxpy_double, BlasTest, ::testing::Values( make_int2(2,36) ));
INSTANTIATE_TEST_CASE_P(multiaxpyBzpcx_double, BlasTest, ::testing::Values( make_int2(2,37) ));
INSTANTIATE_TEST_CASE_P(caxpyBxpz_double, BlasTest, ::testing::Values( make_int2(2,38) ));
INSTANTIATE_TEST_CASE_P(caxpyBzpx_double, BlasTest, ::testing::Values( make_int2(2,39) ));

