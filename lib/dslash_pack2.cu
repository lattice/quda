#include "hip/hip_runtime.h"
#include <color_spinor_field.h>

// STRIPED - spread the blocks throughout the workload to ensure we
// work on all directions/dimensions simultanesouly to maximize NVLink saturation
#define STRIPED
// if not STRIPED then this means we assign one thread block per direction / dimension

#include <dslash_quda.h>
#include <kernels/dslash_pack.cuh>

namespace quda
{

  static int commDim[QUDA_MAX_DIM];

  int* getPackComms() { return commDim; }

  void setPackComms(const int *comm_dim)
  {
    for (int i = 0; i < 4; i++) commDim[i] = comm_dim[i];
    for (int i = 4; i < QUDA_MAX_DIM; i++) commDim[i] = 0;
  }

  template <typename Float, int nSpin, int nColor, bool spin_project>
  std::ostream &operator<<(std::ostream &out, const PackArg<Float, nSpin, nColor, spin_project> &arg)
  {
    out << "parity = " << arg.parity << std::endl;
    out << "nParity = " << arg.nParity << std::endl;
    out << "pc_type = " << arg.pc_type << std::endl;
    out << "nFace = " << arg.nFace << std::endl;
    out << "dagger = " << arg.dagger << std::endl;
    out << "a = " << arg.a << std::endl;
    out << "b = " << arg.b << std::endl;
    out << "c = " << arg.c << std::endl;
    out << "twist = " << arg.twist << std::endl;
    out << "threads = " << arg.threads << std::endl;
    out << "threadDimMapLower = { ";
    for (int i = 0; i < 4; i++) out << arg.threadDimMapLower[i] << (i < 3 ? ", " : " }");
    out << std::endl;
    out << "threadDimMapUpper = { ";
    for (int i = 0; i < 4; i++) out << arg.threadDimMapUpper[i] << (i < 3 ? ", " : " }");
    out << std::endl;
    out << "sites_per_block = " << arg.sites_per_block << std::endl;
    return out;
  }

  // FIXME - add CPU variant

  template <typename Float, int nColor, bool spin_project> class Pack : TunableVectorYZ
  {

protected:
    void **ghost;
    const ColorSpinorField &in;
    MemoryLocation location;
    const int nFace;
    const bool dagger; // only has meaning for nSpin=4
    const int parity;
    const int nParity;
    int threads;
    const double a;
    const double b;
    const double c;
    int twist; // only has meaning for nSpin=4

    bool tuneGridDim() const { return true; } // If striping, always tune grid dimension

    unsigned int maxGridSize() const
    {
      if (location & Host) {
#ifdef STRIPED
        // if zero-copy policy then set a maximum number of blocks to be
        // the 3 * number of dimensions we are communicating
        int max = 3;
#else
        // if zero-copy policy then assign exactly up to four thread blocks
        // per direction per dimension (effectively no grid-size tuning)
        int max = 2 * 4;
#endif
        int nDimComms = 0;
        for (int d = 0; d < in.Ndim(); d++) nDimComms += commDim[d];
        return max * nDimComms;
      } else {
        return TunableVectorYZ::maxGridSize();
      }
    } // use no more than a quarter of the GPU

    unsigned int minGridSize() const
    {
      if (location & Host) {
#ifdef STRIPED
        // if zero-copy policy then set a minimum number of blocks to be
        // the 1 * number of dimensions we are communicating
        int min = 3;
#else
        // if zero-copy policy then assign exactly one thread block
        // per direction per dimension (effectively no grid-size tuning)
        int min = 2;
#endif
        int nDimComms = 0;
        for (int d = 0; d < in.Ndim(); d++) nDimComms += commDim[d];
        return min * nDimComms;
      } else {
        return TunableVectorYZ::minGridSize();
      }
    }

    int gridStep() const
    {
#ifdef STRIPED
      return TunableVectorYZ::gridStep();
#else
      if (location & Host) {
        // the shmem kernel must ensure the grid size autotuner
        // increments in steps of 2 * number partitioned dimensions
        // for equal division of blocks to each direction/dimension
        int nDimComms = 0;
        for (int d = 0; d < in.Ndim(); d++) nDimComms += commDim[d];
        return 2 * nDimComms;
      } else {
        return TunableVectorYZ::gridStep();
      }
#endif
    }

    bool tuneAuxDim() const { return true; } // Do tune the aux dimensions.
    unsigned int minThreads() const { return threads; }

    void fillAux()
    {
      strcpy(aux, "policy_kernel,");
      strcat(aux, in.AuxString());
      char comm[5];
      for (int i = 0; i < 4; i++) comm[i] = (commDim[i] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux, ",comm=");
      strcat(aux, comm);
      strcat(aux, comm_dim_topology_string());
      if (in.PCType() == QUDA_5D_PC) { strcat(aux, ",5D_pc"); }
      if (dagger && in.Nspin() == 4) { strcat(aux, ",dagger"); }
      if (getKernelPackT()) { strcat(aux, ",kernelPackT"); }
      switch (nFace) {
      case 1: strcat(aux, ",nFace=1"); break;
      case 3: strcat(aux, ",nFace=3"); break;
      default: errorQuda("Number of faces not supported");
      }

      twist = ((b != 0.0) ? (c != 0.0 ? 2 : 1) : 0);
      if (twist && a == 0.0) errorQuda("Twisted packing requires non-zero scale factor a");
      if (twist) strcat(aux, twist == 2 ? ",twist-doublet" : ",twist-singlet");

#ifndef STRIPED
      if (location & Host) strcat(aux, ",shmem");
#endif

      // label the locations we are packing to
      // location label is nonp2p-p2p
      switch ((int)location) {
      case Device | Remote: strcat(aux, ",device-remote"); break;
      case Host | Remote: strcat(aux, ",host-remote"); break;
      case Device: strcat(aux, ",device-device"); break;
      case Host: strcat(aux, comm_peer2peer_enabled_global() ? ",host-device" : ",host-host"); break;
      default: errorQuda("Unknown pack target location %d\n", location);
      }
    }

public:
    Pack(void *ghost[], const ColorSpinorField &in, MemoryLocation location, int nFace, bool dagger, int parity,
        double a, double b, double c) :
        TunableVectorYZ((in.Ndim() == 5 ? in.X(4) : 1), in.SiteSubset()),
        ghost(ghost),
        in(in),
        location(location),
        nFace(nFace),
        dagger(dagger),
        parity(parity),
        nParity(in.SiteSubset()),
        threads(0),
        a(a),
        b(b),
        c(c)
    {
      fillAux();

      // compute number of threads - really number of active work items we have to do
      for (int i = 0; i < 4; i++) {
        if (!commDim[i]) continue;
        if (i == 3 && !getKernelPackT()) continue;
        threads += 2 * nFace * in.getDslashConstant().ghostFaceCB[i]; // 2 for forwards and backwards faces
      }
    }

    virtual ~Pack() {}

    template <typename T, typename Arg>
    inline void launch(T *f, const TuneParam &tp, Arg &arg, const qudaStream_t &stream)
    {
      if (deviceProp.major >= 7) { // enable max shared memory mode on GPUs that support it
        this->setMaxDynamicSharedBytesPerBlock(f);
      }

      void *args[] = {&arg};
      qudaLaunchKernel((const void *)f, tp.grid, tp.block, args, tp.shared_bytes, stream);
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

      if (in.Nspin() == 4) {
        using Arg = PackArg<Float, nColor, 4, spin_project>;
        Arg arg(ghost, in, nFace, dagger, parity, threads, a, b, c);
        arg.swizzle = tp.aux.x;
        arg.sites_per_block = (arg.threads + tp.grid.x - 1) / tp.grid.x;
        arg.blocks_per_dir = tp.grid.x / (2 * arg.active_dims); // set number of blocks per direction

#ifdef STRIPED
        if (in.PCType() == QUDA_4D_PC) {
          if (arg.dagger) {
            switch (arg.twist) {
            case 0: launch(packKernel<true, 0, QUDA_4D_PC, Arg>, tp, arg, stream); break;
            case 1: launch(packKernel<true, 1, QUDA_4D_PC, Arg>, tp, arg, stream); break;
            case 2: launch(packKernel<true, 2, QUDA_4D_PC, Arg>, tp, arg, stream); break;
            }
          } else {
            switch (arg.twist) {
            case 0: launch(packKernel<false, 0, QUDA_4D_PC, Arg>, tp, arg, stream); break;
            default: errorQuda("Twisted packing only for dagger");
            }
          }
        } else if (arg.pc_type == QUDA_5D_PC) {
          if (arg.twist) errorQuda("Twist packing not defined");
          if (arg.dagger) {
            launch(packKernel<true, 0, QUDA_5D_PC, Arg>, tp, arg, stream);
          } else {
            launch(packKernel<false, 0, QUDA_5D_PC, Arg>, tp, arg, stream);
          }
        } else {
          errorQuda("Unexpected preconditioning type %d", in.PCType());
        }
#else
        if (in.PCType() == QUDA_4D_PC) {
          if (arg.dagger) {
            switch (arg.twist) {
            case 0:
              launch(location & Host ? packShmemKernel<true, 0, QUDA_4D_PC, Arg> : packKernel<true, 0, QUDA_4D_PC, Arg>,
                  tp, arg, stream);
              break;
            case 1:
              launch(location & Host ? packShmemKernel<true, 1, QUDA_4D_PC, Arg> : packKernel<true, 0, QUDA_4D_PC, Arg>,
                  tp, arg, stream);
              break;
            case 2:
              launch(location & Host ? packShmemKernel<true, 2, QUDA_4D_PC, Arg> : packKernel<true, 2, QUDA_4D_PC, Arg>,
                  tp, arg, stream);
              break;
            }
          } else {
            switch (arg.twist) {
            case 0:
              launch(location & Host ? packShmemKernel<false, 0, QUDA_4D_PC, Arg> : packKernel<false, 0, QUDA_4D_PC, Arg>,
                  tp, arg, stream);
              break;
            default: errorQuda("Twisted packing only for dagger");
            }
          }
        } else if (arg.pc_type == QUDA_5D_PC) {
          if (arg.twist) errorQuda("Twist packing not defined");
          if (arg.dagger) {
            launch(packKernel<true, 0, QUDA_5D_PC, Arg>, tp, arg, stream);
          } else {
            launch(packKernel<false, 0, QUDA_5D_PC, Arg>, tp, arg, stream);
          }
        }
#endif
      } else if (in.Nspin() == 1) {
        using Arg = PackArg<Float, nColor, 1, false>;
        Arg arg(ghost, in, nFace, dagger, parity, threads, a, b, c);
        arg.swizzle = tp.aux.x;
        arg.sites_per_block = (arg.threads + tp.grid.x - 1) / tp.grid.x;
        arg.blocks_per_dir = tp.grid.x / (2 * arg.active_dims); // set number of blocks per direction

#ifdef STRIPED
        launch(packStaggeredKernel<Arg>, tp, arg, stream);
#else
        launch(location & Host ? packStaggeredShmemKernel<Arg> : packStaggeredKernel<Arg>, tp, arg, stream);
#endif
      } else {
        errorQuda("Unsupported nSpin = %d\n", in.Nspin());
      }
    }

    bool tuneSharedBytes() const { return false; }

#if 0
    // not used at present, but if tuneSharedBytes is enabled then
    // this allows tuning up the full dynamic shared memory if needed
    unsigned int maxSharedBytesPerBlock() const { return maxDynamicSharedBytesPerBlock(); }
#endif

    void initTuneParam(TuneParam &param) const
    {
      TunableVectorYZ::initTuneParam(param);
      // if doing a zero-copy policy then ensure that each thread block
      // runs exclusively on a given SM - this is to ensure quality of
      // service for the packing kernel when running concurrently.
      if (location & Host) param.shared_bytes = maxDynamicSharedBytesPerBlock() / 2 + 1;
#ifndef STRIPED
      if (location & Host) param.grid.x = minGridSize();
#endif
    }

    void defaultTuneParam(TuneParam &param) const
    {
      TunableVectorYZ::defaultTuneParam(param);
      // if doing a zero-copy policy then ensure that each thread block
      // runs exclusively on a given SM - this is to ensure quality of
      // service for the packing kernel when running concurrently.
      if (location & Host) param.shared_bytes = maxDynamicSharedBytesPerBlock() / 2 + 1;
#ifndef STRIPED
      if (location & Host) param.grid.x = minGridSize();
#endif
    }

    TuneKey tuneKey() const { return TuneKey(in.VolString(), typeid(*this).name(), aux); }

    int tuningIter() const { return 3; }

    long long flops() const
    {
      // unless we are spin projecting (nSpin = 4), there are no flops to do
      return in.Nspin() == 4 ? 2 * in.Nspin() / 2 * nColor * nParity * in.getDslashConstant().Ls * threads : 0;
    }

    long long bytes() const
    {
      size_t precision = sizeof(Float);
      size_t faceBytes = 2 * ((in.Nspin() == 4 ? in.Nspin() / 2 : in.Nspin()) + in.Nspin()) * nColor * precision;
      if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION)
        faceBytes += 2 * sizeof(float); // 2 is from input and output
      return faceBytes * nParity * in.getDslashConstant().Ls * threads;
    }
  };

  template <typename Float, int nColor>
  void PackGhost(void *ghost[], const ColorSpinorField &in, MemoryLocation location, int nFace, bool dagger, int parity,
                 bool spin_project, double a, double b, double c, const qudaStream_t &stream)
  {
    if (spin_project) {
      Pack<Float, nColor, true> pack(ghost, in, location, nFace, dagger, parity, a, b, c);
      pack.apply(stream);
    } else {
      Pack<Float, nColor, false> pack(ghost, in, location, nFace, dagger, parity, a, b, c);
      pack.apply(stream);
    }
  }

  // template on the number of colors
  template <typename Float>
  void PackGhost(void *ghost[], const ColorSpinorField &in, MemoryLocation location, int nFace, bool dagger, int parity,
                 bool spin_project, double a, double b, double c, const qudaStream_t &stream)
  {
    if (in.Ncolor() == 3) {
      PackGhost<Float, 3>(ghost, in, location, nFace, dagger, parity, spin_project, a, b, c, stream);
    } else {
      errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

  // Pack the ghost for the Dslash operator
  void PackGhost(void *ghost[2 * QUDA_MAX_DIM], const ColorSpinorField &in, MemoryLocation location, int nFace,
                 bool dagger, int parity, bool spin_project, double a, double b, double c, const qudaStream_t &stream)
  {
    int nDimPack = 0;
    for (int d = 0; d < 4; d++) {
      if (!commDim[d]) continue;
      if (d != 3 || getKernelPackT()) nDimPack++;
    }

    if (!nDimPack) return; // if zero then we have nothing to pack

    if (in.Precision() == QUDA_DOUBLE_PRECISION) {
#if QUDA_PRECISION & 8
      PackGhost<double>(ghost, in, location, nFace, dagger, parity, spin_project, a, b, c, stream);
#else
      errorQuda("QUDA_PRECISION=%d does not enable double precision", QUDA_PRECISION);
#endif
    } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
#if QUDA_PRECISION & 4
      PackGhost<float>(ghost, in, location, nFace, dagger, parity, spin_project, a, b, c, stream);
#else
      errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
    } else if (in.Precision() == QUDA_HALF_PRECISION) {
#if QUDA_PRECISION & 2
      PackGhost<short>(ghost, in, location, nFace, dagger, parity, spin_project, a, b, c, stream);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
#if QUDA_PRECISION & 1
      PackGhost<char>(ghost, in, location, nFace, dagger, parity, spin_project, a, b, c, stream);
#else
      errorQuda("QUDA_PRECISION=%d does not enable quarter precision", QUDA_PRECISION);
#endif
    } else {
      errorQuda("Unsupported precision %d\n", in.Precision());
    }
  }

} // namespace quda
