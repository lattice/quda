#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <ks_force_quda.h>
#include <index_helper.cuh>

namespace quda {

  using namespace gauge;

  template<typename Oprod, typename Gauge, typename Mom>
    struct KSForceArg {
      int threads;
      int X[4]; // grid dimensions
#ifndef BUILD_TIFR_INTERFACE
#ifdef MULTI_GPU
      int border[4];
#endif
#endif
      Oprod oprod;
      Gauge gauge;
      Mom mom;

      KSForceArg(Oprod& oprod, Gauge &gauge, Mom& mom, int dim[4])
        : oprod(oprod), gauge(gauge), mom(mom){

          threads = 1;
          for(int dir=0; dir<4; ++dir) threads *= dim[dir];

          for(int dir=0; dir<4; ++dir) X[dir] = dim[dir];
#ifndef BUILD_TIFR_INTERFACE
#ifdef MULTI_GPU
          for(int dir=0; dir<4; ++dir) border[dir] = 2;
#endif
#endif
        }

    };

  template<typename Float, typename Oprod, typename Gauge, typename Mom>
    __host__ __device__ void completeKSForceCore(KSForceArg<Oprod,Gauge,Mom>& arg, int idx){

      int parity = 0;
      if(idx >= arg.threads/2){
        parity = 1;
        idx -= arg.threads/2;
      }

      int X[4];
      for(int dir=0; dir<4; ++dir) X[dir] = arg.X[dir];

      int x[4];
      getCoords(x, idx, X, parity);
#ifndef BUILD_TIFR_INTERFACE
#ifdef MULTI_GPU
      for(int dir=0; dir<4; ++dir){
        x[dir] += arg.border[dir];
        X[dir] += 2*arg.border[dir];
      }
#endif
#endif

      Matrix<complex<Float>,3> O, G, M;

      int dx[4] = {0,0,0,0};
      for(int dir=0; dir<4; ++dir){
        G = arg.gauge(dir, linkIndexShift(x,dx,X), parity);
        O = arg.oprod(dir, linkIndexShift(x,dx,X), parity);
        if(parity==0){
          M = G*O;
        }else{
          M = -G*O;
        }

        makeAntiHerm(M);

        arg.mom(dir, idx, parity) = M;
      }
    }

  template<typename Float, typename Oprod, typename Gauge, typename Mom>
    __global__ void completeKSForceKernel(KSForceArg<Oprod,Gauge,Mom> arg)
    {
      int idx = threadIdx.x + blockIdx.x*blockDim.x;

      if(idx >= arg.threads) return;
      completeKSForceCore<Float,Oprod,Gauge,Mom>(arg,idx);
    }

  template<typename Float, typename Oprod, typename Gauge, typename Mom>
    void completeKSForceCPU(KSForceArg<Oprod,Gauge,Mom>& arg)
    {
      for(int idx=0; idx<arg.threads; idx++){
        completeKSForceCore<Float,Oprod,Gauge,Mom>(arg,idx);
      }
    }

  template<typename Float, typename Oprod, typename Gauge, typename Mom>
    class KSForceComplete : Tunable {

      KSForceArg<Oprod, Gauge, Mom> arg;
      const GaugeField &meta;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneSharedBytes() const { return false; } // Don't tune the shared memory.
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      KSForceComplete(KSForceArg<Oprod,Gauge,Mom> &arg, const GaugeField &meta, QudaFieldLocation location)
        : arg(arg), meta(meta), location(location) {
	writeAuxString("prec=%lu,stride=%d",sizeof(Float),arg.mom.stride);
      }

      virtual ~KSForceComplete() {}

      void apply(const qudaStream_t &stream) {
        if(location == QUDA_CUDA_FIELD_LOCATION){
          // Fix this
          dim3 blockDim(128, 1, 1);
          dim3 gridDim((arg.threads + blockDim.x - 1) / blockDim.x, 1, 1);
          completeKSForceKernel<Float><<<gridDim,blockDim>>>(arg);
        }else{
          completeKSForceCPU<Float>(arg);
        }
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

      long long flops() const { return 792*arg.X[0]*arg.X[1]*arg.X[2]*arg.X[3]; }
      long long bytes() const { return 0; } // Fix this
    };

  template<typename Float, typename Oprod, typename Gauge, typename Mom>
  void completeKSForce(Oprod oprod, Gauge gauge, Mom mom, int dim[4], const GaugeField &meta, QudaFieldLocation location, long long *flops)
    {
      KSForceArg<Oprod,Gauge,Mom> arg(oprod, gauge, mom, dim);
      KSForceComplete<Float,Oprod,Gauge,Mom> completeForce(arg,meta,location);
      completeForce.apply(0);
      if(flops) *flops = completeForce.flops();
      qudaDeviceSynchronize();
    }


  template<typename Float>
    void completeKSForce(GaugeField& mom, const GaugeField& oprod, const GaugeField& gauge, QudaFieldLocation location, long long *flops)
    {

      if(location != QUDA_CUDA_FIELD_LOCATION){
        errorQuda("Only QUDA_CUDA_FIELD_LOCATION currently supported");
      }else{
        if((oprod.Reconstruct() != QUDA_RECONSTRUCT_NO) || (gauge.Reconstruct() != QUDA_RECONSTRUCT_NO) || (mom.Reconstruct() != QUDA_RECONSTRUCT_10)){
          errorQuda("Reconstruct type not supported");
        }else{
          completeKSForce<Float>(FloatNOrder<Float, 18, 2, 18>(oprod),
				 FloatNOrder<Float, 18, 2, 18>(gauge),
				 FloatNOrder<Float, 10, 2, 10>(mom),
				 const_cast<int*>(mom.X()),
				 gauge, location, flops);
        }
      }
      return;
    }


  void completeKSForce(GaugeField &mom, const GaugeField &oprod, const GaugeField &gauge, QudaFieldLocation location, long long *flops)
  {
    if(mom.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported");
    }

    if(mom.Precision() == QUDA_SINGLE_PRECISION){
      completeKSForce<float>(mom, oprod, gauge, location, flops);
    }else if(mom.Precision() == QUDA_DOUBLE_PRECISION){
      completeKSForce<double>(mom, oprod, gauge, location, flops);
    }else{
      errorQuda("Precision %d not supported", mom.Precision());
    }
    return;
  }




  template<typename Result, typename Oprod, typename Gauge>
    struct KSLongLinkArg {
      int threads;
      int X[4]; // grid dimensions
#ifdef MULTI_GPU
      int border[4];
#endif
      double coeff;
      Result res;
      Oprod oprod;
      Gauge gauge;

      KSLongLinkArg(Result& res, Oprod& oprod, Gauge &gauge, int dim[4])
        : coeff(1.0), res(res), oprod(oprod), gauge(gauge){

          threads = 1;
#ifdef MULTI_GPU
          for(int dir=0; dir<4; ++dir) threads *= (dim[dir]-2);
          for(int dir=0; dir<4; ++dir) X[dir] = dim[dir]-2;
          for(int dir=0; dir<4; ++dir) border[dir] = 2;
#else
          for(int dir=0; dir<4; ++dir) threads *= dim[dir];
          for(int dir=0; dir<4; ++dir) X[dir] = dim[dir];
#endif
        }

    };



  template<typename Float, typename Result, typename Oprod, typename Gauge>
    __host__ __device__ void computeKSLongLinkForceCore(KSLongLinkArg<Result,Oprod,Gauge>& arg, int idx){

      /*
         int parity = 0;
         if(idx >= arg.threads/2){
         parity = 1;
         idx -= arg.threads/2;
         }

         int X[4];
         for(int dir=0; dir<4; ++dir) X[dir] = arg.X[dir];

         int x[4];
         getCoords(x, idx, X, parity);
#ifndef BUILD_TIFR_INTERFACE
#ifdef MULTI_GPU
for(int dir=0; dir<4; ++dir){
x[dir] += arg.border[dir];
X[dir] += 2*arg.border[dir];
}
#endif
#endif

typedef complex<Float> Cmplx;

Matrix<Cmplx,3> O;
Matrix<Cmplx,3> G;
Matrix<Cmplx,3> M;


int dx[4] = {0,0,0,0};
for(int dir=0; dir<4; ++dir){
arg.gauge.load((Float*)(G.data), linkIndexShift(x,dx,X), dir, parity);
arg.oprod.load((Float*)(O.data), linkIndexShift(x,dx,X), dir, parity);
if(parity==0){
M = G*O;
}else{
M = -G*O;
}

Float sub = getTrace(M).y/(static_cast<Float>(3));
Float temp[10];


temp[0] = (M.data[1].x - M.data[3].x)*0.5;
temp[1] = (M.data[1].y + M.data[3].y)*0.5;

temp[2] = (M.data[2].x - M.data[6].x)*0.5;
temp[3] = (M.data[2].y + M.data[6].y)*0.5;

temp[4] = (M.data[5].x - M.data[7].x)*0.5;
temp[5] = (M.data[5].y + M.data[7].y)*0.5;

temp[6] = (M.data[0].y-sub);
temp[7] = (M.data[4].y-sub);
temp[8] = (M.data[8].y-sub);
temp[9] = 0.0;

arg.mom.save(temp, idx, dir, parity);
}
       */
    }

  template<typename Float, typename Result, typename Oprod, typename Gauge>
__global__ void computeKSLongLinkForceKernel(KSLongLinkArg<Result,Oprod,Gauge> arg)
{
  int idx = threadIdx.x + blockIdx.x*blockDim.x;

  if(idx >= arg.threads) return;
  computeKSLongLinkForceCore<Float,Result,Oprod,Gauge>(arg,idx);
}




  template<typename Float, typename Result, typename Oprod, typename Gauge>
void computeKSLongLinkForceCPU(KSLongLinkArg<Result,Oprod,Gauge>& arg)
{
  for(int idx=0; idx<arg.threads; idx++){
    computeKSLongLinkForceCore<Float,Result,Oprod,Gauge>(arg,idx);
  }
}



// should be tunable
template<typename Float, typename Result, typename Oprod, typename Gauge>
class KSLongLinkForce : Tunable {


  KSLongLinkArg<Result,Oprod,Gauge> arg;
  const GaugeField &meta;
  const QudaFieldLocation location;

  private:
  unsigned int sharedBytesPerThread() const { return 0; }
  unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  bool tuneSharedBytes() const { return false; } // Don't tune the shared memory.
  bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
  unsigned int minThreads() const { return arg.threads; }

  public:
  KSLongLinkForce(KSLongLinkArg<Result,Oprod,Gauge> &arg, const GaugeField &meta, QudaFieldLocation location)
    : arg(arg), meta(meta), location(location) {
    writeAuxString("prec=%lu,stride=%d",sizeof(Float),arg.res.stride);
  }

  virtual ~KSLongLinkForce() {}

  void apply(const qudaStream_t &stream) {
    if(location == QUDA_CUDA_FIELD_LOCATION){
      // Fix this
      dim3 blockDim(128, 1, 1);
      dim3 gridDim((arg.threads + blockDim.x - 1) / blockDim.x, 1, 1);
      computeKSLongLinkForceKernel<Float><<<gridDim,blockDim>>>(arg);
    }else{
      computeKSLongLinkForceCPU<Float>(arg);
    }
  }

  TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

  long long flops() const { return 0; } // Fix this
  long long bytes() const { return 0; } // Fix this
};




template<typename Float, typename Result, typename Oprod, typename Gauge>
void computeKSLongLinkForce(Result res, Oprod oprod, Gauge gauge, int dim[4], const GaugeField &meta, QudaFieldLocation location)
{
  KSLongLinkArg<Result,Oprod,Gauge> arg(res, oprod, gauge, dim);
  KSLongLinkForce<Float,Result,Oprod,Gauge> computeLongLink(arg,meta,location);
  computeLongLink.apply(0);
  qudaDeviceSynchronize();
}

  template<typename Float>
void computeKSLongLinkForce(GaugeField& result, const GaugeField &oprod, const GaugeField &gauge, QudaFieldLocation location)
{
  if(location != QUDA_CUDA_FIELD_LOCATION){
    errorQuda("Only QUDA_CUDA_FIELD_LOCATION currently supported");
  }else{
    if((oprod.Reconstruct() != QUDA_RECONSTRUCT_NO) || (gauge.Reconstruct() != QUDA_RECONSTRUCT_NO) ||
        (result.Reconstruct() != QUDA_RECONSTRUCT_10)){

      errorQuda("Reconstruct type not supported");
    }else{
      computeKSLongLinkForce<Float>(FloatNOrder<Float, 18, 2, 18>(result),
				    FloatNOrder<Float, 18, 2, 18>(oprod),
				    FloatNOrder<Float, 18, 2, 18>(gauge),
				    const_cast<int*>(result.X()),
				    gauge, location);
    }
  }
  return;
}


void computeKSLongLinkForce(GaugeField &result, const GaugeField &oprod, const GaugeField &gauge, QudaFieldLocation location)
{
  if(result.Precision() == QUDA_HALF_PRECISION){
    errorQuda("Half precision not supported");
  }

  if(result.Precision() == QUDA_SINGLE_PRECISION){
    computeKSLongLinkForce<float>(result, oprod, gauge, location);
  }else if(result.Precision() == QUDA_DOUBLE_PRECISION){
    computeKSLongLinkForce<double>(result, oprod, gauge, location);
  }
  errorQuda("Precision %d not supported", result.Precision());
  return;
}

} // namespace quda
