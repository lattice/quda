#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <comm_quda.h>
#include <pgauge_monte.h>
#include <gauge_tools.h>
#include <random_quda.h>
#include <index_helper.cuh>
#include <atomic.cuh>
#include <cub_helper.cuh>



#ifndef PI
#define PI    3.1415926535897932384626433832795    // pi
#endif
#ifndef PII
#define PII   6.2831853071795864769252867665590    // 2 * pi
#endif

namespace quda {

#ifdef GPU_GAUGE_ALG


/**
    @brief Calculate the SU(2) index block in the SU(Nc) matrix
    @param block number to calculate the index's, the total number of blocks is NCOLORS * ( NCOLORS - 1) / 2.
    @return Returns two index's in int2 type, accessed by .x and .y.
 */
  template<int NCOLORS>
  __host__ __device__ static inline int2 IndexBlock(int block){
    int2 id;
    int i1;
    int found = 0;
    int del_i = 0;
    int index = -1;
    while ( del_i < (NCOLORS - 1) && found == 0 ) {
      del_i++;
      for ( i1 = 0; i1 < (NCOLORS - del_i); i1++ ) {
        index++;
        if ( index == block ) {
          found = 1;
          break;
        }
      }
    }
    id.y = i1 + del_i;
    id.x = i1;
    return id;
  }
/**
    @brief Calculate the SU(2) index block in the SU(Nc) matrix
    @param block number to calculate de index's, the total number of blocks is NCOLORS * ( NCOLORS - 1) / 2.
    @param p store the first index
    @param q store the second index
 */
  template<int NCOLORS>
  __host__ __device__ static inline void   IndexBlock(int block, int &p, int &q){
    if ( NCOLORS == 3 ) {
      if ( block == 0 ) { p = 0; q = 1; }
      else if ( block == 1 ) { p = 1; q = 2; }
      else{ p = 0; q = 2; }
    }
    else if ( NCOLORS > 3 ) {
      int i1;
      int found = 0;
      int del_i = 0;
      int index = -1;
      while ( del_i < (NCOLORS - 1) && found == 0 ) {
        del_i++;
        for ( i1 = 0; i1 < (NCOLORS - del_i); i1++ ) {
          index++;
          if ( index == block ) {
            found = 1;
            break;
          }
        }
      }
      q = i1 + del_i;
      p = i1;
    }
  }

/**
    @brief Generate full SU(2) matrix (four real numbers instead of 2x2 complex matrix) and update link matrix.
    Get from MILC code.
    @param al weight
    @param localstate CURAND rng state
 */
  template <class T>
  __device__ static inline Matrix<T,2> generate_su2_matrix_milc(T al, cuRNGState& localState){
    T xr1, xr2, xr3, xr4, d, r;
    int k;
    xr1 = Random<T>(localState);
    xr1 = (log((xr1 + 1.e-10)));
    xr2 = Random<T>(localState);
    xr2 = (log((xr2 + 1.e-10)));
    xr3 = Random<T>(localState);
    xr4 = Random<T>(localState);
    xr3 = cos(PII * xr3);
    d = -(xr2  + xr1 * xr3 * xr3 ) / al;
    //now  beat each  site into submission
    int nacd = 0;
    if ((1.00 - 0.5 * d) > xr4 * xr4 ) nacd = 1;
    if ( nacd == 0 && al > 2.0 ) { //k-p algorithm
      for ( k = 0; k < 20; k++ ) {
        //get four random numbers (add a small increment to prevent taking log(0.)
        xr1 = Random<T>(localState);
        xr1 = (log((xr1 + 1.e-10)));
        xr2 = Random<T>(localState);
        xr2 = (log((xr2 + 1.e-10)));
        xr3 = Random<T>(localState);
        xr4 = Random<T>(localState);
        xr3 = cos(PII * xr3);
        d = -(xr2 + xr1 * xr3 * xr3) / al;
        if ((1.00 - 0.5 * d) > xr4 * xr4 ) break;
      }
    } //endif nacd
    Matrix<T,2> a;
    if ( nacd == 0 && al <= 2.0 ) { //creutz algorithm
      xr3 = exp(-2.0 * al);
      xr4 = 1.0 - xr3;
      for ( k = 0; k < 20; k++ ) {
        //get two random numbers
        xr1 = Random<T>(localState);
        xr2 = Random<T>(localState);
        r = xr3 + xr4 * xr1;
        a(0,0) = 1.00 + log(r) / al;
        if ((1.0 - a(0,0) * a(0,0)) > xr2 * xr2 ) break;
      }
      d = 1.0 - a(0,0);
    } //endif nacd
      //generate the four su(2) elements
      //find a0  = 1 - d
    a(0,0) = 1.0 - d;
    //compute r
    xr3 = 1.0 - a(0,0) * a(0,0);
    xr3 = abs(xr3);
    r = sqrt(xr3);
    //compute a3
    a(1,1) = (2.0 * Random<T>(localState) - 1.0) * r;
    //compute a1 and a2
    xr1 = xr3 - a(1,1) * a(1,1);
    xr1 = abs(xr1);
    xr1 = sqrt(xr1);
    //xr2 is a random number between 0 and 2*pi
    xr2 = PII * Random<T>(localState);
    a(0,1) = xr1 * cos(xr2);
    a(1,0) = xr1 * sin(xr2);
    return a;
  }


/**
    @brief Return SU(2) subgroup (4 real numbers) from SU(3) matrix
    @param tmp1 input SU(3) matrix
    @param block to retrieve from 0 to 2.
    @return 4 real numbers
 */
  template < class T>
  __host__ __device__ static inline Matrix<T,2> get_block_su2( Matrix<complex<T>,3> tmp1, int block ){
    Matrix<T,2> r;
    switch ( block ) {
    case 0:
      r(0,0) = tmp1(0,0).x + tmp1(1,1).x;
      r(0,1) = tmp1(0,1).y + tmp1(1,0).y;
      r(1,0) = tmp1(0,1).x - tmp1(1,0).x;
      r(1,1) = tmp1(0,0).y - tmp1(1,1).y;
      break;
    case 1:
      r(0,0) = tmp1(1,1).x + tmp1(2,2).x;
      r(0,1) = tmp1(1,2).y + tmp1(2,1).y;
      r(1,0) = tmp1(1,2).x - tmp1(2,1).x;
      r(1,1) = tmp1(1,1).y - tmp1(2,2).y;
      break;
    case 2:
      r(0,0) = tmp1(0,0).x + tmp1(2,2).x;
      r(0,1) = tmp1(0,2).y + tmp1(2,0).y;
      r(1,0) = tmp1(0,2).x - tmp1(2,0).x;
      r(1,1) = tmp1(0,0).y - tmp1(2,2).y;
      break;
    }
    return r;
  }

/**
    @brief Return SU(2) subgroup (4 real numbers) from SU(Nc) matrix
    @param tmp1 input SU(Nc) matrix
    @param id the two indices to retrieve SU(2) block
    @return 4 real numbers
 */
  template <class T, int NCOLORS>
  __host__ __device__ static inline Matrix<T,2> get_block_su2( Matrix<complex<T>,NCOLORS> tmp1, int2 id ){
    Matrix<T,2> r;
    r(0,0) = tmp1(id.x,id.x).x + tmp1(id.y,id.y).x;
    r(0,1) = tmp1(id.x,id.y).y + tmp1(id.y,id.x).y;
    r(1,0) = tmp1(id.x,id.y).x - tmp1(id.y,id.x).x;
    r(1,1) = tmp1(id.x,id.x).y - tmp1(id.y,id.y).y;
    return r;
  }

/**
    @brief Create a SU(Nc) identity matrix and fills with the SU(2) block
    @param rr SU(2) matrix represented only by four real numbers
    @param id the two indices to fill in the SU(3) matrix
    @return SU(Nc) matrix
 */
  template <class T, int NCOLORS>
  __host__ __device__ static inline Matrix<complex<T>,NCOLORS> block_su2_to_sun( Matrix<T,2> rr, int2 id ){
    Matrix<complex<T>,NCOLORS> tmp1;
    setIdentity(&tmp1);
    tmp1(id.x,id.x) = complex<T>( rr(0,0), rr(1,1) );
    tmp1(id.x,id.y) = complex<T>( rr(1,0), rr(0,1) );
    tmp1(id.y,id.x) = complex<T>(-rr(1,0), rr(0,1) );
    tmp1(id.y,id.y) = complex<T>( rr(0,0),-rr(1,1) );
    return tmp1;
  }
/**
    @brief Update the SU(Nc) link with the new SU(2) matrix, link <- u * link
    @param u SU(2) matrix represented by four real numbers
    @param link SU(Nc) matrix
    @param id indices
 */
  template <class T, int NCOLORS>
  __host__ __device__ static inline void mul_block_sun( Matrix<T,2> u, Matrix<complex<T>,NCOLORS> &link, int2 id ){
    for ( int j = 0; j < NCOLORS; j++ ) {
      complex<T> tmp = complex<T>( u(0,0), u(1,1) ) * link(id.x, j) + complex<T>( u(1,0), u(0,1) ) * link(id.y, j);
      link(id.y, j) = complex<T>(-u(1,0), u(0,1) ) * link(id.x, j) + complex<T>( u(0,0),-u(1,1) ) * link(id.y, j);
      link(id.x, j) = tmp;
    }
  }

/**
    @brief Update the SU(3) link with the new SU(2) matrix, link <- u * link
    @param U SU(3) matrix
    @param a00 element (0,0) of the SU(2) matrix
    @param a01 element (0,1) of the SU(2) matrix
    @param a10 element (1,0) of the SU(2) matrix
    @param a11 element (1,1) of the SU(2) matrix
    @param block of the SU(3) matrix, 0,1 or 2
 */
  template <class Cmplx>
  __host__ __device__ static inline void block_su2_to_su3( Matrix<Cmplx,3> &U, Cmplx a00, Cmplx a01, Cmplx a10, Cmplx a11, int block ){
    Cmplx tmp;
    switch ( block ) {
    case 0:
      tmp = a00 * U(0,0) + a01 * U(1,0);
      U(1,0) = a10 * U(0,0) + a11 * U(1,0);
      U(0,0) = tmp;
      tmp = a00 * U(0,1) + a01 * U(1,1);
      U(1,1) = a10 * U(0,1) + a11 * U(1,1);
      U(0,1) = tmp;
      tmp = a00 * U(0,2) + a01 * U(1,2);
      U(1,2) = a10 * U(0,2) + a11 * U(1,2);
      U(0,2) = tmp;
      break;
    case 1:
      tmp = a00 * U(1,0) + a01 * U(2,0);
      U(2,0) = a10 * U(1,0) + a11 * U(2,0);
      U(1,0) = tmp;
      tmp = a00 * U(1,1) + a01 * U(2,1);
      U(2,1) = a10 * U(1,1) + a11 * U(2,1);
      U(1,1) = tmp;
      tmp = a00 * U(1,2) + a01 * U(2,2);
      U(2,2) = a10 * U(1,2) + a11 * U(2,2);
      U(1,2) = tmp;
      break;
    case 2:
      tmp = a00 * U(0,0) + a01 * U(2,0);
      U(2,0) = a10 * U(0,0) + a11 * U(2,0);
      U(0,0) = tmp;
      tmp = a00 * U(0,1) + a01 * U(2,1);
      U(2,1) = a10 * U(0,1) + a11 * U(2,1);
      U(0,1) = tmp;
      tmp = a00 * U(0,2) + a01 * U(2,2);
      U(2,2) = a10 * U(0,2) + a11 * U(2,2);
      U(0,2) = tmp;
      break;
    }
  }



// v * u^dagger
  template <class Float>
  __host__ __device__ static inline Matrix<Float,2> mulsu2UVDagger(Matrix<Float,2> v, Matrix<Float,2> u){
    Matrix<Float,2> b;
    b(0,0) = v(0,0) * u(0,0) + v(0,1) * u(0,1) + v(1,0) * u(1,0) + v(1,1) * u(1,1);
    b(0,1) = v(0,1) * u(0,0) - v(0,0) * u(0,1) + v(1,0) * u(1,1) - v(1,1) * u(1,0);
    b(1,0) = v(1,0) * u(0,0) - v(0,0) * u(1,0) + v(1,1) * u(0,1) - v(0,1) * u(1,1);
    b(1,1) = v(1,1) * u(0,0) - v(0,0) * u(1,1) + v(0,1) * u(1,0) - v(1,0) * u(0,1);
    return b;
  }

/**
    @brief Link update by pseudo-heatbath
    @param U link to be updated
    @param F staple
    @param localstate CURAND rng state
 */
  template <class Float, int NCOLORS>
  __device__ inline void heatBathSUN( Matrix<complex<Float>,NCOLORS>& U, Matrix<complex<Float>,NCOLORS> F,
                                      cuRNGState& localState, Float BetaOverNc ){

    if ( NCOLORS == 3 ) {
      //////////////////////////////////////////////////////////////////
      /*
         for( int block = 0; block < NCOLORS; block++ ) {
         Matrix<complex<T>,3> tmp1 = U * F;
         Matrix<T,2> r = get_block_su2<T>(tmp1, block);
         T k = sqrt(r(0,0)*r(0,0)+r(0,1)*r(0,1)+r(1,0)*r(1,0)+r(1,1)*r(1,1));
         T ap = BetaOverNc * k;
         k = (T)1.0 / k;
         r *= k;
         //Matrix<T,2> a = generate_su2_matrix<T4, T>(ap, localState);
         Matrix<T,2> a = generate_su2_matrix_milc<T>(ap, localState);
         r = mulsu2UVDagger_4<T>( a, r);
         ///////////////////////////////////////
         block_su2_to_su3<T>( U, complex( r(0,0), r(1,1) ), complex( r(1,0), r(0,1) ), complex(-r(1,0), r(0,1) ), complex( r(0,0),-r(1,1) ), block );
         //FLOP_min = (198 + 4 + 15 + 28 + 28 + 84) * 3 = 1071
         }*/
      //////////////////////////////////////////////////////////////////

      for ( int block = 0; block < NCOLORS; block++ ) {
        int p,q;
        IndexBlock<NCOLORS>(block, p, q);
        complex<Float> a0((Float)0.0, (Float)0.0);
        complex<Float> a1 = a0;
        complex<Float> a2 = a0;
        complex<Float> a3 = a0;

        for ( int j = 0; j < NCOLORS; j++ ) {
          a0 += U(p,j) * F(j,p);
          a1 += U(p,j) * F(j,q);
          a2 += U(q,j) * F(j,p);
          a3 += U(q,j) * F(j,q);
        }
        Matrix<Float,2> r;
        r(0,0) = a0.x + a3.x;
        r(0,1) = a1.y + a2.y;
        r(1,0) = a1.x - a2.x;
        r(1,1) = a0.y - a3.y;
        Float k = sqrt(r(0,0) * r(0,0) + r(0,1) * r(0,1) + r(1,0) * r(1,0) + r(1,1) * r(1,1));;
        Float ap = BetaOverNc * k;
        k = 1.0 / k;
        r *= k;
        Matrix<Float,2> a = generate_su2_matrix_milc<Float>(ap, localState);
        r = mulsu2UVDagger<Float>( a, r);
        ///////////////////////////////////////
        a0 = complex<Float>( r(0,0), r(1,1) );
        a1 = complex<Float>( r(1,0), r(0,1) );
        a2 = complex<Float>(-r(1,0), r(0,1) );
        a3 = complex<Float>( r(0,0),-r(1,1) );
        complex<Float> tmp0;

        for ( int j = 0; j < NCOLORS; j++ ) {
          tmp0 = a0 * U(p,j) + a1 * U(q,j);
          U(q,j) = a2 * U(p,j) + a3 * U(q,j);
          U(p,j) = tmp0;
        }
        //FLOP_min = (NCOLORS * 64 + 19 + 28 + 28) * 3 = NCOLORS * 192 + 225
      }
      //////////////////////////////////////////////////////////////////
    }
    else if ( NCOLORS > 3 ) {
      //////////////////////////////////////////////////////////////////
      //TESTED IN SU(4) SP THIS IS WORST
      Matrix<complex<Float>,NCOLORS> M = U * F;
      for ( int block = 0; block < NCOLORS * ( NCOLORS - 1) / 2; block++ ) {
        int2 id = IndexBlock<NCOLORS>( block );
        Matrix<Float,2> r = get_block_su2<Float>(M, id);
        Float k = sqrt(r(0,0) * r(0,0) + r(0,1) * r(0,1) + r(1,0) * r(1,0) + r(1,1) * r(1,1));
        Float ap = BetaOverNc * k;
        k = 1.0 / k;
        r *= k;
        Matrix<Float,2> a = generate_su2_matrix_milc<Float>(ap, localState);
        Matrix<Float,2> rr = mulsu2UVDagger<Float>( a, r);
        ///////////////////////////////////////
        mul_block_sun<Float, NCOLORS>( rr, U, id);
        mul_block_sun<Float, NCOLORS>( rr, M, id);
        ///////////////////////////////////////
      }
      /* / TESTED IN SU(4) SP THIS IS FASTER
         for ( int block = 0; block < NCOLORS * ( NCOLORS - 1) / 2; block++ ) {
         int2 id = IndexBlock<NCOLORS>( block );
         complex a0 = complex::zero();
         complex a1 = complex::zero();
         complex a2 = complex::zero();
         complex a3 = complex::zero();

         for ( int j = 0; j < NCOLORS; j++ ) {
          a0 += U(id.x, j) * F.e[j][id.x];
          a1 += U(id.x, j) * F.e[j][id.y];
          a2 += U(id.y, j) * F.e[j][id.x];
          a3 += U(id.y, j) * F.e[j][id.y];
         }
         Matrix<T,2> r;
         r(0,0) = a0.x + a3.x;
         r(0,1) = a1.y + a2.y;
         r(1,0) = a1.x - a2.x;
         r(1,1) = a0.y - a3.y;
         T k = sqrt(r(0,0) * r(0,0) + r(0,1) * r(0,1) + r(1,0) * r(1,0) + r(1,1) * r(1,1));
         T ap = BetaOverNc * k;
         k = (T)1.0 / k;
         r *= k;
         //Matrix<T,2> a = generate_su2_matrix<T4, T>(ap, localState);
         Matrix<T,2> a = generate_su2_matrix_milc<T>(ap, localState);
         r = mulsu2UVDagger<T>( a, r);
         mul_block_sun<T>( r, U, id); */
         /*
           a0 = complex( r(0,0), r(1,1) );
           a1 = complex( r(1,0), r(0,1) );
           a2 = complex(-r(1,0), r(0,1) );
           a3 = complex( r(0,0),-r(1,1) );
           complex tmp0;

           for ( int j = 0; j < NCOLORS; j++ ) {
           tmp0 = a0 * U(id.x, j) + a1 * U(id.y, j);
           U(id.y, j) = a2 * U(id.x, j) + a3 * U(id.y, j);
           U(id.x, j) = tmp0;
           } */
      // }

    }
    //////////////////////////////////////////////////////////////////
  }

  //////////////////////////////////////////////////////////////////////////
  /**
     @brief Link update by overrelaxation
     @param U link to be updated
     @param F staple
   */
  template <class Float, int NCOLORS>
  __device__ inline void overrelaxationSUN( Matrix<complex<Float>,NCOLORS>& U, Matrix<complex<Float>,NCOLORS> F ){

    if ( NCOLORS == 3 ) {
      //////////////////////////////////////////////////////////////////
      /*
         for( int block = 0; block < 3; block++ ) {
         Matrix<complex<T>,3> tmp1 = U * F;
         Matrix<T,2> r = get_block_su2<T>(tmp1, block);
         //normalize and conjugate
         Float norm = 1.0 / sqrt(r(0,0)*r(0,0)+r(0,1)*r(0,1)+r(1,0)*r(1,0)+r(1,1)*r(1,1));;
         r(0,0) *= norm;
         r(0,1) *= -norm;
         r(1,0) *= -norm;
         r(1,1) *= -norm;
         ///////////////////////////////////////
         complex a00 = complex( r(0,0), r(1,1) );
         complex a01 = complex( r(1,0), r(0,1) );
         complex a10 = complex(-r(1,0), r(0,1) );
         complex a11 = complex( r(0,0),-r(1,1) );
         block_su2_to_su3<T>( U, a00, a01, a10, a11, block );
         block_su2_to_su3<T>( U, a00, a01, a10, a11, block );

         //FLOP = (198 + 17 + 84 * 2) * 3 = 1149
         }*/
      ///////////////////////////////////////////////////////////////////
      //This version does not need to multiply all matrix at each block: tmp1 = U * F;
      //////////////////////////////////////////////////////////////////

      for ( int block = 0; block < 3; block++ ) {
        int p,q;
        IndexBlock<NCOLORS>(block, p, q);
        complex<Float> a0((Float)0., (Float)0.);
        complex<Float> a1 = a0;
        complex<Float> a2 = a0;
        complex<Float> a3 = a0;

        for ( int j = 0; j < NCOLORS; j++ ) {
          a0 += U(p,j) * F(j,p);
          a1 += U(p,j) * F(j,q);
          a2 += U(q,j) * F(j,p);
          a3 += U(q,j) * F(j,q);
        }
        Matrix<Float,2> r;
        r(0,0) = a0.x + a3.x;
        r(0,1) = a1.y + a2.y;
        r(1,0) = a1.x - a2.x;
        r(1,1) = a0.y - a3.y;
        //normalize and conjugate
        //r = r.conj_normalize();
        Float norm = 1.0 / sqrt(r(0,0) * r(0,0) + r(0,1) * r(0,1) + r(1,0) * r(1,0) + r(1,1) * r(1,1));;
        r(0,0) *= norm;
        r(0,1) *= -norm;
        r(1,0) *= -norm;
        r(1,1) *= -norm;


        ///////////////////////////////////////
        a0 = complex<Float>( r(0,0), r(1,1) );
        a1 = complex<Float>( r(1,0), r(0,1) );
        a2 = complex<Float>(-r(1,0), r(0,1) );
        a3 = complex<Float>( r(0,0),-r(1,1) );
        complex<Float> tmp0, tmp1;

        for ( int j = 0; j < NCOLORS; j++ ) {
          tmp0 = a0 * U(p,j) + a1 * U(q,j);
          tmp1 = a2 * U(p,j) + a3 * U(q,j);
          U(p,j) = a0 * tmp0 + a1 * tmp1;
          U(q,j) = a2 * tmp0 + a3 * tmp1;
        }
        //FLOP = (NCOLORS * 88 + 17) * 3
      }
      ///////////////////////////////////////////////////////////////////
    }
    else if ( NCOLORS > 3 ) {
      ///////////////////////////////////////////////////////////////////
      Matrix<complex<Float>,NCOLORS> M = U * F;
      for ( int block = 0; block < NCOLORS * ( NCOLORS - 1) / 2; block++ ) {
        int2 id = IndexBlock<NCOLORS>( block );
        Matrix<Float,2> r = get_block_su2<Float, NCOLORS>(M, id);
        //normalize and conjugate
        Float norm = 1.0 / sqrt(r(0,0) * r(0,0) + r(0,1) * r(0,1) + r(1,0) * r(1,0) + r(1,1) * r(1,1));;
        r(0,0) *= norm;
        r(0,1) *= -norm;
        r(1,0) *= -norm;
        r(1,1) *= -norm;
        mul_block_sun<Float, NCOLORS>( r, U, id);
        mul_block_sun<Float, NCOLORS>( r, U, id);
        mul_block_sun<Float, NCOLORS>( r, M, id);
        mul_block_sun<Float, NCOLORS>( r, M, id);
        ///////////////////////////////////////
      }
      /*  //TESTED IN SU(4) SP THIS IS WORST
          for( int block = 0; block < NCOLORS * ( NCOLORS - 1) / 2; block++ ) {
         int2 id = IndexBlock<NCOLORS>( block );
          complex a0 = complex::zero();
          complex a1 = complex::zero();
          complex a2 = complex::zero();
          complex a3 = complex::zero();

          for(int j = 0; j < NCOLORS; j++){
         a0 += U(id.x, j) * F.e[j][id.x];
         a1 += U(id.x, j) * F.e[j][id.y];
         a2 += U(id.y, j) * F.e[j][id.x];
         a3 += U(id.y, j) * F.e[j][id.y];
          }
          Matrix<T,2> r;
          r(0,0) = a0.x + a3.x;
          r(0,1) = a1.y + a2.y;
          r(1,0) = a1.x - a2.x;
          r(1,1) = a0.y - a3.y;
          //normalize and conjugate
          Float norm = 1.0 / sqrt(r(0,0)*r(0,0)+r(0,1)*r(0,1)+r(1,0)*r(1,0)+r(1,1)*r(1,1));;
          r(0,0) *= norm;
          r(0,1) *= -norm;
          r(1,0) *= -norm;
          r(1,1) *= -norm;
          //mul_block_sun<T>( r, U, id);
          //mul_block_sun<T>( r, U, id);
          ///////////////////////////////////////
          a0 = complex( r(0,0), r(1,1) );
          a1 = complex( r(1,0), r(0,1) );
          a2 = complex(-r(1,0), r(0,1) );
          a3 = complex( r(0,0),-r(1,1) );
          complex tmp0, tmp1;

          for(int j = 0; j < NCOLORS; j++){
          tmp0 = a0 * U(id.x, j) + a1 * U(id.y, j);
          tmp1 = a2 * U(id.x, j) + a3 * U(id.y, j);
          U(id.x, j) = a0 * tmp0 + a1 * tmp1;
          U(id.y, j) = a2 * tmp0 + a3 * tmp1;
          }
          }
       */
    }
  }


  template <typename Gauge, typename Float, int NCOLORS>
  struct MonteArg {
    int threads;       // number of active threads required
    int X[4];       // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    cudaGaugeField &data;
    Float BetaOverNc;
    RNG rngstate;
    MonteArg(const Gauge &dataOr, cudaGaugeField & data, Float Beta, RNG &rngstate)
      : dataOr(dataOr), data(data), rngstate(rngstate) {
      BetaOverNc = Beta / (Float)NCOLORS;
#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir] * 2;
      } 
#else
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
#endif
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
    }
  };


  template<typename Float, typename Gauge, int NCOLORS, bool HeatbathOrRelax>
  __global__ void compute_heatBath(MonteArg<Gauge, Float, NCOLORS> arg, int mu, int parity){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx >= arg.threads ) return;
    int id = idx;
    int X[4];
    #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];

    int x[4];
    getCoords(x, idx, X, parity);
#ifdef MULTI_GPU
    #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
    idx = linkIndex(x,X);
#endif

    Matrix<complex<Float>,NCOLORS> staple;
    setZero(&staple);

    Matrix<complex<Float>,NCOLORS> U;
    for ( int nu = 0; nu < 4; nu++ ) if ( mu != nu ) {
        int dx[4] = { 0, 0, 0, 0 };
        Matrix<complex<Float>,NCOLORS> link = arg.dataOr(nu, idx, parity);
        dx[nu]++;
        U = arg.dataOr(mu, linkIndexShift(x,dx,X), 1 - parity);
        link *= U;
        dx[nu]--;
        dx[mu]++;
        U = arg.dataOr(nu, linkIndexShift(x,dx,X), 1 - parity);
        link *= conj(U);
        staple += link;
        dx[mu]--;
        dx[nu]--;
        link = arg.dataOr(nu, linkIndexShift(x,dx,X), 1 - parity);
        U = arg.dataOr(mu, linkIndexShift(x,dx,X), 1 - parity);
        link = conj(link) * U;
        dx[mu]++;
        U = arg.dataOr(nu, linkIndexShift(x,dx,X), parity);
        link *= U;
        staple += link;
      }
    U = arg.dataOr(mu, idx, parity);
    if ( HeatbathOrRelax ) {
      cuRNGState localState = arg.rngstate.State()[ id ];
      heatBathSUN<Float, NCOLORS>( U, conj(staple), localState, arg.BetaOverNc );
      arg.rngstate.State()[ id ] = localState;
    }
    else{
      overrelaxationSUN<Float, NCOLORS>( U, conj(staple) );
    }
    arg.dataOr(mu, idx, parity) = U;
  }


  template<typename Float, typename Gauge, int NCOLORS, int NElems, bool HeatbathOrRelax>
  class GaugeHB : Tunable {
    MonteArg<Gauge, Float, NCOLORS> arg;
    int mu;
    int parity;
    mutable char aux_string[128];       // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false;  } // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeHB(MonteArg<Gauge, Float, NCOLORS> &arg)
      : arg(arg), mu(0), parity(0) {
    }
    ~GaugeHB () {
    }
    void SetParam(int _mu, int _parity){
      mu = _mu;
      parity = _parity;
    }
    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      compute_heatBath<Float, Gauge, NCOLORS, HeatbathOrRelax > <<< tp.grid,tp.block, tp.shared_bytes, stream >>> (arg, mu, parity);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu",arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    }

    void preTune() {
      arg.data.backup();
      if(HeatbathOrRelax) arg.rngstate.backup();
    }
    void postTune() {
      arg.data.restore();
      if(HeatbathOrRelax) arg.rngstate.restore();
    }
    long long flops() const {

      //NEED TO CHECK THIS!!!!!!
      if ( NCOLORS == 3 ) {
        long long flop = 2268LL;
        if ( HeatbathOrRelax ) {
          flop += 801LL;
        }
        else{
          flop += 843LL;
        }
        flop *= arg.threads;
        return flop;
      }
      else{
        long long flop = NCOLORS * NCOLORS * NCOLORS * 84LL;
        if ( HeatbathOrRelax ) {
          flop += NCOLORS * NCOLORS * NCOLORS + (NCOLORS * ( NCOLORS - 1) / 2) * (46LL + 48LL + 56LL * NCOLORS);
        }
        else{
          flop += NCOLORS * NCOLORS * NCOLORS + (NCOLORS * ( NCOLORS - 1) / 2) * (17LL + 112LL * NCOLORS);
        }
        flop *= arg.threads;
        return flop;
      }
    }
    long long bytes() const {
      //NEED TO CHECK THIS!!!!!!
      if ( NCOLORS == 3 ) {
        long long byte = 20LL * NElems * sizeof(Float);
        if ( HeatbathOrRelax ) byte += 2LL * sizeof(cuRNGState);
        byte *= arg.threads;
        return byte;
      }
      else{
        long long byte = 20LL * NCOLORS * NCOLORS * 2 * sizeof(Float);
        if ( HeatbathOrRelax ) byte += 2LL * sizeof(cuRNGState);
        byte *= arg.threads;
        return byte;
      }
    }
  };









  template<typename Float, int NElems, int NCOLORS, typename Gauge>
  void Monte( Gauge dataOr,  cudaGaugeField& data, RNG &rngstate, Float Beta, int nhb, int nover) {

    TimeProfile profileHBOVR("HeatBath_OR_Relax", false);
    MonteArg<Gauge, Float, NCOLORS> montearg(dataOr, data, Beta, rngstate);
    if ( getVerbosity() >= QUDA_SUMMARIZE ) profileHBOVR.TPSTART(QUDA_PROFILE_COMPUTE);
    GaugeHB<Float, Gauge, NCOLORS, NElems, true> hb(montearg);
    for ( int step = 0; step < nhb; ++step ) {
      for ( int parity = 0; parity < 2; ++parity ) {
        for ( int mu = 0; mu < 4; ++mu ) {
          hb.SetParam(mu, parity);
          hb.apply(0);
        #ifdef MULTI_GPU
          PGaugeExchange( data, mu, parity);
        #endif
        }
      }
    }
    if ( getVerbosity() >= QUDA_VERBOSE ) {
      qudaDeviceSynchronize();
      profileHBOVR.TPSTOP(QUDA_PROFILE_COMPUTE);
      double secs = profileHBOVR.Last(QUDA_PROFILE_COMPUTE);
      double gflops = (hb.flops() * 8 * nhb * 1e-9) / (secs);
      double gbytes = hb.bytes() * 8 * nhb / (secs * 1e9);
    #ifdef MULTI_GPU
      printfQuda("HB: Time = %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops * comm_size(), gbytes * comm_size());
    #else
      printfQuda("HB: Time = %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
    #endif
    }

    if ( getVerbosity() >= QUDA_VERBOSE ) profileHBOVR.TPSTART(QUDA_PROFILE_COMPUTE);
    GaugeHB<Float, Gauge, NCOLORS, NElems, false> relax(montearg);
    for ( int step = 0; step < nover; ++step ) {
      for ( int parity = 0; parity < 2; ++parity ) {
        for ( int mu = 0; mu < 4; ++mu ) {
          relax.SetParam(mu, parity);
          relax.apply(0);
        #ifdef MULTI_GPU
          PGaugeExchange( data, mu, parity);
        #endif
        }
      }
    }
    if ( getVerbosity() >= QUDA_VERBOSE ) {
      qudaDeviceSynchronize();
      profileHBOVR.TPSTOP(QUDA_PROFILE_COMPUTE);
      double secs = profileHBOVR.Last(QUDA_PROFILE_COMPUTE);
      double gflops = (relax.flops() * 8 * nover * 1e-9) / (secs);
      double gbytes = relax.bytes() * 8 * nover / (secs * 1e9);
    #ifdef MULTI_GPU
      printfQuda("OVR: Time = %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops * comm_size(), gbytes * comm_size());
    #else
      printfQuda("OVR: Time = %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
    #endif
    }
  }



  template<typename Float>
  void Monte( cudaGaugeField& data, RNG &rngstate, Float Beta, int nhb, int nover) {

    if ( data.isNative() ) {
      if ( data.Reconstruct() == QUDA_RECONSTRUCT_NO ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;	
        Monte<Float, 18, 3>(Gauge(data), data, rngstate, Beta, nhb, nover);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_12 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;	
        Monte<Float, 12, 3>(Gauge(data), data, rngstate, Beta, nhb, nover);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_8 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;	
        Monte<Float, 8, 3>(Gauge(data), data, rngstate, Beta, nhb, nover);
      } else {
        errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }
    } else {
      errorQuda("Invalid Gauge Order\n");
    }
  }
#endif // GPU_GAUGE_ALG

/** @brief Perform heatbath and overrelaxation. Performs nhb heatbath steps followed by nover overrelaxation steps.
 *
 * @param[in,out] data Gauge field
 * @param[in,out] rngstate state of the CURAND random number generator
 * @param[in] Beta inverse of the gauge coupling, beta = 2 Nc / g_0^2
 * @param[in] nhb number of heatbath steps
 * @param[in] nover number of overrelaxation steps
 */
  void Monte( cudaGaugeField& data, RNG &rngstate, double Beta, int nhb, int nover) {
#ifdef GPU_GAUGE_ALG
    if ( data.Precision() == QUDA_SINGLE_PRECISION ) {
      Monte<float> (data, rngstate, (float)Beta, nhb, nover);
    } else if ( data.Precision() == QUDA_DOUBLE_PRECISION ) {
      Monte<double>(data, rngstate, Beta, nhb, nover);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
#else
    errorQuda("Pure gauge code has not been built");
#endif // GPU_GAUGE_ALG
  }


}
