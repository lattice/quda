#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <stack>

#include <color_spinor_field.h>
#include <clover_field.h>
#include <dslash_quda.h>
#include <color_spinor_field_order.h>
#include <clover_field_order.h>
#include <index_helper.cuh>
#include <color_spinor.h>
#include <linalg.cuh>
#include <dslash_policy.cuh>

namespace quda {

  // these should not be namespaced!!
  // determines whether the temporal ghost zones are packed with a gather kernel,
  // as opposed to multiple calls to hipMemcpy()
  static bool kernelPackT = false;

  void setKernelPackT(bool packT) { kernelPackT = packT; }

  bool getKernelPackT() { return kernelPackT; }

  static std::stack<bool> kptstack;

  void pushKernelPackT(bool packT)
  {
    kptstack.push(getKernelPackT());
    setKernelPackT(packT);

    if (kptstack.size() > 10)
    {
      warningQuda("KernelPackT stack contains %u elements.  Is there a missing popKernelPackT() somewhere?",
      static_cast<unsigned int>(kptstack.size()));
    }
  }

  void popKernelPackT()
  {
    if (kptstack.empty())
    {
      errorQuda("popKernelPackT() called with empty stack");
    }
    setKernelPackT(kptstack.top());
    kptstack.pop();
  }

  namespace dslash {
    int it = 0;

    hipEvent_t packEnd[2];
    hipEvent_t gatherStart[Nstream];
    hipEvent_t gatherEnd[Nstream];
    hipEvent_t scatterStart[Nstream];
    hipEvent_t scatterEnd[Nstream];
    hipEvent_t dslashStart[2];

    // these variables are used for benchmarking the dslash components in isolation
    bool dslash_pack_compute;
    bool dslash_interior_compute;
    bool dslash_exterior_compute;
    bool dslash_comms;
    bool dslash_copy;

    // whether the dslash policy tuner has been enabled
    bool dslash_policy_init;

    // used to keep track of which policy to start the autotuning
    int first_active_policy;
    int first_active_p2p_policy;

    // list of dslash policies that are enabled
    std::vector<QudaDslashPolicy> policies;

    // list of p2p policies that are enabled
    std::vector<QudaP2PPolicy> p2p_policies;

    // string used as a tunekey to ensure we retune if the dslash policy env changes
    char policy_string[TuneKey::aux_n];

    // FIX this is a hack from hell
    // Auxiliary work that can be done while waiting on comms to finis
    Worker *aux_worker;

#if CUDA_VERSION >= 8000
    cuuint32_t *commsEnd_h;
    hipDeviceptr_t commsEnd_d[Nstream];
#endif
  }

  void createDslashEvents()
  {
    using namespace dslash;
    // add hipEventDisableTiming for lower sync overhead
    for (int i=0; i<Nstream; i++) {
      hipEventCreateWithFlags(&gatherStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&gatherEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterEnd[i], hipEventDisableTiming);
    }
    for (int i=0; i<2; i++) {
      hipEventCreateWithFlags(&packEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&dslashStart[i], hipEventDisableTiming);
    }

    aux_worker = NULL;

#if CUDA_VERSION >= 8000
    commsEnd_h = static_cast<cuuint32_t*>(mapped_malloc(Nstream*sizeof(int)));
    for (int i=0; i<Nstream; i++) {
      hipHostGetDevicePointer((void**)&commsEnd_d[i], commsEnd_h+i, 0);
      commsEnd_h[i] = 0;
    }
#endif

    checkCudaError();

    dslash_pack_compute = true;
    dslash_interior_compute = true;
    dslash_exterior_compute = true;
    dslash_comms = true;
    dslash_copy = true;

    dslash_policy_init = false;
    first_active_policy = 0;
    first_active_p2p_policy = 0;

    // list of dslash policies that are enabled
    policies = std::vector<QudaDslashPolicy>(
        static_cast<int>(QudaDslashPolicy::QUDA_DSLASH_POLICY_DISABLED), QudaDslashPolicy::QUDA_DSLASH_POLICY_DISABLED);

    // list of p2p policies that are enabled
    p2p_policies = std::vector<QudaP2PPolicy>(
        static_cast<int>(QudaP2PPolicy::QUDA_P2P_POLICY_DISABLED), QudaP2PPolicy::QUDA_P2P_POLICY_DISABLED);

    strcat(policy_string, ",pol=");
  }


  void destroyDslashEvents()
  {
    using namespace dslash;

#if CUDA_VERSION >= 8000
    host_free(commsEnd_h);
    commsEnd_h = 0;
#endif

    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(gatherStart[i]);
      hipEventDestroy(gatherEnd[i]);
      hipEventDestroy(scatterStart[i]);
      hipEventDestroy(scatterEnd[i]);
    }

    for (int i=0; i<2; i++) {
      hipEventDestroy(packEnd[i]);
      hipEventDestroy(dslashStart[i]);
    }

    checkCudaError();
  }

  /**
     @brief Parameter structure for driving the Gamma operator
   */
  template <typename Float, int nColor>
  struct GammaArg {
    typedef typename colorspinor_mapper<Float,4,nColor>::type F;
    typedef typename mapper<Float>::type RegType;

    F out;                // output vector field
    const F in;           // input vector field
    const int d;          // which gamma matrix are we applying
    const int nParity;    // number of parities we're working on
    bool doublet;         // whether we applying the operator to a doublet
    const int volumeCB;   // checkerboarded volume
    RegType a;            // scale factor
    RegType b;            // chiral twist
    RegType c;            // flavor twist

    GammaArg(ColorSpinorField &out, const ColorSpinorField &in, int d,
	     RegType kappa=0.0, RegType mu=0.0, RegType epsilon=0.0,
	     bool dagger=false, QudaTwistGamma5Type twist=QUDA_TWIST_GAMMA5_INVALID)
      : out(out), in(in), d(d), nParity(in.SiteSubset()),
	doublet(in.TwistFlavor() == QUDA_TWIST_DEG_DOUBLET || in.TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET),
	volumeCB(doublet ? in.VolumeCB()/2 : in.VolumeCB()), a(0.0), b(0.0), c(0.0)
    {
      if (d < 0 || d > 4) errorQuda("Undefined gamma matrix %d", d);
      if (in.Nspin() != 4) errorQuda("Cannot apply gamma5 to nSpin=%d field", in.Nspin());
      if (!in.isNative() || !out.isNative()) errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());

      if (in.TwistFlavor() == QUDA_TWIST_SINGLET) {
	if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
          b = 2.0 * kappa * mu;
          a = 1.0;
        } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
          b = -2.0 * kappa * mu;
          a = 1.0 / (1.0 + b * b);
        }
	c = 0.0;
        if (dagger) b *= -1.0;
      } else if (doublet) {
        if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
          b = 2.0 * kappa * mu;
          c = -2.0 * kappa * epsilon;
          a = 1.0;
        } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
          b = -2.0 * kappa * mu;
          c = 2.0 * kappa * epsilon;
          a = 1.0 / (1.0 + b * b - c * c);
          if (a <= 0) errorQuda("Invalid twisted mass parameters (kappa=%e, mu=%e, epsilon=%e)\n", kappa, mu, epsilon);
        }
        if (dagger) b *= -1.0;
      }
    }
  };

  // CPU kernel for applying the gamma matrix to a colorspinor
  template <typename Float, int nColor, typename Arg>
  void gammaCPU(Arg arg)
  {
    typedef typename mapper<Float>::type RegType;
    for (int parity= 0; parity < arg.nParity; parity++) {

      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	ColorSpinor<RegType,nColor,4> in = arg.in(x_cb, parity);
	arg.out(x_cb, parity) = in.gamma(arg.d);
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying the gamma matrix to a colorspinor
  template <typename Float, int nColor, int d, typename Arg>
  __global__ void gammaGPU(Arg arg)
  {
    typedef typename mapper<Float>::type RegType;
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = blockDim.y*blockIdx.y + threadIdx.y;

    if (x_cb >= arg.volumeCB) return;
    if (parity >= arg.nParity) return;

    ColorSpinor<RegType,nColor,4> in = arg.in(x_cb, parity);
    arg.out(x_cb, parity) = in.gamma(d);
  }

  template <typename Float, int nColor, typename Arg>
  class Gamma : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const { return arg.out.Bytes() + arg.in.Bytes(); }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    Gamma(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
    }
    virtual ~Gamma() { }

    void apply(const qudaStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	gammaCPU<Float,nColor>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	switch (arg.d) {
	case 4: gammaGPU<Float,nColor,4> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg); break;
	default: errorQuda("%d not instantiated", arg.d);
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    void preTune() { arg.out.save(); }
    void postTune() { arg.out.load(); }
  };


  template <typename Float, int nColor>
  void ApplyGamma(ColorSpinorField &out, const ColorSpinorField &in, int d)
  {
    GammaArg<Float,nColor> arg(out, in, d);
    Gamma<Float,nColor,GammaArg<Float,nColor> > gamma(arg, in);
    gamma.apply(streams[Nstream-1]);
  }

  // template on the number of colors
  template <typename Float>
  void ApplyGamma(ColorSpinorField &out, const ColorSpinorField &in, int d)
  {
    if (in.Ncolor() == 3) {
      ApplyGamma<Float,3>(out, in, d);
    } else {
      errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

  //Apply the Gamma matrix to a colorspinor field
  //out(x) = gamma_d*in
  void ApplyGamma(ColorSpinorField &out, const ColorSpinorField &in, int d)
  {
    checkPrecision(out, in);    // check all precisions match
    checkLocation(out, in);     // check all locations match

    if (in.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyGamma<double>(out, in, d);
    } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyGamma<float>(out, in, d);
    } else if (in.Precision() == QUDA_HALF_PRECISION) {
      ApplyGamma<short>(out, in, d);
    } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyGamma<char>(out, in, d);
    } else {
      errorQuda("Unsupported precision %d\n", in.Precision());
    }
  }

  // CPU kernel for applying the gamma matrix to a colorspinor
  template <bool doublet, typename Float, int nColor, typename Arg>
  void twistGammaCPU(Arg arg)
  {
    typedef typename mapper<Float>::type RegType;
    for (int parity= 0; parity < arg.nParity; parity++) {
      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	if (!doublet) {
	  ColorSpinor<RegType,nColor,4> in = arg.in(x_cb, parity);
          arg.out(x_cb, parity) = arg.a * (in + arg.b * in.igamma(arg.d));
        } else {
	  ColorSpinor<RegType,nColor,4> in_1 = arg.in(x_cb+0*arg.volumeCB, parity);
	  ColorSpinor<RegType,nColor,4> in_2 = arg.in(x_cb+1*arg.volumeCB, parity);
          arg.out(x_cb + 0 * arg.volumeCB, parity) = arg.a * (in_1 + arg.b * in_1.igamma(arg.d) + arg.c * in_2);
          arg.out(x_cb + 1 * arg.volumeCB, parity) = arg.a * (in_2 - arg.b * in_2.igamma(arg.d) + arg.c * in_1);
        }
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying the gamma matrix to a colorspinor
  template <bool doublet, typename Float, int nColor, int d, typename Arg>
  __global__ void twistGammaGPU(Arg arg)
  {
    typedef typename mapper<Float>::type RegType;
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = blockDim.y*blockIdx.y + threadIdx.y;
    if (x_cb >= arg.volumeCB) return;

    if (!doublet) {
      ColorSpinor<RegType,nColor,4> in = arg.in(x_cb, parity);
      arg.out(x_cb, parity) = arg.a * (in + arg.b * in.igamma(d));
    } else {
      ColorSpinor<RegType,nColor,4> in_1 = arg.in(x_cb+0*arg.volumeCB, parity);
      ColorSpinor<RegType,nColor,4> in_2 = arg.in(x_cb+1*arg.volumeCB, parity);
      arg.out(x_cb + 0 * arg.volumeCB, parity) = arg.a * (in_1 + arg.b * in_1.igamma(d) + arg.c * in_2);
      arg.out(x_cb + 1 * arg.volumeCB, parity) = arg.a * (in_2 - arg.b * in_2.igamma(d) + arg.c * in_1);
    }
  }

  template <typename Float, int nColor, typename Arg>
  class TwistGamma : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const { return 0; }
    long long bytes() const { return arg.out.Bytes() + arg.in.Bytes(); }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    TwistGamma(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
    }
    virtual ~TwistGamma() { }

    void apply(const qudaStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (arg.doublet) twistGammaCPU<true,Float,nColor>(arg);
	twistGammaCPU<false,Float,nColor>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	if (arg.doublet)
	  switch (arg.d) {
	  case 4: twistGammaGPU<true,Float,nColor,4> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg); break;
	  default: errorQuda("%d not instantiated", arg.d);
	  }
	else
	  switch (arg.d) {
	  case 4: twistGammaGPU<false,Float,nColor,4> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg); break;
	  default: errorQuda("%d not instantiated", arg.d);
	  }
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    void preTune() { if (arg.out.field == arg.in.field) arg.out.save(); }
    void postTune() { if (arg.out.field == arg.in.field) arg.out.load(); }
  };


  template <typename Float, int nColor>
  void ApplyTwistGamma(ColorSpinorField &out, const ColorSpinorField &in, int d, double kappa, double mu, double epsilon, int dagger, QudaTwistGamma5Type type)
  {
    GammaArg<Float,nColor> arg(out, in, d, kappa, mu, epsilon, dagger, type);
    TwistGamma<Float,nColor,GammaArg<Float,nColor> > gamma(arg, in);
    gamma.apply(streams[Nstream-1]);

    checkCudaError();
  }

  // template on the number of colors
  template <typename Float>
  void ApplyTwistGamma(ColorSpinorField &out, const ColorSpinorField &in, int d, double kappa, double mu, double epsilon, int dagger, QudaTwistGamma5Type type)
  {
    if (in.Ncolor() == 3) {
      ApplyTwistGamma<Float,3>(out, in, d, kappa, mu, epsilon, dagger, type);
    } else {
      errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

  //Apply the Gamma matrix to a colorspinor field
  //out(x) = gamma_d*in
  void ApplyTwistGamma(ColorSpinorField &out, const ColorSpinorField &in, int d, double kappa, double mu, double epsilon, int dagger, QudaTwistGamma5Type type)
  {
    checkPrecision(out, in);    // check all precisions match
    checkLocation(out, in);     // check all locations match

#ifdef GPU_TWISTED_MASS_DIRAC
    if (in.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyTwistGamma<double>(out, in, d, kappa, mu, epsilon, dagger, type);
    } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyTwistGamma<float>(out, in, d, kappa, mu, epsilon, dagger, type);
    } else if (in.Precision() == QUDA_HALF_PRECISION) {
      ApplyTwistGamma<short>(out, in, d, kappa, mu, epsilon, dagger, type);
    } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyTwistGamma<char>(out, in, d, kappa, mu, epsilon, dagger, type);
    } else {
      errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

  // Applies a gamma5 matrix to a spinor (wrapper to ApplyGamma)
  void gamma5(ColorSpinorField &out, const ColorSpinorField &in) { ApplyGamma(out,in,4); }

  /**
     @brief Parameteter structure for driving the clover and twist-clover application kernels
     @tparam Float Underlying storage precision
     @tparam nSpin Number of spin components
     @tparam nColor Number of colors
     @tparam dynamic_clover Whether we are inverting the clover field on the fly
  */
  template <typename Float, int nSpin, int nColor>
  struct CloverArg {
    static constexpr int length = (nSpin / (nSpin/2)) * 2 * nColor * nColor * (nSpin/2) * (nSpin/2) / 2;
    static constexpr bool dynamic_clover = dynamic_clover_inverse();

    typedef typename colorspinor_mapper<Float,nSpin,nColor>::type F;
    typedef typename clover_mapper<Float,length>::type C;
    typedef typename mapper<Float>::type RegType;

    F out;                // output vector field
    const F in;           // input vector field
    const C clover;       // clover field
    const C cloverInv;    // inverse clover field (only set if not dynamic clover and doing twisted clover)
    const int nParity;    // number of parities we're working on
    const int parity;     // which parity we're acting on (if nParity=1)
    bool inverse;         // whether we are applying the inverse
    bool doublet;         // whether we applying the operator to a doublet
    const int volumeCB;   // checkerboarded volume
    RegType a;
    RegType b;
    RegType c;
    QudaTwistGamma5Type twist;

    CloverArg(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover,
	      bool inverse, int parity, RegType kappa=0.0, RegType mu=0.0, RegType epsilon=0.0,
	      bool dagger = false, QudaTwistGamma5Type twist=QUDA_TWIST_GAMMA5_INVALID)
      : out(out), clover(clover, twist == QUDA_TWIST_GAMMA5_INVALID ? inverse : false),
	cloverInv(clover, (twist != QUDA_TWIST_GAMMA5_INVALID && !dynamic_clover) ? true : false),
	in(in), nParity(in.SiteSubset()), parity(parity), inverse(inverse),
	doublet(in.TwistFlavor() == QUDA_TWIST_DEG_DOUBLET || in.TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET),
        volumeCB(doublet ? in.VolumeCB()/2 : in.VolumeCB()), a(0.0), b(0.0), c(0.0), twist(twist)
    {
      if (in.TwistFlavor() == QUDA_TWIST_SINGLET) {
	if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
	  a = 2.0 * kappa * mu;
	  b = 1.0;
	} else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
	  a = -2.0 * kappa * mu;
	  b = 1.0 / (1.0 + a*a);
	}
	c = 0.0;
	if (dagger) a *= -1.0;
      } else if (doublet) {
	errorQuda("ERROR: Non-degenerated twisted-mass not supported in this regularization\n");
      }
    }
  };

  template <typename Float, int nSpin, int nColor, typename Arg>
  __device__ __host__ inline void cloverApply(Arg &arg, int x_cb, int parity) {
    using namespace linalg; // for Cholesky
    typedef typename mapper<Float>::type RegType;
    typedef ColorSpinor<RegType, nColor, nSpin> Spinor;
    typedef ColorSpinor<RegType, nColor, nSpin / 2> HalfSpinor;
    int spinor_parity = arg.nParity == 2 ? parity : 0;
    Spinor in = arg.in(x_cb, spinor_parity);
    Spinor out;

    in.toRel(); // change to chiral basis here

#pragma unroll
    for (int chirality=0; chirality<2; chirality++) {

      HMatrix<RegType,nColor*nSpin/2> A = arg.clover(x_cb, parity, chirality);
      HalfSpinor chi = in.chiral_project(chirality);

      if (arg.dynamic_clover) {
        Cholesky<HMatrix, RegType, nColor * nSpin / 2> cholesky(A);
        chi = static_cast<RegType>(0.25) * cholesky.backward(cholesky.forward(chi));
      } else {
        chi = A * chi;
      }

      out += chi.chiral_reconstruct(chirality);
    }

    out.toNonRel(); // change basis back

    arg.out(x_cb, spinor_parity) = out;
  }

  template <typename Float, int nSpin, int nColor, typename Arg>
  void cloverCPU(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;
      for (int x_cb=0; x_cb<arg.volumeCB; x_cb++) cloverApply<Float,nSpin,nColor>(arg, x_cb, parity);
    }
  }

  template <typename Float, int nSpin, int nColor, typename Arg>
  __global__ void cloverGPU(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = (arg.nParity == 2) ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
    if (x_cb >= arg.volumeCB) return;
    cloverApply<Float,nSpin,nColor>(arg, x_cb, parity);
  }

  template <typename Float, int nSpin, int nColor, typename Arg>
  class Clover : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

  protected:
    long long flops() const { return arg.nParity*arg.volumeCB*504ll; }
    long long bytes() const { return arg.out.Bytes() + arg.in.Bytes() + arg.nParity*arg.volumeCB*arg.clover.Bytes(); }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    Clover(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
    }
    virtual ~Clover() { }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	cloverCPU<Float,nSpin,nColor>(arg);
      } else {
	cloverGPU<Float,nSpin,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    void preTune() { if (arg.out.field == arg.in.field) arg.out.save(); }  // Need to save the out field if it aliases the in field
    void postTune() { if (arg.out.field == arg.in.field) arg.out.load(); } // Restore if the in and out fields alias
  };


  template <typename Float, int nColor>
  void ApplyClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover, bool inverse, int parity)
  {
    if (in.Nspin() != 4) errorQuda("Unsupported nSpin=%d", in.Nspin());
    constexpr int nSpin = 4;

    if (inverse) {
      CloverArg<Float, nSpin, nColor> arg(out, in, clover, inverse, parity);
      Clover<Float, nSpin, nColor, decltype(arg)> worker(arg, in);
      worker.apply(streams[Nstream - 1]);
    } else {
      CloverArg<Float, nSpin, nColor> arg(out, in, clover, inverse, parity);
      Clover<Float, nSpin, nColor, decltype(arg)> worker(arg, in);
      worker.apply(streams[Nstream - 1]);
    }

    checkCudaError();
  }

  // template on the number of colors
  template <typename Float>
  void ApplyClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover, bool inverse, int parity)
  {
    if (in.Ncolor() == 3) {
      ApplyClover<Float,3>(out, in, clover, inverse, parity);
    } else {
      errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

  //Apply the clvoer matrix field to a colorspinor field
  //out(x) = clover*in
  void ApplyClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover, bool inverse, int parity)
  {
    checkPrecision(out, clover, in);    // check all precisions match
    checkLocation(out, clover, in);     // check all locations match

#ifdef GPU_CLOVER_DIRAC
    if (in.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyClover<double>(out, in, clover, inverse, parity);
    } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyClover<float>(out, in, clover, inverse, parity);
    } else if (in.Precision() == QUDA_HALF_PRECISION) {
      ApplyClover<short>(out, in, clover, inverse, parity);
    } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyClover<char>(out, in, clover, inverse, parity);
    } else {
      errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Clover dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

  // if (!inverse) apply (Clover + i*a*gamma_5) to the input spinor
  // else apply (Clover + i*a*gamma_5)/(Clover^2 + a^2) to the input spinor
  template <bool inverse, typename Float, int nSpin, int nColor, typename Arg>
  __device__ __host__ inline void twistCloverApply(Arg &arg, int x_cb, int parity) {
    using namespace linalg; // for Cholesky
    constexpr int N = nColor*nSpin/2;
    typedef typename mapper<Float>::type RegType;
    typedef ColorSpinor<RegType,nColor,nSpin> Spinor;
    typedef ColorSpinor<RegType,nColor,nSpin/2> HalfSpinor;
    typedef HMatrix<RegType,N> Mat;
    int spinor_parity = arg.nParity == 2 ? parity : 0;
    Spinor in = arg.in(x_cb, spinor_parity);
    Spinor out;

    in.toRel(); // change to chiral basis here

#pragma unroll
    for (int chirality=0; chirality<2; chirality++) {
      // factor of 2 comes from clover normalization we need to correct for
      const complex<RegType> j(0.0, chirality == 0 ? static_cast<RegType>(0.5) : -static_cast<RegType>(0.5));

      Mat A = arg.clover(x_cb, parity, chirality);

      HalfSpinor in_chi = in.chiral_project(chirality);
      HalfSpinor out_chi = A*in_chi + j*arg.a*in_chi;

      if (inverse) {
	if (arg.dynamic_clover) {
	  Mat A2 = A.square();
	  A2 += arg.a*arg.a*static_cast<RegType>(0.25);
	  Cholesky<HMatrix,RegType,N> cholesky(A2);
	  out_chi = static_cast<RegType>(0.25)*cholesky.backward(cholesky.forward(out_chi));
	} else {
	  Mat Ainv = arg.cloverInv(x_cb, parity, chirality);
	  out_chi = static_cast<RegType>(2.0)*(Ainv*out_chi);
	}
      }

      out += (out_chi).chiral_reconstruct(chirality);
    }

    out.toNonRel(); // change basis back

    arg.out(x_cb, spinor_parity) = out;
  }

  template <bool inverse, typename Float, int nSpin, int nColor, typename Arg>
  void twistCloverCPU(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;
      for (int x_cb=0; x_cb<arg.volumeCB; x_cb++) twistCloverApply<inverse,Float,nSpin,nColor>(arg, x_cb, parity);
    }
  }

  template <bool inverse, typename Float, int nSpin, int nColor, typename Arg>
  __global__ void twistCloverGPU(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = (arg.nParity == 2) ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
    if (x_cb >= arg.volumeCB) return;
    twistCloverApply<inverse,Float,nSpin,nColor>(arg, x_cb, parity);
  }

  template <typename Float, int nSpin, int nColor, typename Arg>
  class TwistClover : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

  protected:
    long long flops() const { return (arg.inverse ? 1056ll : 552ll) * arg.nParity*arg.volumeCB; }
    long long bytes() const {
      long long rtn = arg.out.Bytes() + arg.in.Bytes() + arg.nParity*arg.volumeCB*arg.clover.Bytes();
      if (arg.twist == QUDA_TWIST_GAMMA5_INVERSE && !arg.dynamic_clover)
	rtn += arg.nParity*arg.volumeCB*arg.cloverInv.Bytes();
      return rtn;
    }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    TwistClover(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, arg.inverse ? ",inverse" : ",direct");
    }
    virtual ~TwistClover() { }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (arg.inverse) twistCloverCPU<true,Float,nSpin,nColor>(arg);
	else twistCloverCPU<false,Float,nSpin,nColor>(arg);
      } else {
	if (arg.inverse) twistCloverGPU<true,Float,nSpin,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	else twistCloverGPU<false,Float,nSpin,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    void preTune() { if (arg.out.field == arg.in.field) arg.out.save(); }  // Need to save the out field if it aliases the in field
    void postTune() { if (arg.out.field == arg.in.field) arg.out.load(); } // Restore if the in and out fields alias
  };


  template <typename Float, int nColor>
  void ApplyTwistClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover,
			double kappa, double mu, double epsilon, int parity, int dagger, QudaTwistGamma5Type twist)
  {
    if (in.Nspin() != 4) errorQuda("Unsupported nSpin=%d", in.Nspin());
    constexpr int nSpin = 4;
    bool inverse = twist == QUDA_TWIST_GAMMA5_DIRECT ? false : true;

    CloverArg<Float,nSpin,nColor> arg(out, in, clover, inverse, parity, kappa, mu, epsilon, dagger, twist);
    TwistClover<Float,nSpin,nColor,decltype(arg)> worker(arg, in);
    worker.apply(streams[Nstream-1]);

    checkCudaError();
  }

  // template on the number of colors
  template <typename Float>
  void ApplyTwistClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover,
			double kappa, double mu, double epsilon, int parity, int dagger, QudaTwistGamma5Type twist)
  {
    if (in.Ncolor() == 3) {
      ApplyTwistClover<Float,3>(out, in, clover, kappa, mu, epsilon, parity, dagger, twist);
    } else {
      errorQuda("Unsupported number of colors %d\n", in.Ncolor());
    }
  }

  //Apply the twisted-clover matrix field to a colorspinor field
  void ApplyTwistClover(ColorSpinorField &out, const ColorSpinorField &in, const CloverField &clover,
			double kappa, double mu, double epsilon, int parity, int dagger, QudaTwistGamma5Type twist)
  {
    checkPrecision(out, clover, in);    // check all precisions match
    checkLocation(out, clover, in);     // check all locations match

#ifdef GPU_CLOVER_DIRAC
    if (in.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyTwistClover<double>(out, in, clover, kappa, mu, epsilon, parity, dagger, twist);
    } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyTwistClover<float>(out, in, clover, kappa, mu, epsilon, parity, dagger, twist);
    } else if (in.Precision() == QUDA_HALF_PRECISION) {
      ApplyTwistClover<short>(out, in, clover, kappa, mu, epsilon, parity, dagger, twist);
    } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyTwistClover<char>(out, in, clover, kappa, mu, epsilon, parity, dagger, twist);
    } else {
      errorQuda("Unsupported precision %d\n", in.Precision());
    }
#else
    errorQuda("Clover dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

} // namespace quda
