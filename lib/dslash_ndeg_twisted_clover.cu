#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <clover_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_ndeg_twisted_clover.cuh>

/**
   This is the gauged non-degenerate twisted-clover operator acting on a 
   quark doublet.
*/

namespace quda
{

  template <typename Arg> class NdegTwistedClover : public Dslash<nDegTwistedClover, Arg>
    {
      using Dslash = Dslash<nDegTwistedClover, Arg>;
      using Dslash::arg;
      using Dslash::in;

      unsigned int sharedBytesPerThread() const
      {
        return 2 * in.Ncolor() * 4 * sizeof(typename mapper<typename Arg::Float>::type);
      }

    public:
    NdegTwistedClover(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
        {
          TunableKernel3D::resizeVector(2, arg.nParity);
          TunableKernel3D::resizeStep(2, 1);
        }
      
      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        Dslash::setParam(tp);
        if (arg.xpay)
          Dslash::template instantiate<packShmem, true>(tp, stream);
        else
          errorQuda("Non-degenerate twisted-clover operator only defined for xpay=true");
      }
      
      long long flops() const
      {
        int clover_flops = 504;
        long long flops = Dslash::flops();
        switch (arg.kernel_type) {
        case INTERIOR_KERNEL:
        case KERNEL_POLICY:
          // b and c multiply (= 2 * 48 * in.Volume())
          flops += 2 * in.Ncolor() * 4 * 4 * in.Volume(); // complex * Nc * Ns * fma * vol
          flops += clover_flops * in.Volume();
          break;
        default: break; // twisted-mass flops are in the interior kernel
        }
        return flops;
      }
      long long bytes() const
      {
        int clover_bytes = 72 * in.Precision() + (isFixed<typename Arg::Float>::value ? 2 * sizeof(float) : 0);
        
        long long bytes = Dslash::bytes();
        switch (arg.kernel_type) {
        case INTERIOR_KERNEL:
        case KERNEL_POLICY: bytes += clover_bytes * in.Volume(); break;
        default: break;
        }
        
        return bytes;
      }
    };
  
  template <typename Float, int nColor, QudaReconstructType recon> struct NdegTwistedCloverApply {
    
    inline NdegTwistedCloverApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                                  const CloverField &A, double a,
                                  double b, double c, const ColorSpinorField &x, int parity, bool dagger,
                                  const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      NdegTwistedCloverArg<Float, nColor, nDim, recon> arg(out, in, U, A, a, b, c, x, parity, dagger, comm_override);
      NdegTwistedClover<decltype(arg)> twisted(arg, out, in);
      // in.VolumeCB() and in.GhostFaceCB() are inappropriate for a two-flavour operator
      // (since we abuse the fifth dimension for the flavour dof)
      dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, in.getDslashConstant().volume_4d_cb, in.getDslashConstant().ghostFaceCB, profile);
    }
  };

#ifdef GPU_NDEG_TWISTED_CLOVER_DIRAC
  void ApplyNdegTwistedClover(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const CloverField &A,
                              double a, double b,
                              double c, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
                              TimeProfile &profile)
  {
    instantiate<NdegTwistedCloverApply>(out, in, U, A, a, b, c, x, parity, dagger, comm_override, profile);
  }
#else
  void ApplyNdegTwistedClover(ColorSpinorField &, const ColorSpinorField &, const GaugeField &, const CloverField &,
                              double, double, double, const ColorSpinorField &, int, bool, const int *,
                              TimeProfile &)
  {
    errorQuda("Non-degenerate twisted-clover dslash has not been built");
  }
#endif // GPU_NDEG_TWISTED_CLOVER_DIRAC
  
} // namespace quda
