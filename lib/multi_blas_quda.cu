#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstring> // needed for memset
#include <typeinfo>

#include <tune_quda.h>
#include <blas_quda.h>
#include <color_spinor_field.h>

#include <jitify_helper.cuh>
#include <kernels/multi_blas_core.cuh>

namespace quda {

  namespace blas {

    hipStream_t* getStream();

    template <int NXZ, typename FloatN, int M, typename SpinorX, typename SpinorY, typename SpinorZ, typename SpinorW,
        typename Functor, typename T>
    class MultiBlas : public TunableVectorY
    {

  private:
    typedef typename scalar<FloatN>::type Float;
    typedef typename vector<Float, 2>::type Float2;
    static constexpr int NYW_max = max_YW_size<NXZ, SpinorX, SpinorY, SpinorZ, SpinorW, Functor>();
    const int NYW;
    int max_warp_split;
    mutable int warp_split; // helper used to keep track of current warp splitting
    const int nParity;
    mutable MultiBlasArg<NXZ, SpinorX, SpinorY, SpinorZ, SpinorW, Functor> arg;
    const coeff_array<T> &a, &b, &c;
    std::vector<ColorSpinorField *> &x, &y, &z, &w;

    // host pointers used for backing up fields when tuning
    // don't curry into the Spinors to minimize parameter size
    char *Y_h[NYW_max], *W_h[NYW_max], *Ynorm_h[NYW_max], *Wnorm_h[NYW_max];

    bool tuneSharedBytes() const { return false; }

    // for these streaming kernels, there is no need to tune the grid size, just use max
    unsigned int minGridSize() const { return maxGridSize(); }

  public:
    MultiBlas(SpinorX X[], SpinorY Y[], SpinorZ Z[], SpinorW W[], Functor &f, const coeff_array<T> &a,
              const coeff_array<T> &b, const coeff_array<T> &c, std::vector<ColorSpinorField *> &x,
              std::vector<ColorSpinorField *> &y, std::vector<ColorSpinorField *> &z,
              std::vector<ColorSpinorField *> &w, int NYW, int length) :
      TunableVectorY(NYW),
      NYW(NYW),
      warp_split(1),
      nParity(x[0]->SiteSubset()),
      arg(X, Y, Z, W, f, NYW, length / nParity),
      a(a),
      b(b),
      c(c),
      x(x),
      y(y),
      z(z),
      w(w),
      Y_h(),
      W_h(),
      Ynorm_h(),
      Wnorm_h()
    {
      // heuristic for enabling if we need the warp-splitting optimization
      const int gpu_size = 2 * deviceProp.maxThreadsPerBlock * deviceProp.multiProcessorCount;
      switch (gpu_size / (x[0]->Length() * NYW)) {
      case 0: max_warp_split = 1; break; // we have plenty of work, no need to split
      case 1: max_warp_split = 2; break; // double the thread count
      case 2:                            // quadruple the thread count
      default: max_warp_split = 4;
      }
      max_warp_split = std::min(NXZ, max_warp_split); // ensure we only split if valid

      Amatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(a.data));
      Bmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(b.data));
      Cmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(c.data));

      strcpy(aux, x[0]->AuxString());
      if (x[0]->Precision() != y[0]->Precision()) {
        strcat(aux, ",");
        strcat(aux, y[0]->AuxString());
      }

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/multi_blas_core.cuh");
#endif
      }

      virtual ~MultiBlas() {}

      inline TuneKey tuneKey() const
      {
        char name[TuneKey::name_n];
        strcpy(name, num_to_string<NXZ>::value);
        strcat(name, std::to_string(NYW).c_str());
        strcat(name, typeid(arg.f).name());
        return TuneKey(x[0]->VolString(), name, aux);
      }

      inline void apply(const hipStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        typedef typename scalar<FloatN>::type Float;
        typedef typename vector<Float, 2>::type Float2;

#ifdef JITIFY
        using namespace jitify::reflection;
        auto instance = program->kernel("quda::blas::multiBlasKernel")
                          .instantiate(Type<FloatN>(), M, NXZ, tp.aux.x, Type<decltype(arg)>());

        if (a.data) {
          Float2 A[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) A[NYW * i + j] = make_Float2<Float2>(Complex(a.data[NYW * i + j]));

          auto Amatrix_d = instance.get_constant_ptr("quda::blas::Amatrix_d");
          hipMemcpyHtoDAsync(Amatrix_d, A, NXZ * NYW * sizeof(decltype(A[0])), stream);
        }

        if (b.data) {
          Float2 B[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) B[NYW * i + j] = make_Float2<Float2>(Complex(b.data[NYW * i + j]));

          auto Bmatrix_d = instance.get_constant_ptr("quda::blas::Bmatrix_d");
          hipMemcpyHtoDAsync(Bmatrix_d, B, NXZ * NYW * sizeof(decltype(B[0])), stream);
        }

        if (c.data) {
          Float2 C[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) C[NYW * i + j] = make_Float2<Float2>(Complex(c.data[NYW * i + j]));
          auto Cmatrix_d = instance.get_constant_ptr("quda::blas::Cmatrix_d");
          hipMemcpyHtoDAsync(Cmatrix_d, C, NXZ * NYW * sizeof(decltype(C[0])), stream);
        }

        tp.block.x *= tp.aux.x; // include warp-split factor
        jitify_error = instance.configure(tp.grid, tp.block, tp.shared_bytes, stream).launch(arg);
        tp.block.x /= tp.aux.x; // restore block size
#else
        if (a.data) {
          Float2 A[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) A[NYW * i + j] = make_Float2<Float2>(Complex(a.data[NYW * i + j]));
          hipMemcpyToSymbolAsync(HIP_SYMBOL(Amatrix_d), A, NXZ * NYW * sizeof(decltype(A[0])), 0, hipMemcpyHostToDevice, stream);
        }

        if (b.data) {
          Float2 B[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) B[NYW * i + j] = make_Float2<Float2>(Complex(b.data[NYW * i + j]));
          hipMemcpyToSymbolAsync(HIP_SYMBOL(Bmatrix_d), B, NXZ * NYW * sizeof(decltype(B[0])), 0, hipMemcpyHostToDevice, stream);
        }

        if (c.data) {
          Float2 C[MAX_MATRIX_SIZE / sizeof(Float2)];
          for (int i = 0; i < NXZ; i++)
            for (int j = 0; j < NYW; j++) C[NYW * i + j] = make_Float2<Float2>(Complex(c.data[NYW * i + j]));
          hipMemcpyToSymbolAsync(HIP_SYMBOL(Cmatrix_d), C, NXZ * NYW * sizeof(decltype(C[0])), 0, hipMemcpyHostToDevice, stream);
        }

        tp.block.x *= tp.aux.x; // include warp-split factor

        switch (tp.aux.x) {
        case 1: multiBlasKernel<FloatN, M, NXZ, 1><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
#ifdef WARP_SPLIT
        case 2: multiBlasKernel<FloatN, M, NXZ, 2><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
        case 4: multiBlasKernel<FloatN, M, NXZ, 4><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
#endif
        default: errorQuda("warp-split factor %d not instantiated", tp.aux.x);
        }

        tp.block.x /= tp.aux.x; // restore block size
#endif
      }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (SpinorY::write) arg.Y[i].backup(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          if (SpinorW::write) arg.W[i].backup(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (SpinorY::write) arg.Y[i].restore(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          if (SpinorW::write) arg.W[i].restore(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      bool advanceAux(TuneParam &param) const
      {
#ifdef WARP_SPLIT
        if (2 * param.aux.x <= max_warp_split) {
          param.aux.x *= 2;
          warp_split = param.aux.x;
          return true;
        } else {
          param.aux.x = 1;
          warp_split = param.aux.x;
          // reset the block dimension manually here to pick up the warp_split parameter
          resetBlockDim(param);
          return false;
        }
#else
        warp_split = 1;
        return false;
#endif
      }

      int blockStep() const { return deviceProp.warpSize / warp_split; }
      int blockMin() const { return deviceProp.warpSize / warp_split; }

      void initTuneParam(TuneParam &param) const
      {
        TunableVectorY::initTuneParam(param);
        param.grid.z = nParity;
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      void defaultTuneParam(TuneParam &param) const
      {
        TunableVectorY::defaultTuneParam(param);
        param.grid.z = nParity;
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      long long flops() const { return arg.f.flops() * vec_length<FloatN>::value * (long)arg.length * nParity * M; }

      long long bytes() const
      {
        // the factor two here assumes we are reading and writing to the high precision vector
        return ((arg.f.streams() - 2) * x[0]->Bytes() + 2 * y[0]->Bytes());
      }

      int tuningIter() const { return 3; }
    };

    template <int NXZ_, typename RegType, typename StoreType, typename yType, int M,
              template <int, typename, typename> class Functor, typename write, typename T>
    void multiBlas(const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                   std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                   std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w, int length)
    {
      typedef typename scalar<RegType>::type Float;
      typedef typename vector<Float, 2>::type Float2;
      typedef vector<Float, 2> vec2;

      const int NYW = y.size();
      // the below line enable NXZ = 128 for floating point types, which is invalid for fixed-point types
      constexpr int NXZ = isFixed<StoreType>::value && NXZ_ == 128 ? 64 : NXZ_;
      Functor<NXZ, Float2, RegType> f(NYW);
      constexpr int NYW_max = max_YW_size<NXZ, StoreType, yType, write, decltype(f)>();
      const int NYW_max_check
        = max_YW_size<write>(x.size(), x[0]->Precision(), y[0]->Precision(), f.use_z, f.use_w, false);

      if (!is_valid_NXZ(NXZ, false, x[0]->Precision() < QUDA_SINGLE_PRECISION))
        errorQuda("NXZ=%d is not a valid size ( MAX_MULTI_BLAS_N %d)", NXZ, MAX_MULTI_BLAS_N);
      if (NYW_max != NYW_max_check) errorQuda("Runtime %d and compile time %d limits disagree", NYW_max, NYW_max_check);
      if (NYW > NYW_max) errorQuda("NYW exceeds max size (%d > %d)", NYW, NYW_max);
      if (NXZ * NYW * sizeof(Float2) > MAX_MATRIX_SIZE)
        errorQuda("Coefficient matrix exceeds max size (%lu > %d)", NXZ * NYW * sizeof(Float2), MAX_MATRIX_SIZE);

      SpinorTexture<RegType, StoreType, M> X[NXZ];
      Spinor<RegType, yType, M, write::Y> Y[NYW_max];
      SpinorTexture<RegType, StoreType, M> Z[NXZ];
      Spinor<RegType, StoreType, M, write::W> W[NYW_max];

      for (int i = 0; i < NXZ; i++) {
        X[i].set(*dynamic_cast<cudaColorSpinorField *>(x[i]));
        Z[i].set(*dynamic_cast<cudaColorSpinorField *>(z[i]));
      }
      for (int i = 0; i < NYW; i++) {
        Y[i].set(*dynamic_cast<cudaColorSpinorField *>(y[i]));
        W[i].set(*dynamic_cast<cudaColorSpinorField *>(w[i]));
      }

      MultiBlas<NXZ, RegType, M, typename std::remove_reference<decltype(X[0])>::type,
                typename std::remove_reference<decltype(Y[0])>::type, typename std::remove_reference<decltype(Z[0])>::type,
                typename std::remove_reference<decltype(W[0])>::type, decltype(f), T>
        blas(X, Y, Z, W, f, a, b, c, x, y, z, w, NYW, length);
      blas.apply(*getStream());

      blas::bytes += blas.bytes();
      blas::flops += blas.flops();

      checkCudaError();
    }

    /**
       Driver for generic blas routine with four loads and two store.
    */
    template <int NXZ, template <int MXZ, typename Float, typename FloatN> class Functor, typename write, typename T>
    void uniMultiBlas(const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                      CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                      CompositeColorSpinorField &w)
    {

      if (checkLocation(*x[0], *y[0], *z[0], *w[0]) == QUDA_CUDA_FIELD_LOCATION) {

        if (y[0]->Precision() == QUDA_DOUBLE_PRECISION && x[0]->Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
#if defined(NSPIN4) || defined(NSPIN2) || defined(NSPIN1)
          const int M = 1;
          multiBlas<NXZ, double2, double2, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Length() / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

        } else if (y[0]->Precision() == QUDA_SINGLE_PRECISION && x[0]->Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
          if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
            const int M = 1;
            multiBlas<NXZ, float4, float4, float4, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Length() / (4 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif

          } else if (x[0]->Nspin() == 2 || x[0]->Nspin() == 1) {

#if defined(NSPIN2) || defined(NSPIN1)
            const int M = 1;
            multiBlas<NXZ, float2, float2, float2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Length() / (2 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

        } else if (y[0]->Precision() == QUDA_HALF_PRECISION && x[0]->Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
          if (x[0]->Ncolor() != 3) { errorQuda("nColor = %d is not supported", x[0]->Ncolor()); }
          if (x[0]->Nspin() == 4) { // wilson
#if defined(NSPIN4)
            const int M = 6;
            multiBlas<NXZ, float4, short4, short4, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
          } else if (x[0]->Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = 3;
            multiBlas<NXZ, float2, short2, short2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

        } else if (y[0]->Precision() == QUDA_QUARTER_PRECISION && x[0]->Precision() == QUDA_QUARTER_PRECISION) {

#if QUDA_PRECISION & 1
          if (x[0]->Ncolor() != 3) { errorQuda("nColor = %d is not supported", x[0]->Ncolor()); }
          if (x[0]->Nspin() == 4) { // wilson
#if defined(NSPIN4)
            const int M = 6;
            multiBlas<NXZ, float4, char4, char4, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
          } else if (x[0]->Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = 3;
            multiBlas<NXZ, float2, char2, char2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

        } else {

          errorQuda("Precision combination x=%d not supported\n", x[0]->Precision());
        }
      } else { // fields on the cpu
        errorQuda("Not implemented");
      }
    }

    /**
       Driver for generic blas routine with four loads and two store.
    */
    template <int NXZ, template <int MXZ, typename Float, typename FloatN> class Functor, typename write, typename T>
    void mixedMultiBlas(const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w)
    {
      if (checkLocation(*x[0], *y[0], *z[0], *w[0]) == QUDA_CUDA_FIELD_LOCATION) {

        if (y[0]->Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
          if (x[0]->Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
            if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
              const int M = 12;
              multiBlas<NXZ, double2, float4, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            } else if (x[0]->Nspin() == 1) {

#if defined(NSPIN1)
              const int M = 3;
              multiBlas<NXZ, double2, float2, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            }

#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

          } else if (x[0]->Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
            if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
              const int M = 12;
              multiBlas<NXZ, double2, short4, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif

            } else if (x[0]->Nspin() == 1) {

#if defined(NSPIN1)
              const int M = 3;
              multiBlas<NXZ, double2, short2, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

          } else if (x[0]->Precision() == QUDA_QUARTER_PRECISION) {

#if QUDA_PRECISION & 1
            if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
              const int M = 12;
              multiBlas<NXZ, double2, char4, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif

            } else if (x[0]->Nspin() == 1) {

#if defined(NSPIN1)
              const int M = 3;
              multiBlas<NXZ, double2, char2, double2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x[0]->Precision());
#endif

          } else {
            errorQuda("Not implemented for this precision combination %d %d", x[0]->Precision(), y[0]->Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, y[0]->Precision());
#endif

        } else if (y[0]->Precision() == QUDA_SINGLE_PRECISION) {

#if (QUDA_PRECISION & 4)
          if (x[0]->Precision() == QUDA_HALF_PRECISION) {

#if (QUDA_PRECISION & 2)
            if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
              const int M = 6;
              multiBlas<NXZ, float4, short4, float4, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif

            } else if (x[0]->Nspin() == 1) {

#if defined(NSPIN1)
              const int M = 3;
              multiBlas<NXZ, float2, short2, float2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            } else {
              errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
            }

#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, y[0]->Precision());
#endif

          } else if (x[0]->Precision() == QUDA_QUARTER_PRECISION) {

#if (QUDA_PRECISION & 1)
            if (x[0]->Nspin() == 4) {
#if defined(NSPIN4)
              const int M = 6;
              multiBlas<NXZ, float4, char4, float4, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif

            } else if (x[0]->Nspin() == 1) {

#if defined(NSPIN1)
              const int M = 3;
              multiBlas<NXZ, float2, char2, float2, M, Functor, write>(a, b, c, x, y, z, w, x[0]->Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x[0]->Nspin(), x[0]->FieldOrder());
#endif
            } else {
              errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
            }

#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, y[0]->Precision());
#endif

          } else {
            errorQuda("Precision combination x=%d y=%d not supported\n", x[0]->Precision(), y[0]->Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, y[0]->Precision());
#endif
        } else {
          errorQuda("Precision combination x=%d y=%d not supported\n", x[0]->Precision(), y[0]->Precision());
        }
      } else { // fields on the cpu
        errorQuda("Not implemented");
      }
    }

    template <int NXZ, template <int MXZ, typename Float, typename FloatN> class Functor, typename write, typename T>
    void multiBlas(const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                   CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                   CompositeColorSpinorField &w)
    {
      if (x[0]->Precision() != y[0]->Precision()) {
        mixedMultiBlas<NXZ, Functor, write>(a, b, c, x, y, x, w);
      } else {
        uniMultiBlas<NXZ, Functor, write>(a, b, c, x, y, x, w);
      }
    }

    template <template <int MXZ, typename Float, typename FloatN> class Functor, typename write, typename T>
    void multiBlas(const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                   CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                   CompositeColorSpinorField &w)
    {
      // instantiate the loop unrolling template
      switch (x.size()) {
      // by default all powers of two <= 64 are instantiated
      case 1: multiBlas<1, Functor, write>(a, b, c, x, y, x, w); break;
      case 2: multiBlas<2, Functor, write>(a, b, c, x, y, x, w); break;
      case 4: multiBlas<4, Functor, write>(a, b, c, x, y, x, w); break;
      case 8: multiBlas<8, Functor, write>(a, b, c, x, y, x, w); break;
      case 16: multiBlas<16, Functor, write>(a, b, c, x, y, x, w); break;
      case 32: multiBlas<32, Functor, write>(a, b, c, x, y, x, w); break;
      case 64: multiBlas<64, Functor, write>(a, b, c, x, y, x, w); break;
      case 128: multiBlas<128, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 3
      case 3: multiBlas<3, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 5
      case 5: multiBlas<5, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 6
      case 6: multiBlas<6, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 7
      case 7: multiBlas<7, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 9
      case 9: multiBlas<9, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 10
      case 10: multiBlas<10, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 11
      case 11: multiBlas<11, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 12
      case 12: multiBlas<12, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 13
      case 13: multiBlas<13, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 14
      case 14: multiBlas<14, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 15
      case 15: multiBlas<15, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 17
      case 17: multiBlas<17, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 18
      case 18: multiBlas<18, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 19
      case 19: multiBlas<19, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 20
      case 20: multiBlas<20, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 21
      case 21: multiBlas<21, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 22
      case 22: multiBlas<22, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 23
      case 23: multiBlas<23, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 24
      case 24: multiBlas<24, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 25
      case 25: multiBlas<25, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 26
      case 26: multiBlas<26, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 27
      case 27: multiBlas<27, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 28
      case 28: multiBlas<28, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 29
      case 29: multiBlas<29, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 30
      case 30: multiBlas<30, Functor, write>(a, b, c, x, y, x, w); break;
#if MAX_MULTI_BLAS_N >= 31
      case 31: multiBlas<31, Functor, write>(a, b, c, x, y, x, w); break;
#endif // 31
#endif // 30
#endif // 29
#endif // 28
#endif // 27
#endif // 26
#endif // 25
#endif // 24
#endif // 23
#endif // 22
#endif // 21
#endif // 20
#endif // 19
#endif // 18
#endif // 17
#endif // 15
#endif // 14
#endif // 13
#endif // 12
#endif // 11
#endif // 10
#endif // 9
#endif // 7
#endif // 6
#endif // 5
#endif // 3
      default: errorQuda("x.size %lu greater than MAX_MULTI_BLAS_N %d", x.size(), MAX_MULTI_BLAS_N);
      }
    }

    using range = std::pair<size_t,size_t>;

    template <template <int MXZ, typename Float, typename FloatN> class Functor, typename T>
    void axpy_recurse(const T *a_, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                      const range &range_x, const range &range_y, int upper)
    {
      using write_ = write<0, 1, 0, 0>;
      // if greater than max single-kernel size, recurse
      if (y.size() > (size_t)max_YW_size<write_>(x.size(), x[0]->Precision(), y[0]->Precision(), false, false, false)) {
        // We need to split up 'a' carefully since it's row-major.
        T *tmpmajor = new T[x.size() * y.size()];
        T *tmpmajor0 = &tmpmajor[0];
        T *tmpmajor1 = &tmpmajor[x.size() * (y.size() / 2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i = 0; i < xlen; i++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        axpy_recurse<Functor>(tmpmajor0, x, y0, range_x, range(range_y.first, range_y.first + y0.size()), upper);
        axpy_recurse<Functor>(tmpmajor1, x, y1, range_x, range(range_y.first + y0.size(), range_y.second), upper);

        delete[] tmpmajor;
      } else {
        // if at the bottom of recursion,
        if (is_valid_NXZ(x.size(), false, x[0]->Precision() < QUDA_SINGLE_PRECISION)) {
          // since tile range is [first,second), e.g., [first,second-1], we need >= here
          // if upper triangular and upper-right tile corner is below diagonal return
          if (upper == 1 && range_y.first >= range_x.second) { return; }
          // if lower triangular and lower-left tile corner is above diagonal return
          if (upper == -1 && range_x.first >= range_y.second) { return; }

          // mark true since we will copy the "a" matrix into constant memory
          coeff_array<T> a(a_), b, c;
          multiBlas<Functor, write_>(a, b, c, x, y, x, y);
        } else {
          // split the problem in half and recurse
          const T *a0 = &a_[0];
          const T *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          axpy_recurse<Functor>(a0, x0, y, range(range_x.first, range_x.first + x0.size()), range_y, upper);
          axpy_recurse<Functor>(a1, x1, y, range(range_x.first + x0.size(), range_x.second), range_y, upper);
        }
      } // end if (y.size() > max_YW_size())
    }

    void caxpy(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 0);
    }

    void caxpy_U(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_U with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 1);
    }

    void caxpy_L(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_L with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), -1);
    }

    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy(a, x.Components(), y.Components()); }

    void caxpy_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_U(a, x.Components(), y.Components()); }

    void caxpy_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_L(a, x.Components(), y.Components()); }

    void caxpyz_recurse(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y,
                        std::vector<ColorSpinorField*> &z, const range &range_x, const range &range_y,
                        int pass, int upper)
    {
      // if greater than max single-kernel size, recurse
      using write_ = write<0, 0, 0, 1>;

      if (y.size() > (size_t)max_YW_size<write_>(x.size(), x[0]->Precision(), y[0]->Precision(), false, true, false)) {
        // We need to split up 'a' carefully since it's row-major.
        Complex* tmpmajor = new Complex[x.size()*y.size()];
        Complex* tmpmajor0 = &tmpmajor[0];
        Complex* tmpmajor1 = &tmpmajor[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
        std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i_ = 0; i_ < xlen; i_++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        caxpyz_recurse(tmpmajor0, x, y0, z0, range_x, range(range_y.first, range_y.first + y0.size()), pass, upper);
        caxpyz_recurse(tmpmajor1, x, y1, z1, range_x, range(range_y.first + y0.size(), range_y.second), pass, upper);

        delete[] tmpmajor;
      } else {
        // if at bottom of recursion check where we are
        if (is_valid_NXZ(x.size(), false, x[0]->Precision() < QUDA_SINGLE_PRECISION)) {
          // check if tile straddles diagonal
          bool is_diagonal = (range_x.first < range_y.second) && (range_y.first < range_x.second);
          if (pass==1) {
            if (!is_diagonal) {
              // if upper triangular and upper-right tile corner is below diagonal return
              if (upper == 1 && range_y.first >= range_x.second) { return; }
              // if lower triangular and lower-left tile corner is above diagonal return
              if (upper == -1 && range_x.first >= range_y.second) { return; }
              caxpy(a_, x, z); return;  // off diagonal
            }
            return;
      	  } else {
            if (!is_diagonal) return; // We're on the first pass, so we only want to update the diagonal.
          }

          coeff_array<Complex> a(a_), b, c;
          multiBlas<multicaxpyz_, write_>(a, b, c, x, y, x, z);
        } else {
          // split the problem in half and recurse
          const Complex *a0 = &a_[0];
          const Complex *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          caxpyz_recurse(a0, x0, y, z, range(range_x.first, range_x.first + x0.size()), range_y, pass, upper);
          caxpyz_recurse(a1, x1, y, z, range(range_x.first + x0.size(), range_x.second), range_y, pass, upper);
        }
      } // end if (y.size() > max_YW_size())
    }

    void caxpyz(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z) {
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 0);
    }

    void caxpyz_U(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z) {
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void caxpyz_L(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z) {
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1);
    }


    void caxpyz(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      caxpyz(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      caxpyz_U(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      caxpyz_L(a, x.Components(), y.Components(), z.Components());
    }

    void axpyBzpcx(const double *a_, std::vector<ColorSpinorField *> &x_, std::vector<ColorSpinorField *> &y_,
                   const double *b_, ColorSpinorField &z_, const double *c_)
    {
      using write_ = write<0, 1, 0, 1>;

      if (y_.size() <= (size_t)max_YW_size<write_>(1, z_.Precision(), y_[0]->Precision(), false, true, false)) {
        // swizzle order since we are writing to x_ and y_, but the
	// multi-blas only allow writing to y and w, and moreover the
	// block width of y and w must match, and x and z must match.
	std::vector<ColorSpinorField*> &y = y_;
	std::vector<ColorSpinorField*> &w = x_;

	// wrap a container around the third solo vector
	std::vector<ColorSpinorField*> x;
	x.push_back(&z_);

        coeff_array<double> a(a_), b(b_), c(c_);
        multiBlas<1, multi_axpyBzpcx_, write_>(a, b, c, x, y, x, w);
      } else {
        // split the problem in half and recurse
	const double *a0 = &a_[0];
	const double *b0 = &b_[0];
	const double *c0 = &c_[0];

	std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);
	std::vector<ColorSpinorField*> y0(y_.begin(), y_.begin() + y_.size()/2);

	axpyBzpcx(a0, x0, y0, b0, z_, c0);

	const double *a1 = &a_[y_.size()/2];
	const double *b1 = &b_[y_.size()/2];
	const double *c1 = &c_[y_.size()/2];

	std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());
	std::vector<ColorSpinorField*> y1(y_.begin() + y_.size()/2, y_.end());

	axpyBzpcx(a1, x1, y1, b1, z_, c1);
      }
    }

    void caxpyBxpz(const Complex *a_, std::vector<ColorSpinorField*> &x_, ColorSpinorField &y_,
		   const Complex *b_, ColorSpinorField &z_)
    {
      using write_ = write<0, 1, 0, 1>;

      if (is_valid_NXZ(x_.size(), false, x_[0]->Precision() < QUDA_SINGLE_PRECISION)) // only swizzle if we have to.
      {
        // swizzle order since we are writing to y_ and z_, but the
        // multi-blas only allow writing to y and w, and moreover the
        // block width of y and w must match, and x and z must match.
        // Also, wrap a container around them.
        std::vector<ColorSpinorField*> y;
        y.push_back(&y_);
        std::vector<ColorSpinorField*> w;
        w.push_back(&z_);

        // we're reading from x
        std::vector<ColorSpinorField*> &x = x_;

        coeff_array<Complex> a(a_), b(b_), c;
        multiBlas<multi_caxpyBxpz_, write_>(a, b, c, x, y, x, w);
      } else {
        // split the problem in half and recurse
        const Complex *a0 = &a_[0];
        const Complex *b0 = &b_[0];

        std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);

        caxpyBxpz(a0, x0, y_, b0, z_);

        const Complex *a1 = &a_[x_.size()/2];
        const Complex *b1 = &b_[x_.size()/2];

        std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());

        caxpyBxpz(a1, x1, y_, b1, z_);
      }
    }

    void axpy(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 0);
    }

    void axpy_U(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_U with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 1);
    }

    void axpy_L(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_L with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), -1);
    }

    // Composite field version
    void axpy(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy(a, x.Components(), y.Components()); }

    void axpy_U(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_U(a, x.Components(), y.Components()); }

    void axpy_L(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_L(a, x.Components(), y.Components()); }


  } // namespace blas

} // namespace quda
