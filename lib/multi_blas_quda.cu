#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <color_spinor_field.h>
#include <kernels/multi_blas_core.cuh>
#include <tunable_nd.h>

namespace quda {

  namespace blas {

    template <template <typename ...> class Functor, typename store_t, typename y_store_t, int nSpin, typename T>
    class MultiBlas : public TunableGridStrideKernel3D
    {
      using real = typename mapper<y_store_t>::type;
      const int NXZ;
      const int NYW;
      Functor<real> f;
      int max_warp_split;
      mutable int warp_split; // helper used to keep track of current warp splitting
      const int nParity;
      const T &a, &b, &c;
      cvector_ref<ColorSpinorField> &x, &y, &z, &w;

      bool tuneSharedBytes() const override { return false; }

      // for these streaming kernels, there is no need to tune the grid size, just use max
      unsigned int minGridSize() const override { return maxGridSize(); }

    public:
      template <typename Vx, typename Vy, typename Vz, typename Vw>
      MultiBlas(const T &a, const T &b, const T &c, const ColorSpinorField &x0, const ColorSpinorField &y0,
                Vx &x, Vy &y, Vz &z, Vw &w) :
        TunableGridStrideKernel3D(x0, y.size(), x0.SiteSubset()),
        NXZ(x.size()),
        NYW(y.size()),
        f(NXZ, NYW),
        warp_split(1),
        nParity(x0.SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(reinterpret_cast<cvector_ref<ColorSpinorField>&>(x)),
        y(reinterpret_cast<cvector_ref<ColorSpinorField>&>(y)),
        z(reinterpret_cast<cvector_ref<ColorSpinorField>&>(z)),
        w(reinterpret_cast<cvector_ref<ColorSpinorField>&>(w))
      {
        checkLocation(x[0], y[0], z[0], w[0]);
        checkLength(x[0], y[0], z[0], w[0]);
        auto x_prec = checkPrecision(x[0], z[0], w[0]);
        auto y_prec = y0.Precision();
        auto x_order = checkOrder(x[0], z[0], w[0]);
        auto y_order = y0.FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        // heuristic for enabling if we need the warp-splitting optimization
        const int gpu_size = 2 * device::max_threads_per_block() * device::processor_count();
        switch (gpu_size / (x0.Length() * NYW)) {
        case 0: max_warp_split = 1; break; // we have plenty of work, no need to split
        case 1: max_warp_split = 2; break; // double the thread count
        case 2:                            // quadruple the thread count
        default: max_warp_split = 4;
        }
        max_warp_split = std::min(NXZ, max_warp_split); // ensure we only split if valid

        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y0.AuxString().c_str());
        }
        char NXZ_str[16];
        char NYW_str[16];
        u32toa(NXZ_str, NXZ);
        u32toa(NYW_str, NYW);
        strcat(aux, ",Nxz=");
        strcat(aux, NXZ_str);
        strcat(aux, ",Nyw=");
        strcat(aux, NYW_str);

#ifdef QUDA_FAST_COMPILE_REDUCE
        strcat(aux, ",fast_compile");
#endif

        apply(device::get_default_stream());

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const override { return TuneKey(vol, typeid(f).name(), aux); }

      template <typename Arg> void Launch(const TuneParam &tp, const qudaStream_t &stream, Arg &&arg)
      {
        constexpr bool multi_1d = Arg::Functor::multi_1d;
        if (a.size()) { set_param<multi_1d>(arg, 'a', a); }
        if (b.size()) { set_param<multi_1d>(arg, 'b', b); }
        if (c.size()) { set_param<multi_1d>(arg, 'c', c); }
        launch<MultiBlas_>(tp, stream, arg);
      }

      template <int NXZ> void compute(const qudaStream_t &stream)
      {
        staticCheck<NXZ, store_t, y_store_t, decltype(f)>(f, x, y);

        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll_check && (x[0].Ncolor() != 3 || x[0].Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x[0].Nspin(), x[0].Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(x[0], y[0], z[0], w[0]); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Functor<device_real_t> f_(NXZ, NYW);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x[0].Length() / (nParity * M);

          if (tp.aux.x > 1 && (length * tp.aux.x) % device::warp_size() != 0) {
            // if problem size isn't divisible by the warp size then we can't use warp splitting
            launchError() = QUDA_ERROR;
          } else {
            tp.block.x *= tp.aux.x; // include warp-split factor
            switch (tp.aux.x) {
            case 1:
              Launch(tp, stream, MultiBlasArg<1, device_real_t, M, NXZ, device_store_t, N,
                     device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
              break;
            case 2:
              if constexpr (enable_warp_split()) {
                Launch(tp, stream, MultiBlasArg<2, device_real_t, M, NXZ, device_store_t, N,
                       device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
                break;
              }
            case 4:
              if constexpr (enable_warp_split()) {
                Launch(tp, stream, MultiBlasArg<4, device_real_t, M, NXZ, device_store_t, N,
                       device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
                break;
              }
            default: errorQuda("warp-split factor %d not instantiated", static_cast<int>(tp.aux.x));
            }

            tp.block.x /= tp.aux.x; // restore block size
          }
        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      template <int n> std::enable_if_t<n!=1, void> instantiateLinear(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiateLinear<n-1>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiateLinear(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      template <int n> std::enable_if_t<n!=1, void> instantiatePow2(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiatePow2<n/2>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiatePow2(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      // instantiate the loop unrolling template
      template <int NXZ_max> std::enable_if_t<NXZ_max!=1, void> instantiate(const qudaStream_t &stream)
      {
        // if multi-1d then constrain the templates to no larger than max-1d size
        constexpr auto max_nxz_pow2 = max_NXZ_power2(false, static_cast<QudaPrecision>(sizeof(y_store_t)));
        constexpr auto pow2_max = !decltype(f)::multi_1d ? max_nxz_pow2 : std::min(max_N_multi_1d_pow2(), max_nxz_pow2);
        constexpr auto linear_max = !decltype(f)::multi_1d ? MAX_MULTI_BLAS_N : std::min(max_N_multi_1d(), MAX_MULTI_BLAS_N);

        if (NXZ <= pow2_max && is_power2(NXZ)) instantiatePow2<pow2_max>(stream);
        else if (NXZ <= linear_max) instantiateLinear<linear_max>(stream);
        else errorQuda("x.size %lu greater than maximum supported size (pow2 = %d, linear = %d)", x.size(), pow2_max, linear_max);
      }

      template <int NXZ_max> std::enable_if_t<NXZ_max==1, void> instantiate(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      void apply(const qudaStream_t &stream) override { instantiate<decltype(f)::NXZ_max>(stream); }

      void preTune() override
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.Y) y[i].backup();
          if (f.write.W) w[i].backup();
        }
      }

      void postTune() override
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.Y) y[i].restore();
          if (f.write.W) w[i].restore();
        }
      }

      bool advanceAux(TuneParam &param) const override
      {
        if (enable_warp_split()) {
          if (2 * param.aux.x <= max_warp_split) {
            param.aux.x *= 2;
            warp_split = param.aux.x;
            return true;
          } else {
            param.aux.x = 1;
            warp_split = param.aux.x;
            // reset the block dimension manually here to pick up the warp_split parameter
            resetBlockDim(param);
            return false;
          }
        } else {
          warp_split = 1;
          return false;
        }
      }

      int blockStep() const override { return device::warp_size() / warp_split; }
      int blockMin() const override { return device::warp_size() / warp_split; }

      void initTuneParam(TuneParam &param) const override
      {
        TunableGridStrideKernel3D::initTuneParam(param);
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      void defaultTuneParam(TuneParam &param) const override
      {
        TunableGridStrideKernel3D::defaultTuneParam(param);
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      long long flops() const override
      {
        return NYW * NXZ * f.flops() * x[0].Length();
      }

      long long bytes() const override
      {
        // X and Z reads are repeated (and hopefully cached) across NYW
        // each Y and W read/write is done once
        return NYW * NXZ * (f.read.X + f.write.X) * x[0].Bytes() +
          NYW * (f.read.Y + f.write.Y) * y[0].Bytes() +
          NYW * NXZ * (f.read.Z + f.write.Z) * z[0].Bytes() +
          NYW * (f.read.W + f.write.W) * w[0].Bytes();
      }
    };

    using range = std::pair<size_t,size_t>;

    template <template <typename...> class Functor, typename T>
    void axpy_recurse(const std::vector<T> &a, cvector_ref<const ColorSpinorField> &x,
                      cvector_ref<ColorSpinorField> &y,
                      const range &range_x, const range &range_y, int upper)
    {
      if (a.size() != x.size() * y.size())
        errorQuda("coefficient size %lu does not match vector set %lu * %lu", a.size(), x.size(), y.size());

      // if greater than max single-kernel size, recurse
      size_t max_yw_size = y[0].Precision() == QUDA_DOUBLE_PRECISION ?
        max_YW_size<Functor<double>>(x.size(), x[0].Precision(), y[0].Precision()) :
        max_YW_size<Functor<float>>(x.size(), x[0].Precision(), y[0].Precision());

      if (y.size() > max_yw_size) {
        // We need to split up 'a' carefully since it's row-major.
        auto a_ = bisect_col(a, x.size(), y.size() / 2, y.size() - y.size() / 2);
        auto y_ = bisect(y);

        axpy_recurse<Functor>(a_.first, x, y_.first, range_x, range(range_y.first, range_y.first + y_.first.size()), upper);
        axpy_recurse<Functor>(a_.second, x, y_.second, range_x, range(range_y.first + y_.first.size(), range_y.second), upper);
      } else {
        // if at the bottom of recursion,
        if (is_valid_NXZ(x.size(), false, y[0].Precision())) {
          // since tile range is [first,second), e.g., [first,second-1], we need >= here
          // if upper triangular and upper-right tile corner is below diagonal return
          if (upper == 1 && range_y.first >= range_x.second) { return; }
          // if lower triangular and lower-left tile corner is above diagonal return
          if (upper == -1 && range_x.first >= range_y.second) { return; }

          // mark true since we will copy the "a" matrix into constant memory
          constexpr bool mixed = true;
          instantiate<Functor, MultiBlas, mixed>(a, std::vector<T>(), std::vector<T>(), x[0], y[0], x, y, x, x);
        } else {
          // split the problem in half and recurse
          auto x_ = bisect(x);
          auto a_ = bisect(a, y.size() * (x.size() / 2));

          axpy_recurse<Functor>(a_.first, x_.first, y, range(range_x.first, range_x.first + x_.first.size()), range_y, upper);
          axpy_recurse<Functor>(a_.second, x_.second, y, range(range_x.first + x_.first.size(), range_x.second), range_y, upper);
        }
      } // end if (y.size() > max_YW_size())
    }

    template <>
    void axpy<double>(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multiaxpy_>(a, x, y, range(0,x.size()), range(0,y.size()), 0);
    }

    template <>
    void axpy_U<double>(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_U with non-square 'a' (%lu != %lu) has not yet been implemented. Use block axpy instead",
                  x.size(), y.size());
      }
      axpy_recurse<multiaxpy_>(a, x, y, range(0, x.size()), range(0, y.size()), 1);
    }

    template <>
    void axpy_L<double>(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_L with non-square 'a' (%lu != %lu) has not yet been implemented. Use block axpy instead",
                  x.size(), y.size());
      }
      axpy_recurse<multiaxpy_>(a, x, y, range(0, x.size()), range(0, y.size()), -1);
    }

    template <>
    void axpy<Complex>(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multicaxpy_>(a, x, y, range(0,x.size()), range(0,y.size()), 0);
    }

    void caxpy(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      axpy(a, std::move(x), std::move(y));
    }

    template <>
    void axpy_U<Complex>(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpy_U with non-square 'a' (%lu != %lu) has not yet been implemented. Use block caxpy instead",
                  x.size(), y.size());
      }
      axpy_recurse<multicaxpy_>(a, x, y, range(0,x.size()), range(0,y.size()), 1);
    }

    void caxpy_U(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      axpy_U(a, std::move(x), std::move(y));
    }

    template <>
    void axpy_L<Complex>(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpy_L with non-square 'a' (%lu != %lu) has not yet been implemented. Use block caxpy instead",
                  x.size(), y.size());
      }
      axpy_recurse<multicaxpy_>(a, x, y, range(0,x.size()), range(0,y.size()), -1);
    }

    void caxpy_L(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<ColorSpinorField> &y)
    {
      axpy_L(a, std::move(x), std::move(y));
    }

    template <template <typename...> class Functor, typename T>
    void axpyz_recurse(const std::vector<T> &a, cvector_ref<const ColorSpinorField> &x,
                       cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z,
                       const range &range_x, const range &range_y, int pass, int upper)
    {
      if (a.size() != x.size() * y.size())
        errorQuda("coefficient size %lu does not match vector set %lu * %lu", a.size(), x.size(), y.size());

      // if greater than max single-kernel size, recurse
      size_t max_yw_size = y[0].Precision() == QUDA_DOUBLE_PRECISION ?
        max_YW_size<Functor<double>>(x.size(), x[0].Precision(), y[0].Precision()) :
        max_YW_size<Functor<float>>(x.size(), x[0].Precision(), y[0].Precision());

      if (y.size() > max_yw_size) {
        // We need to split up 'a' carefully since it's row-major.
        auto a_ = bisect_col(a, x.size(), y.size() / 2, y.size() - y.size() / 2);
        auto y_ = bisect(y);
        auto z_ = bisect(z);

        axpyz_recurse<Functor>(a_.first, x, y_.first, z_.first, range_x, range(range_y.first, range_y.first + y_.first.size()), pass, upper);
        axpyz_recurse<Functor>(a_.second, x, y_.second, z_.second, range_x, range(range_y.first + y_.first.size(), range_y.second), pass, upper);
      } else {
        // if at bottom of recursion check where we are
        if (is_valid_NXZ(x.size(), false, y[0].Precision())) {
          // check if tile straddles diagonal for L/U variants
          bool is_diagonal = (upper != 0) && (range_x.first < range_y.second) && (range_y.first < range_x.second);
          // check if tile is first to be updated for full matrices
          bool is_first = (upper == 0) && (range_x.first == 0);
          // whether to do axpyz
          bool do_axpyz = (upper != 0 && is_diagonal && pass == 0) || (upper == 0 && is_first);
          // whether to do axpy
          bool do_axpy = (upper != 0 && !is_diagonal && pass == 1) || (upper == 0 && !is_first);

          if (do_axpyz) {
            constexpr bool mixed = false;
            instantiate<Functor, MultiBlas, mixed>(a, std::vector<T>(), std::vector<T>(), x[0], y[0], x, y, x, z);
          } else if (do_axpy) {
            // if upper triangular and upper-right tile corner is below diagonal return
            if (upper == 1 && range_y.first >= range_x.second) { return; }
            // if lower triangular and lower-left tile corner is above diagonal return
            if (upper == -1 && range_x.first >= range_y.second) { return; }

            // off diagonal
            axpy(a, x, z);
          }
        } else {
          // split the problem in half and recurse
          auto x_ = bisect(x);
          auto a_ = bisect(a, y.size() * (x.size() / 2));

          axpyz_recurse<Functor>(a_.first, x_.first, y, z, range(range_x.first, range_x.first + x_.first.size()), range_y, pass, upper);
          axpyz_recurse<Functor>(a_.second, x_.second, y, z, range(range_x.first + x_.first.size(), range_x.second), range_y, pass, upper);
        }
      } // end if (y.size() > max_YW_size())
    }

    void axpyz(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0);
    }

    void axpyz_U(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block axpyz_U with non-square 'a' (%lu != %lu) has not yet been implemented. Use block axpyz instead",
                  x.size(), y.size());
      }
      // a is upper triangular.
      // first pass does the axpyz on the diagonal
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1);
      // second pass does axpy on the off diagonals
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void axpyz_L(const std::vector<double> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block axpyz_L with non-square 'a' (%lu != %lu) has not yet been implemented. Use block axpyz instead",
                  x.size(), y.size());
      }
      // a is upper triangular.
      // first pass does the axpyz on the diagonal
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1);
      // second pass does axpy on the off diagonals
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1);
    }

    void caxpyz(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0);
    }

    void caxpyz_U(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpyz_U with non-square 'a' (%lu != %lu) has not yet been implemented. Use block caxpyz instead",
                  x.size(), y.size());
      }
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1);
      // second pass does caxpy on the off diagonals
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void axpyz_L(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x, cvector_ref<const ColorSpinorField> &y, cvector_ref<ColorSpinorField> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block axpyz_U with non-square 'a' (%lu != %lu) has not yet been implemented. Use block axpyz instead",
                  x.size(), y.size());
      }
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1);
      // second pass does caxpy on the off diagonals
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1);
    }

    void axpyBzpcx(const std::vector<double> &a, cvector_ref<ColorSpinorField> &x_, cvector_ref<ColorSpinorField> &y_,
                   const std::vector<double> &b, ColorSpinorField &z_, const std::vector<double> &c)
    {
      if (y_.size() <= (size_t)max_N_multi_1d()) {
        // swizzle order since we are writing to x_ and y_, but the
	// multi-blas only allow writing to y and w, and moreover the
	// block width of y and w must match, and x and z must match.
	auto &y = y_;
	auto &w = x_;

	// wrap a container around the third solo vector
	cvector_ref<ColorSpinorField> x{z_};

        constexpr bool mixed = true;
        instantiate<multi_axpyBzpcx_, MultiBlas, mixed>(a, b, c, x[0], y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
        auto a_ = bisect(a);
        auto b_ = bisect(b);
        auto c_ = bisect(c);
        auto x = bisect(x_);
        auto y = bisect(y_);

	axpyBzpcx(a_.first, x.first, y.first, b_.first, z_, c_.first);
	axpyBzpcx(a_.second, x.second, y.second, b_.second, z_, c_.second);
      }
    }

    void caxpyBxpz(const std::vector<Complex> &a, cvector_ref<const ColorSpinorField> &x_, ColorSpinorField &y_,
                   const std::vector<Complex> &b, ColorSpinorField &z_)
    {
      if (x_.size() <= (size_t)max_N_multi_1d() && is_valid_NXZ(x_.size(), false, y_.Precision())) // only split if we have to.
      {
        // swizzle order since we are writing to y_ and z_, but the
        // multi-blas only allow writing to y and w, and moreover the
        // block width of y and w must match, and x and z must match.
        // Also, wrap a container around them.
        cvector_ref<ColorSpinorField> y{y_};
        cvector_ref<ColorSpinorField> w{z_};

        // we're reading from x
        auto &x = x_;

        constexpr bool mixed = true;
        instantiate<multi_caxpyBxpz_, MultiBlas, mixed>(a, b, std::vector<Complex>(), x[0], y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
        auto a_ = bisect(a);
        auto b_ = bisect(b);
        auto x = bisect(x_);

        caxpyBxpz(a_.first, x.first, y_, b_.first, z_);
        caxpyBxpz(a_.second, x.second, y_, b_.second, z_);
      }
    }

    // temporary wrappers
    void axpy(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<double> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(double));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      axpy(a_, x_, y_);
    }

    void axpy_U(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<double> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(double));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      axpy_U(a_, x_, y_);
    }

    void axpy_L(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<double> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(double));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      axpy_L(a_, x_, y_);
    }

    void caxpy(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      std::vector<Complex> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(Complex));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      caxpy(a_, x_, y_);
    }

    void caxpy_U(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      std::vector<Complex> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(Complex));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      caxpy_U(a_, x_, y_);
    }

    void caxpy_L(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      std::vector<Complex> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(Complex));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      caxpy_L(a_, x_, y_);
    }

    void axpyz(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y,
               std::vector<ColorSpinorField*> &z)
    {
      std::vector<double> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(double));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<const ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      vector_ref<ColorSpinorField> z_;
      for (auto &zi : z) z_.push_back(*zi);
      axpyz(a_, x_, y_, z_);
    }

    void caxpyz(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y,
               std::vector<ColorSpinorField*> &z)
    {
      std::vector<Complex> a_(x.size() * y.size());
      memcpy(a_.data(), a, x.size() * y.size() * sizeof(Complex));
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<const ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      vector_ref<ColorSpinorField> z_;
      for (auto &zi : z) z_.push_back(*zi);
      caxpyz(a_, x_, y_, z_);
    }

    void axpyBzpcx(const double *a, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                   const double *b, ColorSpinorField &z, const double *c)
    {
      std::vector<double> a_(x.size());
      memcpy(a_.data(), a, x.size() * sizeof(double));
      std::vector<double> b_(x.size());
      memcpy(b_.data(), b, x.size() * sizeof(double));
      std::vector<double> c_(x.size());
      memcpy(c_.data(), c, x.size() * sizeof(double));

      vector_ref<ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      axpyBzpcx(a_, x_, y_, b_, z, c_);
    }

    void caxpyBxpz(const Complex *a, std::vector<ColorSpinorField*> &x, ColorSpinorField &y,
		   const Complex *b, ColorSpinorField &z)
    {
      std::vector<Complex> a_(x.size());
      memcpy(a_.data(), a, x.size() * sizeof(Complex));
      std::vector<Complex> b_(x.size());
      memcpy(b_.data(), b, x.size() * sizeof(Complex));

      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      caxpyBxpz(a_, x_, y, b_, z);
    }

    // Composite field version
    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y){ caxpy(a, x.Components(), y.Components()); }
    void caxpy_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_U(a, x.Components(), y.Components()); }
    void caxpy_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_L(a, x.Components(), y.Components()); }

    void axpy(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy(a, x.Components(), y.Components()); }
    void axpy_U(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_U(a, x.Components(), y.Components()); }
    void axpy_L(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_L(a, x.Components(), y.Components()); }

    void axpyz(const double *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      axpyz(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz(a, x.Components(), y.Components(), z.Components());
    }

  } // namespace blas

} // namespace quda
