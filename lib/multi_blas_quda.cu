#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <color_spinor_field.h>
#include <kernels/multi_blas_core.cuh>
#include <tunable_nd.h>

namespace quda {

  namespace blas {

    template <template <typename ...> class Functor, typename store_t, typename y_store_t, int nSpin, typename T>
    class MultiBlas : public TunableGridStrideKernel3D
    {
      using real = typename mapper<y_store_t>::type;
      const int NXZ;
      const int NYW;
      Functor<real> f;
      int max_warp_split;
      mutable int warp_split; // helper used to keep track of current warp splitting
      const int nParity;
      const T &a, &b, &c;
      std::vector<ColorSpinorField *> &x, &y, &z, &w;

      bool tuneSharedBytes() const { return false; }

      // for these streaming kernels, there is no need to tune the grid size, just use max
      unsigned int minGridSize() const { return maxGridSize(); }

    public:
      MultiBlas(const T &a, const T &b, const T &c, const ColorSpinorField &, const ColorSpinorField &,
                std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w) :
        TunableGridStrideKernel3D(*x[0], y.size(), x[0]->SiteSubset()),
        NXZ(x.size()),
        NYW(y.size()),
        f(NXZ, NYW),
        warp_split(1),
        nParity(x[0]->SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(x),
        y(y),
        z(z),
        w(w)
      {
        checkLocation(*x[0], *y[0], *z[0], *w[0]);
        checkLength(*x[0], *y[0], *z[0], *w[0]);
        auto x_prec = checkPrecision(*x[0], *z[0], *w[0]);
        auto y_prec = y[0]->Precision();
        auto x_order = checkOrder(*x[0], *z[0], *w[0]);
        auto y_order = y[0]->FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        // heuristic for enabling if we need the warp-splitting optimization
        const int gpu_size = 2 * device::max_threads_per_block() * device::processor_count();
        switch (gpu_size / (x[0]->Length() * NYW)) {
        case 0: max_warp_split = 1; break; // we have plenty of work, no need to split
        case 1: max_warp_split = 2; break; // double the thread count
        case 2:                            // quadruple the thread count
        default: max_warp_split = 4;
        }
        max_warp_split = std::min(NXZ, max_warp_split); // ensure we only split if valid

        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y[0]->AuxString());
        }
        char NXZ_str[16];
        char NYW_str[16];
        u32toa(NXZ_str, NXZ);
        u32toa(NYW_str, NYW);
        strcat(aux, ",Nxz=");
        strcat(aux, NXZ_str);
        strcat(aux, ",Nyw=");
        strcat(aux, NYW_str);

#ifdef QUDA_FAST_COMPILE_REDUCE
        strcat(aux, ",fast_compile");
#endif

        apply(device::get_default_stream());

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const { return TuneKey(vol, typeid(f).name(), aux); }

      template <typename Arg> void Launch(const TuneParam &tp, const qudaStream_t &stream, Arg &&arg)
      {
        constexpr bool multi_1d = Arg::Functor::multi_1d;
        if (a.data) { set_param<multi_1d>(arg, 'a', a); }
        if (b.data) { set_param<multi_1d>(arg, 'b', b); }
        if (c.data) { set_param<multi_1d>(arg, 'c', c); }
        launch<MultiBlas_>(tp, stream, arg);
      }

      template <int NXZ> void compute(const qudaStream_t &stream)
      {
        staticCheck<NXZ, store_t, y_store_t, decltype(f)>(f, x, y);

        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll_check && (x[0]->Ncolor() != 3 || x[0]->Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x[0]->Nspin(), x[0]->Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(*x[0], *y[0], *z[0], *w[0]); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Functor<device_real_t> f_(NXZ, NYW);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x[0]->Length() / (nParity * M);

          if (tp.aux.x > 1 && (length * tp.aux.x) % device::warp_size() != 0) {
            // if problem size isn't divisible by the warp size then we can't use warp splitting
            launchError() = QUDA_ERROR;
          } else {
            tp.block.x *= tp.aux.x; // include warp-split factor
            switch (tp.aux.x) {
            case 1:
              Launch(tp, stream, MultiBlasArg<1, device_real_t, M, NXZ, device_store_t, N,
                     device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
              break;
#ifdef WARP_SPLIT
            case 2:
              Launch(tp, stream, MultiBlasArg<2, device_real_t, M, NXZ, device_store_t, N,
                     device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
              break;
            case 4:
              Launch(tp, stream, MultiBlasArg<4, device_real_t, M, NXZ, device_store_t, N,
                     device_y_store_t, Ny, decltype(f_)>(x, y, z, w, f_, NYW, length));
              break;
#endif
            default: errorQuda("warp-split factor %d not instantiated", static_cast<int>(tp.aux.x));
            }

            tp.block.x /= tp.aux.x; // restore block size
          }
        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      template <int n> std::enable_if_t<n!=1, void> instantiateLinear(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiateLinear<n-1>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiateLinear(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      template <int n> std::enable_if_t<n!=1, void> instantiatePow2(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiatePow2<n/2>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiatePow2(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      // instantiate the loop unrolling template
      template <int NXZ_max> std::enable_if_t<NXZ_max!=1, void> instantiate(const qudaStream_t &stream)
      {
        // if multi-1d then constrain the templates to no larger than max-1d size
        constexpr int pow2_max = !decltype(f)::multi_1d ? max_NXZ_power2<false>() :
          std::min(max_N_multi_1d_pow2(), max_NXZ_power2<false>());
        constexpr int linear_max = !decltype(f)::multi_1d ? MAX_MULTI_BLAS_N : std::min(max_N_multi_1d(), MAX_MULTI_BLAS_N);

        if (NXZ <= pow2_max && is_power2(NXZ)) instantiatePow2<pow2_max>(stream);
        else if (NXZ <= linear_max) instantiateLinear<linear_max>(stream);
        else errorQuda("x.size %lu greater than maximum supported size (pow2 = %d, linear = %d)", x.size(), pow2_max, linear_max);
      }

      template <int NXZ_max> std::enable_if_t<NXZ_max==1, void> instantiate(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      void apply(const qudaStream_t &stream) { instantiate<decltype(f)::NXZ_max>(stream); }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.Y) y[i]->backup();
          if (f.write.W) w[i]->backup();
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.Y) y[i]->restore();
          if (f.write.W) w[i]->restore();
        }
      }

#ifdef WARP_SPLIT
      bool advanceAux(TuneParam &param) const
      {
        if (2 * param.aux.x <= max_warp_split) {
          param.aux.x *= 2;
          warp_split = param.aux.x;
          return true;
        } else {
          param.aux.x = 1;
          warp_split = param.aux.x;
          // reset the block dimension manually here to pick up the warp_split parameter
          resetBlockDim(param);
          return false;
        }
      }
#else
      bool advanceAux(TuneParam &) const
      {
        warp_split = 1;
        return false;
      }
#endif

      int blockStep() const { return device::warp_size() / warp_split; }
      int blockMin() const { return device::warp_size() / warp_split; }

      void initTuneParam(TuneParam &param) const
      {
        TunableGridStrideKernel3D::initTuneParam(param);
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      void defaultTuneParam(TuneParam &param) const
      {
        TunableGridStrideKernel3D::defaultTuneParam(param);
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      long long flops() const
      {
        return NYW * NXZ * f.flops() * x[0]->Length();
      }

      long long bytes() const
      {
        // X and Z reads are repeated (and hopefully cached) across NYW
        // each Y and W read/write is done once
        return NYW * NXZ * (f.read.X + f.write.X) * x[0]->Bytes() +
          NYW * (f.read.Y + f.write.Y) * y[0]->Bytes() +
          NYW * NXZ * (f.read.Z + f.write.Z) * z[0]->Bytes() +
          NYW * (f.read.W + f.write.W) * w[0]->Bytes();
      }
    };

    using range = std::pair<size_t,size_t>;

    template <template <typename...> class Functor, typename T>
    void axpy_recurse(const T *a_, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                      const range &range_x, const range &range_y, int upper, int coeff_width)
    {
      // if greater than max single-kernel size, recurse
      size_t max_yw_size = y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
        max_YW_size<Functor<double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
        max_YW_size<Functor<float>>(x.size(), x[0]->Precision(), y[0]->Precision());

      if (y.size() > max_yw_size) {
        // We need to split up 'a' carefully since it's row-major.
        std::vector<T> tmpmajor(x.size() * y.size());
        T *tmpmajor0 = tmpmajor.data();
        T *tmpmajor1 = tmpmajor0 + x.size() * (y.size() / 2);
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i = 0; i < xlen; i++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        axpy_recurse<Functor>(tmpmajor0, x, y0, range_x, range(range_y.first, range_y.first + y0.size()), upper, coeff_width);
        axpy_recurse<Functor>(tmpmajor1, x, y1, range_x, range(range_y.first + y0.size(), range_y.second), upper, coeff_width);
      } else {
        // if at the bottom of recursion,
        if (is_valid_NXZ(x.size(), false)) {
          // since tile range is [first,second), e.g., [first,second-1], we need >= here
          // if upper triangular and upper-right tile corner is below diagonal return
          if (upper == 1 && range_y.first >= range_x.second) { return; }
          // if lower triangular and lower-left tile corner is above diagonal return
          if (upper == -1 && range_x.first >= range_y.second) { return; }

          // mark true since we will copy the "a" matrix into constant memory
          coeff_array<T> a(a_), b, c;
          constexpr bool mixed = true;
          instantiate<Functor, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, x);
        } else {
          // split the problem in half and recurse
          const T *a0 = &a_[0];
          const T *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          axpy_recurse<Functor>(a0, x0, y, range(range_x.first, range_x.first + x0.size()), range_y, upper, coeff_width);
          axpy_recurse<Functor>(a1, x1, y, range(range_x.first + x0.size(), range_x.second), range_y, upper, coeff_width);
        }
      } // end if (y.size() > max_YW_size())
    }

    void caxpy(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 0, 2);
    }

    void caxpy_U(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpy_U with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 1, 2);
    }

    void caxpy_L(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpy_L with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), -1, 2);
    }

    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy(a, x.Components(), y.Components()); }

    void caxpy_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_U(a, x.Components(), y.Components()); }

    void caxpy_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_L(a, x.Components(), y.Components()); }

    template <template <typename...> class Functor, typename T>
    void axpyz_recurse(const T *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y,
                       std::vector<ColorSpinorField*> &z, const range &range_x, const range &range_y,
                       int pass, int upper, int coeff_width)
    {
      // if greater than max single-kernel size, recurse
      size_t max_yw_size = y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
        max_YW_size<Functor<double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
        max_YW_size<Functor<float>>(x.size(), x[0]->Precision(), y[0]->Precision());

      if (y.size() > max_yw_size) {
        // We need to split up 'a' carefully since it's row-major.
        std::vector<T> tmpmajor(x.size() * y.size());
        T *tmpmajor0 = tmpmajor.data();
        T *tmpmajor1 = tmpmajor0 + x.size() * (y.size() / 2);
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
        std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i_ = 0; i_ < xlen; i_++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        axpyz_recurse<Functor>(tmpmajor0, x, y0, z0, range_x, range(range_y.first, range_y.first + y0.size()), pass, upper, coeff_width);
        axpyz_recurse<Functor>(tmpmajor1, x, y1, z1, range_x, range(range_y.first + y0.size(), range_y.second), pass, upper, coeff_width);
      } else {
        // if at bottom of recursion check where we are
        if (is_valid_NXZ(x.size(), false)) {
          // check if tile straddles diagonal for L/U variants
          bool is_diagonal = (upper != 0) && (range_x.first < range_y.second) && (range_y.first < range_x.second);
          // check if tile is first to be updated for full matrices
          bool is_first = (upper == 0) && (range_x.first == 0);
          // whether to do axpyz
          bool do_axpyz = (upper != 0 && is_diagonal && pass == 0) || (upper == 0 && is_first);
          // whether to do axpy
          bool do_axpy = (upper != 0 && !is_diagonal && pass == 1) || (upper == 0 && !is_first);

          if (do_axpyz) {
            coeff_array<T> a(a_), b, c;
            constexpr bool mixed = false;
            instantiate<Functor, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, z);
          } else if (do_axpy) {
            // if upper triangular and upper-right tile corner is below diagonal return
            if (upper == 1 && range_y.first >= range_x.second) { return; }
            // if lower triangular and lower-left tile corner is above diagonal return
            if (upper == -1 && range_x.first >= range_y.second) { return; }

            // off diagonal
            if (coeff_width == 1) axpy(reinterpret_cast<const double*>(a_), x, z);
            else caxpy(reinterpret_cast<const Complex*>(a_), x, z);
          }
        } else {
          // split the problem in half and recurse
          const T *a0 = &a_[0];
          const T *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          axpyz_recurse<Functor>(a0, x0, y, z, range(range_x.first, range_x.first + x0.size()), range_y, pass, upper, coeff_width);
          axpyz_recurse<Functor>(a1, x1, y, z, range(range_x.first + x0.size(), range_x.second), range_y, pass, upper, coeff_width);
        }
      } // end if (y.size() > max_YW_size())
    }

    void axpyz(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0, 1);
    }

    void axpyz_U(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpyz_U with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      // a is upper triangular.
      // first pass does the axpyz on the diagonal
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1, 1);
      // second pass does axpy on the off diagonals
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1, 1);
    }

    void axpyz_L(const double *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpyz_L with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      // a is upper triangular.
      // first pass does the axpyz on the diagonal
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1, 1);
      // second pass does axpy on the off diagonals
      axpyz_recurse<multiaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1, 1);
    }


    void axpyz(const double *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      axpyz(a, x.Components(), y.Components(), z.Components());
    }

    void axpyz_U(const double *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      axpyz_U(a, x.Components(), y.Components(), z.Components());
    }

    void axpyz_L(const double *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      axpyz_L(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0, 2);
    }

    void caxpyz_U(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpyz_U with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1, 2);
      // second pass does caxpy on the off diagonals
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1, 2);
    }

    void caxpyz_L(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      if (x.size() != y.size()) {
        errorQuda("An optimal block caxpyz_L with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1, 2);
      // second pass does caxpy on the off diagonals
      axpyz_recurse<multicaxpyz_>(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1, 2);
    }


    void caxpyz(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz_U(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz_L(a, x.Components(), y.Components(), z.Components());
    }

    void axpyBzpcx(const double *a_, std::vector<ColorSpinorField *> &x_, std::vector<ColorSpinorField *> &y_,
                   const double *b_, ColorSpinorField &z_, const double *c_)
    {
      if (y_.size() <= (size_t)max_N_multi_1d()) {
        // swizzle order since we are writing to x_ and y_, but the
	// multi-blas only allow writing to y and w, and moreover the
	// block width of y and w must match, and x and z must match.
	std::vector<ColorSpinorField*> &y = y_;
	std::vector<ColorSpinorField*> &w = x_;

	// wrap a container around the third solo vector
	std::vector<ColorSpinorField*> x;
	x.push_back(&z_);

        coeff_array<double> a(a_), b(b_), c(c_);
        constexpr bool mixed = true;
        instantiate<multi_axpyBzpcx_, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
	const double *a0 = &a_[0];
	const double *b0 = &b_[0];
	const double *c0 = &c_[0];

	std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);
	std::vector<ColorSpinorField*> y0(y_.begin(), y_.begin() + y_.size()/2);

	axpyBzpcx(a0, x0, y0, b0, z_, c0);

	const double *a1 = &a_[y_.size()/2];
	const double *b1 = &b_[y_.size()/2];
	const double *c1 = &c_[y_.size()/2];

	std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());
	std::vector<ColorSpinorField*> y1(y_.begin() + y_.size()/2, y_.end());

	axpyBzpcx(a1, x1, y1, b1, z_, c1);
      }
    }

    void caxpyBxpz(const Complex *a_, std::vector<ColorSpinorField*> &x_, ColorSpinorField &y_,
		   const Complex *b_, ColorSpinorField &z_)
    {
      if (x_.size() <= (size_t)max_N_multi_1d() && is_valid_NXZ(x_.size(), false)) // only split if we have to.
      {
        // swizzle order since we are writing to y_ and z_, but the
        // multi-blas only allow writing to y and w, and moreover the
        // block width of y and w must match, and x and z must match.
        // Also, wrap a container around them.
        std::vector<ColorSpinorField*> y;
        y.push_back(&y_);
        std::vector<ColorSpinorField*> w;
        w.push_back(&z_);

        // we're reading from x
        std::vector<ColorSpinorField*> &x = x_;

        coeff_array<Complex> a(a_), b(b_), c;
        constexpr bool mixed = true;
        instantiate<multi_caxpyBxpz_, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
        const Complex *a0 = &a_[0];
        const Complex *b0 = &b_[0];

        std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);

        caxpyBxpz(a0, x0, y_, b0, z_);

        const Complex *a1 = &a_[x_.size()/2];
        const Complex *b1 = &b_[x_.size()/2];

        std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());

        caxpyBxpz(a1, x1, y_, b1, z_);
      }
    }

    void axpy(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 0, 1);
    }

    void axpy_U(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_U with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void axpy_L(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_L with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), -1, 1);
    }

    // Composite field version
    void axpy(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy(a, x.Components(), y.Components()); }

    void axpy_U(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_U(a, x.Components(), y.Components()); }

    void axpy_L(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_L(a, x.Components(), y.Components()); }

  } // namespace blas

} // namespace quda
