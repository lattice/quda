#include "hip/hip_runtime.h"
#include <clover_field_order.h>
#include <tune_quda.h>

namespace quda {

  using namespace clover;

#ifdef GPU_CLOVER_DIRAC

  /** 
      Kernel argument struct
  */
  template <typename Out, typename In>
  struct CopyCloverArg {
    Out out;
    const In in;
    int volumeCB;
    CopyCloverArg (const Out &out, const In in, int volume) : out(out), in(in), volumeCB(in.volumeCB) { }
  };

  /** 
      Generic CPU clover reordering and packing
  */
  template <typename FloatOut, typename FloatIn, int length, typename Out, typename In>
  void copyClover(CopyCloverArg<Out,In> arg) {
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    for (int parity=0; parity<2; parity++) {
      for (int x=0; x<arg.volumeCB; x++) {
	RegTypeIn in[length];
	RegTypeOut out[length];
	arg.in.load(in, x, parity);
	for (int i=0; i<length; i++) out[i] = in[i];
	arg.out.save(out, x, parity);
      }
    }

  }

  /** 
      Generic CUDA clover reordering and packing
  */
  template <typename FloatOut, typename FloatIn, int length, typename Out, typename In>
  __global__ void copyCloverKernel(CopyCloverArg<Out,In> arg) {
    typedef typename mapper<FloatIn>::type RegTypeIn;
    typedef typename mapper<FloatOut>::type RegTypeOut;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= arg.volumeCB) return;
    int parity = blockIdx.y * blockDim.y + threadIdx.y;

    RegTypeIn in[length];
    RegTypeOut out[length];
    arg.in.load(in, x, parity);
#pragma unroll
    for (int i=0; i<length; i++) out[i] = in[i];
    arg.out.save(out, x, parity);

  }  

  template <typename FloatOut, typename FloatIn, int length, typename Out, typename In>
    class CopyClover : TunableVectorY {
    CopyCloverArg<Out,In> arg;
    const CloverField &meta;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    CopyClover(CopyCloverArg<Out,In> &arg, const CloverField &meta)
      : TunableVectorY(2), arg(arg), meta(meta) {
      writeAuxString("out_stride=%d,in_stride=%d", arg.out.stride, arg.in.stride);
    }
    virtual ~CopyClover() { ; }
  
    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      copyCloverKernel<FloatOut, FloatIn, length, Out, In> 
	<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; } 
    long long bytes() const { return 2*arg.volumeCB*(arg.in.Bytes() + arg.out.Bytes()); } 
  };

 template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
 void copyClover(OutOrder outOrder, const InOrder inOrder, const CloverField &out, QudaFieldLocation location) {

   CopyCloverArg<OutOrder,InOrder> arg(outOrder, inOrder, out.Volume());
   
   if (location == QUDA_CPU_FIELD_LOCATION) {
     copyClover<FloatOut, FloatIn, length, OutOrder, InOrder>(arg);
   } else if (location == QUDA_CUDA_FIELD_LOCATION) {
     CopyClover<FloatOut, FloatIn, length, OutOrder, InOrder> cloverCopier(arg, out);
     cloverCopier.apply(0);
   } else {
     errorQuda("Undefined field location %d for copyClover", location);
   }

 }

 template <typename FloatOut, typename FloatIn, int length, typename InOrder>
 void copyClover(const InOrder &inOrder, CloverField &out, bool inverse, QudaFieldLocation location, FloatOut *Out, float *outNorm) {

    if (out.isNative()) {
      const bool override = true;
      typedef typename clover_mapper<FloatOut>::type C;
      copyClover<FloatOut,FloatIn,length>(C(out, inverse, Out, outNorm, override), inOrder, out, location);
    } else if (out.Order() == QUDA_PACKED_CLOVER_ORDER) {
      copyClover<FloatOut,FloatIn,length>
	(QDPOrder<FloatOut,length>(out, inverse, Out), inOrder, out, location);
    } else if (out.Order() == QUDA_QDPJIT_CLOVER_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyClover<FloatOut,FloatIn,length>
	(QDPJITOrder<FloatOut,length>(out, inverse, Out), inOrder, out, location);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_BQCD_CLOVER_ORDER) {
      errorQuda("BQCD output not supported");
    } else {
      errorQuda("Clover field %d order not supported", out.Order());
    }

  }

 template <typename FloatOut, typename FloatIn, int length>
 void copyClover(CloverField &out, const CloverField &in, bool inverse, QudaFieldLocation location, 
		 FloatOut *Out, FloatIn *In, float *outNorm, float *inNorm) {

    // reconstruction only supported on FloatN fields currently
   if (in.isNative()) {
      const bool override = true;
      typedef typename clover_mapper<FloatIn>::type C;
      copyClover<FloatOut,FloatIn,length>(C(in, inverse, In, inNorm, override), out, inverse, location, Out, outNorm);
    } else if (in.Order() == QUDA_PACKED_CLOVER_ORDER) {
      copyClover<FloatOut,FloatIn,length>
	(QDPOrder<FloatIn,length>(in, inverse, In), out, inverse, location, Out, outNorm);
    } else if (in.Order() == QUDA_QDPJIT_CLOVER_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      copyClover<FloatOut,FloatIn,length>
	(QDPJITOrder<FloatIn,length>(in, inverse, In), out, inverse, location, Out, outNorm);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_BQCD_CLOVER_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      copyClover<FloatOut,FloatIn,length>
	(BQCDOrder<FloatIn,length>(in, inverse, In), out, inverse, location, Out, outNorm);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else {
      errorQuda("Clover field %d order not supported", in.Order());
    }

  }

#endif

  // this is the function that is actually called, from here on down we instantiate all required templates
  void copyGenericClover(CloverField &out, const CloverField &in, bool inverse, QudaFieldLocation location,
			void *Out, void *In, void *outNorm, void *inNorm) {
#ifdef GPU_CLOVER_DIRAC
    if (out.Precision() == QUDA_HALF_PRECISION && out.Order() > 4) 
      errorQuda("Half precision not supported for order %d", out.Order());
    if (in.Precision() == QUDA_HALF_PRECISION && in.Order() > 4) 
      errorQuda("Half precision not supported for order %d", in.Order());

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
	copyClover<double,double,72>(out, in, inverse, location, (double*)Out, (double*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
	copyClover<double,float,72>(out, in, inverse, location, (double*)Out, (float*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_HALF_PRECISION) {
	copyClover<double,short,72>(out, in, inverse, location, (double*)Out, (short*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
        copyClover<double, char, 72>(
            out, in, inverse, location, (double *)Out, (char *)In, (float *)outNorm, (float *)inNorm);
      } else {
        errorQuda("Unknown precision %d", in.Precision());
      }
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
	copyClover<float,double,72>(out, in, inverse, location, (float*)Out, (double*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
	copyClover<float,float,72>(out, in, inverse, location, (float*)Out, (float*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_HALF_PRECISION) {
	copyClover<float,short,72>(out, in, inverse, location, (float*)Out, (short*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_HALF_PRECISION) {
        copyClover<float, char, 72>(
            out, in, inverse, location, (float *)Out, (char *)In, (float *)outNorm, (float *)inNorm);
      } else {
        errorQuda("Unknown precision %d", in.Precision());
      }
    } else if (out.Precision() == QUDA_HALF_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION){
	copyClover<short,double,72>(out, in, inverse, location, (short*)Out, (double*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
	copyClover<short,float,72>(out, in, inverse, location, (short*)Out, (float*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_HALF_PRECISION) {
	copyClover<short,short,72>(out, in, inverse, location, (short*)Out, (short*)In, (float*)outNorm, (float*)inNorm);
      } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
        copyClover<short, char, 72>(
            out, in, inverse, location, (short *)Out, (char *)In, (float *)outNorm, (float *)inNorm);
      } else {
        errorQuda("Unknown precision %d", in.Precision());
      }
    } else if (out.Precision() == QUDA_QUARTER_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
        copyClover<char, double, 72>(
            out, in, inverse, location, (char *)Out, (double *)In, (float *)outNorm, (float *)inNorm);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
        copyClover<char, float, 72>(
            out, in, inverse, location, (char *)Out, (float *)In, (float *)outNorm, (float *)inNorm);
      } else if (in.Precision() == QUDA_HALF_PRECISION) {
        copyClover<char, short, 72>(
            out, in, inverse, location, (char *)Out, (short *)In, (float *)outNorm, (float *)inNorm);
      } else if (in.Precision() == QUDA_QUARTER_PRECISION) {
        copyClover<char, char, 72>(
            out, in, inverse, location, (char *)Out, (char *)In, (float *)outNorm, (float *)inNorm);
      } else {
        errorQuda("Unknown precision %d", in.Precision());
      }
    } else {
      errorQuda("Unknown precision %d", out.Precision());
    }
#else
    errorQuda("Clover has not been built");
#endif
  }


} // namespace quda
