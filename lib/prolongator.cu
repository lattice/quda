#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <multigrid_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID
  using namespace quda::colorspinor;
  
  /** 
      Kernel argument struct
  */
  template <typename Float, typename vFloat, int fineSpin, int fineColor, int coarseSpin, int coarseColor, QudaFieldOrder order>
  struct ProlongateArg {
    FieldOrderCB<Float,fineSpin,fineColor,1,order> out;
    const FieldOrderCB<Float,coarseSpin,coarseColor,1,order> in;
    const FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order,vFloat> V;
    const int *geo_map;  // need to make a device copy of this
    const spin_mapper<fineSpin,coarseSpin> spin_map;
    const int parity; // the parity of the output field (if single parity)
    const int nParity; // number of parities of input fine field

    ProlongateArg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &V,
                  const int *geo_map,  const int parity)
      : out(out), in(in), V(V), geo_map(geo_map), spin_map(), parity(parity), nParity(out.SiteSubset()) { }

    ProlongateArg(const ProlongateArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,order> &arg)
      : out(arg.out), in(arg.in), V(arg.V), geo_map(arg.geo_map), spin_map(),
        parity(arg.parity), nParity(arg.nParity) { }
  };

  /**
     Applies the grid prolongation operator (coarse to fine)
  */
  template <typename Float, int fineSpin, int coarseColor, class Coarse, typename S>
  __device__ __host__ inline void prolongate(complex<Float> out[fineSpin*coarseColor], const Coarse &in, 
                                             int parity, int x_cb, const int *geo_map, const S& spin_map, int fineVolumeCB) {
    int x = parity*fineVolumeCB + x_cb;
    int x_coarse = geo_map[x];
    int parity_coarse = (x_coarse >= in.VolumeCB()) ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*in.VolumeCB();

#pragma unroll
    for (int s=0; s<fineSpin; s++) {
#pragma unroll
      for (int c=0; c<coarseColor; c++) {
        out[s*coarseColor+c] = in(parity_coarse, x_coarse_cb, spin_map(s,parity), c);
      }
    }
  }

  /**
     Rotates from the coarse-color basis into the fine-color basis.  This
     is the second step of applying the prolongator.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseColor, int fine_colors_per_thread,
            class FineColor, class Rotator>
  __device__ __host__ inline void rotateFineColor(FineColor &out, const complex<Float> in[fineSpin*coarseColor],
                                                  const Rotator &V, int parity, int nParity, int x_cb, int fine_color_block) {
    const int spinor_parity = (nParity == 2) ? parity : 0;
    const int v_parity = (V.Nparity() == 2) ? parity : 0;

    constexpr int color_unroll = 2;

#pragma unroll
    for (int s=0; s<fineSpin; s++)
#pragma unroll
      for (int fine_color_local=0; fine_color_local<fine_colors_per_thread; fine_color_local++)
        out(spinor_parity, x_cb, s, fine_color_block+fine_color_local) = 0.0; // global fine color index
    
#pragma unroll
    for (int s=0; s<fineSpin; s++) {
#pragma unroll
      for (int fine_color_local=0; fine_color_local<fine_colors_per_thread; fine_color_local++) {
        int i = fine_color_block + fine_color_local; // global fine color index

        complex<Float> partial[color_unroll];
#pragma unroll
        for (int k=0; k<color_unroll; k++) partial[k] = 0.0;

#pragma unroll
        for (int j=0; j<coarseColor; j+=color_unroll) {
          // V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
#pragma unroll
          for (int k=0; k<color_unroll; k++)
            partial[k] += V(v_parity, x_cb, s, i, j+k) * in[s*coarseColor + j + k];
        }

#pragma unroll
        for (int k=0; k<color_unroll; k++) out(spinor_parity, x_cb, s, i) += partial[k];
      }
    }

  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread, typename Arg>
  void Prolongate(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb=0; x_cb<arg.out.VolumeCB(); x_cb++) {
        complex<Float> tmp[fineSpin*coarseColor];
        prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.VolumeCB());
        for (int fine_color_block=0; fine_color_block<fineColor; fine_color_block+=fine_colors_per_thread) {
          rotateFineColor<Float,fineSpin,fineColor,coarseColor,fine_colors_per_thread>
            (arg.out, tmp, arg.V, parity, arg.nParity, x_cb, fine_color_block);
        }
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread, typename Arg>
  __global__ void ProlongateKernel(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = arg.nParity == 2 ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
    if (x_cb >= arg.out.VolumeCB()) return;

    int fine_color_block = (blockDim.z*blockIdx.z + threadIdx.z) * fine_colors_per_thread;
    if (fine_color_block >= fineColor) return;

    complex<Float> tmp[fineSpin*coarseColor];
    prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.VolumeCB());
    rotateFineColor<Float,fineSpin,fineColor,coarseColor,fine_colors_per_thread>
      (arg.out, tmp, arg.V, parity, arg.nParity, x_cb, fine_color_block);
  }
  
  template <typename Float, typename vFloat, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread>
  class ProlongateLaunch : public TunableVectorYZ {

  protected:
    ColorSpinorField &out;
    const ColorSpinorField &in;
    const ColorSpinorField &V;
    const int *fine_to_coarse;
    int parity;
    QudaFieldLocation location;
    char vol[TuneKey::volume_n];

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return out.VolumeCB(); } // fine parity is the block y dimension

  public:
    ProlongateLaunch(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &V,
                     const int *fine_to_coarse, int parity)
      : TunableVectorYZ(out.SiteSubset(), fineColor/fine_colors_per_thread), out(out), in(in), V(V),
        fine_to_coarse(fine_to_coarse), parity(parity), location(checkLocation(out, in, V))
    {
      strcpy(vol, out.VolString());
      strcat(vol, ",");
      strcat(vol, in.VolString());

      strcpy(aux, out.AuxString());
      strcat(aux, ",");
      strcat(aux, in.AuxString());
    }

    virtual ~ProlongateLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
        if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
          ProlongateArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
            arg(out, in, V, fine_to_coarse, parity);
          Prolongate<Float,fineSpin,fineColor,coarseSpin,coarseColor,fine_colors_per_thread>(arg);
        } else {
          errorQuda("Unsupported field order %d", out.FieldOrder());
        }
      } else {
        if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          ProlongateArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_FLOAT2_FIELD_ORDER>
            arg(out, in, V, fine_to_coarse, parity);
          ProlongateKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,fine_colors_per_thread>
            <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
        } else {
          errorQuda("Unsupported field order %d", out.FieldOrder());
        }
      }
    }

    TuneKey tuneKey() const { return TuneKey(vol, typeid(*this).name(), aux); }

    long long flops() const { return 8 * fineSpin * fineColor * coarseColor * out.SiteSubset()*(long long)out.VolumeCB(); }

    long long bytes() const {
      size_t v_bytes = V.Bytes() / (V.SiteSubset() == out.SiteSubset() ? 1 : 2);
      return in.Bytes() + out.Bytes() + v_bytes + out.SiteSubset()*out.VolumeCB()*sizeof(int);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
                  const int *fine_to_coarse, int parity) {

    // for all grids use 1 color per thread
    constexpr int fine_colors_per_thread = 1;

    if (v.Precision() == QUDA_HALF_PRECISION) {
#if QUDA_PRECISION & 2
      ProlongateLaunch<Float, short, fineSpin, fineColor, coarseSpin, coarseColor, fine_colors_per_thread>
      prolongator(out, in, v, fine_to_coarse, parity);
      prolongator.apply(0);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else if (v.Precision() == in.Precision()) {
      ProlongateLaunch<Float, Float, fineSpin, fineColor, coarseSpin, coarseColor, fine_colors_per_thread>
      prolongator(out, in, v, fine_to_coarse, parity);
      prolongator.apply(0);
    } else {
      errorQuda("Unsupported V precision %d", v.Precision());
    }

    if (checkLocation(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }


  template <typename Float, int fineSpin>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
                  int nVec, const int *fine_to_coarse, const int * const * spin_map, int parity) {

    if (in.Nspin() != 2) errorQuda("Coarse spin %d is not supported", in.Nspin());
    const int coarseSpin = 2;

    // first check that the spin_map matches the spin_mapper
    spin_mapper<fineSpin,coarseSpin> mapper;
    for (int s=0; s<fineSpin; s++) 
      for (int p=0; p<2; p++)
        if (mapper(s,p) != spin_map[s][p]) errorQuda("Spin map does not match spin_mapper");

    if (out.Ncolor() == 3) {
      const int fineColor = 3;
#ifdef NSPIN4
      if (nVec == 6) { // Free field Wilson
        Prolongate<Float,fineSpin,fineColor,coarseSpin,6>(out, in, v, fine_to_coarse, parity);
      } else
#endif // NSPIN4
      if (nVec == 24) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, parity);
#ifdef NSPIN4
      } else if (nVec == 32) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
#endif // NSPIN4
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
#ifdef NSPIN4
    } else if (out.Ncolor() == 6) { // for coarsening coarsened Wilson free field.
      const int fineColor = 6;
      if (nVec == 6) { // these are probably only for debugging only
        Prolongate<Float,fineSpin,fineColor,coarseSpin,6>(out, in, v, fine_to_coarse, parity);
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
#endif // NSPIN4
    } else if (out.Ncolor() == 24) {
      const int fineColor = 24;
      if (nVec == 24) { // to keep compilation under control coarse grids have same or more colors
        Prolongate<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, parity);
#ifdef NSPIN4
      } else if (nVec == 32) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
#endif // NSPIN4
#ifdef NSPIN1
      } else if (nVec == 64) { 
        Prolongate<Float,fineSpin,fineColor,coarseSpin,64>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 96) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,96>(out, in, v, fine_to_coarse, parity);
#endif // NSPIN1
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
#ifdef NSPIN4
    } else if (out.Ncolor() == 32) {
      const int fineColor = 32;
      if (nVec == 32) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
#endif // NSPIN4
#ifdef NSPIN1
    } else if (out.Ncolor() == 64) {
      const int fineColor = 64;
      if (nVec == 64) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,64>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 96) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,96>(out, in, v, fine_to_coarse, parity);
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (out.Ncolor() == 96) {
      const int fineColor = 96;
      if (nVec == 96) {
        Prolongate<Float,fineSpin,fineColor,coarseSpin,96>(out, in, v, fine_to_coarse, parity);
      } else {
        errorQuda("Unsupported nVec %d", nVec);
      }
#endif // NSPIN1
    } else {
      errorQuda("Unsupported nColor %d", out.Ncolor());
    }
  }

  template <typename Float>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
                  int Nvec, const int *fine_to_coarse, const int * const * spin_map, int parity) {

    if (out.Nspin() == 2) {
      Prolongate<Float,2>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#ifdef NSPIN4
    } else if (out.Nspin() == 4) {
      Prolongate<Float,4>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#endif
#if 0 // Not needed until we have Laplace MG or staggered MG Lanczos
//#ifdef NSPIN1
    } else if (out.Nspin() == 1) {
      Prolongate<Float,1>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", out.Nspin());
    }
  }

#endif // GPU_MULTIGRID

  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
                  int Nvec, const int *fine_to_coarse, const int * const * spin_map, int parity) {
#ifdef GPU_MULTIGRID
    if (out.FieldOrder() != in.FieldOrder() || out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)", 
                out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    QudaPrecision precision = checkPrecision(out, in);

    if (precision == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      Prolongate<double>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (precision == QUDA_SINGLE_PRECISION) {
      Prolongate<float>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }

    if (checkLocation(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
#else
    errorQuda("Multigrid has not been built");
#endif
  }

} // end namespace quda
