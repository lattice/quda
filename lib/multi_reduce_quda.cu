#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <uint_to_char.h>
#include <kernels/multi_reduce_core.cuh>
#include <tunable_reduction.h>

namespace quda {

  namespace blas {

    template <template <typename ...> class Reducer, typename store_t, typename y_store_t, int nSpin, typename T>
    class MultiReduce : public TunableMultiReduction<1>
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      const int NXZ;
      const int NYW;
      Reducer<device_reduce_t, real> r;
      const int nParity;
      const T &a, &b, &c;
      std::vector<ColorSpinorField *> &x, &y, &z, &w;
      host_reduce_t *result;
      QudaFieldLocation location;

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
          sharedBytesPerThread() * nthreads : sharedBytesPerBlock(param);
        return false;
      }

    public:
      MultiReduce(const T &a, const T &b, const T &c, const ColorSpinorField &, const ColorSpinorField &,
                  std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                  std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w,
                  host_reduce_t *result) :
        TunableMultiReduction(*x[0], y.size(), max_n_batch_block_multi_reduce()),
        NXZ(x.size()),
        NYW(y.size()),
        r(NXZ, NYW),
        nParity(x[0]->SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(x),
        y(y),
        z(z),
        w(w),
        result(result),
        location(checkLocation(*x[0], *y[0], *z[0], *w[0]))
      {
        checkLength(*x[0], *y[0], *z[0], *w[0]);
        auto x_prec = checkPrecision(*x[0], *z[0], *w[0]);
        auto y_prec = y[0]->Precision();
        auto x_order = checkOrder(*x[0], *z[0], *w[0]);
        auto y_order = y[0]->FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        char aux2[TuneKey::aux_n];
        strcpy(aux2, aux);
        strcpy(aux, "policy_kernel,");
        strcat(aux, aux2);
        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y[0]->AuxString());
        }

        char NXZ_str[16];
        char NYW_str[16];
        u32toa(NXZ_str, NXZ);
        u32toa(NYW_str, NYW);
        strcat(aux, ",Nxz=");
        strcat(aux, NXZ_str);
        strcat(aux, ",Nyw=");
        strcat(aux, NYW_str);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // since block dot product and block norm use the same functors, we need to distinguish them
        bool is_norm = false;
        if (NXZ == NYW) {
          is_norm = true;
          for (int i = 0; i < NXZ; i++) {
            if (x[i]->V() != y[i]->V() || x[i]->V() != z[i]->V() || x[i]->V() != w[i]->V()) {
              is_norm = false;
              break;
            }
          }
        }
        if (is_norm) strcat(aux, ",norm");

        apply(device::get_default_stream());

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const { return TuneKey(vol, typeid(r).name(), aux); }

      template <int NXZ> void compute(const qudaStream_t &stream)
      {
        staticCheck<NXZ, store_t, y_store_t, decltype(r)>(r, x, y);

        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll_check && (x[0]->Ncolor() != 3 || x[0]->Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x[0]->Nspin(), x[0]->Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(*x[0], *y[0], *z[0], *w[0]); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Reducer<device_reduce_t, device_real_t> r_(NXZ, NYW);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x[0]->Length() / M;

          MultiReduceArg<device_real_t, M, NXZ, device_store_t, N, device_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, r_, NYW, length, nParity);

          std::vector<host_reduce_t> result_(NXZ * arg.NYW);

#if 0 // no parameters to set so far
          constexpr bool multi_1d = false;
          if (a.data) { set_param<multi_1d>(arg, 'a', a); }
          if (b.data) { set_param<multi_1d>(arg, 'b', b); }
          if (c.data) { set_param<multi_1d>(arg, 'c', c); }
#endif
          // we intentionally do not do a global reduction in the launch, and defer until the entire "tile" is complete
          launch<MultiReduce_, host_reduce_t, comm_reduce_null<host_reduce_t>>(result_, tp, stream, arg);

          // need to transpose for same order with vector thread reduction
          for (int i = 0; i < NXZ; i++) {
            for (int j = 0; j < arg.NYW; j++) {
              result[i * arg.NYW + j] = result_[j * NXZ + i];
            }
          }

        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      template <int n> std::enable_if_t<n!=1, void> instantiateLinear(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiateLinear<n-1>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiateLinear(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      template <int n> std::enable_if_t<n!=1, void> instantiatePow2(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiatePow2<n/2>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiatePow2(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      void apply(const qudaStream_t &stream)
      {
        constexpr int pow2_max = max_NXZ_power2<true>();
        if (NXZ <= pow2_max && is_power2(NXZ)) instantiatePow2<pow2_max>(stream);
        else if (NXZ <= MAX_MULTI_BLAS_N) instantiateLinear<MAX_MULTI_BLAS_N>(stream);
        else errorQuda("x.size %lu greater than MAX_MULTI_BLAS_N %d", x.size(), MAX_MULTI_BLAS_N);
      }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.X) x[i]->backup();
          if (r.write.Y) y[i]->backup();
          if (r.write.Z) z[i]->backup();
          if (r.write.W) w[i]->backup();
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.X) x[i]->restore();
          if (r.write.Y) y[i]->restore();
          if (r.write.Z) z[i]->restore();
          if (r.write.W) w[i]->restore();
        }
      }

      long long flops() const
      {
        return NYW * NXZ * r.flops() * x[0]->Length();
      }

      long long bytes() const
      {
        // X and Z reads are repeated (and hopefully cached) across NYW
        // each Y and W read/write is done once
        return NYW * NXZ * (r.read.X + r.write.X) * x[0]->Bytes() +
          NYW * (r.read.Y + r.write.Y) * y[0]->Bytes() +
          NYW * NXZ * (r.read.Z + r.write.Z) * z[0]->Bytes() +
          NYW * (r.read.W + r.write.W) * w[0]->Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <template <typename ...> class ReducerDiagonal, template <typename ...> class ReducerOffDiagonal, typename T>
    void multiReduce(T result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
                     CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
                     CompositeColorSpinorField &w, int i, int j)
    {
      if (i == j) { // we are on the diagonal so invoke the diagonal reducer
        using host_reduce_t = typename ReducerDiagonal<double, double>::reduce_t;
        instantiate<ReducerDiagonal, MultiReduce, true>(a, b, c, *x[0], *y[0], x, y, z, w, (host_reduce_t*)result);
      } else { // we are on the diagonal so invoke the off-diagonal reducer
        using host_reduce_t = typename ReducerOffDiagonal<double, double>::reduce_t;
        instantiate<ReducerOffDiagonal, MultiReduce, true>(a, b, c, *x[0], *y[0], x, y, z, w, (host_reduce_t*)result);
      }
    }

    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    void multiReduce_recurse(T *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                             std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w, int i_idx,
                             int j_idx, bool hermitian, uint2 tile_size)
    {
      if (y.size() > tile_size.y) { // if greater than max single-kernel size, split and recurse
        // Do the recurse first.
        T* result0 = &result[0];
        T* result1 = &result[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());
        multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result0, x, y0, z, w, i_idx, 2*j_idx+0, hermitian, tile_size);
        multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result1, x, y1, z, w, i_idx, 2*j_idx+1, hermitian, tile_size);
      } else {
        T* tmp_dot = new T[x.size()*y.size()];

	// if at bottom of recursion, return if on lower left
        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true) && hermitian) {
          if (j_idx < i_idx) { return; }
        }

        coeff_array<T> a, b, c;


        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true)) {
          // problem will fit, so do the computation
          multiReduce<ReducerDiagonal, ReducerOffDiagonal>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);
        } else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).
          // Use a few temporary variables.

          T* tmpmajor = new T[x.size()*y.size()];
          T* result0 = &tmpmajor[0];
          T* result1 = &tmpmajor[(x.size()/2)*y.size()];
          std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
          std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());
          std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
          std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());
          std::vector<ColorSpinorField*> w0(w.begin(), w.begin() + w.size()/2);
          std::vector<ColorSpinorField*> w1(w.begin() + w.size()/2, w.end());

          multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result0, x0, y, z0, w0, 2*i_idx+0, j_idx, hermitian, tile_size);
          multiReduce_recurse<ReducerDiagonal,ReducerOffDiagonal>(result1, x1, y, z1, w1, 2*i_idx+1, j_idx, hermitian, tile_size);

          const unsigned int xlen0 = x.size()/2;
          const unsigned int xlen1 = x.size() - xlen0;
          const unsigned int ylen = y.size();

          // Copy back into result.
          int count = 0, count0 = 0, count1 = 0;
          for (unsigned int i = 0; i < ylen; i++)
          {
            for (unsigned int j = 0; j < xlen0; j++)
              result[count++] = result0[count0++];
            for (unsigned int j = 0; j < xlen1; j++)
              result[count++] = result1[count1++];
          }

          delete[] tmpmajor;
        }

        // we are at the leaf of the binary tree (e.g., we ran the kernel): perform the row-to-column-major transpose here.
        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true)) {
          const unsigned int xlen = x.size();
          const unsigned int ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = tmp_dot[j*ylen + i];
        }
        delete[] tmp_dot;
      }
    }

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    class TileSizeTune : public Tunable
    {
      typedef std::vector<ColorSpinorField*> vec;
      T *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

      int NYW_max;
      uint2 max_tile_size;

    public:
      TileSizeTune(T *result, vec &x, vec &y, vec &z, vec &w, bool hermitian, bool Anorm = false,
                   bool nested_policy = false) :
        result(result),
        x(x),
        y(y),
        z(z),
        w(w),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        NYW_max = std::min(
          (y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
           max_YW_size<ReducerDiagonal<device_reduce_t, double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
           max_YW_size<ReducerDiagonal<device_reduce_t, float>>(x.size(), x[0]->Precision(), y[0]->Precision())),
          (y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
           max_YW_size<ReducerOffDiagonal<device_reduce_t, double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
           max_YW_size<ReducerOffDiagonal<device_reduce_t, float>>(x.size(), x[0]->Precision(), y[0]->Precision()))
                               );

        max_tile_size = make_uint2(1, 1);

        strcpy(aux, nested_policy ? "nested_policy," : "policy,");
        strcat(aux, x[0]->AuxString());
      	strcat(aux, ",");
      	strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        // FIXME this will break if the kernels are destructive - which they aren't here
        if (!tuned()) {
          if (!nested_policy) disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundant now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(1, std::min(NYW_max, (int)y.size()));
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else if (y.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(std::min((size_t)max_NXZ_power2(true), x.size()), 1);
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else { // 2-d reduction

            // max_tile_size should be set to the largest power of 2,
            // since we have a requirement that the tile size is a
            // power of 2.
            // FIXME - we only do simple square tiling here
            max_tile_size = make_uint2(max_NXZ_power2(true), max_NXZ_power2(true));

            // Make sure constituents are tuned.
            for (unsigned int tile_size = 1;
                 tile_size <= max_tile_size.x && tile_size <= x.size() && (tile_size <= y.size() || y.size() == 1);
                 tile_size *= 2) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tile_size, tile_size));
            }

            // also test case using a single kernel if both dimensions are less than max
            if (is_valid_NXZ(x.size(), true) && y.size() <= (unsigned int)NYW_max) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(x.size(), y.size()));
            }
          }

          if (!nested_policy) enableProfileCount();
          setPolicyTuning(true);
        }

        apply(device::get_default_stream());
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size.
        multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tp.aux.x, tp.aux.y));
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {
        // for 1-d reductions we don't do any tuning and just use the largest tile
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else { // 2-d reduction

          if ((unsigned int)(2 * param.aux.x) <= max_tile_size.x && (unsigned int)(2 * param.aux.y) <= max_tile_size.y
              && (unsigned int)(2 * param.aux.x) <= x.size() && (unsigned int)(2 * param.aux.y) <= y.size()) {
            // only tune powers of two
            param.aux.x *= 2;
            param.aux.y *= 2;
            return true;
          } else if (is_valid_NXZ(x.size(), true) && y.size() <= (size_t)NYW_max
                     && ((size_t)param.aux.x != x.size() || (size_t)param.aux.y != y.size())) {
            // we've run out of power of two tiles to try, but before
            // we finish, try a single kernel if it fits
            param.aux.x = x.size();
            param.aux.y = y.size();
            return true;
          } else {
            // reset to the beginning (which we'd need for multi-dimensional tuning)
            param.aux.x = 1;
            param.aux.y = 1;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
        Tunable::initTuneParam(param);
        if (x.size() == 1 || y.size() == 1) {
          param.aux.x = max_tile_size.x;
          param.aux.y = max_tile_size.y;
        } else { // only do non-trivial tuning for 2-d reductions
          param.aux.x = 1;
          param.aux.y = 1;
        }
        param.aux.z = 0;
        param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const  {
        Tunable::defaultTuneParam(param); // default is max tile size
        param.aux.x = max_tile_size.x;
        param.aux.y = max_tile_size.y;
        param.aux.z = 0;
        param.aux.w = 0;
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T>
    class TransposeTune : public Tunable
    {
      using TileTuner = TileSizeTune<ReducerDiagonal, ReducerOffDiagonal, T>;
      using vec = std::vector<ColorSpinorField *>;
      T *result;
      vec &x, &y;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    public:
      TransposeTune(T *result, vec &x, vec &y, bool hermitian, bool Anorm = false) :
        result(result),
        x(x),
        y(y),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        strcpy(aux, "policy,");
        strcat(aux, x[0]->AuxString());
        strcat(aux, ",");
        strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
        strcat(aux, ",n=");
        char size[8];
        u64toa(size, x.size());
        strcat(aux, size);
        strcat(aux, ",m=");
        u64toa(size, y.size());
        strcat(aux, size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        if (!tuned()) {
          disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundent now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) {
            TileTuner tile(result, x, y, x, x, hermitian, Anorm, true);
          } else if (y.size() == 1) {
            TileTuner tile(result, y, x, y, y, hermitian, Anorm, true);
          } else {

            { // tune regular inner product
              TileTuner tile(result, x, y, x, x, hermitian, Anorm, true);
            }

            { // tune transpose inner product
              TileTuner tile(result, y, x, y, y, hermitian, Anorm, true);
            }
          }

          enableProfileCount();
          setPolicyTuning(true);
        }

        apply(device::get_default_stream());
      }

      virtual ~TransposeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (tp.aux.x == 0) {
          TileTuner(result, x, y, x, x, hermitian, Anorm, true);
        } else if (tp.aux.x == 1) {
          T *result_trans = new T[x.size() * y.size()];

          // swap (x<->y and w<-z> when doing transpose calculation)
          TileTuner(result_trans, y, x, y, y, hermitian, Anorm, true);

          // tranpose the result if we are doing the transpose calculation
          const auto xlen = x.size();
          const auto ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++) result[i * xlen + j] = conj(result_trans[j * ylen + i]);

          delete[] result_trans;
        } else {
          errorQuda("Unexpected transpose parameter %d", static_cast<int>(tp.aux.x));
        }
      }

      bool advanceAux(TuneParam &param) const
      {
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else {
          if (param.aux.x == 0) {
            param.aux.x = 1;
            return true;
          } else {
            param.aux.x = 0;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        if (x.size() == 1)
          param.aux = make_int4(0, 0, 0, 0);
        else if (y.size() == 1)
          param.aux = make_int4(1, 0, 0, 0);
        else
          param.aux = make_int4(0, 0, 0, 0); // default is not to transpose
      }

      void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

      TuneKey tuneKey() const { return TuneKey(x[0]->VolString(), typeid(*this).name(), aux); }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() {}  // FIXME - use write to determine what needs to be saved
      void postTune() {} // FIXME - use write to determine what needs to be saved
    };

    void reDotProduct(double *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      double *result_tmp = new double[x.size() * y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      if (x.size() == 1) {
        auto NYW_max = y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiDot<device_reduce_t, double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
          max_YW_size<multiDot<device_reduce_t, float>>(x.size(), x[0]->Precision(), y[0]->Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x[0]->Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<multiDot, multiDot>(result_tmp, x, y, x, x, 0, 0, false, max_tile_size);
      } else if (y.size() == 1 && x[0]->Precision() == y[0]->Precision()) {

        double *result_trans = new double[x.size() * y.size()];

        // swap (x<->y and w<-z> when doing transpose calculation)
        auto NXZ_max = x[0]->Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiDot<device_reduce_t, double>>(y.size(), y[0]->Precision(), x[0]->Precision()) :
          max_YW_size<multiDot<device_reduce_t, float>>(y.size(), y[0]->Precision(), x[0]->Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x[0]->Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<multiDot, multiDot>(result_trans, y, x, y, y, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++)
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = result_trans[j * ylen + i];

        delete[] result_trans;

      } else if (x[0]->Precision() == y[0]->Precision()) {
        TransposeTune<multiDot, multiDot, double>(result_tmp, x, y, false);
      } else {
        TileSizeTune<multiDot, multiDot, double>(result_tmp, x, y, x, x, false);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = x.size() * y.size();
      reduceDoubleArray(result_tmp, Nreduce);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++) result[j * ylen + i] = result_tmp[i * xlen + j];

      delete[] result_tmp;
    }

    void cDotProduct(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      Complex *result_tmp = new Complex[x.size() * y.size()];
      for (unsigned int i = 0; i < x.size() * y.size(); i++) result_tmp[i] = 0.0;

      if (x.size() == 1) {
        auto NYW_max = y[0]->Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiCdot<device_reduce_t, double>>(x.size(), x[0]->Precision(), y[0]->Precision()) :
          max_YW_size<multiCdot<device_reduce_t, float>>(x.size(), x[0]->Precision(), y[0]->Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x[0]->Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<multiCdot, multiCdot>(result_tmp, x, y, x, x, 0, 0, false, max_tile_size);
      } else if (y.size() == 1 && x[0]->Precision() == y[0]->Precision()) {

        Complex *result_trans = new Complex[x.size() * y.size()];

        // swap (x<->y and w<-z> when doing transpose calculation)
        auto NXZ_max = x[0]->Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiCdot<device_reduce_t, double>>(y.size(), y[0]->Precision(), x[0]->Precision()) :
          max_YW_size<multiCdot<device_reduce_t, float>>(y.size(), y[0]->Precision(), x[0]->Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x[0]->Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<multiCdot, multiCdot>(result_trans, y, x, y, y, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++)
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = conj(result_trans[j * ylen + i]);

        delete[] result_trans;

      } else if (x[0]->Precision() == y[0]->Precision()) {
        TransposeTune<multiCdot, multiCdot, Complex>(result_tmp, x, y, false);
      } else {
        TileSizeTune<multiCdot, multiCdot, Complex>(result_tmp, x, y, x, x, false);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
    }

    void hDotProduct(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<multiCdot, multiCdot, Complex>(result_tmp, x, y, x, x, true, false); // last false is b/c L2 norm

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
	}

      delete[] result_tmp;
    }

    // for (p, Ap) norms in CG which are Hermitian.
    void hDotProduct_Anorm(Complex *result, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<multiCdot, multiCdot, Complex>(result_tmp, x, y, x, x, true, true); // last true is b/c A norm

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
        }

      delete[] result_tmp;
    }

    // takes the outer product of inner products between and y and copies y into z
    void cDotProductCopy(Complex* , std::vector<ColorSpinorField*>&, std::vector<ColorSpinorField*>&,
			 std::vector<ColorSpinorField*>&)
    {
#if 0
      // FIXME - if this is enabled we need to ensure that use_w is
      // enabled above.  Also, I think this might break if the diagonal
      // write is different from the off-diagonal write
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (y.size() != z.size()) errorQuda("Cannot copy input y of size %lu into z of size %lu\n", y.size(), z.size());

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // When recursing, only the diagonal tiles will do the copy, the rest just do the outer product
      TileSizeTune<double2, typename vector<device_reduce_t,2>::type,multiCdotCopy,multiCdot,Complex>(result_tmp, x, y, x, y, true);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
#else
      errorQuda("cDotProductCopy not enabled");
#endif
    }

   } // namespace blas

} // namespace quda
