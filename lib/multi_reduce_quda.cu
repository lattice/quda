#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <uint_to_char.h>
#include <kernels/multi_reduce_core.cuh>
#include <tunable_reduction.h>

namespace quda {

  namespace blas {

    template <template <typename ...> class Reducer, typename store_t, typename y_store_t, int nSpin,
              typename T>
    class MultiReduce : public TunableMultiReduction
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      const int NXZ;
      const int NYW;
      Reducer<device_reduce_t, real> r;
      const int nParity;
      const T &a, &b, &c;
      cvector_ref<ColorSpinorField> &x, &y, &z, &w;
      T &result;
      QudaFieldLocation location;

      virtual bool advanceSharedBytes(TuneParam &param) const override
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
          sharedBytesPerThread() * nthreads : sharedBytesPerBlock(param);
        return false;
      }

    public:
      template <typename Vx, typename Vy, typename Vz, typename Vw>
      MultiReduce(const T &a, const T &b, const T &c, const ColorSpinorField &x0, const ColorSpinorField &y0,
                  Vx &x, Vy &y, Vz &z, Vw &w, T &result) :
        TunableMultiReduction(x[0], 1u, y.size(), max_n_batch_block_multi_reduce()),
        NXZ(x.size()),
        NYW(y.size()),
        r(NXZ, NYW),
        nParity(x0.SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(reinterpret_cast<cvector_ref<ColorSpinorField>&>(x)),
        y(reinterpret_cast<cvector_ref<ColorSpinorField>&>(y)),
        z(reinterpret_cast<cvector_ref<ColorSpinorField>&>(z)),
        w(reinterpret_cast<cvector_ref<ColorSpinorField>&>(w)),
        result(result),
        location(checkLocation(x[0], y[0], z[0], w[0]))
      {
        checkLength(x[0], y[0], z[0], w[0]);
        auto x_prec = checkPrecision(x[0], z[0], w[0]);
        auto y_prec = y0.Precision();
        auto x_order = checkOrder(x[0], z[0], w[0]);
        auto y_order = y0.FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        char aux2[TuneKey::aux_n];
        strcpy(aux2, aux);
        strcpy(aux, "policy_kernel,");
        strcat(aux, aux2);
        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y0.AuxString().c_str());
        }

        char NXZ_str[16];
        char NYW_str[16];
        u32toa(NXZ_str, NXZ);
        u32toa(NYW_str, NYW);
        strcat(aux, ",Nxz=");
        strcat(aux, NXZ_str);
        strcat(aux, ",Nyw=");
        strcat(aux, NYW_str);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // since block dot product and block norm use the same functors, we need to distinguish them
        bool is_norm = false;
        if (NXZ == NYW) {
          is_norm = true;
          for (int i = 0; i < NXZ; i++) {
            if (x[i].V() != y[i].V() || x[i].V() != z[i].V() || x[i].V() != w[i].V()) {
              is_norm = false;
              break;
            }
          }
        }
        if (is_norm) strcat(aux, ",norm");

        apply(device::get_default_stream());

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const override { return TuneKey(vol, typeid(r).name(), aux); }

      template <int NXZ> void compute(const qudaStream_t &stream)
      {
        staticCheck<NXZ, store_t, y_store_t, decltype(r)>(r, x, y);

        auto &x0 = x[0];
        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll_check && (x0.Ncolor() != 3 || x0.Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x0.Nspin(), x0.Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(x[0], y[0], z[0], w[0]); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Reducer<device_reduce_t, device_real_t> r_(NXZ, NYW);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x0.Length() / M;

          MultiReduceArg<device_real_t, M, NXZ, device_store_t, N, device_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, r_, NYW, length, nParity);

          std::vector<host_reduce_t> result_(NXZ * arg.NYW);

#if 0 // no parameters to set so far
          constexpr bool multi_1d = false;
          if (a.size()) { set_param<multi_1d>(arg, 'a', a); }
          if (b.size()) { set_param<multi_1d>(arg, 'b', b); }
          if (c.size()) { set_param<multi_1d>(arg, 'c', c); }
#endif
          launch<MultiReduce_>(result_, tp, stream, arg);

          // need to transpose for same order with vector thread reduction
          for (int i = 0; i < NXZ; i++) {
            for (int j = 0; j < arg.NYW; j++) {
              reinterpret_cast<host_reduce_t*>(result.data())[i * arg.NYW + j] = result_[j * NXZ + i];
            }
          }

        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      template <int n> std::enable_if_t<n!=1, void> instantiateLinear(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiateLinear<n-1>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiateLinear(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      template <int n> std::enable_if_t<n!=1, void> instantiatePow2(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiatePow2<n/2>(stream);
      }

      template <int n> std::enable_if_t<n==1, void> instantiatePow2(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      void apply(const qudaStream_t &stream) override
      {
        constexpr int pow2_max = max_NXZ_power2(true);
        if (NXZ <= pow2_max && is_power2(NXZ)) instantiatePow2<pow2_max>(stream);
        else if (NXZ <= MAX_MULTI_BLAS_N) instantiateLinear<MAX_MULTI_BLAS_N>(stream);
        else errorQuda("x.size %lu greater than MAX_MULTI_BLAS_N %d", x.size(), MAX_MULTI_BLAS_N);
      }

      void preTune() override
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.X) x[i].backup();
          if (r.write.Y) y[i].backup();
          if (r.write.Z) z[i].backup();
          if (r.write.W) w[i].backup();
        }
      }

      void postTune() override
      {
        for (int i = 0; i < NYW; ++i) {
          if (r.write.X) x[i].restore();
          if (r.write.Y) y[i].restore();
          if (r.write.Z) z[i].restore();
          if (r.write.W) w[i].restore();
        }
      }

      long long flops() const override
      {
        return NYW * NXZ * r.flops() * x[0].Length();
      }

      long long bytes() const override
      {
        // X and Z reads are repeated (and hopefully cached) across NYW
        // each Y and W read/write is done once
        return NYW * NXZ * (r.read.X + r.write.X) * x[0].Bytes() +
          NYW * (r.read.Y + r.write.Y) * y[0].Bytes() +
          NYW * NXZ * (r.read.Z + r.write.Z) * z[0].Bytes() +
          NYW * (r.read.W + r.write.W) * w[0].Bytes();
      }
    };

    template <template <typename ...> class ReducerDiagonal, template <typename ...> class ReducerOffDiagonal,
              typename T, typename Vx, typename Vy, typename Vz, typename Vw>
    void multiReduce(std::vector<T> &result, const std::vector<T> &a, const std::vector<T> &b, const std::vector<T> &c,
                     Vx &x, Vy &y, Vz &z, Vw &w, int i, int j)
    {
      if (i == j) { // we are on the diagonal so invoke the diagonal reducer
        instantiate<ReducerDiagonal, MultiReduce, true>(a, b, c, x[0], y[0], x, y, z, w, result);
      } else { // we are on the diagonal so invoke the off-diagonal reducer
        instantiate<ReducerOffDiagonal, MultiReduce, true>(a, b, c, x[0], y[0], x, y, z, w, result);
      }
    }

    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <typename ...> class reducer_diag,
              template <typename ...> class reducer_off, typename T, typename Vx, typename Vy, typename Vz, typename Vw>
    void multiReduce_recurse(std::vector<T> &result, Vx &x, Vy &y, Vz &z, Vw &w, int i_idx, int j_idx,
                             bool hermitian, uint2 tile_size)
    {
      if (y.size() > tile_size.y) { // if greater than max single-kernel size, split and recurse
        // Do the recurse first.
        auto result_ = std::make_pair( std::vector<T>((y.size() / 2) * x.size()),
                                       std::vector<T>(result.size() - (y.size() / 2) * x.size()) );
        auto y_ = bisect(y);
        multiReduce_recurse<reducer_diag, reducer_off>(result_.first, x, y_.first, z, w, i_idx, j_idx, hermitian, tile_size);
        multiReduce_recurse<reducer_diag, reducer_off>(result_.second, x, y_.second, z, w, i_idx, j_idx + y_.first.size(), hermitian, tile_size);

        result = join(result_);
      } else {

        if (x.size() <= tile_size.x && is_valid_NXZ(x.size(), true)) { // problem fits, so do the computation
          // if at bottom of recursion, return if on strict sub-diagonal
          if (hermitian && (j_idx + y.size() < i_idx + x.size())) return;

          std::vector<T> tmp_dot(x.size()*y.size());
          std::vector<T> a, b, c;

          multiReduce<reducer_diag, reducer_off>(tmp_dot, a, b, c, x, y, z, w, i_idx, j_idx);

          // perform the row-to-column-major transpose here.
          result = transpose(tmp_dot, x.size(), y.size());
        } else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).

          auto result_ = std::make_pair( std::vector<T>((x.size() / 2) * y.size()),
                                         std::vector<T>(result.size() - (x.size() / 2) * y.size()) );
          auto x_ = bisect(x);
          auto z_ = bisect(z);
          auto w_ = bisect(w);

          multiReduce_recurse<reducer_diag, reducer_off>(result_.first, x_.first, y, z_.first, w_.first, i_idx, j_idx, hermitian, tile_size);
          multiReduce_recurse<reducer_diag, reducer_off>(result_.second, x_.second, y, z_.second, w_.second, i_idx + x_.first.size(), j_idx, hermitian, tile_size);

          result = join_row(result_, x.size() / 2, x.size() - x.size() / 2, y.size());
        }
      }
    }

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T,
              typename Vx, typename Vy, typename Vz = Vx, typename Vw = Vx>
    class TileSizeTune : public Tunable
    {
      std::vector<T> &result;
      Vx &x;
      Vy &y;
      Vz &z;
      Vw &w;
      bool hermitian;
      bool Anorm;

      int NYW_max;
      uint2 max_tile_size;

    public:
      TileSizeTune(std::vector<T> &result, Vx &x, Vy &y, Vz &z, Vw &w, bool hermitian, bool Anorm = false,
                   bool nested_policy = false) :
        result(result),
        x(x),
        y(y),
        z(z),
        w(w),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        auto &x0 = x[0];
        auto &y0 = y[0];

        NYW_max = std::min(
          (y0.Precision() == QUDA_DOUBLE_PRECISION ?
           max_YW_size<ReducerDiagonal<device_reduce_t, double>>(x.size(), x0.Precision(), y0.Precision()) :
           max_YW_size<ReducerDiagonal<device_reduce_t, float>>(x.size(), x0.Precision(), y0.Precision())),
          (y0.Precision() == QUDA_DOUBLE_PRECISION ?
           max_YW_size<ReducerOffDiagonal<device_reduce_t, double>>(x.size(), x0.Precision(), y0.Precision()) :
           max_YW_size<ReducerOffDiagonal<device_reduce_t, float>>(x.size(), x0.Precision(), y0.Precision()))
                               );

        max_tile_size = make_uint2(1, 1);

        strcpy(aux, nested_policy ? "nested_policy," : "policy,");
        strcat(aux, x[0].AuxString().c_str());
      	strcat(aux, ",");
        strcat(aux, y[0].AuxString().c_str());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        // FIXME this will break if the kernels are destructive - which they aren't here
        if (!tuned()) {
          if (!nested_policy) disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundant now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(1, std::min(NYW_max, (int)y.size()));
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else if (y.size() == 1) { // 1-d reduction

            max_tile_size = make_uint2(std::min((size_t)max_NXZ_power2(true), x.size()), 1);
            multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, max_tile_size);

          } else { // 2-d reduction

            // max_tile_size should be set to the largest power of 2,
            // since we have a requirement that the tile size is a
            // power of 2.
            // FIXME - we only do simple square tiling here
            max_tile_size = make_uint2(max_NXZ_power2(true), max_NXZ_power2(true));

            // Make sure constituents are tuned.
            for (unsigned int tile_size = 1;
                 tile_size <= max_tile_size.x && tile_size <= x.size() && (tile_size <= y.size() || y.size() == 1);
                 tile_size *= 2) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tile_size, tile_size));
            }

            // also test case using a single kernel if both dimensions are less than max
            if (is_valid_NXZ(x.size(), true) && y.size() <= (unsigned int)NYW_max) {
              multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(x.size(), y.size()));
            }
          }

          if (!nested_policy) enableProfileCount();
          setPolicyTuning(true);
        }

        apply(device::get_default_stream());
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &) override {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size.
        multiReduce_recurse<ReducerDiagonal, ReducerOffDiagonal>(result, x, y, z, w, 0, 0, hermitian, make_uint2(tp.aux.x, tp.aux.y));
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const override
      {
        // for 1-d reductions we don't do any tuning and just use the largest tile
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else { // 2-d reduction

          if ((unsigned int)(2 * param.aux.x) <= max_tile_size.x && (unsigned int)(2 * param.aux.y) <= max_tile_size.y
              && (unsigned int)(2 * param.aux.x) <= x.size() && (unsigned int)(2 * param.aux.y) <= y.size()) {
            // only tune powers of two
            param.aux.x *= 2;
            param.aux.y *= 2;
            return true;
          } else if (is_valid_NXZ(x.size(), true) && y.size() <= (size_t)NYW_max
                     && ((size_t)param.aux.x != x.size() || (size_t)param.aux.y != y.size())) {
            // we've run out of power of two tiles to try, but before
            // we finish, try a single kernel if it fits
            param.aux.x = x.size();
            param.aux.y = y.size();
            return true;
          } else {
            // reset to the beginning (which we'd need for multi-dimensional tuning)
            param.aux.x = 1;
            param.aux.y = 1;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const override { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const override {
        Tunable::initTuneParam(param);
        if (x.size() == 1 || y.size() == 1) {
          param.aux.x = max_tile_size.x;
          param.aux.y = max_tile_size.y;
        } else { // only do non-trivial tuning for 2-d reductions
          param.aux.x = 1;
          param.aux.y = 1;
        }
        param.aux.z = 0;
        param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const override {
        Tunable::defaultTuneParam(param); // default is max tile size
        param.aux.x = max_tile_size.x;
        param.aux.y = max_tile_size.y;
        param.aux.z = 0;
        param.aux.w = 0;
      }

      TuneKey tuneKey() const override {
        return TuneKey(x[0].VolString().c_str(), typeid(*this).name(), aux);
      }

      long long bytes() const override { return 0; } // FIXME

      void preTune() override { } // FIXME - use write to determine what needs to be saved
      void postTune() override { } // FIXME - use write to determine what needs to be saved
    };

    template <template <typename ...> class ReducerDiagonal,
              template <typename ...> class ReducerOffDiagonal, typename T, typename Vx, typename Vy>
    class TransposeTune : public Tunable
    {
      using TileTuner = TileSizeTune<ReducerDiagonal, ReducerOffDiagonal, T, Vx, Vy, Vx, Vx>;
      std::vector<T> &result;
      Vx &x;
      Vy &y;
      bool hermitian;
      bool Anorm;

    public:
      TransposeTune(std::vector<T> &result, Vx &x, Vy &y, bool hermitian, bool Anorm = false) :
        result(result),
        x(x),
        y(y),
        hermitian(hermitian),
        Anorm(Anorm)
      {
        strcpy(aux, "policy,");
        strcat(aux, x[0].AuxString().c_str());
        strcat(aux, ",");
        strcat(aux, y[0].AuxString().c_str());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
        strcat(aux, ",n=");
        char size[8];
        u64toa(size, x.size());
        strcat(aux, size);
        strcat(aux, ",m=");
        u64toa(size, y.size());
        strcat(aux, size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        char max_nyw_tile[8];
        u32toa(max_nyw_tile, max_n_batch_block_multi_reduce());
        strcat(aux, ",max_nyw_tile=");
        strcat(aux, max_nyw_tile);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        if (!tuned()) {
          disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          // note the 1-d tuning is all redundent now that we call
          // multiReduce_recurse directly now for 1-d multi
          // reductions, but I'll keep this code here for now
          if (x.size() == 1) {
            TileTuner tile(result, x, y, x, x, hermitian, Anorm, true);
          } else if (y.size() == 1) {
            TileTuner tile(result, y, x, y, y, hermitian, Anorm, true);
          } else {

            { // tune regular inner product
              TileTuner tile(result, x, y, x, x, hermitian, Anorm, true);
            }

            { // tune transpose inner product
              TileTuner tile(result, y, x, y, y, hermitian, Anorm, true);
            }
          }

          enableProfileCount();
          setPolicyTuning(true);
        }

        apply(device::get_default_stream());
      }

      virtual ~TransposeTune() { setPolicyTuning(false); }

      void apply(const qudaStream_t &) override
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (tp.aux.x == 0) {
          TileTuner(result, x, y, x, x, hermitian, Anorm, true);
        } else if (tp.aux.x == 1) {
          std::vector<T> result_trans(x.size() * y.size());

          // swap (x<->y and w<-z> when doing transpose calculation)
          TileTuner(result_trans, y, x, y, y, hermitian, Anorm, true);

          // tranpose the result if we are doing the transpose calculation
          const auto xlen = x.size();
          const auto ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++) result[i * xlen + j] = conj(result_trans[j * ylen + i]);
        } else {
          errorQuda("Unexpected transpose parameter %d", static_cast<int>(tp.aux.x));
        }
      }

      bool advanceAux(TuneParam &param) const override
      {
        if (x.size() == 1 || y.size() == 1) {
          return false;
        } else {
          if (param.aux.x == 0) {
            param.aux.x = 1;
            return true;
          } else {
            param.aux.x = 0;
            return false;
          }
        }
      }

      bool advanceTuneParam(TuneParam &param) const override { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const override
      {
        Tunable::initTuneParam(param);
        if (x.size() == 1)
          param.aux = make_int4(0, 0, 0, 0);
        else if (y.size() == 1)
          param.aux = make_int4(1, 0, 0, 0);
        else
          param.aux = make_int4(0, 0, 0, 0); // default is not to transpose
      }

      void defaultTuneParam(TuneParam &param) const override { initTuneParam(param); }

      TuneKey tuneKey() const override { return TuneKey(x[0].VolString().c_str(), typeid(*this).name(), aux); }

      long long bytes() const override { return 0; } // FIXME

      void preTune() override {}  // FIXME - use write to determine what needs to be saved
      void postTune() override {} // FIXME - use write to determine what needs to be saved
    };

    void reDotProduct(std::vector<double> &result, cvector_ref<const ColorSpinorField> &x,
                      cvector_ref<const ColorSpinorField> &y)
    {
      auto &x0 = x[0];
      auto &y0 = y[0];

      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      std::vector<double> result_tmp(x.size() * y.size(), 0.0);

      if (x.size() == 1) {
        auto NYW_max = y0.Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiDot<device_reduce_t, double>>(x.size(), x0.Precision(), y0.Precision()) :
          max_YW_size<multiDot<device_reduce_t, float>>(x.size(), x0.Precision(), y0.Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x0.Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<multiDot, multiDot>(result_tmp, x, y, x, x, 0, 0, false, max_tile_size);
      } else if (y.size() == 1 && x0.Precision() == y0.Precision()) {

        std::vector<double> result_trans(x.size() * y.size());

        // swap (x<->y and w<-z> when doing transpose calculation)
        auto NXZ_max = x0.Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiDot<device_reduce_t, double>>(y.size(), y0.Precision(), x0.Precision()) :
          max_YW_size<multiDot<device_reduce_t, float>>(y.size(), y0.Precision(), x0.Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x0.Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<multiDot, multiDot>(result_trans, y, x, y, y, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++)
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = result_trans[j * ylen + i];

      } else if (x0.Precision() == y0.Precision()) {
        TransposeTune<multiDot, multiDot, double, decltype(x), decltype(y)>(result_tmp, x, y, false);
      } else {
        TileSizeTune<multiDot, multiDot, double, decltype(x), decltype(y)>(result_tmp, x, y, x, x, false);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      comm_allreduce_sum(result_tmp);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      result = transpose(result_tmp, y.size(), x.size());
    }

    void cDotProduct(std::vector<Complex> &result, cvector_ref<const ColorSpinorField> &x,
                      cvector_ref<const ColorSpinorField> &y)
    {
      auto &x0 = x[0];
      auto &y0 = y[0];

      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      std::vector<Complex> result_tmp(x.size() * y.size(), 0.0);

      if (x.size() == 1) {
        auto NYW_max = y0.Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiCdot<device_reduce_t, double>>(x.size(), x0.Precision(), y0.Precision()) :
          max_YW_size<multiCdot<device_reduce_t, float>>(x.size(), x0.Precision(), y0.Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NYW_max, (int)y.size(), x0.Ncolor() == 3 ? 32 : NYW_max} ));
        multiReduce_recurse<multiCdot, multiCdot>(result_tmp, x, y, x, x, 0, 0, false, max_tile_size);
      } else if (y.size() == 1 && x0.Precision() == y0.Precision()) {

        std::vector<Complex> result_trans(x.size() * y.size());

        // swap (x<->y and w<-z> when doing transpose calculation)
        auto NXZ_max = x0.Precision() == QUDA_DOUBLE_PRECISION ?
          max_YW_size<multiCdot<device_reduce_t, double>>(y.size(), y0.Precision(), x0.Precision()) :
          max_YW_size<multiCdot<device_reduce_t, float>>(y.size(), y0.Precision(), x0.Precision());

        // if fine-grid then we set max tile size to 32 to avoid unnecessary tuning
        uint2 max_tile_size = make_uint2(1, std::min( {NXZ_max, (int)x.size(), x0.Ncolor() == 3 ? 32 : NXZ_max} ));
        multiReduce_recurse<multiCdot, multiCdot>(result_trans, y, x, y, y, 0, 0, false, max_tile_size);

        // transpose the result if we are doing the transpose calculation
        const auto xlen = x.size();
        const auto ylen = y.size();
        for (unsigned int j = 0; j < xlen; j++) {
          for (unsigned int i = 0; i < ylen; i++) result_tmp[i * xlen + j] = conj(result_trans[j * ylen + i]);
        }
      } else if (x0.Precision() == y0.Precision()) {
        TransposeTune<multiCdot, multiCdot, Complex, decltype(x), decltype(y)>(result_tmp, x, y, false);
      } else {
        TileSizeTune<multiCdot, multiCdot, Complex, decltype(x), decltype(y)>(result_tmp, x, y, x, x, false);
      }

      // do a single multi-node reduction only once we have computed all local dot products
      comm_allreduce_sum(result_tmp);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      result = transpose(result_tmp, y.size(), x.size());
    }

    void hDotProduct(std::vector<Complex> &result, cvector_ref<const ColorSpinorField> &x,
                     cvector_ref<const ColorSpinorField> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      std::vector<Complex> result_tmp(x.size() * y.size(), 0.0);
      TileSizeTune<multiCdot, multiCdot, Complex, decltype(x), decltype(y)>(result_tmp, x, y, x, x, true, false); // last false is b/c L2 norm

      // do a single multi-node reduction only once we have computed all local dot products
      comm_allreduce_sum(result_tmp); // FIXME - could optimize this for Hermiticity as well

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      result = transpose(result_tmp, y.size(), x.size());

      // we have only computed result on upper block trinagular part
      // so copy over to lower block tringaular
      for (auto i = 0u; i < x.size(); i++)
        for (auto j = 0u; j < i; j++)
          result[i * y.size() + j] = conj(result[j * x.size() + i]);
    }

    // for (p, Ap) norms in CG which are Hermitian.
    void hDotProduct_Anorm(std::vector<Complex> &result, cvector_ref<const ColorSpinorField> &x,
                     cvector_ref<const ColorSpinorField> &y)
    {
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      std::vector<Complex> result_tmp(x.size() * y.size(), 0.0);
      TileSizeTune<multiCdot, multiCdot, Complex, decltype(x), decltype(y)>(result_tmp, x, y, x, x, true, true); // last true is b/c A norm

      // do a single multi-node reduction only once we have computed all local dot products
      comm_allreduce_sum(result_tmp);

      // multiReduce_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      result = transpose(result_tmp, y.size(), x.size());

      // we have only computed result on upper block trinagular part
      // so copy over to lower block tringaular
      for (auto i = 0u; i < x.size(); i++)
        for (auto j = 0u; j < i; j++)
          result[i * y.size() + j] = conj(result[j * x.size() + i]);
    }

    void reDotProduct(double *result, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<double> result_(x.size() * y.size());
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<const ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      reDotProduct(result_, std::move(x_), std::move(y_));
      memcpy(result, result_.data(), x.size() * y.size() * sizeof(double));
    }

    void cDotProduct(Complex *result, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<Complex> result_(x.size() * y.size());
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<const ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      cDotProduct(result_, std::move(x_), std::move(y_));
      memcpy(result, result_.data(), x.size() * y.size() * sizeof(Complex));
    }

    void hDotProduct(Complex *result, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      std::vector<Complex> result_(x.size() * y.size());
      vector_ref<const ColorSpinorField> x_;
      for (auto &xi : x) x_.push_back(*xi);
      vector_ref<const ColorSpinorField> y_;
      for (auto &yi : y) y_.push_back(*yi);
      hDotProduct(result_, std::move(x_), std::move(y_));
      memcpy(result, result_.data(), x.size() * y.size() * sizeof(Complex));
    }

  } // namespace blas

} // namespace quda
