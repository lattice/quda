#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_domain_wall_4d.cuh>

/**
   This is the gauged domain-wall 4-d preconditioned operator.

   Note, for now, this just applies a batched 4-d dslash across the fifth
   dimension.
*/

namespace quda
{

  template <typename Arg> class DomainWall4D : public Dslash<domainWall4D, Arg>
  {
    using Dslash = Dslash<domainWall4D, Arg>;
    using Dslash::arg;
    using Dslash::in;

  public:
    DomainWall4D(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
    {
      TunableVectorYZ::resizeVector(in.X(4), arg.nParity);
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);
      typedef typename mapper<typename Arg::Float>::type real;
#ifdef JITIFY
      // we need to break the dslash launch abstraction here to get a handle on the constant memory pointer in the kernel module
      auto instance = Dslash::template kernel_instance<packShmem>();
      hipMemcpyHtoDAsync(instance.get_constant_ptr("quda::mobius_d"), arg.a_5, QUDA_MAX_DWF_LS * sizeof(complex<real>),
                        stream);
      Tunable::jitify_error = instance.configure(tp.grid, tp.block, tp.shared_bytes, stream).launch(arg);
#else
      hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_d), arg.a_5, QUDA_MAX_DWF_LS * sizeof(complex<real>), 0, hipMemcpyHostToDevice,
                              streams[Nstream - 1]);
      Dslash::template instantiate<packShmem>(tp, stream);
#endif
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct DomainWall4DApply {

    inline DomainWall4DApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
                             double m_5, const Complex *b_5, const Complex *c_5, const ColorSpinorField &x, int parity,
                             bool dagger, const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      DomainWall4DArg<Float, nColor, nDim, recon> arg(out, in, U, a, m_5, b_5, c_5, a != 0.0, x, parity, dagger,
                                                      comm_override);
      DomainWall4D<decltype(arg)> dwf(arg, out, in);

      dslash::DslashPolicyTune<decltype(dwf)> policy(
        dwf, const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)),
        in.getDslashConstant().volume_4d_cb, in.getDslashConstant().ghostFaceCB, profile);
      policy.apply(0);

      checkCudaError();
    }
  };

  // Apply the 4-d preconditioned domain-wall Dslash operator
  // out(x) = M*in = in(x) + a*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  void ApplyDomainWall4D(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a, double m_5,
                         const Complex *b_5, const Complex *c_5, const ColorSpinorField &x, int parity, bool dagger,
                         const int *comm_override, TimeProfile &profile)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    instantiate<DomainWall4DApply>(out, in, U, a, m_5, b_5, c_5, x, parity, dagger, comm_override, profile);
#else
    errorQuda("Domain-wall dslash has not been built");
#endif // GPU_DOMAIN_WALL_DIRAC
  }

} // namespace quda
