#include "hip/hip_runtime.h"
#include <clover_field.h>
#include <instantiate.h>
#include <tunable_reduction.h>
#include <kernels/clover_invert.cuh>

namespace quda {

  template <typename store_t>
  class CloverInvert : TunableReduction2D {
    CloverField &clover;
    bool compute_tr_log;

  public:
    CloverInvert(CloverField &clover, bool compute_tr_log) :
      TunableReduction2D(clover),
      clover(clover),
      compute_tr_log(compute_tr_log)
    {
      strcat(aux, compute_tr_log ? ",trlog=true" : "trlog=false");
      strcat(aux, clover.TwistFlavor() == QUDA_TWIST_SINGLET || clover.TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET ?
             ",twist=true" : ",twist=false");
      apply(device::get_default_stream());

      if (compute_tr_log && (std::isnan(clover.TrLog()[0]) || std::isnan(clover.TrLog()[1]))) {
	printfQuda("clover.TrLog()[0]=%e, clover.TrLog()[1]=%e\n", clover.TrLog()[0], clover.TrLog()[1]);
	errorQuda("Clover trlog has returned -nan, likey due to the clover matrix being singular.");
      }
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (clover.TwistFlavor() == QUDA_TWIST_SINGLET ||
          clover.TwistFlavor() == QUDA_TWIST_NONDEG_DOUBLET) {
        CloverInvertArg<store_t, true> arg(clover, compute_tr_log);
        launch<InvertClover>(clover.TrLog(), tp, stream, arg);
      } else {
        CloverInvertArg<store_t, false> arg(clover, compute_tr_log);
        launch<InvertClover>(clover.TrLog(), tp, stream, arg);
      }
    }
    
    long long flops() const { return 0; }
    long long bytes() const { return 2 * clover.Bytes(); }
    void preTune() { if (clover::dynamic_inverse()) clover.backup(); }
    void postTune() { if (clover::dynamic_inverse()) clover.restore(); }
  };

#ifdef GPU_CLOVER_DIRAC
  void cloverInvert(CloverField &clover, bool computeTraceLog)
  {
    if (clover.Reconstruct()) errorQuda("Cannot store the inverse with a reconstruct field");
    if (clover.Precision() < QUDA_SINGLE_PRECISION) errorQuda("Cannot use fixed-point precision here");
    instantiate<CloverInvert>(clover, computeTraceLog);
  }
#else
  void cloverInvert(CloverField &, bool)
  {
    errorQuda("Clover has not been built");
  }
#endif

} // namespace quda
