#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>
#include <float_vector.h>
#include <complex_quda.h>
#include <instantiate.h>

namespace quda {

  template <typename Float_, int nColor_, QudaReconstructType recon_u, QudaReconstructType recon_m, int N_>
  struct UpdateGaugeArg {
    using Float = Float_;
    static constexpr int nColor = nColor_;
    static constexpr int N = N_;
    static_assert(nColor == 3, "Only nColor=3 enabled at this time");
    typedef typename gauge_mapper<Float,recon_u>::type Gauge;
    typedef typename gauge_mapper<Float,recon_m>::type Mom;
    Gauge out;
    Gauge in;
    Mom mom;
    Float dt;
    int nDim;
    UpdateGaugeArg(GaugeField &out, const GaugeField &in, const GaugeField &mom, Float dt, int nDim)
      : out(out), in(in), mom(mom), dt(dt), nDim(nDim) { }
  };

  template <bool conj_mom, bool exact, typename Arg>
  __device__ __host__  void compute(Arg &arg, int x, int parity)
  {
    using Float = typename Arg::Float;
    typedef complex<Float> Complex;
    Matrix<Complex, Arg::nColor> link, result, mom;

    for (int dir=0; dir<arg.nDim; ++dir) {
      link = arg.in(dir, x, parity);
      mom = arg.mom(dir, x, parity);

      Complex trace = getTrace(mom);
      for (int c=0; c<Arg::nColor; c++) mom(c,c) -= trace/static_cast<Float>(Arg::nColor);

      if (!exact) {
	result = link;

	// Nth order expansion of exponential
	if (!conj_mom) {
	  for (int r= Arg::N; r>0; r--)
	    result = (arg.dt/r)*mom*result + link;
	} else {
	  for (int r= Arg::N; r>0; r--)
	    result = (arg.dt/r)*conj(mom)*result + link;
	}
      } else {
	mom = arg.dt * mom;
        expsu3<Float>(mom);

        if (!conj_mom) {
          link = mom * link;
        } else {
          link = conj(mom) * link;
        }

        result = link;
      }

      arg.out(dir, x, parity) = result;
    } // dir
  }

  template <bool conj_mom, bool exact, typename Arg>
  __global__ void updateGaugeFieldKernel(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    if (x_cb >= arg.out.volumeCB) return;
    int parity = blockIdx.y*blockDim.y + threadIdx.y;
    compute<conj_mom,exact>(arg, x_cb, parity);
  }

  template <typename Arg, bool conj_mom, bool exact>
   class UpdateGaugeField : public TunableVectorY {
    Arg &arg;
    const GaugeField &meta; // meta data

    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.in.volumeCB; }

  public:
    UpdateGaugeField(Arg &arg, const GaugeField &meta) :
      TunableVectorY(2),
      arg(arg),
      meta(meta) {}

    void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      updateGaugeFieldKernel<conj_mom,exact><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
    } // apply

    long long flops() const {
      const int Nc = Arg::nColor;
      return arg.nDim*2*arg.in.volumeCB*Arg::N*(Nc*Nc*2 +                 // scalar-matrix multiply
                                                (8*Nc*Nc*Nc - 2*Nc*Nc) +  // matrix-matrix multiply
                                                Nc*Nc*2);                 // matrix-matrix addition
    }

    long long bytes() const { return arg.nDim*2*arg.in.volumeCB*(arg.in.Bytes() + arg.out.Bytes() + arg.mom.Bytes()); }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }
  };

  template <typename Float, int nColor, QudaReconstructType recon_u> struct UpdateGauge
  {
    UpdateGauge(GaugeField &out, const GaugeField &in, const GaugeField &mom, double dt, bool conj_mom, bool exact)
    {
      if (mom.Reconstruct() != QUDA_RECONSTRUCT_10) errorQuda("Reconstruction type %d not supported", mom.Reconstruct());
      constexpr QudaReconstructType recon_m = QUDA_RECONSTRUCT_10;
      constexpr int N = 8; // degree of exponential expansion
      UpdateGaugeArg<Float, nColor, recon_u, recon_m, N> arg(out, in, mom, dt, 4);
      if (conj_mom) {
        if (exact) {
          UpdateGaugeField<decltype(arg),true,true> updateGauge(arg, in);
          updateGauge.apply(0);
        } else {
          UpdateGaugeField<decltype(arg),true,false> updateGauge(arg, in);
          updateGauge.apply(0);
        }
      } else {
        if (exact) {
          UpdateGaugeField<decltype(arg),false,true> updateGauge(arg, in);
          updateGauge.apply(0);
        } else {
          UpdateGaugeField<decltype(arg),false,false> updateGauge(arg, in);
          updateGauge.apply(0);
        }
      }
      checkCudaError();
    }
  };

  void updateGaugeField(GaugeField &out, double dt, const GaugeField& in, const GaugeField& mom, bool conj_mom, bool exact)
  {
#ifdef GPU_GAUGE_TOOLS
    checkPrecision(out, in, mom);
    checkLocation(out, in, mom);
    checkReconstruct(out, in);
    instantiate<UpdateGauge,ReconstructNo12>(out, in, mom, dt, conj_mom, exact);
#else
    errorQuda("Gauge tools are not build");
#endif
  }

} // namespace quda
