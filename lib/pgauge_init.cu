#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <comm_quda.h>
#include <unitarization_links.h>
#include <pgauge_monte.h>
#include <random_quda.h>
#include <cub_helper.cuh>
#include <index_helper.cuh>

#ifndef PI
#define PI    3.1415926535897932384626433832795    // pi
#endif
#ifndef PII
#define PII   6.2831853071795864769252867665590    // 2 * pi
#endif

namespace quda {

#ifdef GPU_GAUGE_ALG

  template <typename Gauge>
  struct InitGaugeColdArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    Gauge dataOr;
    InitGaugeColdArg(const Gauge &dataOr, const cudaGaugeField &data)
      : dataOr(dataOr) {
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      threads = X[0] * X[1] * X[2] * X[3];
    }
  };

  template<typename Float, typename Gauge, int NCOLORS>
  __global__ void compute_InitGauge_ColdStart(InitGaugeColdArg<Gauge> arg){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx >= arg.threads ) return;
    int parity = 0;
    if ( idx >= arg.threads / 2 ) {
      parity = 1;
      idx -= arg.threads / 2;
    }
    Matrix<complex<Float>,NCOLORS> U;
    setIdentity(&U);
    for ( int d = 0; d < 4; d++ ) arg.dataOr(d, idx, parity) = U;
  }


  template<typename Float, typename Gauge, int NCOLORS>
  class InitGaugeCold : Tunable {
    InitGaugeColdArg<Gauge> arg;
    mutable char aux_string[128]; // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false; }  // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    InitGaugeCold(InitGaugeColdArg<Gauge> &arg)
      : arg(arg) {
    }
    ~InitGaugeCold () {
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      compute_InitGauge_ColdStart<Float, Gauge, NCOLORS> <<< tp.grid,tp.block >>> (arg);
      //hipDeviceSynchronize();
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    long long flops() const {
      return 0;
    }                                  // Only correct if there is no link reconstruction, no cub reduction accounted also
    long long bytes() const {
      return 0;
    }                                  //no accounting the reduction!!!!

  };

  template<typename Float, int NCOLORS, typename Gauge>
  void InitGaugeField( Gauge dataOr,  cudaGaugeField& data) {
    InitGaugeColdArg<Gauge> initarg(dataOr, data);
    InitGaugeCold<Float, Gauge, NCOLORS> init(initarg);
    init.apply(0);
    checkCudaError();
  }



  template<typename Float>
  void InitGaugeField( cudaGaugeField& data) {

    if ( data.isNative() ) {
      if ( data.Reconstruct() == QUDA_RECONSTRUCT_NO ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_12 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_8 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data);
      } else {
        errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }
    } else {
      errorQuda("Invalid Gauge Order\n");
    }

  }

/** @brief Perform a cold start to the gauge field, identity SU(3) matrix, also fills the ghost links in multi-GPU case (no need to exchange data)
 *
 * @param[in,out] data Gauge field
 */
  void InitGaugeField( cudaGaugeField& data) {

    if ( data.Precision() == QUDA_SINGLE_PRECISION ) {
      InitGaugeField<float> (data);
    } else if ( data.Precision() == QUDA_DOUBLE_PRECISION ) {
      InitGaugeField<double>(data);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
    
  }








  template <typename Gauge>
  struct InitGaugeHotArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    RNG rngstate;
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    InitGaugeHotArg(const Gauge &dataOr, const cudaGaugeField &data, RNG &rngstate)
      : dataOr(dataOr), rngstate(rngstate) {
#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir] * 2;
      } 
#else
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
#endif
      //the optimal number of RNG states in rngstate array must be equal to half the lattice volume
      //this number is the same used in heatbath...
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
    }
  };


  template <typename Float>
  __host__ __device__ static inline void reunit_link( Matrix<complex<Float>,3> &U ){

    complex<Float> t2((Float)0.0, (Float)0.0);
    Float t1 = 0.0;
    //first normalize first row
    //sum of squares of row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(0,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(0,c) *= t1;
    //6
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t2 += conj(U(0,c)) * U(1,c);
    //24
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1,c) -= t2 * U(0,c);
    //24
    //normalize second row
    //sum of squares of row
    t1 = 0.0;
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(1,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1, c) *= t1;
    //6
    //Reconstruct lat row
    U(2,0) = conj(U(0,1) * U(1,2) - U(0,2) * U(1,1));
    U(2,1) = conj(U(0,2) * U(1,0) - U(0,0) * U(1,2));
    U(2,2) = conj(U(0,0) * U(1,1) - U(0,1) * U(1,0));
    //42
    //T=130
  }






/**
    @brief Generate the four random real elements of the SU(2) matrix
    @param localstate CURAND rng state
    @return four real numbers of the SU(2) matrix
 */
  template <class T>
  __device__ static inline Matrix<T,2> randomSU2(cuRNGState& localState){
    Matrix<T,2> a;
    T aabs, ctheta, stheta, phi;
    a(0,0) = Random<T>(localState, (T)-1.0, (T)1.0);
    aabs = sqrt( 1.0 - a(0,0) * a(0,0));
    ctheta = Random<T>(localState, (T)-1.0, (T)1.0);
    phi = PII * Random<T>(localState);
    stheta = ( hiprand(&localState) & 1 ? 1 : -1 ) * sqrt( (T)1.0 - ctheta * ctheta );
    a(0,1) = aabs * stheta * cos( phi );
    a(1,0) = aabs * stheta * sin( phi );
    a(1,1) = aabs * ctheta;
    return a;
  }


/**
    @brief Update the SU(Nc) link with the new SU(2) matrix, link <- u * link
    @param u SU(2) matrix represented by four real numbers
    @param link SU(Nc) matrix
    @param id indices
 */
  template <class T, int NCOLORS>
  __host__ __device__ static inline void mul_block_sun( Matrix<T,2> u, Matrix<complex<T>,NCOLORS> &link, int2 id ){
    for ( int j = 0; j < NCOLORS; j++ ) {
      complex<T> tmp = complex<T>( u(0,0), u(1,1) ) * link(id.x, j) + complex<T>( u(1,0), u(0,1) ) * link(id.y, j);
      link(id.y, j) = complex<T>(-u(1,0), u(0,1) ) * link(id.x, j) + complex<T>( u(0,0),-u(1,1) ) * link(id.y, j);
      link(id.x, j) = tmp;
    }
  }


/**
    @brief Calculate the SU(2) index block in the SU(Nc) matrix
    @param block number to calculate the index's, the total number of blocks is NCOLORS * ( NCOLORS - 1) / 2.
    @return Returns two index's in int2 type, accessed by .x and .y.
 */
  template<int NCOLORS>
  __host__ __device__ static inline int2 IndexBlock(int block){
    int2 id;
    int i1;
    int found = 0;
    int del_i = 0;
    int index = -1;
    while ( del_i < (NCOLORS - 1) && found == 0 ) {
      del_i++;
      for ( i1 = 0; i1 < (NCOLORS - del_i); i1++ ) {
        index++;
        if ( index == block ) {
          found = 1;
          break;
        }
      }
    }
    id.y = i1 + del_i;
    id.x = i1;
    return id;
  }

/**
    @brief Generate a SU(Nc) random matrix
    @param localstate CURAND rng state
    @return SU(Nc) matrix
 */
  template <class Float, int NCOLORS>
  __device__ inline Matrix<complex<Float>,NCOLORS> randomize( cuRNGState& localState ){
    Matrix<complex<Float>,NCOLORS> U;

    for ( int i = 0; i < NCOLORS; i++ )
      for ( int j = 0; j < NCOLORS; j++ )
        U(i,j) = complex<Float>( (Float)(Random<Float>(localState) - 0.5), (Float)(Random<Float>(localState) - 0.5) );
    reunit_link<Float>(U);
    return U;

    /*setIdentity(&U);
       for( int block = 0; block < NCOLORS * ( NCOLORS - 1) / 2; block++ ) {
       Matrix<Float,2> rr = randomSU2<Float>(localState);
       int2 id = IndexBlock<NCOLORS>( block );
       mul_block_sun<Float, NCOLORS>(rr, U, id);
       //U = block_su2_to_su3<Float>( U, a00, a01, a10, a11, block );
       }
       return U;*/
  }

  template<typename Float, typename Gauge, int NCOLORS>
  __global__ void compute_InitGauge_HotStart(InitGaugeHotArg<Gauge> arg){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx >= arg.threads ) return;
  #ifdef MULTI_GPU
    int X[4], x[4];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] += 2 * arg.border[dr];
    int id = idx;
    cuRNGState localState = arg.rngstate.State()[ id ];
  #else
    cuRNGState localState = arg.rngstate.State()[ idx ];
  #endif
    for ( int parity = 0; parity < 2; parity++ ) {
    #ifdef MULTI_GPU
      getCoords(x, id, arg.X, parity);
      for ( int dr = 0; dr < 4; ++dr ) x[dr] += arg.border[dr];
      idx = linkIndex(x,X);
    #endif
      for ( int d = 0; d < 4; d++ ) {
        Matrix<complex<Float>,NCOLORS> U;
        U = randomize<Float, NCOLORS>(localState);
        arg.dataOr(d, idx, parity) = U;
      }
    }
  #ifdef MULTI_GPU
    arg.rngstate.State()[ id ] = localState;
  #else
    arg.rngstate.State()[ idx ] = localState;
  #endif
  }




  template<typename Float, typename Gauge, int NCOLORS>
  class InitGaugeHot : Tunable {
    InitGaugeHotArg<Gauge> arg;
    mutable char aux_string[128]; // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    bool tuneSharedBytes() const {
      return false;
    }                                            // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    InitGaugeHot(InitGaugeHotArg<Gauge> &arg)
      : arg(arg) {
    }
    ~InitGaugeHot () {
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      compute_InitGauge_HotStart<Float, Gauge, NCOLORS> <<< tp.grid,tp.block >>> (arg);
      //hipDeviceSynchronize();
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lud", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    void preTune(){ arg.rngstate.backup(); }
    void postTune(){ arg.rngstate.restore(); }
    long long flops() const {
      return 0;
    }                                  // Only correct if there is no link reconstruction, no cub reduction accounted also
    long long bytes() const {
      return 0;
    }                                  //no accounting the reduction!!!!

  };


  template<typename Float, int NCOLORS, typename Gauge>
  void InitGaugeField( Gauge dataOr,  cudaGaugeField& data, RNG &rngstate) {
    InitGaugeHotArg<Gauge> initarg(dataOr, data, rngstate);
    InitGaugeHot<Float, Gauge, NCOLORS> init(initarg);
    init.apply(0);
    checkCudaError();
    qudaDeviceSynchronize();

    data.exchangeExtendedGhost(data.R(),false);
  }

  template<typename Float>
  void InitGaugeField( cudaGaugeField& data, RNG &rngstate) {

    if ( data.isNative() ) {
      if ( data.Reconstruct() == QUDA_RECONSTRUCT_NO ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data, rngstate);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_12 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data, rngstate);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_8 ) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
        InitGaugeField<Float, 3>(Gauge(data), data, rngstate);
      } else {
        errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }
    } else {
      errorQuda("Invalid Gauge Order\n");
    }
  }
#endif // GPU_GAUGE_ALG

/** @brief Perform a hot start to the gauge field, random SU(3) matrix, followed by reunitarization, also exchange borders links in multi-GPU case.
 *
 * @param[in,out] data Gauge field
 * @param[in,out] rngstate state of the CURAND random number generator
 */
  void InitGaugeField( cudaGaugeField& data, RNG &rngstate) {
#ifdef GPU_GAUGE_ALG
    if ( data.Precision() == QUDA_SINGLE_PRECISION ) {
      InitGaugeField<float> (data, rngstate);
    } else if ( data.Precision() == QUDA_DOUBLE_PRECISION ) {
      InitGaugeField<double>(data, rngstate);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
#else
    errorQuda("Pure gauge code has not been built");
#endif
  }
}
