#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>

#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>

#define PRESERVE_SPINOR_NORM

#ifdef PRESERVE_SPINOR_NORM // Preserve the norm regardless of basis
#define kP (1.0/sqrt(2.0))
#define kU (1.0/sqrt(2.0))
#else // More numerically accurate not to preserve the norm between basis
#define kP (0.5)
#define kU (1.0)
#endif



namespace quda {

  using namespace colorspinor;
  
  void exchangeExtendedGhost(cudaColorSpinorField* spinor, int R[], int parity, qudaStream_t *stream_p)
  {
#ifdef MULTI_GPU
    int nFace = 0;
    for(int i=0; i<4; i++){
      if(R[i] > nFace) nFace = R[i];
    }

    int dagger = 0;

    int gatherCompleted[2] = {0,0};
    int commsCompleted[2] = {0,0};

    hipEvent_t gatherEnd[2];
    for(int dir=0; dir<2; dir++) hipEventCreate(&gatherEnd[dir], hipEventDisableTiming);

    for(int dim=3; dim<=0; dim--){
      if(!commDim(dim)) continue;

      spinor->packExtended(nFace, R, parity, dagger, dim, stream_p); // packing in the dim dimension complete
      qudaDeviceSynchronize(); // Need this since packing is performed in stream[Nstream-1]
      for(int dir=1; dir<=0; dir--){
        spinor->gather(nFace, dagger, 2*dim + dir);
        qudaEventRecord(gatherEnd[dir], streams[2*dim+dir]); // gatherEnd[1], gatherEnd[0]
      } 

      int completeSum = 0;
      int dir = 1;
      while(completeSum < 2){
        if(!gatherCompleted[dir]){
          if(hipSuccess == hipEventQuery(gatherEnd[dir])){
            spinor->commsStart(nFace, 2*dim+dir, dagger);
            completeSum++;
            gatherCompleted[dir--] = 1;
          }
        }
      }
      gatherCompleted[0] = gatherCompleted[1] = 0;

      // Query if comms has completed
      dir = 1;
      while(completeSum < 4){
        if(!commsCompleted[dir]){
          if(spinor->commsQuery(nFace, 2*dim+dir, dagger)){
            spinor->scatterExtended(nFace, parity, dagger, 2*dim+dir);
            completeSum++;
            commsCompleted[dir--] = 1;
          }
        }
      } 
      commsCompleted[0] = commsCompleted[1] = 0;
      qudaDeviceSynchronize(); // Wait for scatters to complete before next iteration
    } // loop over dim

    for(int dir=0; dir<2; dir++) hipEventDestroy(gatherEnd[dir]);
#endif
    return;
  }


  /** Straight copy with no basis change */
  template <typename FloatOut, typename FloatIn, int Ns, int Nc>
    class PreserveBasis {
      typedef typename mapper<FloatIn>::type RegTypeIn;
      typedef typename mapper<FloatOut>::type RegTypeOut;
      public:
      __device__ __host__ inline void operator()(ColorSpinor<RegTypeOut,Nc,Ns> &out, const ColorSpinor<RegTypeIn,Nc,Ns> &in) {
        for (int s=0; s<Ns; s++) {
          for (int c=0; c<Nc; c++) {
            out(s,c) = in(s,c);
          }
        }
      }
    };

  /** Transform from relativistic into non-relavisitic basis */
  template <typename FloatOut, typename FloatIn, int Ns, int Nc>
    struct NonRelBasis {
      typedef typename mapper<FloatIn>::type RegTypeIn;
      typedef typename mapper<FloatOut>::type RegTypeOut;
      __device__ __host__ inline void operator()(ColorSpinor<RegTypeOut,Nc,Ns> &out, const ColorSpinor<RegTypeIn,Nc,Ns> &in) {
        int s1[4] = {1, 2, 3, 0};
        int s2[4] = {3, 0, 1, 2};
        RegTypeOut K1[4] = {static_cast<RegTypeOut>(kP), static_cast<RegTypeOut>(-kP),
			    static_cast<RegTypeOut>(-kP), static_cast<RegTypeOut>(-kP)};
        RegTypeOut K2[4] = {static_cast<RegTypeOut>(kP), static_cast<RegTypeOut>(-kP),
			    static_cast<RegTypeOut>(kP), static_cast<RegTypeOut>(kP)};
        for (int s=0; s<Ns; s++) {
          for (int c=0; c<Nc; c++) {
            out(s,c).real(K1[s]*in(s1[s],c).real() + K2[s]*in(s2[s],c).real());
            out(s,c).imag(K1[s]*in(s1[s],c).imag() + K2[s]*in(s2[s],c).imag());
          }
        }
      }
    };

  /** Transform from non-relativistic into relavisitic basis */
  template <typename FloatOut, typename FloatIn, int Ns, int Nc>
    struct RelBasis {
      typedef typename mapper<FloatIn>::type RegTypeIn;
      typedef typename mapper<FloatOut>::type RegTypeOut;
      __device__ __host__ inline void operator()(ColorSpinor<RegTypeOut,Nc,Ns> &out, const ColorSpinor<RegTypeIn,Nc,Ns> &in) {
        int s1[4] = {1, 2, 3, 0};
        int s2[4] = {3, 0, 1, 2};
        RegTypeOut K1[4] = {static_cast<RegTypeOut>(-kU), static_cast<RegTypeOut>(kU),
			    static_cast<RegTypeOut>(kU), static_cast<RegTypeOut>(kU)};
        RegTypeOut K2[4] = {static_cast<RegTypeOut>(-kU), static_cast<RegTypeOut>(kU),
			    static_cast<RegTypeOut>(-kU), static_cast<RegTypeOut>(-kU)};
        for (int s=0; s<Ns; s++) {
          for (int c=0; c<Nc; c++) {
            out(s,c).real(K1[s]*in(s1[s],c).real() + K2[s]*in(s2[s],c).real());
            out(s,c).imag(K1[s]*in(s1[s],c).imag() + K2[s]*in(s2[s],c).imag());
          }
        }
      }
    };

  template<typename OutOrder, typename InOrder, typename Basis>
    struct CopySpinorExArg{
      OutOrder out;
      const InOrder in;
      Basis basis;
      int E[QUDA_MAX_DIM];
      int X[QUDA_MAX_DIM];
      int length;
      int parity;

      CopySpinorExArg(const OutOrder &out, const InOrder &in, const Basis& basis, const int *E, const int *X, const int parity)
        : out(out), in(in), basis(basis), parity(parity) 
      {
        this->length = 1;
        for(int d=0; d<4; d++){
          this->E[d] = E[d];
          this->X[d] = X[d];
          this->length *= X[d]; // smaller volume
        }
      }
    };


  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder, typename Basis, bool extend>
    __device__ __host__ void copyInterior(CopySpinorExArg<OutOrder,InOrder,Basis>& arg, int X)
    {
      int x[4];
      int R[4];
      for(int d=0; d<4; d++) R[d] = (arg.E[d] - arg.X[d]) >> 1;

      int za = X/(arg.X[0]/2);
      int x0h = X - za*(arg.X[0]/2);
      int zb = za/arg.X[1];
      x[1] = za - zb*arg.X[1];
      x[3] = zb / arg.X[2];
      x[2] = zb - x[3]*arg.X[2];
      x[0] = 2*x0h + ((x[1] + x[2] + x[3] + arg.parity) & 1);

      // Y is the cb spatial index into the extended gauge field
      int Y = ((((x[3]+R[3])*arg.E[2] + (x[2]+R[2]))*arg.E[1] + (x[1]+R[1]))*arg.E[0]+(x[0]+R[0])) >> 1;

      typedef typename mapper<FloatIn>::type RegTypeIn;
      typedef typename mapper<FloatOut>::type RegTypeOut;

      ColorSpinor<RegTypeIn,Nc,Ns> in;
      ColorSpinor<RegTypeOut,Nc,Ns> out;
      int parity = 0;

      if(extend){
        in = arg.in(X, parity);
        arg.basis(out, in);
        arg.out(Y, parity) = out;
      }else{
        in = arg.in(Y, parity);
        arg.basis(out, in);
        arg.out(Y, parity) = out;
      }
    }


  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder, typename Basis, bool extend>
    __global__ void copyInteriorKernel(CopySpinorExArg<OutOrder,InOrder,Basis> arg)
    {
      int cb_idx = blockIdx.x*blockDim.x + threadIdx.x;

      while(cb_idx < arg.length){
        copyInterior<FloatOut,FloatIn,Ns,Nc,OutOrder,InOrder,Basis,extend>(arg,cb_idx);
        cb_idx += gridDim.x*blockDim.x;
      }
    }

  /*
     Host function
   */
  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder, typename Basis, bool extend>
    void copyInterior(CopySpinorExArg<OutOrder,InOrder,Basis>& arg)
    {
      for(int cb_idx=0; cb_idx<arg.length; cb_idx++){
        copyInterior<FloatOut,FloatIn,Ns,Nc,OutOrder,InOrder,Basis,extend>(arg, cb_idx);
      }
    }




  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder, typename Basis, bool extend>
    class CopySpinorEx : Tunable {

      CopySpinorExArg<OutOrder,InOrder,Basis> arg;
      const ColorSpinorField &meta;
      QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
      bool advanceSharedBytes(TuneParam &param) const { return false; } // Don't tune shared mem
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.length; }

      public: 
      CopySpinorEx(CopySpinorExArg<OutOrder,InOrder,Basis> &arg, const ColorSpinorField &meta, QudaFieldLocation location)
        : arg(arg), meta(meta), location(location) {
	writeAuxString("out_stride=%d,in_stride=%d",arg.out.stride,arg.in.stride);
      }
      virtual ~CopySpinorEx() {}

      void apply(const qudaStream_t &stream){
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if(location == QUDA_CPU_FIELD_LOCATION){
          copyInterior<FloatOut,FloatIn,Ns,Nc,OutOrder,InOrder,Basis,extend>(arg);    
        }else if(location == QUDA_CUDA_FIELD_LOCATION){
          copyInteriorKernel<FloatOut,FloatIn,Ns,Nc,OutOrder,InOrder,Basis,extend>
            <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);    
        }
      } 

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

      long long flops() const { return 0; }
      long long bytes() const {
        return arg.length*2*Nc*Ns*(sizeof(FloatIn) + sizeof(FloatOut));
      }

    }; // CopySpinorEx



  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder, typename Basis>
    void copySpinorEx(OutOrder outOrder, const InOrder inOrder, const Basis basis, const int *E, 
		      const int *X, const int parity, const bool extend, const ColorSpinorField &meta, QudaFieldLocation location)
    {
      CopySpinorExArg<OutOrder,InOrder,Basis> arg(outOrder, inOrder, basis, E, X, parity);
      if(extend){
        CopySpinorEx<FloatOut, FloatIn, Ns, Nc, OutOrder, InOrder, Basis, true> copier(arg, meta, location);
        copier.apply(0);
      }else{
        CopySpinorEx<FloatOut, FloatIn, Ns, Nc, OutOrder, InOrder, Basis, false> copier(arg, meta, location);
        copier.apply(0);
      }
      if(location == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
    }

  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename OutOrder, typename InOrder>
    void copySpinorEx(OutOrder outOrder, InOrder inOrder, const QudaGammaBasis outBasis, const QudaGammaBasis inBasis,
		      const int* E, const int* X, const int parity, const bool extend, 
		      const ColorSpinorField &meta, QudaFieldLocation location)
    {
      if(inBasis == outBasis){
        PreserveBasis<FloatOut,FloatIn,Ns,Nc> basis;
        copySpinorEx<FloatOut, FloatIn, Ns, Nc, OutOrder, InOrder, PreserveBasis<FloatOut,FloatIn,Ns,Nc> >
          (outOrder, inOrder, basis, E, X, parity, extend, meta, location);
      }else if(outBasis == QUDA_UKQCD_GAMMA_BASIS && inBasis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS){
        if(Ns != 4) errorQuda("Can only change basis with Nspin = 4, not Nspin = %d", Ns);
        NonRelBasis<FloatOut,FloatIn,4,Nc> basis;
        copySpinorEx<FloatOut, FloatIn, 4, Nc, OutOrder, InOrder, NonRelBasis<FloatOut,FloatIn,4,Nc> >
          (outOrder, inOrder, basis, E, X, parity, extend, meta, location);
      }else if(inBasis == QUDA_UKQCD_GAMMA_BASIS && outBasis == QUDA_DEGRAND_ROSSI_GAMMA_BASIS){
        if(Ns != 4) errorQuda("Can only change basis with Nspin = 4, not Nspin = %d", Ns);
        RelBasis<FloatOut,FloatIn,4,Nc> basis;
        copySpinorEx<FloatOut, FloatIn, 4, Nc, OutOrder, InOrder, RelBasis<FloatOut,FloatIn,4,Nc> >
          (outOrder, inOrder, basis, E, X, parity, extend, meta, location);
      }else{
        errorQuda("Basis change not supported");
      }
    }


  // Need to rewrite the following two functions...
  // Decide on the output order 
  template<typename FloatOut, typename FloatIn, int Ns, int Nc, typename InOrder>
    void extendedCopyColorSpinor(InOrder &inOrder, ColorSpinorField &out,
        QudaGammaBasis inBasis, const int *E, const int *X,  const int parity, const bool extend,
        QudaFieldLocation location, FloatOut *Out, float *outNorm){

    if (out.isNative()) {
      typedef typename colorspinor_mapper<FloatOut,Ns,Nc>::type ColorSpinor;
      ColorSpinor outOrder(out, 1, Out, outNorm);
      copySpinorEx<FloatOut,FloatIn,Ns,Nc>
	(outOrder, inOrder, out.GammaBasis(), inBasis, E, X, parity, extend, out, location);
    } else {
      errorQuda("Order not defined");
    }

  }

  template<typename FloatOut, typename FloatIn, int Ns, int Nc> 
    void extendedCopyColorSpinor(ColorSpinorField &out, const ColorSpinorField &in, 
        const int parity, const QudaFieldLocation location, FloatOut *Out, FloatIn *In, 
        float* outNorm, float *inNorm){

    int E[4];
    int X[4];
    const bool extend = (out.Volume() >= in.Volume());
    if (extend) {
      for (int d=0; d<4; d++) {
	E[d] = out.X()[d];
	X[d] = in.X()[d];
      }
    } else {
      for (int d=0; d<4; d++) {
	E[d] = in.X()[d];
	X[d] = out.X()[d];
      }
    }
    X[0] *= 2; E[0] *= 2; // Since we consider only a single parity at a time

    if (in.isNative()) {
      typedef typename colorspinor_mapper<FloatIn,Ns,Nc>::type ColorSpinor;
      ColorSpinor inOrder(in, 1, In, inNorm);
      extendedCopyColorSpinor<FloatOut,FloatIn,Ns,Nc>(inOrder, out, in.GammaBasis(), E, X, parity, extend, location, Out, outNorm);
    } else {
      errorQuda("Order not defined");
    }

  }

  template<int Ns, typename dstFloat, typename srcFloat>
    void copyExtendedColorSpinor(ColorSpinorField &dst, const ColorSpinorField &src, 
        const int parity, const QudaFieldLocation location, dstFloat *Dst, srcFloat *Src,
        float *dstNorm, float *srcNorm) {


      if(dst.Ndim() != src.Ndim())
        errorQuda("Number of dimensions %d %d don't match", dst.Ndim(), src.Ndim());

      if(!(dst.SiteOrder() == src.SiteOrder() ||
            (dst.SiteOrder() == QUDA_EVEN_ODD_SITE_ORDER &&
             src.SiteOrder() == QUDA_ODD_EVEN_SITE_ORDER) ||
            (dst.SiteOrder() == QUDA_ODD_EVEN_SITE_ORDER &&
             src.SiteOrder() == QUDA_EVEN_ODD_SITE_ORDER) ) ){

        errorQuda("Subset orders %d %d don't match", dst.SiteOrder(), src.SiteOrder());
      }

      if(dst.SiteSubset() != src.SiteSubset())
        errorQuda("Subset types do not match %d %d", dst.SiteSubset(), src.SiteSubset());

      if(dst.Ncolor() != 3 || src.Ncolor() != 3) errorQuda("Nc != 3 not yet supported");

      const int Nc = 3;

      // We currently only support parity-ordered fields; even-odd or odd-even
      if(dst.SiteOrder() == QUDA_LEXICOGRAPHIC_SITE_ORDER){
        errorQuda("Copying to full fields with lexicographical ordering is not currently supported");
      }

      if(dst.SiteSubset() == QUDA_FULL_SITE_SUBSET){
        if(src.FieldOrder() == QUDA_QDPJIT_FIELD_ORDER ||
            dst.FieldOrder() == QUDA_QDPJIT_FIELD_ORDER){
          errorQuda("QDPJIT field ordering not supported for full site fields");
        }

        // set for the source subset ordering
        srcFloat *srcEven = Src ? Src : (srcFloat*)src.V();
        srcFloat* srcOdd = (srcFloat*)((char*)srcEven + src.Bytes()/2);
        float *srcNormEven = srcNorm ? srcNorm : (float*)src.Norm();
        float *srcNormOdd = (float*)((char*)srcNormEven + src.NormBytes()/2);
        if(src.SiteOrder() == QUDA_ODD_EVEN_SITE_ORDER){
          std::swap<srcFloat*>(srcEven, srcOdd);
          std::swap<float*>(srcNormEven, srcNormOdd);
        }

        // set for the destination subset ordering
        dstFloat *dstEven = Dst ? Dst : (dstFloat*)dst.V();
        dstFloat *dstOdd = (dstFloat*)((char*)dstEven + dst.Bytes()/2);
        float *dstNormEven = dstNorm ? dstNorm : (float*)dst.Norm();
        float *dstNormOdd = (float*)((char*)dstNormEven + dst.NormBytes()/2);
        if(dst.SiteOrder() == QUDA_ODD_EVEN_SITE_ORDER){
          std::swap<dstFloat*>(dstEven, dstOdd);
          std::swap<float*>(dstNormEven, dstNormOdd);
        }

        // should be able to apply to select either even or odd parity at this point as well.
        extendedCopyColorSpinor<dstFloat, srcFloat, Ns, Nc>
          (dst, src, 0, location, dstEven, srcEven, dstNormEven, srcNormEven);
        extendedCopyColorSpinor<dstFloat, srcFloat, Ns, Nc>
          (dst, src, 1, location, dstOdd, srcOdd, dstNormOdd, srcNormOdd); 
      }else{
        extendedCopyColorSpinor<dstFloat, srcFloat, Ns, Nc>
          (dst, src, parity, location, Dst, Src, dstNorm, srcNorm);
      } // N.B. Need to update this to account for differences in parity
    }


  template<typename dstFloat, typename srcFloat>
    void CopyExtendedColorSpinor(ColorSpinorField &dst, const ColorSpinorField &src, 
        const int parity, const QudaFieldLocation location, dstFloat *Dst, srcFloat *Src,
        float *dstNorm=0, float *srcNorm=0)
    {
      if(dst.Nspin() != src.Nspin())
        errorQuda("source and destination spins must match");

      if(dst.Nspin() == 4){
#ifdef NSPIN4
        copyExtendedColorSpinor<4>(dst, src, parity, location, Dst, Src, dstNorm, srcNorm);
#else
	errorQuda("Extended copy has not been built for Nspin=%d fields",dst.Nspin());
#endif
      }else if(dst.Nspin() == 1){
#ifdef NSPIN1
        copyExtendedColorSpinor<1>(dst, src, parity, location, Dst, Src, dstNorm, srcNorm);
#else
	errorQuda("Extended copy has not been built for Nspin=%d fields", dst.Nspin());
#endif
      }else{
        errorQuda("Nspin=%d unsupported", dst.Nspin());
      }
    }


  // There's probably no need to have the additional Dst and Src arguments here!
  void copyExtendedColorSpinor(ColorSpinorField &dst, const ColorSpinorField &src, 
      QudaFieldLocation location, const int parity, void *Dst, void *Src, 
      void *dstNorm, void *srcNorm){

#if 0
    if(dst.Precision() == QUDA_DOUBLE_PRECISION){
      if(src.Precision() == QUDA_DOUBLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<double*>(Dst), static_cast<double*>(Src));
      }else if(src.Precision() == QUDA_SINGLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location,  static_cast<double*>(Dst), static_cast<float*>(Src));
      }else if(src.Precision() == QUDA_HALF_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<double*>(Dst), static_cast<short*>(Src), 0, static_cast<float*>(srcNorm));
      } else {
        errorQuda("Unsupported Precision %d", src.Precision());
      }
    } else if (dst.Precision() == QUDA_SINGLE_PRECISION){
      if(src.Precision() == QUDA_DOUBLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<float*>(Dst), static_cast<double*>(Src));
      }else if(src.Precision() == QUDA_SINGLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<float*>(Dst), static_cast<float*>(Src));
      }else if(src.Precision() == QUDA_HALF_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<float*>(Dst), static_cast<short*>(Src), 0, static_cast<float*>(srcNorm));
      }else{
        errorQuda("Unsupported Precision %d", src.Precision());
      }
    } else if (dst.Precision() == QUDA_HALF_PRECISION){
      if(src.Precision() == QUDA_DOUBLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<short*>(Dst), static_cast<double*>(Src), static_cast<float*>(dstNorm), 0);
      }else if(src.Precision() == QUDA_SINGLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<short*>(Dst), static_cast<float*>(Src), static_cast<float*>(dstNorm), 0);
      }else if(src.Precision() == QUDA_HALF_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<short*>(Dst), static_cast<short*>(Src), static_cast<float*>(dstNorm), static_cast<float*>(srcNorm));
      }else{
        errorQuda("Unsupported Precision %d", src.Precision());
      }
    } else if (dst.Precision() == QUDA_QUARTER_PRECISION){
      if(src.Precision() == QUDA_DOUBLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<char*>(Dst), static_cast<double*>(Src), static_cast<float*>(dstNorm), 0);
      }else if(src.Precision() == QUDA_SINGLE_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<char*>(Dst), static_cast<float*>(Src), static_cast<float*>(dstNorm), 0);
      }else if(src.Precision() == QUDA_HALF_PRECISION){
        CopyExtendedColorSpinor(dst, src, parity, location, static_cast<char*>(Dst), static_cast<short*>(Src), static_cast<float*>(dstNorm), static_cast<float*>(srcNorm));
      }else{
        errorQuda("Unsupported Precision %d", src.Precision());
      }
    }else{
      errorQuda("Unsupported Precision %d", dst.Precision());
    }
#else
    errorQuda("Disabled");
#endif
  }

} // quda
