#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_ndeg_twisted_mass.cuh>

/**
   This is the gauged twisted-mass operator acting on a non-generate
   quark doublet.
*/

namespace quda
{

  template <typename Arg> class NdegTwistedMass : public Dslash<nDegTwistedMass, Arg>
  {
    using Dslash = Dslash<nDegTwistedMass, Arg>;
    using Dslash::arg;
    using Dslash::in;

  public:
    NdegTwistedMass(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
    {
      TunableKernel3D::resizeVector(2, arg.nParity);
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);
      if (arg.xpay)
        Dslash::template instantiate<packShmem, true>(tp, stream);
      else
        errorQuda("Non-degenerate twisted-mass operator only defined for xpay=true");
    }

    long long flops() const
    {
      long long flops = Dslash::flops();
      switch (arg.kernel_type) {
      case INTERIOR_KERNEL:
      case UBER_KERNEL:
      case KERNEL_POLICY:
        flops += 2 * in.Ncolor() * 4 * 4 * in.Volume(); // complex * Nc * Ns * fma * vol
        break;
      default: break; // twisted-mass flops are in the interior kernel
      }
      return flops;
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct NdegTwistedMassApply {

    inline NdegTwistedMassApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
                                double b, double c, const ColorSpinorField &x, int parity, bool dagger,
                                const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      NdegTwistedMassArg<Float, nColor, nDim, recon> arg(out, in, U, a, b, c, x, parity, dagger, comm_override);
      NdegTwistedMass<decltype(arg)> twisted(arg, out, in);

      dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, in.getDslashConstant().volume_4d_cb, in.getDslashConstant().ghostFaceCB, profile);
    }
  };

#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
  void ApplyNdegTwistedMass(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a, double b,
                            double c, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
                            TimeProfile &profile)
  {
    instantiate<NdegTwistedMassApply>(out, in, U, a, b, c, x, parity, dagger, comm_override, profile);
  }
#else
  void ApplyNdegTwistedMass(ColorSpinorField &, const ColorSpinorField &, const GaugeField &, double, double,
                            double, const ColorSpinorField &, int, bool, const int *, TimeProfile &)
  {
    errorQuda("Non-degenerate twisted-mass dslash has not been built");
  }
#endif // GPU_NDEG_TWISTED_MASS_DIRAC

} // namespace quda
