#include <tunable_nd.h>
#include <kernels/copy_gauge_extended.cuh>

namespace quda {

  template <typename FloatOut, typename FloatIn, int length, typename OutOrder, typename InOrder>
  class CopyGaugeEx : TunableKernel2D {
    template <bool expand> using Arg = CopyGaugeExArg<FloatOut, FloatIn, length, OutOrder, InOrder, expand>;
    GaugeField &out;
    const GaugeField &in;
    QudaFieldLocation location;
    FloatOut *Out;
    FloatIn *In;

    unsigned int minThreads() const { return in.VolumeCB() == out.VolumeCB() ? in.VolumeCB() : in.LocalVolumeCB(); }

  public:
    CopyGaugeEx(GaugeField &out, const GaugeField &in, QudaFieldLocation location,
                FloatOut *Out, FloatIn *In) :
      TunableKernel2D(in, 2, location),
      out(out),
      in(in),
      location(location),
      Out(Out),
      In(In)
    {
      strcat(aux, out.AuxString());
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      constexpr bool enable_host = true;
      if (out.Volume() > in.Volume()) launch<CopyGaugeEx_, enable_host>(tp, stream, Arg<true>(out, in, Out, In));
      else                            launch<CopyGaugeEx_, enable_host>(tp, stream, Arg<false>(out, in, Out, In));
    }

    long long flops() const { return 0; }
    long long bytes() const
    { // only count interior sites
      return (out.LocalVolume() * out.Bytes()) / out.Volume() +  (in.LocalVolume() * in.Bytes()) / in.Volume();
    }
  };

  template <typename FloatOut, typename FloatIn, int length, typename InOrder>
  void copyGaugeEx(GaugeField &out, const GaugeField &in, QudaFieldLocation location, FloatOut *Out, FloatIn *In)
  {
    if (out.isNative()) {
      if (out.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        typedef typename gauge_mapper<FloatOut, QUDA_RECONSTRUCT_NO>::type G;
        CopyGaugeEx<FloatOut, FloatIn, length, G, InOrder>(out, in, location, Out, In);
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_12) {
#if QUDA_RECONSTRUCT & 2
        typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_12>::type G;
	CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-12", QUDA_RECONSTRUCT);
#endif
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_8) {
#if QUDA_RECONSTRUCT & 1
        typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_8>::type G;
	CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-8", QUDA_RECONSTRUCT);
#endif
#ifdef GPU_STAGGERED_DIRAC
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_13) {
#if QUDA_RECONSTRUCT & 2
        typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_13>::type G;
        CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-13", QUDA_RECONSTRUCT);
#endif
      } else if (out.Reconstruct() == QUDA_RECONSTRUCT_9) {
#if QUDA_RECONSTRUCT & 1
        typedef typename gauge_mapper<FloatOut,QUDA_RECONSTRUCT_9>::type G;
        CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-9", QUDA_RECONSTRUCT);
#endif
#endif // GPU_STAGGERED_DIRAC
      } else {
	errorQuda("Reconstruction %d and order %d not supported", out.Reconstruct(), out.Order());
      }
    } else if (out.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      using G = QDPOrder<FloatOut,length>;
      CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      using G = MILCOrder<FloatOut, length>;
      CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (out.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      using G = TIFROrder<FloatOut,length>;
      CopyGaugeEx<FloatOut,FloatIn,length, G, InOrder>(out, in, location, Out, In);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", out.Order());
    }

  }

  template <typename FloatOut, typename FloatIn, int length>
  void copyGaugeEx(GaugeField &out, const GaugeField &in, QudaFieldLocation location,
		   FloatOut *Out, FloatIn *In)
  {
    if (in.isNative()) {
      if (in.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        typedef typename gauge_mapper<FloatIn, QUDA_RECONSTRUCT_NO>::type G;
        copyGaugeEx<FloatOut, FloatIn, length, G>(out, in, location, Out, In);
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_12) {
#if QUDA_RECONSTRUCT & 2
        typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_12>::type G;
	copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-12", QUDA_RECONSTRUCT);
#endif
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_8) {
#if QUDA_RECONSTRUCT & 1
        typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_8>::type G;
	copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-8", QUDA_RECONSTRUCT);
#endif
#ifdef GPU_STAGGERED_DIRAC
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_13) {
#if QUDA_RECONSTRUCT & 2
        typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_13>::type G;
	copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-13", QUDA_RECONSTRUCT);
#endif
      } else if (in.Reconstruct() == QUDA_RECONSTRUCT_9) {
#if QUDA_RECONSTRUCT & 1
        typedef typename gauge_mapper<FloatIn,QUDA_RECONSTRUCT_9>::type G;
	copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
        errorQuda("QUDA_RECONSTRUCT=%d does not enable reconstruct-9", QUDA_RECONSTRUCT);
#endif
#endif // GPU_STAGGERED_DIRAC
      } else {
	errorQuda("Reconstruction %d and order %d not supported", in.Reconstruct(), in.Order());
      }
    } else if (in.Order() == QUDA_QDP_GAUGE_ORDER) {

#ifdef BUILD_QDP_INTERFACE
      using G = QDPOrder<FloatIn, length>;
      copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
      errorQuda("QDP interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      using G = MILCOrder<FloatIn, length>;
      copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (in.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      using G = TIFROrder<FloatIn,length>;
      copyGaugeEx<FloatOut,FloatIn,length, G>(out, in, location, Out, In);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", in.Order());
    }

  }

  template <typename FloatOut, typename FloatIn>
  void copyGaugeEx(GaugeField &out, const GaugeField &in, QudaFieldLocation location,
		   FloatOut *Out, FloatIn *In) {

    if (in.Ncolor() != 3 && out.Ncolor() != 3) {
      errorQuda("Unsupported number of colors; out.Nc=%d, in.Nc=%d", out.Ncolor(), in.Ncolor());
    }

    if (out.Geometry() != in.Geometry()) {
      errorQuda("Field geometries %d %d do not match", out.Geometry(), in.Geometry());
    }

    if (in.LinkType() != QUDA_ASQTAD_MOM_LINKS && out.LinkType() != QUDA_ASQTAD_MOM_LINKS) {
      // we are doing gauge field packing
      copyGaugeEx<FloatOut,FloatIn,18>(out, in, location, Out, In);
    } else {
      errorQuda("Not supported");
    }
  }

  void copyExtendedGauge(GaugeField &out, const GaugeField &in,
			 QudaFieldLocation location, void *Out, void *In) {

    for (int d=0; d<in.Ndim(); d++) {
      if ( (out.X()[d] - in.X()[d]) % 2 != 0)
	errorQuda("Cannot copy into an asymmetrically extended gauge field");
    }

    if (out.Precision() == QUDA_DOUBLE_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
	copyGaugeEx(out, in, location, (double*)Out, (double*)In);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
#if QUDA_PRECISION & 4
        copyGaugeEx(out, in, location, (double*)Out, (float*)In);
#else
        errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
      } else {
        errorQuda("Precision %d not instantiated", in.Precision());
      }
    } else if (out.Precision() == QUDA_SINGLE_PRECISION) {
      if (in.Precision() == QUDA_DOUBLE_PRECISION) {
        copyGaugeEx(out, in, location, (float *)Out, (double *)In);
      } else if (in.Precision() == QUDA_SINGLE_PRECISION) {
#if QUDA_PRECISION & 4
        copyGaugeEx(out, in, location, (float *)Out, (float *)In);
#else
        errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
      } else {
        errorQuda("Precision %d not instantiated", in.Precision());
      }
    } else if (out.Precision() == QUDA_HALF_PRECISION) {
      if (in.Precision() == QUDA_HALF_PRECISION) {
#if QUDA_PRECISION & 2
        copyGaugeEx(out, in, location, (short *)Out, (short *)In);
#else
        errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
      } else {
        errorQuda("Precision %d not instantiated", in.Precision());
      }
    } else if (out.Precision() == QUDA_QUARTER_PRECISION) {
      if (in.Precision() == QUDA_QUARTER_PRECISION) {
#if QUDA_PRECISION & 1
        copyGaugeEx(out, in, location, (int8_t *)Out, (int8_t *)In);
#else
        errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
      } else {
        errorQuda("Precision %d not instantiated", in.Precision());
      }
    } else {
      errorQuda("Precision %d not instantiated", out.Precision());
    }
  }

} // namespace quda
