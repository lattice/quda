#include <utility>
#include <quda_internal.h>
#include <gauge_field.h>
#include <ks_improved_force.h>
#include <tune_quda.h>
#include <instantiate.h>
#include <tunable_nd.h>
#include <kernels/hisq_paths_force.cuh>

namespace quda {

  namespace fermion_force {

    template <typename Arg> class FatLinkForce : public TunableKernel3D {
      Arg &arg;
      const GaugeField &outA;
      const GaugeField &outB;
      const GaugeField &pMu;
      const GaugeField &qMu;
      const GaugeField &p3;
      const GaugeField &link;
      const HisqForceType type;
      unsigned int minThreads() const { return arg.threads.x; }

    public:
      FatLinkForce(Arg &arg, const GaugeField &link, int sig, int mu, HisqForceType type,
                   const GaugeField &outA, const GaugeField &outB, const GaugeField &pMu,
                   const GaugeField &qMu, const GaugeField &p3) :
        TunableKernel3D(link, 2, type == FORCE_ONE_LINK ? 4 : 1),
        arg(arg),
        outA(outA),
        outB(outB),
        pMu(pMu),
        qMu(qMu),
        p3(p3),
        link(link),
        type(type)
      {
        arg.sig = sig;
        arg.mu = mu;

        strcat(aux, (std::string(comm_dim_partitioned_string()) + "threads=" + std::to_string(arg.threads.x)).c_str());
        if (type == FORCE_MIDDLE_LINK || type == FORCE_LEPAGE_MIDDLE_LINK)
          strcat(aux, (std::string(",sig=") + std::to_string(arg.sig) +
                       std::string(",mu=") + std::to_string(arg.mu) +
                       std::string(",pMu=") + std::to_string(arg.p_mu) +
                       std::string(",q_mu=") + std::to_string(arg.q_mu) +
                       std::string(",q_prev=") + std::to_string(arg.q_prev)).c_str());
        else if (type != FORCE_ONE_LINK)
          strcat(aux, (std::string(",mu=") + std::to_string(arg.mu)).c_str()); // no sig dependence needed for side link

        switch (type) {
        case FORCE_ONE_LINK:           strcat(aux, ",ONE_LINK");           break;
        case FORCE_ALL_LINK:           strcat(aux, ",ALL_LINK");           break;
        case FORCE_MIDDLE_LINK:        strcat(aux, ",MIDDLE_LINK");        break;
        case FORCE_LEPAGE_MIDDLE_LINK: strcat(aux, ",LEPAGE_MIDDLE_LINK"); break;
        case FORCE_SIDE_LINK:          strcat(aux, ",SIDE_LINK");          break;
        case FORCE_SIDE_LINK_SHORT:    strcat(aux, ",SIDE_LINK_SHORT");    break;
        default: errorQuda("Undefined force type %d", type);
        }

        apply(device::get_default_stream());
      }

      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        switch (type) {
        case FORCE_ONE_LINK:
          launch<OneLinkTerm>(tp, stream, arg);
          break;
        case FORCE_ALL_LINK:
          if (goes_forward(arg.sig) && goes_forward(arg.mu)) {
            launch<AllLink>(tp, stream, FatLinkParam<Arg, 1, 1>(arg));
          } else if (goes_forward(arg.sig) && goes_backward(arg.mu)) {
            launch<AllLink>(tp, stream, FatLinkParam<Arg, 0, 1>(arg));
          } else if (goes_backward(arg.sig) && goes_forward(arg.mu)) {
            launch<AllLink>(tp, stream, FatLinkParam<Arg, 1, 0>(arg));
          } else {
            launch<AllLink>(tp, stream, FatLinkParam<Arg, 0, 0>(arg));
          }
          break;
        case FORCE_MIDDLE_LINK:
          if (!arg.p_mu || !arg.q_mu) errorQuda("Expect p_mu=%d and q_mu=%d to both be true", arg.p_mu, arg.q_mu);
          if (arg.q_prev) {
            if (goes_forward(arg.sig) && goes_forward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 1, true, true, true>(arg));
            } else if (goes_forward(arg.sig) && goes_backward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 1, true, true, true>(arg));
            } else if (goes_backward(arg.sig) && goes_forward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 0, true, true, true>(arg));
            } else {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 0, true, true, true>(arg));
            }
          } else {
            if (goes_forward(arg.sig) && goes_forward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 1, true, true, false>(arg));
            } else if (goes_forward(arg.sig) && goes_backward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 1, true, true, false>(arg));
            } else if (goes_backward(arg.sig) && goes_forward(arg.mu)) {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 0, true, true, false>(arg));
            } else {
              launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 0, true, true, false>(arg));
            }
          }
          break;
        case FORCE_LEPAGE_MIDDLE_LINK:
          if (arg.p_mu || arg.q_mu || !arg.q_prev)
            errorQuda("Expect p_mu=%d and q_mu=%d to both be false and q_prev=%d true", arg.p_mu, arg.q_mu, arg.q_prev);
          if (goes_forward(arg.sig) && goes_forward(arg.mu)) {
            launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 1, false, false, true>(arg));
          } else if (goes_forward(arg.sig) && goes_backward(arg.mu)) {
            launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 1, false, false, true>(arg));
          } else if (goes_backward(arg.sig) && goes_forward(arg.mu)) {
            launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 1, 0, false, false, true>(arg));
          } else {
            launch<MiddleLink>(tp, stream, FatLinkParam<Arg, 0, 0, false, false, true>(arg));
          }
          break;
        case FORCE_SIDE_LINK:
          if (goes_forward(arg.mu)) {
            launch<SideLink>(tp, stream, FatLinkParam<Arg, 1>(arg));
          } else {
            launch<SideLink>(tp, stream, FatLinkParam<Arg, 0>(arg));
          }
          break;
        case FORCE_SIDE_LINK_SHORT:
          if (goes_forward(arg.mu)) {
            launch<SideLinkShort>(tp, stream, FatLinkParam<Arg, 1>(arg));
          } else {
            launch<SideLinkShort>(tp, stream, FatLinkParam<Arg, 0>(arg));
          }
          break;
        default:
          errorQuda("Undefined force type %d", type);
        }
      }

      void preTune() {
        switch (type) {
        case FORCE_ONE_LINK:
          outA.backup();
          break;
        case FORCE_ALL_LINK:
          outA.backup();
          outB.backup();
          break;
        case FORCE_MIDDLE_LINK:
          pMu.backup();
          qMu.backup();
          outA.backup();
          p3.backup();
          break;
        case FORCE_LEPAGE_MIDDLE_LINK:
          outA.backup();
          p3.backup();
          break;
        case FORCE_SIDE_LINK:
          outB.backup();
          outA.backup();
          break;
        case FORCE_SIDE_LINK_SHORT:
          outA.backup();
          break;
        default: errorQuda("Undefined force type %d", type);
        }
      }

      void postTune() {
        switch (type) {
        case FORCE_ONE_LINK:
          outA.restore();
          break;
        case FORCE_ALL_LINK:
          outA.restore();
          outB.restore();
          break;
        case FORCE_MIDDLE_LINK:
          pMu.restore();
          qMu.restore();
          outA.restore();
          p3.restore();
          break;
        case FORCE_LEPAGE_MIDDLE_LINK:
          outA.restore();
          p3.restore();
          break;
        case FORCE_SIDE_LINK:
          outB.restore();
          outA.restore();
          break;
        case FORCE_SIDE_LINK_SHORT:
          outA.restore();
          break;
        default: errorQuda("Undefined force type %d", type);
        }
      }

      long long flops() const {
        switch (type) {
        case FORCE_ONE_LINK:
          return 2*4*arg.threads.x*36ll;
        case FORCE_ALL_LINK:
          return 2*arg.threads.x*(goes_forward(arg.sig) ? 1242ll : 828ll);
        case FORCE_MIDDLE_LINK:
        case FORCE_LEPAGE_MIDDLE_LINK:
          return 2*arg.threads.x*(2 * 198 +
                                (!arg.q_prev && goes_forward(arg.sig) ? 198 : 0) +
                                (arg.q_prev && (arg.q_mu || goes_forward(arg.sig) ) ? 198 : 0) +
                                ((arg.q_prev && goes_forward(arg.sig) ) ?  198 : 0) +
                                ( goes_forward(arg.sig) ? 216 : 0) );
        case FORCE_SIDE_LINK:       return 2*arg.threads.x*2*234;
        case FORCE_SIDE_LINK_SHORT: return 2*arg.threads.x*36;
        default: errorQuda("Undefined force type %d", type);
        }
        return 0;
      }

      long long bytes() const {
        switch (type) {
        case FORCE_ONE_LINK:
          return 2*4*arg.threads.x*( arg.oProd.Bytes() + 2*arg.outA.Bytes() );
        case FORCE_ALL_LINK:
          return 2*arg.threads.x*( (goes_forward(arg.sig) ? 4 : 2)*arg.outA.Bytes() + 3*arg.link.Bytes()
                                 + arg.oProd.Bytes() + arg.qPrev.Bytes() + 2*arg.outB.Bytes());
        case FORCE_MIDDLE_LINK:
        case FORCE_LEPAGE_MIDDLE_LINK:
          return 2*arg.threads.x*( ( goes_forward(arg.sig) ? 2*arg.outA.Bytes() : 0 ) +
                                 (arg.p_mu ? arg.pMu.Bytes() : 0) +
                                 (arg.q_mu ? arg.qMu.Bytes() : 0) +
                                 ( ( goes_forward(arg.sig) || arg.q_mu ) ? arg.qPrev.Bytes() : 0) +
                                 arg.p3.Bytes() + 3*arg.link.Bytes() + arg.oProd.Bytes() );
        case FORCE_SIDE_LINK:
          return 2*arg.threads.x*( 2*arg.outA.Bytes() + 2*arg.outB.Bytes() +
                                 arg.p3.Bytes() + arg.link.Bytes() + arg.qProd.Bytes() );
        case FORCE_SIDE_LINK_SHORT:
          return 2*arg.threads.x*( 2*arg.outA.Bytes() + arg.p3.Bytes() );
        default: errorQuda("Undefined force type %d", type);
        }
        return 0;
      }
    };

    template <typename real, int nColor, QudaReconstructType recon>
    struct HisqStaplesForce {
      HisqStaplesForce(GaugeField &Pmu, GaugeField &P3, GaugeField &P5, GaugeField &Pnumu,
                       GaugeField &Qmu, GaugeField &Qnumu, GaugeField &newOprod,
                       const GaugeField &oprod, const GaugeField &link,
                       const double *path_coeff_array)
      {
        PathCoefficients<real> act_path_coeff(path_coeff_array);
        real OneLink = act_path_coeff.one;
        real ThreeSt = act_path_coeff.three;
        real mThreeSt = -ThreeSt;
        real FiveSt  = act_path_coeff.five;
        real mFiveSt  = -FiveSt;
        real SevenSt = act_path_coeff.seven;
        real Lepage  = act_path_coeff.lepage;
        real mLepage  = -Lepage;

        {
          FatLinkArg<real, nColor> arg(newOprod, oprod, link, OneLink, FORCE_ONE_LINK);
          arg.threads.z = 4;
          FatLinkForce<decltype(arg)> oneLink(arg, link, 0, 0, FORCE_ONE_LINK, newOprod, newOprod, oprod, oprod, oprod);
        }

        for (int sig=0; sig<8; sig++) {
          for (int mu=0; mu<8; mu++) {
            if ( (mu == sig) || (mu == opp_dir(sig))) continue;

            //3-link
            //Kernel A: middle link
            FatLinkArg<real, nColor> middleLinkArg(newOprod, Pmu, P3, Qmu, oprod, link, mThreeSt, 2, FORCE_MIDDLE_LINK);
            FatLinkForce<decltype(middleLinkArg)> middleLink(middleLinkArg, link, sig, mu, FORCE_MIDDLE_LINK, newOprod, newOprod, Pmu, P3, Qmu);

            for (int nu=0; nu < 8; nu++) {
              if (nu == sig || nu == opp_dir(sig) || nu == mu || nu == opp_dir(mu)) continue;

              //5-link: middle link
              //Kernel B
              FatLinkArg<real, nColor> middleLinkArg(newOprod, Pnumu, P5, Qnumu, Pmu, Qmu, link, FiveSt, 1, FORCE_MIDDLE_LINK);
              FatLinkForce<decltype(middleLinkArg)> middleLink(middleLinkArg, link, sig, nu, FORCE_MIDDLE_LINK, newOprod, newOprod, Pnumu, P5, Qnumu);

              for (int rho = 0; rho < 8; rho++) {
                if (rho == sig || rho == opp_dir(sig) || rho == mu || rho == opp_dir(mu) || rho == nu || rho == opp_dir(nu)) continue;

                //7-link: middle link and side link
                FatLinkArg<real, nColor> arg(newOprod, P5, Pnumu, Qnumu, link, SevenSt, FiveSt != 0 ? SevenSt/FiveSt : 0, 1, FORCE_ALL_LINK, true);
                FatLinkForce<decltype(arg)> all(arg, link, sig, rho, FORCE_ALL_LINK, newOprod, P5, P5, P5, Qnumu);

              }//rho

              //5-link: side link
              FatLinkArg<real, nColor> arg(newOprod, P3, P5, Qmu, link, mFiveSt, (ThreeSt != 0 ? FiveSt/ThreeSt : 0), 1, FORCE_SIDE_LINK);
              FatLinkForce<decltype(arg)> side(arg, link, sig, nu, FORCE_SIDE_LINK, newOprod, P3, P5, P5, Qmu);

            } //nu

            //lepage
            if (Lepage != 0.) {
              FatLinkArg<real, nColor> middleLinkArg(newOprod, P5, Pmu, Qmu, link, Lepage, 2, FORCE_LEPAGE_MIDDLE_LINK);
              FatLinkForce<decltype(middleLinkArg)> middleLink(middleLinkArg, link, sig, mu, FORCE_LEPAGE_MIDDLE_LINK, newOprod, newOprod, P5, P5, Qmu);

              FatLinkArg<real, nColor> arg(newOprod, P3, P5, Qmu, link, mLepage, (ThreeSt != 0 ? Lepage/ThreeSt : 0), 2, FORCE_SIDE_LINK);
              FatLinkForce<decltype(arg)> side(arg, link, sig, mu, FORCE_SIDE_LINK, newOprod, P3, P5, P5, Qmu);
            } // Lepage != 0.0

            // 3-link side link
            FatLinkArg<real, nColor> arg(newOprod, P3, link, ThreeSt, 1, FORCE_SIDE_LINK_SHORT);
            FatLinkForce<decltype(arg)> side(arg, P3, sig, mu, FORCE_SIDE_LINK_SHORT, newOprod, newOprod, P3, P3, P3);
          }//mu
        }//sig
      }
    };

#ifdef GPU_STAGGERED_DIRAC
    void hisqStaplesForce(GaugeField &newOprod, const GaugeField &oprod, const GaugeField &link, const double path_coeff_array[6])
    {
      checkNative(link, oprod, newOprod);
      checkLocation(newOprod, oprod, link);
      checkPrecision(oprod, link, newOprod);

      // create color matrix fields with zero padding
      GaugeFieldParam gauge_param(link);
      gauge_param.reconstruct = QUDA_RECONSTRUCT_NO;
      gauge_param.geometry = QUDA_SCALAR_GEOMETRY;
      gauge_param.setPrecision(gauge_param.Precision(), true);

      auto Pmu = GaugeField::Create(gauge_param);
      auto P3 = GaugeField::Create(gauge_param);
      auto P5 = GaugeField::Create(gauge_param);
      auto Pnumu = GaugeField::Create(gauge_param);
      auto Qmu = GaugeField::Create(gauge_param);
      auto Qnumu = GaugeField::Create(gauge_param);

      instantiate<HisqStaplesForce, ReconstructNone>(*Pmu, *P3, *P5, *Pnumu, *Qmu, *Qnumu, newOprod, oprod, link, path_coeff_array);

      delete Pmu;
      delete P3;
      delete P5;
      delete Pnumu;
      delete Qmu;
      delete Qnumu;
    }
#else
    void hisqStaplesForce(GaugeField &, const GaugeField &, const GaugeField &, const double[6])
    {
      errorQuda("HISQ force not enabled");
    }
#endif

    template <typename Arg>
    class HisqForce : public TunableKernel2D {

      Arg &arg;
      GaugeField &force;
      const GaugeField &meta;
      const HisqForceType type;
      unsigned int minThreads() const { return arg.threads.x; }

    public:
      HisqForce(Arg &arg, GaugeField &force, const GaugeField &meta, int sig, int mu, HisqForceType type) :
        TunableKernel2D(meta, 2),
        arg(arg),
        force(force),
        meta(meta),
        type(type)
      {
        arg.sig = sig;
        arg.mu = mu;
        apply(device::get_default_stream());
      }

      void apply(const qudaStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        switch (type) {
        case FORCE_LONG_LINK: launch<LongLink>(tp, stream, arg); break;
        case FORCE_COMPLETE:  launch<CompleteForce>(tp, stream, arg); break;
        default: errorQuda("Undefined force type %d", type);
        }
      }

      TuneKey tuneKey() const {
        std::stringstream aux;
        aux << meta.AuxString() << comm_dim_partitioned_string() << ",threads=" << arg.threads.x;
        switch (type) {
        case FORCE_LONG_LINK: aux << ",LONG_LINK"; break;
        case FORCE_COMPLETE:  aux << ",COMPLETE";  break;
        default: errorQuda("Undefined force type %d", type);
        }
        return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
      }

      void preTune() {
        switch (type) {
        case FORCE_LONG_LINK:
        case FORCE_COMPLETE:
          force.backup(); break;
        default: errorQuda("Undefined force type %d", type);
        }
      }

      void postTune() {
        switch (type) {
        case FORCE_LONG_LINK:
        case FORCE_COMPLETE:
          force.restore(); break;
        default: errorQuda("Undefined force type %d", type);
        }
      }

      long long flops() const {
        switch (type) {
        case FORCE_LONG_LINK: return 2*arg.threads.x*4968ll;
        case FORCE_COMPLETE:  return 2*arg.threads.x*792ll;
        default: errorQuda("Undefined force type %d", type);
        }
        return 0;
      }

      long long bytes() const {
        switch (type) {
        case FORCE_LONG_LINK: return 4*2*arg.threads.x*(2*arg.outA.Bytes() + 4*arg.link.Bytes() + 3*arg.oProd.Bytes());
        case FORCE_COMPLETE:  return 4*2*arg.threads.x*(arg.outA.Bytes() + arg.link.Bytes() + arg.oProd.Bytes());
        default: errorQuda("Undefined force type %d", type);
        }
        return 0;
      }
    };

    template <typename real, int nColor, QudaReconstructType recon>
    struct HisqLongLinkForce {
      HisqLongLinkForce(GaugeField &newOprod, const GaugeField &oldOprod, const GaugeField &link, double coeff)
      {
        LongLinkArg<real, nColor, recon> arg(newOprod, link, oldOprod, coeff);
        HisqForce<decltype(arg)> longLink(arg, newOprod, link, 0, 0, FORCE_LONG_LINK);
      }
    };

#ifdef GPU_STAGGERED_DIRAC
    void hisqLongLinkForce(GaugeField &newOprod, const GaugeField &oldOprod, const GaugeField &link, double coeff)
    {
      checkNative(link, oldOprod, newOprod);
      checkLocation(newOprod, oldOprod, link);
      checkPrecision(newOprod, link, oldOprod);
      instantiate<HisqLongLinkForce, ReconstructNone>(newOprod, oldOprod, link, coeff);
    }
#else
    void hisqLongLinkForce(GaugeField &, const GaugeField &, const GaugeField &, double)
    {
      errorQuda("HISQ force not enabled");
    }
#endif

    template <typename real, int nColor, QudaReconstructType recon>
    struct HisqCompleteForce {
      HisqCompleteForce(GaugeField &force, const GaugeField &link)
      {
        CompleteForceArg<real, nColor, recon> arg(force, link);
        HisqForce<decltype(arg)> completeForce(arg, force, link, 0, 0, FORCE_COMPLETE);
      }
    };

#ifdef GPU_STAGGERED_DIRAC
    void hisqCompleteForce(GaugeField &force, const GaugeField &link)
    {
      checkNative(link, force);
      checkLocation(force, link);
      checkPrecision(link, force);
      instantiate<HisqCompleteForce, ReconstructNone>(force, link);
    }
#else
    void hisqCompleteForce(GaugeField &, const GaugeField &)
    {
      errorQuda("HISQ force not enabled");
    }
#endif

  } // namespace fermion_force

} // namespace quda
