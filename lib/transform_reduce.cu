#include <reduce_helper.h>
#include <transform_reduce.h>
#include <tunable_reduction.h>
#include <kernels/transform_reduce.cuh>

namespace quda
{

  /**
     Trait that returns the correct comm reduce class for a given reducer
   */
  template <typename T, typename reducer> struct get_comm_reducer_t { };
  template <> struct get_comm_reducer_t<double, plus<double>> { using type = comm_reduce_sum<double>; };
  template <> struct get_comm_reducer_t<double, maximum<double>> { using type = comm_reduce_max<double>; };
  template <> struct get_comm_reducer_t<double, maximum<float>> { using type = comm_reduce_max<double>; };
  template <> struct get_comm_reducer_t<double, minimum<double>> { using type = comm_reduce_min<double>; };
  template <> struct get_comm_reducer_t<double, minimum<float>> { using type = comm_reduce_min<double>; };

  template <typename reduce_t, typename T, typename count_t, typename transformer, typename reducer>
  class TransformReduce : TunableMultiReduction<1>
  {
    using Arg = TransformReduceArg<reduce_t, T, count_t, transformer, reducer>;
    QudaFieldLocation location;
    std::vector<reduce_t> &result;
    const std::vector<T *> &v;
    count_t n_items;
    transformer &h;
    reduce_t init;
    reducer &r;

    bool tuneSharedBytes() const { return false; }

  public:
    TransformReduce(QudaFieldLocation location, std::vector<reduce_t> &result, const std::vector<T *> &v, count_t n_items,
                    transformer &h, reduce_t init, reducer &r) :
      TunableMultiReduction(n_items, v.size(), Arg::max_n_batch_block, location),
      location(location),
      result(result),
      v(v),
      n_items(n_items),
      h(h),
      init(init),
      r(r)
    {
      char aux2[TuneKey::aux_n];
      strcpy(aux2, "batch_size=");
      u32toa(aux2 + 11, v.size());
      strcat(aux, aux2);
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Arg arg(v, n_items, h, init, r);
      launch<transform_reducer, reduce_t, typename get_comm_reducer_t<reduce_t, reducer>::type, true>(result, tp, stream, arg);
    }

    long long bytes() const { return v.size() * n_items * sizeof(T); }
  };

  template <typename reduce_t, typename T, typename count_t, typename transformer, typename reducer>
  void transform_reduce(QudaFieldLocation location, std::vector<reduce_t> &result, const std::vector<T *> &v, count_t n_items,
                        transformer h, reduce_t init, reducer r)
  {
    if (result.size() != v.size()) errorQuda("result %lu and input %lu set sizes do not match", result.size(), v.size());
    TransformReduce<reduce_t, T, count_t, transformer, reducer> reduce(location, result, v, n_items, h, init, r);
  }

  template <typename reduce_t, typename T, typename count_t, typename transformer, typename reducer>
  reduce_t transform_reduce(QudaFieldLocation location, const T *v, count_t n_items, transformer h, reduce_t init, reducer r)
  {
    std::vector<reduce_t> result = {0.0};
    std::vector<const T *> v_ = {v};
    transform_reduce(location, result, v_, n_items, h, init, r);
    return result[0];
  }

  template <typename reduce_t, typename T, typename count_t, typename transformer, typename reducer>
  void reduce(QudaFieldLocation location, std::vector<reduce_t> &result, const std::vector<T *> &v, count_t n_items,
              reduce_t init, reducer r)
  {
    transform_reduce(location, result, v, n_items, identity<T>(), init, r);
  }

  template <typename reduce_t, typename T, typename count_t, typename reducer>
  reduce_t reduce(QudaFieldLocation location, const T *v, count_t n_items, reduce_t init, reducer r)
  {
    std::vector<reduce_t> result = {0.0};
    std::vector<const T *> v_ = {v};
    transform_reduce(location, result, v_, n_items, identity<T>(), init, r);
    return result[0];
  }

  // explicit instantiation list for transform_reduce

  template void transform_reduce<double, complex<float>, unsigned int, square_<double, float>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, square_<double, float>,
    double, plus<double>);
  template void transform_reduce<double, complex<int>, unsigned int, abs_<double, int>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<quda::complex<int> *> const &, unsigned int, abs_<double, int>, double,
    plus<double>);
  template void transform_reduce<double, complex<double>, unsigned int, abs_<double, double>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_<double, double>, double,
    plus<double>);
  template void transform_reduce<double, complex<float>, unsigned int, abs_min_<float, float>, minimum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, abs_min_<float, float>, double,
    minimum<float>);
  template void transform_reduce<double, complex<int>, unsigned int, square_<double, int>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<int> *> const &, unsigned int, square_<double, int>, double,
    plus<double>);
  template void transform_reduce<double, complex<int>, unsigned int, abs_max_<float, int>, maximum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<int> *> const &, unsigned int, abs_max_<float, int>, double,
    maximum<float>);
  template void transform_reduce<double, complex<double>, unsigned int, abs_min_<double, double>, minimum<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_min_<double, double>, double,
    minimum<double>);
  template void transform_reduce<double, complex<float>, unsigned int, abs_max_<float, float>, maximum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, abs_max_<float, float>, double,
    maximum<float>);
  template void transform_reduce<double, complex<double>, unsigned int, square_<double, double>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, square_<double, double>,
    double, plus<double>);
  template void transform_reduce<double, complex<int>, unsigned int, abs_min_<float, int>, minimum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<int> *> const &, unsigned int, abs_min_<float, int>, double,
    minimum<float>);
  template void transform_reduce<double, complex<double>, unsigned int, abs_max_<double, double>, maximum<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_max_<double, double>, double,
    maximum<double>);
  template void transform_reduce<double, complex<float>, unsigned int, abs_<double, float>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, abs_<double, float>, double,
    plus<double>);
  template void transform_reduce<double, complex<signed char>, unsigned int, square_<double, signed char>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int,
    square_<double, signed char>, double, plus<double>);
  template void transform_reduce<double, complex<short>, unsigned int, abs_min_<float, short>, minimum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, abs_min_<float, short>, double,
    minimum<float>);
  template void transform_reduce<double, complex<short>, unsigned int, square_<double, short>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, square_<double, short>,
    double, plus<double>);
  template void transform_reduce<double, complex<signed char>, unsigned int, abs_max_<float, signed char>, maximum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int,
    abs_max_<float, signed char>, double, maximum<float>);
  template void transform_reduce<double, complex<signed char>, unsigned int, abs_min_<float, signed char>, minimum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int,
    abs_min_<float, signed char>, double, minimum<float>);
  template void transform_reduce<double, complex<signed char>, unsigned int, abs_<double, signed char>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int,
    abs_<double, signed char>, double, plus<double>);
  template void transform_reduce<double, complex<short>, unsigned int, abs_<double, short>, plus<double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, abs_<double, short>, double,
    plus<double>);
  template void transform_reduce<double, complex<short>, unsigned int, abs_max_<float, short>, maximum<float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, abs_max_<float, short>, double,
    maximum<float>);

  template double transform_reduce<double, complex<double>, unsigned long, abs_min_<double, double>, minimum<double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_min_<double, double>, double, minimum<double>);
  template double transform_reduce<double, complex<float>, unsigned int, abs_max_<double, float>, maximum<double>>(
    QudaFieldLocation, complex<float> const *, unsigned int, abs_max_<double, float>, double, maximum<double>);
  template double transform_reduce<double, complex<double>, unsigned long, square_<double, double>, plus<double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, square_<double, double>, double, plus<double>);
  template double transform_reduce<double, complex<double>, unsigned int, abs_max_<double, double>, maximum<double>>(
    QudaFieldLocation, complex<double> const *, unsigned int, abs_max_<double, double>, double, maximum<double>);
  template double transform_reduce<double, complex<float>, unsigned long, abs_<double, float>, plus<double>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_<double, float>, double, plus<double>);
  template double transform_reduce<double, complex<double>, unsigned long, abs_<double, double>, plus<double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_<double, double>, double, plus<double>);
  template double transform_reduce<double, complex<float>, unsigned long, abs_max_<float, float>, maximum<float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_max_<float, float>, double, maximum<float>);
  template double transform_reduce<double, complex<short>, unsigned long, abs_max_<float, short>, maximum<float>>(
    QudaFieldLocation, complex<short> const *, unsigned long, abs_max_<float, short>, double, maximum<float>);
  template double transform_reduce<double, complex<float>, unsigned long, square_<double, float>, plus<double>>(
    QudaFieldLocation, complex<float> const *, unsigned long, square_<double, float>, double, plus<double>);
  template double transform_reduce<double, complex<float>, unsigned long, abs_min_<float, float>, minimum<float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_min_<float, float>, double, minimum<float>);
  template double transform_reduce<double, complex<double>, unsigned long, abs_max_<double, double>, maximum<double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_max_<double, double>, double, maximum<double>);
  template double transform_reduce<double, complex<float>, unsigned int, square_<double, float>, plus<double>>(
    QudaFieldLocation, complex<float> const *, unsigned int, square_<double, float>, double, plus<double>);
  template double transform_reduce<double, complex<short>, unsigned int, square_<double, short>, plus<double>>(
    QudaFieldLocation, complex<short> const *, unsigned int, square_<double, short>, double, plus<double>);
  template double transform_reduce<double, complex<short>, unsigned int, abs_max_<double, short>, maximum<double>>(
    QudaFieldLocation, complex<short> const*, unsigned int, abs_max_<double, short>, double, maximum<double>);

} // namespace quda
