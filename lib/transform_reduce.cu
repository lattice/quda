#include "hip/hip_runtime.h"
#include <reduce_helper.h>
#include <transform_reduce.h>
#include <tunable_reduction.h>
#include <kernels/transform_reduce.cuh>

namespace quda
{

  template <typename reducer, typename T, typename count_t, typename transformer>
  class TransformReduce : TunableMultiReduction<1>
  {
    using reduce_t = typename reducer::reduce_t;
    using Arg = TransformReduceArg<reducer, T, count_t, transformer>;
    QudaFieldLocation location;
    std::vector<reduce_t> &result;
    const std::vector<T *> &v;
    count_t n_items;
    transformer &h;

    bool tuneSharedBytes() const { return false; }

  public:
    TransformReduce(QudaFieldLocation location, std::vector<reduce_t> &result, const std::vector<T *> &v, count_t n_items,
                    transformer &h) :
      TunableMultiReduction(n_items, v.size(), Arg::max_n_batch_block, location),
      location(location),
      result(result),
      v(v),
      n_items(n_items),
      h(h)
    {
      char aux2[TuneKey::aux_n];
      strcpy(aux2, "batch_size=");
      u32toa(aux2 + 11, v.size());
      strcat(aux, aux2);
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Arg arg(v, n_items, h);
      launch<transform_reducer, true>(result, tp, stream, arg);
    }

    long long bytes() const { return v.size() * n_items * sizeof(T); }
  };

  template <typename reducer, typename T, typename count_t, typename transformer>
  void transform_reduce(QudaFieldLocation location, std::vector<typename reducer::reduce_t> &result, const std::vector<T *> &v,
			count_t n_items, transformer h)
  {
    if (result.size() != v.size()) errorQuda("result %lu and input %lu set sizes do not match", result.size(), v.size());
    TransformReduce<reducer, T, count_t, transformer> reduce(location, result, v, n_items, h);
  }

  template <typename reducer, typename T, typename count_t, typename transformer>
  typename reducer::reduce_t transform_reduce(QudaFieldLocation location, const T *v, count_t n_items, transformer h)
  {
    std::vector<typename reducer::reduce_t> result = {0.0};
    std::vector<const T *> v_ = {v};
    transform_reduce<reducer>(location, result, v_, n_items, h);
    return result[0];
  }

  template <typename reducer, typename T, typename count_t, typename transformer>
  void reduce(QudaFieldLocation location, std::vector<typename reducer::reduce_t> &result, const std::vector<T *> &v, count_t n_items)
  {
    transform_reduce<reducer>(location, result, v, n_items, identity<T>());
  }

  template <typename reducer, typename T, typename count_t>
  typename reducer::reduce_t reduce(QudaFieldLocation location, const T *v, count_t n_items)
  {
    std::vector<typename reducer::reduce_t> result = {0.0};
    std::vector<const T *> v_ = {v};
    transform_reduce<reducer>(location, result, v_, n_items, identity<T>());
    return result[0];
  }

  // explicit instantiation list for transform_reduce
  // abs
  template void transform_reduce<plus<double>, complex<double>, unsigned int, abs_<double, double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_<double, double>);
  template void transform_reduce<plus<double>, complex<float>, unsigned int, abs_<double, float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, abs_<double, float>);
  template void transform_reduce<plus<double>, complex<int>, unsigned int, abs_<double, int>>(
    QudaFieldLocation, std::vector<double> &, std::vector<quda::complex<int> *> const &, unsigned int, abs_<double, int>);
  template void transform_reduce<plus<double>, complex<short>, unsigned int, abs_<double, short>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, abs_<double, short>);
  template void transform_reduce<plus<double>, complex<signed char>, unsigned int, abs_<double, signed char>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int, abs_<double, signed char>);
  template double transform_reduce<plus<double>, complex<double>, unsigned long, abs_<double, double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_<double, double>);
  template double transform_reduce<plus<double>, complex<float>, unsigned long, abs_<double, float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_<double, float>);
  // square
  template void transform_reduce<plus<double>, complex<double>, unsigned int, square_<double, double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, square_<double, double>);
  template void transform_reduce<plus<double>, complex<float>, unsigned int, square_<double, float>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<float> *> const &, unsigned int, square_<double, float>);
  template void transform_reduce<plus<double>, complex<int>, unsigned int, square_<double, int>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<int> *> const &, unsigned int, square_<double, int>);
  template void transform_reduce<plus<double>, complex<signed char>, unsigned int, square_<double, signed char>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<signed char> *> const &, unsigned int, square_<double, signed char>);
  template void transform_reduce<plus<double>, complex<short>, unsigned int, square_<double, short>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<short> *> const &, unsigned int, square_<double, short>);
  template double transform_reduce<plus<double>, complex<double>, unsigned long, square_<double, double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, square_<double, double>);
  template double transform_reduce<plus<double>, complex<float>, unsigned long, square_<double, float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, square_<double, float>);
  template double transform_reduce<plus<double>, complex<float>, unsigned int, square_<double, float>>(
    QudaFieldLocation, complex<float> const *, unsigned int, square_<double, float>);
  template double transform_reduce<plus<double>, complex<short>, unsigned int, square_<double, short>>(
    QudaFieldLocation, complex<short> const *, unsigned int, square_<double, short>);
  // abs_max
  template void transform_reduce<maximum<double>, complex<double>, unsigned int, abs_max_<double, double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_max_<double, double>);
  template void transform_reduce<maximum<float>, complex<int>, unsigned int, abs_max_<float, int>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<int> *> const &, unsigned int, abs_max_<float, int>);
  template void transform_reduce<maximum<float>, complex<float>, unsigned int, abs_max_<float, float>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<float> *> const &, unsigned int, abs_max_<float, float>);
  template void transform_reduce<maximum<float>, complex<signed char>, unsigned int, abs_max_<float, signed char>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<signed char> *> const &, unsigned int, abs_max_<float, signed char>);
  template void transform_reduce<maximum<float>, complex<short>, unsigned int, abs_max_<float, short>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<short> *> const &, unsigned int, abs_max_<float, short>);
  template double transform_reduce<maximum<double>, complex<double>, unsigned long, abs_max_<double, double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_max_<double, double>);
  template double transform_reduce<maximum<double>, complex<double>, unsigned int, abs_max_<double, double>>(
    QudaFieldLocation, complex<double> const *, unsigned int, abs_max_<double, double>);
  template float transform_reduce<maximum<float>, complex<float>, unsigned long, abs_max_<float, float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_max_<float, float>);
  template float transform_reduce<maximum<float>, complex<float>, unsigned int, abs_max_<float, float>>(
    QudaFieldLocation, complex<float> const *, unsigned int, abs_max_<float, float>);
  template float transform_reduce<maximum<float>, complex<short>, unsigned long, abs_max_<float, short>>(
    QudaFieldLocation, complex<short> const *, unsigned long, abs_max_<float, short>);
  template float transform_reduce<maximum<float>, complex<short>, unsigned int, abs_max_<float, short>>(
    QudaFieldLocation, complex<short> const*, unsigned int, abs_max_<float, short>);
  // abs_min
  template void transform_reduce<minimum<double>, complex<double>, unsigned int, abs_min_<double, double>>(
    QudaFieldLocation, std::vector<double> &, std::vector<complex<double> *> const &, unsigned int, abs_min_<double, double>);
  template void transform_reduce<minimum<float>, complex<float>, unsigned int, abs_min_<float, float>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<float> *> const &, unsigned int, abs_min_<float, float>);
  template void transform_reduce<minimum<float>, complex<int>, unsigned int, abs_min_<float, int>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<int> *> const &, unsigned int, abs_min_<float, int>);
  template void transform_reduce<minimum<float>, complex<short>, unsigned int, abs_min_<float, short>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<short> *> const &, unsigned int, abs_min_<float, short>);
  template void transform_reduce<minimum<float>, complex<signed char>, unsigned int, abs_min_<float, signed char>>(
    QudaFieldLocation, std::vector<float> &, std::vector<complex<signed char> *> const &, unsigned int, abs_min_<float, signed char>);
  template double transform_reduce<minimum<double>, complex<double>, unsigned long, abs_min_<double, double>>(
    QudaFieldLocation, complex<double> const *, unsigned long, abs_min_<double, double>);
  template float transform_reduce<minimum<float>, complex<float>, unsigned long, abs_min_<float, float>>(
    QudaFieldLocation, complex<float> const *, unsigned long, abs_min_<float, float>);

} // namespace quda
