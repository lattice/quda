#include "hip/hip_runtime.h"
/*
  Spinor reordering and copying routines.  These are implemented to
  un on both CPU and GPU.  Here we are templating on the following:
  - input precision
  - output precision
  - number of colors
  - number of spins
  - field ordering
*/

#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <utility> // for std::swap
#include <random_quda.h>

namespace quda {

  using namespace colorspinor;

  template<typename real, int Ns, int Nc, QudaFieldOrder order>
  struct Arg {
    typedef typename colorspinor::FieldOrderCB<real,Ns,Nc,1,order> V;
    V v;
    const int nParity;
    const int volumeCB;
    RNG rng;
    Arg(ColorSpinorField &v, RNG &rng) : v(v), nParity(v.SiteSubset()), volumeCB(v.VolumeCB()), rng(rng) { }
  };

  template<typename real, typename Arg> // Gauss
  __device__ __host__ inline void genGauss(Arg &arg, cuRNGState& localState, int parity, int x_cb, int s, int c) {
    real phi = 2.0*M_PI*Random<real>(localState);
    real radius = Random<real>(localState);
    radius = sqrt(-1.0 * log(radius));
    arg.v(parity, x_cb, s, c) = complex<real>(radius*cos(phi),radius*sin(phi));
  }

  template<typename real, typename Arg> // Uniform
  __device__ __host__ inline void genUniform(Arg &arg, cuRNGState& localState, int parity, int x_cb, int s, int c) {
    real x = Random<real>(localState);
    real y = Random<real>(localState);
    arg.v(parity, x_cb, s, c) = complex<real>(x, y);
  }

  /** CPU function to reorder spinor fields.  */
  template <typename real, int Ns, int Nc, QudaNoiseType type, typename Arg> void SpinorNoiseCPU(Arg &arg)
  {

    for (int parity = 0; parity < arg.nParity; parity++) {
      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) {
        for (int s = 0; s < Ns; s++) {
          for (int c = 0; c < Nc; c++) {
            cuRNGState localState = arg.rng.State()[parity * arg.volumeCB + x_cb];
            if (type == QUDA_NOISE_GAUSS)
              genGauss<real>(arg, localState, parity, x_cb, s, c);
            else if (type == QUDA_NOISE_UNIFORM)
              genUniform<real>(arg, localState, parity, x_cb, s, c);
            arg.rng.State()[parity * arg.volumeCB + x_cb] = localState;
          }
        }
      }
    }
  }

  /** CUDA kernel to reorder spinor fields.  Adopts a similar form as the CPU version, using the same inlined functions. */
  template <typename real, int Ns, int Nc, QudaNoiseType type, typename Arg>
    __global__ void SpinorNoiseGPU(Arg arg) {

    int x_cb = blockIdx.x * blockDim.x + threadIdx.x;
    if (x_cb >= arg.volumeCB) return;

    int parity = blockIdx.y * blockDim.y + threadIdx.y;
    if (parity >= arg.nParity) return;

    cuRNGState localState = arg.rng.State()[parity * arg.volumeCB + x_cb];
    for (int s=0; s<Ns; s++) {
      for (int c=0; c<Nc; c++) {
        if (type == QUDA_NOISE_GAUSS) genGauss<real>(arg, localState, parity, x_cb, s, c);
        else if (type == QUDA_NOISE_UNIFORM) genUniform<real>(arg, localState, parity, x_cb, s, c);
      }
    }
    arg.rng.State()[parity * arg.volumeCB + x_cb] = localState;
  }

  template <typename real, int Ns, int Nc, QudaNoiseType type, typename Arg>
  class SpinorNoise : TunableVectorY {
    Arg &arg;
    const ColorSpinorField &meta; // this reference is for meta data only

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return meta.VolumeCB(); }

  public:
    SpinorNoise(Arg &arg, const ColorSpinorField &meta)
      : TunableVectorY(meta.SiteSubset()), arg(arg), meta(meta) {
      strcpy(aux, meta.AuxString());
      strcat(aux, meta.Location()==QUDA_CUDA_FIELD_LOCATION ? ",GPU" : ",CPU");
    }

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      SpinorNoiseGPU<real, Ns, Nc, type><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    }

    bool advanceTuneParam(TuneParam &param) const {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) return Tunable::advanceTuneParam(param);
      else return false;
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    long long flops() const { return 0; }
    long long bytes() const { return meta.Bytes(); }
    void preTune() { arg.rng.backup(); }
    void postTune(){ arg.rng.restore(); }
  };

  template <typename real, int Ns, int Nc, QudaFieldOrder order>
  void spinorNoise(ColorSpinorField &in, RNG &rngstate, QudaNoiseType type) {
    Arg<real, Ns, Nc, order> arg(in, rngstate);
    switch (type) {
    case QUDA_NOISE_GAUSS:
      {
        SpinorNoise<real, Ns, Nc, QUDA_NOISE_GAUSS, Arg<real, Ns, Nc, order> > noise(arg, in);
        noise.apply(0);
        break;
      }
    case QUDA_NOISE_UNIFORM:
      {
        SpinorNoise<real, Ns, Nc, QUDA_NOISE_UNIFORM, Arg<real, Ns, Nc, order> > noise(arg, in);
        noise.apply(0);
        break;
      }
    default:
      errorQuda("Noise type %d not implemented", type);
    }
  }

  /** Decide on the input order*/
  template <typename real, int Ns, int Nc>
  void spinorNoise(ColorSpinorField &in, RNG &rngstate, QudaNoiseType type)
  {
    if (in.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
      spinorNoise<real,Ns,Nc,QUDA_FLOAT2_FIELD_ORDER>(in, rngstate, type);
    } else if (in.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) {
      spinorNoise<real,Ns,Nc,QUDA_FLOAT4_FIELD_ORDER>(in, rngstate, type);
    } else {
      errorQuda("Order %d not defined (Ns=%d, Nc=%d)", in.FieldOrder(), Ns, Nc);
    }
  }

  template <typename real, int Ns>
  void spinorNoise(ColorSpinorField &src, RNG& randstates, QudaNoiseType type)
  {
    if (src.Ncolor() == 3) {
      spinorNoise<real,Ns,3>(src, randstates, type);
    } else if (src.Ncolor() == 6) {
      spinorNoise<real,Ns,6>(src, randstates, type);
    } else if (src.Ncolor() == 24) {
      spinorNoise<real,Ns,24>(src, randstates, type);
    } else if (src.Ncolor() == 32) {
      spinorNoise<real,Ns,32>(src, randstates, type);
    } else if (src.Ncolor() == 64) {
      spinorNoise<real,Ns,64>(src, randstates, type);
    } else if (src.Ncolor() == 96) {
      spinorNoise<real,Ns,96>(src, randstates, type);
    } else {
      errorQuda("nColor = %d not implemented", src.Ncolor());
    }
  }

  template <typename real>
  void spinorNoise(ColorSpinorField &src, RNG& randstates, QudaNoiseType type)
  {
    if (src.Nspin() == 4) {
#ifdef NSPIN4
      spinorNoise<real,4>(src, randstates, type);
#else
      errorQuda("spinorNoise has not been built for nSpin=%d fields", src.Nspin());
#endif
    } else if (src.Nspin() == 2) {
#ifdef NSPIN2
      spinorNoise<real,2>(src, randstates, type);
#else
      errorQuda("spinorNoise has not been built for nSpin=%d fields", src.Nspin());
#endif
    } else if (src.Nspin() == 1) {
#ifdef NSPIN1
      spinorNoise<real,1>(src, randstates, type);
#else
      errorQuda("spinorNoise has not been built for nSpin=%d fields", src.Nspin());
#endif
    } else {
      errorQuda("Nspin = %d not implemented", src.Nspin());
    }
  }

  void spinorNoise(ColorSpinorField &src_, RNG &randstates, QudaNoiseType type)
  {
    // if src is a CPU field then create GPU field
    ColorSpinorField *src = &src_;
    if (src_.Location() == QUDA_CPU_FIELD_LOCATION || src_.Precision() < QUDA_SINGLE_PRECISION) {
      ColorSpinorParam param(src_);
      QudaPrecision prec = std::max(src_.Precision(), QUDA_SINGLE_PRECISION);
      param.setPrecision(prec, prec, true); // change to native field order
      param.create = QUDA_NULL_FIELD_CREATE;
      param.location = QUDA_CUDA_FIELD_LOCATION;
      src = ColorSpinorField::Create(param);
    }

    switch (src->Precision()) {
    case QUDA_DOUBLE_PRECISION: spinorNoise<double>(*src, randstates, type); break;
    case QUDA_SINGLE_PRECISION: spinorNoise<float>(*src, randstates, type); break;
    default: errorQuda("Precision %d not implemented", src->Precision());
    }

    if (src != &src_) {
      src_ = *src; // upload result
      delete src;
    }
  }

  void spinorNoise(ColorSpinorField &src, unsigned long long seed, QudaNoiseType type)
  {
    RNG *randstates = new RNG(src, seed);
    randstates->Init();
    spinorNoise(src, *randstates, type);
    randstates->Release();
    delete randstates;
  }

} // namespace quda
