#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <unitarization_links.h>
#include <gauge_tools.h>

#include <FFT_Plans.h>
#include <instantiate.h>

#include <tunable_nd.h>
#include <tunable_reduction.h>
#include <kernels/gauge_fix_fft.cuh>

namespace quda {

  template <typename Float>
  class GaugeFixFFTRotate : TunableKernel1D {
    template <int dir> using Arg = GaugeFixFFTRotateArg<Float, dir>;
    GaugeField &data;
    complex<Float> *tmp0;
    complex<Float> *tmp1;
    int dir;
    unsigned int minThreads() const { return data.Volume(); }

  public:
    GaugeFixFFTRotate(GaugeField &data) :
      TunableKernel1D(data),
      data(data),
      dir(0) {}

    void setDirection(int dir_, complex<Float> *data_in, complex<Float> *data_out)
    {
      dir = dir_;
      tmp0 = data_in;
      tmp1 = data_out;
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      switch (dir) {
      case 0: launch<FFTrotate>(tp, stream, Arg<0>(data, tmp0, tmp1)); break;
      case 1: launch<FFTrotate>(tp, stream, Arg<1>(data, tmp0, tmp1)); break;
      default: errorQuda("Error in GaugeFixFFTRotate option");
      }
    }

    long long flops() const { return 0; }
    long long bytes() const { return 4 * sizeof(Float) * data.Volume(); }
  };

  template <typename Arg>
  class GaugeFixQuality : TunableReduction2D<> {
    Arg &arg;
    const GaugeField &meta;

  public:
    GaugeFixQuality(Arg &arg, const GaugeField &meta) :
      TunableReduction2D(meta),
      arg(arg),
      meta(meta) {}

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      launch<FixQualityFFT>(arg.result, tp, stream, arg);

      arg.result[0] /= static_cast<double>(3 * Arg::gauge_dir * meta.Volume());
      arg.result[1] /= static_cast<double>(3 * meta.Volume());
    }

    long long flops() const { return (36 * Arg::gauge_dir + 65) * meta.Volume(); }
    long long bytes() const
    { return (Arg::gauge_dir * meta.Bytes() / 4) + 12 * meta.Volume() * meta.Precision(); }
  };

  enum GaugeFixFFTKernel {
    KERNEL_SET_INVPSQ,
    KERNEL_NORMALIZE,
    KERNEL_GX,
    KERNEL_UEO
  };

  template <typename Arg> class GaugeFixerFFT : TunableKernel2D {
    Arg &arg;
    const GaugeField &field;
    GaugeFixFFTKernel type;
    char aux_tmp[TuneKey::aux_n];
    unsigned int minThreads() const { return arg.threads.x; }

  public:
    GaugeFixerFFT(Arg &arg, const GaugeField &field) :
      TunableKernel2D(field, 2),
      arg(arg),
      field(field)
    {
      strcpy(aux_tmp, aux);
    }

    void set_type(GaugeFixFFTKernel type) {
      this->type = type;
      strcpy(aux, aux_tmp);
      switch (type) {
      case KERNEL_SET_INVPSQ: strcat(aux, ",set_invpsq"); break;
      case KERNEL_NORMALIZE: strcat(aux, ",normalize"); break;
      case KERNEL_GX: strcat(aux, ",gx"); break;
      case KERNEL_UEO:
        strcat(aux, ",ueo");
#ifdef GAUGEFIXING_DONT_USE_GX
        strcat(aux, "_new");
#endif
        break;
      default: errorQuda("Unknown kernel type %d", type);
      }
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      switch (type) {
      case KERNEL_SET_INVPSQ: launch<set_invpsq>(tp, stream, arg); break;
      case KERNEL_NORMALIZE: launch<mult_norm_2d>(tp, stream, arg); break;
      case KERNEL_GX: launch<GX>(tp, stream, arg); break;
#ifdef GAUGEFIXING_DONT_USE_GX
      case KERNEL_UEO: launch<U_EO_NEW>(tp, stream, arg); break;
#else
      case KERNEL_UEO: launch<U_EO>(tp, stream, arg); break;
#endif //GAUGEFIXING_DONT_USE_GX
      default: errorQuda("Unexpected kernel type %d", type);
      }
    }

    void preTune()
    {
      switch (type) {
      case KERNEL_NORMALIZE: std::swap(arg.gx, arg.delta); break; // delta is irrelevant here, so use as backup
      case KERNEL_UEO: field.backup(); break;
      default: break;
      }
    }

    void postTune()
    {
      switch (type) {
      case KERNEL_NORMALIZE: std::swap(arg.gx, arg.delta); break;
      case KERNEL_UEO: field.restore(); break;
      default: break;
      }
    }

    long long flops() const
    {
      switch (type) {
      case KERNEL_SET_INVPSQ: return 2 * field.Volume();
      case KERNEL_NORMALIZE: return 2 * field.Volume();
      case KERNEL_GX: return (arg.elems == 6 ? 208 : 166) * field.Volume();
#ifdef GAUGEFIXING_DONT_USE_GX
      case KERNEL_UEO: return 2414 * field.Volume();
#else
      case KERNEL_UEO: return (arg.elems == 6 ? 1794 : 1536) * field.Volume();
#endif
      default: errorQuda("Unexpected kernel type %d", type); return 0;
      }
    }

    long long bytes() const
    {
      switch (type) {
      case KERNEL_SET_INVPSQ: return sizeof(typename Arg::Float) * field.Volume();
      case KERNEL_NORMALIZE: return 3 * sizeof(typename Arg::Float) * field.Volume();
      case KERNEL_GX: return 4 * arg.elems * field.Precision() * field.Volume();
#ifdef GAUGEFIXING_DONT_USE_GX
      case KERNEL_UEO: return field.Bytes() + (5 * 12 * sizeof(typename Arg::Float)) * field.Volume();
#else
      case KERNEL_UEO: return 26 * arg.elems * field.Precision() * field.Volume();
#endif
      default: errorQuda("Unexpected kernel type %d", type); return 0;
      }
    }
  };

  template <typename Float, QudaReconstructType recon, int gauge_dir>
  void gaugeFixingFFT(GaugeField& data, QudaGaugeFixParam &fix_param)
  {
    TimeProfile profileInternalGaugeFixFFT("InternalGaugeFixQudaFFT", false);
    
    QudaBoolean autotune = fix_param.fft_autotune;
    double alpha0 = fix_param.fft_alpha;
    double tolerance = fix_param.tolerance;
    QudaBoolean theta_condition = fix_param.theta_condition;
    int steps = fix_param.maxiter;
    int verbose_interval = fix_param.verbosity_interval;
    
    profileInternalGaugeFixFFT.TPSTART(QUDA_PROFILE_COMPUTE);

    if (getVerbosity() >= QUDA_SUMMARIZE) {
      if(autotune == QUDA_BOOLEAN_TRUE) printfQuda("\tAuto tune active: alpha will be adjusted as the algorithm progresses\n");
      else printfQuda("\tAuto tune not active: alpha will remain constant as the algorithm progresses\n");      
      printfQuda("\tAlpha parameter of the Steepest Descent Method: %e\n", alpha0);
      printfQuda("\tTolerance: %e\n", tolerance);
      printfQuda("\tStop criterion method: %s\n", theta_condition == QUDA_BOOLEAN_TRUE ? "Theta" : "Delta");
      printfQuda("\tMaximum number of iterations: %d\n", steps);
      printfQuda("\tPrint convergence results at every %d steps\n", verbose_interval);
    }
    
    unsigned int delta_pad = data.X()[0] * data.X()[1] * data.X()[2] * data.X()[3];
    int4 size = make_int4(data.X()[0], data.X()[1], data.X()[2], data.X()[3]);
    FFTPlanHandle plan_xy;
    FFTPlanHandle plan_zt;

    GaugeFixArg<Float, recon> arg(data, alpha0);
    SetPlanFFT2DMany(plan_zt, size, 0, data.Precision());     //for space and time ZT
    SetPlanFFT2DMany(plan_xy, size, 1, data.Precision());    //with space only XY

    GaugeFixFFTRotate<Float> GFRotate(data);

    GaugeFixerFFT<decltype(arg)> gfix(arg, data);
    gfix.set_type(KERNEL_SET_INVPSQ);
    gfix.apply(device::get_default_stream());

    GaugeFixQualityFFTArg<Float, recon, gauge_dir> argQ(data, arg.delta);
    GaugeFixQuality<decltype(argQ)> gfixquality(argQ, data);
    gfixquality.apply(device::get_default_stream());
    double action0 = argQ.getAction();
    if(getVerbosity() >= QUDA_SUMMARIZE) printf("Step: %05d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());

    double diff = 0.0;
    int iter = 0;
    for (iter = 0; iter < steps; iter++) {
      for (int k = 0; k < 6; k++) {
        //------------------------------------------------------------------------
        // Set a pointer do the element k in lattice volume
        // each element is stored with stride lattice volume
        // it uses gx as temporary array!!!!!!
        //------------------------------------------------------------------------
        complex<Float> *_array = arg.delta + k * delta_pad;
        //////  2D FFT + 2D FFT
        //------------------------------------------------------------------------
        // Perform FFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, _array, arg.gx, FFT_FORWARD);
        //------------------------------------------------------------------------
        // Rotate hypercube, xyzt -> ztxy
        //------------------------------------------------------------------------
        GFRotate.setDirection(0, arg.gx, _array);
        GFRotate.apply(device::get_default_stream());
        //------------------------------------------------------------------------
        // Perform FFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, _array, arg.gx, FFT_FORWARD);
        //------------------------------------------------------------------------
        // Normalize FFT and apply pmax^2/p^2
        //------------------------------------------------------------------------
        gfix.set_type(KERNEL_NORMALIZE);
        gfix.apply(device::get_default_stream());
        //------------------------------------------------------------------------
        // Perform IFFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, arg.gx, _array, FFT_INVERSE);
        //------------------------------------------------------------------------
        // Rotate hypercube, ztxy -> xyzt
        //------------------------------------------------------------------------
        GFRotate.setDirection(1, _array, arg.gx);
        GFRotate.apply(device::get_default_stream());
        //------------------------------------------------------------------------
        // Perform IFFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, arg.gx, _array, FFT_INVERSE);
      }

#ifndef GAUGEFIXING_DONT_USE_GX
      //------------------------------------------------------------------------
      // Calculate g(x)
      // ------------------------------------------------------------------------
      // (using GX - else without using GX, gx will be created only
      // for plane rotation but with less size)
      gfix.set_type(KERNEL_GX);
      gfix.apply(device::get_default_stream());
#endif
      //------------------------------------------------------------------------
      // Apply gauge fix to current gauge field
      //------------------------------------------------------------------------
      gfix.set_type(KERNEL_UEO);
      gfix.apply(device::get_default_stream());

      //------------------------------------------------------------------------
      // Measure gauge quality and recalculate new Delta(x)
      //------------------------------------------------------------------------
      gfixquality.apply(device::get_default_stream());
      double action = argQ.getAction();
      diff = abs(action0 - action);
      if ((iter % verbose_interval) == (verbose_interval - 1) && getVerbosity() >= QUDA_SUMMARIZE)
        printf("Step: %05d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
      if ( autotune == QUDA_BOOLEAN_TRUE && ((action - action0) < -1e-14) ) {
        if ( arg.alpha > 0.01 ) {
          arg.alpha = 0.95 * arg.alpha;
          if(getVerbosity() >= QUDA_SUMMARIZE) printf("Changing alpha down -> %.4e\n", arg.alpha);
        }
      }
      //------------------------------------------------------------------------
      // Check gauge fix quality criterion
      //------------------------------------------------------------------------
      if ( theta_condition == QUDA_BOOLEAN_TRUE ) {   if ( argQ.getTheta() < tolerance ) break; }
      else { if ( diff < tolerance ) break; }

      action0 = action;
    }
    if ((iter % verbose_interval) != 0 && getVerbosity() >= QUDA_SUMMARIZE)
      printf("Step: %05d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter, argQ.getAction(), argQ.getTheta(), diff);
    
    // Reunitarize at end
    const double unitarize_eps = 1e-14;
    const double max_error = 1e-10;
    const int reunit_allow_svd = 1;
    const int reunit_svd_only  = 0;
    const double svd_rel_error = 1e-6;
    const double svd_abs_error = 1e-6;
    setUnitarizeLinksConstants(unitarize_eps, max_error,
                               reunit_allow_svd, reunit_svd_only,
                               svd_rel_error, svd_abs_error);
    int *num_failures_h = static_cast<int*>(mapped_malloc(sizeof(int)));
    int *num_failures_d = static_cast<int*>(get_mapped_device_pointer(num_failures_h));

    *num_failures_h = 0;
    unitarizeLinks(data, data, num_failures_d);
    if (*num_failures_h > 0) errorQuda("Error in the unitarization (%d errors)\n", *num_failures_h);
    // end reunitarize

    arg.free();
    FFTDestroyPlan(plan_zt);
    FFTDestroyPlan(plan_xy);
    qudaDeviceSynchronize();
    profileInternalGaugeFixFFT.TPSTOP(QUDA_PROFILE_COMPUTE);


    double secs = profileInternalGaugeFixFFT.Last(QUDA_PROFILE_COMPUTE);
    double fftflop = 5.0 * (log2((double)( data.X()[0] * data.X()[1]) ) + log2( (double)(data.X()[2] * data.X()[3] )));
    fftflop *= (double)data.Volume();
    gfix.set_type(KERNEL_SET_INVPSQ);
    double gflops = gfix.flops() + gfixquality.flops();
    double gbytes = gfix.bytes() + gfixquality.bytes();
    gfix.set_type(KERNEL_NORMALIZE);
    double flop = gfix.flops() * recon / 2;
    double byte = gfix.bytes() * recon / 2;
    flop += (GFRotate.flops() + fftflop) * (recon / 2) * 2;
    byte += GFRotate.bytes() * (recon / 2) * 4;     //includes FFT reads, assuming 1 read and 1 write per site
#ifndef GAUGEFIXING_DONT_USE_GX
    gfix.set_type(KERNEL_GX);
    flop += gfix.flops();
    byte += gfix.bytes();
#endif
    gfix.set_type(KERNEL_UEO);
    flop += gfix.flops();
    byte += gfix.bytes();
    flop += gfixquality.flops();
    byte += gfixquality.bytes();
    gflops += flop * iter;
    gbytes += byte * iter;
    gflops += 4588.0 * data.Volume(); //Reunitarize at end
    gbytes += 2 * data.Bytes(); //Reunitarize at end
    
    gflops = (gflops * 1e-9) / (secs);
    gbytes = gbytes / (secs * 1e9);
    if (getVerbosity() > QUDA_SUMMARIZE)
      printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
    
    host_free(num_failures_h);
  }

  template<typename Float, int nColors, QudaReconstructType recon> struct GaugeFixingFFT {
    GaugeFixingFFT(GaugeField& data, QudaGaugeFixParam &fix_param)
    {
      if (fix_param.gauge_dir == 4) {
	if (getVerbosity() > QUDA_SUMMARIZE) printfQuda("Starting Landau gauge fixing with FFTs\n");
        gaugeFixingFFT<Float, recon, 4>(data, fix_param);
      } else if (fix_param.gauge_dir == 3) {
	if (getVerbosity() > QUDA_SUMMARIZE) printfQuda("Starting Coulomb gauge fixing with FFTs\n");
        gaugeFixingFFT<Float, recon, 3>(data, fix_param);	
      } else {
	errorQuda("Unexpected gauge_dir = %d", fix_param.gauge_dir); 
      }
    }
  };

  /**
   * @brief Gauge fixing with Steepest descent method with FFTs with support for single GPU only.
   * @param[in,out] data, quda gauge field
   * @param[in] fix_param Parameter struct defining the gauge fixing
   */
#if defined(GPU_GAUGE_ALG)
  void gaugeFixingFFT(GaugeField& data, QudaGaugeFixParam &fix_param)
  {
    if (comm_partitioned()) errorQuda("Gauge Fixing with FFTs in multi-GPU support NOT implemented yet!");
    instantiate<GaugeFixingFFT, ReconstructNo12>(data, fix_param);
  }
#else
  void gaugeFixingFFT(GaugeField&, QudaGaugeFixParam &)
  {
    errorQuda("Gauge fixing has bot been built");
  }
#endif

}
