#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <unitarization_links.h>
#include <atomic.cuh>
#include <cub_helper.cuh>
#include <index_helper.cuh>

#include <hipfft/hipfft.h>

#ifdef GPU_GAUGE_ALG
#include <CUFFT_Plans.h>
#endif

namespace quda {

#ifdef GPU_GAUGE_ALG

//UNCOMMENT THIS IF YOU WAN'T TO USE LESS MEMORY
#define GAUGEFIXING_DONT_USE_GX
//Without using the precalculation of g(x),
//we loose some performance, because Delta(x) is written in normal lattice coordinates need for the FFTs
//and the gauge array in even/odd format

#ifdef HOST_DEBUG
#ifdef GAUGEFIXING_DONT_USE_GX
#warning Not using precalculated g(x)
#else
#warning Using precalculated g(x)
#endif
#endif


#ifndef FL_UNITARIZE_PI
#define FL_UNITARIZE_PI 3.14159265358979323846
#endif

  template <typename Float>
  struct GaugeFixFFTRotateArg {
    int threads;     // number of active threads required
    int X[4];     // grid dimensions
    complex<Float> *tmp0;
    complex<Float> *tmp1;
    GaugeFixFFTRotateArg(const cudaGaugeField &data){
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      threads = X[0] * X[1] * X[2] * X[3];
      tmp0 = 0;
      tmp1 = 0;
    }
  };

  template <int direction, typename Float>
  __global__ void fft_rotate_kernel_2D2D(GaugeFixFFTRotateArg<Float> arg){ //Cmplx *data_in, Cmplx *data_out){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= arg.threads ) return;
    if ( direction == 0 ) {
      int x3 = id / (arg.X[0] * arg.X[1] * arg.X[2]);
      int x2 = (id / (arg.X[0] * arg.X[1])) % arg.X[2];
      int x1 = (id / arg.X[0]) % arg.X[1];
      int x0 = id % arg.X[0];

      int id  =  x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0];
      int id_out =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
      arg.tmp1[id_out] = arg.tmp0[id];
      //data_out[id_out] = data_in[id];
    }
    if ( direction == 1 ) {

      int x1 = id / (arg.X[2] * arg.X[3] * arg.X[0]);
      int x0 = (id / (arg.X[2] * arg.X[3])) % arg.X[0];
      int x3 = (id / arg.X[2]) % arg.X[3];
      int x2 = id % arg.X[2];

      int id  =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
      int id_out =  x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0];
      arg.tmp1[id_out] = arg.tmp0[id];
      //data_out[id_out] = data_in[id];
    }
  }






  template<typename Float>
  class GaugeFixFFTRotate : Tunable {
    GaugeFixFFTRotateArg<Float> arg;
    int direction;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false; } // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                              // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeFixFFTRotate(GaugeFixFFTRotateArg<Float> &arg) : arg(arg) {
      direction = 0;
    }
    ~GaugeFixFFTRotate () {
    }
    void setDirection(int dir, complex<Float> *data_in, complex<Float> *data_out){
      direction = dir;
      arg.tmp0 = data_in;
      arg.tmp1 = data_out;
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if ( direction == 0 )
        fft_rotate_kernel_2D2D<0, Float > <<< tp.grid, tp.block, 0, stream >>> (arg);
      else if ( direction == 1 )
        fft_rotate_kernel_2D2D<1, Float > <<< tp.grid, tp.block, 0, stream >>> (arg);
      else
        errorQuda("Error in GaugeFixFFTRotate option.\n");
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    long long flops() const {
      return 0;
    }
    long long bytes() const {
      return 4LL * sizeof(Float) * arg.threads;
    }

  };


  template <typename Float, typename Gauge>
  struct GaugeFixQualityArg : public ReduceArg<double2> {
    int threads;     // number of active threads required
    int X[4];     // grid dimensions
    Gauge dataOr;
    complex<Float> *delta;

    GaugeFixQualityArg(const Gauge &dataOr, const cudaGaugeField &data, complex<Float> * delta)
      : ReduceArg<double2>(), dataOr(dataOr), delta(delta) {
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      threads = data.VolumeCB();
    }
    double getAction(){ return result_h[0].x; }
    double getTheta(){ return result_h[0].y; }
  };

  template<int blockSize, int Elems, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFix_quality(GaugeFixQualityArg<Float, Gauge> argQ){
    int idx_cb = threadIdx.x + blockIdx.x * blockDim.x;
    int parity = threadIdx.y;

    double2 data = make_double2(0.0,0.0);
    while (idx_cb < argQ.threads) {
      typedef complex<Float> Cmplx;

      int x[4];
      getCoords(x, idx_cb, argQ.X, parity);
      Matrix<Cmplx,3> delta;
      setZero(&delta);
      //idx = linkIndex(x,X);
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, idx_cb, parity);
        delta -= U;
      }
      //18*gauge_dir
      data.x += -delta(0, 0).x - delta(1, 1).x - delta(2, 2).x;
      //2
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, linkIndexM1(x,argQ.X,mu), 1 - parity);
        delta += U;
      }
      //18*gauge_dir
      delta -= conj(delta);
      //18
      //SAVE DELTA!!!!!
      SubTraceUnit(delta);
      int idx = getIndexFull(idx_cb, argQ.X, parity);
      //Saving Delta
      argQ.delta[idx] = delta(0,0);
      argQ.delta[idx + 2 * argQ.threads] = delta(0,1);
      argQ.delta[idx + 4 * argQ.threads] = delta(0,2);
      argQ.delta[idx + 6 * argQ.threads] = delta(1,1);
      argQ.delta[idx + 8 * argQ.threads] = delta(1,2);
      argQ.delta[idx + 10 * argQ.threads] = delta(2,2);
      //12
      data.y += getRealTraceUVdagger(delta, delta);
      //35
      //T=36*gauge_dir+65

      idx_cb += blockDim.x * gridDim.x;
    }

    reduce2d<blockSize,2>(argQ, data);
  }



  template<int Elems, typename Float, typename Gauge, int gauge_dir>
  class GaugeFixQuality : TunableLocalParity {
    GaugeFixQualityArg<Float, Gauge> argQ;
    mutable char aux_string[128];     // used as a label in the autotuner

  private:
    bool tuneGridDim() const { return true; }

  public:
    GaugeFixQuality(GaugeFixQualityArg<Float, Gauge> &argQ)
      : argQ(argQ) {
    }
    ~GaugeFixQuality () { }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      argQ.result_h[0] = make_double2(0.0,0.0);
      LAUNCH_KERNEL_LOCAL_PARITY(computeFix_quality, (*this), tp, stream, argQ, Elems, Float, Gauge, gauge_dir);
      qudaDeviceSynchronize();
      argQ.result_h[0].x  /= (double)(3 * gauge_dir * 2 * argQ.threads);
      argQ.result_h[0].y  /= (double)(3 * 2 * argQ.threads);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << argQ.X[0] << "x" << argQ.X[1] << "x" << argQ.X[2] << "x" << argQ.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu,gaugedir=%d", argQ.threads, sizeof(Float), gauge_dir);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    }

    long long flops() const {
      return (36LL * gauge_dir + 65LL) * 2 * argQ.threads;
    }                                                                         // Only correct if there is no link reconstruction, no cub reduction accounted also
    long long bytes() const {
      return (2LL * gauge_dir + 2LL) * Elems * 2 * argQ.threads * sizeof(Float);
    }                                                                                                    //Not accounting the reduction!!!

  };



  template <typename Float>
  struct GaugeFixArg {
    int threads;     // number of active threads required
    int X[4];     // grid dimensions
    cudaGaugeField &data;
    Float *invpsq;
    complex<Float> *delta;
    complex<Float> *gx;

    GaugeFixArg( cudaGaugeField & data, const int Elems) : data(data){
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
      threads = X[0] * X[1] * X[2] * X[3];
      invpsq = (Float*)device_malloc(sizeof(Float) * threads);
      delta = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads * 6);
#ifdef GAUGEFIXING_DONT_USE_GX
      gx = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads);
#else
      gx = (complex<Float>*)device_malloc(sizeof(complex<Float>) * threads * Elems);
#endif
    }
    void free(){
      device_free(invpsq);
      device_free(delta);
      device_free(gx);
    }
  };




  template <typename Float>
  __global__ void kernel_gauge_set_invpsq(GaugeFixArg<Float> arg){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id >= arg.threads ) return;
    int x1 = id / (arg.X[2] * arg.X[3] * arg.X[0]);
    int x0 = (id / (arg.X[2] * arg.X[3])) % arg.X[0];
    int x3 = (id / arg.X[2]) % arg.X[3];
    int x2 = id % arg.X[2];
    //id  =  x2 + (x3 +  (x0 + x1 * arg.X[0]) * arg.X[3]) * arg.X[2];
    Float sx = sin( (Float)x0 * FL_UNITARIZE_PI / (Float)arg.X[0]);
    Float sy = sin( (Float)x1 * FL_UNITARIZE_PI / (Float)arg.X[1]);
    Float sz = sin( (Float)x2 * FL_UNITARIZE_PI / (Float)arg.X[2]);
    Float st = sin( (Float)x3 * FL_UNITARIZE_PI / (Float)arg.X[3]);
    Float sinsq = sx * sx + sy * sy + sz * sz + st * st;
    Float prcfact = 0.0;
    //The FFT normalization is done here
    if ( sinsq > 0.00001 ) prcfact = 4.0 / (sinsq * (Float)arg.threads);
    arg.invpsq[id] = prcfact;
  }


  template<typename Float>
  class GaugeFixSETINVPSP : Tunable {
    GaugeFixArg<Float> arg;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    bool tuneSharedBytes() const {
      return false;
    }                                                  // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                              // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeFixSETINVPSP(GaugeFixArg<Float> &arg) : arg(arg) { }
    ~GaugeFixSETINVPSP () { }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_gauge_set_invpsq<Float> <<< tp.grid, tp.block, 0, stream >>> (arg);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    long long flops() const {
      return 21 * arg.threads;
    }
    long long bytes() const {
      return sizeof(Float) * arg.threads;
    }

  };

  template<typename Float>
  __global__ void kernel_gauge_mult_norm_2D(GaugeFixArg<Float> arg){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if ( id < arg.threads ) arg.gx[id] = arg.gx[id] * arg.invpsq[id];
  }


  template<typename Float>
  class GaugeFixINVPSP : Tunable {
    GaugeFixArg<Float> arg;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false; } // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                              // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeFixINVPSP(GaugeFixArg<Float> &arg)
      : arg(arg){
      hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_gauge_mult_norm_2D<Float>),   hipFuncCachePreferL1);
    }
    ~GaugeFixINVPSP () {
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_gauge_mult_norm_2D<Float> <<< tp.grid, tp.block, 0, stream >>> (arg);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    void preTune(){
      //since delta contents are irrelevant at this point, we can swap gx with delta
      complex<Float> *tmp = arg.gx;
      arg.gx = arg.delta;
      arg.delta = tmp;
    }
    void postTune(){
      arg.gx = arg.delta;
    }
    long long flops() const {
      return 2LL * arg.threads;
    }
    long long bytes() const {
      return 5LL * sizeof(Float) * arg.threads;
    }

  };



  template <typename Float>
  __host__ __device__ inline void reunit_link( Matrix<complex<Float>,3> &U ){

    complex<Float> t2((Float)0.0, (Float)0.0);
    Float t1 = 0.0;
    //first normalize first row
    //sum of squares of row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(0,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(0,c) *= t1;
    //6
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t2 += conj(U(0,c)) * U(1,c);
    //24
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1,c) -= t2 * U(0,c);
    //24
    //normalize second row
    //sum of squares of row
    t1 = 0.0;
#pragma unroll
    for ( int c = 0; c < 3; c++ ) t1 += norm(U(1,c));
    t1 = (Float)1.0 / sqrt(t1);
    //14
    //used to normalize row
#pragma unroll
    for ( int c = 0; c < 3; c++ ) U(1, c) *= t1;
    //6
    //Reconstruct lat row
    U(2,0) = conj(U(0,1) * U(1,2) - U(0,2) * U(1,1));
    U(2,1) = conj(U(0,2) * U(1,0) - U(0,0) * U(1,2));
    U(2,2) = conj(U(0,0) * U(1,1) - U(0,1) * U(1,0));
    //42
    //T=130
  }

#ifdef GAUGEFIXING_DONT_USE_GX

  template <typename Float, typename Gauge>
  __global__ void kernel_gauge_fix_U_EO_NEW( GaugeFixArg<Float> arg, Gauge dataOr, Float half_alpha){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int parity = threadIdx.y;

    if ( id >= arg.threads/2 ) return;

    typedef complex<Float> Cmplx;

    int x[4];
    getCoords(x, id, arg.X, parity);
    int idx = ((x[3] * arg.X[2] + x[2]) * arg.X[1] + x[1]) * arg.X[0] + x[0];
    Matrix<Cmplx,3> de;
    //Read Delta
    de(0,0) = arg.delta[idx + 0 * arg.threads];
    de(0,1) = arg.delta[idx + 1 * arg.threads];
    de(0,2) = arg.delta[idx + 2 * arg.threads];
    de(1,1) = arg.delta[idx + 3 * arg.threads];
    de(1,2) = arg.delta[idx + 4 * arg.threads];
    de(2,2) = arg.delta[idx + 5 * arg.threads];

    de(1,0) = Cmplx(-de(0,1).x, de(0,1).y);
    de(2,0) = Cmplx(-de(0,2).x, de(0,2).y);
    de(2,1) = Cmplx(-de(1,2).x, de(1,2).y);
    Matrix<Cmplx,3> g;
    setIdentity(&g);
    g += de * half_alpha;
    //36
    reunit_link<Float>( g );
    //130


    for ( int mu = 0; mu < 4; mu++ ) {
      Matrix<Cmplx,3> U = dataOr(mu, id, parity);
      Matrix<Cmplx,3> g0;
      U = g * U;
      //198
      idx = linkNormalIndexP1(x,arg.X,mu);
      //Read Delta
      de(0,0) = arg.delta[idx + 0 * arg.threads];
      de(0,1) = arg.delta[idx + 1 * arg.threads];
      de(0,2) = arg.delta[idx + 2 * arg.threads];
      de(1,1) = arg.delta[idx + 3 * arg.threads];
      de(1,2) = arg.delta[idx + 4 * arg.threads];
      de(2,2) = arg.delta[idx + 5 * arg.threads];

      de(1,0) = Cmplx(-de(0,1).x, de(0,1).y);
      de(2,0) = Cmplx(-de(0,2).x, de(0,2).y);
      de(2,1) = Cmplx(-de(1,2).x, de(1,2).y);

      setIdentity(&g0);
      g0 += de * half_alpha;
      //36
      reunit_link<Float>( g0 );
      //130

      U = U * conj(g0);
      //198
      dataOr(mu, id, parity) = U;
    }
  }


  template<typename Float, typename Gauge>
  class GaugeFixNEW : TunableLocalParity {
    GaugeFixArg<Float> arg;
    Float half_alpha;
    Gauge dataOr;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:

    // since GaugeFixArg is used by other kernels that don't use
    // tunableLocalParity, arg.threads stores Volume and not VolumeCB
    // so we need to divide by two
    unsigned int minThreads() const { return arg.threads/2; }

    public:
    GaugeFixNEW(Gauge & dataOr, GaugeFixArg<Float> &arg, Float alpha)
      : dataOr(dataOr), arg(arg) {
      half_alpha = alpha * 0.5;
      hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_gauge_fix_U_EO_NEW<Float), Gauge>,   hipFuncCachePreferL1);
    }
    ~GaugeFixNEW () { }

    void setAlpha(Float alpha){ half_alpha = alpha * 0.5; }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_gauge_fix_U_EO_NEW<Float, Gauge> <<< tp.grid, tp.block, 0, stream >>> (arg, dataOr, half_alpha);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x" << arg.X[1] << "x" << arg.X[2] << "x" << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    //need this
    void preTune() {
      arg.data.backup();
    }
    void postTune() {
      arg.data.restore();
    }
    long long flops() const {
      return 2414LL * arg.threads;
      //Not accounting here the reconstruction of the gauge if 12 or 8!!!!!!
    }
    long long bytes() const {
      return ( dataOr.Bytes() * 4LL + 5 * 12LL * sizeof(Float)) * arg.threads;
    }

  };



#else
  template <int Elems, typename Float>
  __global__ void kernel_gauge_GX(GaugeFixArg<Float> arg, Float half_alpha){

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if ( id >= arg.threads ) return;

    typedef complex<Float> Cmplx;

    Matrix<Cmplx,3> de;
    //Read Delta
    de(0,0) = arg.delta[id];
    de(0,1) = arg.delta[id + arg.threads];
    de(0,2) = arg.delta[id + 2 * arg.threads];
    de(1,1) = arg.delta[id + 3 * arg.threads];
    de(1,2) = arg.delta[id + 4 * arg.threads];
    de(2,2) = arg.delta[id + 5 * arg.threads];

    de(1,0) = makeComplex(-de(0,1).x, de(0,1).y);
    de(2,0) = makeComplex(-de(0,2).x, de(0,2).y);
    de(2,1) = makeComplex(-de(1,2).x, de(1,2).y);


    Matrix<Cmplx,3> g;
    setIdentity(&g);
    g += de * half_alpha;
    //36
    reunit_link<Float>( g );
    //130
    //gx is represented in even/odd order
    //normal lattice index to even/odd index
    int x3 = id / (arg.X[0] * arg.X[1] * arg.X[2]);
    int x2 = (id / (arg.X[0] * arg.X[1])) % arg.X[2];
    int x1 = (id / arg.X[0]) % arg.X[1];
    int x0 = id % arg.X[0];
    id  =  (x0 + (x1 + (x2 + x3 * arg.X[2]) * arg.X[1]) * arg.X[0]) >> 1;
    id += ((x0 + x1 + x2 + x3) & 1 ) * arg.threads / 2;

    for ( int i = 0; i < Elems; i++ ) arg.gx[id + i * arg.threads] = g.data[i];
    //T=166 for Elems 9
    //T=208 for Elems 6
  }




  template<int Elems, typename Float>
  class GaugeFix_GX : Tunable {
    GaugeFixArg<Float> arg;
    Float half_alpha;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false; } // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                              // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeFix_GX(GaugeFixArg<Float> &arg, Float alpha)
      : arg(arg) {
      half_alpha = alpha * 0.5;
      hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_gauge_GX<Elems), Float>,   hipFuncCachePreferL1);
    }
    ~GaugeFix_GX () {
    }

    void setAlpha(Float alpha){
      half_alpha = alpha * 0.5;
    }


    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_gauge_GX<Elems, Float> <<< tp.grid, tp.block, 0, stream >>> (arg, half_alpha);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    long long flops() const {
      if ( Elems == 6 ) return 208LL * arg.threads;
      else return 166LL * arg.threads;
    }
    long long bytes() const {
      return 4LL * Elems * sizeof(Float) * arg.threads;
    }

  };


  template <int Elems, typename Float, typename Gauge>
  __global__ void kernel_gauge_fix_U_EO( GaugeFixArg<Float> arg, Gauge dataOr){
    int idd = threadIdx.x + blockIdx.x * blockDim.x;

    if ( idd >= arg.threads ) return;

    int parity = 0;
    int id = idd;
    if ( idd >= arg.threads / 2 ) {
      parity = 1;
      id -= arg.threads / 2;
    }
    typedef complex<Float> Cmplx;

    Matrix<Cmplx,3> g;
    //for(int i = 0; i < Elems; i++) g.data[i] = arg.gx[idd + i * arg.threads];
    for ( int i = 0; i < Elems; i++ ) {
      g.data[i] = arg.gx[idd + i * arg.threads];
    }
    if ( Elems == 6 ) {
      g(2,0) = conj(g(0,1) * g(1,2) - g(0,2) * g(1,1));
      g(2,1) = conj(g(0,2) * g(1,0) - g(0,0) * g(1,2));
      g(2,2) = conj(g(0,0) * g(1,1) - g(0,1) * g(1,0));
      //42
    }
    int x[4];
    getCoords(x, id, arg.X, parity);
    for ( int mu = 0; mu < 4; mu++ ) {
      Matrix<Cmplx,3> U = dataOr(mu, id, parity);
      Matrix<Cmplx,3> g0;
      U = g * U;
      //198
      int idm1 = linkIndexP1(x,arg.X,mu);
      idm1 += (1 - parity) * arg.threads / 2;
      //for(int i = 0; i < Elems; i++) g0.data[i] = arg.gx[idm1 + i * arg.threads];
      for ( int i = 0; i < Elems; i++ ) {
        g0.data[i] = arg.gx[idm1 + i * arg.threads];
      }
      if ( Elems == 6 ) {
        g0(2,0) = conj(g0(0,1) * g0(1,2) - g0(0,2) * g0(1,1));
        g0(2,1) = conj(g0(0,2) * g0(1,0) - g0(0,0) * g0(1,2));
        g0(2,2) = conj(g0(0,0) * g0(1,1) - g0(0,1) * g0(1,0));
        //42
      }
      U = U * conj(g0);
      //198
      dataOr.save(mu, id, parity) = U;
    }
    //T=42+4*(198*2+42) Elems=6
    //T=4*(198*2) Elems=9
    //Not accounting here the reconstruction of the gauge if 12 or 8!!!!!!
  }


  template<int Elems, typename Float, typename Gauge>
  class GaugeFix : Tunable {
    GaugeFixArg<Float> arg;
    Gauge dataOr;
    mutable char aux_string[128];     // used as a label in the autotuner
    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      return 0;
    }
    //bool tuneSharedBytes() const { return false; } // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                              // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

    public:
    GaugeFix(Gauge & dataOr, GaugeFixArg<Float> &arg)
      : dataOr(dataOr), arg(arg) {
      hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_gauge_fix_U_EO<Elems), Float, Gauge>,   hipFuncCachePreferL1);
    }
    ~GaugeFix () { }


    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      kernel_gauge_fix_U_EO<Elems, Float, Gauge> <<< tp.grid, tp.block, 0, stream >>> (arg, dataOr);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu", arg.threads, sizeof(Float));
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    //need this
    void preTune() {
      arg.data.backup();
    }
    void postTune() {
      arg.data.restore();
    }
    long long flops() const {
      if ( Elems == 6 ) return 1794LL * arg.threads;
      else return 1536LL * arg.threads;
      //Not accounting here the reconstruction of the gauge if 12 or 8!!!!!!
    }
    long long bytes() const {
      return 26LL * Elems * sizeof(Float) * arg.threads;
    }

  };
#endif
//GAUGEFIXING_DONT_USE_GX


  template<int Elems, typename Float, typename Gauge, int gauge_dir>
  void gaugefixingFFT( Gauge dataOr,  cudaGaugeField& data, \
                       const int Nsteps, const int verbose_interval, \
                       const Float alpha0, const int autotune, const double tolerance, \
                       const int stopWtheta) {

    TimeProfile profileInternalGaugeFixFFT("InternalGaugeFixQudaFFT", false);

    profileInternalGaugeFixFFT.TPSTART(QUDA_PROFILE_COMPUTE);

    Float alpha = alpha0;
    std::cout << "\tAlpha parameter of the Steepest Descent Method: " << alpha << std::endl;
    if ( autotune ) std::cout << "\tAuto tune active: yes" << std::endl;
    else std::cout << "\tAuto tune active: no" << std::endl;
    std::cout << "\tStop criterium: " << tolerance << std::endl;
    if ( stopWtheta ) std::cout << "\tStop criterium method: theta" << std::endl;
    else std::cout << "\tStop criterium method: Delta" << std::endl;
    std::cout << "\tMaximum number of iterations: " << Nsteps << std::endl;
    std::cout << "\tPrint convergence results at every " << verbose_interval << " steps" << std::endl;


    unsigned int delta_pad = data.X()[0] * data.X()[1] * data.X()[2] * data.X()[3];
    int4 size = make_int4( data.X()[0], data.X()[1], data.X()[2], data.X()[3] );
    hipfftHandle plan_xy;
    hipfftHandle plan_zt;

    GaugeFixArg<Float> arg(data, Elems);
    SetPlanFFT2DMany( plan_zt, size, 0, arg.delta);     //for space and time ZT
    SetPlanFFT2DMany( plan_xy, size, 1, arg.delta);    //with space only XY


    GaugeFixFFTRotateArg<Float> arg_rotate(data);
    GaugeFixFFTRotate<Float> GFRotate(arg_rotate);

    GaugeFixSETINVPSP<Float> setinvpsp(arg);
    setinvpsp.apply(0);
    GaugeFixINVPSP<Float> invpsp(arg);


#ifdef GAUGEFIXING_DONT_USE_GX
    //without using GX, gx will be created only for plane rotation but with less size
    GaugeFixNEW<Float, Gauge> gfixNew(dataOr, arg, alpha);
#else
    //using GX
    GaugeFix_GX<Elems, Float> calcGX(arg, alpha);
    GaugeFix<Elems, Float, Gauge> gfix(dataOr, arg);
#endif

    GaugeFixQualityArg<Float, Gauge> argQ(dataOr, data, arg.delta);
    GaugeFixQuality<Elems, Float, Gauge, gauge_dir> gfixquality(argQ);

    gfixquality.apply(0);
    double action0 = argQ.getAction();
    printf("Step: %d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());

    double diff = 0.0;
    int iter = 0;
    for ( iter = 0; iter < Nsteps; iter++ ) {
      for ( int k = 0; k < 6; k++ ) {
        //------------------------------------------------------------------------
        // Set a pointer do the element k in lattice volume
        // each element is stored with stride lattice volume
        // it uses gx as temporary array!!!!!!
        //------------------------------------------------------------------------
        complex<Float> *_array = arg.delta + k * delta_pad;
        //////  2D FFT + 2D FFT
        //------------------------------------------------------------------------
        // Perform FFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, _array, arg.gx, HIPFFT_FORWARD);
        //------------------------------------------------------------------------
        // Rotate hypercube, xyzt -> ztxy
        //------------------------------------------------------------------------
        GFRotate.setDirection(0, arg.gx, _array);
        GFRotate.apply(0);
        //------------------------------------------------------------------------
        // Perform FFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, _array, arg.gx, HIPFFT_FORWARD);
        //------------------------------------------------------------------------
        // Normalize FFT and apply pmax^2/p^2
        //------------------------------------------------------------------------
        invpsp.apply(0);
        //------------------------------------------------------------------------
        // Perform IFFT on zt plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_zt, arg.gx, _array, HIPFFT_BACKWARD);
        //------------------------------------------------------------------------
        // Rotate hypercube, ztxy -> xyzt
        //------------------------------------------------------------------------
        GFRotate.setDirection(1, _array, arg.gx);
        GFRotate.apply(0);
        //------------------------------------------------------------------------
        // Perform IFFT on xy plane
        //------------------------------------------------------------------------
        ApplyFFT(plan_xy, arg.gx, _array, HIPFFT_BACKWARD);
      }
                #ifdef GAUGEFIXING_DONT_USE_GX
      //------------------------------------------------------------------------
      // Apply gauge fix to current gauge field
      //------------------------------------------------------------------------
      gfixNew.apply(0);
                #else
      //------------------------------------------------------------------------
      // Calculate g(x)
      //------------------------------------------------------------------------
      calcGX.apply(0);
      //------------------------------------------------------------------------
      // Apply gauge fix to current gauge field
      //------------------------------------------------------------------------
      gfix.apply(0);
                #endif
      //------------------------------------------------------------------------
      // Measure gauge quality and recalculate new Delta(x)
      //------------------------------------------------------------------------
      gfixquality.apply(0);
      double action = argQ.getAction();
      diff = abs(action0 - action);
      if ((iter % verbose_interval) == (verbose_interval - 1))
        printf("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
      if ( autotune && ((action - action0) < -1e-14) ) {
        if ( alpha > 0.01 ) {
          alpha = 0.95 * alpha;
                                #ifdef GAUGEFIXING_DONT_USE_GX
          gfixNew.setAlpha(alpha);
                                #else
          calcGX.setAlpha(alpha);
                                #endif
          printf(">>>>>>>>>>>>>> Warning: changing alpha down -> %.4e\n", alpha );
        }
      }
      //------------------------------------------------------------------------
      // Check gauge fix quality criterium
      //------------------------------------------------------------------------
      if ( stopWtheta ) {   if ( argQ.getTheta() < tolerance ) break; }
      else { if ( diff < tolerance ) break; }

      action0 = action;
    }
    if ((iter % verbose_interval) != 0 )
      printf("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter, argQ.getAction(), argQ.getTheta(), diff);

    // Reunitarize at end
    const double unitarize_eps = 1e-14;
    const double max_error = 1e-10;
    const int reunit_allow_svd = 1;
    const int reunit_svd_only  = 0;
    const double svd_rel_error = 1e-6;
    const double svd_abs_error = 1e-6;
    setUnitarizeLinksConstants(unitarize_eps, max_error,
                               reunit_allow_svd, reunit_svd_only,
                               svd_rel_error, svd_abs_error);
    int num_failures = 0;
    int* num_failures_dev = static_cast<int*>(pool_device_malloc(sizeof(int)));
    hipMemset(num_failures_dev, 0, sizeof(int));
    unitarizeLinks(data, data, num_failures_dev);
    qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);

    pool_device_free(num_failures_dev);
    if ( num_failures > 0 ) {
      errorQuda("Error in the unitarization\n");
      exit(1);
    }
    // end reunitarize


    arg.free();
    CUFFT_SAFE_CALL(hipfftDestroy(plan_zt));
    CUFFT_SAFE_CALL(hipfftDestroy(plan_xy));
    checkCudaError();
    qudaDeviceSynchronize();
    profileInternalGaugeFixFFT.TPSTOP(QUDA_PROFILE_COMPUTE);

    if (getVerbosity() > QUDA_SUMMARIZE){
      double secs = profileInternalGaugeFixFFT.Last(QUDA_PROFILE_COMPUTE);
      double fftflop = 5.0 * (log2((double)( data.X()[0] * data.X()[1]) ) + log2( (double)(data.X()[2] * data.X()[3] )));
      fftflop *= (double)( data.X()[0] * data.X()[1] * data.X()[2] * data.X()[3] );
      double gflops = setinvpsp.flops() + gfixquality.flops();
      double gbytes = setinvpsp.bytes() + gfixquality.bytes();
      double flop = invpsp.flops() * Elems;
      double byte = invpsp.bytes() * Elems;
      flop += (GFRotate.flops() + fftflop) * Elems * 2;
      byte += GFRotate.bytes() * Elems * 4;     //includes FFT reads, assuming 1 read and 1 write per site
      #ifdef GAUGEFIXING_DONT_USE_GX
      flop += gfixNew.flops();
      byte += gfixNew.bytes();
      #else
      flop += calcGX.flops();
      byte += calcGX.bytes();
      flop += gfix.flops();
      byte += gfix.bytes();
      #endif
      flop += gfixquality.flops();
      byte += gfixquality.bytes();
      gflops += flop * iter;
      gbytes += byte * iter;
      gflops += 4588.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3]; //Reunitarize at end
      gbytes += 8.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3] * dataOr.Bytes() ; //Reunitarize at end

      gflops = (gflops * 1e-9) / (secs);
      gbytes = gbytes / (secs * 1e9);
      printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
    }
  }

  template<int Elems, typename Float, typename Gauge>
  void gaugefixingFFT( Gauge dataOr,  cudaGaugeField& data, const int gauge_dir, \
                       const int Nsteps, const int verbose_interval, const Float alpha, const int autotune, \
                       const double tolerance, const int stopWtheta) {
    if ( gauge_dir != 3 ) {
      printf("Starting Landau gauge fixing with FFTs...\n");
      gaugefixingFFT<Elems, Float, Gauge, 4>(dataOr, data, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
    }
    else {
      printf("Starting Coulomb gauge fixing with FFTs...\n");
      gaugefixingFFT<Elems, Float, Gauge, 3>(dataOr, data, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
    }
  }



  template<typename Float>
  void gaugefixingFFT( cudaGaugeField& data, const int gauge_dir, \
                       const int Nsteps, const int verbose_interval, const Float alpha, const int autotune, \
                       const double tolerance, const int stopWtheta) {

    // Switching to FloatNOrder for the gauge field in order to support RECONSTRUCT_12
    // Need to fix this!!
    //9 and 6 means the number of complex elements used to store g(x) and Delta(x)
    if ( data.isNative() ) {
      if ( data.Reconstruct() == QUDA_RECONSTRUCT_NO ) {
        //printfQuda("QUDA_RECONSTRUCT_NO\n");
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
        gaugefixingFFT<9, Float>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_12 ) {
        //printfQuda("QUDA_RECONSTRUCT_12\n");
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
        gaugefixingFFT<6, Float>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_8 ) {
        //printfQuda("QUDA_RECONSTRUCT_8\n");
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
        gaugefixingFFT<6, Float>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);

      } else {
        errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }
    } else {
      errorQuda("Invalid Gauge Order\n");
    }
  }

#endif // GPU_GAUGE_ALG


  /**
   * @brief Gauge fixing with Steepest descent method with FFTs with support for single GPU only.
   * @param[in,out] data, quda gauge field
   * @param[in] gauge_dir, 3 for Coulomb gauge fixing, other for Landau gauge fixing
   * @param[in] Nsteps, maximum number of steps to perform gauge fixing
   * @param[in] verbose_interval, print gauge fixing info when iteration count is a multiple of this
   * @param[in] alpha, gauge fixing parameter of the method, most common value is 0.08
   * @param[in] autotune, 1 to autotune the method, i.e., if the Fg inverts its tendency we decrease the alpha value 
   * @param[in] tolerance, torelance value to stop the method, if this value is zero then the method stops when iteration reachs the maximum number of steps defined by Nsteps
   * @param[in] stopWtheta, 0 for MILC criterium and 1 to use the theta value
   */
  void gaugefixingFFT( cudaGaugeField& data, const int gauge_dir, \
                       const int Nsteps, const int verbose_interval, const double alpha, const int autotune, \
                       const double tolerance, const int stopWtheta) {

#ifdef GPU_GAUGE_ALG
#ifdef MULTI_GPU
    if(comm_dim_partitioned(0) || comm_dim_partitioned(1) || comm_dim_partitioned(2) || comm_dim_partitioned(3))
      errorQuda("Gauge Fixing with FFTs in multi-GPU support NOT implemented yet!\n");
#endif
    if ( data.Precision() == QUDA_HALF_PRECISION ) {
      errorQuda("Half precision not supported\n");
    }
    if ( data.Precision() == QUDA_SINGLE_PRECISION ) {
      gaugefixingFFT<float> (data, gauge_dir, Nsteps, verbose_interval, (float)alpha, autotune, tolerance, stopWtheta);
    } else if ( data.Precision() == QUDA_DOUBLE_PRECISION ) {
      gaugefixingFFT<double>(data, gauge_dir, Nsteps, verbose_interval, alpha, autotune, tolerance, stopWtheta);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
#else
    errorQuda("Gauge fixing has bot been built");
#endif
  }



}
