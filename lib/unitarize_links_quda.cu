#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <gauge_field.h>
#include <gauge_field_order.h>

#include <tune_quda.h>
#include <quda_matrix.h>
#include <hisq_links_quda.h>

namespace quda{
#ifdef GPU_UNITARIZE

namespace{
  #include <svd_quda.h>
}

#ifndef FL_UNITARIZE_PI
#define FL_UNITARIZE_PI 3.14159265358979323846
#endif
#ifndef FL_UNITARIZE_PI23
#define FL_UNITARIZE_PI23 FL_UNITARIZE_PI*2.0/3.0
#endif 
 
  __constant__ int INPUT_PADDING=0;
  __constant__ int OUTPUT_PADDING=0;
  __constant__ int DEV_MAX_ITER = 20;

  static int HOST_MAX_ITER = 20;

  __constant__ double DEV_FL_MAX_ERROR;
  __constant__ double DEV_FL_UNITARIZE_EPS;
  __constant__ bool   DEV_FL_REUNIT_ALLOW_SVD;
  __constant__ bool   DEV_FL_REUNIT_SVD_ONLY;
  __constant__ double DEV_FL_REUNIT_SVD_REL_ERROR;
  __constant__ double DEV_FL_REUNIT_SVD_ABS_ERROR;
  __constant__ bool   DEV_FL_CHECK_UNITARIZATION;

  static double HOST_FL_MAX_ERROR;
  static double HOST_FL_UNITARIZE_EPS;
  static bool   HOST_FL_REUNIT_ALLOW_SVD;
  static bool   HOST_FL_REUNIT_SVD_ONLY;
  static double HOST_FL_REUNIT_SVD_REL_ERROR;
  static double HOST_FL_REUNIT_SVD_ABS_ERROR;
  static bool   HOST_FL_CHECK_UNITARIZATION;

  void setUnitarizeLinksPadding(int input_padding_h, int output_padding_h)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(INPUT_PADDING), &input_padding_h, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(OUTPUT_PADDING), &output_padding_h, sizeof(int));
    return;
  }


  template<class Cmplx>
  __device__ __host__
  bool isUnitary(const Matrix<Cmplx,3>& matrix, double max_error)
  {
    const Matrix<Cmplx,3> identity = conj(matrix)*matrix;

    for(int i=0; i<3; ++i){
      if( fabs(identity(i,i).x - 1.0) > max_error || fabs(identity(i,i).y) > max_error) return false;
      for(int j=i+1; j<3; ++j){
	if( fabs(identity(i,j).x) > max_error || fabs(identity(i,j).y) > max_error
	    ||  fabs(identity(j,i).x) > max_error || fabs(identity(j,i).y) > max_error ){
	  return false;
	}
      }
    }
    return true;
  }



  template<class Cmplx>
  __device__ __host__
  bool isUnitarizedLinkConsistent(const Matrix<Cmplx,3>& initial_matrix,
				  const Matrix<Cmplx,3>& unitary_matrix,
				  double max_error)	
  {
    Matrix<Cmplx,3> temporary; 
    temporary = conj(initial_matrix)*unitary_matrix;
    temporary = temporary*temporary - conj(initial_matrix)*initial_matrix;
   
    for(int i=0; i<3; ++i){
      for(int j=0; j<3; ++j){
	if( fabs(temporary(i,j).x) > max_error || fabs(temporary(i,j).y) > max_error){
	  return false;
	}
      }
    }
    return true;
  }


  template<class T>
  __device__ __host__
  T getAbsMin(const T* const array, int size){
    T min = fabs(array[0]);
    for(int i=1; i<size; ++i){
      T abs_val = fabs(array[i]);
      if((abs_val) < min){ min = abs_val; }   
    }
    return min;
  }


  template<class Real>
  __device__ __host__
  inline bool checkAbsoluteError(Real a, Real b, Real epsilon)
  {
    if( fabs(a-b) <  epsilon) return true;
    return false;
  }


  template<class Real>
  __device__ __host__ 
  inline bool checkRelativeError(Real a, Real b, Real epsilon)
  {
    if( fabs((a-b)/b)  < epsilon ) return true;
    return false;
  }
    



  // Compute the reciprocal square root of the matrix q
  // Also modify q if the eigenvalues are dangerously small.
  template<class Cmplx> 
  __device__  __host__ 
  bool reciprocalRoot(const Matrix<Cmplx,3>& q, Matrix<Cmplx,3>* res){

    Matrix<Cmplx,3> qsq, tempq;


    typename RealTypeId<Cmplx>::Type c[3];
    typename RealTypeId<Cmplx>::Type g[3];

    qsq = q*q;
    tempq = qsq*q;

    c[0] = getTrace(q).x;
    c[1] = getTrace(qsq).x/2.0;
    c[2] = getTrace(tempq).x/3.0;

    g[0] = g[1] = g[2] = c[0]/3.;
    typename RealTypeId<Cmplx>::Type r,s,theta;
    s = c[1]/3. - c[0]*c[0]/18;

#ifdef __CUDA_ARCH__
#define FL_UNITARIZE_EPS DEV_FL_UNITARIZE_EPS
#else
#define FL_UNITARIZE_EPS HOST_FL_UNITARIZE_EPS
#endif


#ifdef __CUDA_ARCH__
#define FL_REUNIT_SVD_REL_ERROR DEV_FL_REUNIT_SVD_REL_ERROR
#define FL_REUNIT_SVD_ABS_ERROR DEV_FL_REUNIT_SVD_ABS_ERROR
#else // cpu
#define FL_REUNIT_SVD_REL_ERROR HOST_FL_REUNIT_SVD_REL_ERROR
#define FL_REUNIT_SVD_ABS_ERROR HOST_FL_REUNIT_SVD_ABS_ERROR
#endif


    typename RealTypeId<Cmplx>::Type cosTheta; 
    if(fabs(s) >= FL_UNITARIZE_EPS){
      const typename RealTypeId<Cmplx>::Type sqrt_s = sqrt(s);
      r = c[2]/2. - (c[0]/3.)*(c[1] - c[0]*c[0]/9.);
      cosTheta = r/(sqrt_s*sqrt_s*sqrt_s);
      if(fabs(cosTheta) >= 1.0){
	if( r > 0 ){ 
	  theta = 0.0;
	}else{
	  theta = FL_UNITARIZE_PI;
	}
      }else{ 
	theta = acos(cosTheta);
      }
      g[0] = c[0]/3 + 2*sqrt_s*cos( theta/3 );
      g[1] = c[0]/3 + 2*sqrt_s*cos( theta/3 + FL_UNITARIZE_PI23 );
      g[2] = c[0]/3 + 2*sqrt_s*cos( theta/3 + 2*FL_UNITARIZE_PI23 );
    }
                
    // Check the eigenvalues, if the determinant does not match the product of the eigenvalues
    // return false. Then call SVD instead.
    typename RealTypeId<Cmplx>::Type det = getDeterminant(q).x;
    if( fabs(det) < FL_REUNIT_SVD_ABS_ERROR ){ 
      return false;
    }
    if( checkRelativeError(g[0]*g[1]*g[2],det,FL_REUNIT_SVD_REL_ERROR) == false ) return false;


    // At this point we have finished with the c's 
    // use these to store sqrt(g)
    for(int i=0; i<3; ++i) c[i] = sqrt(g[i]);

    // done with the g's, use these to store u, v, w
    g[0] = c[0]+c[1]+c[2];
    g[1] = c[0]*c[1] + c[0]*c[2] + c[1]*c[2];
    g[2] = c[0]*c[1]*c[2];
        
    const typename RealTypeId<Cmplx>::Type & denominator  = g[2]*(g[0]*g[1]-g[2]); 
    c[0] = (g[0]*g[1]*g[1] - g[2]*(g[0]*g[0]+g[1]))/denominator;
    c[1] = (-g[0]*g[0]*g[0] - g[2] + 2.*g[0]*g[1])/denominator;
    c[2] =  g[0]/denominator;

    tempq = c[1]*q + c[2]*qsq;
    // Add a real scalar
    tempq(0,0).x += c[0];
    tempq(1,1).x += c[0];
    tempq(2,2).x += c[0];

    *res = tempq;
        	
    return true;
  }




  template<class Cmplx>
  __host__ __device__
  bool unitarizeLinkMILC(const Matrix<Cmplx,3>& in, Matrix<Cmplx,3>* const result)
  {
    Matrix<Cmplx,3> u;
#ifdef __CUDA_ARCH__
#define FL_REUNIT_SVD_ONLY  DEV_FL_REUNIT_SVD_ONLY
#define FL_REUNIT_ALLOW_SVD DEV_FL_REUNIT_ALLOW_SVD
#else
#define FL_REUNIT_SVD_ONLY  HOST_FL_REUNIT_SVD_ONLY
#define FL_REUNIT_ALLOW_SVD HOST_FL_REUNIT_ALLOW_SVD
#endif
    if( !FL_REUNIT_SVD_ONLY ){
      if( reciprocalRoot<Cmplx>(conj(in)*in,&u) ){
	*result = in*u;
	return true;
      }
    }

    // If we've got this far, then the Caley-Hamilton unitarization 
    // has failed. If SVD is not allowed, the unitarization has failed.
    if( !FL_REUNIT_ALLOW_SVD ) return false;

    Matrix<Cmplx,3> v;
    typename RealTypeId<Cmplx>::Type singular_values[3];
    computeSVD<Cmplx>(in, u, v, singular_values); // should pass pointers to u, v I guess
    *result = u*conj(v);
    return true;
  } // unitarizeMILC
    

  template<class Cmplx>
  __host__ __device__
  bool unitarizeLinkSVD(const Matrix<Cmplx,3>& in, Matrix<Cmplx,3>* const result)
  {
    Matrix<Cmplx,3> u, v;
    typename RealTypeId<Cmplx>::Type singular_values[3];
    computeSVD<Cmplx>(in, u, v, singular_values); // should pass pointers to u,v I guess	

    *result = u*conj(v);

#ifdef __CUDA_ARCH__ 
#define FL_MAX_ERROR  DEV_FL_MAX_ERROR
#else 
#define FL_MAX_ERROR  HOST_FL_MAX_ERROR
#endif
    if(isUnitary(*result,FL_MAX_ERROR)==false)
      {
#if (!defined(__CUDA_ARCH__) || (__COMPUTE_CAPABILITY__>=200))
	printf("ERROR: Link unitarity test failed\n");
	printf("TOLERANCE: %g\n", FL_MAX_ERROR);
#endif
	return false;
      }
    return true;
  }
#undef FL_MAX_ERROR


  template<class Cmplx>
  __host__ __device__
  bool unitarizeLinkNewton(const Matrix<Cmplx,3>& in, Matrix<Cmplx,3>* const result)
  {
    Matrix<Cmplx,3> u, uinv;
    u = in;

#ifdef __CUDA_ARCH__
#define MAX_ITER DEV_MAX_ITER
#else
#define MAX_ITER HOST_MAX_ITER
#endif
    for(int i=0; i<MAX_ITER; ++i){
      computeMatrixInverse(u, &uinv);
      u = 0.5*(u + conj(uinv));
    }

#undef MAX_ITER	
    if(isUnitarizedLinkConsistent(in,u,0.0000001)==false)
      {
#if (!defined(__CUDA_ARCH__) || (__COMPUTE_CAPABILITY__>=200))
        printf("ERROR: Unitarized link is not consistent with incoming link\n");
#endif
	return false;
      }
    *result = u;

    return true;
  }   


  




  template<class Cmplx>
  __global__ void getUnitarizedField(const Cmplx* inlink_even, const Cmplx*  inlink_odd,
				     Cmplx*  outlink_even, Cmplx*  outlink_odd,
				     int* num_failures, const int threads)
  {
    int mem_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (mem_idx >= threads) return;

    const Cmplx* inlink;
    Cmplx* outlink;

    inlink  = inlink_even;
    outlink = outlink_even;
    
    if(mem_idx >= threads/2){
      mem_idx = mem_idx - (threads/2);
      inlink  = inlink_odd;
      outlink = outlink_odd;
    }

    // Unitarization is always done in double precision
    Matrix<double2,3> v, result;
    for(int dir=0; dir<4; ++dir){
      loadLinkVariableFromArray(inlink, dir, mem_idx, (threads/2)+INPUT_PADDING, &v); 
      unitarizeLinkMILC(v, &result);
#ifdef __CUDA_ARCH__
#define FL_MAX_ERROR DEV_FL_MAX_ERROR
#define FL_CHECK_UNITARIZATION DEV_FL_CHECK_UNITARIZATION
#else
#define FL_MAX_ERROR HOST_FL_MAX_ERROR
#define FL_CHECK_UNITARIZATION HOST_FL_CHECK_UNITARIZATION
#endif
      if(FL_CHECK_UNITARIZATION){
        if(isUnitary(result,FL_MAX_ERROR) == false)
	  {
#ifdef __CUDA_ARCH__
	    atomicAdd(num_failures, 1);
#else 
	    (*num_failures)++;
#endif
	  }
      }
      writeLinkVariableToArray(result, dir, mem_idx, (threads/2)+OUTPUT_PADDING, outlink); 
    }
    return;
  }

  class UnitarizeLinksCuda : public Tunable {
  private:
    const cudaGaugeField &inField;
    cudaGaugeField &outField;
    int *fails;
    
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }
    
    // don't tune the grid dimension
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return inField.Volume(); }

  public:
    UnitarizeLinksCuda(const cudaGaugeField& inField, cudaGaugeField& outField,  int* fails) : 
      inField(inField), outField(outField), fails(fails) { ; }
    virtual ~UnitarizeLinksCuda() { ; }
    
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      
      if(inField.Precision() == QUDA_SINGLE_PRECISION){
	getUnitarizedField<<<tp.grid,tp.block>>>((float2*)inField.Even_p(), (float2*)inField.Odd_p(),
						 (float2*)outField.Even_p(), (float2*)outField.Odd_p(),
						 fails, inField.Volume());
      }else if(inField.Precision() == QUDA_DOUBLE_PRECISION){
	getUnitarizedField<<<tp.grid,tp.block>>>((double2*)inField.Even_p(), (double2*)inField.Odd_p(),
						 (double2*)outField.Even_p(), (double2*)outField.Odd_p(),
						 fails, inField.Volume());
      } else {
	errorQuda("UnitarizeLinks not implemented for precision %d", inField.Precision());
      }
      
    }
    void preTune() { ; }
    void postTune() { hipMemset(fails, 0, sizeof(int)); } // reset fails counter
    
    long long flops() const { return 0; } // FIXME: add flops counter

    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << inField.X()[0] << "x";
      vol << inField.X()[1] << "x";
      vol << inField.X()[2] << "x";
      vol << inField.X()[3] << "x";
      aux << "threads=" << inField.Volume() << ",prec=" << inField.Precision();
      aux << "stride=" << inField.Stride();
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
    }  
  }; // UnitarizeLinksCuda
    
  void unitarizeLinksCuda(const QudaGaugeParam& param,
			  cudaGaugeField& inField,
			  cudaGaugeField* outField, 
			  int* fails) { 
    UnitarizeLinksCuda unitarizeLinks(inField, *outField, fails);
    unitarizeLinks.apply(0);
  }

  void unitarizeLinksCPU(const QudaGaugeParam& param, cpuGaugeField& infield, cpuGaugeField* outfield)
  {
    int num_failures = 0;
    Matrix<double2,3> inlink, outlink;
      
    for(int i=0; i<infield.Volume(); ++i){
      for(int dir=0; dir<4; ++dir){
	if(param.cpu_prec == QUDA_SINGLE_PRECISION){
	  copyArrayToLink(&inlink, ((float*)(infield.Gauge_p()) + (i*4 + dir)*18)); // order of arguments?
	  if( unitarizeLinkNewton<double2>(inlink, &outlink) == false ) num_failures++; 
	  copyLinkToArray(((float*)(outfield->Gauge_p()) + (i*4 + dir)*18), outlink); 
	}else if(param.cpu_prec == QUDA_DOUBLE_PRECISION){
	  copyArrayToLink(&inlink, ((double*)(infield.Gauge_p()) + (i*4 + dir)*18)); // order of arguments?
	  if( unitarizeLinkNewton<double2>(inlink, &outlink) == false ) num_failures++; 
	  copyLinkToArray(((double*)(outfield->Gauge_p()) + (i*4 + dir)*18), outlink); 
	} // precision?
      } // dir
    }  // loop over volume
    return;
  }
    
  // CPU function which checks that the gauge field is unitary
  bool isUnitary(const QudaGaugeParam& param, cpuGaugeField& field, double max_error)
  {
    Matrix<double2,3> link, identity;
      
    for(int i=0; i<field.Volume(); ++i){
      for(int dir=0; dir<4; ++dir){
	if(param.cpu_prec == QUDA_SINGLE_PRECISION){
	  copyArrayToLink(&link, ((float*)(field.Gauge_p()) + (i*4 + dir)*18)); // order of arguments?
	}else if(param.cpu_prec == QUDA_DOUBLE_PRECISION){     
	  copyArrayToLink(&link, ((double*)(field.Gauge_p()) + (i*4 + dir)*18)); // order of arguments?
	}else{
	  errorQuda("Unsupported precision\n");
	}
	if(isUnitary(link,max_error) == false){ 
	  printf("Unitarity failure\n");
	  printf("site index = %d,\t direction = %d\n", i, dir);
	  printLink(link);
	  identity = conj(link)*link;
	  printLink(identity);
	  return false;
	}
      } // dir
    } // i	  
    return true;
  } // is unitary





  


















  template <typename Gauge>
  struct UnitarizeLinksQudaArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    Gauge links;
    int *fails;
    UnitarizeLinksQudaArg(const Gauge &links, const GaugeField &data,  int* fails) : links(links), fails(fails) {
    for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];  
    threads = X[0]*X[1]*X[2]*X[3];
    }
  };


__device__ __host__ inline int linkIndex(int x[], const int X[4]) {
  int idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
  return idx;
}

__device__ __host__ inline void getCoords3(int x[4], int cb_index, const int X[4], int parity) {
  x[3] = cb_index/(X[2]*X[1]*X[0]/2);
  x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
  x[1] = (cb_index/(X[0]/2)) % X[1];
  x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);

  return;
}


template<typename Float, typename Gauge>
__global__ void DoUnitarizedLink(UnitarizeLinksQudaArg<Gauge> arg){
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  if(idx >= arg.threads) return;
  typedef typename ComplexTypeId<Float>::Type Cmplx;
  int parity = 0;
  if(idx >= arg.threads/2) {
    parity = 1;
    idx -= arg.threads/2;
  }
  int X[4]; 
  for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];
  int x[4];
  getCoords3(x, idx, X, parity);
 
  idx = linkIndex(x,X);
  Matrix<double2,3> v, result;
  Matrix<Cmplx,3> tmp;
  for (int mu = 0; mu < 4; mu++) { 
    arg.links.load((Float*)(tmp.data),idx, mu, parity);
      for(int i = 0; i < 9;i++) {
        v.data[i].x = (double)tmp.data[i].x;
        v.data[i].y = (double)tmp.data[i].y;
      }
      unitarizeLinkMILC(v, &result);
#ifdef __CUDA_ARCH__
#define FL_MAX_ERROR DEV_FL_MAX_ERROR
#define FL_CHECK_UNITARIZATION DEV_FL_CHECK_UNITARIZATION
#else
#define FL_MAX_ERROR HOST_FL_MAX_ERROR
#define FL_CHECK_UNITARIZATION HOST_FL_CHECK_UNITARIZATION
#endif
      if(FL_CHECK_UNITARIZATION){
        if(isUnitary(result,FL_MAX_ERROR) == false)
    {
#ifdef __CUDA_ARCH__
      atomicAdd(arg.fails, 1);
#else 
      (*arg.fails)++;
#endif
    }
      }
        //WRITE BACK IF FAIL??????????
        for(int i = 0; i < 9;i++) {
          tmp.data[i].x = (Float)result.data[i].x;
          tmp.data[i].y = (Float)result.data[i].y;
        }
        arg.links.save((Float*)(tmp.data),idx, mu, parity); 
  }
}



template<typename Float, typename Gauge>
  class UnitarizeLinksQuda : Tunable {    
    UnitarizeLinksQudaArg<Gauge> arg;
    
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }
    
    // don't tune the grid dimension
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

  public:
    UnitarizeLinksQuda(UnitarizeLinksQudaArg<Gauge> &arg) : arg(arg) { }


      void apply(const hipStream_t &stream){
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          DoUnitarizedLink<Float,Gauge><<<tp.grid, tp.block, 0, stream>>>(arg);
      }
    void preTune() { ; }
    void postTune() { hipMemset(arg.fails, 0, sizeof(int)); } // reset fails counter
    
    long long flops() const { return 0; } // FIXME: add flops counter

    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3] << "x";
      aux << "threads=" << arg.threads << ",prec=" << sizeof(Float);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
    }  
}; 


template<typename Float, typename Gauge>
void unitarizeLinksQuda( Gauge links,  cudaGaugeField& data, int* fails) {

      UnitarizeLinksQudaArg<Gauge> arg(links, data, fails);
      UnitarizeLinksQuda<Float, Gauge> unitlinks(arg) ;
      unitlinks.apply(0);
}

template<typename Float>
void unitarizeLinksQuda( cudaGaugeField& links, int* fails) {

        if(links.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
        if(links.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        //printf("QUDA_RECONSTRUCT_NO\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 2, 18>(links), links, fails) ;
        } else if(links.Reconstruct() == QUDA_RECONSTRUCT_12){
        //printf("QUDA_RECONSTRUCT_12\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 2, 12>(links), links, fails) ;
        } else if(links.Reconstruct() == QUDA_RECONSTRUCT_8){
        //printf("QUDA_RECONSTRUCT_8\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 2, 8>(links), links, fails) ;
        
        } else {
          errorQuda("Reconstruction type %d of gauge field not supported", links.Reconstruct());
        }
      } else if(links.Order() == QUDA_FLOAT4_GAUGE_ORDER) {
        if(links.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        //printf("QUDA_RECONSTRUCT_NO\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 4, 18>(links), links, fails) ;
        } else if(links.Reconstruct() == QUDA_RECONSTRUCT_12){
        //printf("QUDA_RECONSTRUCT_12\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 4, 12>(links), links, fails) ;
        } else if(links.Reconstruct() == QUDA_RECONSTRUCT_8){
        //printf("QUDA_RECONSTRUCT_8\n");
          unitarizeLinksQuda<Float>(FloatNOrder<Float, 18, 4, 8>(links), links, fails) ;
        } else {
          errorQuda("Reconstruction type %d of gauge field not supported", links.Reconstruct());
        }
      } else {
        errorQuda("Invalid Gauge Order\n");
      }
    }
#endif
  
  void unitarizeLinksQuda(cudaGaugeField& links, int* fails) {
#ifdef GPU_UNITARIZE
    if(links.Precision() == QUDA_HALF_PRECISION) {
      errorQuda("Half precision not supported\n");
    }
    if (links.Precision() == QUDA_SINGLE_PRECISION) {
      unitarizeLinksQuda<float>(links, fails);
    } else if(links.Precision() == QUDA_DOUBLE_PRECISION) {
      unitarizeLinksQuda<double>(links, fails);
    } else {
      errorQuda("Precision %d not supported", links.Precision());
    }
#else
    errorQuda("Unitarization has not been built");
#endif
  }

  void setUnitarizeLinksConstants(double unitarize_eps_h, double max_error_h, 
				  bool allow_svd_h, bool svd_only_h,
				  double svd_rel_error_h, double svd_abs_error_h, 
				  bool check_unitarization_h)
  {
#ifdef GPU_UNITARIZE
    // not_set is only initialised once
    static bool not_set=true;
		
    if(not_set){
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_UNITARIZE_EPS), &unitarize_eps_h, sizeof(double));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_REUNIT_ALLOW_SVD), &allow_svd_h, sizeof(bool));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_REUNIT_SVD_ONLY), &svd_only_h, sizeof(bool));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_REUNIT_SVD_REL_ERROR), &svd_rel_error_h, sizeof(double));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_REUNIT_SVD_ABS_ERROR), &svd_abs_error_h, sizeof(double));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_MAX_ERROR), &max_error_h, sizeof(double));
      hipMemcpyToSymbol(HIP_SYMBOL(DEV_FL_CHECK_UNITARIZATION), &check_unitarization_h, sizeof(bool));
	  

      HOST_FL_UNITARIZE_EPS = unitarize_eps_h;
      HOST_FL_REUNIT_ALLOW_SVD = allow_svd_h;
      HOST_FL_REUNIT_SVD_ONLY = svd_only_h;
      HOST_FL_REUNIT_SVD_REL_ERROR = svd_rel_error_h;
      HOST_FL_REUNIT_SVD_ABS_ERROR = svd_abs_error_h;
      HOST_FL_MAX_ERROR = max_error_h;     
      HOST_FL_CHECK_UNITARIZATION = check_unitarization_h;

      not_set = false;
    }
    checkCudaError();
#else
    errorQuda("Unitarization has not been built");
#endif
    return;
  }
  
} // namespace quda

