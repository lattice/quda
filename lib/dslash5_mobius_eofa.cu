#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <dslash_quda.h>
#include <index_helper.cuh>
#include <inline_ptx.h>
#include <math_helper.cuh>
#include <shared_memory_cache_helper.cuh>

namespace quda
{
  namespace mobius_eofa
  {
#ifdef GPU_DOMAIN_WALL_DIRAC

    /**
      @brief Structure containing the EOFA coefficients
     */
    template <typename real> struct eofa_coeff {
      real u[QUDA_MAX_DWF_LS]; // xpay coefficients
      real x[QUDA_MAX_DWF_LS];
      real y[QUDA_MAX_DWF_LS];
    };

    constexpr int size = 4096;
    static __constant__ char mobius_eofa_d[size];
    static char mobius_eofa_h[size];

    /**
      @brief Helper function for grabbing the constant struct, whether
      we are on the GPU or CPU.
     */
    template <typename real> inline __device__ __host__ const eofa_coeff<real> *get_eofa_coeff()
    {
#ifdef __CUDA_ARCH__
      return reinterpret_cast<const eofa_coeff<real> *>(mobius_eofa_d);
#else
      return reinterpret_cast<const eofa_coeff<real> *>(mobius_eofa_h);
#endif
    }

    template <typename storage_type, int nColor> struct Dslash5Arg {
      typedef typename colorspinor_mapper<storage_type, 4, nColor>::type F;
      typedef typename mapper<storage_type>::type real;

      F out;                  // output vector field
      const F in;             // input vector field
      const F x;              // auxiliary input vector field
      const int nParity;      // number of parities we're working on
      const int volume_cb;    // checkerboarded volume
      const int volume_4d_cb; // 4-d checkerboarded volume
      const int_fastdiv Ls;   // length of 5th dimension

      const real m_f; // fermion mass parameter
      const real m_5; // Wilson mass shift

      const bool dagger; // dagger
      const bool xpay;   // whether we are doing xpay or not

      real b = 0.; // real constant Mobius coefficient
      real c = 0.; // real constant Mobius coefficient
      real a;      // real xpay coefficient

      real kappa;
      real inv;

      int eofa_pm;
      real sherman_morrison;

      Dslash5Type type;

      Dslash5Arg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, const double m_f_,
                 const double m_5_, const Complex *b_5_, const Complex *c_5_, double a_, int eofa_pm_, double inv_,
                 double kappa_, const double *eofa_u, const double *eofa_x, const double *eofa_y,
                 double sherman_morrison_, bool dagger_, Dslash5Type type_) :
        out(out),
        in(in),
        x(x),
        nParity(in.SiteSubset()),
        volume_cb(in.VolumeCB()),
        volume_4d_cb(volume_cb / in.X(4)),
        Ls(in.X(4)),
        m_f(m_f_),
        m_5(m_5_),
        a(a_),
        dagger(dagger_),
        xpay(a_ == 0. ? false : true),
        type(type_),
        eofa_pm(eofa_pm_),
        inv(inv_),
        kappa(kappa_),
        sherman_morrison(sherman_morrison_)
      {
        if (in.Nspin() != 4) errorQuda("nSpin = %d not support", in.Nspin());
        if (!in.isNative() || !out.isNative())
          errorQuda("Unsupported field order out=%d in=%d\n", out.FieldOrder(), in.FieldOrder());
        if (sizeof(eofa_coeff<real>) > size)
          errorQuda("Coefficient buffer too large at %lu bytes\n", sizeof(eofa_coeff<real>));
        // printfQuda("Coefficient buffer too large at %lu bytes\n", sizeof(eofa_coeff<real>));

        eofa_coeff<real> *eofa_coeffs = reinterpret_cast<eofa_coeff<real> *>(mobius_eofa_h);

        switch (type) {
        case M5_EOFA:
          for (int s = 0; s < Ls; s++) {
            eofa_coeffs->u[s] = eofa_u[s];
            // printfQuda("eofa_coeffs->u[%02d]=%.6f\n", s, eofa_coeffs->u[s]);
          }
          hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_eofa_d), mobius_eofa_h, sizeof(eofa_coeff<real>) / 3, 0, hipMemcpyHostToDevice,
                                  streams[Nstream - 1]);
          break;
        case M5INV_EOFA:
          for (int s = 0; s < Ls; s++) {
            eofa_coeffs->u[s] = eofa_u[s];
            eofa_coeffs->x[s] = eofa_x[s];
            eofa_coeffs->y[s] = eofa_y[s];
          }
          hipMemcpyToSymbolAsync(HIP_SYMBOL(mobius_eofa_d), mobius_eofa_h, sizeof(eofa_coeff<real>), 0, hipMemcpyHostToDevice,
                                  streams[Nstream - 1]);
          break;
        default: errorQuda("Unknown EOFA Dslash5Type %d", type);
        }
      }
    };

    /**
      @brief Apply the D5 operator at given site
      @param[in] arg    Argument struct containing any meta data and accessors
      @param[in] parity Parity we are on
      @param[in] x_cb   Checkerboarded 4-d space-time index
      @param[in] s      Ls dimension coordinate
     */
    template <typename storage_type, int nColor, bool dagger, bool pm, bool xpay, Dslash5Type type, typename Arg>
    __device__ inline void dslash5(Arg &arg, int parity, int x_cb, int s)
    {
      typedef typename mapper<storage_type>::type real;
      typedef ColorSpinor<real, nColor, 4> Vector;

      VectorCache<real, Vector> cache;

      Vector out;
      cache.save(arg.in(s * arg.volume_4d_cb + x_cb, parity));
      cache.sync();

      auto Ls = arg.Ls;

      { // forwards direction
        const Vector in = cache.load(threadIdx.x, (s + 1) % Ls, 0);
        constexpr int proj_dir = dagger ? +1 : -1;
        if (s == Ls - 1) {
          out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
        } else {
          out += in.project(4, proj_dir).reconstruct(4, proj_dir);
        }
      }

      { // backwards direction
        const Vector in = cache.load(threadIdx.x, (s + Ls - 1) % Ls, 0);
        constexpr int proj_dir = dagger ? -1 : +1;
        if (s == 0) {
          out += (-arg.m_f * in.project(4, proj_dir)).reconstruct(4, proj_dir);
        } else {
          out += in.project(4, proj_dir).reconstruct(4, proj_dir);
        }
      }

      if (type == M5_EOFA) {
        const eofa_coeff<real> *eofa_coeffs = get_eofa_coeff<real>();
        Vector diagonal = cache.load(threadIdx.x, s, 0);
        out = (static_cast<real>(0.5) * arg.kappa) * out + diagonal; // 1 + kappa*D5; the 0.5 for spin projection

        constexpr int proj_dir = pm ? +1 : -1;

        if (dagger) {
          // in Grid:
          // axpby_ssp_pplus(chi, one, chi, shift_coeffs[s], psi, Ls-1, s);
          // axpby_ssp_pminus(chi, one, chi, shift_coeffs[s], psi, 0, s);
          if (s == (pm ? Ls - 1 : 0)) {
            for (int sp = 0; sp < Ls; sp++) {
              out += (static_cast<real>(0.5) * eofa_coeffs->u[sp])
                * cache.load(threadIdx.x, sp, 0).project(4, proj_dir).reconstruct(4, proj_dir);
            }
          }
        } else {
          // in Grid:
          // axpby_ssp_pplus(chi, one, chi, shift_coeffs[s], psi, s, Ls-1);
          // axpby_ssp_pminus(chi, one, chi, shift_coeffs[s], psi, s, 0);
          out += (static_cast<real>(0.5) * eofa_coeffs->u[s])
            * cache.load(threadIdx.x, pm ? Ls - 1 : 0, 0).project(4, proj_dir).reconstruct(4, proj_dir);
        }

        if (xpay) { // really axpy
          Vector x = arg.x(s * arg.volume_4d_cb + x_cb, parity);
          out = arg.a * x + out;
        }
      }
      arg.out(s * arg.volume_4d_cb + x_cb, parity) = out;
    }

    /**
      @brief Apply the M5 inverse operator at a given site on the
      lattice.  This is the original algorithm as described in Kim and
      Izubushi (LATTICE 2013_033), where the b and c coefficients are
      constant along the Ls dimension, so is suitable for Shamir and
      Mobius domain-wall fermions.

      @param[in] arg    Argument struct containing any meta data and accessors
      @param[in] parity Parity we are on
      @param[in] x_cb   Checkerboarded 4-d space-time index
      @param[in] s      Ls dimension coordinate
     */

    template <typename storage_type, int nColor, bool dagger, bool pm, bool xpay, Dslash5Type type, typename Arg>
    __device__ __host__ inline void dslash5inv(Arg &arg, int parity, int x_cb, int s)
    {
      typedef typename mapper<storage_type>::type real;
      typedef ColorSpinor<real, nColor, 4> Vector;

      // const auto k = -arg.kappa; // k is -kappa
      // const auto inv = arg.inv;
      const auto sherman_morrison = arg.sherman_morrison;
      VectorCache<real, Vector> cache;
      cache.save(arg.in(s * arg.volume_4d_cb + x_cb, parity));
      cache.sync();

      Vector out;
      const eofa_coeff<real> *eofa_coeffs = get_eofa_coeff<real>();

      for (int sp = 0; sp < arg.Ls; sp++) {
        Vector in = cache.load(threadIdx.x, sp, parity);
        {
          int exp = s < sp ? arg.Ls - sp + s : s - sp;
          real factorR = 0.5 * eofa_coeffs->y[pm ? arg.Ls - exp - 1 : exp] * (s < sp ? -arg.m_f : static_cast<real>(1.0));
          // real factorR = inv * __fast_pow(k,exp) * ( s < sp ? -arg.m_f :
          // static_cast<real>(1.0) );
          constexpr int proj_dir = dagger ? -1 : +1;
          out += factorR * (in.project(4, proj_dir)).reconstruct(4, proj_dir);
        }
        {
          int exp = s > sp ? arg.Ls - s + sp : sp - s;
          real factorL = 0.5 * eofa_coeffs->y[pm ? arg.Ls - exp - 1 : exp] * (s > sp ? -arg.m_f : static_cast<real>(1.0));
          // real factorL = inv * __fast_pow(k,exp) * ( s > sp ? -arg.m_f :
          // static_cast<real>(1.0) );
          constexpr int proj_dir = dagger ? +1 : -1;
          out += factorL * (in.project(4, proj_dir)).reconstruct(4, proj_dir);
        }
        // The EOFA stuff
        {
          constexpr int proj_dir = pm ? +1 : -1;
          real t = dagger ? eofa_coeffs->y[s] * eofa_coeffs->x[sp] : eofa_coeffs->x[s] * eofa_coeffs->y[sp];
          out += (t * sherman_morrison) * (in.project(4, proj_dir)).reconstruct(4, proj_dir);
        }
      }
      if (xpay) { // really axpy
        Vector x = arg.x(s * arg.volume_4d_cb + x_cb, parity);
        out = x + arg.a * out;
      }
      arg.out(s * arg.volume_4d_cb + x_cb, parity) = out;
    }

    /**
      @brief GPU kernel for applying the D5 operator
      @param[in] arg Argument struct containing any meta data and accessors
     */
    template <typename storage_type, int nColor, bool dagger, bool pm, bool xpay, Dslash5Type type, typename Arg>
    __global__ void dslash5GPU(Arg arg)
    {
      int x_cb = blockIdx.x * blockDim.x + threadIdx.x;
      int s = blockIdx.y * blockDim.y + threadIdx.y;
      int parity = blockIdx.z * blockDim.z + threadIdx.z;

      if (x_cb >= arg.volume_4d_cb) return;
      if (s >= arg.Ls) return;
      if (parity >= arg.nParity) return;

      if (type == M5_EOFA) {
        dslash5<storage_type, nColor, dagger, pm, xpay, type>(arg, parity, x_cb, s);
      } else if (type == M5INV_EOFA) {
        dslash5inv<storage_type, nColor, dagger, pm, xpay, type>(arg, parity, x_cb, s);
      }
    }

    template <typename storage_type, int nColor, typename Arg> class Dslash5 : public Tunable
    {
    protected:
      Arg &arg;
      const ColorSpinorField &meta;
      static constexpr bool shared = true; // whether to use shared memory cache blocking for M5inv

      long long flops() const
      {
        // FIXME: Fix the flop count
        long long Ls = meta.X(4);
        long long bulk = (Ls - 2) * (meta.Volume() / Ls);
        long long wall = 2 * meta.Volume() / Ls;
        long long n = meta.Ncolor() * meta.Nspin();

        long long flops_ = 0;
        switch (arg.type) {
        case M5_EOFA:
        case M5INV_EOFA: flops_ = n * (8ll * bulk + 10ll * wall + (arg.xpay ? 4ll * meta.Volume() : 0)); break;
        default: errorQuda("Unknown Dslash5Type %d for EOFA", arg.type);
        }

        return flops_;
      }

      long long bytes() const
      {
        long long Ls = meta.X(4);
        switch (arg.type) {
        case M5_EOFA:
        case M5INV_EOFA: return arg.out.Bytes() + 2 * arg.in.Bytes() + (arg.xpay ? arg.x.Bytes() : 0);
        default: errorQuda("Unknown Dslash5Type %d for EOFA", arg.type);
        }
        return 0ll;
      }

      bool tuneGridDim() const { return false; }
      unsigned int minThreads() const { return arg.volume_4d_cb; }
      int blockStep() const { return 4; }
      int blockMin() const { return 4; }
      unsigned int sharedBytesPerThread() const
      {
        // spin components in shared depend on inversion algorithm
        int nSpin = meta.Nspin();
        return 2 * nSpin * nColor * sizeof(typename mapper<storage_type>::type);
      }

      unsigned int sharedBytesPerBlock(const TuneParam &param) const
      {
        return 0; 
      }

      // overloaded to return max dynamic shared memory if doing shared-memory
      // inverse
      unsigned int maxSharedBytesPerBlock() const
      {
        if (shared && (arg.type == M5_EOFA || arg.type == M5INV_EOFA)) {
          return maxDynamicSharedBytesPerBlock();
        } else {
          return Tunable::maxSharedBytesPerBlock();
        }
      }

    public:
      Dslash5(Arg &arg, const ColorSpinorField &meta) : arg(arg), meta(meta)
      {
        strcpy(aux, meta.AuxString());
        if (arg.dagger) strcat(aux, ",Dagger");
        if (arg.xpay) strcat(aux, ",xpay");
        if (arg.eofa_pm) {
          strcat(aux, ",eofa_plus");
        } else {
          strcat(aux, ",eofa_minus");
        }
        switch (arg.type) {
        case M5_EOFA: strcat(aux, ",mobius_M5_EOFA"); break;
        case M5INV_EOFA: strcat(aux, ",mobius_M5INV_EOFA"); break;
        default: errorQuda("Unknown Dslash5Type %d", arg.type);
        }
      }
      virtual ~Dslash5() {}

      template <typename T> inline void launch(T *f, const TuneParam &tp, Arg &arg, const hipStream_t &stream)
      {
        if (shared && (arg.type == M5_EOFA || arg.type == M5INV_EOFA)) {
          // if inverse kernel uses shared memory then maximize total shared memory
          setMaxDynamicSharedBytesPerBlock(f);
        }
        void *args[] = {&arg};
        qudaLaunchKernel((const void *)f, tp.grid, tp.block, args, tp.shared_bytes, stream);
      }

      void apply(const hipStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        if (arg.type == M5_EOFA) {
          if (arg.eofa_pm) {
            if (arg.xpay) {
              arg.dagger ? launch(dslash5GPU<storage_type, nColor, true, true, true, M5_EOFA, Arg>, tp, arg, stream) :
                           launch(dslash5GPU<storage_type, nColor, false, true, true, M5_EOFA, Arg>, tp, arg, stream);
            } else {
              arg.dagger ? launch(dslash5GPU<storage_type, nColor, true, true, false, M5_EOFA, Arg>, tp, arg, stream) :
                           launch(dslash5GPU<storage_type, nColor, false, true, false, M5_EOFA, Arg>, tp, arg, stream);
            }
          } else {
            if (arg.xpay) {
              arg.dagger ? launch(dslash5GPU<storage_type, nColor, true, false, true, M5_EOFA, Arg>, tp, arg, stream) :
                           launch(dslash5GPU<storage_type, nColor, false, false, true, M5_EOFA, Arg>, tp, arg, stream);
            } else {
              arg.dagger ? launch(dslash5GPU<storage_type, nColor, true, false, false, M5_EOFA, Arg>, tp, arg, stream) :
                           launch(dslash5GPU<storage_type, nColor, false, false, false, M5_EOFA, Arg>, tp, arg, stream);
            }
          }
        } else if (arg.type == M5INV_EOFA) {
          if (arg.eofa_pm) {
            if (arg.xpay) {
              arg.dagger ? launch(dslash5GPU<storage_type, nColor, true, true, true, M5INV_EOFA, Arg>, tp, arg, stream) :
                           launch(dslash5GPU<storage_type, nColor, false, true, true, M5INV_EOFA, Arg>, tp, arg, stream);
            } else {
              arg.dagger ?
                launch(dslash5GPU<storage_type, nColor, true, true, false, M5INV_EOFA, Arg>, tp, arg, stream) :
                launch(dslash5GPU<storage_type, nColor, false, true, false, M5INV_EOFA, Arg>, tp, arg, stream);
            }
          } else {
            if (arg.xpay) {
              arg.dagger ?
                launch(dslash5GPU<storage_type, nColor, true, false, true, M5INV_EOFA, Arg>, tp, arg, stream) :
                launch(dslash5GPU<storage_type, nColor, false, false, true, M5INV_EOFA, Arg>, tp, arg, stream);
            } else {
              arg.dagger ?
                launch(dslash5GPU<storage_type, nColor, true, false, false, M5INV_EOFA, Arg>, tp, arg, stream) :
                launch(dslash5GPU<storage_type, nColor, false, false, false, M5INV_EOFA, Arg>, tp, arg, stream);
            }
          }
        } else {
          errorQuda("Unknown Dslash5Type %d", arg.type);
        }
      }

      bool advanceBlockDim(TuneParam &param) const
      {
        const unsigned int max_threads = maxBlockSize(param);
        const unsigned int max_shared = maxSharedBytesPerBlock();
        bool ret;

        param.block.x += blockStep();
        int nthreads = param.block.x*param.block.y*param.block.z;
        if (param.block.x > max_threads || sharedBytesPerThread() * nthreads > max_shared
            || sharedBytesPerBlock(param) > max_shared) {
          resetBlockDim(param);
          ret = false;
        } else {
          ret = true;
        }

        return ret;
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.block.y = arg.Ls; // Ls must be contained in the block
        param.grid.y = 1;
        param.block.z = 1;
        param.grid.z = arg.nParity;
        param.shared_bytes = sharedBytesPerThread() * param.block.x * param.block.y * param.block.z;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        initTuneParam(param);
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    };

    template <typename storage_type, int nColor>
    void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, double m_f,
                      double m_5, const Complex *b_5, const Complex *c_5, double a, int eofa_pm, double inv,
                      double kappa, const double *eofa_u, const double *eofa_x, const double *eofa_y,
                      double sherman_morrison, bool dagger, Dslash5Type type)
    {
      Dslash5Arg<storage_type, nColor> arg(out, in, x, m_f, m_5, b_5, c_5, a, eofa_pm, inv, kappa, eofa_u, eofa_x,
                                           eofa_y, sherman_morrison, dagger, type);
      Dslash5<storage_type, nColor, Dslash5Arg<storage_type, nColor>> dslash(arg, in);
      dslash.apply(streams[Nstream - 1]);
    }

    // template on the number of colors
    template <typename storage_type>
    void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, double m_f,
                      double m_5, const Complex *b_5, const Complex *c_5, double a, int eofa_pm, double inv,
                      double kappa, const double *eofa_u, const double *eofa_x, const double *eofa_y,
                      double sherman_morrison, bool dagger, Dslash5Type type)
    {
      switch (in.Ncolor()) {
      case 3:
        ApplyDslash5<storage_type, 3>(out, in, x, m_f, m_5, b_5, c_5, a, eofa_pm, inv, kappa, eofa_u, eofa_x, eofa_y,
                                      sherman_morrison, dagger, type);
        break;
      default: errorQuda("Unsupported number of colors %d\n", in.Ncolor());
      }
    }

#endif
    // Apply the 5th dimension dslash operator to a colorspinor field
    // out = Dslash5*in
    void apply_dslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, double m_f,
                       double m_5, const Complex *b_5, const Complex *c_5, double a, int eofa_pm, double inv,
                       double kappa, const double *eofa_u, const double *eofa_x, const double *eofa_y,
                       double sherman_morrison, bool dagger, Dslash5Type type)
    {
#ifdef GPU_DOMAIN_WALL_DIRAC
      checkLocation(out, in); // check all locations match

      switch (checkPrecision(out, in)) {
      case QUDA_DOUBLE_PRECISION:
        ApplyDslash5<double>(out, in, x, m_f, m_5, b_5, c_5, a, eofa_pm, inv, kappa, eofa_u, eofa_x, eofa_y,
                             sherman_morrison, dagger, type);
        break;
      case QUDA_SINGLE_PRECISION:
        ApplyDslash5<float>(out, in, x, m_f, m_5, b_5, c_5, a, eofa_pm, inv, kappa, eofa_u, eofa_x, eofa_y,
                            sherman_morrison, dagger, type);
        break;
      case QUDA_HALF_PRECISION:
        ApplyDslash5<short>(out, in, x, m_f, m_5, b_5, c_5, a, eofa_pm, inv, kappa, eofa_u, eofa_x, eofa_y,
                            sherman_morrison, dagger, type);
        break;
      default: errorQuda("Unsupported precision %d\n", in.Precision());
      }
#else
      errorQuda("Mobius EOFA dslash has not been built");
#endif
    }
  } // namespace mobius_eofa
} // namespace quda
