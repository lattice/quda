#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <malloc_quda.h>
#include <reduce_helper.h>
#include <tunable_nd.h>
#include <kernels/reduce_init.cuh>

// These are used for reduction kernels
static device_reduce_t *d_reduce = nullptr;
static device_reduce_t *h_reduce = nullptr;
static device_reduce_t *hd_reduce = nullptr;

static count_t *reduce_count = nullptr;
static qudaEvent_t reduceEnd;

namespace quda
{

  namespace reducer
  {

    // FIXME need to dynamically resize these
    void *get_device_buffer() { return d_reduce; }
    void *get_mapped_buffer() { return hd_reduce; }
    void *get_host_buffer() { return h_reduce; }
    template <> count_t *get_count() { return reduce_count; }
    qudaEvent_t &get_event() { return reduceEnd; }

    size_t buffer_size()
    {
      /* we have these different reductions to cater for:

         - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), and a
           grid-stride loop with max number of blocks = 2 x SM count

         - multi-reductions where we are reducing to a matrix of size
           of size QUDA_MAX_MULTI_REDUCE of vectors (max length 4),
           and a grid-stride loop with maximum number of blocks = 2 x
           SM count
      */

      int reduce_size = 4 * sizeof(device_reduce_t);
      int max_reduce = reduce_size;
      int max_multi_reduce = max_n_reduce() * reduce_size;
      int max_reduce_blocks = 2 * device::processor_count();

      // reduction buffer size
      size_t bytes = max_reduce_blocks * std::max(max_reduce, max_multi_reduce);
      return bytes;
    }

    template <typename T>
    struct init_reduce : public TunableKernel1D {
      T *reduce_count;
      long long bytes() const { return max_n_reduce() * sizeof(T); }
      unsigned int minThreads() const { return max_n_reduce(); }

      init_reduce(T *reduce_count) :
        TunableKernel1D(max_n_reduce()),
        reduce_count(reduce_count)
      { apply(device::get_default_stream()); }

      void apply(const qudaStream_t &stream)
      {
        auto tp = tuneLaunch(*this, getTuning(), getVerbosity());
        launch_device<init_count>(tp, stream, init_arg<T>(reduce_count));
      }
    };

    void init()
    {
      auto bytes = buffer_size();
      if (!d_reduce) d_reduce = (device_reduce_t *)device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be device_reduce_t x 3)

      if (!h_reduce) {
        h_reduce = (device_reduce_t *)mapped_malloc(bytes);
        hd_reduce = (device_reduce_t *)get_mapped_device_pointer(h_reduce); // set the matching device pointer

        using system_atomic_t = device_reduce_t;
        size_t n_reduce = bytes / sizeof(system_atomic_t);
        auto *atomic_buf = reinterpret_cast<system_atomic_t *>(h_reduce);
        for (size_t i = 0; i < n_reduce; i++) new (atomic_buf + i) system_atomic_t {0}; // placement new constructor
      }

      if (!reduce_count) {
        reduce_count = static_cast<count_t *>(device_malloc(max_n_reduce() * sizeof(decltype(*reduce_count))));
        init_reduce<count_t> init(reduce_count);
      }

      reduceEnd = qudaEventCreate();
    }

    void destroy()
    {
      qudaEventDestroy(reduceEnd);

      if (reduce_count) {
        device_free(reduce_count);
        reduce_count = nullptr;
      }
      if (d_reduce) {
        device_free(d_reduce);
        d_reduce = 0;
      }
      if (h_reduce) {
        host_free(h_reduce);
        h_reduce = 0;
      }
      hd_reduce = 0;
    }

  } // namespace reducer
} // namespace quda
