#include "hip/hip_runtime.h"
#include <cstdio>

#include <quda_internal.h>
#include <gauge_field.h>
#include <llfat_quda.h>
#include <index_helper.cuh>
#include <gauge_field_order.h>
#include <fast_intdiv.h>
#include <tune_quda.h>
#include <instantiate.h>

#define MIN_COEFF 1e-7

namespace quda {

  template <typename Float, int nColor, QudaReconstructType recon>
  struct LinkArg {
    typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_NO>::type Link;
    typedef typename gauge_mapper<Float, recon, 18, QUDA_STAGGERED_PHASE_MILC>::type Gauge;

    Link link;
    Gauge u;
    Float coeff;

    unsigned int threads;

    int_fastdiv X[4];
    int_fastdiv E[4];
    int border[4];

    /** This keeps track of any parity changes that result in using a
    radius of 1 for the extended border (the staple computations use
    such an extension, and if an odd number of dimensions are
    partitioned then we have to correct for this when computing the local index */
    int odd_bit;

    LinkArg(GaugeField &link, const GaugeField &u, Float coeff) :
      threads(link.VolumeCB()),
      link(link),
      u(u),
      coeff(coeff)
    {
      if (u.StaggeredPhase() != QUDA_STAGGERED_PHASE_MILC && u.Reconstruct() != QUDA_RECONSTRUCT_NO)
        errorQuda("Staggered phase type %d not supported", u.StaggeredPhase());
      for (int d=0; d<4; d++) {
        X[d] = link.X()[d];
        E[d] = u.X()[d];
        border[d] = (E[d] - X[d]) / 2;
      }
    }
  };

  template <typename Float, int dir, typename Arg>
  __device__ void longLinkDir(Arg &arg, int idx, int parity) {
    int x[4];
    int dx[4] = {0, 0, 0, 0};

    auto y = arg.u.coords;
    getCoords(x, idx, arg.X, parity);
    for (int d=0; d<4; d++) x[d] += arg.border[d];

    typedef Matrix<complex<Float>,3> Link;

    Link a = arg.u(dir, linkIndex(y, x, arg.E), parity);

    dx[dir]++;
    Link b = arg.u(dir, linkIndexShift(y, x, dx, arg.E), 1-parity);

    dx[dir]++;
    Link c = arg.u(dir, linkIndexShift(y, x, dx, arg.E), parity);
    dx[dir]-=2;

    arg.link(dir, idx, parity) = arg.coeff * a * b * c;
  }

  template <typename Float, typename Arg>
  __global__ void computeLongLink(Arg arg) {

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = blockIdx.y*blockDim.y + threadIdx.y;
    int dir = blockIdx.z*blockDim.z + threadIdx.z;
    if (idx >= arg.threads) return;
    if (dir >= 4) return;

    switch(dir) {
    case 0: longLinkDir<Float, 0>(arg, idx, parity); break;
    case 1: longLinkDir<Float, 1>(arg, idx, parity); break;
    case 2: longLinkDir<Float, 2>(arg, idx, parity); break;
    case 3: longLinkDir<Float, 3>(arg, idx, parity); break;
    }
    return;
  }

  template <typename Float, int nColor, QudaReconstructType recon>
  class LongLink : public TunableVectorYZ {
    LinkArg<Float, nColor, recon> arg;
    const GaugeField &meta;
    unsigned int minThreads() const { return arg.threads; }
    bool tuneGridDim() const { return false; }

  public:
    LongLink(const GaugeField &u, GaugeField &lng, double coeff) :
      TunableVectorYZ(2,4),
      arg(lng, u, coeff),
      meta(lng)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, comm_dim_partitioned_string());

      apply(0);
    }

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      computeLongLink<Float><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 2*4*arg.threads*198; }
    long long bytes() const { return 2*4*arg.threads*(3*arg.u.Bytes()+arg.link.Bytes()); }
  };

  void computeLongLink(GaugeField &lng, const GaugeField &u, double coeff)
  {
    instantiate<LongLink, ReconstructNo12>(u, lng, coeff); // u first arg so we pick its recon
  }

  template <typename Float, typename Arg>
  __global__ void computeOneLink(Arg arg)
  {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = blockIdx.y * blockDim.y + threadIdx.y;
    int dir =  blockIdx.z * blockDim.z + threadIdx.z;
    if (idx >= arg.threads) return;
    if (dir >= 4) return;

    auto x = arg.u.coords;
    getCoords(x, idx, arg.X, parity);
    for (int d=0; d<4; d++) x[d] += arg.border[d];

    typedef Matrix<complex<Float>,3> Link;

    Link a = arg.u(dir, linkIndex(x,arg.E), parity);

    arg.link(dir, idx, parity) = arg.coeff*a;

    return;
  }

  template <typename Float, int nColor, QudaReconstructType recon>
  class OneLink : public TunableVectorYZ {
    LinkArg<Float, nColor, recon> arg;
    const GaugeField &meta;
    unsigned int minThreads() const { return arg.threads; }
    bool tuneGridDim() const { return false; }

  public:
    OneLink(const GaugeField &u, GaugeField &fat, double coeff) :
      TunableVectorYZ(2,4),
      arg(fat, u, coeff),
      meta(fat)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, comm_dim_partitioned_string());

      apply(0);
    }

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      computeOneLink<Float><<<tp.grid,tp.block>>>(arg);
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
    long long flops() const { return 2*4*arg.threads*18; }
    long long bytes() const { return 2*4*arg.threads*(arg.u.Bytes()+arg.link.Bytes()); }
  };

  void computeOneLink(GaugeField &fat, const GaugeField &u, double coeff)
  {
    if (u.StaggeredPhase() != QUDA_STAGGERED_PHASE_MILC && u.Reconstruct() != QUDA_RECONSTRUCT_NO)
      errorQuda("Staggered phase type %d not supported", u.StaggeredPhase());
    instantiate<OneLink, ReconstructNo12>(u, fat, coeff);
  }

  template <typename Float, typename Fat, typename Staple, typename Mulink, typename Gauge>
  struct StapleArg {
    unsigned int threads;

    int_fastdiv X[4];
    int_fastdiv E[4];
    int border[4];

    int_fastdiv inner_X[4];
    int inner_border[4];

    /** This keeps track of any parity changes that result in using a
    radius of 1 for the extended border (the staple computations use
    such an extension, and if an odd number of dimensions are
    partitioned then we have to correct for this when computing the local index */
    int odd_bit;

    Gauge u;
    Fat fat;
    Staple staple;
    Mulink mulink;
    Float coeff;

    int n_mu;
    int mu_map[4];

    StapleArg(Fat fat, Staple staple, Mulink mulink, Gauge u, Float coeff,
	      const GaugeField &fat_meta, const GaugeField &u_meta) :
      threads(1), fat(fat), staple(staple), mulink(mulink), u(u), coeff(coeff),
      odd_bit( (commDimPartitioned(0)+commDimPartitioned(1) +
                commDimPartitioned(2)+commDimPartitioned(3))%2 )
    {
      for (int d=0; d<4; d++) {
        X[d] = (fat_meta.X()[d] + u_meta.X()[d]) / 2;
        E[d] = u_meta.X()[d];
        border[d] = (E[d] - X[d]) / 2;
        threads *= X[d];

        inner_X[d] = fat_meta.X()[d];
        inner_border[d] = (E[d] - inner_X[d]) / 2;
      }
      threads /= 2; // account for parity in y dimension
    }
  };

  template<typename Float, int mu, int nu, typename Arg>
  __device__ inline void computeStaple(Matrix<complex<Float>,3> &staple, Arg &arg, int x[], int parity) {
    typedef Matrix<complex<Float>,3> Link;
    int *y = arg.u.coords, *y_mu = arg.mulink.coords, dx[4] = {0, 0, 0, 0};

    /* Computes the upper staple :
     *                 mu (B)
     *               +-------+
     *       nu	   |	   |
     *	     (A)   |	   |(C)
     *		   X	   X
     */
    {
      /* load matrix A*/
      Link a = arg.u(nu, linkIndex(y, x, arg.E), parity);

      /* load matrix B*/
      dx[nu]++;
      Link b = arg.mulink(mu, linkIndexShift(y_mu, x, dx, arg.E), 1-parity);
      dx[nu]--;

      /* load matrix C*/
      dx[mu]++;
      Link c = arg.u(nu, linkIndexShift(y, x, dx, arg.E), 1-parity);
      dx[mu]--;

      staple = a * b * conj(c);
    }

    /* Computes the lower staple :
     *                 X       X
     *           nu    |       |
     *	         (A)   |       | (C)
     *		       +-------+
     *                  mu (B)
     */
    {
      /* load matrix A*/
      dx[nu]--;
      Link a = arg.u(nu, linkIndexShift(y, x, dx, arg.E), 1-parity);

      /* load matrix B*/
      Link b = arg.mulink(mu, linkIndexShift(y_mu, x, dx, arg.E), 1-parity);

      /* load matrix C*/
      dx[mu]++;
      Link c = arg.u(nu, linkIndexShift(y, x, dx, arg.E), parity);
      dx[mu]--;
      dx[nu]++;

      staple = staple + conj(a)*b*c;
    }
  }

  template<typename Float, bool save_staple, typename Arg>
  __global__ void computeStaple(Arg arg, int nu)
  {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = blockIdx.y*blockDim.y + threadIdx.y;
    if (idx >= arg.threads) return;

    int mu_idx = blockIdx.z*blockDim.z + threadIdx.z;
    if (mu_idx >= arg.n_mu) return;
    int mu;
    switch(mu_idx) {
    case 0: mu = arg.mu_map[0]; break;
    case 1: mu = arg.mu_map[1]; break;
    case 2: mu = arg.mu_map[2]; break;
    }

    int x[4];
    getCoords(x, idx, arg.X, (parity+arg.odd_bit)%2);
    for (int d=0; d<4; d++) x[d] += arg.border[d];

    typedef Matrix<complex<Float>,3> Link;
    Link staple;
    switch(mu) {
    case 0:
      switch(nu) {
      case 1: computeStaple<Float,0,1>(staple, arg, x, parity); break;
      case 2: computeStaple<Float,0,2>(staple, arg, x, parity); break;
      case 3: computeStaple<Float,0,3>(staple, arg, x, parity); break;
      } break;
    case 1:
      switch(nu) {
      case 0: computeStaple<Float,1,0>(staple, arg, x, parity); break;
      case 2: computeStaple<Float,1,2>(staple, arg, x, parity); break;
      case 3: computeStaple<Float,1,3>(staple, arg, x, parity); break;
      } break;
    case 2:
      switch(nu) {
      case 0: computeStaple<Float,2,0>(staple, arg, x, parity); break;
      case 1: computeStaple<Float,2,1>(staple, arg, x, parity); break;
      case 3: computeStaple<Float,2,3>(staple, arg, x, parity); break;
      } break;
    case 3:
      switch(nu) {
      case 0: computeStaple<Float,3,0>(staple, arg, x, parity); break;
      case 1: computeStaple<Float,3,1>(staple, arg, x, parity); break;
      case 2: computeStaple<Float,3,2>(staple, arg, x, parity); break;
      } break;
    }

    // exclude inner halo
    if ( !(x[0] < arg.inner_border[0] || x[0] >= arg.inner_X[0] + arg.inner_border[0] ||
	   x[1] < arg.inner_border[1] || x[1] >= arg.inner_X[1] + arg.inner_border[1] ||
	   x[2] < arg.inner_border[2] || x[2] >= arg.inner_X[2] + arg.inner_border[2] ||
	   x[3] < arg.inner_border[3] || x[3] >= arg.inner_X[3] + arg.inner_border[3]) ) {
      // convert to inner coords
      int inner_x[] = {x[0]-arg.inner_border[0], x[1]-arg.inner_border[1], x[2]-arg.inner_border[2], x[3]-arg.inner_border[3]};
      Link fat = arg.fat(mu, linkIndex(inner_x, arg.inner_X), parity);
      fat += arg.coeff * staple;
      arg.fat(mu, linkIndex(inner_x, arg.inner_X), parity) = fat;
    }

    if (save_staple) arg.staple(mu, linkIndex(x, arg.E), parity) = staple;
    return;
  }

  template <typename Float, typename Arg>
  class Staple : public TunableVectorYZ {
    Arg &arg;
    const GaugeField &meta;
    unsigned int minThreads() const { return arg.threads; }
    bool tuneGridDim() const { return false; }
    int nu;
    int dir1;
    int dir2;
    bool save_staple;

  public:
    Staple(Arg &arg, int nu, int dir1, int dir2, bool save_staple, const GaugeField &meta)
      : TunableVectorYZ(2,(3 - ( (dir1 > -1) ? 1 : 0 ) - ( (dir2 > -1) ? 1 : 0 ))),
	arg(arg), meta(meta), nu(nu), dir1(dir1), dir2(dir2), save_staple(save_staple)
	{
	  // compute the map for z thread index to mu index in the kernel
	  // mu != nu 3 -> n_mu = 3
	  // mu != nu != rho 2 -> n_mu = 2
	  // mu != nu != rho != sig 1 -> n_mu = 1
	  arg.n_mu = 3 - ( (dir1 > -1) ? 1 : 0 ) - ( (dir2 > -1) ? 1 : 0 );
	  int j=0;
	  for (int i=0; i<4; i++) {
	    if (i==nu || i==dir1 || i==dir2) continue; // skip these dimensions
	    arg.mu_map[j++] = i;
	  }
	  assert(j == arg.n_mu);
	}

    void apply(const qudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (save_staple)
	computeStaple<Float,true><<<tp.grid,tp.block>>>(arg, nu);
      else
	computeStaple<Float,false><<<tp.grid,tp.block>>>(arg, nu);
    }

    TuneKey tuneKey() const {
      std::stringstream aux;
      aux << meta.AuxString() << comm_dim_partitioned_string();
      aux << ",nu=" << nu << ",dir1=" << dir1 << ",dir2=" << dir2 << ",save=" << save_staple;
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    void preTune() { arg.fat.save(); arg.staple.save(); }
    void postTune() { arg.fat.load(); arg.staple.load(); }

    long long flops() const {
      return 2*arg.n_mu*arg.threads*( 4*198 + 18 + 36 );
    }
    long long bytes() const {
      return arg.n_mu*2*meta.VolumeCB()*arg.fat.Bytes()*2 // fat load/store is only done on interior
	+ arg.n_mu*2*arg.threads*(4*arg.u.Bytes() + 2*arg.mulink.Bytes() + (save_staple ? arg.staple.Bytes() : 0));
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon>
  struct Staple_ {
    Staple_(const GaugeField &u, GaugeField &fat, GaugeField &staple, const GaugeField &mulink,
            int nu, int dir1, int dir2, double coeff, bool save_staple)
    { // FIXME - incorporate another level of reconstruct peel off in instantiate
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type L;
      typedef typename gauge_mapper<Float,recon,18,QUDA_STAGGERED_PHASE_MILC>::type G;
      if (mulink.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        StapleArg<Float,L,L,L,G> arg(L(fat), L(staple), L(mulink), G(u), coeff, fat, u);
        Staple<Float,decltype(arg)> stapler(arg, nu, dir1, dir2, save_staple, fat);
        stapler.apply(0);
      } else if (mulink.Reconstruct() == recon) {
        StapleArg<Float,L,L,G,G> arg(L(fat), L(staple), G(mulink), G(u), coeff, fat, u);
        Staple<Float,decltype(arg)> stapler(arg, nu, dir1, dir2, save_staple, fat);
        stapler.apply(0);
      } else {
        errorQuda("Reconstruct %d is not supported\n", u.Reconstruct());
      }
    }
  };

  // Compute the staple field for direction nu,excluding the directions dir1 and dir2.
  void computeStaple(GaugeField &fat, GaugeField &staple, const GaugeField &mulink, const GaugeField &u,
		     int nu, int dir1, int dir2, double coeff, bool save_staple)
  {
    instantiate<Staple_, ReconstructNo12>(u, fat, staple, mulink, nu, dir1, dir2, coeff, save_staple);
  }

  void fatLongKSLink(GaugeField *fat, GaugeField *lng, const GaugeField& u, const double *coeff)
  {
#ifdef GPU_FATLINK
    GaugeFieldParam gParam(u);
    gParam.reconstruct = QUDA_RECONSTRUCT_NO;
    gParam.setPrecision(gParam.Precision());
    gParam.create = QUDA_NULL_FIELD_CREATE;
    auto staple = GaugeField::Create(gParam);
    auto staple1 = GaugeField::Create(gParam);

    if ( ((fat->X()[0] % 2 != 0) || (fat->X()[1] % 2 != 0) || (fat->X()[2] % 2 != 0) || (fat->X()[3] % 2 != 0))
	&& (u.Reconstruct()  != QUDA_RECONSTRUCT_NO)){
      errorQuda("Reconstruct %d and odd dimensionsize is not supported by link fattening code (yet)\n",
		u.Reconstruct());
    }

    computeOneLink(*fat, u, coeff[0]-6.0*coeff[5]);

    // if this pointer is not NULL, compute the long link
    if (lng) computeLongLink(*lng, u, coeff[1]);

    // Check the coefficients. If all of the following are zero, return.
    if (fabs(coeff[2]) >= MIN_COEFF || fabs(coeff[3]) >= MIN_COEFF ||
	fabs(coeff[4]) >= MIN_COEFF || fabs(coeff[5]) >= MIN_COEFF) {

      for (int nu = 0; nu < 4; nu++) {
        computeStaple(*fat, *staple, u, u, nu, -1, -1, coeff[2], 1);

        if (coeff[5] != 0.0) computeStaple(*fat, *staple, *staple, u, nu, -1, -1, coeff[5], 0);

        for (int rho = 0; rho < 4; rho++) {
          if (rho != nu) {

            computeStaple(*fat, *staple1, *staple, u, rho, nu, -1, coeff[3], 1);

            if (fabs(coeff[4]) > MIN_COEFF) {
              for (int sig = 0; sig < 4; sig++) {
                if (sig != nu && sig != rho) {
                  computeStaple(*fat, *staple, *staple1, u, sig, nu, rho, coeff[4], 0);
                }
              } //sig
            } // MIN_COEFF
          }
        } //rho
      } //nu
    }

    qudaDeviceSynchronize();
    checkCudaError();

    delete staple;
    delete staple1;
#else
    errorQuda("Fat-link computation not enabled");
#endif
  }

#undef MIN_COEFF

} // namespace quda
