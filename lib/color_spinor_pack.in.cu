#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <tunable_nd.h>
#include <kernels/color_spinor_pack.cuh>
#include <instantiate.h>
#include <multigrid.h>

/**
   @file color_spinor_pack.cu

   @brief This is the implementation of the color-spinor halo packer
   for an arbitrary field.  This implementation uses the fine-grained
   accessors and should support all field types reqgardless of
   precision, number of color or spins etc.

   Using a different precision of the field and of the halo is
   supported, though only QUDA_SINGLE_PRECISION fields with
   QUDA_HALF_PRECISION or QUDA_QUARTER_PRECISION halos are
   instantiated. When an integer format is requested for the halos
   then block-float format is used.

   As well as tuning basic block sizes, the autotuner also tunes for
   the dimensions to assign to each thread.  E.g., dim_thread=1 means
   we have one thread for all dimensions, dim_thread=4 means we have
   four threads (e.g., one per dimension).  We always uses seperate
   threads for forwards and backwards directions.  Dimension,
   direction and parity are assigned to the z thread dimension.

   If doing block-float format, since all spin and color components of
   a given site have to reside in the same thread block (to allow us
   to compute the max element) we override the autotuner to keep the z
   thread dimensions in the grid and not the block, and allow for
   smaller tuning increments of the thread block dimension in x to
   ensure that we can always fit within a single thread block.  It is
   this constraint that gives rise for the need to cap the limit for
   block-float support, e.g., max_block_float_nc.
 */

namespace quda {

  // this is the maximum number of colors for which we support block-float format

  template <typename store_t, typename ghost_store_t, QudaFieldOrder order, int nSpin, int nColor>
  class GhostPack : public TunableKernel3D {
    void **ghost;
    const ColorSpinorField &a;
    cvector_ref<const ColorSpinorField> &v;
    const QudaParity parity;
    const int nFace;
    const int dagger;
    static constexpr bool block_float = sizeof(store_t) == QUDA_SINGLE_PRECISION && isFixed<ghost_store_t>::value;
    size_t work_items;
    int shmem;
    static constexpr int get_max_block_float_nc() { return 6144; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const
    {
      if (block_float) {
        auto thread_width_x = a.isNative() ?
          ((param.block.x + device::warp_size() - 1) / device::warp_size()) * device::warp_size() :
          param.block.x;
        return sizeof(store_t) * thread_width_x * param.block.y * param.block.z;
      } else {
        return 0;
      }
    }

    bool tuneSharedBytes() const { return false; }
    unsigned int minThreads() const { return work_items; }

  public:
    GhostPack(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
              MemoryLocation *destination, int shmem_, cvector_ref<const ColorSpinorField> &v) :
      TunableKernel3D(a, (a.Nspin() / spins_per_thread(a)) * (a.Ncolor() / colors_per_thread(a)), a.SiteSubset()),
      ghost(ghost),
      a(a),
      v(v),
      parity(parity),
      nFace(nFace),
      dagger(dagger),
      work_items(0),
      shmem(shmem_)
    {
      // if doing block float then all spin-color components must be within the same block
      if (block_float) resizeStep((a.Nspin()/spins_per_thread(a))*(a.Ncolor()/colors_per_thread(a)), step_z);
      switch (a.GhostPrecision()) {
      case QUDA_DOUBLE_PRECISION:  strcat(aux,",halo_prec=8"); break;
      case QUDA_SINGLE_PRECISION:  strcat(aux,",halo_prec=4"); break;
      case QUDA_HALF_PRECISION:    strcat(aux,",halo_prec=2"); break;
      case QUDA_QUARTER_PRECISION: strcat(aux,",halo_prec=1"); break;
      default: errorQuda("Unexpected precision = %d", a.GhostPrecision());
      }
      strcat(aux,comm_dim_partitioned_string());
      strcat(aux,comm_dim_topology_string());

      // record the location of where each pack buffer is in [2*dim+dir] ordering
      // 0 - no packing
      // 1 - pack to local GPU memory
      // 2 - pack to local mapped CPU memory
      // 3 - pack to remote mapped GPU memory
      char label[15] = ",dest=";
      for (int dim=0; dim<4; dim++) {
	for (int dir=0; dir<2; dir++) {
	  label[2*dim+dir+6] = !comm_dim_partitioned(dim) ? '0' : destination[2*dim+dir] == Device ? '1' : destination[2*dim+dir] == Host ? '2' : '3';
	}
      }
      label[14] = '\0';
      strcat(aux, label);
      strcat(aux, ",nFace=");
      u32toa(aux + strlen(aux), nFace);
      strcat(aux, ",spins_per_thread=");
      u32toa(aux + strlen(aux), spins_per_thread(a));
      strcat(aux, ",colors_per_thread=");
      u32toa(aux + strlen(aux), colors_per_thread(a));
      strcat(aux, ",shmem=");
      u32toa(aux + strlen(aux), shmem);
      if (v.size()) strcat(aux, ",batched");

      // compute number of number of work items we have to do
      // unlike the dslash kernels, we include the fifth dimension here
      for (int i = 0; i < 4; i++) {
        if (!comm_dim_partitioned(i)) continue;
        work_items += 2 * nFace * a.getDslashConstant().ghostFaceCB[i] * a.getDslashConstant().Ls; // 2 for forwards and backwards faces
      }

      apply(device::get_default_stream());
    }

    template <int nDim> using Arg = PackGhostArg<store_t, ghost_store_t, nSpin, nColor, nDim, order>;

    template <bool enable>
    std::enable_if_t<enable, void> launch_(const TuneParam &tp, const qudaStream_t &stream)
    {
      if (a.Ndim() == 5)
        launch<GhostPacker, true>(tp, stream, Arg<5>(a, work_items, ghost, parity, nFace, dagger, shmem, v));
      else
        launch<GhostPacker, true>(tp, stream, Arg<4>(a, work_items, ghost, parity, nFace, dagger, shmem, v));
    }

    template <bool enable>
    std::enable_if_t<!enable, void> launch_(TuneParam &, const qudaStream_t &)
    {
      errorQuda("block-float halo format not supported for nColor = %d", nColor);
    }

    void apply(const qudaStream_t &stream)
    {
      auto tp = tuneLaunch(*this, getTuning(), getVerbosity());
      launch_<(!block_float || nColor <= get_max_block_float_nc())>(tp, stream);
    }

    int blockStep() const { return block_float ? 1 : TunableKernel3D::blockStep(); }
    int blockMin() const { return block_float ? 1 : TunableKernel3D::blockMin(); }

    long long bytes() const { return work_items * 2 * a.Nspin() * a.Ncolor() * (a.Precision() + a.GhostPrecision()); }
  };

  template <int...> struct IntList { };

  template <typename Float, typename ghostFloat, int Ns, bool native, int fineColor, int coarseColor, int...N>
  bool genericPackGhostC(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
                         MemoryLocation *destination, int shmem, cvector_ref<const ColorSpinorField> &v,
                         IntList<coarseColor, N...>)
  {
    constexpr int Nc = fineColor * coarseColor;
    if (a.Ncolor() == Nc) {
      
      // don't compile if
      // 1. double precision MG unless enabled
      // 2. block-float format with arbitrary colors
      constexpr bool do_not_compile =
        (std::is_same_v<Float, double> && Nc != 3 && !is_enabled_multigrid_double()) ||
        (std::is_same_v<Float, float> && std::is_same_v<ghostFloat, short> && Nc != 3 && Ns != 2) ||
        (std::is_same_v<Float, float> && std::is_same_v<ghostFloat, int8_t> && Nc != 3 && Ns != 2);

      if constexpr (!do_not_compile) {
        constexpr QudaFieldOrder order = native ? colorspinor::getNative<Float>(Ns) : QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
        GhostPack<Float, ghostFloat, order, Ns, Nc>(ghost, a, parity, nFace, dagger, destination, shmem, v);
      } else {
        errorQuda("Not supported (Nc = %d, Ns = %d, Precision = %d, Ghost Precision = %d)",
                  a.Ncolor(), a.Nspin(), a.Precision(), a.GhostPrecision());
      }
      return true;
    } else {
      if constexpr (sizeof...(N) > 0) {
        return genericPackGhostC<Float, ghostFloat, Ns, native, fineColor>
          (ghost, a, parity, nFace, dagger, destination, shmem, v, IntList<N...>());
      }
    }
    return false;
  }

  template <typename Float, typename ghostFloat, int Ns, bool native, int fineColor, int...N>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
                        MemoryLocation *destination, int shmem, cvector_ref<const ColorSpinorField> &v,
                        IntList<fineColor, N...>)
  {
    // 1 ensures we generate templates for just the fineColor with no multiplication by coarseColor
    IntList<1, @QUDA_MULTIGRID_MRHS_LIST@, @QUDA_MULTIGRID_NC_NVEC_LIST@> coarseColors;

    if (!genericPackGhostC<Float, ghostFloat, Ns, native, fineColor>
        (ghost, a, parity, nFace, dagger, destination, shmem, v, coarseColors)) {
      if constexpr (sizeof...(N) > 0) {
        genericPackGhost<Float, ghostFloat, Ns, native>
          (ghost, a, parity, nFace, dagger, destination, shmem, v, IntList<N...>());
      } else {
        errorQuda("Nc = %d has not been instantiated", a.Ncolor());        
      }
    }
  }

  template <typename Float, typename ghostFloat, bool native>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
                        MemoryLocation *destination, int shmem, cvector_ref<const ColorSpinorField> &v)
  {
    if (!is_enabled_spin(a.Nspin())) errorQuda("nSpin=%d not enabled for this build", a.Nspin());

    IntList<@QUDA_MULTIGRID_NC_NVEC_LIST@> fineColors;
    if (a.Nspin() == 4) {
      if constexpr (is_enabled_spin(4))
        genericPackGhost<Float, ghostFloat, 4, native>(ghost, a, parity, nFace, dagger, destination, shmem, v, fineColors);
    } else if (a.Nspin() == 2) {
      if constexpr (is_enabled_spin(2))
        genericPackGhost<Float, ghostFloat, 2, native>(ghost, a, parity, nFace, dagger, destination, shmem, v, fineColors);
    } else if (a.Nspin() == 1) {
      if constexpr (is_enabled_spin(1))
        genericPackGhost<Float, ghostFloat, 1, native>(ghost, a, parity, nFace, dagger, destination, shmem, v, fineColors);
    } else {
      errorQuda("Unsupported nSpin = %d", a.Nspin());
    }
  }

  template <typename Float, typename ghostFloat>
  void genericPackGhost(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
                        MemoryLocation *destination, int shmem, cvector_ref<const ColorSpinorField> &v)
  {
    if (a.isNative()) {
      genericPackGhost<Float, ghostFloat, true>(ghost, a, parity, nFace, dagger, destination, shmem, v);
    } else if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericPackGhost<Float, ghostFloat, false>(ghost, a, parity, nFace, dagger, destination, shmem, v);
    } else {
      errorQuda("Unsupported field order = %d", a.FieldOrder());
    }
  }

  void genericPackGhost(void **ghost, const ColorSpinorField &a, QudaParity parity, int nFace, int dagger,
                        MemoryLocation *destination_, int shmem, cvector_ref<const ColorSpinorField> v)
  {
    if (a.FieldOrder() == QUDA_QOP_DOMAIN_WALL_FIELD_ORDER) {
      errorQuda("Field order %d not supported", a.FieldOrder());
    }

    // set default location to match field type
    MemoryLocation destination[2*QUDA_MAX_DIM];
    for (int i=0; i<4*2; i++) {
      destination[i] = destination_ ? destination_[i] : a.Location() == QUDA_CUDA_FIELD_LOCATION ? Device : Host;
    }

    // only do packing if one of the dimensions is partitioned
    bool partitioned = false;
    for (int d=0; d<4; d++)
      if (comm_dim_partitioned(d)) partitioned = true;
    if (!partitioned) return;

    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      if (a.GhostPrecision() == QUDA_DOUBLE_PRECISION) {
        genericPackGhost<double, double>(ghost, a, parity, nFace, dagger, destination, shmem, v);
      } else {
        errorQuda("precision = %d and ghost precision = %d not supported", a.Precision(), a.GhostPrecision());
      }
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      if (a.GhostPrecision() == QUDA_SINGLE_PRECISION) {
        genericPackGhost<float, float>(ghost, a, parity, nFace, dagger, destination, shmem, v);
      } else if (a.GhostPrecision() == QUDA_HALF_PRECISION) {
        if constexpr (is_enabled(QUDA_HALF_PRECISION))
          genericPackGhost<float, short>(ghost, a, parity, nFace, dagger, destination, shmem, v);
        else
          errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
      } else if (a.GhostPrecision() == QUDA_QUARTER_PRECISION) {
        if constexpr (is_enabled(QUDA_QUARTER_PRECISION))
          genericPackGhost<float, int8_t>(ghost, a, parity, nFace, dagger, destination, shmem, v);
        else
          errorQuda("QUDA_PRECISION=%d does not enable quarter precision", QUDA_PRECISION);
      } else {
        errorQuda("precision = %d and ghost precision = %d not supported", a.Precision(), a.GhostPrecision());
      }
    } else if (a.Precision() == QUDA_HALF_PRECISION) {
      if (a.GhostPrecision() == QUDA_HALF_PRECISION) {
        if constexpr (is_enabled(QUDA_HALF_PRECISION))
          genericPackGhost<short, short>(ghost, a, parity, nFace, dagger, destination, shmem, v);
        else
          errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
      } else {
        errorQuda("precision = %d and ghost precision = %d not supported", a.Precision(), a.GhostPrecision());
      }
    } else {
      errorQuda("Unsupported precision %d", a.Precision());
    }
  }

} // namespace quda
