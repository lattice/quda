#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstring> // needed for memset

#include <float_vector.h>

#include <tune_quda.h>
#include <typeinfo>

#include <quda_internal.h>
#include <blas_quda.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <face_quda.h> // this is where the MPI / QMP depdendent code is

#define checkSpinor(a, b)						\
  {									\
    if (a.Precision() != b.Precision())					\
      errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision()); \
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

#define checkLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

namespace quda {

  namespace blas {

#include <texture.h>

    unsigned long long flops;
    unsigned long long bytes;

    void zero(ColorSpinorField &a) { 
      if (typeid(a) == typeid(cudaColorSpinorField)) {
	static_cast<cudaColorSpinorField&>(a).zero(); 
      } else {
	static_cast<cpuColorSpinorField&>(a).zero();
      }
    }

    static hipStream_t *blasStream;

    static struct {
      const char *vol_str;
      const char *aux_str;
      char aux_tmp[TuneKey::aux_n];
    } blasStrings;

    void initReduce();
    void endReduce();

    void init()
    { 
      blasStream = &streams[Nstream-1];
      initReduce();
    }
  
    void end(void)
    {
      endReduce();
    }

    hipStream_t* getStream() { return blasStream; }

#include <blas_core.h>
#include <blas_mixed_core.h>


    template <typename Float2, typename FloatN>
    struct BlasFunctor {

      //! pre-computation routine before the main loop
      virtual __device__ __host__ void init() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) = 0;
    };

    /**
       Functor to perform the operation y = a*x + b*y
    */
    template <typename Float2, typename FloatN>
    struct axpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y = a.x*x + b.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 3; } //! flops per element
    };

    void axpby(const double &a, ColorSpinorField &x, const double &b, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpby_,0,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpby_,0,1,0,0>(make_double2(a, 0.0), make_double2(b, 0.0), make_double2(0.0, 0.0),
				 x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y += x
    */
    template <typename Float2, typename FloatN>
    struct xpy_ : public BlasFunctor<Float2,FloatN> {
      xpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y += x ; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void xpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<xpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0), 
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation y += a*x
    */
    template <typename Float2, typename FloatN>
    struct axpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      axpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = a.x*x + y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void axpy(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpy_,0,1,0,0>(make_double2(a,0.0), make_double2(1.0,0.0), make_double2(0.0,0.0),
				       x, y, x, x);
      } else {
	blasCuda<axpy_,0,1,0,0>(make_double2(a, 0.0), make_double2(1.0, 0.0), make_double2(0.0, 0.0), 
			       x, y, x, x);
      }
    }

    /**
       Functor to perform the operation y = x + a*y
    */
    template <typename Float2, typename FloatN>
    struct xpay_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      xpay_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y = x + a.x*y; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void xpay(ColorSpinorField &x, const double &a, ColorSpinorField &y) {
      blasCuda<xpay_,0,1,0,0>(make_double2(a,0.0), make_double2(0.0, 0.0), make_double2(0.0, 0.0),
			     x, y, x, x);
    }

    /**
       Functor to perform the operation y -= x;
    */
    template <typename Float2, typename FloatN>
    struct mxpy_ : public BlasFunctor<Float2,FloatN> {
      mxpy_(const Float2 &a, const Float2 &b, const Float2 &c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { y -= x; }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void mxpy(ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<mxpy_,0,1,0,0>(make_double2(1.0, 0.0), make_double2(1.0, 0.0), 
			     make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation x *= a
    */
    template <typename Float2, typename FloatN>
    struct ax_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      ax_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w) { x *= a.x; }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 1; } //! flops per element
    };

    void ax(const double &a, ColorSpinorField &x) {
      blasCuda<ax_,1,0,0,0>(make_double2(a, 0.0), make_double2(0.0, 0.0), 
			   make_double2(0.0, 0.0), x, x, x, x);
    }

    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */

    __device__ __host__ void _caxpy(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    __device__ __host__ void _caxpy(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    __device__ __host__ void _caxpy(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }

    template <typename Float2, typename FloatN>
    struct caxpy_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      caxpy_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    void caxpy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      blasCuda<caxpy_,0,1,0,0>(make_double2(real(a),imag(a)), make_double2(0.0, 0.0), 
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to perform the operation y = a*x + b*y  (complex-valued)
    */

    __device__ __host__ void _caxpby(const float2 &a, const float4 &x, const float2 &b, float4 &y)					
    { float4 yy;								
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;	
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;	
      yy.z = a.x*x.z; yy.z -= a.y*x.w; yy.z += b.x*y.z; yy.z -= b.y*y.w;	
      yy.w = a.y*x.z; yy.w += a.x*x.w; yy.w += b.y*y.z; yy.w += b.x*y.w;	
      y = yy; }

    __device__ __host__ void _caxpby(const float2 &a, const float2 &x, const float2 &b, float2 &y)
    { float2 yy;								
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;	
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;	
      y = yy; }

    __device__ __host__ void _caxpby(const double2 &a, const double2 &x, const double2 &b, double2 &y)				 
    { double2 yy;								
      yy.x = a.x*x.x; yy.x -= a.y*x.y; yy.x += b.x*y.x; yy.x -= b.y*y.y;	
      yy.y = a.y*x.x; yy.y += a.x*x.y; yy.y += b.y*y.x; yy.y += b.x*y.y;	
      y = yy; }

    template <typename Float2, typename FloatN>
    struct caxpby_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpby_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpby(a, x, b, y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 7; } //! flops per element
    };

    void caxpby(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y) {
      blasCuda<caxpby_,0,1,0,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)), 
			       make_double2(0.0, 0.0), x, y, x, x);
    }

    /**
       Functor to performs the operation z[i] = x[i] + a*y[i] + b*z[i]
    */

    __device__ __host__ void _cxpaypbz(const float4 &x, const float2 &a, const float4 &y, const float2 &b, float4 &z) {
      float4 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      zz.z = x.z + a.x*y.z; zz.z -= a.y*y.w; zz.z += b.x*z.z; zz.z -= b.y*z.w;
      zz.w = x.w + a.y*y.z; zz.w += a.x*y.w; zz.w += b.y*z.z; zz.w += b.x*z.w;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const float2 &x, const float2 &a, const float2 &y, const float2 &b, float2 &z) {
      float2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    __device__ __host__ void _cxpaypbz(const double2 &x, const double2 &a, const double2 &y, const double2 &b, double2 &z) {
      double2 zz;
      zz.x = x.x + a.x*y.x; zz.x -= a.y*y.y; zz.x += b.x*z.x; zz.x -= b.y*z.y;
      zz.y = x.y + a.y*y.x; zz.y += a.x*y.y; zz.y += b.y*z.x; zz.y += b.x*z.y;
      z = zz;
    }

    template <typename Float2, typename FloatN>
    struct cxpaypbz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cxpaypbz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _cxpaypbz(x, a, y, b, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void cxpaypbz(ColorSpinorField &x, const Complex &a, ColorSpinorField &y, 
		  const Complex &b, ColorSpinorField &z) {
      blasCuda<cxpaypbz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)), 
				 make_double2(0.0, 0.0), x, y, z, z);
    }

    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = b*z[i] + c*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyBzpcx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      axpyBzpcx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = b.x*z + c.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };

    void axpyBzpcx(const double &a, ColorSpinorField& x, ColorSpinorField& y, const double &b, 
		   ColorSpinorField& z, const double &c) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), 
					    make_double2(c,0.0),	x, y, z, x);
      } else {
	// swap arguments around 
	blasCuda<axpyBzpcx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), 
				     make_double2(c,0.0), x, y, z, x);
      }
    }
  
    /**
       Functor performing the operations: y[i] = a*x[i] + y[i]; x[i] = z[i] + b*x[i]
    */
    template <typename Float2, typename FloatN>
    struct axpyZpbx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      axpyZpbx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { y += a.x*x; x = z + b.x*x; }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void axpyZpbx(const double &a, ColorSpinorField& x, ColorSpinorField& y,
		  ColorSpinorField& z, const double &b) {
      if (x.Precision() != y.Precision()) {
	// call hacked mixed precision kernel
	mixed::blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
					   x, y, z, x);
      } else {
	// swap arguments around 
	blasCuda<axpyZpbx_,1,1,0,0>(make_double2(a,0.0), make_double2(b,0.0), make_double2(0.0,0.0),
				    x, y, z, x);
      }
    }

    /**
       Functor performing the operations z[i] = a*x[i] + b*y[i] + z[i] and y[i] -= b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypzYmbw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypzYmbw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); _caxpy(-b, w, y); }

      static int streams() { return 6; } //! total number of input and output streams
      static int flops() { return 12; } //! flops per element
    };

    void caxpbypzYmbw(const Complex &a, ColorSpinorField &x, const Complex &b, 
		      ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      blasCuda<caxpbypzYmbw_,0,1,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b), IMAG(b)), 
				     make_double2(0.0,0.0), x, y, z, w);
    }

    /**
       Functor performing the operation y[i] += a*b*x[i], x[i] *= a
    */
    template <typename Float2, typename FloatN>
    struct cabxpyAx_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      cabxpyAx_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { x *= a.x; _caxpy(b, x, y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void cabxpyAx(const double &a, const Complex &b, 
		  ColorSpinorField &x, ColorSpinorField &y) {
      // swap arguments around 
      blasCuda<cabxpyAx_,1,1,0,0>(make_double2(a,0.0), make_double2(REAL(b),IMAG(b)), 
				  make_double2(0.0,0.0), x, y, x, x);
    }

    /**
       Functor performing the operation z[i] = a*x[i] + b*y[i] + z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypz_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      caxpbypz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, z); _caxpy(b, y, z); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void caxpbypz(const Complex &a, ColorSpinorField &x, const Complex &b, 
		  ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpbypz_,0,0,1,0>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)), 
				  make_double2(0.0,0.0), x, y, z, z);
    }

    /**
       Functor Performing the operation w[i] = a*x[i] + b*y[i] + c*z[i] + w[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpbypczpw_ : public BlasFunctor<Float2,FloatN> {
      const Float2 a;
      const Float2 b;
      const Float2 c;
      caxpbypczpw_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b), c(c) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, w); _caxpy(b, y, w); _caxpy(c, z, w); }

      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 5; } //! flops per element
    };

    void caxpbypczpw(const Complex &a, ColorSpinorField &x, const Complex &b, 
		     ColorSpinorField &y, const Complex &c, ColorSpinorField &z, 
		     ColorSpinorField &w) {
      blasCuda<caxpbypczpw_,0,0,0,1>(make_double2(REAL(a),IMAG(a)), make_double2(REAL(b),IMAG(b)), 
				     make_double2(REAL(c),IMAG(c)), x, y, z, w);
    }

    /**
       double caxpyXmaz(c a, V x, V y, V z){}
   
       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmaz_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      caxpyxmaz_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmaz(const Complex &a, ColorSpinorField &x, 
		   ColorSpinorField &y, ColorSpinorField &z) {
      blasCuda<caxpyxmaz_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), 
				   make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double caxpyXmazMR(c a, V x, V y, V z){}

       First performs the operation y[i] += a*x[i]
       Second performs the operator x[i] -= a*z[i]
    */
    template <typename Float2, typename FloatN>
    struct caxpyxmazMR_ : public BlasFunctor<Float2,FloatN> {
      Float2 a;
      double3 *Ar3;
      caxpyxmazMR_(const Float2 &a, const Float2 &b, const Float2 &c)
	: a(a), Ar3(static_cast<double3*>(blas::getDeviceReduceBuffer())) { ; }

      inline __device__ __host__ void init() {
#ifdef __CUDA_ARCH__
	typedef decltype(a.x) real;
	double3 result = __ldg(Ar3);
	a.y = a.x * (real)(result.y) * ((real)1.0 / (real)result.z);
	a.x = a.x * (real)(result.x) * ((real)1.0 / (real)result.z);
#endif
      }

      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      { _caxpy(a, x, y); _caxpy(-a, z, x); }

      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    void caxpyXmazMR(const Complex &a, ColorSpinorField &x,
		     ColorSpinorField &y, ColorSpinorField &z) {
      if (!commAsyncReduction())
	errorQuda("This kernel requires asynchronous reductions to be set");
      if (x.Location() == QUDA_CPU_FIELD_LOCATION)
	errorQuda("This kernel cannot be run on CPU fields");

      blasCuda<caxpyxmazMR_,1,1,0,0>(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0),
				     make_double2(0.0, 0.0), x, y, z, x);
    }

    /**
       double tripleCGUpdate(d a, d b, V x, V y, V z, V w){}
   
       First performs the operation y[i] = y[i] + a*w[i]
       Second performs the operation z[i] = z[i] - a*x[i] 
       Third performs the operation w[i] = z[i] + b*w[i]
    */
    template <typename Float2, typename FloatN>
    struct tripleCGUpdate_ : public BlasFunctor<Float2,FloatN> {
      Float2 a, b;
      tripleCGUpdate_(const Float2 &a, const Float2 &b, const Float2 &c) : a(a), b(b) { ; }
      __device__ __host__ void operator()(FloatN &x, FloatN &y, FloatN &z, FloatN &w)
      //{ y -= a.x*x; z += a.x*w; w = y + b.x*w; }
      { y += a.x*w; z -= a.x*x; w = z + b.x*w; }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    void tripleCGUpdate(const double &a, const double &b, ColorSpinorField &x, 
			ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w) {
      if (x.Precision() != y.Precision()) {
      // call hacked mixed precision kernel
	mixed::blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a,0.0), make_double2(b,0.0), 
						 make_double2(0.0,0.0), x, y, z, w);
      } else {
	blasCuda<tripleCGUpdate_,0,1,1,1>(make_double2(a, 0.0), make_double2(b, 0.0), 
					  make_double2(0.0, 0.0), x, y, z, w);
      }
    }
  
  } // namespace blas

} // namespace quda
