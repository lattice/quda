#include "hip/hip_runtime.h"
#include <tuple>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <blas_quda.h>
#include <instantiate.h>

namespace quda {

  using namespace colorspinor;

  /**
     Random number insertion over all field elements
  */
  template <class T>
  void random(T &t) {
    for (int parity=0; parity<t.Nparity(); parity++) {
      for (int x_cb=0; x_cb<t.VolumeCB(); x_cb++) {
      	for (int s=0; s<t.Nspin(); s++) {
      	  for (int c=0; c<t.Ncolor(); c++) {
            t(parity,x_cb,s,c) = complex<typename T::real>(comm_drand(), comm_drand());
      	  }
      	}
      }
    }
  }

  /**
     Create a point source at spacetime point x, spin s and colour c
  */
  template <class T>
  void point(T &t, int x, int s, int c) { t(x%2, x/2, s, c) = 1.0; }

  /**
     Set all space-time real elements at spin s and color c of the
     field equal to k
  */
  template <class T>
  void constant(T &t, int k, int s, int c) {
    for (int parity=0; parity<t.Nparity(); parity++) {
      for (int x_cb=0; x_cb<t.VolumeCB(); x_cb++) {
      	// set all color-spin components to zero
      	for (int s2=0; s2<t.Nspin(); s2++) {
      	  for (int c2=0; c2<t.Ncolor(); c2++) {
      	    t(parity,x_cb,s2,c2) = 0.0;
      	  }
      	}
        t(parity,x_cb,s,c) = k; // now set the one we want
      }
    }
  }

  /**
     Insert a sinusoidal wave sin ( n * (x[d] / X[d]) * pi ) in dimension d
   */
  template <class P>
  void sin(P &p, int d, int n, int offset) {
    int coord[4];
    int X[4] = { p.X(0), p.X(1), p.X(2), p.X(3)};
    X[0] *= (p.Nparity() == 1) ? 2 : 1; // need full lattice dims

    for (int parity=0; parity<p.Nparity(); parity++) {
      for (int x_cb=0; x_cb<p.VolumeCB(); x_cb++) {
        getCoords(coord, x_cb, X, parity);

        double mode = n * (double)coord[d] / X[d];
        double k = (double)offset + sin (M_PI * mode);

        for (int s=0; s<p.Nspin(); s++)
          for (int c=0; c<p.Ncolor(); c++)
            p(parity, x_cb, s, c) = k;
      }
    }
  }

  /**
     Create a corner source with value "v" on color "c"
     on a single corner overloaded into "s". "s" is
     encoded via a bitmap: 1010 -> x = 0, y = 1, z = 0, t = 1
     corner, for ex.
  */
  template <class T>
  void corner(T &p, int v, int s, int c) {
    if (p.Nspin() != 1) errorQuda("corner() is only defined for Nspin = 1 fields");

    int coord[4];
    int X[4] = { p.X(0), p.X(1), p.X(2), p.X(3)};
    X[0] *= (p.Nparity() == 1) ? 2 : 1; // need full lattice dims

    for (int parity=0; parity<p.Nparity(); parity++) {
      for (int x_cb=0; x_cb<p.VolumeCB(); x_cb++) {

        // get coords
        getCoords(coord, x_cb, X, parity);

        // Figure out corner of current site.
        int corner = 8*(coord[3]%2)+4*(coord[2]%2)+2*(coord[1]%2)+(coord[0]%2);

        // set all color components to zero
        for (int c2=0; c2<p.Ncolor(); c2++) {
          p(parity,x_cb,0,c2) = 0.0;
        }
        // except the corner and color we want
        if (s == corner)
          p(parity,x_cb,0,c) = (double)v;
      }
    }
  }

  // print out the vector at volume point x
  template <typename Float, int nSpin, int nColor, QudaFieldOrder order, typename pack_t>
  void genericSource(const pack_t &pack)
  {
    auto &a = std::get<0>(pack);
    auto &sourceType = std::get<1>(pack);
    auto &x = std::get<2>(pack);
    auto &s = std::get<3>(pack);
    auto &c = std::get<4>(pack);

    FieldOrderCB<Float,nSpin,nColor,1,order> A(a);
    if (sourceType == QUDA_RANDOM_SOURCE) random(A);
    else if (sourceType == QUDA_POINT_SOURCE) point(A, x, s, c);
    else if (sourceType == QUDA_CONSTANT_SOURCE) constant(A, x, s, c);
    else if (sourceType == QUDA_SINUSOIDAL_SOURCE) sin(A, x, s, c);
    else if (sourceType == QUDA_CORNER_SOURCE) corner(A, x, s, c);
    else errorQuda("Unsupported source type %d", sourceType);
  }

  template <typename Float, int nSpin, QudaFieldOrder order, typename pack_t>
  void genericSource(const pack_t &pack)
  {
    auto &a = std::get<0>(pack);
    if (a.Ncolor() == 3) {
      genericSource<Float,nSpin,3,order>(pack);
#ifdef GPU_MULTIGRID
    } else if (a.Ncolor() == 4) {
      genericSource<Float,nSpin,4,order>(pack);
    } else if (a.Ncolor() == 6) { // for Wilson free field
      genericSource<Float,nSpin,6,order>(pack);
    } else if (a.Ncolor() == 8) {
      genericSource<Float,nSpin,8,order>(pack);
    } else if (a.Ncolor() == 12) {
      genericSource<Float,nSpin,12,order>(pack);
    } else if (a.Ncolor() == 16) {
      genericSource<Float,nSpin,16,order>(pack);
    } else if (a.Ncolor() == 20) {
      genericSource<Float,nSpin,20,order>(pack);
    } else if (a.Ncolor() == 24) {
      genericSource<Float,nSpin,24,order>(pack);
#ifdef NSPIN4
    } else if (a.Ncolor() == 32) {
      genericSource<Float,nSpin,32,order>(pack);
#endif // NSPIN4
#ifdef NSPIN1
    } else if (a.Ncolor() == 64) {
      genericSource<Float,nSpin,64,order>(pack);
    } else if (a.Ncolor() == 96) {
      genericSource<Float,nSpin,96,order>(pack);
#endif // NSPIN1
#endif // GPU_MULTIGRID
    } else {
      errorQuda("Unsupported nColor=%d", a.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order, typename pack_t>
  void genericSource(const pack_t &pack)
  {
    auto &a = std::get<0>(pack);
    if (a.Nspin() == 1) {
#ifdef NSPIN1
      genericSource<Float,1,order>(pack);
#else
      errorQuda("nSpin=1 not enabled for this build");
#endif
    } else if (a.Nspin() == 2) {
#ifdef NSPIN2
      genericSource<Float,2,order>(pack);
#else
      errorQuda("nSpin=2 not enabled for this build");
#endif
    } else if (a.Nspin() == 4) {
#ifdef NSPIN4
      genericSource<Float,4,order>(pack);
#else
      errorQuda("nSpin=4 not enabled for this build");
#endif
    } else {
      errorQuda("Unsupported nSpin=%d", a.Nspin());
    }
  }

  template <typename Float, typename pack_t>
  void genericSource(const pack_t &pack)
  {
    auto &a = std::get<0>(pack);
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericSource<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(pack);
    } else {
      errorQuda("Unsupported field order %d", a.FieldOrder());
    }
  }

  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c)
  {
    using pack_t = std::tuple<cpuColorSpinorField&, QudaSourceType, int, int, int>;
    pack_t pack(a, sourceType, x, s, c);
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericSource<double>(pack);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericSource<float>(pack);
    } else {
      errorQuda("Precision not supported");
    }
  }

  template <class U, class V>
  int compareSpinor(const U &u, const V &v, const int tol) {
    int fail_check = 16*tol;
    int *fail = new int[fail_check];
    for (int f=0; f<fail_check; f++) fail[f] = 0;

    int N = 2*u.Nspin()*u.Ncolor();
    int *iter = new int[N];
    for (int i=0; i<N; i++) iter[i] = 0;

    for (int parity=0; parity<v.Nparity(); parity++) {
      for (int x_cb=0; x_cb<u.VolumeCB(); x_cb++) {

	for (int s=0; s<u.Nspin(); s++) {
	  for (int c=0; c<u.Ncolor(); c++) {
            complex<double> u_ = u(parity, x_cb, s, c);
            complex<double> v_ = v(parity, x_cb, s, c);

            double diff_real = fabs(u_.real() - v_.real());
            double diff_imag = fabs(u_.imag() - v_.imag());

            for (int f=0; f<fail_check; f++) {
              if (diff_real > pow(10.0,-(f+1)/(double)tol) || std::isnan(diff_real)) fail[f]++;
              if (diff_imag > pow(10.0,-(f+1)/(double)tol) || std::isnan(diff_imag)) fail[f]++;
            }

            int j = (s * u.Ncolor() + c) * 2;
            if (diff_real > 1e-3 || std::isnan(diff_real)) iter[j+0]++;
            if (diff_imag > 1e-3 || std::isnan(diff_imag)) iter[j+1]++;
	  }
	}
      }
    }

    // reduce over all processes
    for (int i=0; i<N; i++) comm_allreduce_int(&iter[i]);
    for (int f=0; f<fail_check; f++) comm_allreduce_int(&fail[f]);

    for (int i=0; i<N; i++) printfQuda("%d fails = %d\n", i, iter[i]);

    int accuracy_level =0;
    for (int f=0; f<fail_check; f++) {
      if (fail[f] == 0) accuracy_level = f+1;
    }

    size_t total = u.Nparity()*u.VolumeCB()*N*comm_size();
    for (int f=0; f<fail_check; f++) {
      printfQuda("%e Failures: %d / %lu  = %e\n", pow(10.0,-(f+1)/(double)tol),
		 fail[f], total, fail[f] / (double)total);
    }

    delete []iter;
    delete []fail;

    return accuracy_level;
  }

  template <typename oFloat, typename iFloat, QudaFieldOrder order>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.Ncolor() == 3) {
      constexpr int Nc = 3;
      if (a.Nspin() == 4) {
        constexpr int Ns = 4;
        FieldOrderCB<oFloat,Ns,Nc,1,order> A(a);
	FieldOrderCB<iFloat,Ns,Nc,1,order> B(b);

        double rescale = 1.0 / A.abs_max();

        auto a_(a), b_(b);
        blas::ax(rescale, a_);
        blas::ax(rescale, b_);
        FieldOrderCB<oFloat, Ns, Nc, 1, order> A_(a_);
        FieldOrderCB<iFloat, Ns, Nc, 1, order> B_(b_);

        ret = compareSpinor(A_, B_, tol);
      } else if (a.Nspin() == 1) {
        constexpr int Ns = 1;
        FieldOrderCB<oFloat,Ns,Nc,1,order> A(a);
	FieldOrderCB<iFloat,Ns,Nc,1,order> B(b);

        double rescale = 1.0 / A.abs_max();

        auto a_(a), b_(b);
        blas::ax(rescale, a_);
        blas::ax(rescale, b_);
        FieldOrderCB<oFloat, Ns, Nc, 1, order> A_(a_);
        FieldOrderCB<iFloat, Ns, Nc, 1, order> B_(b_);

        ret = compareSpinor(A_, B_, tol);
      }
    } else {
      errorQuda("Number of colors %d not supported", a.Ncolor());
    }
    return ret;
  }


  template <typename oFloat, typename iFloat>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && b.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      ret = genericCompare<oFloat,iFloat,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(a, b, tol);
    } else {
      errorQuda("Unsupported field order %d", a.FieldOrder());
    }
    return ret;
  }


  template <typename oFloat>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (b.Precision() == QUDA_DOUBLE_PRECISION) {
      ret = genericCompare<oFloat,double>(a, b, tol);
    } else if (b.Precision() == QUDA_SINGLE_PRECISION) {
      ret = genericCompare<oFloat,float>(a, b, tol);
    } else {
      errorQuda("Precision not supported");
    }
    return ret;
  }


  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      ret = genericCompare<double>(a, b, tol);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      ret = genericCompare<float>(a, b, tol);
    } else {
      errorQuda("Precision not supported");
    }
    return ret;
  }


  template <class Order>
  void print_vector(const Order &o, unsigned int x)
  {
    int x_cb = x / o.Nparity();
    int parity = x % o.Nparity();

    for (int s = 0; s < o.Nspin(); s++) {
      printfQuda("x = %u, s = %d, { ", x_cb, s);
      for (int c = 0; c < o.Ncolor(); c++) {
        auto value = complex<double>(o(parity, x_cb, s, c));
        printfQuda("(%f,%f) ", value.real(), value.imag());
      }
      printfQuda("}\n");
    }
  }

  // print out the vector at volume point x
  template <typename Float, QudaFieldOrder order> void genericPrintVector(const cpuColorSpinorField &a, unsigned int x)
  {
    if (a.Ncolor() == 3 && a.Nspin() == 1)  {
      FieldOrderCB<Float,1,3,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 3 && a.Nspin() == 4)  {
      FieldOrderCB<Float,4,3,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 3 && a.Nspin() == 1)  {
      FieldOrderCB<Float,1,3,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 2 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,2,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 24 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,24,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 6 && a.Nspin() == 4) {
      FieldOrderCB<Float,4,6,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 72 && a.Nspin() == 4) {
      FieldOrderCB<Float,4,72,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 576 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,576,1,order> A(a);
      print_vector(A, x);
    }
#ifdef GPU_STAGGERED_DIRAC
    else if (a.Ncolor() == 64 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,64,1,order> A(a);
      print_vector(A, x);
    } 
else if (a.Ncolor() == 96 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,96,1,order> A(a);
      print_vector(A, x);
    } 
#endif
    else {
      errorQuda("Not supported Ncolor = %d, Nspin = %d", a.Ncolor(), a.Nspin());	 
    }
  }

  // print out the vector at volume point x
  template <typename Float> void genericPrintVector(const cpuColorSpinorField &a, unsigned int x)
  {
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericPrintVector<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(a,x);
    } else {
      errorQuda("Unsupported field order %d", a.FieldOrder());
    }
  }

  // print out the vector at volume point x
  void genericPrintVector(const cpuColorSpinorField &a, unsigned int x)
  {
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericPrintVector<double>(a,x);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericPrintVector<float>(a,x);
    } else {
      errorQuda("Precision %d not implemented", a.Precision());
    }
  }

  // Eventually we should merge the below device print function and
  // the above host print function.

  template <typename StoreType, int Ns, int Nc, QudaFieldOrder FieldOrder>
  void genericCudaPrintVector(const cudaColorSpinorField &field, unsigned int i)
  {

    typedef colorspinor::AccessorCB<StoreType, Ns, Nc, 1, FieldOrder> AccessorType;

    AccessorType A(field);

    // Register type
    typedef typename scalar<typename mapper<StoreType>::type>::type Float;

    // Allocate a real+imag component for the storage type.
    StoreType indiv_num[2];

    // Allocate space for the full site.
    Float *data_cpu = new Float[2 * Ns * Nc];

    // Grab the pointer to the field.
    complex<StoreType> *field_ptr = (complex<StoreType> *)field.V();

    // Grab the pointer to the norm field. Might be ignored as appropriate.
    float *norm_ptr = (float *)field.Norm();
    float scale = 1.0;

    if (isFixed<StoreType>::value) {
      qudaMemcpy(&scale, &norm_ptr[i], sizeof(float), qudaMemcpyDeviceToHost);
      scale *= fixedInvMaxValue<StoreType>::value;
    }

    for (int s = 0; s < Ns; s++) {
      for (int c = 0; c < Nc; c++) {
        qudaMemcpy(indiv_num, &field_ptr[A.index(i % 2, i / 2, s, c, 0)], 2 * sizeof(StoreType), qudaMemcpyDeviceToHost);
        data_cpu[2 * (c + Nc * s)] = scale * static_cast<Float>(indiv_num[0]);
        data_cpu[2 * (c + Nc * s) + 1] = scale * static_cast<Float>(indiv_num[1]);
      }
    }
    // print
    for (int s = 0; s < Ns; s++) {
      printfQuda("x = %u, s = %d, { ", i, s);
      for (int c = 0; c < Nc; c++) {
        printfQuda("(%f,%f) ", data_cpu[(s * Nc + c) * 2], data_cpu[(s * Nc + c) * 2 + 1]);
      }
      printfQuda("}\n");
    }

    delete[] data_cpu;
  }

  template <typename Float, int Ns, int Nc>
  void genericCudaPrintVector(const cudaColorSpinorField &field, unsigned int i)
  {
    switch (field.FieldOrder()) {
    case QUDA_FLOAT_FIELD_ORDER: genericCudaPrintVector<Float, Ns, Nc, QUDA_FLOAT_FIELD_ORDER>(field, i); break;
    case QUDA_FLOAT2_FIELD_ORDER: genericCudaPrintVector<Float, Ns, Nc, QUDA_FLOAT2_FIELD_ORDER>(field, i); break;
    case QUDA_FLOAT4_FIELD_ORDER: genericCudaPrintVector<Float, Ns, Nc, QUDA_FLOAT4_FIELD_ORDER>(field, i); break;
    case QUDA_FLOAT8_FIELD_ORDER: genericCudaPrintVector<Float, Ns, Nc, QUDA_FLOAT8_FIELD_ORDER>(field, i); break;
    case QUDA_SPACE_SPIN_COLOR_FIELD_ORDER:
      genericCudaPrintVector<Float, Ns, Nc, QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(field, i);
      break;
    case QUDA_SPACE_COLOR_SPIN_FIELD_ORDER:
      genericCudaPrintVector<Float, Ns, Nc, QUDA_SPACE_COLOR_SPIN_FIELD_ORDER>(field, i);
      break;
    default: errorQuda("Unsupported field order %d", field.FieldOrder());
    }
  }

  template <typename Float> struct GenericCudaPrintVector {
    GenericCudaPrintVector(const cudaColorSpinorField &field, unsigned int i)
    {
      if (field.Ncolor() == 3 && field.Nspin() == 4) {
        genericCudaPrintVector<Float, 4, 3>(field, i);
      } else if (field.Ncolor() == 3 && field.Nspin() == 1) {
        genericCudaPrintVector<Float, 1, 3>(field, i);
      } else if (field.Ncolor() == 6 && field.Nspin() == 2) { // wilson free field MG
        genericCudaPrintVector<Float, 2, 6>(field, i);
      } else if (field.Ncolor() == 24 && field.Nspin() == 2) { // common value for Wilson, also staggered free field
        genericCudaPrintVector<Float, 2, 24>(field, i);
      } else if (field.Ncolor() == 32 && field.Nspin() == 2) {
        genericCudaPrintVector<Float, 2, 32>(field, i);
#ifdef GPU_STAGGERED_DIRAC
      } else if (field.Ncolor() == 64 && field.Nspin() == 2) {
        genericCudaPrintVector<Float, 2, 64>(field, i);
      } else if (field.Ncolor() == 96 && field.Nspin() == 2) {
        genericCudaPrintVector<Float, 2, 96>(field, i);
#endif
      } else {
        errorQuda("Not supported Ncolor = %d, Nspin = %d", field.Ncolor(), field.Nspin());
      }
    }
  };

  void genericCudaPrintVector(const cudaColorSpinorField &field, unsigned int i)
  {
    instantiatePrecision<GenericCudaPrintVector>(field, i);
  }

} // namespace quda
