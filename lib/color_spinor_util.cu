#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>

namespace quda {

  using namespace colorspinor;

  /**
     Random number insertion over all field elements
  */
  template <class T>
  void random(T &t) {
    for (int parity=0; parity<t.Nparity(); parity++) {
      for (int x_cb=0; x_cb<t.VolumeCB(); x_cb++) {
	for (int s=0; s<t.Nspin(); s++) {
	  for (int c=0; c<t.Ncolor(); c++) {
	    t(parity,x_cb,s,c).real(comm_drand());
	    t(parity,x_cb,s,c).imag(comm_drand());
	  }
	}
      }
    }
  }

  /**
     Create a point source at spacetime point x, spin s and colour c
  */
  template <class T>
  void point(T &t, int x, int s, int c) { t(x%2, x/2, s, c) = 1.0; }

  /**
     Set all space-time real elements at spin s and color c of the
     field equal to k
  */
  template <class T>
  void constant(T &t, int k, int s, int c) {
    for (int parity=0; parity<t.Nparity(); parity++) {
      for (int x_cb=0; x_cb<t.VolumeCB(); x_cb++) {
	// set all color-spin components to zero
	for (int s2=0; s2<t.Nspin(); s2++) {
	  for (int c2=0; c2<t.Ncolor(); c2++) {
	    t(parity,x_cb,s2,c2) = 0.0;
	  }
	}
	t(parity,x_cb,s,c) = k; // now set the one we want
      }
    }
  }

  /**
     Insert a sinusoidal wave sin ( n * (x[d] / X[d]) * pi ) in dimension d
   */
  template <class P>
  void sin(P &p, int d, int n, int offset) {
    int coord[4];
    int X[4] = { p.X(0), p.X(1), p.X(2), p.X(3)};
    X[0] *= (p.Nparity() == 1) ? 2 : 1; // need full lattice dims

    for (int parity=0; parity<p.Nparity(); parity++) {
      for (int x_cb=0; x_cb<p.VolumeCB(); x_cb++) {
	getCoords(coord, x_cb, X, parity);

	double mode = n * (double)coord[d] / X[d];
	double k = (double)offset + sin (M_PI * mode);

	for (int s=0; s<p.Nspin(); s++)
	  for (int c=0; c<p.Ncolor(); c++)
	    p(parity, x_cb, s, c) = k;
      }
    }
  }

  // print out the vector at volume point x
  template <typename Float, int nSpin, int nColor, QudaFieldOrder order>
  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {
    FieldOrderCB<Float,nSpin,nColor,1,order> A(a);
    if (sourceType == QUDA_RANDOM_SOURCE) random(A);
    else if (sourceType == QUDA_POINT_SOURCE) point(A, x, s, c);
    else if (sourceType == QUDA_CONSTANT_SOURCE) constant(A, x, s, c);
    else if (sourceType == QUDA_SINUSOIDAL_SOURCE) sin(A, x, s, c);
    else errorQuda("Unsupported source type %d", sourceType);
  }

  template <typename Float, int nSpin, QudaFieldOrder order>
  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {
    if (a.Ncolor() == 2) {
      genericSource<Float,nSpin,2,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 3) {
      genericSource<Float,nSpin,3,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 4) {
      genericSource<Float,nSpin,4,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 8) {
      genericSource<Float,nSpin,8,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 12) {
      genericSource<Float,nSpin,12,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 16) {
      genericSource<Float,nSpin,16,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 20) {
      genericSource<Float,nSpin,20,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 24) {
      genericSource<Float,nSpin,24,order>(a,sourceType, x, s, c);
    } else if (a.Ncolor() == 32) {
      genericSource<Float,nSpin,32,order>(a,sourceType, x, s, c);
    } else {
      errorQuda("Unsupported nColor=%d\n", a.Ncolor());
    }
  }

  template <typename Float, QudaFieldOrder order>
  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {
    if (a.Nspin() == 1) {
      genericSource<Float,1,order>(a,sourceType, x, s, c);
    } else if (a.Nspin() == 2) {
      genericSource<Float,2,order>(a,sourceType, x, s, c);
    } else if (a.Nspin() == 4) {
      genericSource<Float,4,order>(a,sourceType, x, s, c);
    } else {
      errorQuda("Unsupported nSpin=%d\n", a.Nspin());
    }
  }

  template <typename Float>
  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericSource<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(a,sourceType, x, s, c);
    } else {
      errorQuda("Unsupported field order %d\n", a.FieldOrder());
    }

  }

  void genericSource(cpuColorSpinorField &a, QudaSourceType sourceType, int x, int s, int c) {

    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericSource<double>(a,sourceType, x, s, c);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericSource<float>(a,sourceType, x, s, c);      
    } else {
      errorQuda("Precision not supported");
    }

  }


  template <class U, class V>
  int compareSpinor(const U &u, const V &v, const int tol) {
    int fail_check = 16*tol;
    int *fail = new int[fail_check];
    for (int f=0; f<fail_check; f++) fail[f] = 0;

    int N = 2*u.Nspin()*u.Ncolor();
    int *iter = new int[N];
    for (int i=0; i<N; i++) iter[i] = 0;

    for (int parity=0; parity<v.Nparity(); parity++) {
      for (int x_cb=0; x_cb<u.VolumeCB(); x_cb++) {

	for (int s=0; s<u.Nspin(); s++) {
	  for (int c=0; c<u.Ncolor(); c++) {
	    for (int z=0; z<2; z++) {
	      int j = (s*u.Ncolor() + c)*2+z;

	      double diff = z==0 ? fabs(u(parity,x_cb,s,c,z).real() - v(parity,x_cb,s,c,z).real()) :
		fabs(u(parity,x_cb,s,c).imag() - v(parity,x_cb,s,c).imag());

	      for (int f=0; f<fail_check; f++) {
		if (diff > pow(10.0,-(f+1)/(double)tol)) {
		  fail[f]++;
		}
	      }

	      if (diff > 1e-3) iter[j]++;
	    }
	  }
	}
      }
    }

    // reduce over all processes
    for (int i=0; i<N; i++) comm_allreduce_int(&iter[i]);
    for (int f=0; f<fail_check; f++) comm_allreduce_int(&fail[f]);

    for (int i=0; i<N; i++) printfQuda("%d fails = %d\n", i, iter[i]);

    int accuracy_level =0;
    for (int f=0; f<fail_check; f++) {
      if (fail[f] == 0) accuracy_level = f+1;
    }

    size_t total = u.Nparity()*u.VolumeCB()*N*comm_size();
    for (int f=0; f<fail_check; f++) {
      printfQuda("%e Failures: %d / %lu  = %e\n", pow(10.0,-(f+1)/(double)tol),
		 fail[f], total, fail[f] / (double)total);
    }

    delete []iter;
    delete []fail;

    return accuracy_level;
  }

  template <typename oFloat, typename iFloat, QudaFieldOrder order>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.Ncolor() == 3) {
      const int Nc = 3;
      if (a.Nspin() == 4) {
	const int Ns = 4;
	FieldOrderCB<oFloat,Ns,Nc,1,order> A(a);
	FieldOrderCB<iFloat,Ns,Nc,1,order> B(b);
	ret = compareSpinor(A, B, tol);
      } else if (a.Nspin() == 1) {
	const int Ns = 1;
	FieldOrderCB<oFloat,Ns,Nc,1,order> A(a);
	FieldOrderCB<iFloat,Ns,Nc,1,order> B(b);
	ret = compareSpinor(A, B, tol);
      }
    } else {
      errorQuda("Number of colors %d not supported", a.Ncolor());
    }
    return ret;
  }


  template <typename oFloat, typename iFloat>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER &&
	a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      ret = genericCompare<oFloat,iFloat,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(a, b, tol);
    } else {
      errorQuda("Unsupported field order %d\n", a.FieldOrder());
    }
    return ret;
  }


  template <typename oFloat>
  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (b.Precision() == QUDA_DOUBLE_PRECISION) {
      ret = genericCompare<oFloat,double>(a, b, tol);
    } else if (b.Precision() == QUDA_SINGLE_PRECISION) {
      ret = genericCompare<oFloat,float>(a, b, tol);
    } else {
      errorQuda("Precision not supported");
    }
    return ret;
  }


  int genericCompare(const cpuColorSpinorField &a, const cpuColorSpinorField &b, int tol) {
    int ret = 0;
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      ret = genericCompare<double>(a, b, tol);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      ret = genericCompare<float>(a, b, tol);
    } else {
      errorQuda("Precision not supported");
    }
    return ret;
  }


  template <class Order>
  void print_vector(const Order &o, unsigned int x) {

    int x_cb = x / o.Nparity();
    int parity = x%o.Nparity();

    for (int s=0; s<o.Nspin(); s++) {
      std::cout << "x = " << x << ", s = " << s << ", { ";
      for (int c=0; c<o.Ncolor(); c++) {
	std::cout << o(parity, x_cb, s, c) ;
	std::cout << ((c<o.Ncolor()-1) ? " , "  : " " ) ;
      }
      std::cout << "}" << std::endl;
    }

  }

  // print out the vector at volume point x
  template <typename Float, QudaFieldOrder order>
  void genericPrintVector(cpuColorSpinorField &a, unsigned int x) {
    if (a.Ncolor() == 3 && a.Nspin() == 4)  {
      FieldOrderCB<Float,4,3,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 2 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,2,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 24 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,24,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 6 && a.Nspin() == 4) {
      FieldOrderCB<Float,4,6,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 72 && a.Nspin() == 4) {
      FieldOrderCB<Float,4,72,1,order> A(a);
      print_vector(A, x);
    }
    else if (a.Ncolor() == 576 && a.Nspin() == 2) {
      FieldOrderCB<Float,2,576,1,order> A(a);
      print_vector(A, x);
    }    
    else {
      errorQuda("Not supported Ncolor = %d, Nspin = %d", a.Ncolor(), a.Nspin());	 
    }
  }

  // print out the vector at volume point x
  template <typename Float>
  void genericPrintVector(cpuColorSpinorField &a, unsigned int x) {
    if (a.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      genericPrintVector<Float,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(a,x);
    } else {
      errorQuda("Unsupported field order %d\n", a.FieldOrder());
    }
  }

  // print out the vector at volume point x
  void genericPrintVector(cpuColorSpinorField &a, unsigned int x) {
    if (a.Precision() == QUDA_DOUBLE_PRECISION) {
      genericPrintVector<double>(a,x);
    } else if (a.Precision() == QUDA_SINGLE_PRECISION) {
      genericPrintVector<float>(a,x);
    } else {
      errorQuda("Precision %d not implemented", a.Precision()); 
    }    
  }

} // namespace quda
