#include <dslash.h>
#include <worker.h>
#include <dslash_helper.cuh>
#include <color_spinor_field_order.h>
#include <gauge_field_order.h>
#include <color_spinor.h>
#include <dslash_helper.cuh>
#include <index_helper.cuh>
#include <gauge_field.h>
#include <uint_to_char.h>

#include <dslash_policy.hpp>
#include <kernels/laplace.cuh>

/**
   This is the laplacian derivative based on the basic gauged differential operator
*/

namespace quda
{

  template <typename Arg> class Laplace : public Dslash<laplace, Arg>
  {
    using Dslash = Dslash<laplace, Arg>;
    using Dslash::arg;
    using Dslash::halo;
    using Dslash::in;

  public:
    Laplace(Arg &arg, cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
            const ColorSpinorField &halo) :
      Dslash(arg, out, in, halo)
    {
    }

    void apply(const qudaStream_t &stream) override
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);

      // operator is Hermitian so do not instantiate dagger
      if (arg.nParity == 1) {
        if (arg.xpay)
          Dslash::template instantiate<packStaggeredShmem, 1, false, true>(tp, stream);
        else
          Dslash::template instantiate<packStaggeredShmem, 1, false, false>(tp, stream);
      } else if (arg.nParity == 2) {
        if (arg.xpay)
          Dslash::template instantiate<packStaggeredShmem, 2, false, true>(tp, stream);
        else
          Dslash::template instantiate<packStaggeredShmem, 2, false, false>(tp, stream);
      }
    }

    long long flops() const override
    {
      int mv_flops = (8 * in.Ncolor() - 2) * in.Ncolor(); // SU(3) matrix-vector flops
      int ghost_flops = (in.Nspin() * mv_flops + 2 * in.Ncolor() * in.Nspin());
      int xpay_flops = 2 * 2 * in.Ncolor() * in.Nspin(); // multiply and add per real component
      int num_dir = (arg.dir == 4 ? 2 * 4 : 2 * 3);      // 3D or 4D operator

      long long flops_ = 0;

      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
        flops_ = (ghost_flops + (arg.xpay ? xpay_flops : xpay_flops / 2)) * 2 * halo.GhostFace()[arg.kernel_type];
        break;
      case EXTERIOR_KERNEL_ALL: {
        long long ghost_sites
          = 2 * (halo.GhostFace()[0] + halo.GhostFace()[1] + halo.GhostFace()[2] + halo.GhostFace()[3]);
        flops_ = (ghost_flops + (arg.xpay ? xpay_flops : xpay_flops / 2)) * ghost_sites;
        break;
      }
      case INTERIOR_KERNEL:
      case UBER_KERNEL:
      case KERNEL_POLICY: {
        long long sites = halo.Volume();
        flops_ = (num_dir * in.Nspin() * mv_flops +                  // SU(3) matrix-vector multiplies
                  ((num_dir - 1) * 2 * in.Ncolor() * in.Nspin()))
          * sites; // accumulation
        if (arg.xpay) flops_ += xpay_flops * sites;

        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for flops done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * halo.GhostFace()[d];
        flops_ -= ghost_flops * ghost_sites;

        break;
      }
      }

      return flops_;
    }

    virtual long long bytes() const override
    {
      int gauge_bytes = arg.reconstruct * in.Precision();
      int spinor_bytes = 2 * in.Ncolor() * in.Nspin() * in.Precision() + (isFixed<typename Arg::Float>::value ? sizeof(float) : 0);
      int proj_spinor_bytes = in.Nspin() == 4 ? spinor_bytes / 2 : spinor_bytes;
      int ghost_bytes = (proj_spinor_bytes + gauge_bytes) + 2 * spinor_bytes; // 2 since we have to load the partial
      int num_dir = (arg.dir == 4 ? 2 * 4 : 2 * 3);                           // 3D or 4D operator

      long long bytes_ = 0;

      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T: bytes_ = ghost_bytes * 2 * halo.GhostFace()[arg.kernel_type]; break;
      case EXTERIOR_KERNEL_ALL: {
        long long ghost_sites
          = 2 * (halo.GhostFace()[0] + halo.GhostFace()[1] + halo.GhostFace()[2] + halo.GhostFace()[3]);
        bytes_ = ghost_bytes * ghost_sites;
        break;
      }
      case INTERIOR_KERNEL:
      case UBER_KERNEL:
      case KERNEL_POLICY: {
        long long sites = halo.Volume();
        bytes_ = (num_dir * gauge_bytes + ((num_dir - 2) * spinor_bytes + 2 * proj_spinor_bytes) + spinor_bytes) * sites;
        if (arg.xpay) bytes_ += spinor_bytes;
	
        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for bytes done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * halo.GhostFace()[d];
        bytes_ -= ghost_bytes * ghost_sites;
	
        break;
      }
      }
      return bytes_;
    }
    
    TuneKey tuneKey() const override
    { // add laplace transverse dir to the key
      auto key = Dslash::tuneKey();
      strcat(key.aux, ",laplace=");
      u32toa(key.aux + strlen(key.aux), arg.dir);
      return key;
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct LaplaceApply {

    LaplaceApply(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
                 cvector_ref<const ColorSpinorField> &x, const GaugeField &U, int dir, double a, double b, int parity,
                 bool dagger, const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      auto halo = ColorSpinorField::create_comms_batch(in);
      if (in.Nspin() == 1) {
        constexpr int nSpin = 1;
        LaplaceArg<Float, nSpin, nColor, nDim, recon> arg(out, in, halo, U, dir, a, b, x, parity, dagger, comm_override);
        Laplace<decltype(arg)> laplace(arg, out, in, halo);
        dslash::DslashPolicyTune<decltype(laplace)> policy(laplace, in, halo, profile);
      } else if (in.Nspin() == 4) {
        constexpr int nSpin = 4;
        LaplaceArg<Float, nSpin, nColor, nDim, recon> arg(out, in, halo, U, dir, a, b, x, parity, dagger, comm_override);
        Laplace<decltype(arg)> laplace(arg, out, in, halo);
        dslash::DslashPolicyTune<decltype(laplace)> policy(laplace, in, halo, profile);
      } else {
        errorQuda("Unsupported nSpin= %d", in.Nspin());
      }
    }
  };

  // Apply the Laplace operator
  // out(x) = M*in = - a*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu) + b*in(x)
  // Omits direction 'dir' from the operator.
  void ApplyLaplace(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in, const GaugeField &U,
                    int dir, double a, double b, cvector_ref<const ColorSpinorField> &x, int parity, bool dagger,
                    const int *comm_override, TimeProfile &profile)
  {
    if constexpr (is_enabled<QUDA_LAPLACE_DSLASH>()) {
      instantiate<LaplaceApply>(out, in, x, U, dir, a, b, parity, dagger, comm_override, profile);
    } else {
      errorQuda("Laplace operator has not been enabled");
    }
  }
} // namespace quda
