#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
//#include "cuda_common.h"
#include "random.h"
#include <hip/hip_runtime.h>
#include <quda_internal.h>

#include <comm_quda.h>


namespace quda {


#define BLOCKSDIVUP(a, b)  (((a)+(b)-1)/(b))


dim3 GetBlockDim(size_t threads, size_t size){
    /*uint blockx = BLOCKSDIVUP(size, threads);
    uint blocky = 1;
    if(blockx > PARAMS::GPUGridDimX){
        blocky = BLOCKSDIVUP(blockx, PARAMS::GPUGridDimX);
        blockx = PARAMS::GPUGridDimX;
    }
    dim3 blocks(blockx,blocky,1);
    return blocks;*/

    int blockx = BLOCKSDIVUP(size, threads);
    dim3 blocks(blockx,1,1);
    return blocks;
}




#  define CUDA_SAFE_CALL_NO_SYNC( call) {                               \
        hipError_t err = call;                                           \
        if( hipSuccess != err) {                                       \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString( err) );     \
            exit(EXIT_FAILURE);                                         \
        } }

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);   

/**
    @brief CUDA kernel to initialize CURAND RNG states
    @param state CURAND RNG state array
    @param seed initial seed for RNG
    @param rng_size size of the CURAND RNG state array
    @param node_offset this parameter is used to skip ahead the index in the sequence, usefull for multigpu. 
*/
__global__ void 
kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset ){
//#if (__CUDA_ARCH__ >= 300)
    int id = blockIdx.x * blockDim.x + threadIdx.x;
/*#else
    int id = gridDim.x * blockIdx.y + blockIdx.x;
    id = blockDim.x * id + threadIdx.x; 
#endif*/
    if(id < rng_size){
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(seed, id + node_offset, 0, &state[id]);
    }
}

struct rngArg{
    int comm_dim[4];
    int comm_coord[4];
    int X[4];
};


static __device__ __host__ inline void getCoords3(int x[4], int cb_index, const int X[4], int parity) {
  /*x[3] = cb_index/(X[2]*X[1]*X[0]/2);
  x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
  x[1] = (cb_index/(X[0]/2)) % X[1];
  x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);*/
  int za = (cb_index / (X[0]/2));
  int zb =  (za / X[1]);
  x[1] = za - zb * X[1];
  x[3] = (zb / X[2]);
  x[2] = zb - x[3] * X[2];
  int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
  x[0] = (2 * cb_index + x1odd)  - za * X[0];
  return;
}


__global__ void 
kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset, rngArg arg ){
//#if (__CUDA_ARCH__ >= 300)
    int id = blockIdx.x * blockDim.x + threadIdx.x;
/*#else
    int id = gridDim.x * blockIdx.y + blockIdx.x;
    id = blockDim.x * id + threadIdx.x; 
#endif*/
    if(id < rng_size){
        /* Each thread gets same seed, a different sequence number, no offset */
    #ifndef MULTI_GPU
        hiprand_init(seed, id + node_offset, 0, &state[id]);
    #else

    int x[4];
    getCoords3(x, id, arg.X, 0);
    for(int i=0; i<4;i++) x[i] += arg.comm_coord[i] * arg.X[i];
    int idd = ((((x[3] * arg.comm_dim[2] * arg.X[2] + x[2]) * arg.comm_dim[1] * arg.X[1]) + x[1] ) * arg.comm_dim[0] * arg.X[0] + x[0]) >> 1 ;
    hiprand_init(seed, idd, 0, &state[id]);
    #endif
    }
}

/**
    @brief Call CUDA kernel to initialize CURAND RNG states
    @param state CURAND RNG state array
    @param seed initial seed for RNG
    @param rng_size size of the CURAND RNG state array
    @param node_offset this parameter is used to skip ahead the index in the sequence, usefull for multigpu. 
*/
void launch_kernel_random(cuRNGState *state, int seed, int rng_size, int node_offset, int X[4]){  
    dim3 nthreads(128,1,1);
    dim3 nblocks = GetBlockDim(nthreads.x, rng_size);
    //CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>( kernel_random),	hipFuncCachePreferL1));
    #ifndef MULTI_GPU
    kernel_random<<<nblocks,nthreads>>>(state, seed, rng_size, node_offset);
    #else
    rngArg arg;
    for(int i=0; i < 4; i++){
        arg.comm_dim[i] = comm_dim(i);
        arg.comm_coord[i] = comm_coord(i);
        arg.X[i] = X[i];
    }
    kernel_random<<<nblocks,nthreads>>>(state, seed, rng_size, 0, arg);
    #endif
    hipDeviceSynchronize();
}

RNG::RNG(int rng_sizes, int seedin){
    rng_size = rng_sizes;
    seed = seedin;
    state = NULL;
    node_offset = 0;
    #ifdef MULTI_GPU
    for(int i=0; i<4;i++) X[i]=0;
    node_offset = comm_rank() * rng_sizes;
    #endif
#if defined(XORWOW)
    printfQuda("Using hiprandStateXORWOW\n");
#elif defined(RG32k3a)
    printfQuda("Using hiprandStateMRG32k3a\n");
#else
    printfQuda("Using hiprandStateMRG32k3a\n");
#endif
} 
RNG::RNG(int rng_sizes, int seedin, int XX[4]){
    rng_size = rng_sizes;
    seed = seedin;
    state = NULL;
    node_offset = 0;
    #ifdef MULTI_GPU
    for(int i=0; i<4;i++) X[i]=XX[i];
    node_offset = comm_rank() * rng_sizes;
    #endif
#if defined(XORWOW)
    printfQuda("Using hiprandStateXORWOW\n");
#elif defined(RG32k3a)
    printfQuda("Using hiprandStateMRG32k3a\n");
#else
    printfQuda("Using hiprandStateMRG32k3a\n");
#endif
} 




/**
    @brief Initialize CURAND RNG states
*/
void RNG::Init(){
	AllocateRNG();
	launch_kernel_random(state, seed, rng_size, node_offset, X);
}		
					

/**
    @brief Allocate Device memory for CURAND RNG states
*/
void RNG::AllocateRNG(){
    if(rng_size>0 && state == NULL){
        //CUDA_SAFE_CALL(hipMalloc((void **)&state, rng_size * sizeof(cuRNGState)));
        state = (cuRNGState*)device_malloc(rng_size * sizeof(cuRNGState));
        CUDA_SAFE_CALL(hipMemset( state , 0 , rng_size * sizeof(cuRNGState) ));
        printfQuda("Allocated array of random numbers with rng_size: %.2f MB\n", rng_size * sizeof(cuRNGState)/(float)(1048576));
    }
    else{
        errorQuda("Array of random numbers not allocated, array size: %d !\nExiting...\n",rng_size);
    }
}
/**
    @brief Release Device memory for CURAND RNG states
*/
void RNG::Release(){
    if(rng_size>0 && state != NULL){
        //hipFree(state);
        device_free(state);
        printfQuda("Free array of random numbers with rng_size: %.2f MB\n", rng_size * sizeof(cuRNGState)/(float)(1048576));
        rng_size = 0;
        state = NULL;
    }
}

}