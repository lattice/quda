#include <quda_internal.h>
#include <gauge_field.h>
#include <gauge_tools.h>
#include <unitarization_links.h>
#include <comm_quda.h>
#include <instantiate.h>
#include <tunable_nd.h>
#include <kernels/gauge_fix_ovr2.cuh>

namespace quda
{

  template <typename Float, int nColor, QudaReconstructType recon> class GaugeFixingOVR : TunableKernel1D
  {
    GaugeField &rot;
    const GaugeField &u;
    const Float omega;
    const int dir_ignore;
    const int fixDim;
    const int parity;
    unsigned int minThreads() const { return u.LocalVolumeCB(); }

  public:
    GaugeFixingOVR(GaugeField &rot, const GaugeField &u, double omega, int dir_ignore, int parity) :
      TunableKernel1D(u),
      rot(rot),
      u(u),
      omega(static_cast<Float>(omega)),
      dir_ignore(dir_ignore),
      fixDim((dir_ignore == 4) ? 4 : 3),
      parity(parity)
    {
      strcat(aux, ",dir_ignore=");
      i32toa(aux + strlen(aux), dir_ignore);
      strcat(aux, comm_dim_partitioned_string());
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (omega == 1.0) {
        if (parity == 0) {
          GaugeFixArg<Float, nColor, recon, 0, false> arg(rot, u, omega, dir_ignore);
          launch<GaugeFix>(tp, stream, arg);
        } else if (parity == 1) {
          GaugeFixArg<Float, nColor, recon, 1, false> arg(rot, u, omega, dir_ignore);
          launch<GaugeFix>(tp, stream, arg);
        }
      } else {
        if (parity == 0) {
          GaugeFixArg<Float, nColor, recon, 0, true> arg(rot, u, omega, dir_ignore);
          launch<GaugeFix>(tp, stream, arg);
        } else if (parity == 1) {
          GaugeFixArg<Float, nColor, recon, 1, true> arg(rot, u, omega, dir_ignore);
          launch<GaugeFix>(tp, stream, arg);
        }
      }
    }

    void preTune() { rot.backup(); } // defensive measure in case they alias
    void postTune() { rot.restore(); }

    long long flops() const
    {
      auto mat_flops = u.Ncolor() * u.Ncolor() * (8ll * u.Ncolor() - 2ll);
      return (2 + (fixDim - 1) * 4) * mat_flops * fixDim * u.LocalVolume();
    }

    long long bytes() const // 2 links per dim, 2 rot in per dim, 1 rot in, 1 rot out.
    {
      return ((fixDim * 2) * u.Reconstruct() * u.Precision() + (fixDim * 2 + 1 + 1) * rot.Reconstruct() * rot.Precision())
        * u.LocalVolume();
    }

  }; // GaugeFixingOVR

  void gaugeFixingOVR2(GaugeField &rot, const GaugeField &u, double omega, int dir_ignore)
  {
    checkPrecision(rot, u);
    checkReconstruct(rot, u);
    checkNative(rot, u);

    if (dir_ignore < 0 || dir_ignore > 3) { dir_ignore = 4; }

    getProfile().TPSTART(QUDA_PROFILE_COMPUTE);
    instantiate<GaugeFixingOVR>(rot, u, omega, dir_ignore, 0);
    rot.exchangeExtendedGhost(rot.R(), false);
    instantiate<GaugeFixingOVR>(rot, u, omega, dir_ignore, 1);
    rot.exchangeExtendedGhost(rot.R(), false);
    getProfile().TPSTOP(QUDA_PROFILE_COMPUTE);
  }

} // namespace quda
