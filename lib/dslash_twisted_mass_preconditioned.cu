#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_twisted_mass_preconditioned.cuh>

/**
   This is the preconditioned gauged twisted-mass operator
*/

namespace quda
{

  // trait to ensure we don't instantiate asymmetric & xpay
  template <bool symmetric> constexpr bool xpay_() { return true; }
  template <> constexpr bool xpay_<true>() { return false; }

  // trait to ensure we don't instantiate asymmetric & !dagger
  template <bool symmetric> constexpr bool not_dagger_() { return false; }
  template <> constexpr bool not_dagger_<true>() { return true; }

  template <typename Arg> class TwistedMassPreconditioned : public Dslash<twistedMassPreconditioned, Arg>
  {
    using Dslash = Dslash<twistedMassPreconditioned, Arg>;
    using Dslash::arg;
    using Dslash::in;

  public:
    TwistedMassPreconditioned(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
    {
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);
      if (arg.asymmetric && !arg.dagger) errorQuda("asymmetric operator only defined for dagger");
      if (arg.asymmetric && arg.xpay) errorQuda("asymmetric operator not defined for xpay");
      if (arg.nParity != 1) errorQuda("Preconditioned twisted-mass operator not defined nParity=%d", arg.nParity);

      if (arg.dagger) {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, true, xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, true, false>(tp, stream);
      } else {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), false>(tp, stream);
      }
    }

    long long flops() const
    {
      long long flops = Dslash::flops();
      switch (arg.kernel_type) {
      case INTERIOR_KERNEL:
      case UBER_KERNEL:
      case KERNEL_POLICY:
        flops += 2 * in.Ncolor() * 4 * 2 * in.Volume(); // complex * Nc * Ns * fma * vol
        break;
      default: break;
      }
      return flops;
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct TwistedMassPreconditionedApply {

    inline TwistedMassPreconditionedApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
        double a, double b, bool xpay, const ColorSpinorField &x, int parity, bool dagger, bool asymmetric,
        const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      if (asymmetric) {
        TwistedMassArg<Float, nColor, nDim, recon, true> arg(out, in, U, a, b, xpay, x, parity, dagger, comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, in.VolumeCB(), in.GhostFaceCB(), profile);
      } else {
        TwistedMassArg<Float, nColor, nDim, recon, false> arg(out, in, U, a, b, xpay, x, parity, dagger, comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, in.VolumeCB(), in.GhostFaceCB(), profile);
      }
    }
  };

  /*
    Apply the preconditioned twisted-mass Dslash operator

    out = x + A^{-1} D * in = x + a*(1 + i*b*gamma_5)*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  */
#ifdef GPU_TWISTED_MASS_DIRAC
  void ApplyTwistedMassPreconditioned(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
      double b, bool xpay, const ColorSpinorField &x, int parity, bool dagger, bool asymmetric,
      const int *comm_override, TimeProfile &profile)
  {
    instantiate<TwistedMassPreconditionedApply>(
        out, in, U, a, b, xpay, x, parity, dagger, asymmetric, comm_override, profile);
  }
#else
  void ApplyTwistedMassPreconditioned(ColorSpinorField &, const ColorSpinorField &, const GaugeField &, double,
                                      double, bool, const ColorSpinorField &, int, bool, bool, const int *, TimeProfile &)
  {
    errorQuda("Twisted-mass dslash has not been built");
  }
#endif // GPU_TWISTED_MASS_DIRAC

} // namespace quda
