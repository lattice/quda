#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.hpp>
#include <kernels/dslash_twisted_mass_preconditioned.cuh>

/**
   This is the preconditioned gauged twisted-mass operator
*/

namespace quda
{

  // trait to ensure we don't instantiate asymmetric & xpay
  template <bool symmetric> constexpr bool xpay_() { return true; }
  template <> constexpr bool xpay_<true>() { return false; }

  // trait to ensure we don't instantiate asymmetric & !dagger
  template <bool symmetric> constexpr bool not_dagger_() { return false; }
  template <> constexpr bool not_dagger_<true>() { return true; }

  template <typename Arg> class TwistedMassPreconditioned : public Dslash<twistedMassPreconditioned, Arg>
  {
    using Dslash = Dslash<twistedMassPreconditioned, Arg>;
    using Dslash::arg;
    using Dslash::in;

  public:
    TwistedMassPreconditioned(Arg &arg, cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
                              const ColorSpinorField &halo) :
      Dslash(arg, out, in, halo)
    {
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);
      if (arg.asymmetric && !arg.dagger) errorQuda("asymmetric operator only defined for dagger");
      if (arg.asymmetric && arg.xpay) errorQuda("asymmetric operator not defined for xpay");
      if (arg.nParity != 1) errorQuda("Preconditioned twisted-mass operator not defined nParity=%d", arg.nParity);

      if (arg.dagger) {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, true, xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, true, false>(tp, stream);
      } else {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), false>(tp, stream);
      }
    }

    long long flops() const
    {
      long long flops = Dslash::flops();
      switch (arg.kernel_type) {
      case INTERIOR_KERNEL:
      case UBER_KERNEL:
      case KERNEL_POLICY:
        flops += in.size() * 2 * in.Ncolor() * 4 * 2 * in.Volume(); // complex * Nc * Ns * fma * vol
        break;
      default: break;
      }
      return flops;
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct TwistedMassPreconditionedApply {

    TwistedMassPreconditionedApply(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
                                   cvector_ref<const ColorSpinorField> &x, const GaugeField &U, double a, double b,
                                   bool xpay, int parity, bool dagger, bool asymmetric, const int *comm_override,
                                   TimeProfile &profile)
    {
      constexpr int nDim = 4;
      auto halo = ColorSpinorField::create_comms_batch(in);
      if (asymmetric) {
        TwistedMassArg<Float, nColor, nDim, recon, true> arg(out, in, halo, U, a, b, xpay, x, parity, dagger,
                                                             comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in, halo);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, halo, profile);
      } else {
        TwistedMassArg<Float, nColor, nDim, recon, false> arg(out, in, halo, U, a, b, xpay, x, parity, dagger,
                                                              comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in, halo);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted, in, halo, profile);
      }
    }
  };

  /*
    Apply the preconditioned twisted-mass Dslash operator

    out = x + A^{-1} D * in = x + a*(1 + i*b*gamma_5)*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  */
  void ApplyTwistedMassPreconditioned(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
                                      const GaugeField &U, double a, double b, bool xpay,
                                      cvector_ref<const ColorSpinorField> &x, int parity, bool dagger, bool asymmetric,
                                      const int *comm_override, TimeProfile &profile)
  {
    if constexpr (is_enabled<QUDA_TWISTED_MASS_DSLASH>()) {
      instantiate<TwistedMassPreconditionedApply>(out, in, x, U, a, b, xpay, parity, dagger, asymmetric, comm_override,
                                                  profile);
    } else {
      errorQuda("Twisted-mass operator has not been built");
    }
  }

} // namespace quda
