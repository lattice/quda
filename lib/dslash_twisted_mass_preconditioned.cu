#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_twisted_mass_preconditioned.cuh>

/**
   This is the preconditioned gauged twisted-mass operator
*/

namespace quda
{

  // trait to ensure we don't instantiate asymmetric & xpay
  template <bool symmetric> constexpr bool xpay_() { return true; }
  template <> constexpr bool xpay_<true>() { return false; }

  // trait to ensure we don't instantiate asymmetric & !dagger
  template <bool symmetric> constexpr bool not_dagger_() { return false; }
  template <> constexpr bool not_dagger_<true>() { return true; }

  template <typename Arg> class TwistedMassPreconditioned : public Dslash<twistedMassPreconditioned, Arg>
  {
    using Dslash = Dslash<twistedMassPreconditioned, Arg>;
    using Dslash::arg;
    using Dslash::in;

  public:
    TwistedMassPreconditioned(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) : Dslash(arg, out, in)
    {
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash::setParam(tp);
      if (arg.asymmetric && !arg.dagger) errorQuda("asymmetric operator only defined for dagger");
      if (arg.asymmetric && arg.xpay) errorQuda("asymmetric operator not defined for xpay");
      if (arg.nParity != 1) errorQuda("Preconditioned twisted-mass operator not defined nParity=%d", arg.nParity);

      if (arg.dagger) {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, true, xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, true, false>(tp, stream);
      } else {
        if (arg.xpay)
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), xpay_<Arg::asymmetric>()>(tp, stream);
        else
          Dslash::template instantiate<packShmem, 1, not_dagger_<Arg::asymmetric>(), false>(tp, stream);
      }
    }

    long long flops() const
    {
      long long flops = Dslash::flops();
      switch (arg.kernel_type) {
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
        flops += 2 * in.Ncolor() * 4 * 2 * in.Volume(); // complex * Nc * Ns * fma * vol
        break;
      default: break;
      }
      return flops;
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct TwistedMassPreconditionedApply {

    inline TwistedMassPreconditionedApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
        double a, double b, bool xpay, const ColorSpinorField &x, int parity, bool dagger, bool asymmetric,
        const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      if (asymmetric) {
        TwistedMassArg<Float, nColor, nDim, recon, true> arg(out, in, U, a, b, xpay, x, parity, dagger, comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted,
          const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)), in.VolumeCB(),
          in.GhostFaceCB(), profile);
        policy.apply(0);
      } else {
        TwistedMassArg<Float, nColor, nDim, recon, false> arg(out, in, U, a, b, xpay, x, parity, dagger, comm_override);
        TwistedMassPreconditioned<decltype(arg)> twisted(arg, out, in);

        dslash::DslashPolicyTune<decltype(twisted)> policy(twisted,
          const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)), in.VolumeCB(),
          in.GhostFaceCB(), profile);
        policy.apply(0);
      }

      checkCudaError();
    }
  };

  /*
    Apply the preconditioned twisted-mass Dslash operator

    out = x + A^{-1} D * in = x + a*(1 + i*b*gamma_5)*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  */
  void ApplyTwistedMassPreconditioned(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
      double b, bool xpay, const ColorSpinorField &x, int parity, bool dagger, bool asymmetric,
      const int *comm_override, TimeProfile &profile)
  {
#ifdef GPU_TWISTED_MASS_DIRAC
    // with symmetric dagger operator we must use kernel packing
    if (dagger && !asymmetric) pushKernelPackT(true);

    instantiate<TwistedMassPreconditionedApply>(
        out, in, U, a, b, xpay, x, parity, dagger, asymmetric, comm_override, profile);

    if (dagger && !asymmetric) popKernelPackT();
#else
    errorQuda("Twisted-mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

} // namespace quda
