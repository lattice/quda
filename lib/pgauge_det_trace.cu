#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <comm_quda.h>
#include <instantiate.h>

#include <tunable_reduction.h>
#include <kernels/gauge_det_trace.cuh>

namespace quda {

  template <typename Float, int nColor, QudaReconstructType recon>
  class CalcFunc : TunableReduction2D {
    const GaugeField &u;
    array<double, 2> &result;
    const compute_type type;

  public:
    CalcFunc(const GaugeField &u, array<double, 2> &result, compute_type type) :
      TunableReduction2D(u),
      u(u),
      result(result),
      type(type)
    {
      strcat(aux, type == compute_type::determinant ? ",det" : ",trace");
      apply(device::get_default_stream());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (type == compute_type::determinant) {
        KernelArg<Float, nColor, recon, compute_type::determinant> arg(u);
        launch<DetTrace>(result, tp, stream, arg);
      } else {
        KernelArg<Float, nColor, recon, compute_type::trace> arg(u);
        launch<DetTrace>(result, tp, stream, arg);
      }

      for (int i = 0; i < 2; i++) result[i] /= (double)(4*u.LocalVolume()*comm_size());
    }

    long long flops() const {
      if (u.Ncolor() == 3 && type == compute_type::determinant) return 264LL*u.LocalVolume();
      else if (type == compute_type::trace) return 2*u.Geometry()*u.Ncolor()*u.LocalVolume();
      else return 0;
    }

    long long bytes() const { return u.Bytes(); }
  };

  double2 getLinkDeterminant(GaugeField& data)
  {
    array<double, 2> det{0.0, 0.0};
    instantiate<CalcFunc>(data, det, compute_type::determinant);
    return make_double2(det[0], det[1]);
  }

  double2 getLinkTrace(GaugeField& data)
  {
    array<double, 2> tr{0.0, 0.0};
    instantiate<CalcFunc>(data, tr, compute_type::trace);
    return make_double2(tr[0], tr[1]);
  }

} // namespace quda
