#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <tune_quda.h>
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash_quda.h>

#include <jitify_helper.cuh>
#include <kernels/clover_sigma_outer_product.cuh>

namespace quda {

#ifdef GPU_CLOVER_DIRAC

  template <typename Float, typename Arg> class CloverSigmaOprod : public TunableVectorYZ
  {

private:
    Arg &arg;
    const GaugeField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    unsigned int minThreads() const { return arg.length; }
    bool tuneGridDim() const { return false; }

  public:
      CloverSigmaOprod(Arg &arg, const GaugeField &meta) : TunableVectorYZ(2, 6), arg(arg), meta(meta)
      {
        writeAuxString("prec=%lu,stride=%d,nvector=%d", sizeof(Float), arg.inA[0].Stride(), arg.nvector);
        // this sets the communications pattern for the packing kernel
#ifdef JITIFY
        create_jitify_program("kernels/clover_sigma_outer_product.cuh");
#endif
      }

      virtual ~CloverSigmaOprod() {}

      void apply(const qudaStream_t &stream)
      {
        if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
#ifdef JITIFY
          using namespace jitify::reflection;
          jitify_error = program->kernel("quda::sigmaOprodKernel")
                             .instantiate(arg.nvector, Type<Float>(), Type<Arg>())
                             .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                             .launch(arg);
#else
          switch (arg.nvector) {
          case 1: sigmaOprodKernel<1, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 2: sigmaOprodKernel<2, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 3: sigmaOprodKernel<3, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 4: sigmaOprodKernel<4, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 5: sigmaOprodKernel<5, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 6: sigmaOprodKernel<6, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 7: sigmaOprodKernel<7, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 8: sigmaOprodKernel<8, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          case 9: sigmaOprodKernel<9, Float><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg); break;
          }
#endif
        } else { // run the CPU code
          errorQuda("No CPU support for staggered outer-product calculation\n");
        }
      } // apply

      void preTune() { this->arg.oprod.save(); }
      void postTune() { this->arg.oprod.load(); }

      long long flops() const
      {
        return (2 * (long long)arg.length) * 6
            * ((0 + 144 + 18) * arg.nvector + 18); // spin_mu_nu + spin trace + multiply-add
      }
      long long bytes() const
      {
        return (2 * (long long)arg.length) * 6
            * ((arg.inA[0].Bytes() + arg.inB[0].Bytes()) * arg.nvector + 2 * arg.oprod.Bytes());
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), "CloverSigmaOprod", aux); }
  }; // CloverSigmaOprod

  template<typename Float, typename Output, typename InputA, typename InputB>
  void computeCloverSigmaOprod(Output oprod, const GaugeField& out, InputA *inA, InputB *inB,
			       std::vector<std::vector<double> > &coeff, int nvector) {
    // Create the arguments
    typedef CloverSigmaOprodArg<Float, Output, InputA, InputB> Arg;
    Arg arg(oprod, inA, inB, coeff, out, nvector);
    CloverSigmaOprod<Float, Arg> sigma_oprod(arg, out);
    sigma_oprod.apply(0);
  } // computeCloverSigmaOprod

#endif // GPU_CLOVER_FORCE

  void computeCloverSigmaOprod(GaugeField& oprod,
			       std::vector<ColorSpinorField*> &x,
			       std::vector<ColorSpinorField*> &p,
			       std::vector<std::vector<double> > &coeff)
  {

#ifdef GPU_CLOVER_DIRAC
    if (x.size() > MAX_NVECTOR) {
      // divide and conquer
      std::vector<ColorSpinorField*> x0(x.begin(), x.begin()+x.size()/2);
      std::vector<ColorSpinorField*> p0(p.begin(), p.begin()+p.size()/2);
      std::vector<std::vector<double> > coeff0(coeff.begin(), coeff.begin()+coeff.size()/2);
      for (unsigned int i=0; i<coeff0.size(); i++) {
	coeff0[i].reserve(2); coeff0[i][0] = coeff[i][0]; coeff0[i][1] = coeff[i][1];
      }
      computeCloverSigmaOprod(oprod, x0, p0, coeff0);

      std::vector<ColorSpinorField*> x1(x.begin()+x.size()/2, x.end());
      std::vector<ColorSpinorField*> p1(p.begin()+p.size()/2, p.end());
      std::vector<std::vector<double> > coeff1(coeff.begin()+coeff.size()/2, coeff.end());
      for (unsigned int i=0; i<coeff1.size(); i++) {
	coeff1[i].reserve(2); coeff1[i][0] = coeff[coeff.size()/2 + i][0]; coeff1[i][1] = coeff[coeff.size()/2 + i][1];
      }
      computeCloverSigmaOprod(oprod, x1, p1, coeff1);

      return;
    }

    if (oprod.Order() != QUDA_FLOAT2_GAUGE_ORDER) errorQuda("Unsupported output ordering: %d\n", oprod.Order());

    if(x[0]->Precision() != oprod.Precision())
      errorQuda("Mixed precision not supported: %d %d\n", x[0]->Precision(), oprod.Precision());

    if(oprod.Precision() == QUDA_DOUBLE_PRECISION){

      Spinor<double2, double2, 12, 0> spinorA[MAX_NVECTOR];
      Spinor<double2, double2, 12, 0> spinorB[MAX_NVECTOR];

      for (unsigned int i=0; i<x.size(); i++) {
	spinorA[i].set(*dynamic_cast<cudaColorSpinorField*>(x[i]));
	spinorB[i].set(*dynamic_cast<cudaColorSpinorField*>(p[i]));
      }

      computeCloverSigmaOprod<double>(gauge::FloatNOrder<double, 18, 2, 18>(oprod),
				      oprod, spinorA, spinorB, coeff, x.size());

    } else {
      errorQuda("Unsupported precision: %d\n", oprod.Precision());
    }
#else // GPU_CLOVER_DIRAC not defined
    errorQuda("Clover Dirac operator has not been built!");
#endif

    checkCudaError();
    return;
  } // computeCloverForce

} // namespace quda
