#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>

namespace quda {

  using namespace gauge;

  template <typename Order, int nDim, int dim>
  struct ExtractGhostExArg {
    Order order;
    int X[nDim];
    int R[nDim];
    int surfaceCB[nDim];
    int A0[nDim];
    int A1[nDim];
    int B0[nDim];
    int B1[nDim];
    int C0[nDim];
    int C1[nDim];
    int fBody[nDim][nDim];
    int fBuf[nDim][nDim];
    int localParity[nDim];
    int threads;
    ExtractGhostExArg(const Order &order, const int *X_, const int *R_, 
		      const int *surfaceCB_, 
		      const int *A0_, const int *A1_, const int *B0_, const int *B1_, 
		      const int *C0_, const int *C1_, const int fBody_[nDim][nDim], 
		      const int fBuf_[nDim][nDim], const int *localParity_) 
      : order(order), threads(0) {

      threads = R_[dim]*(A1_[dim]-A0_[dim])*(B1_[dim]-B0_[dim])*(C1_[dim]-C0_[dim])*order.geometry;

      for (int d=0; d<nDim; d++) {
	X[d] = X_[d];
	R[d] = R_[d];
	surfaceCB[d] = surfaceCB_[d];
	A0[d] = A0_[d];
	A1[d] = A1_[d];
	B0[d] = B0_[d];
	B1[d] = B1_[d];
	C0[d] = C0_[d];
	C1[d] = C1_[d];
	for (int e=0; e<nDim; e++) {
	  fBody[d][e] = fBody_[d][e];
	  fBuf[d][e] = fBuf_[d][e];
	}
	localParity[d] = localParity_[d]; 
      }
    }

  };

  template <typename Float, int length, int dim, typename Arg>
  __device__ __host__ void extractor(Arg &arg, int dir, int a, int b, 
				     int c, int d, int g, int parity) {
    int srcIdx = (a*arg.fBody[dim][0] + b*arg.fBody[dim][1] + 
		  c*arg.fBody[dim][2] + d*arg.fBody[dim][3]) >> 1;
    
    int dstIdx = (a*arg.fBuf[dim][0] + b*arg.fBuf[dim][1] + 
		  c*arg.fBuf[dim][2] + (d-(dir?arg.X[dim]:arg.R[dim]))*arg.fBuf[dim][3]) >> 1;
    
    Matrix<complex<typename mapper<Float>::type>, Ncolor(length)> u;

    // load the ghost element from the bulk
    u = arg.order(g, srcIdx, parity); 

    // need dir dependence in write
    // srcIdx is used here to determine boundary condition
    arg.order.saveGhostEx(u.data, dstIdx, srcIdx, dir, dim, g, (parity+arg.localParity[dim])&1, arg.R);
  }


  template <typename Float, int length, int dim, typename Arg>
  __device__ __host__ void injector(Arg &arg, int dir, int a, int b, 
				    int c, int d, int g, int parity) {
    int srcIdx = (a*arg.fBuf[dim][0] + b*arg.fBuf[dim][1] + 
		  c*arg.fBuf[dim][2] + (d-dir*(arg.X[dim]+arg.R[dim]))*arg.fBuf[dim][3]) >> 1;
    
    int dstIdx = (a*arg.fBody[dim][0] + b*arg.fBody[dim][1] + 
		  c*arg.fBody[dim][2] + d*arg.fBody[dim][3]) >> 1;

    int oddness = (parity+arg.localParity[dim])&1;
    
    Matrix<complex<typename mapper<Float>::type>, Ncolor(length)> u;

    // need dir dependence in read
    // dstIdx is used here to determine boundary condition
    arg.order.loadGhostEx(u.data, srcIdx, dstIdx, dir, dim, g, oddness, arg.R);
    
    arg.order(g, dstIdx, parity) = u; // save the ghost element into the bulk
  }
  
  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, int nDim, int dim, typename Order, bool extract>
  void extractGhostEx(ExtractGhostExArg<Order,nDim,dim> arg)
  {
    for (int parity=0; parity<2; parity++) {

      // the following 4-way loop means this is specialized for 4 dimensions 
      // dir = 0 backwards, dir = 1 forwards
      for (int dir = 0; dir<2; dir++) {

	int D0 = extract ? dir*arg.X[dim] + (1-dir)*arg.R[dim] : dir*(arg.X[dim] + arg.R[dim]); 
	  
	for (int d=D0; d<D0+arg.R[dim]; d++) {
	  for (int a=arg.A0[dim]; a<arg.A1[dim]; a++) { // loop over the interior surface
	    for (int b=arg.B0[dim]; b<arg.B1[dim]; b++) { // loop over the interior surface
	      for (int c=arg.C0[dim]; c<arg.C1[dim]; c++) { // loop over the interior surface
		for (int g=0; g<arg.order.geometry; g++) {

		  // we only do the extraction for parity we are currently working on
		  int oddness = (a+b+c+d) & 1;
		  if (oddness == parity) {
		    if (extract) extractor<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
		    else injector<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
		  } // oddness == parity
		} // g
	      } // c
	    } // b
	  } // a
	} // d
      } // dir
      
    } // parity

  }

  /**
     Generic GPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
     FIXME this implementation will have two-way warp divergence
  */
  
  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
  */
  template <typename Float, int length, int nDim, int dim, typename Order, bool extract>
  __global__ void extractGhostExKernel(ExtractGhostExArg<Order,nDim,dim> arg)
  {
    // parallelize over parity and dir using block or grid 
    /*for (int parity=0; parity<2; parity++) {*/
    {
      int parity = blockIdx.z;

      // the following 4-way loop means this is specialized for 4 dimensions 
      // dir = 0 backwards, dir = 1 forwards
      //for (int dir = 0; dir<2; dir++) {
      {
	int dir = blockIdx.y;

	// this will have two-warp divergence since we only do work on
	// one parity but parity alternates between threads
	// linear index used for writing into ghost buffer
	int X = blockIdx.x * blockDim.x + threadIdx.x; 	
	if (X >= arg.threads) return;

	int dA = arg.A1[dim]-arg.A0[dim];
	int dB = arg.B1[dim]-arg.B0[dim];
	int dC = arg.C1[dim]-arg.C0[dim];
	int D0 = extract ? dir*arg.X[dim] + (1-dir)*arg.R[dim] : dir*(arg.X[dim] + arg.R[dim]); 

	// thread order is optimized to maximize coalescing
	// X = (((g*R + d) * dA + a)*dB + b)*dC + c
	int gdab = X / dC;
	int c    = arg.C0[dim] + X    - gdab*dC;
	int gda  = gdab / dB;
	int b    = arg.B0[dim] + gdab - gda *dB;
	int gd   = gda / dA;
	int a    = arg.A0[dim] + gda  - gd  *dA;
	int g    = gd / arg.R[dim];
	int d    = D0          + gd   - g   *arg.R[dim];

	// we only do the extraction for parity we are currently working on
	int oddness = (a+b+c+d) & 1;
	if (oddness == parity) {
	  if (extract) extractor<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
	  else injector<Float,length,dim>(arg, dir, a, b, c, d, g, parity);
	} // oddness == parity
      } // dir
      
    } // parity

  }

  template <typename Float, int length, int nDim, int dim, typename Order>
  class ExtractGhostEx : Tunable {
    ExtractGhostExArg<Order,nDim,dim> arg;
    int size;
    bool extract;
    const GaugeField &meta;
    QudaFieldLocation location;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return size; }

  public:
    ExtractGhostEx(ExtractGhostExArg<Order,nDim,dim> &arg, bool extract, 
		   const GaugeField &meta, QudaFieldLocation location)
      : arg(arg), extract(extract), meta(meta), location(location) {
      int dA = arg.A1[dim]-arg.A0[dim];
      int dB = arg.B1[dim]-arg.B0[dim];
      int dC = arg.C1[dim]-arg.C0[dim];
      size = arg.R[dim]*dA*dB*dC*arg.order.geometry;
      writeAuxString("prec=%lu,stride=%d,extract=%d,dimension=%d,geometry=%d",
		     sizeof(Float),arg.order.stride, extract, dim, arg.order.geometry);
    }
  
    void apply(const qudaStream_t &stream) {
      if (extract) {
	if (location==QUDA_CPU_FIELD_LOCATION) {
	  extractGhostEx<Float,length,nDim,dim,Order,true>(arg);
	} else {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  tp.grid.y = 2;
	  tp.grid.z = 2;
	  extractGhostExKernel<Float,length,nDim,dim,Order,true> 
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	}
      } else { // we are injecting
	if (location==QUDA_CPU_FIELD_LOCATION) {
	  extractGhostEx<Float,length,nDim,dim,Order,false>(arg);
	} else {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  tp.grid.y = 2;
	  tp.grid.z = 2;
	  extractGhostExKernel<Float,length,nDim,dim,Order,false> 
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; } 
    long long bytes() const { return 2 * 2 * 2 * size * arg.order.Bytes(); } // 2 for i/o    
  };


  /**
     Generic CPU gauge ghost extraction and packing
     NB This routines is specialized to four dimensions
     @param E the extended gauge dimensions
     @param R array holding the radius of the extended region 
     @param extract Whether we are extracting or injecting the ghost zone
  */
  template <typename Float, int length, typename Order>
  void extractGhostEx(Order order, const int dim, const int *surfaceCB, const int *E, 
		      const int *R, bool extract, const GaugeField &u, QudaFieldLocation location) {
    const int nDim = 4;
    //loop variables: a, b, c with a the most signifcant and c the least significant
    //A0, B0, C0 the minimum value
    //A0, B0, C0 the maximum value

    int X[nDim]; // compute interior dimensions
    for (int d=0; d<nDim; d++) X[d] = E[d] - 2*R[d];

    //..........x..........y............z.............t
    int A0[nDim] = {R[3],      R[3],        R[3],         0};
    int A1[nDim] = {X[3]+R[3], X[3]+R[3],   X[3]+R[3],    X[2]+2*R[2]};
    
    int B0[nDim] = {R[2],      R[2],        0,            0};
    int B1[nDim] = {X[2]+R[2], X[2]+R[2],   X[1]+2*R[1],  X[1]+2*R[1]};
    
    int C0[nDim] = {R[1],      0,           0,            0};
    int C1[nDim] = {X[1]+R[1], X[0]+2*R[0], X[0]+2*R[0],  X[0]+2*R[0]};

    int fSrc[nDim][nDim] = {
      {E[2]*E[1]*E[0], E[1]*E[0], E[0],              1},
      {E[2]*E[1]*E[0], E[1]*E[0],    1,           E[0]},
      {E[2]*E[1]*E[0],      E[0],    1,      E[1]*E[0]},
      {E[1]*E[0],           E[0],    1, E[2]*E[1]*E[0]}
    };  
  
    int fBuf[nDim][nDim]={
      {E[2]*E[1], E[1], 1, E[3]*E[2]*E[1]},
      {E[2]*E[0], E[0], 1, E[3]*E[2]*E[0]}, 
      {E[1]*E[0], E[0], 1, E[3]*E[1]*E[0]},
      {E[1]*E[0], E[0], 1, E[2]*E[1]*E[0]}
    };

    //set the local processor parity 
    //switching odd and even ghost gauge when that dimension size is odd
    //only switch if X[dir] is odd and the gridsize in that dimension is greater than 1
    // FIXME - I don't understand this, shouldn't it be commDim(dim) == 0 ?
    int localParity[nDim];
    for (int d=0; d<nDim; d++) 
      localParity[dim] = ((X[dim] % 2 ==1) && (commDim(dim) > 1)) ? 1 : 0;
    //      localParity[dim] = (X[dim]%2==0 || commDim(dim)) ? 0 : 1;

    if (dim==0) {
      ExtractGhostExArg<Order,nDim,0> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,0,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==1) {
      ExtractGhostExArg<Order,nDim,1> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,1,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==2) {
      ExtractGhostExArg<Order,nDim,2> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,2,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else if (dim==3) {
      ExtractGhostExArg<Order,nDim,3> arg(order, X, R, surfaceCB, A0, A1, B0, B1, 
					  C0, C1, fSrc, fBuf, localParity);
      ExtractGhostEx<Float,length,nDim,3,Order> extractor(arg, extract, u, location);
      extractor.apply(0);
    } else {
      errorQuda("Invalid dim=%d", dim);
    }

    checkCudaError();
  }

  /** This is the template driver for extractGhost */
  template <typename Float>
  void extractGhostEx(const GaugeField &u, int dim, const int *R, Float **Ghost, bool extract) {

    const int length = 18;

    QudaFieldLocation location = 
      (typeid(u)==typeid(cudaGaugeField)) ? QUDA_CUDA_FIELD_LOCATION : QUDA_CPU_FIELD_LOCATION;

    if (u.isNative()) {
      if (u.Reconstruct() == QUDA_RECONSTRUCT_NO) {
        typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_NO>::type G;
        extractGhostEx<Float, length>(G(u, 0, Ghost), dim, u.SurfaceCB(), u.X(), R, extract, u, location);
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_12) {
#if QUDA_RECONSTRUCT & 2
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
        errorQuda("QUDA_RECONSTRUCT = %d does not enable QUDA_RECONSTRUCT_12", QUDA_RECONSTRUCT);
#endif
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_8) {
#if QUDA_RECONSTRUCT & 1
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost), 
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
        errorQuda("QUDA_RECONSTRUCT = %d does not enable QUDA_RECONSTRUCT_8", QUDA_RECONSTRUCT);
#endif
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_13) {
#if QUDA_RECONSTRUCT & 2
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_13>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
        errorQuda("QUDA_RECONSTRUCT = %d does not enable QUDA_RECONSTRUCT_13", QUDA_RECONSTRUCT);
#endif
      } else if (u.Reconstruct() == QUDA_RECONSTRUCT_9) {
#if QUDA_RECONSTRUCT & 1
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_9>::type G;
	extractGhostEx<Float,length>(G(u, 0, Ghost),
				     dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
        errorQuda("QUDA_RECONSTRUCT = %d does not enable QUDA_RECONSTRUCT_9", QUDA_RECONSTRUCT);
#endif
      }
    } else if (u.Order() == QUDA_QDP_GAUGE_ORDER) {
      
#ifdef BUILD_QDP_INTERFACE
      extractGhostEx<Float,length>(QDPOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("QDP interface has not been built\n");
#endif
      
    } else if (u.Order() == QUDA_QDPJIT_GAUGE_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      extractGhostEx<Float,length>(QDPJITOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_CPS_WILSON_GAUGE_ORDER) {

#ifdef BUILD_CPS_INTERFACE
      extractGhostEx<Float,length>(CPSOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("CPS interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_MILC_GAUGE_ORDER) {

#ifdef BUILD_MILC_INTERFACE
      extractGhostEx<Float,length>(MILCOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("MILC interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_BQCD_GAUGE_ORDER) {

#ifdef BUILD_BQCD_INTERFACE
      extractGhostEx<Float,length>(BQCDOrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("BQCD interface has not been built\n");
#endif

    } else if (u.Order() == QUDA_TIFR_GAUGE_ORDER) {

#ifdef BUILD_TIFR_INTERFACE
      extractGhostEx<Float,length>(TIFROrder<Float,length>(u, 0, Ghost),
				   dim, u.SurfaceCB(), u.X(), R, extract, u, location);
#else
      errorQuda("TIFR interface has not been built\n");
#endif

    } else {
      errorQuda("Gauge field %d order not supported", u.Order());
    }

  }

  void extractExtendedGaugeGhost(const GaugeField &u, int dim, const int *R, 
				 void **ghost, bool extract) {

    if (u.Precision() == QUDA_DOUBLE_PRECISION) {
      extractGhostEx(u, dim, R, (double**)ghost, extract);
    } else if (u.Precision() == QUDA_SINGLE_PRECISION) {
#if QUDA_PRECISION & 4
      extractGhostEx(u, dim, R, (float**)ghost, extract);
#else
      errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
    } else if (u.Precision() == QUDA_HALF_PRECISION) {
#if QUDA_PRECISION & 2
      extractGhostEx(u, dim, R, (short **)ghost, extract);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else if (u.Precision() == QUDA_QUARTER_PRECISION) {
#if QUDA_PRECISION & 1
      extractGhostEx(u, dim, R, (char **)ghost, extract);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else {
      errorQuda("Unknown precision type %d", u.Precision());
    }
  }

} // namespace quda
