#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>

#define  DOUBLE_TOL	1e-15
#define  SINGLE_TOL	2e-6

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename GaugeOr, typename GaugeDs>
  struct GaugeSTOUTArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4]; 
#endif
    GaugeOr origin;
    const Float rho;
    const Float tolerance;
    
    GaugeDs dest;

    GaugeSTOUTArg(GaugeOr &origin, GaugeDs &dest, const GaugeField &data, const Float rho, const Float tolerance) 
      : origin(origin), dest(dest), rho(rho), tolerance(tolerance) {
#ifdef MULTI_GPU
        for(int dir=0; dir<4; ++dir){
          border[dir] = 2;
        }
        for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir] - border[dir]*2;
#else
        for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];
#endif
	threads = X[0]*X[1]*X[2]*X[3];
    }
  };


  __device__ __host__ inline int linkIndex2(int x[], int dx[], const int X[4]) {
    int y[4];
    for (int i=0; i<4; i++) y[i] = (x[i] + dx[i] + X[i]) % X[i];
    int idx = (((y[3]*X[2] + y[2])*X[1] + y[1])*X[0] + y[0]) >> 1;
    return idx;
  }


  __device__ __host__ inline void getCoords2(int x[4], int cb_index, const int X[4], int parity) 
  {
    x[3] = cb_index/(X[2]*X[1]*X[0]/2);
    x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
    x[1] = (cb_index/(X[0]/2)) % X[1];
    x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);

    return;
  }

  template <typename Float2, typename Float>
  __host__ __device__ int checkUnitary(Matrix<Float2,3> in, Matrix<Float2,3> *inv, const Float tol)
  {
    computeMatrixInverse(in, inv);

    for (int i=0;i<3;i++)
      for (int j=0;j<3;j++)
      {
        if (fabs(in(i,j).x - (*inv)(j,i).x) > tol)
          return 1;
        if (fabs(in(i,j).y + (*inv)(j,i).y) > tol)
          return 1;
      }
    return 0;
  }

  template <typename Float2>
  __host__ __device__ int checkUnitaryPrint(Matrix<Float2,3> in, Matrix<Float2,3> *inv)
  {
    computeMatrixInverse(in, inv);
    for (int i=0;i<3;i++)
      for (int j=0;j<3;j++)
      {
        printf("TESTR: %+.3le %+.3le %+.3le\n", in(i,j).x, (*inv)(j,i).x, fabs(in(i,j).x - (*inv)(j,i).x));
	printf("TESTI: %+.3le %+.3le %+.3le\n", in(i,j).y, (*inv)(j,i).y, fabs(in(i,j).y + (*inv)(j,i).y));
        hipDeviceSynchronize();
        if (fabs(in(i,j).x - (*inv)(j,i).x) > 1e-14)
          return 1;
        if (fabs(in(i,j).y + (*inv)(j,i).y) > 1e-14)
          return 1;
      }
    return 0;  
  }

  template <typename Float2,typename Float>
  __host__ __device__ void polarSu3(Matrix<Float2,3> *in, Float tol)
  {
    typedef typename ComplexTypeId<Float>::Type Cmplx;
    Matrix<Cmplx,3> inv, out;

    out = *in;
    computeMatrixInverse(out, &inv);

    do
    {
      out = out + conj(inv);
      out = out*0.5;
    } while(checkUnitary(out, &inv, tol));
/*
    printf("Convergence after %d iterations\n", N);
    hipDeviceSynchronize();
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", out(0,0).x, out(0,0).y, out(0,1).x, out(0,1).y, out(0,2).x, out(0,2).y);
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", out(1,0).x, out(1,0).y, out(1,1).x, out(1,1).y, out(1,2).x, out(1,2).y);
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", out(2,0).x, out(2,0).y, out(2,1).x, out(2,1).y, out(2,2).x, out(2,2).y);
    printf("\n\n");
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", inv(0,0).x, inv(0,0).y, inv(0,1).x, inv(0,1).y, inv(0,2).x, inv(0,2).y);
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", inv(1,0).x, inv(1,0).y, inv(1,1).x, inv(1,1).y, inv(1,2).x, inv(1,2).y);
    printf("%+.3lf %+.3lfi    %+.3lf %+.3lfi    %+.3lf %+.3lfi\n", inv(2,0).x, inv(2,0).y, inv(2,1).x, inv(2,1).y, inv(2,2).x, inv(2,2).y);
    printf("\n\n\n\n");
    hipDeviceSynchronize();
*/
    Cmplx  det = getDeterminant(out);
    double mod = det.x*det.x + det.y*det.y;
    mod = pow(mod, (1./6.));
    double angle = atan2(det.y, det.x);
    angle /= -3.;
    
    Cmplx cTemp;

    cTemp.x = cos(angle)/mod;
    cTemp.y = sin(angle)/mod;

//    out = out*cTemp;
    *in = out*cTemp;
/*    if (checkUnitary(out, &inv))
    {
    	cTemp = getDeterminant(out);
	printf ("DetX: %+.3lf  %+.3lfi, %.3lf %.3lf\nDetN: %+.3lf  %+.3lfi", det.x, det.y, mod, angle, cTemp.x, cTemp.y);
        hipDeviceSynchronize();
	checkUnitaryPrint(out, &inv);
	setIdentity(in);
        *in = *in * 0.5;
    }
    else
    {
      cTemp = getDeterminant(out);
//      printf("Det: %+.3lf %+.3lf\n", cTemp.x, cTemp.y);
      hipDeviceSynchronize();

      if (fabs(cTemp.x - 1.0) > 1e-8)
	setIdentity(in);
      else if (fabs(cTemp.y) > 1e-8)
      {
	setIdentity(in);
        printf("DadadaUnitary failed\n");
        *in = *in * 0.1;
      }
      else
        *in = out;
    }*/
  }


  template <typename Float, typename GaugeOr, typename GaugeDs, typename Float2>
  __host__ __device__ void computeStaple(GaugeSTOUTArg<Float,GaugeOr,GaugeDs>& arg, int idx, int parity, int dir, Matrix<Float2,3> &staple) {

    typedef typename ComplexTypeId<Float>::Type Cmplx;
      // compute spacetime dimensions and parity

    int X[4]; 
    for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

    int x[4];
    getCoords2(x, idx, X, parity);
#ifdef MULTI_GPU
    for(int dr=0; dr<4; ++dr) {
         x[dr] += arg.border[dr];
         X[dr] += 2*arg.border[dr];
    }
#endif

    setZero(&staple);

    for (int mu=0; mu<4; mu++) {
      if (mu == dir) {
        continue;
      }

      int nu = dir;

      {
        int dx[4] = {0, 0, 0, 0};
        Matrix<Cmplx,3> U1;
        arg.origin.load((Float*)(U1.data),linkIndex2(x,dx,X), mu, parity); 

        Matrix<Cmplx,3> U2;
        dx[mu]++;
        arg.origin.load((Float*)(U2.data),linkIndex2(x,dx,X), nu, 1-parity); 

        Matrix<Cmplx,3> U3;
        dx[mu]--;
        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndex2(x,dx,X), mu, 1-parity); 
   
        Matrix<Cmplx,3> tmpS;

        tmpS	= U1 * U2;
	tmpS	= tmpS * conj(U3);

	staple = staple + tmpS;

        dx[mu]--;
        dx[nu]--;
        arg.origin.load((Float*)(U1.data),linkIndex2(x,dx,X), mu, 1-parity); 
        arg.origin.load((Float*)(U2.data),linkIndex2(x,dx,X), nu, 1-parity); 

        dx[nu]++;
        arg.origin.load((Float*)(U3.data),linkIndex2(x,dx,X), mu, parity); 

        tmpS	= conj(U1);
	tmpS	= tmpS * U2;
	tmpS	= tmpS * U3;

	staple = staple + tmpS;
      }
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    __global__ void computeSTOUTStep(GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg){
      int idx = threadIdx.x + blockIdx.x*blockDim.x;
      if(idx >= arg.threads) return;
      typedef typename ComplexTypeId<Float>::Type Cmplx;

      int parity = 0;
      if(idx >= arg.threads/2) {
        parity = 1;
        idx -= arg.threads/2;
      }

      int X[4]; 
      for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

      int x[4];
      getCoords2(x, idx, X, parity);
#ifdef MULTI_GPU
      for(int dr=0; dr<4; ++dr) {
           x[dr] += arg.border[dr];
           X[dr] += 2*arg.border[dr];
      }
#endif
      
      int dx[4] = {0, 0, 0, 0};
      for (int dir=0; dir < 3; dir++) {	 //Only spatial dimensions are smeared
        Matrix<Cmplx,3> U, UDag, Stap, Omega, OmegaDag, OmegaDiff, ODT, Q,
	  exp_iQ, tmp1, tmp2;
	Cmplx OmegaDiffTr;
	Cmplx i_2 = makeComplex<Cmplx>(0,0.5);

	//This function gets stap = S_{mu,nu} i.e., the staple of length 3,
        computeStaple<Float,GaugeOr,GaugeDs,Cmplx>(arg,idx,parity,dir,Stap);
	//
	// |- > -|
	// ^     v
	// |     |
	//          +  |     |
	//             v     ^
	//             |- > -|

	// Get link U
        arg.origin.load((Float*)(U.data),linkIndex2(x,dx,X), dir, parity);

	//Compute Omega_{mu}=[Sum_{mu neq nu}rho_{mu,nu}C_{mu,nu}]*U_{mu}^dag

	//Get U^{\dagger}
	computeMatrixInverse(U,&UDag);
	
	//Compute \Omega = \rho * S * U^{\dagger}
	tmp1 = arg.rho * Stap;
	Omega = tmp1 * UDag;

	//Compute \Q_{mu} = i/2[Omega_{mu}^dag - Omega_{mu} 
	//                      - 1/3 Tr(Omega_{mu}^dag - Omega_{mu})]

	OmegaDag = conj(Omega);
	OmegaDiff = OmegaDag - Omega;

	Q = OmegaDiff;
	OmegaDiffTr = getTrace(OmegaDiff);
	OmegaDiffTr =  1.0/3.0 * OmegaDiffTr;

	//Matrix proportional to OmegaDiffTr
	setIdentity(&ODT);
	tmp1 = OmegaDiffTr * ODT;

	Q = Q - tmp1;
	Q = i_2 * Q;
	//Q is now defined.

	//exponentiate_iMat(Q,&exp_iQ);

	//No need to project back down to SU(3)
        //polarSu3<Cmplx,Float>(&U, arg.tolerance);
        arg.dest.save((Float*)(U.data),linkIndex2(x,dx,X), dir, parity); 
    }
  }

  template<typename Float, typename GaugeOr, typename GaugeDs>
    class GaugeSTOUT : Tunable {
      GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg;
      const QudaFieldLocation location;

      private:
      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool tuneSharedBytes() const { return false; } // Don't tune shared memory
      bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
      unsigned int minThreads() const { return arg.threads; }

      public:
      GaugeSTOUT(GaugeSTOUTArg<Float,GaugeOr, GaugeDs> &arg, QudaFieldLocation location)
        : arg(arg), location(location) {}
      virtual ~GaugeSTOUT () {}

      void apply(const hipStream_t &stream){
        if(location == QUDA_CUDA_FIELD_LOCATION){
#if (__COMPUTE_CAPABILITY__ >= 200)
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
          computeSTOUTStep<<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
#else
	  errorQuda("GaugeSTOUT not supported on pre-Fermi architecture");
#endif
        }else{
          errorQuda("CPU not supported yet\n");
          //computeSTOUTStepCPU(arg);
        }
      }

      TuneKey tuneKey() const {
        std::stringstream vol, aux;
        vol << arg.X[0] << "x";
        vol << arg.X[1] << "x";
        vol << arg.X[2] << "x";
        vol << arg.X[3];
        aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
        return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
      }


      std::string paramString(const TuneParam &param) const {
        std::stringstream ps;
        ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
        ps << "shared=" << param.shared_bytes;
        return ps.str();
      }

      void preTune(){}
      void postTune(){}
      long long flops() const { return (1)*6*arg.threads; }
      long long bytes() const { return (1)*6*arg.threads*sizeof(Float); } // Only correct if there is no link reconstruction

    }; // GaugeSTOUT

  template<typename Float,typename GaugeOr, typename GaugeDs>
    void STOUTStep(GaugeOr origin, GaugeDs dest, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {
      if (dataOr.Precision() == QUDA_DOUBLE_PRECISION) {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, DOUBLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      } else {
        GaugeSTOUTArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, rho, SINGLE_TOL);
        GaugeSTOUT<Float,GaugeOr,GaugeDs> gaugeSTOUT(arg, location);
        gaugeSTOUT.apply(0);
      }
      hipDeviceSynchronize();
    }

  template<typename Float>
    void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, Float rho, QudaFieldLocation location) {

      // Switching to FloatNOrder for the gauge field in order to support RECONSTRUCT_12
      // Need to fix this!!

      if(dataDs.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
        if(dataOr.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
          if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else {
            errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
          }
        } else if(dataOr.Order() == QUDA_FLOAT4_GAUGE_ORDER) {
          if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 2, 18>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 2, 12>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 2,  8>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else {
            errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
          }
        } else {
	  errorQuda("Invalid Gauge Order origin field\n");
        }
      } else if(dataDs.Order() == QUDA_FLOAT4_GAUGE_ORDER) {
        if(dataOr.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
          if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 2, 18>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 2, 12>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 2,  8>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else {
            errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
          }
        } else if(dataOr.Order() == QUDA_FLOAT4_GAUGE_ORDER) {
          if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 4, 18>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 4, 12>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
            if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
              STOUTStep(FloatNOrder<Float, 18, 4, 18>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
              STOUTStep(FloatNOrder<Float, 18, 4, 12>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
              STOUTStep(FloatNOrder<Float, 18, 4,  8>(dataOr), FloatNOrder<Float, 18, 4,  8>(dataDs), dataOr, rho, location);
            }else{
              errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
          } else {
            errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
          }
        } else {
	  errorQuda("Invalid Gauge Order origin field\n");
        }
      } else {
        errorQuda("Invalid Gauge Order destination field\n");
      }
  }
#endif

  void STOUTStep(GaugeField &dataDs, const GaugeField& dataOr, double rho, QudaFieldLocation location) {

#ifdef GPU_GAUGE_TOOLS

    if(dataOr.Precision() != dataDs.Precision()) {
      errorQuda("Oriign and destination fields must have the same precision\n");
    }

    if(dataDs.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported\n");
    }

    if (dataDs.Precision() == QUDA_SINGLE_PRECISION){
      STOUTStep<float>(dataDs, dataOr, (float) rho, location);
    } else if(dataDs.Precision() == QUDA_DOUBLE_PRECISION) {
      STOUTStep<double>(dataDs, dataOr, rho, location);
    } else {
      errorQuda("Precision %d not supported", dataDs.Precision());
    }
    return;
#else
  errorQuda("Gauge tools are not build");
#endif
  }


}
