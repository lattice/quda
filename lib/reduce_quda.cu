#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>

//#define QUAD_SUM
#ifdef QUAD_SUM
#include <dbldbl.h>
#endif

#include <cub_helper.cuh>

template<typename> struct ScalarType { };
template<> struct ScalarType<double> { typedef double type; };
template<> struct ScalarType<double2> { typedef double type; };
template<> struct ScalarType<double3> { typedef double type; };
template<> struct ScalarType<double4> { typedef double type; };

template<typename> struct Vec2Type { };
template<> struct Vec2Type<double> { typedef double2 type; };

#ifdef QUAD_SUM
#define QudaSumFloat doubledouble
#define QudaSumFloat2 doubledouble2
#define QudaSumFloat3 doubledouble3
template<> struct ScalarType<doubledouble> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble2> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble3> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble4> { typedef doubledouble type; };
template<> struct Vec2Type<doubledouble> { typedef doubledouble2 type; };
#else
#define QudaSumFloat double
#define QudaSumFloat2 double2
#define QudaSumFloat3 double3
#define QudaSumFloat4 double4
#endif


#define REDUCE_MAX_BLOCKS 65536

void checkSpinor(const ColorSpinorField &a, const ColorSpinorField &b) {
  if (a.Precision() != b.Precision())
    errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision());
  if (a.Length() != b.Length())
    errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length());
  if (a.Stride() != b.Stride())
    errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());
}

void checkLength(const ColorSpinorField &a, ColorSpinorField &b) {									\
  if (a.Length() != b.Length())
    errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length());
  if (a.Stride() != b.Stride())
    errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());
}

static struct {
  const char *vol_str;
  const char *aux_str;
  char aux_tmp[quda::TuneKey::aux_n];
} blasStrings;

// These are used for reduction kernels
static QudaSumFloat *d_reduce=0;
static QudaSumFloat *h_reduce=0;
static QudaSumFloat *hd_reduce=0;
static hipEvent_t reduceEnd;

namespace quda {
  namespace blas {

    hipStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }
    hipEvent_t* getReduceEvent() { return &reduceEnd; }

    void initReduce()
    {

      const int MaxReduce = 16;
      // reduction buffer size
      size_t bytes = 2*MaxReduce*3*REDUCE_MAX_BLOCKS*sizeof(QudaSumFloat); // Factor of N for composite reductions

      if (!d_reduce) d_reduce = (QudaSumFloat *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be QudaSumFloat3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (QudaSumFloat *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (QudaSumFloat *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    namespace reduce {

#include <texture.h>
#include <reduce_core.cuh>
#include <reduce_core.h>
#include <reduce_mixed_core.h>

    } // namespace reduce

    /**
       Base class from which all reduction functors should derive.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct ReduceFunctor {

      //! pre-computation routine called before the "M-loop"
      virtual __device__ __host__ void pre() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y,
							   FloatN &z, FloatN &w, FloatN &v) = 0;

      //! post-computation routine called after the "M-loop"
      virtual __device__ __host__ void post(ReduceType &sum) { ; }

    };

    /**
       Return the L1 norm of x
    */
    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const double2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float4 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y) + (ReduceType)fabs(a.z) + (ReduceType)fabs(a.w);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Norm1 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm1(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { sum += norm1_<ReduceType>(x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm1(const ColorSpinorField &x) {
#ifdef HOST_DEBUG
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x); // FIXME
      return reduce::reduceCuda<double,QudaSumFloat,Norm1,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
#else
	errorQuda("L1 norm kernel only built when HOST_DEBUG is enabled");
      return 0.0;
#endif
    }

    /**
       Return the L2 norm of x
    */
    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const double2 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const float2 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ void norm2_(ReduceType &sum, const float4 &a) {
      sum += (ReduceType)a.x*(ReduceType)a.x;
      sum += (ReduceType)a.y*(ReduceType)a.y;
      sum += (ReduceType)a.z*(ReduceType)a.z;
      sum += (ReduceType)a.w*(ReduceType)a.w;
    }


    template <typename ReduceType, typename Float2, typename FloatN>
      struct Norm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm2(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { norm2_<ReduceType>(sum,x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm2(const ColorSpinorField &x) {
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x);
      return reduce::reduceCuda<double,QudaSumFloat,Norm2,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }


    /**
       Return the real dot product of x and y
    */
    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const double2 &a, const double2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float2 &a, const float2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float4 &a, const float4 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
      sum += (ReduceType)a.z*(ReduceType)b.z;
      sum += (ReduceType)a.w*(ReduceType)b.w;
    }

   template <typename ReduceType, typename Float2, typename FloatN>
    struct Dot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Dot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
     { dot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,Dot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
     * Returns the real component of the dot product of a and b and
     * the norm of a
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ ReduceType dotNormA_(const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      ReduceType c;
      dot_<scalar>(c.x,a,b);
      norm2_<scalar>(c.y,a);
      return c;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct DotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
      DotNormA(const Float2 &a, const Float2 &b){}
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,  FloatN &w, FloatN &v)
      {sum += dotNormA_<ReduceType,FloatN>(x,y);}
      static int streams() { return 2; }
      static int flops() { return 4; }
    };

    double2 reDotProductNormA(ColorSpinorField &x,ColorSpinorField &y){
      return reduce::reduceCuda<double2,QudaSumFloat2,DotNormA,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] += a*x[i]
       Return the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      axpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y += a.x*x; norm2_<ReduceType>(sum,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpyNorm(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,axpyNorm2,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] += a*x[i]
       Return real dot product (x,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct AxpyReDot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      AxpyReDot(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y += a.x*x; dot_<ReduceType>(sum,x,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpyReDot(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,AxpyReDot,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] = x[i] - y[i]
       Second returns the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xmyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      xmyNorm2(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y = x - y; norm2_<ReduceType>(sum,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 3; } //! flops per element
    };

    double xmyNorm(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,xmyNorm2,0,1,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */
    __device__ __host__ void Caxpy_(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    /**
       First performs the operation y[i] = a*x[i] + y[i] (complex-valued)
       Second returns the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	Caxpy_(a, x, y); norm2_<ReduceType>(sum,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,caxpyNorm2,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       double caxpyXmayNormCuda(float a, float *x, float *y, n){}
       First performs the operation y[i] = a*x[i] + y[i]
       Second performs the operator x[i] -= a*z[i]
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyxmaznormx : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyxmaznormx(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); Caxpy_(-a,z,x); norm2_<ReduceType>(sum,x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x,
			  ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,caxpyxmaznormx,1,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
    }


    /**
       double cabxpyAxNorm(float a, complex b, float *x, float *y, n){}
       First performs the operation y[i] += a*b*x[i]
       Second performs x[i] *= a
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct cabxpyaxnorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      cabxpyaxnorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { x *= a.x; Caxpy_(b, x, y); norm2_<ReduceType>(sum,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };


    double cabxpyAxNorm(const double &a, const Complex &b,
			ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,cabxpyaxnorm,1,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(REAL(b), IMAG(b)), x, y, x, x, x);
    }


    /**
       Returns complex-valued dot product of x and y
    */
    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const double2 &a, const double2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float2 &a, const float2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float4 &a, const float4 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.x += (scalar)a.z*(scalar)b.z;
      sum.x += (scalar)a.w*(scalar)b.w;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
      sum.y += (scalar)a.z*(scalar)b.w;
      sum.y -= (scalar)a.w*(scalar)b.z;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Cdot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Cdot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { cdot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };


    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,Cdot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    /**
       double2 xpaycDotzyCuda(float2 *x, float a, float2 *y, float2 *z, int n) {}
       First performs the operation y = x + a*y
       Second returns cdot product (z,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xpaycdotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      xpaycdotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { y = x + a.x*y; cdot_<ReduceType>(sum,z,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    Complex xpaycDotzy(ColorSpinorField &x, const double &a, ColorSpinorField &y, ColorSpinorField &z) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,xpaycdotzy,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       double caxpyDotzyCuda(float a, float *x, float *y, float *z, n){}
       First performs the operation y[i] = a*x[i] + y[i]
       Second returns the dot product (z,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpydotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpydotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); cdot_<ReduceType>(sum,z,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };


    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,caxpydotzy,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       First returns the dot product (x,y)
       Returns the norm of x
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ void cdotNormA_(ReduceType &sum, const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      cdot_<ReduceType>(sum,a,b);
      norm2_<scalar>(sum.z,a);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct CdotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
      CdotNormA(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { cdotNormA_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double3,QudaSumFloat3,CdotNormA,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First returns the dot product (x,y)
       Returns the norm of y
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ void cdotNormB_(ReduceType &sum, const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      cdot_<ReduceType>(sum,a,b);
      norm2_<scalar>(sum.z,b);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct CdotNormB : public ReduceFunctor<ReduceType, Float2, FloatN> {
      CdotNormB(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { cdotNormB_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 cDotProductNormB(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double3,QudaSumFloat3,CdotNormB,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       This convoluted kernel does the following:
       z += a*x + b*y, y -= b*w, norm = (y,y), dot = (u, y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpbypzYmbwcDotProductUYNormY_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      caxpbypzYmbwcDotProductUYNormY_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { Caxpy_(a, x, z); Caxpy_(b, y, z); Caxpy_(-b, w, y); cdotNormB_<ReduceType>(sum,v,y); }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 18; } //! flops per element
    };

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x,
					   const Complex &b, ColorSpinorField &y,
					   ColorSpinorField &z, ColorSpinorField &w,
					   ColorSpinorField &u) {
      if (x.Precision() != z.Precision()) {
	return reduce::mixed::reduceCuda<double3,QudaSumFloat3,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      } else {
	return reduce::reduceCuda<double3,QudaSumFloat3,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      }
    }


    /**
       Specialized kernel for the modified CG norm computation for
       computing beta.  Computes y = y + a*x and returns norm(y) and
       dot(y, delta(y)) where delta(y) is the difference between the
       input and out y vector.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpyCGNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      axpyCGNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	FloatN z_new = z + a.x*x;
	norm2_<scalar>(sum.x,z_new);
	dot_<scalar>(sum.y,z_new,z_new-z);
	z = z_new;
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per real element
    };

    Complex axpyCGNorm(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      // swizzle since mixed is on z
      double2 cg_norm ;
      if (x.Precision() != y.Precision()) {
	cg_norm = reduce::mixed::reduceCuda<double2,QudaSumFloat2,axpyCGNorm2,0,0,1,0,0,false>
	  (make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      } else {
	cg_norm = reduce::reduceCuda<double2,QudaSumFloat2,axpyCGNorm2,0,0,1,0,0,false>
	  (make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      }
      return Complex(cg_norm.x, cg_norm.y);
    }


    /**
       This kernel returns (x, x) and (r,r) and also returns the so-called
       heavy quark norm as used by MILC: 1 / N * \sum_i (r, r)_i / (x, x)_i, where
       i is site index and N is the number of sites.
       When this kernel is launched, we must enforce that the parameter M
       in the launcher corresponds to the number of FloatN fields used to
       represent the spinor, e.g., M=6 for Wilson and M=3 for staggered.
       This is only the case for half-precision kernels by default.  To
       enable this, the siteUnroll template parameter must be set true
       when reduceCuda is instantiated.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct HeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      typedef typename scalar<ReduceType>::type real;
      Float2 a;
      Float2 b;
      ReduceType aux;
      HeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	norm2_<real>(aux.x,x); norm2_<real>(aux.y,y);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : static_cast<real>(1.0);
      }

      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! undercounts since it excludes the per-site division
    };

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r) {
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,HeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }


    /**
      Variant of the HeavyQuarkResidualNorm kernel: this takes three
      arguments, the first two are summed together to form the
      solution, with the third being the residual vector.  This removes
      the need an additional xpy call in the solvers, impriving
      performance.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xpyHeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
	typedef typename scalar<ReduceType>::type real;
      Float2 a;
      Float2 b;
      ReduceType aux;
      xpyHeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	norm2_<real>(aux.x,x + y); norm2_<real>(aux.y,z);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : static_cast<real>(1.0);
      }

      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 5; }
    };

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y,
				      ColorSpinorField &r) {
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,xpyHeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    /**
       double3 tripleCGReduction(V x, V y, V z){}
       First performs the operation norm2(x)
       Second performs the operatio norm2(y)
       Third performs the operation dotPropduct(y,z)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct tripleCGReduction_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      tripleCGReduction_(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	norm2_<scalar>(sum.x,x); norm2_<scalar>(sum.y,y); dot_<scalar>(sum.z,y,z);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double3,QudaSumFloat3,tripleCGReduction_,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }


#ifdef ALTRELIABLE
    /**
       double4 quadrupleCGReduction(V x, V y, V z){}
       First performs the operation norm2(x)
       Second performs the operatio norm2(y)
       Third performs the operation dotPropduct(y,z)
       Fourth performs the operation norm(z)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct quadrupleCGReduction_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      quadrupleCGReduction_(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
  typedef typename ScalarType<ReduceType>::type scalar;
  norm2_<scalar>(sum.x,x); norm2_<scalar>(sum.y,y); dot_<scalar>(sum.z,y,z); norm2_<scalar>(sum.w,w);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double4,QudaSumFloat4,quadrupleCGReduction_,0,0,0,0,0,false>
  (make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

#endif

   } // namespace blas

} // namespace quda
