#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>

//#define QUAD_SUM
#ifdef QUAD_SUM
#include <dbldbl.h>
#endif

#include <cub_helper.cuh>

template<typename> struct ScalarType { };
template<> struct ScalarType<double> { typedef double type; };
template<> struct ScalarType<double2> { typedef double type; };
template<> struct ScalarType<double3> { typedef double type; };

template<typename> struct Vec2Type { };
template<> struct Vec2Type<double> { typedef double2 type; };

#ifdef QUAD_SUM
#define QudaSumFloat doubledouble
#define QudaSumFloat2 doubledouble2
#define QudaSumFloat3 doubledouble3
template<> struct ScalarType<doubledouble> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble2> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble3> { typedef doubledouble type; };
template<> struct Vec2Type<doubledouble> { typedef doubledouble2 type; };
#else
#define QudaSumFloat double
#define QudaSumFloat2 double2
#define QudaSumFloat3 double3
#endif


#define REDUCE_MAX_BLOCKS 65536

#define checkSpinor(a, b)						\
  {									\
    if (a.Precision() != b.Precision())					\
      errorQuda("precisions do not match: %d %d", a.Precision(), b.Precision()); \
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

#define checkLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length()); \
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

static struct {
  const char *vol_str;
  const char *aux_str;
  char aux_tmp[quda::TuneKey::aux_n];
} blasStrings;

// These are used for reduction kernels
static QudaSumFloat *d_reduce=0;
static QudaSumFloat *h_reduce=0;
static QudaSumFloat *hd_reduce=0;
static hipEvent_t reduceEnd;

namespace quda {
  namespace blas {

    hipStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }

    void initReduce()
    {

      const int MaxReduce = 12;
      // reduction buffer size
      size_t bytes = MaxReduce*3*REDUCE_MAX_BLOCKS*sizeof(QudaSumFloat); // Factor of N for composite reductions

      if (!d_reduce) d_reduce = (QudaSumFloat *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be QudaSumFloat3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (QudaSumFloat *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (QudaSumFloat *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    namespace reduce {

#include <texture.h>
#include <reduce_core.h>
#include <reduce_mixed_core.h>

    } // namespace reduce

    /**
       Base class from which all reduction functors should derive.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct ReduceFunctor {

      //! pre-computation routine called before the "M-loop"
      virtual __device__ __host__ void pre() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y,
							   FloatN &z, FloatN &w, FloatN &v) = 0;

      //! post-computation routine called after the "M-loop"
      virtual __device__ __host__ void post(ReduceType &sum) { ; }

    };

    /**
       Return the L1 norm of x
    */
    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const double2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float2 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y);
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm1_(const float4 &a) {
      return (ReduceType)fabs(a.x) + (ReduceType)fabs(a.y) + (ReduceType)fabs(a.z) + (ReduceType)fabs(a.w);
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Norm1 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm1(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { sum += norm1_<ReduceType>(x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm1(const ColorSpinorField &x) {
#ifdef HOST_DEBUG
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x); // FIXME
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Norm1,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
#else
	errorQuda("L1 norm kernel only built when HOST_DEBUG is enabled");
      return 0.0;
#endif
    }

    /**
       Return the L2 norm of x
    */
    template<typename ReduceType> __device__ __host__ ReduceType norm2_(const double2 &a) {
      return (ReduceType)a.x*(ReduceType)a.x + (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm2_(const float2 &a) {
      return (ReduceType)a.x*(ReduceType)a.x + (ReduceType)a.y*(ReduceType)a.y;
    }

    template<typename ReduceType> __device__ __host__ ReduceType norm2_(const float4 &a) {
      return (ReduceType)a.x*(ReduceType)a.x + (ReduceType)a.y*(ReduceType)a.y +
	(ReduceType)a.z*(ReduceType)a.z + (ReduceType)a.w*(ReduceType)a.w;
    }


    template <typename ReduceType, typename Float2, typename FloatN>
      struct Norm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Norm2(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,FloatN  &w, FloatN &v)
      { sum += norm2_<ReduceType>(x); }
      static int streams() { return 1; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double norm2(const ColorSpinorField &x) {
      ColorSpinorField &y = const_cast<ColorSpinorField&>(x);
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Norm2,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }


    /**
       Return the real dot product of x and y
    */
    template<typename ReduceType> __device__ __host__ ReduceType dot_(const double2 &a, const double2 &b) {
      return (ReduceType)a.x*(ReduceType)b.x + (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ ReduceType dot_(const float2 &a, const float2 &b) {
      return (ReduceType)a.x*(ReduceType)b.x + (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ ReduceType dot_(const float4 &a, const float4 &b) {
      return (ReduceType)a.x*(ReduceType)b.x + (ReduceType)a.y*(ReduceType)b.y +
	(ReduceType)a.z*(ReduceType)b.z + (ReduceType)a.w*(ReduceType)b.w;
    }

   template <typename ReduceType, typename Float2, typename FloatN>
    struct Dot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Dot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
     { sum += dot_<ReduceType>(x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }

    void reDotProduct(double* result, std::vector<cudaColorSpinorField*>& x, std::vector<cudaColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else
    switch(x.size()){
      case 1:
        reduce::multiReduceCuda<1,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 2:
        reduce::multiReduceCuda<2,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 3:
        reduce::multiReduceCuda<3,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 4:
        reduce::multiReduceCuda<4,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 5:
        reduce::multiReduceCuda<5,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 6:
        reduce::multiReduceCuda<6,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 7:
        reduce::multiReduceCuda<7,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 8:
        reduce::multiReduceCuda<8,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 9:
        reduce::multiReduceCuda<9,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 10:
        reduce::multiReduceCuda<10,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 11:
        reduce::multiReduceCuda<11,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 12:
        reduce::multiReduceCuda<12,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 13:
        reduce::multiReduceCuda<13,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 14:
        reduce::multiReduceCuda<14,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 15:
        reduce::multiReduceCuda<15,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 16:
        reduce::multiReduceCuda<16,double,QudaSumFloat,QudaSumFloat,Dot,0,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      default:
        errorQuda("Unsupported vector size");
        break;
    }
#endif // SSTEP
  }


    /**
     * Returns the real component of the dot product of a and b and
     * the norm of a
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ ReduceType dotNormA_(const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      ReduceType c;
      c.x = dot_<scalar>(a,b);
      c.y = norm2_<scalar>(a);
      return c;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct DotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
      DotNormA(const Float2 &a, const Float2 &b){}
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z,  FloatN &w, FloatN &v)
      {sum += dotNormA_<ReduceType,FloatN>(x,y);}
      static int streams() { return 2; }
      static int flops() { return 4; }
    };

    double2 reDotProductNormA(ColorSpinorField &x,ColorSpinorField &y){
      return reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,DotNormA,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] += a*x[i]
       Return the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      axpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y += a.x*x; sum += norm2_<ReduceType>(y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpyNorm(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,axpyNorm2,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] += a*x[i]
       Return real dot product (x,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct AxpyReDot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      AxpyReDot(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	y += a.x*x; sum += dot_<ReduceType>(x,y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    double axpyReDot(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,AxpyReDot,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First performs the operation y[i] = x[i] - y[i]
       Second returns the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xmyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      xmyNorm2(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
      y = x - y; sum += norm2_<ReduceType>(y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 3; } //! flops per element
    };

    double xmyNorm(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,xmyNorm2,0,1,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       Functor to perform the operation y += a * x  (complex-valued)
    */
    __device__ __host__ void Caxpy_(const double2 &a, const double2 &x, double2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float2 &x, float2 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
    }
    __device__ __host__ void Caxpy_(const float2 &a, const float4 &x, float4 &y) {
      y.x += a.x*x.x; y.x -= a.y*x.y;
      y.y += a.y*x.x; y.y += a.x*x.y;
      y.z += a.x*x.z; y.z -= a.y*x.w;
      y.w += a.y*x.z; y.w += a.x*x.w;
    }

    /**
       First performs the operation y[i] = a*x[i] + y[i] (complex-valued)
       Second returns the norm of y
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	Caxpy_(a, x, y); sum += norm2_<ReduceType>(y); }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,caxpyNorm2,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       double caxpyXmayNormCuda(float a, float *x, float *y, n){}

       First performs the operation y[i] = a*x[i] + y[i]
       Second performs the operator x[i] -= a*z[i]
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpyxmaznormx : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpyxmaznormx(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); Caxpy_(-a,z,x); sum += norm2_<ReduceType>(x); }
      static int streams() { return 5; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x,
			  ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,caxpyxmaznormx,1,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
    }


    /**
       double cabxpyAxNorm(float a, complex b, float *x, float *y, n){}

       First performs the operation y[i] += a*b*x[i]
       Second performs x[i] *= a
       Third returns the norm of x
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct cabxpyaxnorm : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      cabxpyaxnorm(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { x *= a.x; Caxpy_(b, x, y); sum += norm2_<ReduceType>(y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 10; } //! flops per element
    };


    double cabxpyAxNorm(const double &a, const Complex &b,
			ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double,QudaSumFloat,QudaSumFloat,cabxpyaxnorm,1,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(REAL(b), IMAG(b)), x, y, x, x, x);
    }


    /**
       Returns complex-valued dot product of x and y
    */
    template<typename ReduceType>
    __device__ __host__ ReduceType cdot_(const double2 &a, const double2 &b) {
      ReduceType c;
      typedef typename ScalarType<ReduceType>::type scalar;
      c.x = (scalar)a.x*(scalar)b.x + (scalar)a.y*(scalar)b.y;
      c.y = (scalar)a.x*(scalar)b.y - (scalar)a.y*(scalar)b.x;
      return c;
    }

    template<typename ReduceType>
    __device__ __host__ ReduceType cdot_(const float2 &a, const float2 &b) {
      ReduceType c;
      typedef typename ScalarType<ReduceType>::type scalar;
      c.x = (scalar)a.x*(scalar)b.x + (scalar)a.y*(scalar)b.y;
      c.y = (scalar)a.x*(scalar)b.y - (scalar)a.y*(scalar)b.x;
      return c;
    }

    template<typename ReduceType>
    __device__ __host__ ReduceType cdot_(const float4 &a, const float4 &b) {
      ReduceType c;
      typedef typename ScalarType<ReduceType>::type scalar;
      c.x = (scalar)a.x*(scalar)b.x + (scalar)a.y*(scalar)b.y +
	(scalar)a.z*(scalar)b.z + (scalar)a.w*(scalar)b.w;
      c.y = (scalar)a.x*(scalar)b.y - (scalar)a.y*(scalar)b.x +
	(scalar)a.z*(scalar)b.w - (scalar)a.w*(scalar)b.z;
      return c;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct Cdot : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Cdot(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { sum += cdot_<ReduceType>(x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };


    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    void cDotProduct(Complex* result, std::vector<cudaColorSpinorField*>& x, std::vector<cudaColorSpinorField*>& y){
      double2* cdot = new double2[x.size()];

      switch(x.size()){
      case 1:
        reduce::multiReduceCuda<1,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
	  (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 2:
        reduce::multiReduceCuda<2,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
	  (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 3:
        reduce::multiReduceCuda<3,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
	  (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      case 4:
        reduce::multiReduceCuda<4,double2,QudaSumFloat2,QudaSumFloat,Cdot,0,0,0,0,0,false>
	  (cdot, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
        break;
      default:
        errorQuda("Unsupported vector size\n");
        break;
      }

      for (unsigned int i=0; i<x.size(); ++i) result[i] = Complex(cdot[i].x,cdot[i].y);
      delete[] cdot;
    }

    /**
       double2 xpaycDotzyCuda(float2 *x, float a, float2 *y, float2 *z, int n) {}

       First performs the operation y = x + a*y
       Second returns cdot product (z,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xpaycdotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      xpaycdotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { y = x + a.x*y; sum += cdot_<ReduceType>(z,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    Complex xpaycDotzy(ColorSpinorField &x, const double &a, ColorSpinorField &y, ColorSpinorField &z) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,xpaycdotzy,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       double caxpyDotzyCuda(float a, float *x, float *y, float *z, n){}

       First performs the operation y[i] = a*x[i] + y[i]
       Second returns the dot product (z,y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpydotzy : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      caxpydotzy(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { Caxpy_(a, x, y); sum += cdot_<ReduceType>(z,y); }
      static int streams() { return 4; } //! total number of input and output streams
      static int flops() { return 8; } //! flops per element
    };


    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      double2 cdot = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,caxpydotzy,0,1,0,0,0,false>
	(make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }


    /**
       First returns the dot product (x,y)
       Returns the norm of x
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ ReduceType cdotNormA_(const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      vec2 cdot = cdot_<vec2>(a,b);
      ReduceType c;
      c.x = cdot.x; c.y = cdot.y; c.z = norm2_<scalar>(a);
      return c;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct CdotNormA : public ReduceFunctor<ReduceType, Float2, FloatN> {
      CdotNormA(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { sum += cdotNormA_<ReduceType>(x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,CdotNormA,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       First returns the dot product (x,y)
       Returns the norm of y
    */
    template<typename ReduceType, typename InputType>
    __device__ __host__ ReduceType cdotNormB_(const InputType &a, const InputType &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      typedef typename Vec2Type<scalar>::type vec2;
      vec2 cdot = cdot_<vec2>(a,b);
      ReduceType c;
      c.x = cdot.x; c.y = cdot.y; c.z = norm2_<scalar>(b);
      return c;
    }

    template <typename ReduceType, typename Float2, typename FloatN>
    struct CdotNormB : public ReduceFunctor<ReduceType, Float2, FloatN> {
      CdotNormB(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v)
      { sum += cdotNormB_<ReduceType>(x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 cDotProductNormB(ColorSpinorField &x, ColorSpinorField &y) {
      return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,CdotNormB,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }


    /**
       This convoluted kernel does the following:
       z += a*x + b*y, y -= b*w, norm = (y,y), dot = (u, y)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct caxpbypzYmbwcDotProductUYNormY_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      caxpbypzYmbwcDotProductUYNormY_(const Float2 &a, const Float2 &b) : a(a), b(b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) { Caxpy_(a, x, z); Caxpy_(b, y, z); Caxpy_(-b, w, y); sum += cdotNormB_<ReduceType>(v,y); }
      static int streams() { return 7; } //! total number of input and output streams
      static int flops() { return 18; } //! flops per element
    };

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x,
					   const Complex &b, ColorSpinorField &y,
					   ColorSpinorField &z, ColorSpinorField &w,
					   ColorSpinorField &u) {
      if (x.Precision() != z.Precision()) {
	return reduce::mixed::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      } else {
	return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,caxpbypzYmbwcDotProductUYNormY_,0,1,1,0,0,false>
	  (make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      }
    }


    /**
       Specialized kernel for the modified CG norm computation for
       computing beta.  Computes y = y + a*x and returns norm(y) and
       dot(y, delta(y)) where delta(y) is the difference between the
       input and out y vector.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct axpyCGNorm2 : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      axpyCGNorm2(const Float2 &a, const Float2 &b) : a(a) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	FloatN y_new = y + a.x*x;
	sum.x += norm2_<scalar>(y_new);
	sum.y += dot_<scalar>(y_new, y_new-y);
	y = y_new;
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per real element
    };

    Complex axpyCGNorm(const double &a, ColorSpinorField &x, ColorSpinorField &y) {
      double2 cg_norm = reduce::reduceCuda<double2,QudaSumFloat2,QudaSumFloat,axpyCGNorm2,0,1,0,0,0,false>
	(make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
      return Complex(cg_norm.x, cg_norm.y);
    }


    /**
       This kernel returns (x, x) and (r,r) and also returns the so-called
       heavy quark norm as used by MILC: 1 / N * \sum_i (r, r)_i / (x, x)_i, where
       i is site index and N is the number of sites.

       When this kernel is launched, we must enforce that the parameter M
       in the launcher corresponds to the number of FloatN fields used to
       represent the spinor, e.g., M=6 for Wilson and M=3 for staggered.
       This is only the case for half-precision kernels by default.  To
       enable this, the siteUnroll template parameter must be set true
       when reduceCuda is instantiated.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct HeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      ReduceType aux;
      HeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	aux.x += norm2_<scalar>(x); aux.y += norm2_<scalar>(y);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : 1.0;
      }

      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! undercounts since it excludes the per-site division
    };

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r) {
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,HeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }


    /**
      Variant of the HeavyQuarkResidualNorm kernel: this takes three
      arguments, the first two are summed together to form the
      solution, with the third being the residual vector.  This removes
      the need an additional xpy call in the solvers, impriving
      performance.
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct xpyHeavyQuarkResidualNorm_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      Float2 a;
      Float2 b;
      ReduceType aux;
      xpyHeavyQuarkResidualNorm_(const Float2 &a, const Float2 &b) : a(a), b(b), aux{ } { ; }

      __device__ __host__ void pre() { aux.x = 0; aux.y = 0; }

      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	aux.x += norm2_<scalar>(x + y); aux.y += norm2_<scalar>(z);
      }

      //! sum the solution and residual norms, and compute the heavy-quark norm
      __device__ __host__ void post(ReduceType &sum)
      {
	sum.x += aux.x; sum.y += aux.y; sum.z += (aux.x > 0.0) ? (aux.y / aux.x) : 1.0;
      }

      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 5; }
    };

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y,
				      ColorSpinorField &r) {
      double3 rtn = reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,xpyHeavyQuarkResidualNorm_,0,0,0,0,0,true>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    /**
       double3 tripleCGUpdate(V x, V y, V z){}

       First performs the operation norm2(x)
       Second performs the operatio norm2(y)
       Third performs the operation dotPropduct(y,z)
    */
    template <typename ReduceType, typename Float2, typename FloatN>
    struct tripleCGReduction_ : public ReduceFunctor<ReduceType, Float2, FloatN> {
      tripleCGReduction_(const Float2 &a, const Float2 &b) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, FloatN &v) {
	typedef typename ScalarType<ReduceType>::type scalar;
	sum.x += norm2_<scalar>(x); sum.y += norm2_<scalar>(y); sum.z += dot_<scalar>(y,z);
      }
      static int streams() { return 3; } //! total number of input and output streams
      static int flops() { return 6; } //! flops per element
    };

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return reduce::reduceCuda<double3,QudaSumFloat3,QudaSumFloat,tripleCGReduction_,0,0,0,0,0,false>
	(make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

   } // namespace blas

} // namespace quda
