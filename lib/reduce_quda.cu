#include "hip/hip_runtime.h"
#include <atomic>
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/reduce_core.cuh>

// These are used for reduction kernels
static QudaSumFloat *d_reduce=0;
static QudaSumFloat *h_reduce=0;
static QudaSumFloat *hd_reduce=0;
static hipEvent_t reduceEnd;
static bool fast_reduce_enabled = false;

namespace quda {

  namespace blas {

#include <generic_reduce.cuh>

    qudaStream_t* getStream();

    void* getDeviceReduceBuffer() { return d_reduce; }
    void* getMappedHostReduceBuffer() { return hd_reduce; }
    void* getHostReduceBuffer() { return h_reduce; }
    hipEvent_t* getReduceEvent() { return &reduceEnd; }
    bool getFastReduce() { return fast_reduce_enabled; }

    void initFastReduce(int32_t words)
    {
      // initialize the reduction values in 32-bit increments to INT_MIN
      for (int32_t i = 0; i < words; i++) {
        reinterpret_cast<int32_t *>(h_reduce)[i] = std::numeric_limits<int32_t>::min();
      }

      // ensure that the host memory write is complete before we launch the kernel
      atomic_thread_fence(std::memory_order_release);
    }

    void completeFastReduce(int32_t words)
    {
      volatile int32_t *check = reinterpret_cast<int32_t *>(h_reduce);
      int count = 0;
      int complete = 0;
      while (complete < words) {
        // ensure visiblity to any changes in memory
        atomic_thread_fence(std::memory_order_acquire);

        complete = 0;
        for (int32_t i = 0; i < words; i++) {
          // spin-wait until all values have been updated
          if (check[i] != std::numeric_limits<int32_t>::min()) complete++;
        }
        if (count++ % 10000 == 0) { // check error every 10000 iterations
          // if there is an error in the kernel then we need to exit the spin-wait
          if (hipSuccess != hipPeekAtLastError()) break;
        }
      }
    }

    void initReduce()
    {
      /* we have these different reductions to cater for:

         - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), with possibly
           parity dimension, and a grid-stride loop with max number of
           blocks = 2 x SM count

         - multi-reductions where we are reducing to a matrix of size
           of size QUDA_MAX_MULTI_REDUCE of vectors (max length 4), with
           possible parity dimension, and a grid-stride loop with
           maximum number of blocks = 2 x SM count
      */

      const int reduce_size = 4 * sizeof(QudaSumFloat);
      const int max_reduce_blocks = 2*deviceProp.multiProcessorCount;

      const int max_reduce = 2 * max_reduce_blocks * reduce_size;
      const int max_multi_reduce = 2 * QUDA_MAX_MULTI_REDUCE * max_reduce_blocks * reduce_size;

      // reduction buffer size
      size_t bytes = max_reduce > max_multi_reduce ? max_reduce : max_multi_reduce;

      if (!d_reduce) d_reduce = (QudaSumFloat *) device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be QudaSumFloat3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
	// only use zero copy reductions when using 64-bit
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
	if(deviceProp.canMapHostMemory) {
	  h_reduce = (QudaSumFloat *) mapped_malloc(bytes);
	  hipHostGetDevicePointer(&hd_reduce, h_reduce, 0); // set the matching device pointer
	} else
#endif
	  {
	    h_reduce = (QudaSumFloat *) pinned_malloc(bytes);
	    hd_reduce = d_reduce;
	  }
	memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      // enable fast reductions with CPU spin waiting as opposed to using CUDA events
      char *fast_reduce_env = getenv("QUDA_ENABLE_FAST_REDUCE");
      if (fast_reduce_env && strcmp(fast_reduce_env,"1") == 0) {
        warningQuda("Experimental fast reductions enabled");
        fast_reduce_enabled = true;
      }

      checkCudaError();
    }

    void endReduce(void)
    {
      if (d_reduce) {
	device_free(d_reduce);
	d_reduce = 0;
      }
      if (h_reduce) {
	host_free(h_reduce);
	h_reduce = 0;
      }
      hd_reduce = 0;

      hipEventDestroy(reduceEnd);
    }

    /**
       Generic reduction kernel launcher
    */
    template <typename doubleN, typename ReduceType, typename FloatN, int M, typename Arg>
    doubleN reduceLaunch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      const int32_t words = tp.grid.y * sizeof(ReduceType) / sizeof(int32_t);
      if (getFastReduce() && !commAsyncReduction()) initFastReduce(words);

#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::reduceKernel")
                                  .instantiate((int)tp.block.x, Type<ReduceType>(), Type<FloatN>(), M, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
      LAUNCH_KERNEL(reduceKernel, tunable, tp, stream, arg, ReduceType, FloatN, M);
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            completeFastReduce(words);
          } else {
            qudaEventRecord(reduceEnd, stream);
            while (hipSuccess != qudaEventQuery(reduceEnd)) { ; }
          }
        } else
#endif
        {
          qudaMemcpy(h_reduce, hd_reduce, sizeof(ReduceType), hipMemcpyDeviceToHost);
        }
      }
      doubleN cpu_sum = set(((ReduceType *)h_reduce)[0]);
      if (tp.grid.y == 2) sum(cpu_sum, ((ReduceType *)h_reduce)[1]); // add other parity if needed
      return cpu_sum;
    }

    template <typename doubleN, typename ReduceType, typename FloatN, int M, typename SpinorX, typename SpinorY,
        typename SpinorZ, typename SpinorW, typename SpinorV, typename Reducer>
    class ReduceCuda : public Tunable
    {

  private:
      const int nParity; // for composite fields this includes the number of composites
      mutable ReductionArg<ReduceType, SpinorX, SpinorY, SpinorZ, SpinorW, SpinorV, Reducer> arg;
      doubleN &result;

      const ColorSpinorField &x, &y, &z, &w, &v;

      // host pointers used for backing up fields when tuning
      // these can't be curried into the Spinors because of Tesla argument length restriction
      char *X_h, *Y_h, *Z_h, *W_h, *V_h;
      char *Xnorm_h, *Ynorm_h, *Znorm_h, *Wnorm_h, *Vnorm_h;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

  public:
      ReduceCuda(doubleN &result, SpinorX &X, SpinorY &Y, SpinorZ &Z, SpinorW &W, SpinorV &V, Reducer &r,
          ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v,
          int length) :
          nParity((x.IsComposite() ? x.CompositeDim() : 1) * (x.SiteSubset())),
          arg(X, Y, Z, W, V, r, length / nParity),
          x(x),
          y(y),
          z(z),
          w(w),
          v(v),
          result(result),
          X_h(0),
          Y_h(0),
          Z_h(0),
          W_h(0),
          V_h(0),
          Xnorm_h(0),
          Ynorm_h(0),
          Znorm_h(0),
          Wnorm_h(0),
          Vnorm_h(0)
      {
        strcpy(aux, x.AuxString());
        if (x.Precision() != z.Precision()) {
          strcat(aux, ",");
          strcat(aux, z.AuxString());
        }
        if (getFastReduce()) strcat(aux, ",fast_reduce");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/reduce_core.cuh");
#endif
      }
      virtual ~ReduceCuda() {}

      inline TuneKey tuneKey() const { return TuneKey(x.VolString(), typeid(arg.r).name(), aux); }

      void apply(const qudaStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        result = reduceLaunch<doubleN, ReduceType, FloatN, M>(arg, tp, stream, *this);
      }

      void preTune()
      {
        arg.X.backup(&X_h, &Xnorm_h, x.Bytes(), x.NormBytes());
        arg.Y.backup(&Y_h, &Ynorm_h, y.Bytes(), y.NormBytes());
        arg.Z.backup(&Z_h, &Znorm_h, z.Bytes(), z.NormBytes());
        arg.W.backup(&W_h, &Wnorm_h, w.Bytes(), w.NormBytes());
        arg.V.backup(&V_h, &Vnorm_h, v.Bytes(), v.NormBytes());
      }

      void postTune()
      {
        arg.X.restore(&X_h, &Xnorm_h, x.Bytes(), x.NormBytes());
        arg.Y.restore(&Y_h, &Ynorm_h, y.Bytes(), y.NormBytes());
        arg.Z.restore(&Z_h, &Znorm_h, z.Bytes(), z.NormBytes());
        arg.W.restore(&W_h, &Wnorm_h, w.Bytes(), w.NormBytes());
        arg.V.restore(&V_h, &Vnorm_h, v.Bytes(), v.NormBytes());
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.grid.y = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.grid.y = nParity;
      }

      long long flops() const { return arg.r.flops() * vec_length<FloatN>::value * arg.length * nParity * M; }

      long long bytes() const
      {
        // the factor two here assumes we are reading and writing to the high precision vector
        // this will evaluate correctly for non-mixed kernels since the +2/-2 will cancel out
        return (arg.r.streams() - 2) * x.Bytes() + 2 * z.Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <typename doubleN, typename ReduceType, typename RegType, typename StoreType, typename zType, int M,
        template <typename ReducerType, typename Float, typename FloatN> class Reducer, int writeX, int writeY,
        int writeZ, int writeW, int writeV>
    doubleN nativeReduce(const double2 &a, const double2 &b, ColorSpinorField &x, ColorSpinorField &y,
        ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v, int length)
    {

      checkLength(x, y);
      checkLength(x, z);
      checkLength(x, w);
      checkLength(x, v);

      Spinor<RegType, StoreType, M, writeX> X(x);
      Spinor<RegType, StoreType, M, writeY> Y(y);
      Spinor<RegType, zType, M, writeZ> Z(z);
      Spinor<RegType, StoreType, M, writeW> W(w);
      Spinor<RegType, StoreType, M, writeV> V(v);

      doubleN value;
      typedef typename scalar<RegType>::type Float;
      typedef typename vector<Float, 2>::type Float2;
      typedef vector<Float, 2> vec2;

      Reducer<ReduceType, Float2, RegType> r((Float2)vec2(a), (Float2)vec2(b));
      ReduceCuda<doubleN, ReduceType, RegType, M, decltype(X), decltype(Y), decltype(Z), decltype(W), decltype(V),
          Reducer<ReduceType, Float2, RegType>>
          reduce(value, X, Y, Z, W, V, r, x, y, z, w, v, length);
      reduce.apply(*(blas::getStream()));

      blas::bytes += reduce.bytes();
      blas::flops += reduce.flops();

      checkCudaError();
      return value;
    }

    /*
      Wilson
      double double2 M = 1/12
      single float4  M = 1/6
      half   short4  M = 6/6

      Staggered
      double double2 M = 1/3
      single float2  M = 1/3
      half   short2  M = 3/3
    */

    /**
       Driver for generic reduction routine with five loads.
       @param ReduceType
       @param siteUnroll - if this is true, then one site corresponds to exactly one thread
    */
    template <typename doubleN, typename ReduceType, template <typename ReducerType, typename Float, typename FloatN> class Reducer,
        int writeX, int writeY, int writeZ, int writeW, int writeV, bool siteUnroll>
    doubleN uni_reduce(const double2 &a, const double2 &b, ColorSpinorField &x, ColorSpinorField &y,
        ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {

      checkPrecision(x, y, z, w, v);

      doubleN value;
      if (checkLocation(x, y, z, w, v) == QUDA_CUDA_FIELD_LOCATION) {

        if (!x.isNative() && x.FieldOrder() != QUDA_FLOAT2_FIELD_ORDER && x.FieldOrder() != QUDA_FLOAT8_FIELD_ORDER) {
          warningQuda("Device reductions on non-native fields is not supported (prec = %d, order = %d)", x.Precision(),
                      x.FieldOrder());
          doubleN value;
          ::quda::zero(value);
          return value;
        }

        // cannot do site unrolling for arbitrary color (needs JIT)
        if (siteUnroll && x.Ncolor() != 3) errorQuda("Not supported");

        int reduce_length = siteUnroll ? x.RealLength() : x.Length();

        if (x.Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
          if (x.Nspin() == 4 || x.Nspin() == 2) { // wilson
#if defined(NSPIN4) || defined(NSPIN2)
            const int M = siteUnroll ? 12 : 1; // determines how much work per thread to do
            if (x.Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
            value = nativeReduce<doubleN, ReduceType, double2, double2, double2, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, reduce_length / (2 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = siteUnroll ? 3 : 1; // determines how much work per thread to do
            value = nativeReduce<doubleN, ReduceType, double2, double2, double2, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, reduce_length / (2 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = siteUnroll ? 6 : 1; // determines how much work per thread to do
            value = nativeReduce<doubleN, ReduceType, float4, float4, float4, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, reduce_length / (4 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1 || x.Nspin() == 2 || (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER)) {
#if defined(NSPIN1) || defined(NSPIN2) || defined(GPU_MULTIGRID)
            const int M = siteUnroll ? 3 : 1; // determines how much work per thread to do
            if ((x.Nspin() == 2 || x.Nspin() == 4) && siteUnroll) errorQuda("siteUnroll not supported here for nSpin=%d", x.Nspin());
            value = nativeReduce<doubleN, ReduceType, float2, float2, float2, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, reduce_length / (2 * M));
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_HALF_PRECISION) { // half precision

#if QUDA_PRECISION & 2
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = 6; // determines how much work per thread to do
            value = nativeReduce<doubleN, ReduceType, float4, short4, short4, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) { // wilson
#if defined(GPU_MULTIGRID)  // FIXME eventually we should get rid of this and use float4 ordering
            const int M = 12; // determines how much work per thread to do
            value
                = nativeReduce<doubleN, ReduceType, float2, short2, short2, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                    a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT8_FIELD_ORDER) { // wilson
#if defined(NSPIN4) && defined(FLOAT8)
            const int M = 3; // determines how much work per thread to do
            value
                = nativeReduce<doubleN, ReduceType, float8, short8, short8, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                    a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
            const int M = 3; // determines how much work per thread to do
            value = nativeReduce<doubleN, ReduceType, float2, short2, short2, M, Reducer, writeX, writeY, writeZ,
                writeW, writeV>(a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else if (x.Precision() == QUDA_QUARTER_PRECISION) { // quarter precision

#if QUDA_PRECISION & 1
          if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT4_FIELD_ORDER) { // wilson
#if defined(NSPIN4)
            const int M = 6; // determines how much work per thread to do
            value
                = nativeReduce<doubleN, ReduceType, float4, char4, char4, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                    a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) { // wilson
#if defined(GPU_MULTIGRID)  // FIXME eventually we should get rid of this and use float4 ordering
            const int M = 12; // determines how much work per thread to do
            value
              = nativeReduce<doubleN, ReduceType, float2, char2, char2, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT8_FIELD_ORDER) { // wilson
#if defined(NSPIN4) && defined(FLOAT8)
            const int M = 3;
            value
              = nativeReduce<doubleN, ReduceType, float8, char8, char8, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else if (x.Nspin() == 1) { // staggered
#ifdef NSPIN1
            const int M = 3; // determines how much work per thread to do
            value
                = nativeReduce<doubleN, ReduceType, float2, char2, char2, M, Reducer, writeX, writeY, writeZ, writeW, writeV>(
                    a, b, x, y, z, w, v, y.Volume());
#else
            errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
          } else {
            errorQuda("nSpin=%d is not supported\n", x.Nspin());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else {
          errorQuda("precision=%d is not supported\n", x.Precision());
        }
      } else { // fields are on the CPU
        // we don't have quad precision support on the GPU so use doubleN instead of ReduceType
        if (x.Precision() == QUDA_DOUBLE_PRECISION) {
          Reducer<doubleN, double2, double2> r(a, b);
          value = genericReduce<doubleN, doubleN, double, double, writeX, writeY, writeZ, writeW, writeV,
              Reducer<doubleN, double2, double2>>(x, y, z, w, v, r);
        } else if (x.Precision() == QUDA_SINGLE_PRECISION) {
          Reducer<doubleN, float2, float2> r(make_float2(a.x, a.y), make_float2(b.x, b.y));
          value = genericReduce<doubleN, doubleN, float, float, writeX, writeY, writeZ, writeW, writeV,
              Reducer<doubleN, float2, float2>>(x, y, z, w, v, r);
        } else {
          errorQuda("Precision %d not implemented", x.Precision());
        }
      }

      const int Nreduce = sizeof(doubleN) / sizeof(double);
      reduceDoubleArray((double *)&value, Nreduce);

      return value;
    }

    /**
       Driver for generic reduction routine with two loads.
       @param ReduceType
       @param siteUnroll - if this is true, then one site corresponds to exactly one thread
    */
    template <typename doubleN, typename ReduceType, template <typename ReducerType, typename Float, typename FloatN> class Reducer,
        int writeX, int writeY, int writeZ, int writeW, int writeV, bool siteUnroll>
    doubleN mixed_reduce(const double2 &a, const double2 &b, ColorSpinorField &x, ColorSpinorField &y,
        ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      checkPrecision(x, y, w, v);

      doubleN value;
      if (checkLocation(x, y, z, w, v) == QUDA_CUDA_FIELD_LOCATION) {

        if (!x.isNative() && !(x.Nspin() == 4 && x.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER && x.Precision() == QUDA_SINGLE_PRECISION)) {
          warningQuda("Device reductions on non-native fields is not supported (prec = %d, order = %d)", x.Precision(),
                      x.FieldOrder());
          doubleN value;
          ::quda::zero(value);
          return value;
        }

        // cannot do site unrolling for arbitrary color (needs JIT)
        if (x.Ncolor() != 3) errorQuda("Not supported");

        if (z.Precision() == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
          if (x.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 12; // determines how much work per thread to do
              value = nativeReduce<doubleN, ReduceType, double2, float4, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = siteUnroll ? 3 : 1; // determines how much work per thread to do
              const int reduce_length = siteUnroll ? x.RealLength() : x.Length();
              value = nativeReduce<doubleN, ReduceType, double2, float2, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, reduce_length / (2 * M));
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 12; // determines how much work per thread to do
              value = nativeReduce<doubleN, ReduceType, double2, short4, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 3; // determines how much work per thread to do
              value = nativeReduce<doubleN, ReduceType, double2, short2, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_QUARTER_PRECISION) {

#if QUDA_PRECISION & 1
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 12; // determines how much work per thread to do
              value = nativeReduce<doubleN, ReduceType, double2, char4, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 3; // determines how much work per thread to do
              value = nativeReduce<doubleN, ReduceType, double2, char2, double2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else {
            errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, z.Precision());
#endif

        } else if (z.Precision() == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
          if (x.Precision() == QUDA_HALF_PRECISION) {

#if QUDA_PRECISION & 2
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 6;
              value = nativeReduce<doubleN, ReduceType, float4, short4, float4, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 3;
              value = nativeReduce<doubleN, ReduceType, float2, short2, float2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
            blas::bytes
                += Reducer<ReduceType, double2, double2>::streams() * (unsigned long long)x.Volume() * sizeof(float);
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

          } else if (x.Precision() == QUDA_QUARTER_PRECISION) {
#if QUDA_PRECISION & 1
            if (x.Nspin() == 4) { // wilson
#if defined(NSPIN4)
              const int M = 6;
              value = nativeReduce<doubleN, ReduceType, float4, char4, float4, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else if (x.Nspin() == 1) { // staggered
#if defined(NSPIN1)
              const int M = 3;
              value = nativeReduce<doubleN, ReduceType, float2, char2, float2, M, Reducer, writeX, writeY, writeZ,
                  writeW, writeV>(a, b, x, y, z, w, v, x.Volume());
#else
              errorQuda("blas has not been built for Nspin=%d order=%d fields", x.Nspin(), x.FieldOrder());
#endif
            } else {
              errorQuda("ERROR: nSpin=%d is not supported\n", x.Nspin());
            }
            blas::bytes
                += Reducer<ReduceType, double2, double2>::streams() * (unsigned long long)x.Volume() * sizeof(float);
#else
            errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif
          } else {
            errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
          }
#else
          errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, x.Precision());
#endif

        } else {
          errorQuda("Not implemented for this precision combination %d %d", x.Precision(), z.Precision());
        }

      } else {
        // we don't have quad precision support on the GPU so use doubleN instead of ReduceType
        if (x.Precision() == QUDA_SINGLE_PRECISION && z.Precision() == QUDA_DOUBLE_PRECISION) {
          Reducer<doubleN, double2, double2> r(a, b);
          value = genericReduce<doubleN, doubleN, float, double, writeX, writeY, writeZ, writeW, writeV,
              Reducer<doubleN, double2, double2>>(x, y, z, w, v, r);
        } else {
          errorQuda("Precision %d not implemented", x.Precision());
        }
      }

      const int Nreduce = sizeof(doubleN) / sizeof(double);
      reduceDoubleArray((double *)&value, Nreduce);

      return value;
    }

    double norm1(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x); // FIXME
      return uni_reduce<double, QudaSumFloat, Norm1, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }

    double norm2(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x);
      return uni_reduce<double, QudaSumFloat, Norm2, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), y, y, y, y, y);
    }

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<double, QudaSumFloat, Dot, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<double, QudaSumFloat, axpbyzNorm2, 0, 0, 1, 0, 0, false>(
          make_double2(a, 0.0), make_double2(b, 0.0), x, y, z, x, x);
    }

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<double, QudaSumFloat, AxpyReDot, 0, 1, 0, 0, 0, false>(
          make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return uni_reduce<double, QudaSumFloat, caxpyNorm2, 0, 1, 0, 0, 0, false>(
          make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, x, x, x);
    }

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<double, QudaSumFloat, caxpyxmaznormx, 1, 1, 0, 0, 0, false>(
          make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
    }

    double cabxpyzAxNorm(double a, const Complex &b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return uni_reduce<double, QudaSumFloat, cabxpyzaxnorm, 1, 0, 1, 0, 0, false>(
          make_double2(a, 0.0), make_double2(REAL(b), IMAG(b)), x, y, z, x, x);
    }

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      double2 cdot = uni_reduce<double2, QudaSumFloat2, Cdot, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      double2 cdot = uni_reduce<double2, QudaSumFloat2, caxpydotzy, 0, 1, 0, 0, 0, false>(
          make_double2(REAL(a), IMAG(a)), make_double2(0.0, 0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y) {
      return uni_reduce<double3, QudaSumFloat3, CdotNormA, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, x, x);
    }

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x,
					   const Complex &b, ColorSpinorField &y,
					   ColorSpinorField &z, ColorSpinorField &w,
					   ColorSpinorField &u) {
      if (x.Precision() != z.Precision()) {
        return mixed_reduce<double3, QudaSumFloat3, caxpbypzYmbwcDotProductUYNormY_, 0, 1, 1, 0, 0, false>(
            make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      } else {
        return uni_reduce<double3, QudaSumFloat3, caxpbypzYmbwcDotProductUYNormY_, 0, 1, 1, 0, 0, false>(
            make_double2(REAL(a), IMAG(a)), make_double2(REAL(b), IMAG(b)), x, y, z, w, u);
      }
    }

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y) {
      // swizzle since mixed is on z
      double2 cg_norm ;
      if (x.Precision() != y.Precision()) {
        cg_norm = mixed_reduce<double2, QudaSumFloat2, axpyCGNorm2, 0, 0, 1, 0, 0, false>(
            make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      } else {
        cg_norm = uni_reduce<double2, QudaSumFloat2, axpyCGNorm2, 0, 0, 1, 0, 0, false>(
            make_double2(a, 0.0), make_double2(0.0, 0.0), x, x, y, x, x);
      }
      return Complex(cg_norm.x, cg_norm.y);
    }

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r) {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = uni_reduce<double3, QudaSumFloat3, HeavyQuarkResidualNorm_, 0, 0, 0, 0, 0, true>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y,
				      ColorSpinorField &r) {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = uni_reduce<double3, QudaSumFloat3, xpyHeavyQuarkResidualNorm_, 0, 0, 0, 0, 0, true>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return uni_reduce<double3, QudaSumFloat3, tripleCGReduction_, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return uni_reduce<double4, QudaSumFloat4, quadrupleCGReduction_, 0, 0, 0, 0, 0, false>(
          make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, z, x, x);
    }

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v) {
      return uni_reduce<double, QudaSumFloat, quadrupleCG3InitNorm_, 1, 1, 1, 1, 0, false>(
          make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, w, v);
    }

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v) {
      return uni_reduce<double, QudaSumFloat, quadrupleCG3UpdateNorm_, 1, 1, 1, 1, 0, false>(
          make_double2(a, 0.0), make_double2(b, 1. - b), x, y, z, w, v);
    }

    double doubleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return uni_reduce<double, QudaSumFloat, doubleCG3InitNorm_, 1, 1, 0, 0, 0, false>(
          make_double2(a, 0.0), make_double2(0.0, 0.0), x, y, z, z, z);
    }

    double doubleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z) {
      return uni_reduce<double, QudaSumFloat, doubleCG3UpdateNorm_, 1, 1, 0, 0, 0, false>(
          make_double2(a, 0.0), make_double2(b, 1.0 - b), x, y, z, z, z);
    }

   } // namespace blas

} // namespace quda
