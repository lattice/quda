#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <hipcub/hipcub.hpp> 
#include <launch_kernel.cuh>

#include <hip/device_functions.h>

#include <hisq_links_quda.h> //reunit gauge links!!!!!

#include <comm_quda.h>


#include <gauge_fix_ovr_extra.h>


#include <gauge_fix_ovr_hit_devf.cuh>


namespace quda {



static int numParams = 18;

#define LAUNCH_KERNEL_GAUGEFIX(kernel, tp, stream, arg, parity, ...)     \
  if(tp.block.z==0){\
  switch (tp.block.x) {             \
  case 256:                \
    kernel<0, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:                \
    kernel<0, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:                \
    kernel<0, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<0, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else if(tp.block.z==1){\
  switch (tp.block.x) {             \
  case 256:                \
    kernel<1, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:                \
    kernel<1, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:                \
    kernel<1, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<1, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else if(tp.block.z==2){\
  switch (tp.block.x) {             \
  case 256:                \
    kernel<2, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:                \
    kernel<2, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:                \
    kernel<2, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<2, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else if(tp.block.z==3){\
  switch (tp.block.x) {             \
  case 128:                \
    kernel<3, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 256:                \
    kernel<3, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 384:                \
    kernel<3, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:               \
    kernel<3, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 640:               \
    kernel<3, 160,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:               \
    kernel<3, 192,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 896:               \
    kernel<3, 224,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<3, 256,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else if(tp.block.z==4){\
  switch (tp.block.x) {             \
  case 128:                \
    kernel<4, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 256:                \
    kernel<4, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 384:                \
    kernel<4, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:               \
    kernel<4, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 640:               \
    kernel<4, 160,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:               \
    kernel<4, 192,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 896:               \
    kernel<4, 224,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<4, 256,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else if(tp.block.z==5){\
  switch (tp.block.x) {             \
  case 128:                \
    kernel<5, 32,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 256:                \
    kernel<5, 64,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 384:                \
    kernel<5, 96,__VA_ARGS__>           \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 512:               \
    kernel<5, 128,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 640:               \
    kernel<5, 160,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 768:               \
    kernel<5, 192,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 896:               \
    kernel<5, 224,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  case 1024:               \
    kernel<5, 256,__VA_ARGS__>            \
      <<< tp.grid.x, tp.block.x, tp.shared_bytes, stream >>>(arg, parity);   \
    break;                \
  default:                \
    errorQuda("%s not implemented for %d threads", #kernel, tp.block.x); \
    }\
  }\
  else{\
    errorQuda("Not implemented for %d", tp.block.z);\
  }



template<class T>
__device__ __host__ inline Matrix<T,3> getSubTraceUnit(const Matrix<T,3>& a){
  T tr = (a(0,0) + a(1,1) + a(2,2)) / 3.0;
  Matrix<T,3> res;
  res(0,0) = a(0,0)- tr; res(0,1) = a(0,1); res(0,2) = a(0,2);
  res(1,0) = a(1,0); res(1,1) = a(1,1)-tr; res(1,2) = a(1,2);
  res(2,0) = a(2,0); res(2,1) = a(2,1); res(2,2) = a(2,2)-tr;
  return res;
}

template<class T>
__device__ __host__ inline void SubTraceUnit(Matrix<T,3>& a){
  T tr = (a(0,0) + a(1,1) + a(2,2)) / 3.0;
  a(0,0)-= tr; a(1,1) -= tr; a(2,2) -= tr;
}

template<class T>
__device__ __host__ inline double getRealTraceUVdagger(const Matrix<T,3>& a, const Matrix<T,3>& b){
   double sum = (double)(a(0,0).x * b(0,0).x  + a(0,0).y * b(0,0).y);
   sum += (double)(a(0,1).x * b(0,1).x  + a(0,1).y * b(0,1).y);
   sum += (double)(a(0,2).x * b(0,2).x  + a(0,2).y * b(0,2).y);
   sum += (double)(a(1,0).x * b(1,0).x  + a(1,0).y * b(1,0).y);
   sum += (double)(a(1,1).x * b(1,1).x  + a(1,1).y * b(1,1).y);
   sum += (double)(a(1,2).x * b(1,2).x  + a(1,2).y * b(1,2).y);
   sum += (double)(a(2,0).x * b(2,0).x  + a(2,0).y * b(2,0).y);
   sum += (double)(a(2,1).x * b(2,1).x  + a(2,1).y * b(2,1).y);
   sum += (double)(a(2,2).x * b(2,2).x  + a(2,2).y * b(2,2).y);
  return sum;
}



template <typename T>
struct Summ {
    __host__ __device__ __forceinline__ T operator()(const T &a, const T &b){
        return a + b;
    }
};
template <>
struct Summ<double2>{
    __host__ __device__ __forceinline__ double2 operator()(const double2 &a, const double2 &b){
        return make_double2(a.x+b.x, a.y+b.y);
    }
};




static __device__ __host__ inline int linkIndex3(int x[], int dx[], const int X[4]) {
  int y[4];
  for (int i=0; i<4; i++) y[i] = (x[i] + dx[i] + X[i]) % X[i];
  int idx = (((y[3]*X[2] + y[2])*X[1] + y[1])*X[0] + y[0]) >> 1;
  return idx;
}
static __device__ __host__ inline int linkIndex(int x[], const int X[4]) {
  int idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
  return idx;
}
static __device__ __host__ inline int linkIndexM1(int x[], const int X[4], const int mu) {
  int y[4];
  for (int i=0; i<4; i++) y[i] = x[i];
  y[mu] = (y[mu] -1 + X[mu]) % X[mu];
  int idx = (((y[3]*X[2] + y[2])*X[1] + y[1])*X[0] + y[0]) >> 1;
  return idx;
}


static __device__ __host__ inline void getCoords3(int x[4], int cb_index, const int X[4], int parity) {
  /*x[3] = cb_index/(X[2]*X[1]*X[0]/2);
  x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
  x[1] = (cb_index/(X[0]/2)) % X[1];
  x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);*/
  int za = (cb_index / (X[0]/2));
  int zb =  (za / X[1]);
  x[1] = za - zb * X[1];
  x[3] = (zb / X[2]);
  x[2] = zb - x[3] * X[2];
  int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
  x[0] = (2 * cb_index + x1odd)  - za * X[0];
  return;
}








template <typename Gauge>
struct GaugeFixQualityArg {
  int threads; // number of active threads required
  int X[4]; // grid dimensions
#ifdef MULTI_GPU
  int border[4]; 
#endif
  Gauge dataOr;
  double2 *quality;
  double2 *quality_h;
  GaugeFixQualityArg(const Gauge &dataOr, const cudaGaugeField &data)
    : dataOr(dataOr) {
    //: dataOr(dataOr), quality_h(static_cast<double2*>(pinned_malloc(sizeof(double2)))) {

    for(int dir=0; dir<4; ++dir){
      X[dir] = data.X()[dir] - data.R()[dir]*2;
      #ifdef MULTI_GPU
      border[dir] = data.R()[dir];
      #endif
    }
    threads = X[0]*X[1]*X[2]*X[3];
    quality = (double2*)device_malloc(sizeof(double2));
    quality_h = (double2*)safe_malloc(sizeof(double2));
    //hipHostGetDevicePointer(&quality, quality_h, 0);
  }
  double getAction(){return quality_h[0].x;}
  double getTheta(){return quality_h[0].y;}
};



template<int blockSize, typename Float, typename Gauge, int gauge_dir>
__global__ void computeFix_quality(GaugeFixQualityArg<Gauge> argQ){
  int idx = threadIdx.x + blockIdx.x*blockDim.x;

  typedef hipcub::BlockReduce<double2, blockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  
  //AVOID SHAREDMEM PROBLEMS!!!!!!! hipcub::BlockReduce<double2, blockSize> not initialize memory with 0? 
  double2 data = make_double2(0.0,0.0);
  if(idx < argQ.threads) {
    typedef typename ComplexTypeId<Float>::Type Cmplx;
    int parity = 0;
    if(idx >= argQ.threads/2) {
      parity = 1;
      idx -= argQ.threads/2;
    }
    int X[4]; 
    #pragma unroll
    for(int dr=0; dr<4; ++dr) X[dr] = argQ.X[dr];

    int x[4];
    getCoords3(x, idx, X, parity);
#ifdef MULTI_GPU
    #pragma unroll
    for(int dr=0; dr<4; ++dr) {
         x[dr] += argQ.border[dr];
         X[dr] += 2*argQ.border[dr];
    }
#endif
    Matrix<Cmplx,3> delta;
    setZero(&delta);
    idx = linkIndex(x,X);
  
    for (int mu = 0; mu < gauge_dir; mu++) { 
      Matrix<Cmplx,3> U; 
      argQ.dataOr.load((Float*)(U.data),idx, mu, parity);
      delta -= U;
    }
    //18*gauge_dir
    data.x = -delta(0,0).x - delta(1,1).x - delta(2,2).x ;
    //2
    for (int mu = 0; mu < gauge_dir; mu++) {
      Matrix<Cmplx,3> U; 
      argQ.dataOr.load((Float*)(U.data),linkIndexM1(x,X,mu), mu, 1 - parity);
      delta += U;
    }
    //18*gauge_dir
    delta -= conj(delta);
    //18
    SubTraceUnit(delta);
    //12
    data.y = getRealTraceUVdagger(delta, delta);
    //35
    //T=36*gauge_dir+65
  }
  //This must be here for the case when the total number of threads is not multiple of blocksize!!!!
  //HOW TO pre-initialize temp_storage to 0?
  double2 aggregate = BlockReduce(temp_storage).Reduce(data, Summ<double2>());
  if (threadIdx.x == 0) atomicAdd(argQ.quality, aggregate);
}



template<typename Float, typename Gauge, int gauge_dir>
class GaugeFixQuality : Tunable {
  GaugeFixQualityArg<Gauge> argQ;
  mutable char aux_string[128]; // used as a label in the autotuner
  private:
  unsigned int sharedBytesPerThread() const { return 0; }
  unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
  bool tuneSharedBytes() const { return false; } // Don't tune shared memory
  bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
  unsigned int minThreads() const { return argQ.threads; }

  public:
  GaugeFixQuality(GaugeFixQualityArg<Gauge> &argQ)
    : argQ(argQ) {}
  ~GaugeFixQuality () { host_free(argQ.quality_h);device_free(argQ.quality);}

  void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      //argQ.quality_h[0] = make_double2(0.0,0.0);
      hipMemset(argQ.quality, 0, sizeof(double2));
      LAUNCH_KERNEL(computeFix_quality, tp, stream, argQ, Float, Gauge, gauge_dir);
      hipMemcpy(argQ.quality_h, argQ.quality, sizeof(double2), hipMemcpyDeviceToHost);
      //hipDeviceSynchronize();
      #ifdef MULTI_GPU        
        if(comm_size() != 1) comm_allreduce_array((double*)argQ.quality_h, 2);
        const int nNodes = comm_dim(0)*comm_dim(1)*comm_dim(2)*comm_dim(3);
        argQ.quality_h[0].x  /= (double)(3*gauge_dir*argQ.threads*nNodes);
        argQ.quality_h[0].y  /= (double)(3*argQ.threads*nNodes);
      #else
        argQ.quality_h[0].x  /= (double)(3*gauge_dir*argQ.threads);
        argQ.quality_h[0].y  /= (double)(3*argQ.threads);
      #endif
  }

  TuneKey tuneKey() const {
    std::stringstream vol;
    vol << argQ.X[0] << "x";
    vol << argQ.X[1] << "x";
    vol << argQ.X[2] << "x";
    vol << argQ.X[3];
    sprintf(aux_string,"threads=%d,prec=%d,gaugedir=%d",argQ.threads, sizeof(Float),gauge_dir);
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune(){}
  void postTune(){}
  long long flops() const { return (36LL*gauge_dir+65LL)*argQ.threads; }// Only correct if there is no link reconstruction, no cub reduction accounted also
  //long long bytes() const { return (1)*2*gauge_dir*argQ.dataOr.Bytes(); }//no accounting the reduction!!!! argQ.dataOr.Bytes() return 0....
  long long bytes() const { return 2LL*gauge_dir*argQ.threads*numParams*sizeof(Float); }//no accounting the reduction!!!!

}; 


//  template <typename Float, typename Gauge>
template <typename Float, typename Gauge>
struct GaugeFixArg {
  int threads; // number of active threads required
  int X[4]; // grid dimensions
#ifdef MULTI_GPU
  int border[4]; 
#endif
  Gauge dataOr;
  cudaGaugeField &data;
  const Float relax_boost;

  GaugeFixArg(Gauge &dataOr, cudaGaugeField &data, const Float relax_boost)
    : dataOr(dataOr), data(data), relax_boost(relax_boost) {

    for(int dir=0; dir<4; ++dir){
      X[dir] = data.X()[dir] - data.R()[dir]*2;
      #ifdef MULTI_GPU
      border[dir] = data.R()[dir];
      #endif
    }
    threads = X[0]*X[1]*X[2]*X[3] >> 1;
  }
};





template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
__global__ void computeFix(GaugeFixArg<Float, Gauge> arg, int parity){
  int tid = (threadIdx.x + blockSize) % blockSize;  
  int idx = blockIdx.x * blockSize + tid;

  if(idx >= arg.threads) return;

  typedef typename ComplexTypeId<Float>::Type Cmplx;

  if(ImplementationType<3){
    int X[4]; 
    #pragma unroll
    for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

    int x[4];
    getCoords3(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
    for(int dr=0; dr<4; ++dr) {
         x[dr] += arg.border[dr];
         X[dr] += 2*arg.border[dr];
    }
  #endif
    int mu = (threadIdx.x / blockSize);
    int oddbit = parity;
    if(threadIdx.x >= blockSize * 4){
      mu -= 4;
      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      oddbit = 1 - parity;
    }
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, oddbit);
    if(ImplementationType==0) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==1) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==2)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    arg.dataOr.save((Float*)(link.data),idx, mu, oddbit);
  }
  else{
    int X[4]; 
    #pragma unroll
    for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

    int x[4];
    getCoords3(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
    for(int dr=0; dr<4; ++dr) {
         x[dr] += arg.border[dr];
         X[dr] += 2*arg.border[dr];
    }
  #endif
    int mu = (threadIdx.x / blockSize);
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, parity);


    x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
    int idx1 = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link1;
    arg.dataOr.load((Float*)(link1.data),idx1, mu, 1-parity);

    if(ImplementationType==3) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==4) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==5)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

    arg.dataOr.save((Float*)(link.data),idx, mu, parity);
    arg.dataOr.save((Float*)(link1.data),idx1, mu, 1-parity);

  }
}






template<typename Float, typename Gauge, int gauge_dir>
class GaugeFix : Tunable {
  GaugeFixArg<Float, Gauge> arg;
  int parity;
  mutable char aux_string[128]; // used as a label in the autotuner
protected:

  dim3 createGrid   (const dim3 &block) const {
    unsigned int blockx = block.x / 8;
    if(block.z > 2) blockx = block.x / 4;
    unsigned int  gx  = (arg.threads + blockx - 1) / blockx;
    return  dim3(gx, 1, 1);
  }
  bool advanceBlockDim  (TuneParam &param) const {
    //Use param.block.z to tune and save state for best kernel option
    // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
    const unsigned int min_threads0 = 32 * 8;
    const unsigned int min_threads1 = 32 * 4;
    const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
    const unsigned int atmadd = 0;
    unsigned int min_threads = min_threads0;
    param.block.z += atmadd;    //USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
    if(param.block.z > 2) min_threads = 32 * 4;
    param.block.x += min_threads;
    param.block.y = 1;    
    param.grid  = createGrid(param.block);


    
    if  ((param.block.x >= min_threads) && (param.block.x <= max_threads)){
      if(param.block.z == 0) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 4 || param.block.z == 5) param.shared_bytes = param.block.x * sizeof(Float);
      return  true;
    }
    else if(param.block.z == 0){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 1;    //USE FOR ATOMIC ADD
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 1){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 2;    //USE FOR NO ATOMIC ADD and LESS SHARED MEM
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 2){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 3;        //USE FOR NO ATOMIC ADD 
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float);
      return true;
    }
    else if(param.block.z == 3){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 4;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else if(param.block.z == 4){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 5;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else
      return  false;
  }
  private:
  unsigned int sharedBytesPerThread() const { 
    return 0; 
  }
  unsigned int sharedBytesPerBlock(const TuneParam &param) const { 
      if(param.block.z == 0) return param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) return param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) return param.block.x * 4 * sizeof(Float);
      else return param.block.x * sizeof(Float);
  }

  bool tuneSharedBytes() const { return false; } // Don't tune shared memory
  bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
  unsigned int minThreads() const { return arg.threads; }

  public:
  virtual void initTuneParam(TuneParam &param) const{
    param.block = dim3(256, 1, 0);
    param.grid = createGrid(param.block);
    param.shared_bytes = param.block.x * 4 * sizeof(Float);
  }

  GaugeFix(GaugeFixArg<Float, Gauge> &arg) : arg(arg) {
      int parity = 0;
    }
  ~GaugeFix () { }
  void setParity(const int par){ parity = par; }

  void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
    LAUNCH_KERNEL_GAUGEFIX(computeFix, tp, stream, arg, parity, Float, Gauge, gauge_dir);
  }

  /** Sets default values for when tuning is disabled - this is guaranteed to work, but will be slow */
  virtual void defaultTuneParam(TuneParam &param) const{ initTuneParam(param); }

  TuneKey tuneKey() const {
    std::stringstream vol;
    vol << arg.X[0] << "x";
    vol << arg.X[1] << "x";
    vol << arg.X[2] << "x";
    vol << arg.X[3];
    sprintf(aux_string,"threads=%d,prec=%d,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
  }

  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    ps << ", atomicadd=" << param.block.z;
    return ps.str();
  }

  //need this
  void preTune() { arg.data.backup(); }
  void postTune() { arg.data.restore(); }
  long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3)*arg.threads; }// Only correct if there is no link reconstruction
  //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
  long long bytes() const { return 8LL*2*arg.threads*numParams*sizeof(Float); }//no accounting the reduction!!!!
}; 




#ifdef MULTI_GPU
template <typename Float, typename Gauge>
struct GaugeFixInteriorPointsArg {
  int threads; // number of active threads required
  int X[4]; // grid dimensions
#ifdef MULTI_GPU
  int border[4]; 
#endif
  Gauge dataOr;
  cudaGaugeField &data;
  const Float relax_boost;
  GaugeFixInteriorPointsArg(Gauge &dataOr, cudaGaugeField &data, const Float relax_boost)
    : dataOr(dataOr), data(data), relax_boost(relax_boost) {

#ifdef MULTI_GPU   
    for(int dir=0; dir<4; ++dir){
      if(comm_dim_partitioned(dir)) border[dir] = data.R()[dir] + 1; //skip BORDER_RADIUS + face border point
      else border[dir] = 0;
    }
    for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir] - border[dir]*2;
#else
    for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir];
#endif
    threads = X[0]*X[1]*X[2]*X[3] >> 1;
  }
};




template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
__global__ void computeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> arg, int parity){
  int tid = (threadIdx.x + blockSize) % blockSize;  
  int idx = blockIdx.x * blockSize + tid;
  if(idx >= arg.threads) return;
  typedef typename ComplexTypeId<Float>::Type Cmplx;
  int X[4];
  #pragma unroll 
  for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];
  int x[4];
#ifdef MULTI_GPU
  int za = (idx / (X[0]/2));
  int zb =  (za / X[1]);
  x[1] = za - zb * X[1];
  x[3] = (zb / X[2]);
  x[2] = zb - x[3] * X[2];
  int p=0; for(int dr=0; dr<4; ++dr) p += arg.border[dr]; 
  p = p & 1;
  int x1odd = (x[1] + x[2] + x[3] + parity + p) & 1;
  //int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
  x[0] = (2 * idx + x1odd)  - za * X[0];
  for(int dr=0; dr<4; ++dr) {
       x[dr] += arg.border[dr];
       X[dr] += 2 * arg.border[dr];
  }
#else
  getCoords3(x, idx, X, parity);
#endif
  int mu = (threadIdx.x / blockSize);

  if(ImplementationType<3){
    if(threadIdx.x >= blockSize * 4){
      mu -= 4;
      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      parity = 1 - parity;
    }
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, parity);
    if(ImplementationType==0) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==1) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==2)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    arg.dataOr.save((Float*)(link.data),idx, mu, parity);
  }
 else{
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, parity);


    x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
    int idx1 = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link1;
    arg.dataOr.load((Float*)(link1.data),idx1, mu, 1-parity);

    if(ImplementationType==3) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==4) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==5)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

    arg.dataOr.save((Float*)(link.data),idx, mu, parity);
    arg.dataOr.save((Float*)(link1.data),idx1, mu, 1-parity);

  }
}








template<typename Float, typename Gauge, int gauge_dir>
class GaugeFixInteriorPoints : Tunable {
  GaugeFixInteriorPointsArg<Float, Gauge> arg;
  int parity;
  mutable char aux_string[128]; // used as a label in the autotuner
protected:

  dim3 createGrid   (const dim3 &block) const {
    unsigned int blockx = block.x / 8;
    if(block.z > 2) blockx = block.x / 4;
    unsigned int  gx  = (arg.threads + blockx - 1) / blockx;
    return  dim3(gx, 1, 1);
  }
  bool advanceBlockDim  (TuneParam &param) const {
    //Use param.block.z to tune and save state for best kernel option
    // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
    const unsigned int min_threads0 = 32 * 8;
    const unsigned int min_threads1 = 32 * 4;
    const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
    const unsigned int atmadd = 0;
    unsigned int min_threads = min_threads0;
    param.block.z += atmadd;    //USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
    if(param.block.z > 2) min_threads = 32 * 4;
    param.block.x += min_threads;
    param.block.y = 1;    
    param.grid  = createGrid(param.block);


    
    if  ((param.block.x >= min_threads) && (param.block.x <= max_threads)){
      if(param.block.z == 0) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 4 || param.block.z == 5) param.shared_bytes = param.block.x * sizeof(Float);
      return  true;
    }
    else if(param.block.z == 0){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 1;    //USE FOR ATOMIC ADD
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 1){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 2;    //USE FOR NO ATOMIC ADD and LESS SHARED MEM
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 2){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 3;        //USE FOR NO ATOMIC ADD 
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float);
      return true;
    }
    else if(param.block.z == 3){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 4;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else if(param.block.z == 4){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 5;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else
      return  false;
  }
  private:
  unsigned int sharedBytesPerThread() const { 
    return 0; 
  }
  unsigned int sharedBytesPerBlock(const TuneParam &param) const { 
      if(param.block.z == 0) return param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) return param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) return param.block.x * 4 * sizeof(Float);
      else return param.block.x * sizeof(Float);
  }

  bool tuneSharedBytes() const { return false; } // Don't tune shared memory
  bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
  unsigned int minThreads() const { return arg.threads; }

  public:
  virtual void initTuneParam(TuneParam &param) const{
    param.block = dim3(256, 1, 0);
    param.grid = createGrid(param.block);
    param.shared_bytes = param.block.x * 4 * sizeof(Float);
  }
  GaugeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> &arg) : arg(arg) {
      int parity = 0;
    }
  ~GaugeFixInteriorPoints () { }
  void setParity(const int par){ parity = par; }

  void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
    LAUNCH_KERNEL_GAUGEFIX(computeFixInteriorPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
  }


  /** Sets default values for when tuning is disabled - this is guaranteed to work, but will be slow */
  virtual void defaultTuneParam(TuneParam &param) const{ initTuneParam(param); }

  TuneKey tuneKey() const {
    std::stringstream vol;
    vol << arg.X[0] << "x";
    vol << arg.X[1] << "x";
    vol << arg.X[2] << "x";
    vol << arg.X[3];
    sprintf(aux_string,"threads=%d,prec=%d,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
  }

  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    ps << ", atomicadd=" << param.block.z;
    return ps.str();
  }

  //need this
  void preTune() { arg.data.backup(); }
  void postTune() { arg.data.restore(); }
  long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3)*arg.threads; }// Only correct if there is no link reconstruction
  //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
  long long bytes() const { return 8LL*2*arg.threads*numParams*sizeof(Float); } // Only correct if there is no link reconstruction load+save
}; 
















template <typename Float, typename Gauge>
struct GaugeFixBorderPointsArg {
  int threads; // number of active threads required
  int X[4]; // grid dimensions
  int border[4]; 
  int *borderpoints[2];
  int *faceindicessize[2];
  size_t faceVolume[4];
  size_t faceVolumeCB[4];
  Gauge dataOr;
  cudaGaugeField &data;
  const Float relax_boost;

  GaugeFixBorderPointsArg(Gauge &dataOr, cudaGaugeField &data, const Float relax_boost, size_t faceVolume_[4], size_t faceVolumeCB_[4])
    : dataOr(dataOr), data(data), relax_boost(relax_boost) {


    for(int dir=0; dir<4; ++dir){
      X[dir] = data.X()[dir] - data.R()[dir]*2;
      border[dir] = data.R()[dir];
    }

    /*for(int dir=0; dir<4; ++dir){
      if(comm_dim_partitioned(dir)) border[dir] = BORDER_RADIUS;
      else border[dir] = 0;
    }
    for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir] - border[dir]*2;*/
    for(int dir=0; dir<4; ++dir){
      faceVolume[dir] = faceVolume_[dir];
      faceVolumeCB[dir] = faceVolumeCB_[dir];
    }
    if(comm_size() !=1) PreCalculateLatticeIndices(faceVolume, faceVolumeCB, X, border, threads, borderpoints);
  }
};

template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
__global__ void computeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> arg, int parity){
  int tid = (threadIdx.x + blockSize) % blockSize;  
  int idx = blockIdx.x * blockSize + tid;
  if(idx >= arg.threads) return;
  typedef typename ComplexTypeId<Float>::Type Cmplx;
  int mu = (threadIdx.x / blockSize);
  idx = arg.borderpoints[parity][idx];
  int X[4], x[4];
  x[3] = idx/(arg.X[0] * arg.X[1]  * arg.X[2]);
  x[2] = (idx/(arg.X[0] * arg.X[1])) % arg.X[2];
  x[1] = (idx/arg.X[0]) % arg.X[1];
  x[0] = idx % arg.X[0];
  #pragma unroll
  for(int dr=0; dr<4; ++dr) x[dr] += arg.border[dr];
  #pragma unroll
  for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr] + 2 * arg.border[dr];

  if(ImplementationType<3){
    if(threadIdx.x >= blockSize * 4){
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        parity = 1 - parity;
    }
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, parity);
    if(ImplementationType==0) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==1) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    if(ImplementationType==2)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
    arg.dataOr.save((Float*)(link.data),idx, mu, parity);
  }
  else{
    idx = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link;
    arg.dataOr.load((Float*)(link.data),idx, mu, parity);


    x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
    int idx1 = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    Matrix<Cmplx,3> link1;
    arg.dataOr.load((Float*)(link1.data),idx1, mu, 1-parity);

    if(ImplementationType==3) GaugeFixHit_NoAtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==4) GaugeFixHit_AtomicAdd<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
    if(ImplementationType==5)GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Cmplx, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

    arg.dataOr.save((Float*)(link.data),idx, mu, parity);
    arg.dataOr.save((Float*)(link1.data),idx1, mu, 1-parity);
  }
}




template<typename Float, typename Gauge, int gauge_dir>
class GaugeFixBorderPoints : Tunable {
  GaugeFixBorderPointsArg<Float, Gauge> arg;
  int parity;
  mutable char aux_string[128]; // used as a label in the autotuner
protected:

  dim3 createGrid   (const dim3 &block) const {
    unsigned int blockx = block.x / 8;
    if(block.z > 2) blockx = block.x / 4;
    unsigned int  gx  = (arg.threads + blockx - 1) / blockx;
    return  dim3(gx, 1, 1);
  }
  bool advanceBlockDim  (TuneParam &param) const {
    //Use param.block.z to tune and save state for best kernel option
    // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
    const unsigned int min_threads0 = 32 * 8;
    const unsigned int min_threads1 = 32 * 4;
    const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
    const unsigned int atmadd = 0;
    unsigned int min_threads = min_threads0;
    param.block.z += atmadd;    //USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
    if(param.block.z > 2) min_threads = 32 * 4;
    param.block.x += min_threads;
    param.block.y = 1;    
    param.grid  = createGrid(param.block);


    
    if  ((param.block.x >= min_threads) && (param.block.x <= max_threads)){
      if(param.block.z == 0) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) param.shared_bytes = param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 4 || param.block.z == 5) param.shared_bytes = param.block.x * sizeof(Float);
      return  true;
    }
    else if(param.block.z == 0){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 1;    //USE FOR ATOMIC ADD
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 1){
      param.block.x = min_threads0;   
      param.block.y = 1;    
      param.block.z = 2;    //USE FOR NO ATOMIC ADD and LESS SHARED MEM
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
      return true;
    }
    else if(param.block.z == 2){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 3;        //USE FOR NO ATOMIC ADD 
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * 4 * sizeof(Float);
      return true;
    }
    else if(param.block.z == 3){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 4;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else if(param.block.z == 4){
      param.block.x = min_threads1;   
      param.block.y = 1;    
      param.block.z = 5;
      param.grid  = createGrid(param.block);
      param.shared_bytes = param.block.x * sizeof(Float);
      return true;
    }
    else
      return  false;
  }
  private:
  unsigned int sharedBytesPerThread() const { 
    return 0; 
  }
  unsigned int sharedBytesPerBlock(const TuneParam &param) const { 
      if(param.block.z == 0) return param.block.x * 4 * sizeof(Float);
      else if(param.block.z == 1 || param.block.z == 2) return param.block.x * 4 * sizeof(Float) / 8;
      else if(param.block.z == 3) return param.block.x * 4 * sizeof(Float);
      else return param.block.x * sizeof(Float);
  }

  bool tuneSharedBytes() const { return false; } // Don't tune shared memory
  bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
  unsigned int minThreads() const { return arg.threads; }

  public:
  virtual void initTuneParam(TuneParam &param) const{
    param.block = dim3(256, 1, 0);
    param.grid = createGrid(param.block);
    param.shared_bytes = param.block.x * 4 * sizeof(Float);
  }
  GaugeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> &arg) : arg(arg) {
      int parity = 0;
  }
  ~GaugeFixBorderPoints () { 
    if(comm_size() !=1) for(int i = 0; i < 2; i++) hipFree(arg.borderpoints[i]);
   }
  void setParity(const int par){ parity = par; }

  void apply(const hipStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
    LAUNCH_KERNEL_GAUGEFIX(computeFixBorderPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
  }

  /** Sets default values for when tuning is disabled - this is guaranteed to work, but will be slow */
  virtual void defaultTuneParam(TuneParam &param) const{ initTuneParam(param); }

  TuneKey tuneKey() const {
    std::stringstream vol;
    vol << arg.X[0] << "x";
    vol << arg.X[1] << "x";
    vol << arg.X[2] << "x";
    vol << arg.X[3];
    sprintf(aux_string,"threads=%d,prec=%d,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
  }

  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    ps << ", atomicadd=" << param.block.z;
    return ps.str();
  }

  //need this
  void preTune() { arg.data.backup(); }
  void postTune() { arg.data.restore(); }
  long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3)*arg.threads; }// Only correct if there is no link reconstruction
  //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
  long long bytes() const { return 8LL*2*arg.threads*numParams*sizeof(Float); } // Only correct if there is no link reconstruction load+save

}; 














template <typename Gauge>
struct GaugeFixUnPackArg {
  int X[4]; // grid dimensions
#ifdef MULTI_GPU
  int border[4]; 
#endif
  Gauge dataOr;
  GaugeFixUnPackArg(Gauge &dataOr, cudaGaugeField &data)
    : dataOr(dataOr) {
    for(int dir=0; dir<4; ++dir){
      X[dir] = data.X()[dir] - data.R()[dir]*2;
      #ifdef MULTI_GPU
      border[dir] = data.R()[dir];
      #endif
    }
  }
};


template<int NElems, typename Float, typename Gauge, bool pack>
__global__ void Kernel_UnPackGhost(int size, GaugeFixUnPackArg<Gauge> arg, typename ComplexTypeId<Float>::Type *array, int parity, int face, int dir){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= size) return;
  int X[4]; 
  for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];
  int x[4];
  int za, xodd;
  int borderid = 0;
  parity = 1 - parity;
  switch(face){
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
    break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
  }
  for(int dr=0; dr<4; ++dr) {
       x[dr] += arg.border[dr];
       X[dr] += 2*arg.border[dr];
  }
  x[face] -= 1;
  parity = 1 - parity;
  int id = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
  typedef typename ComplexTypeId<Float>::Type Cmplx;
  typedef typename mapper<Float>::type RegType;
  RegType tmp[NElems];
  RegType data[18];
  if(pack){
    arg.dataOr.load(data, id, dir, parity);
    arg.dataOr.reconstruct.Pack(tmp, data, id);
    for(int i=0; i<NElems/2; ++i) array[idx + size * i] = ((Cmplx*)tmp)[i];
  }
else{
    for(int i=0; i<NElems/2; ++i) ((Cmplx*)tmp)[i] = array[idx + size * i];
    arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0);
    arg.dataOr.save(data, id, dir, parity);
  }
}




template<int NElems, typename Float, typename Gauge, bool pack>
__global__ void Kernel_UnPackTop(int size, GaugeFixUnPackArg<Gauge> arg, typename ComplexTypeId<Float>::Type *array, int parity, int face, int dir){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= size) return;
  int X[4]; 
  for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];
  int x[4];
  int za, xodd;
  int borderid = arg.X[face] - 1;
  switch(face){
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
    break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
    break;
  }
  for(int dr=0; dr<4; ++dr) {
       x[dr] += arg.border[dr];
       X[dr] += 2*arg.border[dr];
  }
  int id = (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
  typedef typename ComplexTypeId<Float>::Type Cmplx;
  typedef typename mapper<Float>::type RegType;
  RegType tmp[NElems];
  RegType data[18];
  if(pack){
    arg.dataOr.load(data, id, dir, parity);
    arg.dataOr.reconstruct.Pack(tmp, data, id);
    for(int i=0; i<NElems/2; ++i) array[idx + size * i] = ((Cmplx*)tmp)[i];
  }
  else{
    for(int i=0; i<NElems/2; ++i) ((Cmplx*)tmp)[i] = array[idx + size * i];
    arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0);
    arg.dataOr.save(data, id, dir, parity);
  }
}
#endif


















template<typename Float, typename Gauge, int NElems, int gauge_dir>
void gaugefixingOVR( Gauge dataOr,  cudaGaugeField& data, \
  const unsigned int Nsteps, const unsigned int verbose_interval, \
  const Float relax_boost, const double tolerance, \
  const unsigned int reunit_interval, \
  const unsigned int stopWtheta) {


  TimeProfile profileGaugeFix("GaugeFixCuda");

  profileGaugeFix.Start(QUDA_PROFILE_COMPUTE);
  double flop = 0;
  double byte = 0;

  

  printfQuda("\tOverrelaxation boost parameter: %lf\n", (double)relax_boost);
  printfQuda("\tStop criterium: %lf\n", tolerance);
  if(stopWtheta) printfQuda("\tStop criterium method: theta\n");
  else           printfQuda("\tStop criterium method: Delta\n");
  printfQuda("\tMaximum number of iterations: %d\n", Nsteps);
  printfQuda("\tReunitarize at every %d steps\n", reunit_interval);
  printfQuda("\tPrint convergence results at every %d steps\n", verbose_interval);

  
  const double unitarize_eps = 1e-14;
  const double max_error = 1e-10;
  const int reunit_allow_svd = 1;
  const int reunit_svd_only  = 0;
  const double svd_rel_error = 1e-6;
  const double svd_abs_error = 1e-6;
  setUnitarizeLinksConstants(unitarize_eps, max_error,
      reunit_allow_svd, reunit_svd_only,
      svd_rel_error, svd_abs_error);
  int num_failures=0;
  int* num_failures_dev;
  hipMalloc((void**)&num_failures_dev, sizeof(int));
  hipMemset(num_failures_dev, 0, sizeof(int));
  if(num_failures_dev == NULL) errorQuda("hipMalloc failed for dev_pointer\n");

  GaugeFixQualityArg<Gauge> argQ(dataOr, data);
  GaugeFixQuality<Float,Gauge, gauge_dir> GaugeFixQuality(argQ);


  GaugeFixArg<Float, Gauge> arg(dataOr, data, relax_boost);
  GaugeFix<Float,Gauge, gauge_dir> gaugeFix(arg);





#ifdef MULTI_GPU
  void *send[4];
  void *recv[4];
  void *sendg[4];
  void *recvg[4];
  void *send_d[4];
  void *recv_d[4];
  void *sendg_d[4];
  void *recvg_d[4];
  void *hostbuffer_h[4];
  hipStream_t GFStream[9];
  size_t offset[4];
  size_t bytes[4];
  size_t faceVolume[4];
  size_t faceVolumeCB[4];
  // do the exchange
  MsgHandle *mh_recv_back[4];
  MsgHandle *mh_recv_fwd[4];
  MsgHandle *mh_send_fwd[4];
  MsgHandle *mh_send_back[4];
  int X[4];
  dim3 block[4];
  dim3 grid[4];

  if(comm_size() != 1){

    for(int dir=0; dir<4; ++dir){
      X[dir] = data.X()[dir] - data.R()[dir]*2;
      if (!commDimPartitioned(dir) && data.R()[dir] != 0) errorQuda("Not supported!\n");
    }
    for (int i=0; i<4; i++) {
      faceVolume[i] = 1;
      for (int j=0; j<4; j++) {
        if (i==j) continue;
        faceVolume[i] *= X[j];
      }
      faceVolumeCB[i] = faceVolume[i]/2;
    }

    for (int d=0; d<4; d++) {
      if (!commDimPartitioned(d)) continue;
      offset[d] = faceVolumeCB[d] * NElems;
      bytes[d] =  sizeof(Float) * offset[d];
      send_d[d] = device_malloc(bytes[d]);
      recv_d[d] = device_malloc(bytes[d]);
      sendg_d[d] = device_malloc(bytes[d]);
      recvg_d[d] = device_malloc(bytes[d]);
      hipStreamCreate(&GFStream[d]);
      hipStreamCreate(&GFStream[4 + d]);
      #ifndef GPU_COMMS
      hostbuffer_h[d] = (void*)pinned_malloc(4*bytes[d]);
      #endif
      block[d] = make_uint3(128, 1, 1);
      grid[d] = make_uint3((faceVolumeCB[d] + block[d].x - 1) / block[d].x, 1, 1);
    }
    hipStreamCreate(&GFStream[8]);
    for (int d=0; d<4; d++) {
      if (!commDimPartitioned(d)) continue;
      #ifdef GPU_COMMS
      recv[d] = recv_d[d];
      send[d] = send_d[d];
      recvg[d] = recvg_d[d];
      sendg[d] = sendg_d[d];
      #else
      recv[d] = hostbuffer_h[d];
      send[d] = static_cast<char*>(hostbuffer_h[d]) + bytes[d];
      recvg[d] = static_cast<char*>(hostbuffer_h[d]) + 3*bytes[d];
      sendg[d] = static_cast<char*>(hostbuffer_h[d]) + 2*bytes[d];      
      #endif
      mh_recv_back[d] = comm_declare_receive_relative(recv[d], d, -1, bytes[d]);
      mh_recv_fwd[d]  = comm_declare_receive_relative(recvg[d], d, +1, bytes[d]);
      mh_send_back[d] = comm_declare_send_relative(sendg[d], d, -1, bytes[d]);
      mh_send_fwd[d]  = comm_declare_send_relative(send[d], d, +1, bytes[d]);
    }
  }
  GaugeFixUnPackArg<Gauge> dataexarg(dataOr, data);
  GaugeFixBorderPointsArg<Float, Gauge> argBorder(dataOr, data, relax_boost, faceVolume, faceVolumeCB);
  GaugeFixBorderPoints<Float,Gauge, gauge_dir> gfixBorderPoints(argBorder);
  GaugeFixInteriorPointsArg<Float, Gauge> argInt(dataOr, data, relax_boost);
  GaugeFixInteriorPoints<Float,Gauge, gauge_dir> gfixIntPoints(argInt);
  #endif

  GaugeFixQuality.apply(0);
  flop += (double)GaugeFixQuality.flops();
  byte += (double)GaugeFixQuality.bytes();
  double action0 = argQ.getAction();
  printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());


unitarizeLinksQuda(data, num_failures_dev);
      hipMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
      if(num_failures>0){
        hipFree(num_failures_dev); 
        errorQuda("Error in the unitarization\n"); 
        exit(1);
      }
      hipMemset(num_failures_dev, 0, sizeof(int));

  int iter = 0;
  for(iter = 0; iter < Nsteps; iter++){
    for(int p = 0; p < 2; p++){
      #ifndef MULTI_GPU      
        gaugeFix.setParity(p);
        gaugeFix.apply(0);
        flop += (double)gaugeFix.flops();
        byte += (double)gaugeFix.bytes();
      #else
      if(comm_size() == 1){
        gaugeFix.setParity(p);
        gaugeFix.apply(0);
        flop += (double)gaugeFix.flops();
        byte += (double)gaugeFix.bytes();
      }
      else{
        gfixIntPoints.setParity(p);
        gfixBorderPoints.setParity(p);//compute border points
        gfixBorderPoints.apply(0);
        flop += (double)gfixBorderPoints.flops();
        byte += (double)gfixBorderPoints.bytes();
        flop += (double)gfixIntPoints.flops();
        byte += (double)gfixIntPoints.bytes();
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_start(mh_recv_back[d]);  
          comm_start(mh_recv_fwd[d]);  
        }   
        //wait for the update to the halo points before start packing...
        hipDeviceSynchronize();
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          //extract top face
          Kernel_UnPackTop<NElems, Float, Gauge, true><<<grid[d], block[d], 0, GFStream[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<typename ComplexTypeId<Float>::Type*>(send_d[d]), p, d, d);
          //extract bottom ghost
          Kernel_UnPackGhost<NElems, Float, Gauge, true><<<grid[d], block[d], 0, GFStream[4+d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<typename ComplexTypeId<Float>::Type*>(sendg_d[d]), 1-p, d, d);
        }  
        #ifdef GPU_COMMS
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          hipStreamSynchronize(GFStream[d]);
          comm_start(mh_send_fwd[d]);
          hipStreamSynchronize(GFStream[4+d]);
          comm_start(mh_send_back[d]);
        }   
        #else
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          hipMemcpyAsync(send[d], send_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[d]);
        }
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          hipMemcpyAsync(sendg[d], sendg_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[4+d]);
        }    
        #endif
        //compute interior points
        gfixIntPoints.apply(GFStream[8]);

        #ifndef GPU_COMMS
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          hipStreamSynchronize(GFStream[d]);
          comm_start(mh_send_fwd[d]);
          hipStreamSynchronize(GFStream[4+d]);
          comm_start(mh_send_back[d]);
        }
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_wait(mh_recv_back[d]);
          hipMemcpyAsync(recv_d[d], recv[d], bytes[d], hipMemcpyHostToDevice, GFStream[d]);
        }
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_wait(mh_recv_fwd[d]);
          hipMemcpyAsync(recvg_d[d], recvg[d], bytes[d], hipMemcpyHostToDevice, GFStream[4 + d]);
        }
        #endif
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
          comm_wait(mh_recv_back[d]);
          #endif
          Kernel_UnPackGhost<NElems, Float, Gauge, false><<<grid[d], block[d], 0, GFStream[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<typename ComplexTypeId<Float>::Type*>(recv_d[d]), p, d, d);
        }
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
          comm_wait(mh_recv_fwd[d]);
          #endif
          Kernel_UnPackTop<NElems, Float, Gauge, false><<<grid[d], block[d], 0, GFStream[4 + d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<typename ComplexTypeId<Float>::Type*>(recvg_d[d]), 1-p, d, d); 
        }
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_wait(mh_send_back[d]);
          comm_wait(mh_send_fwd[d]);
          hipStreamSynchronize(GFStream[d]);
          hipStreamSynchronize(GFStream[4+d]);
        }
        hipStreamSynchronize(GFStream[8]);
      }
      #endif 
      /*gaugeFix.setParity(p);
      gaugeFix.apply(0);
      flop += (double)gaugeFix.flops();
      byte += (double)gaugeFix.bytes();
      #ifdef MULTI_GPU
      if(comm_size() != 1){//exchange updated top face links in current parity
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_start(mh_recv_back[d]);      
          //extract top face
          Kernel_UnPackTop<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(send_d[d]), p, d, d, true);
          #ifndef GPU_COMMS
          hipMemcpy(send[d], send_d[d], bytes[d], hipMemcpyDeviceToHost);
          #else
          hipDeviceSynchronize();
          #endif
          comm_start(mh_send_fwd[d]);
          comm_wait(mh_recv_back[d]);
          comm_wait(mh_send_fwd[d]);
          #ifndef GPU_COMMS
          hipMemcpy(recv_d[d], recv[d], bytes[d], hipMemcpyHostToDevice);
          #endif
          //inject top face in ghost
          Kernel_UnPackGhost<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(recv_d[d]), p, d, d, false);
        }
        //exchange updated ghost links in opposite parity
        for (int d=0; d<4; d++) {
          if (!commDimPartitioned(d)) continue;
          comm_start(mh_recv_fwd[d]);  
          Kernel_UnPackGhost<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(sendg_d[d]), 1-p, d, d, true); 
          #ifndef GPU_COMMS
          hipMemcpy(sendg[d], sendg_d[d], bytes[d], hipMemcpyDeviceToHost);
          #else
          hipDeviceSynchronize();
          #endif
          comm_start(mh_send_back[d]);
          comm_wait(mh_recv_fwd[d]);
          comm_wait(mh_send_back[d]);
          #ifndef GPU_COMMS
          hipMemcpy(recvg_d[d], recvg[d], bytes[d], hipMemcpyHostToDevice);
          #endif
          Kernel_UnPackTop<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(recvg_d[d]), 1-p, d, d, false);
        }
      }
      #endif*/
    }
    if((iter % reunit_interval) == (reunit_interval - 1)) {
      unitarizeLinksQuda(data, num_failures_dev);
      hipMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
      if(num_failures>0){
        hipFree(num_failures_dev); 
        errorQuda("Error in the unitarization\n"); 
        exit(1);
      }
      hipMemset(num_failures_dev, 0, sizeof(int));
      //flop += (double)????????????????????????????????????????????????
      //byte += (double)????????????????????????????????????????????????
    }
    GaugeFixQuality.apply(0);
    flop += (double)GaugeFixQuality.flops();
    byte += (double)GaugeFixQuality.bytes();
    double action = argQ.getAction();
    double diff = abs(action0 - action);
    if((iter % verbose_interval) == (verbose_interval - 1))
    printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter+1, argQ.getAction(), argQ.getTheta(), diff);
    if(stopWtheta){
      if(argQ.getTheta() < tolerance) break;
    }
    else{
      if(diff < tolerance) break;
    } 
    action0 = action;
  }
  if((iter % reunit_interval) != 0)  {
    unitarizeLinksQuda(data, num_failures_dev);
    hipMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
    if(num_failures>0){
      hipFree(num_failures_dev); 
      errorQuda("Error in the unitarization\n"); 
      exit(1);
    }
    hipMemset(num_failures_dev, 0, sizeof(int));
    //flop += (double)????????????????????????????????????????????????
    //byte += (double)????????????????????????????????????????????????
  }
  if((iter % verbose_interval) != 0){
    GaugeFixQuality.apply(0);
    flop += (double)GaugeFixQuality.flops();
    byte += (double)GaugeFixQuality.bytes();
    double action = argQ.getAction();
    double diff = abs(action0 - action);
    printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter+1, argQ.getAction(), argQ.getTheta(), diff);
  }
  hipFree(num_failures_dev); 
  #ifdef MULTI_GPU
  if(comm_size() != 1){
    for (int d=0; d<4; d++) {
      if (commDimPartitioned(d)) {
        comm_free(mh_send_fwd[d]);
        comm_free(mh_send_back[d]);
        comm_free(mh_recv_back[d]);
        comm_free(mh_recv_fwd[d]);
        device_free(send_d[d]);
        device_free(recv_d[d]);
        device_free(sendg_d[d]);
        device_free(recvg_d[d]);
        hipStreamDestroy(GFStream[d]);
        hipStreamDestroy(GFStream[4 + d]);
        #ifndef GPU_COMMS
        free(hostbuffer_h[d]);
        #endif
      }
    }
    hipStreamDestroy(GFStream[8]);
  }
  #endif
  checkCudaError();
  hipDeviceSynchronize();
  profileGaugeFix.Stop(QUDA_PROFILE_COMPUTE);
  double secs = profileGaugeFix.Last(QUDA_PROFILE_COMPUTE);
  double gflops = (flop*1e-9)/(secs);
  double gbytes = byte/(secs*1e9);
  #ifdef MULTI_GPU
  printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops*comm_size(), gbytes*comm_size());
  #else
  printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
  #endif
  printfQuda("Reunitarization flops and bandwidth not accounted!!!!!!\n");
}

template<typename Float, int NElems, typename Gauge>
void gaugefixingOVR( Gauge dataOr,  cudaGaugeField& data, const unsigned int gauge_dir, \
  const unsigned int Nsteps, const unsigned int verbose_interval, \
  const Float relax_boost, const double tolerance, const unsigned int reunit_interval, const unsigned int stopWtheta) {
  if( gauge_dir !=3 ){
    printfQuda("Starting Landau gauge fixing...\n");
    gaugefixingOVR<Float, Gauge, NElems, 4>(dataOr, data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
  } 
  else {        
    printfQuda("Starting Coulomb gauge fixing...\n");
    gaugefixingOVR<Float, Gauge, NElems, 3>(dataOr, data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
  }
}



template<typename Float>
void gaugefixingOVR( cudaGaugeField& data, const unsigned int gauge_dir, \
  const unsigned int Nsteps, const unsigned int verbose_interval, const Float relax_boost, const double tolerance, \
  const unsigned int reunit_interval, const unsigned int stopWtheta) {

  // Switching to FloatNOrder for the gauge field in order to support RECONSTRUCT_12
  if(data.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
    if(data.Reconstruct() == QUDA_RECONSTRUCT_NO) {
    printfQuda("QUDA_RECONSTRUCT_NO\n");
      numParams = 18;
      gaugefixingOVR<Float, 18>(FloatNOrder<Float, 18, 2, 18>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else if(data.Reconstruct() == QUDA_RECONSTRUCT_12){
    printfQuda("QUDA_RECONSTRUCT_12\n");
      numParams = 12;
      gaugefixingOVR<Float, 12>(FloatNOrder<Float, 18, 2, 12>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    
    } else if(data.Reconstruct() == QUDA_RECONSTRUCT_8){
    printfQuda("QUDA_RECONSTRUCT_8\n");
      numParams = 8;
      gaugefixingOVR<Float, 8>(FloatNOrder<Float, 18, 2,  8>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    
    } else {
      errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
    }
  } else if(data.Order() == QUDA_FLOAT4_GAUGE_ORDER) {
    if(data.Reconstruct() == QUDA_RECONSTRUCT_NO) {
    printfQuda("QUDA_RECONSTRUCT_NO\n");
      numParams = 18;
      gaugefixingOVR<Float, 18>(FloatNOrder<Float, 18, 4, 18>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else if(data.Reconstruct() == QUDA_RECONSTRUCT_12){
    printfQuda("QUDA_RECONSTRUCT_12\n");
      numParams = 12;
      gaugefixingOVR<Float, 12>(FloatNOrder<Float, 18, 4, 12>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else if(data.Reconstruct() == QUDA_RECONSTRUCT_8){
    printfQuda("QUDA_RECONSTRUCT_8\n");
      numParams = 8;
      gaugefixingOVR<Float, 8>(FloatNOrder<Float, 18, 4,  8>(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else {
      errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
    }
  } else {
    errorQuda("Invalid Gauge Order\n");
  }
}

  void gaugefixingOVR( cudaGaugeField& data, const unsigned int gauge_dir, \
    const unsigned int Nsteps, const unsigned int verbose_interval, const double relax_boost, \
    const double tolerance, const unsigned int reunit_interval, const unsigned int stopWtheta) {

    if(data.Precision() == QUDA_HALF_PRECISION) {
      errorQuda("Half precision not supported\n");
    }
    if (data.Precision() == QUDA_SINGLE_PRECISION) {
      gaugefixingOVR<float> (data, gauge_dir, Nsteps, verbose_interval, (float)relax_boost, tolerance, reunit_interval, stopWtheta);
    } else if(data.Precision() == QUDA_DOUBLE_PRECISION) {
      gaugefixingOVR<double>(data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
  }


} //namespace quda
