#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <unitarization_links.h>
#include <comm_quda.h>
#include <gauge_fix_ovr_extra.h>
#include <gauge_fix_ovr_hit_devf.cuh>
#include <cub_helper.cuh>
#include <index_helper.cuh>

namespace quda {

#ifdef GPU_GAUGE_ALG

  static int numParams = 18;

#define LAUNCH_KERNEL_GAUGEFIX(kernel, tp, stream, arg, parity, ...)                                                   \
  if (tp.aux.x == 0) {                                                                                                 \
    switch (tp.block.x) {                                                                                              \
    case 256: kernel<0, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<0, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 768: kernel<0, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 1024: kernel<0, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 1) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 256: kernel<1, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<1, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 768: kernel<1, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 1024: kernel<1, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 2) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 256: kernel<2, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<2, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 768: kernel<2, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 1024: kernel<2, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 3) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: kernel<3, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 256: kernel<3, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 384: kernel<3, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<3, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 640: kernel<3, 160, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 768: kernel<3, 192, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 896: kernel<3, 224, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 1024: kernel<3, 256, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 4) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: kernel<4, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 256: kernel<4, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 384: kernel<4, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<4, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 640: kernel<4, 160, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 768: kernel<4, 192, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 896: kernel<4, 224, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 1024: kernel<4, 256, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 5) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: kernel<5, 32, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 256: kernel<5, 64, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 384: kernel<5, 96, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;      \
    case 512: kernel<5, 128, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 640: kernel<5, 160, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 768: kernel<5, 192, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 896: kernel<5, 224, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;     \
    case 1024: kernel<5, 256, __VA_ARGS__><<<tp.grid.x, tp.block.x, tp.shared_bytes, stream>>>(arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else {                                                                                                             \
    errorQuda("Not implemented for %d", tp.aux.x);                                                                     \
  }

  /**
   * @brief container to pass parameters for the gauge fixing quality kernel
   */
  template <typename Gauge>
  struct GaugeFixQualityArg : public ReduceArg<double2> {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    GaugeFixQualityArg(const Gauge &dataOr, const cudaGaugeField &data)
      : ReduceArg<double2>(), dataOr(dataOr) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
      threads = X[0]*X[1]*X[2]*X[3]/2;
    }
    double getAction(){ return result_h[0].x; }
    double getTheta(){ return result_h[0].y; }
  };


  /**
   * @brief Measure gauge fixing quality
   */
  template<int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFix_quality(GaugeFixQualityArg<Gauge> argQ){
    typedef complex<Float> Cmplx;

    int idx_cb = threadIdx.x + blockIdx.x * blockDim.x;
    int parity = threadIdx.y;

    double2 data = make_double2(0.0,0.0);
    while (idx_cb < argQ.threads) {
      int X[4];
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = argQ.X[dr];

      int x[4];
      getCoords(x, idx_cb, X, parity);
#ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += argQ.border[dr];
        X[dr] += 2 * argQ.border[dr];
      }
#endif
      Matrix<Cmplx,3> delta;
      setZero(&delta);
      //load upward links
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, linkIndex(x, X), parity);
        delta -= U;
      }
      //18*gauge_dir
      data.x += -delta(0, 0).x - delta(1, 1).x - delta(2, 2).x;
      //2
      //load downward links
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, linkIndexM1(x,X,mu), 1 - parity);
        delta += U;
      }
      //18*gauge_dir
      delta -= conj(delta);
      //18
      SubTraceUnit(delta);
      //12
      data.y += getRealTraceUVdagger(delta, delta);
      //35
      //T=36*gauge_dir+65

      idx_cb += blockDim.x * gridDim.x;
    }
    reduce2d<blockSize,2>(argQ, data);
  }


  /**
   * @brief Tunable object for the gauge fixing quality kernel
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixQuality : TunableLocalParity {
    GaugeFixQualityArg<Gauge> argQ;
    mutable char aux_string[128]; // used as a label in the autotuner

  private:
    bool tuneGridDim() const { return true; }

  public:
    GaugeFixQuality(GaugeFixQualityArg<Gauge> &argQ) : argQ(argQ) { }
    ~GaugeFixQuality () { }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      argQ.result_h[0] = make_double2(0.0,0.0);
      LAUNCH_KERNEL_LOCAL_PARITY(computeFix_quality, (*this), tp, stream, argQ, Float, Gauge, gauge_dir);
      qudaDeviceSynchronize();
      if ( comm_size() != 1 ) comm_allreduce_array((double*)argQ.result_h, 2);
      argQ.result_h[0].x  /= (double)(3 * gauge_dir * 2 * argQ.threads * comm_size());
      argQ.result_h[0].y  /= (double)(3 * 2 * argQ.threads * comm_size());
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << argQ.X[0] << "x";
      vol << argQ.X[1] << "x";
      vol << argQ.X[2] << "x";
      vol << argQ.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu,gaugedir=%d",argQ.threads, sizeof(Float),gauge_dir);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);

    }

    long long flops() const {
      return (36LL * gauge_dir + 65LL) * 2 * argQ.threads;
    }                                                                   // Only correct if there is no link reconstruction, no cub reduction accounted also
    //long long bytes() const { return (1)*2*gauge_dir*argQ.dataOr.Bytes(); }//no accounting the reduction!!!! argQ.dataOr.Bytes() return 0....
    long long bytes() const {
      return 2LL * gauge_dir * 2 * argQ.threads * numParams * sizeof(Float);
    }                                                                                   //no accounting the reduction!!!!

  };


  /**
   * @brief container to pass parameters for the gauge fixing kernel
   */
  template <typename Float, typename Gauge>
  struct GaugeFixArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    cudaGaugeField &data;
    const Float relax_boost;

    GaugeFixArg(Gauge & dataOr, cudaGaugeField & data, const Float relax_boost)
      : dataOr(dataOr), data(data), relax_boost(relax_boost) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
    }
  };




  /**
   * @brief Kernel to perform gauge fixing with overrelaxation for single-GPU
   */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFix(GaugeFixArg<Float, Gauge> arg, int parity){
    typedef complex<Float> Cmplx;

    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;

    if ( idx >= arg.threads ) return;

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      int X[4];
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += arg.border[dr];
        X[dr] += 2 * arg.border[dr];
      }
  #endif
      int mu = (threadIdx.x / blockSize);
      int oddbit = parity;
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        oddbit = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, oddbit);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, oddbit) = link;
    }
    // 4 threads per lattice site
    else{
      int X[4];
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += arg.border[dr];
        X[dr] += 2 * arg.border[dr];
      }
  #endif
      int mu = (threadIdx.x / blockSize);
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      //load upward link
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);

      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      //load downward link
      Matrix<Cmplx,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;

    }
  }


  /**
   * @brief Tunable object for the gauge fixing kernel
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFix : Tunable {
    GaugeFixArg<Float, Gauge> arg;
    int parity;
    mutable char aux_string[128]; // used as a label in the autotuner
protected:
    dim3 createGrid(const TuneParam &param) const
    {
      unsigned int blockx = param.block.x / 8;
      if (param.aux.x > 2) blockx = param.block.x / 4;
      unsigned int gx  = (arg.threads + blockx - 1) / blockx;
      return dim3(gx, 1, 1);
    }

    bool advanceBlockDim  (TuneParam &param) const {
      // Use param.aux.x to tune and save state for best kernel option
      // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
      const unsigned int min_threads0 = 32 * 8;
      const unsigned int min_threads1 = 32 * 4;
      const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int atmadd = 0;
      unsigned int min_threads = min_threads0;
      param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
      if (param.aux.x > 2) min_threads = 32 * 4;
      param.block.x += min_threads;
      param.block.y = 1;
      param.grid = createGrid(param);

      if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
        param.shared_bytes = sharedBytesPerBlock(param);
        return true;
      } else if (param.aux.x == 0) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 1; // USE FOR ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 1) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 2) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 3; // USE FOR NO ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float);
        return true;
      } else if (param.aux.x == 3) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 4;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else if (param.aux.x == 4) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 5;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else {
        return false;
      }
    }

private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const {
      return false;
    }                                            // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

public:
    GaugeFix(GaugeFixArg<Float, Gauge> &arg) : arg(arg), parity(0) { }
    ~GaugeFix () { }

    void setParity(const int par){
      parity = par;
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFix, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const {
      initTuneParam(param);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    //need this
    void preTune() {
      arg.data.backup();
    }
    void postTune() {
      arg.data.restore();
    }
    long long flops() const {
      return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads;
    }                                                                                  // Only correct if there is no link reconstruction
    //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
    long long bytes() const {
      return 8LL * 2 * arg.threads * numParams * sizeof(Float);
    }                                                                          //no accounting the reduction!!!!
  };




#ifdef MULTI_GPU
  template <typename Float, typename Gauge>
  struct GaugeFixInteriorPointsArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    cudaGaugeField &data;
    const Float relax_boost;
    GaugeFixInteriorPointsArg(Gauge & dataOr, cudaGaugeField & data, const Float relax_boost)
      : dataOr(dataOr), data(data), relax_boost(relax_boost) {

#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        if ( comm_dim_partitioned(dir)) border[dir] = data.R()[dir] + 1;  //skip BORDER_RADIUS + face border point
        else border[dir] = 0;
      }
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir] - border[dir] * 2;
#else
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
#endif
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
    }
  };




  /**
   * @brief Kernel to perform gauge fixing with overrelaxation in the interior points for multi-GPU implementation
   */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> arg, int parity){
    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;
    if ( idx >= arg.threads ) return;
    typedef complex<Float> Complex;
    int X[4];
#pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
#ifdef MULTI_GPU
    int za = (idx / (X[0] / 2));
    int zb =  (za / X[1]);
    x[1] = za - zb * X[1];
    x[3] = (zb / X[2]);
    x[2] = zb - x[3] * X[2];
    int p = 0; for ( int dr = 0; dr < 4; ++dr ) p += arg.border[dr];
    p = p & 1;
    int x1odd = (x[1] + x[2] + x[3] + parity + p) & 1;
    //int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
    x[0] = (2 * idx + x1odd)  - za * X[0];
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
#else
    getCoords(x, idx, X, parity);
#endif
    int mu = (threadIdx.x / blockSize);

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        parity = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link = arg.dataOr(mu, idx, parity);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, parity) = link;
    }
    // 4 threads per lattice site
    else{
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link = arg.dataOr(mu, idx, parity);


      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;
    }
  }

  /**
   * @brief Tunable object for the interior points of the gauge fixing
   * kernel in multi-GPU implementation
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixInteriorPoints : Tunable {
    GaugeFixInteriorPointsArg<Float, Gauge> arg;
    int parity;
    mutable char aux_string[128]; // used as a label in the autotuner
protected:
    dim3 createGrid(const TuneParam &param) const
    {
      unsigned int blockx = param.block.x / 8;
      if (param.aux.x > 2) blockx = param.block.x / 4;
      unsigned int gx  = (arg.threads + blockx - 1) / blockx;
      return dim3(gx, 1, 1);
    }

    bool advanceBlockDim  (TuneParam &param) const {
      // Use param.aux.x to tune and save state for best kernel option
      // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
      const unsigned int min_threads0 = 32 * 8;
      const unsigned int min_threads1 = 32 * 4;
      const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int atmadd = 0;
      unsigned int min_threads = min_threads0;
      param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
      if (param.aux.x > 2) min_threads = 32 * 4;
      param.block.x += min_threads;
      param.block.y = 1;
      param.grid = createGrid(param);

      if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
        param.shared_bytes = sharedBytesPerBlock(param);
        return true;
      } else if (param.aux.x == 0) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 1; // USE FOR ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 1) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 2) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 3; // USE FOR NO ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float);
        return true;
      } else if (param.aux.x == 3) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 4;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else if (param.aux.x == 4) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 5;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else {
        return false;
      }
    }

private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const {
      return false;
    }                                            // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

public:
    GaugeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> &arg) : arg(arg), parity(0) {}

    ~GaugeFixInteriorPoints () { }

    void setParity(const int par) { parity = par; }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFixInteriorPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    //need this
    void preTune() {
      arg.data.backup();
    }
    void postTune() {
      arg.data.restore();
    }
    long long flops() const {
      return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads;
    }                                                                                  // Only correct if there is no link reconstruction
    //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
    long long bytes() const {
      return 8LL * 2 * arg.threads * numParams * sizeof(Float);
    }                                                                           // Only correct if there is no link reconstruction load+save
  };


  template <typename Float, typename Gauge>
  struct GaugeFixBorderPointsArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    int border[4];
    int *borderpoints[2];
    int *faceindicessize[2];
    size_t faceVolume[4];
    size_t faceVolumeCB[4];
    Gauge dataOr;
    cudaGaugeField &data;
    const Float relax_boost;

    GaugeFixBorderPointsArg(Gauge & dataOr, cudaGaugeField & data, const Float relax_boost, size_t faceVolume_[4], size_t faceVolumeCB_[4])
      : dataOr(dataOr), data(data), relax_boost(relax_boost) {


      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
        border[dir] = data.R()[dir];
      }

      /*for(int dir=0; dir<4; ++dir){
         if(comm_dim_partitioned(dir)) border[dir] = BORDER_RADIUS;
         else border[dir] = 0;
         }
         for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir] - border[dir]*2;*/
      for ( int dir = 0; dir < 4; ++dir ) {
        faceVolume[dir] = faceVolume_[dir];
        faceVolumeCB[dir] = faceVolumeCB_[dir];
      }
      if ( comm_partitioned() ) PreCalculateLatticeIndices(faceVolume, faceVolumeCB, X, border, threads, borderpoints);
    }
  };

  /**
   * @brief Kernel to perform gauge fixing with overrelaxation in the border points for multi-GPU implementation
  */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> arg, int parity){
    typedef complex<Float> Cmplx;

    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;
    if ( idx >= arg.threads ) return;
    int mu = (threadIdx.x / blockSize);
    idx = arg.borderpoints[parity][idx];
    int X[4], x[4];
    x[3] = idx / (arg.X[0] * arg.X[1]  * arg.X[2]);
    x[2] = (idx / (arg.X[0] * arg.X[1])) % arg.X[2];
    x[1] = (idx / arg.X[0]) % arg.X[1];
    x[0] = idx % arg.X[0];
  #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) x[dr] += arg.border[dr];
  #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr] + 2 * arg.border[dr];

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        parity = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, parity) = link;
    }
    // 4 threads per lattice site
    else{
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);


      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;
    }
  }




  /**
   * @brief Tunable object for the border points of the gauge fixing kernel in multi-GPU implementation
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixBorderPoints : Tunable {
    GaugeFixBorderPointsArg<Float, Gauge> arg;
    int parity;
    mutable char aux_string[128]; // used as a label in the autotuner
    protected:
        dim3 createGrid(const TuneParam &param) const
        {
          unsigned int blockx = param.block.x / 8;
          if (param.aux.x > 2) blockx = param.block.x / 4;
          unsigned int gx = (arg.threads + blockx - 1) / blockx;
          return dim3(gx, 1, 1);
        }

        bool advanceBlockDim(TuneParam &param) const
        {
          // Use param.aux.x to tune and save state for best kernel option
          // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
          const unsigned int min_threads0 = 32 * 8;
          const unsigned int min_threads1 = 32 * 4;
          const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
          const unsigned int atmadd = 0;
          unsigned int min_threads = min_threads0;
          param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
          if (param.aux.x > 2) min_threads = 32 * 4;
          param.block.x += min_threads;
          param.block.y = 1;
          param.grid = createGrid(param);

          if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
            param.shared_bytes = sharedBytesPerBlock(param);
            return true;
          } else if (param.aux.x == 0) {
            param.block.x = min_threads0;
            param.block.y = 1;
            param.aux.x = 1; // USE FOR ATOMIC ADD
            param.grid = createGrid(param);
            param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
            return true;
          } else if (param.aux.x == 1) {
            param.block.x = min_threads0;
            param.block.y = 1;
            param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
            param.grid = createGrid(param);
            param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
            return true;
          } else if (param.aux.x == 2) {
            param.block.x = min_threads1;
            param.block.y = 1;
            param.aux.x = 3; // USE FOR NO ATOMIC ADD
            param.grid = createGrid(param);
            param.shared_bytes = param.block.x * 4 * sizeof(Float);
            return true;
          } else if (param.aux.x == 3) {
            param.block.x = min_threads1;
            param.block.y = 1;
            param.aux.x = 4;
            param.grid = createGrid(param);
            param.shared_bytes = param.block.x * sizeof(Float);
            return true;
          } else if (param.aux.x == 4) {
            param.block.x = min_threads1;
            param.block.y = 1;
            param.aux.x = 5;
            param.grid = createGrid(param);
            param.shared_bytes = param.block.x * sizeof(Float);
            return true;
          } else {
            return false;
          }
        }

    private:
    unsigned int sharedBytesPerThread() const {
      return 0;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const {
      return false;
    }                                            // Don't tune shared memory
    bool tuneGridDim() const {
      return false;
    }                                        // Don't tune the grid dimensions.
    unsigned int minThreads() const {
      return arg.threads;
    }

public:
    GaugeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> &arg) : arg(arg), parity(0) { }
    ~GaugeFixBorderPoints () {
      if ( comm_partitioned() ) for ( int i = 0; i < 2; i++ ) pool_device_free(arg.borderpoints[i]);
    }
    void setParity(const int par){
      parity = par;
    }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFixBorderPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const {
      initTuneParam(param);
    }

    TuneKey tuneKey() const {
      std::stringstream vol;
      vol << arg.X[0] << "x";
      vol << arg.X[1] << "x";
      vol << arg.X[2] << "x";
      vol << arg.X[3];
      sprintf(aux_string,"threads=%d,prec=%lu,gaugedir=%d",arg.threads,sizeof(Float),gauge_dir);
      return TuneKey(vol.str().c_str(), typeid(*this).name(), aux_string);
    }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    //need this
    void preTune() {
      arg.data.backup();
    }
    void postTune() {
      arg.data.restore();
    }
    long long flops() const {
      return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads;
    }                                                                                  // Only correct if there is no link reconstruction
    //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
    long long bytes() const {
      return 8LL * 2 * arg.threads * numParams * sizeof(Float);
    }                                                                           // Only correct if there is no link reconstruction load+save

  };














  template <typename Gauge>
  struct GaugeFixUnPackArg {
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    GaugeFixUnPackArg(Gauge & dataOr, cudaGaugeField & data)
      : dataOr(dataOr) {
      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
    }
  };


  template<int NElems, typename Float, typename Gauge, bool pack>
  __global__ void Kernel_UnPackGhost(int size, GaugeFixUnPackArg<Gauge> arg, complex<Float> *array, int parity, int face, int dir){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= size ) return;
    int X[4];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
    int za, xodd;
    int borderid = 0;
    parity = 1 - parity;
    switch ( face ) {
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
      break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    }
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
    x[face] -= 1;
    parity = 1 - parity;
    int id = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
    typedef complex<Float> Cmplx;
    typedef typename mapper<Float>::type RegType;
    RegType tmp[NElems];
    Cmplx data[9];
    if ( pack ) {
      arg.dataOr.load(data, id, dir, parity);
      arg.dataOr.reconstruct.Pack(tmp, data, id);
      for ( int i = 0; i < NElems / 2; ++i ) {
        array[idx + size * i] = Cmplx(tmp[2*i+0], tmp[2*i+1]);
      }
    } else {
      for ( int i = 0; i < NElems / 2; ++i ) {
        tmp[2*i+0] = array[idx + size * i].real();
        tmp[2*i+1] = array[idx + size * i].imag();
      }
      arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0, arg.dataOr.X, arg.dataOr.R);
      arg.dataOr.save(data, id, dir, parity);
    }
  }


  template<int NElems, typename Float, typename Gauge, bool pack>
  __global__ void Kernel_UnPackTop(int size, GaugeFixUnPackArg<Gauge> arg, complex<Float> *array, int parity, int face, int dir){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= size ) return;
    int X[4];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
    int za, xodd;
    int borderid = arg.X[face] - 1;
    switch ( face ) {
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
      break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    }
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
    int id = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
    typedef complex<Float> Cmplx;
    typedef typename mapper<Float>::type RegType;
    RegType tmp[NElems];
    Cmplx data[9];
    if ( pack ) {
      arg.dataOr.load(data, id, dir, parity);
      arg.dataOr.reconstruct.Pack(tmp, data, id);
      for ( int i = 0; i < NElems / 2; ++i ) array[idx + size * i] = Cmplx(tmp[2*i+0], tmp[2*i+1]);
    }
    else{
      for ( int i = 0; i < NElems / 2; ++i ) {
        tmp[2*i+0] = array[idx + size * i].real();
        tmp[2*i+1] = array[idx + size * i].imag();
      }
      arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0, arg.dataOr.X, arg.dataOr.R);
      arg.dataOr.save(data, id, dir, parity);
    }
  }
#endif


  template<typename Float, typename Gauge, int NElems, int gauge_dir>
  void gaugefixingOVR( Gauge dataOr,  cudaGaugeField& data,
		       const int Nsteps, const int verbose_interval,
		       const Float relax_boost, const double tolerance,
		       const int reunit_interval, const int stopWtheta) {


    TimeProfile profileInternalGaugeFixOVR("InternalGaugeFixQudaOVR", false);

    profileInternalGaugeFixOVR.TPSTART(QUDA_PROFILE_COMPUTE);
    double flop = 0;
    double byte = 0;

    printfQuda("\tOverrelaxation boost parameter: %lf\n", (double)relax_boost);
    printfQuda("\tStop criterium: %lf\n", tolerance);
    if ( stopWtheta ) printfQuda("\tStop criterium method: theta\n");
    else printfQuda("\tStop criterium method: Delta\n");
    printfQuda("\tMaximum number of iterations: %d\n", Nsteps);
    printfQuda("\tReunitarize at every %d steps\n", reunit_interval);
    printfQuda("\tPrint convergence results at every %d steps\n", verbose_interval);


    const double unitarize_eps = 1e-14;
    const double max_error = 1e-10;
    const int reunit_allow_svd = 1;
    const int reunit_svd_only  = 0;
    const double svd_rel_error = 1e-6;
    const double svd_abs_error = 1e-6;
    setUnitarizeLinksConstants(unitarize_eps, max_error,
                               reunit_allow_svd, reunit_svd_only,
                               svd_rel_error, svd_abs_error);
    int num_failures = 0;
    int* num_failures_dev = static_cast<int*>(pool_device_malloc(sizeof(int)));
    hipMemset(num_failures_dev, 0, sizeof(int));

    GaugeFixQualityArg<Gauge> argQ(dataOr, data);
    GaugeFixQuality<Float,Gauge, gauge_dir> GaugeFixQuality(argQ);

    GaugeFixArg<Float, Gauge> arg(dataOr, data, relax_boost);
    GaugeFix<Float,Gauge, gauge_dir> gaugeFix(arg);

#ifdef MULTI_GPU
    void *send[4];
    void *recv[4];
    void *sendg[4];
    void *recvg[4];
    void *send_d[4];
    void *recv_d[4];
    void *sendg_d[4];
    void *recvg_d[4];
    void *hostbuffer_h[4];
    qudaStream_t GFStream[9];
    size_t offset[4];
    size_t bytes[4];
    size_t faceVolume[4];
    size_t faceVolumeCB[4];
    // do the exchange
    MsgHandle *mh_recv_back[4];
    MsgHandle *mh_recv_fwd[4];
    MsgHandle *mh_send_fwd[4];
    MsgHandle *mh_send_back[4];
    int X[4];
    dim3 block[4];
    dim3 grid[4];

    if ( comm_partitioned() ) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
        if ( !commDimPartitioned(dir) && data.R()[dir] != 0 ) errorQuda("Not supported!\n");
      }
      for ( int i = 0; i < 4; i++ ) {
        faceVolume[i] = 1;
        for ( int j = 0; j < 4; j++ ) {
          if ( i == j ) continue;
          faceVolume[i] *= X[j];
        }
        faceVolumeCB[i] = faceVolume[i] / 2;
      }

      for ( int d = 0; d < 4; d++ ) {
        if ( !commDimPartitioned(d)) continue;
        offset[d] = faceVolumeCB[d] * NElems;
        bytes[d] =  sizeof(Float) * offset[d];
        send_d[d] = device_malloc(bytes[d]);
        recv_d[d] = device_malloc(bytes[d]);
        sendg_d[d] = device_malloc(bytes[d]);
        recvg_d[d] = device_malloc(bytes[d]);
        hipStreamCreate(&GFStream[d]);
        hipStreamCreate(&GFStream[4 + d]);
      #ifndef GPU_COMMS
        hostbuffer_h[d] = (void*)pinned_malloc(4 * bytes[d]);
      #endif
        block[d] = make_uint3(128, 1, 1);
        grid[d] = make_uint3((faceVolumeCB[d] + block[d].x - 1) / block[d].x, 1, 1);
      }
      hipStreamCreate(&GFStream[8]);
      for ( int d = 0; d < 4; d++ ) {
        if ( !commDimPartitioned(d)) continue;
      #ifdef GPU_COMMS
        recv[d] = recv_d[d];
        send[d] = send_d[d];
        recvg[d] = recvg_d[d];
        sendg[d] = sendg_d[d];
      #else
        recv[d] = hostbuffer_h[d];
        send[d] = static_cast<char*>(hostbuffer_h[d]) + bytes[d];
        recvg[d] = static_cast<char*>(hostbuffer_h[d]) + 3 * bytes[d];
        sendg[d] = static_cast<char*>(hostbuffer_h[d]) + 2 * bytes[d];
      #endif
        mh_recv_back[d] = comm_declare_receive_relative(recv[d], d, -1, bytes[d]);
        mh_recv_fwd[d]  = comm_declare_receive_relative(recvg[d], d, +1, bytes[d]);
        mh_send_back[d] = comm_declare_send_relative(sendg[d], d, -1, bytes[d]);
        mh_send_fwd[d]  = comm_declare_send_relative(send[d], d, +1, bytes[d]);
      }
    }
    GaugeFixUnPackArg<Gauge> dataexarg(dataOr, data);
    GaugeFixBorderPointsArg<Float, Gauge> argBorder(dataOr, data, relax_boost, faceVolume, faceVolumeCB);
    GaugeFixBorderPoints<Float,Gauge, gauge_dir> gfixBorderPoints(argBorder);
    GaugeFixInteriorPointsArg<Float, Gauge> argInt(dataOr, data, relax_boost);
    GaugeFixInteriorPoints<Float,Gauge, gauge_dir> gfixIntPoints(argInt);
  #endif

    GaugeFixQuality.apply(0);
    flop += (double)GaugeFixQuality.flops();
    byte += (double)GaugeFixQuality.bytes();
    double action0 = argQ.getAction();
    printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());


    unitarizeLinks(data, data, num_failures_dev);
    qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
    if ( num_failures > 0 ) {
      pool_device_free(num_failures_dev);
      errorQuda("Error in the unitarization\n");
      exit(1);
    }
    hipMemset(num_failures_dev, 0, sizeof(int));

    int iter = 0;
    for ( iter = 0; iter < Nsteps; iter++ ) {
      for ( int p = 0; p < 2; p++ ) {
      #ifndef MULTI_GPU
        gaugeFix.setParity(p);
        gaugeFix.apply(0);
        flop += (double)gaugeFix.flops();
        byte += (double)gaugeFix.bytes();
      #else
        if ( !comm_partitioned() ) {
          gaugeFix.setParity(p);
          gaugeFix.apply(0);
          flop += (double)gaugeFix.flops();
          byte += (double)gaugeFix.bytes();
        }
        else{
          gfixIntPoints.setParity(p);
          gfixBorderPoints.setParity(p); //compute border points
          gfixBorderPoints.apply(0);
          flop += (double)gfixBorderPoints.flops();
          byte += (double)gfixBorderPoints.bytes();
          flop += (double)gfixIntPoints.flops();
          byte += (double)gfixIntPoints.bytes();
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_start(mh_recv_back[d]);
            comm_start(mh_recv_fwd[d]);
          }
          //wait for the update to the halo points before start packing...
          qudaDeviceSynchronize();
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            //extract top face
            Kernel_UnPackTop<NElems, Float, Gauge, true> <<< grid[d], block[d], 0, GFStream[d] >>> (faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(send_d[d]), p, d, d);
            //extract bottom ghost
            Kernel_UnPackGhost<NElems, Float, Gauge, true> <<< grid[d], block[d], 0, GFStream[4 + d] >>> (faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(sendg_d[d]), 1 - p, d, d);
          }
        #ifdef GPU_COMMS
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaStreamSynchronize(GFStream[d]);
            comm_start(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
            comm_start(mh_send_back[d]);
          }
        #else
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            hipMemcpyAsync(send[d], send_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            hipMemcpyAsync(sendg[d], sendg_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[4 + d]);
          }
        #endif
          //compute interior points
          gfixIntPoints.apply(GFStream[8]);

        #ifndef GPU_COMMS
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaStreamSynchronize(GFStream[d]);
            comm_start(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
            comm_start(mh_send_back[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_recv_back[d]);
            hipMemcpyAsync(recv_d[d], recv[d], bytes[d], hipMemcpyHostToDevice, GFStream[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_recv_fwd[d]);
            hipMemcpyAsync(recvg_d[d], recvg[d], bytes[d], hipMemcpyHostToDevice, GFStream[4 + d]);
          }
        #endif
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
            comm_wait(mh_recv_back[d]);
          #endif
            Kernel_UnPackGhost<NElems, Float, Gauge, false> <<< grid[d], block[d], 0, GFStream[d] >>> (faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(recv_d[d]), p, d, d);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
            comm_wait(mh_recv_fwd[d]);
          #endif
            Kernel_UnPackTop<NElems, Float, Gauge, false> <<< grid[d], block[d], 0, GFStream[4 + d] >>> (faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(recvg_d[d]), 1 - p, d, d);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_send_back[d]);
            comm_wait(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
          }
          qudaStreamSynchronize(GFStream[8]);
        }
      #endif
        /*gaugeFix.setParity(p);
           gaugeFix.apply(0);
           flop += (double)gaugeFix.flops();
           byte += (double)gaugeFix.bytes();
           #ifdef MULTI_GPU
           if(comm_partitioned()){//exchange updated top face links in current parity
           for (int d=0; d<4; d++) {
            if (!commDimPartitioned(d)) continue;
            comm_start(mh_recv_back[d]);
            //extract top face
            Kernel_UnPackTop<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(send_d[d]), p, d, d, true);
           #ifndef GPU_COMMS
            hipMemcpy(send[d], send_d[d], bytes[d], hipMemcpyDeviceToHost);
           #else
            qudaDeviceSynchronize();
           #endif
            comm_start(mh_send_fwd[d]);
            comm_wait(mh_recv_back[d]);
            comm_wait(mh_send_fwd[d]);
           #ifndef GPU_COMMS
            hipMemcpy(recv_d[d], recv[d], bytes[d], hipMemcpyHostToDevice);
           #endif
            //inject top face in ghost
            Kernel_UnPackGhost<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(recv_d[d]), p, d, d, false);
           }
           //exchange updated ghost links in opposite parity
           for (int d=0; d<4; d++) {
            if (!commDimPartitioned(d)) continue;
            comm_start(mh_recv_fwd[d]);
            Kernel_UnPackGhost<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(sendg_d[d]), 1-p, d, d, true);
           #ifndef GPU_COMMS
            hipMemcpy(sendg[d], sendg_d[d], bytes[d], hipMemcpyDeviceToHost);
           #else
            qudaDeviceSynchronize();
           #endif
            comm_start(mh_send_back[d]);
            comm_wait(mh_recv_fwd[d]);
            comm_wait(mh_send_back[d]);
           #ifndef GPU_COMMS
            hipMemcpy(recvg_d[d], recvg[d], bytes[d], hipMemcpyHostToDevice);
           #endif
            Kernel_UnPackTop<NElems, Float, Gauge><<<grid[d], block[d]>>>(faceVolumeCB[d], dataexarg, reinterpret_cast<Float*>(recvg_d[d]), 1-p, d, d, false);
           }
           }
         #endif*/
      }
      if ((iter % reunit_interval) == (reunit_interval - 1)) {
        unitarizeLinks(data, data, num_failures_dev);
        qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
        if ( num_failures > 0 ) errorQuda("Error in the unitarization\n");
        hipMemset(num_failures_dev, 0, sizeof(int));
        flop += 4588.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3];
        byte += 8.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3] * dataOr.Bytes();
      }
      GaugeFixQuality.apply(0);
      flop += (double)GaugeFixQuality.flops();
      byte += (double)GaugeFixQuality.bytes();
      double action = argQ.getAction();
      double diff = abs(action0 - action);
      if ((iter % verbose_interval) == (verbose_interval - 1))
        printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
      if ( stopWtheta ) {
        if ( argQ.getTheta() < tolerance ) break;
      }
      else{
        if ( diff < tolerance ) break;
      }
      action0 = action;
    }
    if ((iter % reunit_interval) != 0 )  {
      unitarizeLinks(data, data, num_failures_dev);
      qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
      if ( num_failures > 0 ) errorQuda("Error in the unitarization\n");
      hipMemset(num_failures_dev, 0, sizeof(int));
      flop += 4588.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3];
      byte += 8.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3] * dataOr.Bytes();
    }
    if ((iter % verbose_interval) != 0 ) {
      GaugeFixQuality.apply(0);
      flop += (double)GaugeFixQuality.flops();
      byte += (double)GaugeFixQuality.bytes();
      double action = argQ.getAction();
      double diff = abs(action0 - action);
      printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
    }
    pool_device_free(num_failures_dev);
  #ifdef MULTI_GPU
    if ( comm_partitioned() ) {
      data.exchangeExtendedGhost(data.R(),false);
      for ( int d = 0; d < 4; d++ ) {
        if ( commDimPartitioned(d)) {
          comm_free(mh_send_fwd[d]);
          comm_free(mh_send_back[d]);
          comm_free(mh_recv_back[d]);
          comm_free(mh_recv_fwd[d]);
          device_free(send_d[d]);
          device_free(recv_d[d]);
          device_free(sendg_d[d]);
          device_free(recvg_d[d]);
          hipStreamDestroy(GFStream[d]);
          hipStreamDestroy(GFStream[4 + d]);
        #ifndef GPU_COMMS
          host_free(hostbuffer_h[d]);
        #endif
        }
      }
      hipStreamDestroy(GFStream[8]);
    }
  #endif
    checkCudaError();
    qudaDeviceSynchronize();
    profileInternalGaugeFixOVR.TPSTOP(QUDA_PROFILE_COMPUTE);
    if (getVerbosity() > QUDA_SUMMARIZE){
      double secs = profileInternalGaugeFixOVR.Last(QUDA_PROFILE_COMPUTE);
	  double gflops = (flop * 1e-9) / (secs);
	  double gbytes = byte / (secs * 1e9);
	  #ifdef MULTI_GPU
	  printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops * comm_size(), gbytes * comm_size());
	  #else
	  printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops, gbytes);
	  #endif
	}
  }

  template<typename Float, int NElems, typename Gauge>
  void gaugefixingOVR( Gauge dataOr,  cudaGaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval,
                       const Float relax_boost, const double tolerance, const int reunit_interval, const int stopWtheta) {
    if ( gauge_dir != 3 ) {
      printfQuda("Starting Landau gauge fixing...\n");
      gaugefixingOVR<Float, Gauge, NElems, 4>(dataOr, data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    }
    else {
      printfQuda("Starting Coulomb gauge fixing...\n");
      gaugefixingOVR<Float, Gauge, NElems, 3>(dataOr, data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    }
  }



  template<typename Float>
  void gaugefixingOVR( cudaGaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval,
		       const Float relax_boost, const double tolerance, const int reunit_interval, const int stopWtheta) {

    // Switching to FloatNOrder for the gauge field in order to support RECONSTRUCT_12
    if ( data.isNative() ) {
      if ( data.Reconstruct() == QUDA_RECONSTRUCT_NO ) {
        //printfQuda("QUDA_RECONSTRUCT_NO\n");
        numParams = 18;
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type Gauge;
        gaugefixingOVR<Float, 18>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_12 ) {
        //printfQuda("QUDA_RECONSTRUCT_12\n");
        numParams = 12;
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type Gauge;
        gaugefixingOVR<Float, 12>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
      } else if ( data.Reconstruct() == QUDA_RECONSTRUCT_8 ) {
        //printfQuda("QUDA_RECONSTRUCT_8\n");
        numParams = 8;
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type Gauge;
        gaugefixingOVR<Float, 8>(Gauge(data), data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
      } else {
        errorQuda("Reconstruction type %d of gauge field not supported", data.Reconstruct());
      }
    } else {
      errorQuda("Invalid Gauge Order\n");
    }
  }

#endif // GPU_GAUGE_ALG


  /**
   * @brief Gauge fixing with overrelaxation with support for single and multi GPU.
   * @param[in,out] data, quda gauge field
   * @param[in] gauge_dir, 3 for Coulomb gauge fixing, other for Landau gauge fixing
   * @param[in] Nsteps, maximum number of steps to perform gauge fixing
   * @param[in] verbose_interval, print gauge fixing info when iteration count is a multiple of this
   * @param[in] relax_boost, gauge fixing parameter of the overrelaxation method, most common value is 1.5 or 1.7.
   * @param[in] tolerance, torelance value to stop the method, if this value is zero then the method stops when iteration reachs the maximum number of steps defined by Nsteps
   * @param[in] reunit_interval, reunitarize gauge field when iteration count is a multiple of this
   * @param[in] stopWtheta, 0 for MILC criterium and 1 to use the theta value
   */
  void gaugefixingOVR( cudaGaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval, const double relax_boost,
                       const double tolerance, const int reunit_interval, const int stopWtheta) {
#ifdef GPU_GAUGE_ALG
    if ( data.Precision() == QUDA_HALF_PRECISION ) {
      errorQuda("Half precision not supported\n");
    }
    if ( data.Precision() == QUDA_SINGLE_PRECISION ) {
      gaugefixingOVR<float> (data, gauge_dir, Nsteps, verbose_interval, (float)relax_boost, tolerance, reunit_interval, stopWtheta);
    } else if ( data.Precision() == QUDA_DOUBLE_PRECISION ) {
      gaugefixingOVR<double>(data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
    } else {
      errorQuda("Precision %d not supported", data.Precision());
    }
#else
    errorQuda("Gauge fixing has not been built");
#endif // GPU_GAUGE_ALG
  }


}   //namespace quda
