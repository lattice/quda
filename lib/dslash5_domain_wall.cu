#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <dslash_quda.h>
#include <instantiate.h>
#include <tunable_nd.h>
#include <kernels/dslash_domain_wall_m5.cuh>

namespace quda
{

  template <typename Float, int nColor> class Dslash5 : public TunableKernel3D
  {
    cvector_ref<ColorSpinorField> &out;
    cvector_ref<const ColorSpinorField> &in;
    cvector_ref<const ColorSpinorField> &x;
    double m_f;
    double m_5;
    const Complex *b_5;
    const Complex *c_5;
    double a;
    bool dagger;
    bool xpay;
    Dslash5Type type;

    long long flops() const
    {
      long long Ls = in.X(4);
      long long bulk = (Ls - 2) * (in.Volume() / Ls);
      long long wall = 2 * in.Volume() / Ls;
      long long n = in.Ncolor() * in.Nspin();

      long long flops_ = 0;
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: flops_ = n * (8ll * bulk + 10ll * wall + (xpay ? 4ll * in.Volume() : 0)); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE:
        flops_ = n * (8ll * bulk + 10ll * wall + 14ll * in.Volume() + (xpay ? 8ll * in.Volume() : 0));
        break;
      case Dslash5Type::DSLASH5_MOBIUS:
        flops_ = n * (8ll * bulk + 10ll * wall + 8ll * in.Volume() + (xpay ? 8ll * in.Volume() : 0));
        break;
      case Dslash5Type::M5_INV_DWF:
      case Dslash5Type::M5_INV_MOBIUS: // FIXME flops
        flops_ = ((2 + 8 * n) * Ls + (xpay ? 4ll : 0)) * in.Volume();
        break;
      case Dslash5Type::M5_INV_ZMOBIUS: flops_ = ((12 + 16 * n) * Ls + (xpay ? 8ll : 0)) * in.Volume(); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }

      return in.size() * flops_;
    }

    long long bytes() const
    {
      long long Ls = in.X(4);
      size_t bytes = 0u;
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: bytes = out.Bytes() + 2 * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE: bytes = out.Bytes() + 3 * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      case Dslash5Type::DSLASH5_MOBIUS: bytes = out.Bytes() + 3 * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      case Dslash5Type::M5_INV_DWF: bytes = out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      case Dslash5Type::M5_INV_MOBIUS: bytes = out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      case Dslash5Type::M5_INV_ZMOBIUS: bytes = out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }
      return bytes;
    }

    unsigned int minThreads() const { return in.VolumeCB() / in.X(4); }
    int blockStep() const { return 4; }
    int blockMin() const { return 4; }
    unsigned int sharedBytesPerThread() const
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        // spin components in shared depend on inversion algorithm
        bool isInv = type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
          || type == Dslash5Type::M5_INV_ZMOBIUS;
        int nSpin = (!isInv || mobius_m5::var_inverse()) ? mobius_m5::use_half_vector() ? in.Nspin() / 2 : in.Nspin() :
                                                           in.Nspin();
        return 2 * nSpin * nColor * sizeof(typename mapper<Float>::type);
      } else {
        return 0;
      }
    }

    // overloaded to return max dynamic shared memory if doing shared-memory inverse
    unsigned int maxSharedBytesPerBlock() const
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        return maxDynamicSharedBytesPerBlock();
      } else {
        return TunableKernel3D::maxSharedBytesPerBlock();
      }
    }

  public:
    Dslash5(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
            cvector_ref<const ColorSpinorField> &x, double m_f, double m_5, const Complex *b_5, const Complex *c_5,
            double a, bool dagger, Dslash5Type type) :
      TunableKernel3D(in[0], in.size() * in.X(4), in.SiteSubset()),
      out(out),
      in(in),
      x(x),
      m_f(m_f),
      m_5(m_5),
      b_5(b_5),
      c_5(c_5),
      a(a),
      dagger(dagger),
      xpay(a == 0.0 ? false : true),
      type(type)
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        // Ls must be contained in the block and different fields on different blocks
        resizeStep(in.X(4), 1);
        tune_block_y = false;
      }

      if (dagger) strcat(aux, ",Dagger");
      if (xpay) strcat(aux, ",xpay");
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: strcat(aux, ",Dslash5Type::DSLASH5_DWF"); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE: strcat(aux, ",Dslash5Type::DSLASH5_MOBIUS_PRE"); break;
      case Dslash5Type::DSLASH5_MOBIUS: strcat(aux, ",Dslash5Type::DSLASH5_MOBIUS"); break;
      case Dslash5Type::M5_INV_DWF: strcat(aux, ",Dslash5Type::M5_INV_DWF"); break;
      case Dslash5Type::M5_INV_MOBIUS: strcat(aux, ",Dslash5Type::M5_INV_MOBIUS"); break;
      case Dslash5Type::M5_INV_ZMOBIUS: strcat(aux, ",Dslash5Type::M5_INV_ZMOBIUS"); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }
      setRHSstring(aux, in.size());

      apply(device::get_default_stream());
    }

    template <bool dagger, bool xpay, Dslash5Type type> using Arg = Dslash5Arg<Float, nColor, dagger, xpay, type>;

    template <Dslash5Type type, template <typename> class F>
    void Launch(TuneParam &tp, const qudaStream_t &stream)
    {
      if (xpay)
        dagger ?
          launch<F>(tp, stream, Arg<true, true, type>(out, in, x, m_f, m_5, b_5, c_5, a)) :
          launch<F>(tp, stream, Arg<false, true, type>(out, in, x, m_f, m_5, b_5, c_5, a));
      else
        dagger ?
          launch<F>(tp, stream, Arg<true, false, type>(out, in, x, m_f, m_5, b_5, c_5, a)) :
          launch<F>(tp, stream, Arg<false, false, type>(out, in, x, m_f, m_5, b_5, c_5, a));
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        tp.set_max_shared_bytes = true; // if inverse kernel uses shared memory then maximize total shared memory pool
      }

      switch (type) {
      case Dslash5Type::DSLASH5_DWF: Launch<Dslash5Type::DSLASH5_DWF, dslash5>(tp, stream); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE: Launch<Dslash5Type::DSLASH5_MOBIUS_PRE, dslash5>(tp, stream); break;
      case Dslash5Type::DSLASH5_MOBIUS: Launch<Dslash5Type::DSLASH5_MOBIUS, dslash5>(tp, stream); break;
      case Dslash5Type::M5_INV_DWF: Launch<Dslash5Type::M5_INV_DWF, dslash5inv>(tp, stream); break;
      case Dslash5Type::M5_INV_MOBIUS: Launch<Dslash5Type::M5_INV_MOBIUS, dslash5inv>(tp, stream); break;
      case Dslash5Type::M5_INV_ZMOBIUS: Launch<Dslash5Type::M5_INV_ZMOBIUS, dslash5inv>(tp, stream); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }
    }
  };

  // Apply the 5th dimension dslash operator to a colorspinor field
  // out = Dslash5*in
  void ApplyDslash5(cvector_ref<ColorSpinorField> &out, cvector_ref<const ColorSpinorField> &in,
                    cvector_ref<const ColorSpinorField> &x, double m_f, double m_5, const Complex *b_5,
                    const Complex *c_5, double a, bool dagger, Dslash5Type type)
  {
    if (is_enabled<QUDA_DOMAIN_WALL_4D_DSLASH>()) {
      if (in.PCType() != QUDA_4D_PC) errorQuda("Only 4-d preconditioned fields are supported");
      checkLocation(out, in, x); // check all locations match
      instantiate_recurse3<Dslash5>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
    } else {
      errorQuda("Domain wall operator has not been built");
    }
  }

} // namespace quda
