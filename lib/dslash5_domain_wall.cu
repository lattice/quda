#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <dslash_quda.h>
#include <instantiate.h>
#include <tunable_nd.h>
#include <kernels/dslash_domain_wall_m5.cuh>

namespace quda
{

  template <typename Float, int nColor> class Dslash5 : public TunableKernel3D
  {
    ColorSpinorField &out;
    const ColorSpinorField &in;
    const ColorSpinorField &x;
    double m_f;
    double m_5;
    const Complex *b_5;
    const Complex *c_5;
    double a;
    bool dagger;
    bool xpay;
    Dslash5Type type;

    long long flops() const
    {
      long long Ls = in.X(4);
      long long bulk = (Ls - 2) * (in.Volume() / Ls);
      long long wall = 2 * in.Volume() / Ls;
      long long n = in.Ncolor() * in.Nspin();

      long long flops_ = 0;
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: flops_ = n * (8ll * bulk + 10ll * wall + (xpay ? 4ll * in.Volume() : 0)); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE:
        flops_ = n * (8ll * bulk + 10ll * wall + 14ll * in.Volume() + (xpay ? 8ll * in.Volume() : 0));
        break;
      case Dslash5Type::DSLASH5_MOBIUS:
        flops_ = n * (8ll * bulk + 10ll * wall + 8ll * in.Volume() + (xpay ? 8ll * in.Volume() : 0));
        break;
      case Dslash5Type::M5_INV_DWF:
      case Dslash5Type::M5_INV_MOBIUS: // FIXME flops
        flops_ = ((2 + 8 * n) * Ls + (xpay ? 4ll : 0)) * in.Volume();
        break;
      case Dslash5Type::M5_INV_ZMOBIUS: flops_ = ((12 + 16 * n) * Ls + (xpay ? 8ll : 0)) * in.Volume(); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }

      return flops_;
    }

    long long bytes() const
    {
      long long Ls = in.X(4);
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: return out.Bytes() + 2 * in.Bytes() + (xpay ? x.Bytes() : 0);
      case Dslash5Type::DSLASH5_MOBIUS_PRE: return out.Bytes() + 3 * in.Bytes() + (xpay ? x.Bytes() : 0);
      case Dslash5Type::DSLASH5_MOBIUS: return out.Bytes() + 3 * in.Bytes() + (xpay ? x.Bytes() : 0);
      case Dslash5Type::M5_INV_DWF: return out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0);
      case Dslash5Type::M5_INV_MOBIUS: return out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0);
      case Dslash5Type::M5_INV_ZMOBIUS: return out.Bytes() + Ls * in.Bytes() + (xpay ? x.Bytes() : 0);
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }
      return 0ll;
    }

    unsigned int minThreads() const { return in.VolumeCB() / in.X(4); }
    int blockStep() const { return 4; }
    int blockMin() const { return 4; }
    unsigned int sharedBytesPerThread() const
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        // spin components in shared depend on inversion algorithm
        int nSpin = mobius_m5::var_inverse() ? mobius_m5::use_half_vector() ? in.Nspin() / 2 : in.Nspin() : in.Nspin();
        return 2 * nSpin * nColor * sizeof(typename mapper<Float>::type);
      } else {
        return 0;
      }
    }

    // overloaded to return max dynamic shared memory if doing shared-memory inverse
    unsigned int maxSharedBytesPerBlock() const
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        return maxDynamicSharedBytesPerBlock();
      } else {
        return TunableKernel3D::maxSharedBytesPerBlock();
      }
    }

  public:
    Dslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, double m_f,
            double m_5, const Complex *b_5, const Complex *c_5, double a, bool dagger, Dslash5Type type) :
      TunableKernel3D(in, in.X(4), in.SiteSubset()),
      out(out),
      in(in),
      x(x),
      m_f(m_f),
      m_5(m_5),
      b_5(b_5),
      c_5(c_5),
      a(a),
      dagger(dagger),
      xpay(a == 0.0 ? false : true),
      type(type)
    {
      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        TunableKernel2D_base<false>::resizeStep(in.X(4)); // Ls must be contained in the block
      }

      if (dagger) strcat(aux, ",Dagger");
      if (xpay) strcat(aux, ",xpay");
      switch (type) {
      case Dslash5Type::DSLASH5_DWF: strcat(aux, ",Dslash5Type::DSLASH5_DWF"); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE: strcat(aux, ",Dslash5Type::DSLASH5_MOBIUS_PRE"); break;
      case Dslash5Type::DSLASH5_MOBIUS: strcat(aux, ",Dslash5Type::DSLASH5_MOBIUS"); break;
      case Dslash5Type::M5_INV_DWF: strcat(aux, ",Dslash5Type::M5_INV_DWF"); break;
      case Dslash5Type::M5_INV_MOBIUS: strcat(aux, ",Dslash5Type::M5_INV_MOBIUS"); break;
      case Dslash5Type::M5_INV_ZMOBIUS: strcat(aux, ",Dslash5Type::M5_INV_ZMOBIUS"); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }

      apply(device::get_default_stream());
    }

    template <bool dagger, bool xpay, Dslash5Type type> using Arg = Dslash5Arg<Float, nColor, dagger, xpay, type>;
    
    template <Dslash5Type type, template <typename> class F>
    void Launch(TuneParam &tp, const qudaStream_t &stream)
    {
      if (xpay)
        dagger ?
          launch<F>(tp, stream, Arg<true, true, type>(out, in, x, m_f, m_5, b_5, c_5, a)) :
          launch<F>(tp, stream, Arg<false, true, type>(out, in, x, m_f, m_5, b_5, c_5, a));
      else
        dagger ?
          launch<F>(tp, stream, Arg<true, false, type>(out, in, x, m_f, m_5, b_5, c_5, a)) :
          launch<F>(tp, stream, Arg<false, false, type>(out, in, x, m_f, m_5, b_5, c_5, a));
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

      if (mobius_m5::shared()
          && (type == Dslash5Type::M5_INV_DWF || type == Dslash5Type::M5_INV_MOBIUS
              || type == Dslash5Type::M5_INV_ZMOBIUS)) {
        tp.set_max_shared_bytes = true; // if inverse kernel uses shared memory then maximize total shared memory pool
      }

      switch (type) {
      case Dslash5Type::DSLASH5_DWF: Launch<Dslash5Type::DSLASH5_DWF, dslash5>(tp, stream); break;
      case Dslash5Type::DSLASH5_MOBIUS_PRE: Launch<Dslash5Type::DSLASH5_MOBIUS_PRE, dslash5>(tp, stream); break;
      case Dslash5Type::DSLASH5_MOBIUS: Launch<Dslash5Type::DSLASH5_MOBIUS, dslash5>(tp, stream); break;
      case Dslash5Type::M5_INV_DWF: Launch<Dslash5Type::M5_INV_DWF, dslash5inv>(tp, stream); break;
      case Dslash5Type::M5_INV_MOBIUS: Launch<Dslash5Type::M5_INV_MOBIUS, dslash5inv>(tp, stream); break;
      case Dslash5Type::M5_INV_ZMOBIUS: Launch<Dslash5Type::M5_INV_ZMOBIUS, dslash5inv>(tp, stream); break;
      default: errorQuda("Unexpected Dslash5Type %d", static_cast<int>(type));
      }
    }
  };

  // Apply the 5th dimension dslash operator to a colorspinor field
  // out = Dslash5*in
#ifdef GPU_DOMAIN_WALL_DIRAC
  void ApplyDslash5(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &x, double m_f,
                    double m_5, const Complex *b_5, const Complex *c_5, double a, bool dagger, Dslash5Type type)
  {
    if (in.PCType() != QUDA_4D_PC) errorQuda("Only 4-d preconditioned fields are supported");
    checkLocation(out, in, x); // check all locations match
    instantiate<Dslash5>(out, in, x, m_f, m_5, b_5, c_5, a, dagger, type);
  }
#else
  void ApplyDslash5(ColorSpinorField &, const ColorSpinorField &, const ColorSpinorField &, double,
                    double, const Complex *, const Complex *, double, bool, Dslash5Type)
  {
    errorQuda("Domain wall dslash has not been built");
  }
#endif

} // namespace quda
