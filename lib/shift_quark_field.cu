#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <quda_internal.h>

namespace quda {

  template<typename Output, typename Input>
    struct ShiftColorSpinorFieldArg {
      const unsigned int length;
      unsigned int X[4];
#ifdef MULTI_GPU
      const usigned int ghostOffset; // depends on the direction
#endif
      const unsigned int parity;
      const unsigned int dir;
      bool partitioned[4];
      const int shift;
      Input in;
      Output out;
      ShiftColorSpinorFieldArg(const unsigned int length, 
          const unsigned int X[4],
          const unsigned int ghostOffset,
          const unsigned int parity,
          const unsigned int dir,
          const int shift,   
          const Input& in,
          const Output& out) : length(length),
#ifdef MULTI_GPU
      ghostOffset(ghostOffset),
#endif
      parity(parity), dir(dir), shift(shift),  in(in), out(out) 
      {
        for(int i=0; i<4; ++i) this->X[i] = X[i];
        for(int i=0; i<4; ++i) partitioned[i] = commDimPartitioned(i) ? true : false;
      }
    };

  template<IndexType idxType, typename Int>
    __device__ __forceinline__
    int neighborIndex(const unsigned int& cb_idx, const int (&shift)[4], const bool (&partitioned)[4], const unsigned int& parity){

      int idx;
      Int x, y, z, t;

      coordsFromIndex(full_idx, x, y, z, t, cb_idx, parity);

#ifdef MULTI_GPU
      if(partitioned[0])
        if( (x+shift[0])<0 || (x+shift[0])>=X1) return -1;
      if(partitioned[1])
        if( (y+shift[1])<0 || (y+shift[1])>=X2) return -1;
      if(partitioned[2])
        if( (z+shift[2])<0 || (z+shift[2])>=X3) return -1;
      if(partitioned[3])
        if( (z+shift[3])<0 || (z+shift[3])>=X4) return -1;
#endif

      x = shift[0] ? (x + shift[0] + X1) % X1 : x;
      y = shift[1] ? (y + shift[1] + X2) % X2 : y;
      z = shift[2] ? (z + shift[2] + X3) % X3 : z;
      t = shift[3] ? (t + shift[3] + X4) % X4 : t;
      return  (((t*X3 + z)*X2 + y)*X1 + x) >> 1;
    }


  template <typename FloatN, int N, typename Output, typename Input>
    __global__ void shiftColorSpinorFieldKernel(ShiftQuarkArg<Output,Input> arg){

      int shift[4] = {0,0,0,0};
      shift[arg.dir] = arg.shift;

      unsigned int idx = blockIdx.x*(blockDim.x) + threadIdx.x;
      unsigned int gridSize = gridDim.x*blockDim.x;

      FloatN x[N];
      while(idx<arg.length){
        const int new_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity);
#ifdef MULTI_GPU
        if(new_idx > 0){
#endif
          arg.in.load(x, new_idx);
          arg.out.save(x, idx);
#ifdef MULTI_GPU
        }
#endif       
        idx += gridSize;
      }  
      return;
    }

  template<typename FloatN, int N, typename Output, typename Input>
    __global__ void shiftColorSpinorFieldExternalKernel(ShiftQuarkArg<Output,Input> arg){

      unsigned int idx = blockIdx.x*(blockDim.x) + threadIdx.x;
      unsigned int gridSize = gridDim.x*blockDim.x;

      Float x[N];
      unsigned int coord[4];
      while(idx<arg.length){

        // compute the coordinates in the ghost zone 
        coordsFromIndex<1>(coord, idx, arg.X, arg.dir, arg.parity);

        unsigned int ghost_idx = arg.ghostOffset + ghostIndexFromCoords<3,3>(arg.X, coord, arg.dir, arg.shift);

        arg.in.load(x, ghost_idx);
        arg.out.save(x, idx);

        idx += gridSize;
      }


      return;
    }

  template<typename Output, typename Input> 
    class ShiftColorSpinorField : public Tunable {

      private:
        ShiftColorSpinorFieldArg<Output,Input> arg;
        const int *X; // pointer to lattice dimensions

        int sharedBytesPerThread() const { return 0; }
        int sharedBytesPerBlock(const TuneParam &) cont { return 0; }

        // don't tune the grid dimension
        bool advanceGridDim(TuneParam & param) const { return false; }

        bool advanceBlockDim(TuneParam &param) const 
        {
          const unsigned int max_threads = deviceProp.maxThreadsDim[0];
          const unsigned int max_blocks = deviceProp.maxGridSize[0];
          const unsigned int max_shared = 16384;
          const int step = deviceProp.warpSize;
          const int threads = arg.length;
          bool ret;

          param.block.x += step;
          if(param.block.x > max_threads || sharedBytesPerThread()*param.block.x > max_shared){
            param.block = dim3((threads+max_blocks-1)/max_blocks, 1, 1); // ensure the blockDim is large enough given the limit on gridDim
            param.block.x = ((param.block.x+step-1)/step)*step;
            if(param.block.x > max_threads) errorQuda("Local lattice volume is too large for device");
            ret = false;
          }else{
            ret = true;
          }
          param.grid = dim3((threads+param.block.x-1)/param.block.x,1,1);
          return ret;
        }


      public:
        ShiftColorSpinorField(const ShiftColorSpinorField<Output,Input> &arg, 
            QudaFieldLocation location)
          : arg(arg), location(location)  {}
        virtual ~ShiftColorSpinorField() {}

        void apply(const qudaStream_t &stream){
          if(location == QUDA_CUDA_FIELD_LOCATION){
            TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            shiftColorSpinorFieldKernel<Output,Input><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
#ifdef MULTI_GPU
            // Need to perform some communication and call exterior kernel, I guess
#endif
          }else{ // run the CPU code
            errorQuda("ShiftColorSpinorField is not yet implemented on the CPU\n");
          }
        } // apply

        virtual void initTuneParam(TuneParam &param) const
        {
          const unsigned int max_threads = deviceProp.maxThreadsDim[0];
          const unsigned int max_blocks = deviceProp.maxGridSize[0];
          const int threads = arg.length;
          const int step = deviceProp.warpSize;
          param.block = dim3((threads+max_blocks-1)/max_blocks, 1, 1); // ensure the blockDim is large enough, given the limit on gridDim
          param.block.x = ((param.block.x+step-1) / step) * step; // round up to the nearest "step"
          if (param.block.x > max_threads) errorQuda("Local lattice volume is too large for device");
          param.grid = dim3((threads+param.block.x-1)/param.block.x, 1, 1);
          param.shared_bytes = sharedBytesPerThread()*param.block.x > sharedBytesPerBlock(param) ?
            sharedBytesPerThread()*param.block.x : sharedBytesPerBlock(param);
        }

        /** sets default values for when tuning is disabled */
        void defaultTuneParam(TuneParam &param) const {
          initTuneParam(param);
        }

        long long flops() const { return 0; } // fixme
        long long bytes() const { return 0; } // fixme

        TuneKey tuneKey() const {
          std::stringstream vol, aux;
          vol << X[0] << "x";
          vol << X[1] << "x";
          vol << X[2] << "x";
          vol << X[3] << "x";
          aux << "threads=" << 2*arg.in.volumeCB << ",prec=" << sizeof(Complex)/2;
          aux << "stride=" << arg.in.stride;
          return TuneKey(vol.str(), typeid(*this).name(), aux.str());
        }
    };


  // Should really have a parity
  void shiftColorSpinorField(cudaColorSpinorField &dst, const cudaColorSpinorField &src, const unsigned int parity, const unsigned int dim, const int shift) {

    if(&src == &dst){
      errorQuda("destination field is the same as source field\n");
      return;
    }

    if(src.Nspin() != 1 && src.Nspin() !=4) errorQuda("nSpin(%d) not supported\n", src.Nspin());

    if(src.SiteSubset() != dst.SiteSubset())
      errorQuda("Spinor fields do not have matching subsets\n");

    if(src.SiteSubset() == QUDA_FULL_SITE_SUBSET){
      if(shift&1){
        shiftColorSpinorField(dst.Even(), src.Odd(), 0, dim, shift);
        shiftColorSpinorField(dst.Odd(), src.Even(), 1, dim, shift);
      }else{
        shiftColorSpinorField(dst.Even(), src.Even(), 0, dim, shift);
        shiftColorSpinorField(dst.Odd(), src.Odd(), 1, dim, shift);
      }
      return;
    }

#ifdef MULTI_GPU
    const int dir = (shift>0) ? QUDA_BACKWARDS : QUDA_FORWARDS; // pack the start of the field if shift is positive
    const int offset = (shift>0) ? 0 : 1;
#endif


    if(dst.Precision() == QUDA_DOUBLE_PRECISION && src.Precision() == QUDA_DOUBLE_PRECISION){
      if(src.Nspin() == 1){
        Spinor<double2, double2, double2, 3, 0, 0> src_tex(src);
        Spinor<double2, double2, double2, 3, 1> dst_spinor(dst);
        ShiftColorSpinorFieldArg arg(src.Volume(), parity, dim, shift, dst_spinor, src_tex);
        ShiftColorSpinorField shiftColorSpinor(arg, QUDA_CPU_FIELD_LOCATION);

#ifdef MULTI_GPU
        if(commDimPartitioned(dim) && dim!=3){
          face->pack(src, 1-parity, dagger, dim, dir, streams); // pack in stream[1]
          qudaEventRecord(packEnd, streams[1]);
          qudaStreamWaitEvent(streams[1], packEnd, 0); // wait for pack to end in stream[1]
          face->gather(src, dagger, 2*dim+offset, 1); // copy packed data from device buffer to host and do this in stream[1] 
          qudaEventRecord(gatherEnd, streams[1]); // record the completion of face->gather
        }
#endif

        shiftColorSpinor.apply(0); // shift the field in the interior region

#ifdef MULTI_GPU
        if(commDimPartitioned(dim) && dim!=3){
          while(1){
            hipError_t eventQuery = hipEventQuery(gatherEnd);
            if(eventQuery == hipSuccess){
              face->commsStart(2*dim + offset); // if argument is even, send backwards, else send forwards
              break;
            }
          }

          // after communication, load data back on to device
          // do this in stream[1]
          while(1){
            if(face->commsQuery(2*dim + offset)){
              face->scatter(src, dagger, 2*dim+offset, 1);
              break;
            }
          } // while(1) 
          qudaEventRecord(scatterEnd, streams[1]);
          qudaStreamWaitEvent(streams[1], scatterEnd, 0);
          shiftColorSpinor.apply(1);
        }
#endif

      }else{
        errorQuda("Only staggered fermions are currently supported\n");
      }
    }else if(dst.Precision() == QUDA_SINGLE_PRECISION && src.Precision() == QUDA_SINGLE_PRECISION){
      if(src.Nspin() == 1 ){
        Spinor<float2, float2, float2, 3, 0, 0> src_tex(src);
        Spinor<float2, float2, float2, 3, 1> dst_spinor(dst);
        ShiftColorSpinorFieldArg arg(src.Volume(), parity, dim, shift, dst_spinor, src_tex);
        ShiftColorSpinorField shiftColorSpinor(arg, QUDA_CPU_FIELD_LOCATION);
      }else{
        errorQuda("Only staggered fermions are currently supported\n");
      }
    }
    return;
  }


} // namespace quda

