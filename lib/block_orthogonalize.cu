#include "hip/hip_runtime.h"
#include <array>

#include <color_spinor_field.h>
#include <uint_to_char.h>
#include <vector>
#include <assert.h>
#include <utility>

#include <power_of_two_array.h>
#include <kernels/block_orthogonalize.cuh>
#include <tunable_block_reduction.h>

namespace quda {

  struct OrthoAggregates {
    // List of block sizes we wish to instantiate.  The required block
    // size is equal to number of fine points per aggregate, rounded
    // up to a whole power of two.  So for example, 2x2x2x2 and
    // 3x3x3x1 aggregation would both use the same block size 32
#ifndef QUDA_FAST_COMPILE_REDUCE
    using array_type = PowerOfTwoArray<device::warp_size(), device::max_block_size()>;
#else
    using array_type = PowerOfTwoArray<device::max_block_size(), device::max_block_size()>;
#endif
    static constexpr array_type block = array_type();

    /**
       @brief Return the first power of two block that is larger than the required size
    */
    static unsigned int block_mapper(unsigned int raw_block)
    {
      for (unsigned int b = 0; b < block.size();  b++) if (raw_block <= block[b]) return block[b];
      errorQuda("Invalid raw block size %d\n", raw_block);
      return 0;
    }
  };

  constexpr OrthoAggregates::array_type OrthoAggregates::block;

  using namespace quda::colorspinor;

  // B fields in general use float2 ordering except for fine-grid Wilson
  template <typename store_t, int nSpin, int nColor> struct BOrder { static constexpr QudaFieldOrder order = QUDA_FLOAT2_FIELD_ORDER; };
  template<> struct BOrder<float, 4, 3> { static constexpr QudaFieldOrder order = QUDA_FLOAT4_FIELD_ORDER; };
#ifdef FLOAT8
  template<> struct BOrder<short, 4, 3> { static constexpr QudaFieldOrder order = QUDA_FLOAT8_FIELD_ORDER; };
  template<> struct BOrder<int8_t, 4, 3> { static constexpr QudaFieldOrder order = QUDA_FLOAT8_FIELD_ORDER; };
#else
  template<> struct BOrder<short, 4, 3> { static constexpr QudaFieldOrder order = QUDA_FLOAT4_FIELD_ORDER; };
  template<> struct BOrder<int8_t, 4, 3> { static constexpr QudaFieldOrder order = QUDA_FLOAT4_FIELD_ORDER; };
#endif

  template <typename vFloat, typename bFloat, int nSpin, int spinBlockSize, int nColor_, int coarseSpin, int nVec>
  class BlockOrtho : public TunableBlock2D {

    using real = typename mapper<vFloat>::type;
    // we only support block-format on fine grid where Ncolor=3
    static constexpr int nColor = isFixed<bFloat>::value ? 3 : nColor_;
    static constexpr int chiral_blocks = nSpin == 1 ? 2 : nSpin / spinBlockSize;
    template <bool is_device, typename Rotator, typename Vector> using Arg = BlockOrthoArg<is_device, vFloat, Rotator, Vector, nSpin, nColor, coarseSpin, nVec>;

    ColorSpinorField &V;
    const std::vector<ColorSpinorField*> B;
    const int *fine_to_coarse;
    const int *coarse_to_fine;
    const int *geo_bs;
    const int n_block_ortho;
    int aggregate_size;
    int nBlock;
    bool two_pass;
    int iter;
    double max;

  public:
    BlockOrtho(ColorSpinorField &V, const std::vector<ColorSpinorField *> B, const int *fine_to_coarse,
               const int *coarse_to_fine, const int *geo_bs, int n_block_ortho, bool two_pass) :
      TunableBlock2D(V, chiral_blocks),
      V(V),
      B(B),
      fine_to_coarse(fine_to_coarse),
      coarse_to_fine(coarse_to_fine),
      geo_bs(geo_bs),
      n_block_ortho(n_block_ortho),
      two_pass(two_pass),
      iter(0),
      max(1.0)
    {
      if (nColor_ != nColor)
        errorQuda("Number of colors %d not supported with this precision %lu\n", nColor_, sizeof(bFloat));

      strcat(aux,",block_size=");

      aggregate_size = 1;
      char geo_str[16];
      for (int d = 0; d < V.Ndim(); d++) {
        aggregate_size *= geo_bs[d];
        i32toa(geo_str, geo_bs[d]);
        strcat(aux, geo_str);
        if (d < V.Ndim() - 1) strcat(aux, "x");
      }

      if (aggregate_size == 1) errorQuda("Invalid MG aggregate size %d, cannot be 1", aggregate_size);
      if (aggregate_size % 2 != 0) errorQuda("Invalid MG aggregate size %d, must be even", aggregate_size);
      if (aggregate_size > 1024) errorQuda("Invalid MG aggregate size %d, must be <= 1024", aggregate_size);

      nBlock = (V.Volume()/aggregate_size) * chiral_blocks;

      strcat(aux, ",n_block_ortho=");
      char n_ortho_str[2];
      i32toa(n_ortho_str, n_block_ortho);
      strcat(aux, n_ortho_str);
      strcat(aux, ",mVec=");
      char mvec_str[3];
      int active_x_threads = (aggregate_size / 2) * (nSpin == 1 ? 1 : V.SiteSubset());
      i32toa(mvec_str, tile_size<nColor, nVec>(OrthoAggregates::block_mapper(active_x_threads)));
      strcat(aux, mvec_str);

      V.Scale(max); // by definition this is true
      apply(device::get_default_stream());
      if (two_pass && V.Precision() < QUDA_SINGLE_PRECISION) {  // recompute for more precision
        iter++;
        V.Scale(1.05 * max); // the 1.05 gives us some margin
        apply(device::get_default_stream());
      }
    }

    template <typename Rotator, typename Vector, std::size_t... S>
    void launch_host_(const TuneParam &tp, const qudaStream_t &stream,
                     const std::vector<ColorSpinorField*> &B, std::index_sequence<S...>)
    {
      Arg<false, Rotator, Vector> arg(V, fine_to_coarse, coarse_to_fine, QUDA_INVALID_PARITY, geo_bs, n_block_ortho, V, B[S]...);
      launch_host<BlockOrtho_, OrthoAggregates>(tp, stream, arg);
      if (two_pass && iter == 0 && V.Precision() < QUDA_SINGLE_PRECISION && !activeTuning()) max = Rotator(V).abs_max(V);
    }

    template <typename Rotator, typename Vector, std::size_t... S>
    void launch_device_(const TuneParam &tp, const qudaStream_t &stream,
                        const std::vector<ColorSpinorField*> &B, std::index_sequence<S...>)
    {
      Arg<true, Rotator, Vector> arg(V, fine_to_coarse, coarse_to_fine, QUDA_INVALID_PARITY, geo_bs, n_block_ortho, V, B[S]...);
      arg.swizzle_factor = tp.aux.x;
      launch_device<BlockOrtho_, OrthoAggregates>(tp, stream, arg);
      if (two_pass && iter == 0 && V.Precision() < QUDA_SINGLE_PRECISION && !activeTuning()) max = Rotator(V).abs_max(V);
    }

    void apply(const qudaStream_t &stream)
    {
      constexpr bool disable_ghost = DISABLE_GHOST;
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
        if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER && B[0]->FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
          typedef FieldOrderCB<real,nSpin,nColor,nVec,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,vFloat,vFloat,disable_ghost> Rotator;
          typedef FieldOrderCB<real,nSpin,nColor,1,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,bFloat,bFloat,disable_ghost> Vector;
          launch_host_<Rotator, Vector>(tp, stream, B, std::make_index_sequence<nVec>());
        } else {
          errorQuda("Unsupported field order %d", V.FieldOrder());
        }
      } else {
        if (V.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER && B[0]->FieldOrder() == BOrder<bFloat,nSpin,nColor>::order) {
          typedef FieldOrderCB<real,nSpin,nColor,nVec,QUDA_FLOAT2_FIELD_ORDER,vFloat,vFloat,disable_ghost> Rotator;
          typedef FieldOrderCB<real,nSpin,nColor,1,BOrder<bFloat,nSpin,nColor>::order,bFloat,bFloat,disable_ghost,isFixed<bFloat>::value> Vector;
          launch_device_<Rotator, Vector>(tp, stream, B, std::make_index_sequence<nVec>());
        } else {
          errorQuda("Unsupported field order V=%d B=%d", V.FieldOrder(), B[0]->FieldOrder());
        }
      }
    }

#ifdef SWIZZLE
    bool advanceAux(TuneParam &param) const
    {
      if (param.aux.x < 2 * device::processor_count()) {
        param.aux.x++;
	return true;
      } else {
        param.aux.x = 1;
	return false;
      }
    }
#else
    bool advanceAux(TuneParam &) const { return false; }
#endif

    bool advanceTuneParam(TuneParam &param) const
    {
      if (V.Location() == QUDA_CUDA_FIELD_LOCATION) {
	return advanceSharedBytes(param) || advanceAux(param);
      } else {
	return false;
      }
    }

    /** sets default values for when tuning is disabled */
    void initTuneParam(TuneParam &param) const
    {
      TunableBlock2D::initTuneParam(param);
      int active_x_threads = (aggregate_size / 2) * (nSpin == 1 ? 1 : V.SiteSubset());
      param.block = dim3(OrthoAggregates::block_mapper(active_x_threads), 1, 1);
      param.grid = dim3((nSpin == 1 ? V.VolumeCB() : V.Volume()) / active_x_threads, chiral_blocks, 1);
      param.aux.x = 1; // swizzle factor
    }

    void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    long long flops() const
    {
      auto n = nVec;
      auto k = (aggregate_size / 2) * (spinBlockSize == 0 ? 1 : 2 * spinBlockSize) * nColor;
      auto L = 8l + 8l; // dot + caxpy
      auto D = 4l + 2l; // norm + scale
      return n_block_ortho * nBlock * k * ((n - 1) * n / 2 * L + n * D);
    }

    long long bytes() const
    {
      return nVec * B[0]->Bytes() + (nVec - 1) * nVec / 2 * V.Bytes() / nVec + V.Bytes()
        + (n_block_ortho - 1) * (V.Bytes() + (nVec - 1) * nVec / 2 * V.Bytes() / nVec + V.Bytes());
    }

    void preTune() { V.backup(); }
    void postTune() { V.restore(); }
  };

  template <typename vFloat, typename bFloat, int nSpin, int spinBlockSize, int nColor, int nVec>
  void BlockOrthogonalize(ColorSpinorField &V, const std::vector<ColorSpinorField *> &B, const int *fine_to_coarse,
                          const int *coarse_to_fine, const int *geo_bs, int n_block_ortho, bool two_pass)
  {
    int geo_blocksize = 1;
    for (int d = 0; d < V.Ndim(); d++) geo_blocksize *= geo_bs[d];

    int blocksize = geo_blocksize * V.Ncolor();
    if (spinBlockSize == 0) { blocksize /= 2; } else { blocksize *= spinBlockSize; }
    int chiralBlocks = (spinBlockSize == 0) ? 2 : V.Nspin() / spinBlockSize; //always 2 for staggered.
    int numblocks = (V.Volume()/geo_blocksize) * chiralBlocks;
    constexpr int coarseSpin = (nSpin == 4 || nSpin == 2 || spinBlockSize == 0) ? 2 : 1;

    if (getVerbosity() >= QUDA_VERBOSE)
      printfQuda("Block Orthogonalizing %d blocks of %d length and width %d repeating %d times, two_pass = %d\n",
                 numblocks, blocksize, nVec, n_block_ortho, two_pass);

    BlockOrtho<vFloat, bFloat, nSpin, spinBlockSize, nColor, coarseSpin, nVec>
      ortho(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
  }

  template <typename vFloat, typename bFloat>
  void BlockOrthogonalize(ColorSpinorField &V, const std::vector<ColorSpinorField *> &B, const int *fine_to_coarse,
                          const int *coarse_to_fine, const int *geo_bs, int spin_bs, int n_block_ortho, bool two_pass)
  {
    const int Nvec = B.size();
    if (V.Ncolor()/Nvec == 3) {
#ifdef NSPIN4
      if (V.Nspin() == 4) {
        constexpr int nColor = 3;
        constexpr int nSpin = 4;
        if (spin_bs != 2) errorQuda("Unexpected spin block size = %d", spin_bs);
        constexpr int spinBlockSize = 2;

        if (Nvec == 6) { // for Wilson free field
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 6>(V, B, fine_to_coarse, coarse_to_fine,
                                                                              geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 24) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 24>(V, B, fine_to_coarse, coarse_to_fine,
                                                                               geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 32) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 32>(V, B, fine_to_coarse, coarse_to_fine,
                                                                               geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
      } else
#endif // NSPIN4
#ifdef NSPIN1
      if (V.Nspin() == 1) {
        constexpr int nColor = 3;
        constexpr int nSpin = 1;
        if (spin_bs != 0) errorQuda("Unexpected spin block size = %d", spin_bs);
        constexpr int spinBlockSize = 0;

        if (Nvec == 24) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 24>(V, B, fine_to_coarse, coarse_to_fine,
                                                                               geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 64) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 64>(V, B, fine_to_coarse, coarse_to_fine,
                                                                               geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 96) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 96>(V, B, fine_to_coarse, coarse_to_fine,
                                                                               geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }

      } else
#endif // NSPIN1
      {
        errorQuda("Unexpected nSpin = %d", V.Nspin());
      }

    } else { // Nc != 3
      if (V.Nspin() != 2) errorQuda("Unexpected nSpin = %d", V.Nspin());
      constexpr int nSpin = 2;
      if (spin_bs != 1) errorQuda("Unexpected spin block size = %d", spin_bs);
      constexpr int spinBlockSize = 1;

#ifdef NSPIN4
      if (V.Ncolor()/Nvec == 6) {
        constexpr int nColor = 6;
        if (Nvec == 6) {
          BlockOrthogonalize<vFloat, bFloat, nSpin, spinBlockSize, nColor, 6>(V, B, fine_to_coarse, coarse_to_fine,
                                                                              geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
      } else
#endif // NSPIN4
      if (V.Ncolor()/Nvec == 24) {
        constexpr int nColor = 24;
        if (Nvec == 24) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,24>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
#ifdef NSPIN4
        } else if (Nvec == 32) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,32>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
#endif // NSPIN4
#ifdef NSPIN1
        } else if (Nvec == 64) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,64>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 96) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,96>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
#endif // NSPIN1
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
#ifdef NSPIN4
      } else if (V.Ncolor()/Nvec == 32) {
        constexpr int nColor = 32;
        if (Nvec == 32) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,32>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
#endif // NSPIN4
#ifdef NSPIN1
      } else if (V.Ncolor()/Nvec == 64) {
        constexpr int nColor = 64;
        if (Nvec == 64) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,64>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
        } else if (Nvec == 96) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,96>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
      } else if (V.Ncolor()/Nvec == 96) {
        constexpr int nColor = 96;
        if (Nvec == 96) {
          BlockOrthogonalize<vFloat,bFloat,nSpin,spinBlockSize,nColor,96>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, n_block_ortho, two_pass);
        } else {
          errorQuda("Unsupported nVec %d\n", Nvec);
        }
#endif // NSPIN1
      } else {
        errorQuda("Unsupported nColor %d\n", V.Ncolor()/Nvec);
      }
    } // Nc != 3
  }

#ifdef GPU_MULTIGRID
  void BlockOrthogonalize(ColorSpinorField &V, const std::vector<ColorSpinorField *> &B, const int *fine_to_coarse,
                          const int *coarse_to_fine, const int *geo_bs, int spin_bs, int n_block_ortho, bool two_pass)
  {
    if (B[0]->V() == nullptr) {
      warningQuda("Trying to BlockOrthogonalize staggered transform, skipping...");
      return;
    }
    if (V.Precision() == QUDA_DOUBLE_PRECISION && B[0]->Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      BlockOrthogonalize<double>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, spin_bs, n_block_ortho, two_pass);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (V.Precision() == QUDA_SINGLE_PRECISION && B[0]->Precision() == QUDA_SINGLE_PRECISION) {
      BlockOrthogonalize<float, float>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, spin_bs, n_block_ortho, two_pass);
    } else if (V.Precision() == QUDA_HALF_PRECISION && B[0]->Precision() == QUDA_SINGLE_PRECISION) {
#if QUDA_PRECISION & 2
      BlockOrthogonalize<short, float>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, spin_bs, n_block_ortho, two_pass);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else if (V.Precision() == QUDA_HALF_PRECISION && B[0]->Precision() == QUDA_HALF_PRECISION) {
#if QUDA_PRECISION & 2
      BlockOrthogonalize<short, short>(V, B, fine_to_coarse, coarse_to_fine, geo_bs, spin_bs, n_block_ortho, two_pass);
#else
      errorQuda("QUDA_PRECISION=%d does not enable half precision", QUDA_PRECISION);
#endif
    } else {
      errorQuda("Unsupported precision combination V=%d B=%d\n", V.Precision(), B[0]->Precision());
    }
  }
#else
  void BlockOrthogonalize(ColorSpinorField &, const std::vector<ColorSpinorField *> &, const int *,
                          const int *, const int *, int, int, bool)
  {
    errorQuda("Multigrid has not been built");
  }
#endif // GPU_MULTIGRID

} // namespace quda
